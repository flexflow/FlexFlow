/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/parallel_ops/kernels/allreduce_kernels.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

AllReduceMeta::AllReduceMeta(FFHandler handle, AllReduce const *reduct)
    : OpMeta(handle) {}

namespace Kernels {
namespace AllReduce {

void inference_kernel_wrapper(AllReduceMeta const *m,
                              BatchConfig const *bc,
                              GenericTensorAccessorR const &input,
                              GenericTensorAccessorW const &output) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  assert(input.data_type == output.data_type);
  assert(input.domain == output.domain);
  size_t hidden_dim_size = input.domain.hi()[0] - input.domain.lo()[0] + 1;
  size_t num_elements = bc->num_tokens * hidden_dim_size;
#ifdef FF_USE_NCCL
  ncclDataType_t nccl_data_type = ff_to_nccl_datatype(input.data_type);
  checkNCCL(ncclAllReduce(input.ptr,
                          output.ptr,
                          num_elements,
                          nccl_data_type,
                          ncclSum,
                          m->handle.ncclComm,
                          stream));
#else
  assert(false && "Must enable FF_USE_NCCL to use AllReduce operators");
#endif
}

void forward_kernel_wrapper(AllReduceMeta const *m,
                            GenericTensorAccessorR const &input,
                            GenericTensorAccessorW const &output) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  assert(input.data_type == output.data_type);
  assert(input.domain == output.domain);
#ifdef FF_USE_NCCL
  ncclDataType_t nccl_data_type = ff_to_nccl_datatype(input.data_type);
  checkNCCL(ncclAllReduce(input.ptr,
                          output.ptr,
                          input.domain.get_volume(),
                          nccl_data_type,
                          ncclSum,
                          m->handle.ncclComm,
                          stream));
#else
  assert(false && "Must enable FF_USE_NCCL to use AllReduce operators");
#endif
}

void backward_kernel_wrapper(AllReduceMeta const *m,
                             GenericTensorAccessorW const &input_grad,
                             GenericTensorAccessorR const &output_grad) {
  assert(false && "To be implemented");
}

} // namespace AllReduce
} // namespace Kernels
} // namespace FlexFlow
