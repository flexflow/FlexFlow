#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "initializer.h"
#include "accessor.h"
#include "model.h"
#include "cuda_helper.h"
#include <hiprand.h>
#include <random>
#include <ctime>

void UniformInitializer::init_task(const Task* task,
                                   const std::vector<PhysicalRegion>& regions,
                                   Context ctx, Runtime* runtime)
{

  assert(regions.size() == task->regions.size());
  UniformInitializer* initializer = (UniformInitializer*) task->args;
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  hiprandSetStream(gen, stream);
  //fprintf(stderr, "seed = %d\n", initializer->seed);



  for (size_t i = 0; i < regions.size(); i++) {
    Domain domain = runtime->get_index_space_domain(
        ctx, task->regions[i].region.get_index_space());
    float* w;
    switch (domain.get_dim()) {
      case 0:
      {
        // Do not support 0-dim parameters
        assert(false);
        break;
      }
#define DIMFUNC(DIM) \
      case DIM: \
      { \
        TensorAccessorW<float, DIM> accW( \
            regions[i], task->regions[i], FID_DATA, ctx, runtime, false/*readOutput*/); \
        w = accW.ptr; \
        break; \
      }
      LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
      default:
      {
         assert(false);
         break;
      }
    }
    hiprandSetPseudoRandomGeneratorSeed(gen, initializer->seed);
    checkCUDA(hiprandGenerateUniform(gen, w, domain.get_volume()));
    scale_kernel<<<GET_BLOCKS(domain.get_volume()), CUDA_NUM_THREADS>>>(
        w, domain.get_volume(), initializer->min_val, initializer->max_val);
  }
  checkCUDA(hipDeviceSynchronize());
  hiprandDestroyGenerator(gen);
}

void GlorotUniform::init_task(const Task* task,
                              const std::vector<PhysicalRegion>& regions,
                              Context ctx, Runtime* runtime)
{
  assert(regions.size() == 1);
  assert(task->regions.size() == 1);
  Domain domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  float* w;
  float scale = 0;
  switch (domain.get_dim()) {
    case 2:
    {
      TensorAccessorW<float, 2> accW(regions[0], task->regions[0],
          FID_DATA, ctx, runtime, false/*readOutput*/);
      w = accW.ptr;
      int outputDim = accW.rect.hi[1] - accW.rect.lo[1] + 1;
      int inputDim = accW.rect.hi[0] - accW.rect.lo[0] + 1;
      scale = sqrt(6.0 / (inputDim + outputDim));
      break;
    }
    case 3:
    {
      TensorAccessorW<float, 3> accW(regions[0], task->regions[0],
          FID_DATA, ctx, runtime, false/*readOutput*/);
      w = accW.ptr;
      // reference: tensorflow code for computing fan_in/fan_out
      // https://github.com/tensorflow/tensorflow/blob/r2.0/tensorflow/python/ops/init_ops.py#L1415-L1439
      int num_dim = domain.get_dim();
      coord_t receptive_field_size = 1;
      for (int i = 0; i < num_dim - 2; i++)
        receptive_field_size *= (accW.rect.hi[i] - accW.rect.lo[i] + 1);
      coord_t c_in = accW.rect.hi[num_dim-2] - accW.rect.lo[num_dim-2] + 1;
      coord_t c_out = accW.rect.hi[num_dim-1] - accW.rect.lo[num_dim-1] + 1;
      coord_t fan_in = c_in * receptive_field_size;
      coord_t fan_out = c_out * receptive_field_size;
      scale = sqrt(6.0 / (fan_in + fan_out));
      break;
    }
    case 4:
    {
      TensorAccessorW<float, 4> accW(regions[0], task->regions[0],
          FID_DATA, ctx, runtime, false/*readOutput*/);
      w = accW.ptr;
      // reference: tensorflow code for computing fan_in/fan_out
      // https://github.com/tensorflow/tensorflow/blob/r2.0/tensorflow/python/ops/init_ops.py#L1415-L1439
      int num_dim = domain.get_dim();
      coord_t receptive_field_size = 1;
      for (int i = 0; i < num_dim - 2; i++)
        receptive_field_size *= (accW.rect.hi[i] - accW.rect.lo[i] + 1);
      coord_t c_in = accW.rect.hi[num_dim-2] - accW.rect.lo[num_dim-2] + 1;
      coord_t c_out = accW.rect.hi[num_dim-1] - accW.rect.lo[num_dim-1] + 1;
      coord_t fan_in = c_in * receptive_field_size;
      coord_t fan_out = c_out * receptive_field_size;
      scale = sqrt(6.0 / (fan_in + fan_out));
      break;
    }
    default:
      assert(false);
  }
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCURAND(hiprandSetStream(gen, stream));
#endif
  GlorotUniform* initializer = (GlorotUniform*) task->args;
  hiprandSetPseudoRandomGeneratorSeed(gen, initializer->seed);
  fprintf(stderr, "seed = %d scale = %.4lf\n", initializer->seed, scale);
  checkCUDA(hiprandGenerateUniform(gen, w, domain.get_volume()));
  scale_kernel<<<GET_BLOCKS(domain.get_volume()), CUDA_NUM_THREADS>>>(
      w, domain.get_volume(), -scale, scale);
  checkCUDA(hipDeviceSynchronize());
  hiprandDestroyGenerator(gen);
}


void NormInitializer::init_task(const Task* task,
                                const std::vector<PhysicalRegion>& regions,
                                Context ctx, Runtime* runtime)
{
  assert(regions.size() == 1);
  assert(task->regions.size() == 1);
  Domain domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  float* w;
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
      case DIM: \
      { \
        TensorAccessorW<float, DIM> accW( \
            regions[0], task->regions[0], FID_DATA, ctx, runtime, false/*readOutput*/); \
        w = accW.ptr; \
        break; \
      }
      LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCURAND(hiprandSetStream(gen, stream));
#endif
  NormInitializer* initializer = (NormInitializer*) task->args;
  //fprintf(stderr, "seed = %d\n", initializer->seed);
  hiprandSetPseudoRandomGeneratorSeed(gen, initializer->seed);
  //fprintf(stderr, "domain.volume() = %zu mean(%.4lf) var(%.4lf)\n",
  //    domain.get_volume(), initializer->mean, initializer->stddev);
  // FIXME: it seems hiprand has an internal bug with volume < 4
  // double check this later
  if (domain.get_volume() < 4) {
    std::default_random_engine generator;
    std::normal_distribution<float> distribution(
        initializer->mean, initializer->stddev);
    float* w_dram = (float*) malloc(domain.get_volume() * sizeof(float));
    for (size_t i = 0; i < domain.get_volume(); i++)
      w_dram[i] = distribution(generator);
    checkCUDA(hipMemcpy(w, w_dram, sizeof(float) * domain.get_volume(),
                         hipMemcpyHostToDevice));
    checkCUDA(hipDeviceSynchronize());
    free(w_dram);
  } else {
    checkCURAND(hiprandGenerateNormal(gen, w, domain.get_volume(),
        initializer->mean, initializer->stddev));
    checkCUDA(hipDeviceSynchronize());
  }
  hiprandDestroyGenerator(gen);
}

void ZeroInitializer::init_task(const Task* task,
                                const std::vector<PhysicalRegion>& regions,
                                Context ctx, Runtime* runtime)
{
  assert(regions.size() == task->regions.size());
  for (size_t i = 0; i < regions.size(); i++) {
    Domain domain = runtime->get_index_space_domain(
        ctx, task->regions[i].region.get_index_space());
    float* w;
    switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
      case DIM: \
      { \
        TensorAccessorW<float, DIM> accW( \
            regions[i], task->regions[i], FID_DATA, ctx, runtime, false/*readOutput*/); \
        w = accW.ptr; \
        break; \
      }
      LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
      default:
      {
         assert(false);
         break;
      }
    }
    assign_kernel<<<GET_BLOCKS(domain.get_volume()), CUDA_NUM_THREADS>>>(
        w, domain.get_volume(), 0.0f);
  }
  checkCUDA(hipDeviceSynchronize());
}

void ConstantInitializer::init_task(const Task* task,
                                    const std::vector<PhysicalRegion>& regions,
                                    Context ctx, Runtime* runtime)
{
  ConstantInitializer* initializer = (ConstantInitializer*) task->args;
  assert(regions.size() == task->regions.size());
  for (size_t i = 0; i < regions.size(); i++) {
    Domain domain = runtime->get_index_space_domain(
        ctx, task->regions[i].region.get_index_space());
    float* w;
    switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
      case DIM: \
      { \
        TensorAccessorW<float, DIM> accW( \
            regions[i], task->regions[i], FID_DATA, ctx, runtime, false/*readOutput*/); \
        w = accW.ptr; \
        break; \
      }
      LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
      default:
      {
         assert(false);
         break;
      }
    }
    assign_kernel<<<GET_BLOCKS(domain.get_volume()), CUDA_NUM_THREADS>>>(
        w, domain.get_volume(), initializer->value);
  }
  checkCUDA(hipDeviceSynchronize());
}
