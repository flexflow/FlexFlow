#include "hip/hip_runtime.h"
#include "cuda_helper.h"
#include "model.h"

__global__
void scale_kernel(float* ptr, coord_t size, float a, float b)
{
  CUDA_KERNEL_LOOP(i, size)
  {
    ptr[i] = (b - a) * ptr[i] + a;
  }
}

__global__
void ones_kernel(float* ptr, coord_t size)
{
  CUDA_KERNEL_LOOP(i, size)
  {
    ptr[i] = 1.0f;
  }
}

template<typename DT>
__global__
void assign_kernel(DT* ptr, coord_t size, DT value)
{
  CUDA_KERNEL_LOOP(i, size)
  {
    ptr[i] = value;
  }
}

template<typename DT>
__global__
void copy_kernel(DT* dst, const DT* src, coord_t size)
{
  CUDA_KERNEL_LOOP(i, size)
  {
    dst[i] = src[i];
  }
}

__global__
void reluBackward(float *grad_ptr, const float *output, int n)
{
  CUDA_KERNEL_LOOP(i, n)
  {
    grad_ptr[i] = (output[i] > 0.0f) ? grad_ptr[i] : 0;
  }
}

__global__
void apply_add(float *data_ptr, const float *replica_ptr, size_t size)
{
  CUDA_KERNEL_LOOP(i, size)
  {
    data_ptr[i] += replica_ptr[i];
  }
}

__global__
void apply_add_with_scale(float *data_ptr, const float *grad_ptr,
                          size_t size, float scale)
{
  CUDA_KERNEL_LOOP(i, size)
  {
    data_ptr[i] += grad_ptr[i] * scale;
  }
}

__global__
void add_with_stride(float* output,
                     const float* input,
                     int num_blocks,
                     int output_blk_size,
                     int input_blk_size)
{
  int min_blk_size = min(output_blk_size, input_blk_size);
  CUDA_KERNEL_LOOP(i, num_blocks * min_blk_size)
  {
    int blk_idx = i / min_blk_size;
    int blk_offset = i % min_blk_size;
    int input_offset = blk_idx * input_blk_size + blk_offset;
    int output_offset = blk_idx * output_blk_size + blk_offset;
    output[output_offset] += input[input_offset];
  }
}

__global__
void copy_with_stride(float* output,
                      const float* input,
                      int num_blocks,
                      int output_blk_size,
                      int input_blk_size)
{
  int min_blk_size = min(output_blk_size, input_blk_size);
  CUDA_KERNEL_LOOP(i, num_blocks * min_blk_size)
  {
    int blk_idx = i / min_blk_size;
    int blk_offset = i % min_blk_size;
    int input_offset = blk_idx * input_blk_size + blk_offset;
    int output_offset = blk_idx * output_blk_size + blk_offset;
    output[output_offset] = input[input_offset];
  }
}



__host__
void updateGAS(float* para_ptr, const float* grad_ptr, size_t replica_size,
               int num_replica, float learning_rate)
{
  // Step 1: gater gradients to the first replica
  for (int i = 1; i < num_replica; i++) {
    const float *replica = grad_ptr + i * replica_size;
    apply_add<<<GET_BLOCKS(replica_size), CUDA_NUM_THREADS>>>(
        (float*)grad_ptr, replica, replica_size);
  }
  // Step 2: scale the first replica
  float scale_factor = 1.0f / num_replica * (-learning_rate);
  apply_add_with_scale<<<GET_BLOCKS(replica_size), CUDA_NUM_THREADS>>>(
      para_ptr, grad_ptr, replica_size, scale_factor);
}

#ifdef DEADCODE
template<unsigned DIM, typename T>
__host__
void print_tensor(const T* ptr, Rect<DIM> rect, const char* prefix)
{
  // device synchronize to make sure the data are ready
  // checkCUDA(hipDeviceSynchronize());
  T* host_ptr;
  checkCUDA(hipHostAlloc(&host_ptr, sizeof(T) * rect.volume(),
                          hipHostMallocPortable | hipHostMallocMapped));
  checkCUDA(hipMemcpy(host_ptr, ptr, sizeof(T) * rect.volume(),
                       hipMemcpyDeviceToHost));
  // checkCUDA(hipDeviceSynchronize());
  int idx = 0;
  printf("%s", prefix);
  for (PointInRectIterator<DIM> it(rect); it(); it++, idx++) {
    printf(" %.4lf", (float)host_ptr[idx]);
    if (idx >= 16) break;
  }
  printf("\n");
  checkCUDA(hipHostFree(host_ptr));
}
#endif

template<typename T>
__host__
void print_tensor(const T* ptr, size_t num_elements, const char* prefix)
{
  // device synchronize to make sure the data are ready
  // checkCUDA(hipDeviceSynchronize());
  T* host_ptr;
  checkCUDA(hipHostAlloc(&host_ptr, sizeof(T) * num_elements,
                          hipHostMallocPortable | hipHostMallocMapped));
  checkCUDA(hipMemcpy(host_ptr, ptr, sizeof(T) * num_elements,
                       hipMemcpyDeviceToHost));
  // checkCUDA(hipDeviceSynchronize());
  int idx = 0;
  printf("%s", prefix);
  for (idx=0; idx < num_elements; idx++) {
    printf(" %.4lf", (float)host_ptr[idx]);
    if (idx >= 16) break;
  }
  printf("\n");
  checkCUDA(hipHostFree(host_ptr));
}

hipdnnStatus_t cudnnSetTensorDescriptorFromDomain(hipdnnTensorDescriptor_t tensor, Domain domain)
{
  int dims[MAX_TENSOR_DIM];
  switch (domain.get_dim()) {
    case 1:
    {
      Rect<1> rect = domain;
      dims[0] = rect.hi[0] - rect.lo[0] + 1;
      return hipdnnSetTensor4dDescriptor(tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dims[0], 1, 1, 1);
    }
    case 2:
    {
      Rect<2> rect = domain;
      dims[0] = rect.hi[0] - rect.lo[0] + 1;
      dims[1] = rect.hi[1] - rect.lo[1] + 1;
      return hipdnnSetTensor4dDescriptor(tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dims[1], dims[0], 1, 1);
    }
    case 3:
    {
      Rect<3> rect = domain;
      dims[0] = rect.hi[0] - rect.lo[0] + 1;
      dims[1] = rect.hi[1] - rect.lo[1] + 1;
      dims[2] = rect.hi[2] - rect.lo[2] + 1;
      return hipdnnSetTensor4dDescriptor(tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dims[2], dims[1], dims[0], 1);
    }
    case 4:
    {
      Rect<4> rect = domain;
      dims[0] = rect.hi[0] - rect.lo[0] + 1;
      dims[1] = rect.hi[1] - rect.lo[1] + 1;
      dims[2] = rect.hi[2] - rect.lo[2] + 1;
      dims[3] = rect.hi[3] - rect.lo[3] + 1;
      return hipdnnSetTensor4dDescriptor(tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dims[3], dims[2], dims[1], dims[0]);
    }
    default:
      assert(false && "Unsupported dim number");
  }
  return HIPDNN_STATUS_BAD_PARAM;
}

template __global__ void assign_kernel<float>(float* ptr, coord_t size, float value);
template __global__ void assign_kernel<int32_t>(int32_t* ptr, coord_t size, int32_t value);
template __global__ void assign_kernel<int64_t>(int64_t* ptr, coord_t size, int64_t value);

template __global__ void copy_kernel<float>(float* dst, const float* src, coord_t size);
template __global__ void copy_kernel<int>(int* dst, const int* src, coord_t size);

template __host__ void print_tensor<float>(const float* ptr, size_t rect, const char* prefix);
template __host__ void print_tensor<long>(const long* ptr, size_t rect, const char* prefix);
