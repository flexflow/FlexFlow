/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "simulator.h"
#include "model.h"
#include "realm/runtime_impl.h"
#include "realm/cuda/cuda_module.h"
#include "cuda_helper.h"

typedef long long int coord_t;

typedef Realm::Point<1, coord_t> Point1;
typedef Realm::Rect<1, coord_t> Rect1;

Simulator::Simulator(const FFModel* model,
                     FFHandler _handler,
                     Memory _memory)
: memory(_memory), handler(_handler),
  offset(0), warmup_times(5), repeat_times(10)
{
  // Allocate simulator memory
  Rect1 bounds(Point1(0), Point1(0));
  std::vector<size_t> field_sizes;
  field_sizes.push_back(model->config.simulator_work_space_size);
  Realm::RegionInstance::create_instance(simulatorInst,
      memory, bounds, field_sizes, 0, Realm::ProfilingRequestSet()).wait();
  base_ptr = (char*)simulatorInst.pointer_untyped(0, sizeof(char));
  capacity = model->config.simulator_work_space_size;

  float inter_gpu_bandwidth = 20 * 1024 * 1024.0f; /* B/ms*/
  float inter_node_bandwidth = 12 * 1024 * 1024.0f / model->config.numNodes; /* B/ms*/
  float gpu_dram_bandwidth = 16 * 1024 * 1024.0f; /* B/ms*/
  size_t max_num_tasks = 1024 * 1024;

  hipEventCreate(&start_event);
  hipEventCreate(&end_event);
  conv2d_meta = new Conv2DMeta(handler);
  linear_meta = new LinearMeta(handler, 4096);
  pool2d_meta = new Pool2DMeta(handler);
  ele_unary_meta = new ElementUnaryMeta(handler);
  ele_binary_meta = new ElementBinaryMeta(handler);
  softmax_meta = new SoftmaxMeta(handler);
  batch_matmul_meta = new BatchMatmulMeta(handler);
  batch_norm_meta = new BatchNormMeta(handler);
  concat_meta = new ConcatMeta(handler);
  dropout_meta = new DropoutMeta(handler);
  transpose_meta = new TransposeMeta(handler);
  int num_nodes = model->config.numNodes;
  int gpus_per_node = model->config.workersPerNode;
  total_num_devices = num_nodes * gpus_per_node;
  // Create GPU compute device
  for (int i = 0; i < num_nodes; i++)
    for (int j = 0; j < gpus_per_node; j++) {
      id_to_compute_device[i*gpus_per_node+j] = new Device(Device::DEVICE_GPU,
          i, i*gpus_per_node+j);
    }
  // Create inter GPU comm devices:
  for (int i = 0; i < total_num_devices; i++)
    for (int j = 0; j < total_num_devices; j++) {
      Device* src = id_to_compute_device[i];
      Device* dst = id_to_compute_device[j];
      if (src->node_id == dst->node_id && src != dst) {
        int hash = i * total_num_devices + j;
        ids_to_inter_gpu_comm_device[hash] = new Device(Device::DEVICE_COMM,
            inter_gpu_bandwidth);
      }
    }
  // Create gpu<->dram comm devices
  for (int i = 0; i < total_num_devices; i++) {
    id_to_gputodram_comm_device[i] = new Device(Device::DEVICE_COMM,
        gpu_dram_bandwidth);
    id_to_dramtogpu_comm_device[i] = new Device(Device::DEVICE_COMM,
        gpu_dram_bandwidth);
  }
  // Create inter node comm devices
  for (int i = 0; i < num_nodes; i++)
    for (int j = 0; j < num_nodes; j++)
      if (i != j) {
        int hash = i * total_num_devices + j;
        ids_to_inter_node_comm_device[hash] = new Device(Device::DEVICE_COMM,
            inter_node_bandwidth);
      }
  // Initialize task manager
  task_manager = new TaskManager(max_num_tasks);
}

Simulator::~Simulator(void)
{
  simulatorInst.destroy();
}

__host__
void Simulator::strategy_search_task(const Task *task,
                                     const std::vector<PhysicalRegion> &regions,
                                     Context ctx, Runtime *runtime)
{
  const FFModel* model = *((FFModel**) task->args);
  Memory gpu_mem = Machine::MemoryQuery(Machine::get_machine())
         .only_kind(Memory::GPU_FB_MEM).best_affinity_to(task->target_proc).first();
  // Realm::MemoryImpl* memImpl =
  //     Realm::get_runtime()->get_memory_impl(gpu_mem);
  // Realm::Cuda::GPUFBMemory* memFBImpl = (Realm::Cuda::GPUFBMemory*) memImpl;
  // off_t offset = memFBImpl->alloc_bytes_local(model->config.simulator_work_space_size);
  // void* base_ptr = memFBImpl->get_direct_ptr(offset, 0);
  // Assume this task is running on GPU0
  Simulator* simulator = new Simulator(model, model->handlers[0], gpu_mem);
  // Set cublas/cudnn streams to allow Realm catch the events
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(simulator->handler.blas, stream));
  checkCUDNN(hipdnnSetStream(simulator->handler.dnn, stream));
#endif
  std::map<Op*, ParallelConfig> strategies;
  if (model->config.import_strategy_file.length() > 0) {
    // Load the strategy from config.strategies
    for (size_t l = 0; l < model->layers.size(); l++) {
      MappingTagID key = FFConfig::get_hash_id(std::string(model->layers[l]->name));
      std::map<MappingTagID, ParallelConfig>::const_iterator iter;
      iter = model->config.strategies.find(key);
      if (iter == model->config.strategies.end()) {
        fprintf(stderr, "ERROR: Cannot find strategy for operator %s in "
                "strategy file %s\n", model->layers[l]->name,
                model->config.import_strategy_file.c_str());
      }
      strategies[model->layers[l]] = iter->second;
    }
  } else {
    // Start from data parallel
    for (size_t l = 0; l < model->layers.size(); l++) {
      strategies[model->layers[l]] = model->layers[l]->get_data_parallel_config(*model);
    }
  }

  model->optimize(simulator, strategies, model->config.search_budget, model->config.search_alpha);
  if (model->config.export_strategy_file.length() > 0) {
    fprintf(stderr, "Exporting the best discovered strategy to %s\n",
        model->config.export_strategy_file.c_str());
    std::map<Op*, ParallelConfig>::const_iterator iter;
    std::map<std::string, ParallelConfig> strategy_output;
    for (iter = strategies.begin(); iter != strategies.end(); iter++) {
      strategy_output[iter->first->name] = iter->second;
    }
    save_strategies_to_file(model->config.export_strategy_file, strategy_output);
  }
  // Start from data
  // memFBImpl->free_bytes_local(offset, model->config.simulator_work_space_size);
  delete(simulator);
}

