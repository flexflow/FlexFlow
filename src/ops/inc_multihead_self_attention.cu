#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/inc_multihead_self_attention.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

// declare Legion names
using Legion::coord_t;
using Legion::Memory;

/*static*/
void IncMultiHeadSelfAttention::inference_kernel1(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    float const *input_ptr,
    float const *weight_ptr,
    float *output_ptr,
    hipStream_t stream) {

  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha = 1.0f, beta = 0.0f;
  int out_dim = (m->qProjSize + m->kProjSize + m->vProjSize) * m->num_heads;
  int in_dim = m->qSize;
  assert(in_dim == m->vSize && in_dim == m->kSize);
  hipDataType data_type = ff_to_cuda_datatype(DT_FLOAT);
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = HIP_R_32F;
#endif
  checkCUDA(hipblasGemmEx(m->handle.blas,
                         HIPBLAS_OP_T,
                         HIPBLAS_OP_N,
                         out_dim,
                         bc->num_tokens,
                         in_dim,
                         &alpha,
                         weight_ptr,
                         data_type,
                         in_dim,
                         input_ptr,
                         data_type,
                         in_dim,
                         &beta,
                         output_ptr,
                         data_type,
                         out_dim,
                         compute_type,
                         CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

__global__ void store_kv_cache(float const *devQKVProjArray,
                               float *cache_ptr,
                               BatchConfig::token_ids const *id_map,
                               int qProjSize,
                               int kProjSize,
                               int vProjSize,
                               int num_tokens,
                               int num_heads,
                               int max_seq_len,
                               bool k_cache) {
  CUDA_KERNEL_LOOP(i,
                   num_tokens * (k_cache ? kProjSize : vProjSize) * num_heads) {
    int proj_size = k_cache ? kProjSize : vProjSize;
    int head_idx = i / (num_tokens * proj_size);
    int token_idx = (i - head_idx * (num_tokens * proj_size)) / proj_size;
    int data_idx = i % proj_size;

    int qkv_block_size = (qProjSize + kProjSize + vProjSize) * num_tokens;
    int current_head_block_size =
        num_tokens * (k_cache ? qProjSize : qProjSize + kProjSize);
    float val =
        devQKVProjArray[head_idx * qkv_block_size + current_head_block_size +
                        token_idx * proj_size + data_idx];

    int const req_id = id_map[token_idx].request_id;
    int const tok_id = id_map[token_idx].token_id;

    cache_ptr[req_id * (num_heads * max_seq_len * proj_size) +
              head_idx * (max_seq_len * proj_size) + tok_id * proj_size +
              data_idx] = val;
  }
}

/*static*/
void IncMultiHeadSelfAttention::inference_kernel2(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    hipStream_t stream) {
  int num_tokens = bc->num_active_tokens();
  if (num_tokens > 0) {
    int parallelism = m->kProjSize * num_tokens * m->num_heads;
    store_kv_cache<<<GET_BLOCKS(parallelism),
                     min(CUDA_NUM_THREADS, parallelism),
                     0,
                     stream>>>(m->devQKVProjArray,
                               m->keyCache,
                               m->dev_token2ids,
                               m->qProjSize,
                               m->kProjSize,
                               m->vProjSize,
                               num_tokens,
                               m->num_heads,
                               bc->MAX_SEQUENCE_LENGTH,
                               /* k_cache = */ true);
    parallelism = m->vProjSize * num_tokens * m->num_heads;
    store_kv_cache<<<GET_BLOCKS(parallelism),
                     min(CUDA_NUM_THREADS, parallelism),
                     0,
                     stream>>>(m->devQKVProjArray,
                               m->valueCache,
                               m->dev_token2ids,
                               m->qProjSize,
                               m->kProjSize,
                               m->vProjSize,
                               num_tokens,
                               m->num_heads,
                               bc->MAX_SEQUENCE_LENGTH,
                               /* k_cache = */ false);
  }
}

__global__ void fill_above_diagonal_square(float *matrix,
                                           int x_dim,
                                           int num_heads,
                                           int entries_above_diagonal,
                                           float value) {
  CUDA_KERNEL_LOOP(i, entries_above_diagonal * num_heads) {
    int head_idx = i / entries_above_diagonal;
    int y = (-1 + sqrt(8 * (float)i + 1)) / 2 + 1;
    int x = i - y * (y + 1) / 2;
    matrix[head_idx * x_dim * x_dim + x_dim * y + x] = value;
  }
}

__global__ void fill_last_entry_vector(float *matrix,
                                       int y_dim,
                                       int num_heads,
                                       float value) {
  // Fill last entry of each of the num_heads contiguous arrays of size y_dim
  CUDA_KERNEL_LOOP(i, num_heads) {
    matrix[i * y_dim + (y_dim - 1)] = value;
  }
}

/*static*/
void IncMultiHeadSelfAttention::inference_kernel3(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    hipStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  hipDataType cublas_data_type = ff_to_cuda_datatype(DT_FLOAT);
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = HIP_R_32F;
#endif
  int num_requests = bc->num_active_requests();
  int num_tokens = bc->num_active_tokens();
  int tokens_previous_requests = 0;
  int tokens_prev_requests_squares = 0;
  int qkv_block_size =
      (m->qProjSize + m->kProjSize + m->vProjSize) * num_tokens;
  for (int i = 0; i < num_requests; i++) {
    int num_new_tokens = bc->num_processing_tokens[i];
    int total_tokens = bc->token_last_available_idx[i] + 1;
    float alpha = 1.0f / (float)sqrt(m->kProjSize), beta = 0.0f;
    checkCUDA(hipblasGemmStridedBatchedEx(
        m->handle.blas,
        HIPBLAS_OP_T,
        HIPBLAS_OP_N,
        num_new_tokens,
        total_tokens,
        m->kProjSize,
        &alpha,
        (void const *)(m->devQKVProjArray +
                       tokens_previous_requests * m->qProjSize),
        cublas_data_type,
        m->kProjSize,
        qkv_block_size,
        (void const *)(m->keyCache +
                       i * (m->num_heads * bc->MAX_SEQUENCE_LENGTH *
                            m->kProjSize)),
        cublas_data_type,
        m->kProjSize,
        m->kProjSize * total_tokens,
        &beta,
        (void *)(m->qt_prods + m->num_heads * tokens_prev_requests_squares),
        cublas_data_type,
        num_new_tokens,
        num_new_tokens * num_new_tokens,
        m->num_heads,
        compute_type,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    // fill all elements above diagonal with -inf
    // assume that number of rows is either 1 or equal to number of columns
    assert(num_new_tokens == 1 || num_new_tokens == total_tokens);
    if (num_tokens == 1) {
      int parallelism = m->num_heads;
      fill_last_entry_vector<<<GET_BLOCKS(parallelism),
                               min(CUDA_NUM_THREADS, parallelism),
                               0,
                               stream>>>(
          m->qt_prods + m->num_heads * tokens_prev_requests_squares,
          total_tokens,
          m->num_heads,
          FLT_MIN);

    } else {
      int entries_above_diagonal = total_tokens * (total_tokens - 1) / 2;
      int parallelism = m->num_heads * entries_above_diagonal;
      fill_above_diagonal_square<<<GET_BLOCKS(parallelism),
                                   min(CUDA_NUM_THREADS, parallelism),
                                   0,
                                   stream>>>(
          m->qt_prods + m->num_heads * tokens_prev_requests_squares,
          total_tokens,
          m->num_heads,
          entries_above_diagonal,
          FLT_MIN);
    }

    // Softmax
    hipdnnTensorDescriptor_t qt_tensor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&qt_tensor));
    checkCUDNN(
        hipdnnSetTensor4dDescriptor(qt_tensor,
                                   HIPDNN_TENSOR_NCHW,
                                   HIPDNN_DATA_FLOAT,
                                   m->num_heads * total_tokens * total_tokens,
                                   1,
                                   1,
                                   1));
    alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnSoftmaxForward(
        m->handle.dnn,
        HIPDNN_SOFTMAX_ACCURATE,
        HIPDNN_SOFTMAX_MODE_CHANNEL,
        &alpha,
        qt_tensor,
        m->qt_prods + m->num_heads * tokens_prev_requests_squares,
        &beta,
        qt_tensor,
        m->qt_prods_softmax + m->num_heads * tokens_prev_requests_squares));

    // TODO: Matmul by V

    tokens_previous_requests += num_new_tokens;
    tokens_prev_requests_squares += num_new_tokens * num_new_tokens;
  }

  assert(tokens_previous_requests == num_tokens);
}

/*static*/
void IncMultiHeadSelfAttention::inference_kernel_wrapper(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    float const *input_ptr,
    float const *weight_ptr,
    float *output_ptr) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  // phase 1: Implement kernel to compute KQV for input tokens
  IncMultiHeadSelfAttention::inference_kernel1(
      m, bc, input_ptr, weight_ptr, m->devQKVProjArray, stream);

  // phase 2: Update key/val cache
  hipMemcpyAsync(m->dev_token2ids,
                  bc->token2ids,
                  bc->MAX_NUM_TOKENS * sizeof(BatchConfig::token_ids),
                  hipMemcpyHostToDevice,
                  stream);
  IncMultiHeadSelfAttention::inference_kernel2(m, bc, stream);

  // phase 3: Compute attention score
  // 3 kernels for pahse 3: matmul1 - softmax - matmal2
  IncMultiHeadSelfAttention::inference_kernel3(m, bc, stream);

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("IncMultiHeadSelfAttention forward time = %.2fms\n", elapsed);
    // print_tensor<3, float>(acc_query.ptr, acc_query.rect,
    // "[Attention:forward:query]"); print_tensor<3, float>(acc_output.ptr,
    // acc_output.rect, "[Attention:forward:output]");
  }
}

IncMultiHeadSelfAttentionMeta::IncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    IncMultiHeadSelfAttention const *attn,
    BatchConfig const *bc,
    Memory gpu_mem,
    int num_samples,
    int _num_heads)
    : OpMeta(handler, attn) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  // checkCUDNN(hipdnnSetStream(handler.dnn, stream));

  qSize = attn->qSize;
  kSize = attn->kSize;
  vSize = attn->vSize;
  // assume dimensions match for now
  assert(qSize == kSize);
  assert(kSize == vSize);
  qProjSize = attn->qProjSize;
  kProjSize = attn->kProjSize;
  assert(qProjSize == kProjSize); // required for attention QK^T matmul
  vProjSize = attn->vProjSize;
  oProjSize = attn->oProjSize;
  num_heads = _num_heads;
  weightSize = (qSize * qProjSize + kSize * kProjSize + vSize * vProjSize +
                oProjSize * (vProjSize > 0 ? vProjSize : vSize)) *
               num_heads * sizeof(float);

  // Currently do not support adding bias to key/value projection
  assert(!attn->add_bias_kv);

  // allocate memory for the seqArray and reserve space
  {
    // size_t totalSize = reserveSpaceSize + sizeof(int) * num_samples * 2 +
    // bc->MAX_NUM_REQUESTS *bc-> MAX_SEQUENCE_LENGTH * sizeof(int); size_t
    // max_num_tokens = bc->MAX_NUM_REQUESTS * bc->MAX_SEQUENCE_LENGTH;
    size_t qkv_proj_dim = qProjSize + kProjSize + vProjSize;
    size_t qkv_max_proj_size = bc->MAX_NUM_TOKENS * qkv_proj_dim * num_heads;
    size_t key_cache_size =
        num_heads * kProjSize * bc->MAX_NUM_REQUESTS * bc->MAX_SEQUENCE_LENGTH;
    size_t value_cache_size =
        num_heads * vProjSize * bc->MAX_NUM_REQUESTS * bc->MAX_SEQUENCE_LENGTH;
    size_t token2ids_size = bc->MAX_NUM_TOKENS;
    size_t qt_prod_size = bc->MAX_NUM_TOKENS * bc->MAX_NUM_TOKENS * num_heads;
    size_t totalSize =
        (qkv_max_proj_size + key_cache_size + value_cache_size +
         2 * qt_prod_size) *
            sizeof(float) +
        token2ids_size * sizeof(BatchConfig::token_ids); // more components will
                                                         // be added here later

    Realm::Rect<1, coord_t> bounds(Realm::Point<1, coord_t>(0),
                                   Realm::Point<1, coord_t>(totalSize - 1));
    std::vector<size_t> field_sizes;
    field_sizes.push_back(sizeof(char));
    Realm::RegionInstance::create_instance(reserveInst,
                                           gpu_mem,
                                           bounds,
                                           field_sizes,
                                           0,
                                           Realm::ProfilingRequestSet())
        .wait();
    devQKVProjArray = (float *)reserveInst.pointer_untyped(0, sizeof(char));
    keyCache = (float *)devQKVProjArray + qkv_max_proj_size;
    valueCache = (float *)keyCache + key_cache_size;
    dev_token2ids = (BatchConfig::token_ids *)(valueCache + value_cache_size);
    qt_prods = (float *)(dev_token2ids + token2ids_size);
    qt_prods_softmax = (float *)(qt_prods + qt_prod_size);
    // checkCUDA(hipMemcpy(devQoSeqArray,
    //                      qoSeqArray,
    //                      sizeof(int) * num_samples,
    //                      hipMemcpyHostToDevice));
    // devKvSeqArray = (int *)devQoSeqArray + num_samples;
    // checkCUDA(hipMemcpy(devKvSeqArray,
    //                      kvSeqArray,
    //                      sizeof(int) * num_samples,
    //                      hipMemcpyHostToDevice));
    // kvCache = (int *)devKvSeqArray + num_samples;
    // reserveSpace = (int *)kvCache + bc->MAX_NUM_REQUESTS * bc->
    // MAX_SEQUENCE_LENGTH;
  }

  // input_token_ids = new request_token_id[bc->MAX_NUM_TOKENS];
}

IncMultiHeadSelfAttentionMeta::~IncMultiHeadSelfAttentionMeta(void) {
  reserveInst.destroy();
}

//__global__ void store_kv_cache(
//    float const *input_ptr, float const *cache_ptr, request_token_id const
//    *id_map, int max_seq_len, int hid_dim) {
//  int const token_idx = blockIdx.x;
//  int const element_idx = threadIdx.x;
//  int const req_id = id_map[token_idx].request_id;
//  int const tok_id = id_map[token_idx].token_id;
//  cache_ptr[(req_id * max_seq_len + tok_id) * hid_dim + element_idx] =
//  input_ptr[token_idx * hid_dim + element_idx];
//}

}; // namespace FlexFlow
