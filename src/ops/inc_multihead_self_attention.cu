#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/inc_multihead_self_attention.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

// declare Legion names
using Legion::coord_t;
using Legion::Memory;

__global__ void store_kv_cache(
    float const *input_ptr, float const *cache_ptr, request_token_id const *id_map, int max_seq_len, int hid_dim) {
  int const token_idx = blockIdx.x;
  int const element_idx = threadIdx.x;
  int const req_id = id_map[token_idx].request_id;
  int const tok_id = id_map[token_idx].token_id;
  memcpy((float *)input_ptr + token_idx * hid_dim + element_idx, (float *)cache_ptr + (req_id * max_seq_len + tok_id) * hid_dim + element_idx, sizeof(float)) ;
}

/*static*/
void IncMultiHeadSelfAttention::inference_kernel1(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    float const *input_ptr,
    float const *weight_ptr,
    float *output_ptr,
    hipStream_t stream) {

  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));

  float alpha = 1.0f, beta = 0.0f;
  int out_dim = (m->qProjSize + m->kProjSize + m->vProjSize) * m->num_heads;
  int in_dim = m->qSize;
  assert(in_dim == m->vSize && in_dim == m->kSize);
  hipDataType data_type = ff_to_cuda_datatype(DT_FLOAT);
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = HIP_R_32F;
#endif
  checkCUDA(hipblasGemmEx(m->handle.blas,
                         HIPBLAS_OP_T,
                         HIPBLAS_OP_N,
                         out_dim,
                         bc->num_tokens,
                         in_dim,
                         &alpha,
                         weight_ptr,
                         data_type,
                         in_dim,
                         input_ptr,
                         data_type,
                         in_dim,
                         &beta,
                         output_ptr,
                         data_type,
                         out_dim,
                         compute_type,
                         CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

/*static*/
void IncMultiHeadSelfAttention::inference_kernel2(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    float const *input_ptr,
    request_token_id const *id_map,
    hipStream_t stream) {
  store_kv_cache<<<bc->num_tokens, m->kSize>>>((float *)input_ptr + bc->MAX_NUM_TOKENS * m->qProjSize, m->keyCache, id_map, bc->MAX_SEQUENCE_LENGTH, m->kProjSize);
  store_kv_cache<<<bc->num_tokens, m->vSize>>>((float *)input_ptr + bc->MAX_NUM_TOKENS * (m->qProjSize + m->kProjSize), m->valueCache, id_map, bc->MAX_SEQUENCE_LENGTH, m->vProjSize);
}

/*static*/
void IncMultiHeadSelfAttention::inference_kernel_wrapper(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    float const *input_ptr,
    float const *weight_ptr,
    float *output_ptr) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  // phase 0: convert BatchConfig representation to {rid, tid} struct
  int curr_token_idx = 0;
  int curr_request_idx = 0;
  while (curr_request_idx < bc->num_requests) {
    for (int i = 0; i < bc->num_processing_tokens[curr_request_idx]; i++) {
      m->input_token_ids[curr_token_idx].request_id = curr_request_idx;
      m->input_token_ids[curr_token_idx].token_id = bc->token_start_idx[curr_request_idx] + i;
      curr_token_idx += 1;
      if (curr_token_idx >= bc->num_tokens) {
        assert(false); // total number of tokens should matches the batch config
      }
    }
    curr_request_idx += 1;
  }

  // phase 1: Implement kernel to compute KQV for input tokens
  IncMultiHeadSelfAttention::inference_kernel1(
      m, bc, input_ptr, weight_ptr, m->devQKVProjArray, stream);

  // phase 2: Update key/val cache
  IncMultiHeadSelfAttention::inference_kernel2(
      m, bc, m->devQKVProjArray, m->input_token_ids, stream);

  // phase 3: Compute attention score
  // 3 kernels for pahse 3: matmul1 - softmax - matmal2

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("IncMultiHeadSelfAttention forward time = %.2fms\n", elapsed);
    // print_tensor<3, float>(acc_query.ptr, acc_query.rect,
    // "[Attention:forward:query]"); print_tensor<3, float>(acc_output.ptr,
    // acc_output.rect, "[Attention:forward:output]");
  }
}

IncMultiHeadSelfAttentionMeta::IncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    IncMultiHeadSelfAttention const *attn,
    BatchConfig const *bc,
    Memory gpu_mem,
    int num_samples,
    int _num_heads)
    : OpMeta(handler, attn) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  // checkCUDNN(hipdnnSetStream(handler.dnn, stream));

  qSize = attn->qSize;
  kSize = attn->kSize;
  vSize = attn->vSize;
  // assume dimensions match for now
  assert(qSize == kSize);
  assert(kSize == vSize);
  qProjSize = attn->qProjSize;
  kProjSize = attn->kProjSize;
  vProjSize = attn->vProjSize;
  oProjSize = attn->oProjSize;
  num_heads = _num_heads;
  weightSize = (qSize * qProjSize + kSize * kProjSize + vSize * vProjSize +
                oProjSize * (vProjSize > 0 ? vProjSize : vSize)) *
               num_heads * sizeof(float);

  /*checkCUDNN(cudnnCreateAttnDescriptor(&attnDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&qDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&kDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&vDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&oDesc));*/
  // Currently do not support adding bias to key/value projection
  assert(!attn->add_bias_kv);
  // cudnnAttnQueryMap_t attnMode = CUDNN_ATTN_QUERYMAP_ALL_TO_ONE;
  //  Assume no beam search for now
  // int maxBeamSize = 1;
  //  printf("batchSize(%d) qSize(%d) kSize(%d) vSize(%d) qProjSize(%d)
  //  kProjSize(%d)\n",
  //      num_samples, attn->qSize, attn->kSize, attn->vSize, attn->qProjSize,
  //      attn->kProjSize);
  //  printf("vProjSize(%d) oProjSize(%d) qoSeqLength(%d) kvSeqLength(%d)\n",
  //      attn->vProjSize, attn->oProjSize, attn->qoSeqLength,
  //      attn->kvSeqLength);
  //  hipdnnMathType_t math_type;
  //  if (handle.allowTensorOpMathConversion) {
  //    math_type = CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION;
  //  } else {
  //    math_type = HIPDNN_TENSOR_OP_MATH;
  //  }
  //  checkCUDNN(cudnnSetAttnDescriptor(attnDesc,
  //                                    attnMode,
  //                                    num_heads,
  //                                    1.0f /*smScalar*/,
  //                                    HIPDNN_DATA_FLOAT,
  //                                    HIPDNN_DATA_FLOAT,
  //                                    math_type,
  //                                    NULL /*attnDropoutDesc*/,
  //                                    NULL /*postDropoutDesc*/,
  //                                    attn->qSize,
  //                                    attn->kSize,
  //                                    attn->vSize,
  //                                    attn->qProjSize,
  //                                    attn->kProjSize,
  //                                    attn->vProjSize,
  //                                    attn->oProjSize,
  //                                    attn->qoSeqLength,
  //                                    attn->kvSeqLength,
  //                                    num_samples,
  //                                    maxBeamSize));
  //  size_t workSpaceSize;
  //  checkCUDNN(cudnnGetMultiHeadAttnBuffers(
  //      handler.dnn, attnDesc, &weightSize, &workSpaceSize,
  //      &reserveSpaceSize));
  //  assert(workSpaceSize <= handler.workSpaceSize);
  //  printf("weightSize(%zu) workSpaceSize(%zu) reserveSpaceSize(%zu)\n",
  //  weightSize, workSpaceSize, reserveSpaceSize);
  /*int dimA[CUDNN_SEQDATA_DIM_COUNT];
  cudnnSeqDataAxis_t axes[CUDNN_SEQDATA_DIM_COUNT];
  assert(CUDNN_SEQDATA_DIM_COUNT == 4);
  axes[3] = CUDNN_SEQDATA_VECT_DIM; // 3 = nbDims-1
  axes[2] = CUDNN_SEQDATA_BEAM_DIM;
  axes[1] = CUDNN_SEQDATA_TIME_DIM;
  axes[0] = CUDNN_SEQDATA_BATCH_DIM;*/
  /*int *qoSeqArray = (int *)malloc(sizeof(int) * num_samples);
  int *kvSeqArray = (int *)malloc(sizeof(int) * num_samples);
  for (int i = 0; i < num_samples; i++) {
    qoSeqArray[i] = attn->qoSeqLength;
    kvSeqArray[i] = attn->kvSeqLength;
  }*/
  // Set qDesc
  /*{
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = attn->qoSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = attn->qSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(qDesc,
                                         HIPDNN_DATA_FLOAT,
                                         CUDNN_SEQDATA_DIM_COUNT,
                                         dimA,
                                         axes,
                                         num_samples,
                                         qoSeqArray,
                                         NULL));
  }
  // Set kDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = attn->kvSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = attn->kSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(kDesc,
                                         HIPDNN_DATA_FLOAT,
                                         CUDNN_SEQDATA_DIM_COUNT,
                                         dimA,
                                         axes,
                                         num_samples,
                                         kvSeqArray,
                                         NULL));
  }
  // Set vDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = attn->kvSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = attn->vSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(vDesc,
                                         HIPDNN_DATA_FLOAT,
                                         CUDNN_SEQDATA_DIM_COUNT,
                                         dimA,
                                         axes,
                                         num_samples,
                                         kvSeqArray,
                                         NULL));
  }
  // Set oDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = attn->qoSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = attn->oProjSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(oDesc,
                                         HIPDNN_DATA_FLOAT,
                                         CUDNN_SEQDATA_DIM_COUNT,
                                         dimA,
                                         axes,
                                         num_samples,
                                         qoSeqArray,
                                         NULL));
  }*/
  // allocate memory for the seqArray and reserve space
  {
    // size_t totalSize = reserveSpaceSize + sizeof(int) * num_samples * 2 +
    // bc->MAX_NUM_REQUESTS *bc-> MAX_SEQUENCE_LENGTH * sizeof(int); size_t
    // max_num_tokens = bc->MAX_NUM_REQUESTS * bc->MAX_SEQUENCE_LENGTH;
    size_t qkv_proj_dim = qProjSize + kProjSize + vProjSize;
    size_t qkv_max_proj_size = num_samples * qkv_proj_dim * num_heads;
    size_t key_cache_size = kProjSize * bc->MAX_NUM_REQUESTS * bc->MAX_SEQUENCE_LENGTH;
    size_t value_cache_size = vProjSize * bc->MAX_NUM_REQUESTS * bc->MAX_SEQUENCE_LENGTH;

    size_t totalSize =
        (qkv_max_proj_size  + key_cache_size + value_cache_size) *
        sizeof(float); // more components will be added here later

    Realm::Rect<1, coord_t> bounds(Realm::Point<1, coord_t>(0),
                                   Realm::Point<1, coord_t>(totalSize - 1));
    std::vector<size_t> field_sizes;
    field_sizes.push_back(sizeof(char));
    Realm::RegionInstance::create_instance(reserveInst,
                                           gpu_mem,
                                           bounds,
                                           field_sizes,
                                           0,
                                           Realm::ProfilingRequestSet())
        .wait();
    devQKVProjArray = (float *)reserveInst.pointer_untyped(0, sizeof(char));
    keyCache = (float *)devQKVProjArray + qkv_max_proj_size;
    valueCache = (float *)keyCache + key_cache_size;
    // checkCUDA(hipMemcpy(devQoSeqArray,
    //                      qoSeqArray,
    //                      sizeof(int) * num_samples,
    //                      hipMemcpyHostToDevice));
    // devKvSeqArray = (int *)devQoSeqArray + num_samples;
    // checkCUDA(hipMemcpy(devKvSeqArray,
    //                      kvSeqArray,
    //                      sizeof(int) * num_samples,
    //                      hipMemcpyHostToDevice));
    // kvCache = (int *)devKvSeqArray + num_samples;
    // reserveSpace = (int *)kvCache + bc->MAX_NUM_REQUESTS * bc->
    // MAX_SEQUENCE_LENGTH;
  }

  input_token_ids = new request_token_id[bc->MAX_NUM_TOKENS];
  
  /*// allocate memory for loWinIdx/hiWinIdx
  loWinIdx = (int *)malloc(sizeof(int) * attn->qoSeqLength);
  hiWinIdx = (int *)malloc(sizeof(int) * attn->qoSeqLength);
  for (int i = 0; i < attn->qoSeqLength; i++) {
    loWinIdx[i] = 0;
    hiWinIdx[i] = attn->kvSeqLength;
  }*/
  // free(qoSeqArray);
  // free(kvSeqArray);
}

IncMultiHeadSelfAttentionMeta::~IncMultiHeadSelfAttentionMeta(void) {
  reserveInst.destroy();
  /*free(loWinIdx);
  free(hiWinIdx);
  checkCUDNN(cudnnDestroyAttnDescriptor(attnDesc));
  checkCUDNN(cudnnDestroySeqDataDescriptor(qDesc));
  checkCUDNN(cudnnDestroySeqDataDescriptor(kDesc));
  checkCUDNN(cudnnDestroySeqDataDescriptor(vDesc));
  checkCUDNN(cudnnDestroySeqDataDescriptor(oDesc));*/
}

//__global__ void store_kv_cache(
//    float const *input_ptr, float const *cache_ptr, request_token_id const *id_map, int max_seq_len, int hid_dim) {
//  int const token_idx = blockIdx.x;
//  int const element_idx = threadIdx.x;
//  int const req_id = id_map[token_idx].request_id;
//  int const tok_id = id_map[token_idx].token_id;
//  cache_ptr[(req_id * max_seq_len + tok_id) * hid_dim + element_idx] = input_ptr[token_idx * hid_dim + element_idx];
//}

}; // namespace FlexFlow
