#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "hip/hip_complex.h"
#include "flexflow/ffconst_utils.h"
#include "flexflow/ops/inc_multihead_self_attention.h"
#include "flexflow/ops/kernels/decompress_kernels.h"
#include "flexflow/ops/kernels/inc_multihead_self_attention_kernels.h"
#include "flexflow/ops/kernels/inc_multihead_self_attention_utils.cuh"
#include "flexflow/utils/cuda_helper.h"
#include <hip/hip_math_constants.h>

namespace FlexFlow {

// declare Legion names
using Legion::coord_t;
using Legion::Memory;

#define WARP_SIZE 32

namespace Kernels {
namespace IncMultiHeadAttention {

template <typename DT>
__global__ void store_kv_cache(DT const *devQKVProjArray,
                               DT *kCache_ptr,
                               DT *vCache_ptr,
                               BatchConfig::PerTokenInfo const *tokenInfos,
                               int num_tokens,
                               int max_seq_len,
                               int hidden_size) {
  CUDA_KERNEL_LOOP(i, num_tokens * hidden_size) {
    int token_idx = i / hidden_size;
    int offset = i % hidden_size;

    size_t val_idx =
        token_idx * QKV_WEIGHT_NUM * hidden_size + hidden_size + offset;

    DT kVal = devQKVProjArray[val_idx];
    DT vVal = devQKVProjArray[val_idx + hidden_size];
    int const req_id = tokenInfos[token_idx].request_index;
    int const tok_id = tokenInfos[token_idx].abs_depth_in_request;

    // key cache
    kCache_ptr[req_id * (hidden_size * max_seq_len) + tok_id * hidden_size +
               offset] = kVal;
    vCache_ptr[req_id * (hidden_size * max_seq_len) + tok_id * hidden_size +
               offset] = vVal;
  }
}

template <typename DT>
__global__ void store_query_cache(DT const *devQKVProjArray,
                                  DT *qCache_ptr,
                                  int num_tokens,
                                  int hidden_size) {
  CUDA_KERNEL_LOOP(i, num_tokens * hidden_size) {
    int token_idx = i / hidden_size;
    int offset = i % hidden_size;

    size_t val_idx = token_idx * QKV_WEIGHT_NUM * hidden_size + offset;

    DT qVal = devQKVProjArray[val_idx];

    // query cache
    qCache_ptr[i] = qVal;
  }
}

template <typename DT>
__global__ void fill_entries_above_diagonal(DT *matrix,
                                            size_t num_rows,
                                            size_t num_cols,
                                            size_t num_q_heads,
                                            size_t entries_above_diagonal,
                                            DT value) {
  CUDA_KERNEL_LOOP(i, entries_above_diagonal * num_q_heads) {
    size_t head_idx = i / entries_above_diagonal;
    size_t entry_idx = i % entries_above_diagonal;
    size_t y = (-1 + sqrt(8 * (float)entry_idx + 1)) / 2;
    size_t x = entry_idx - y * (y + 1) / 2;
    y += (num_cols - num_rows) + 1;
    matrix[head_idx * num_rows * num_cols + num_cols * y + x] = value;
  }
}

template <typename DT>
void compute_attention_kernel_prompt(IncMultiHeadSelfAttentionMeta *m,
                                     BatchConfig const *bc,
                                     int shard_id,
                                     hipStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  hipDataType cublas_data_type = ff_to_cuda_datatype(m->output_type[0]);
  hipdnnDataType_t cudnn_data_type = ff_to_cudnn_datatype(m->output_type[0]);
  assert(data_type_size(m->output_type[0]) == sizeof(DT));
  hipDataType compute_type = cublas_data_type;

  int num_tokens = bc->num_active_tokens();
  int tokens_previous_requests = 0;
  int q_block_size = m->qProjSize;
  int kt_block_size = m->kProjSize;
  int kt_req_block_size =
      kt_block_size * m->num_q_heads * BatchConfig::max_sequence_length();
  int vt_block_size = m->vProjSize;
  int vt_req_block_size =
      vt_block_size * m->num_q_heads * BatchConfig::max_sequence_length();
  assert(m->qProjSize == m->kProjSize);

  for (int i = 0; i < bc->max_requests_per_batch(); i++) {
    if (bc->request_completed[i] ||
        (!bc->requestsInfo[i].prompt_phase && !bc->requestsInfo[i].peft_bwd)) {
      continue;
    }
    int num_new_tokens = bc->requestsInfo[i].num_tokens_in_batch;
    int total_tokens = bc->requestsInfo[i].first_token_depth_in_request +
                       bc->requestsInfo[i].num_tokens_in_batch;
    int max_peft_tokens = bc->requestsInfo[i].max_length;
    // Copy query to m->query_activation_buffer if we need to compute
    // PEFT backward
    if (bc->requestsInfo[i].peft_bwd) {
      size_t activation_size_needed =
          sizeof(DT) * max_peft_tokens * m->num_q_heads * m->qProjSize;
      if (activation_size_needed > m->allocated_peft_buffer_size1) {
        MemoryAllocator *allocator = m->handle.peft_activation_allocator;
        m->query_activation_buffer =
            allocator->allocate_instance_untyped(activation_size_needed);
        m->allocated_peft_buffer_size1 = activation_size_needed;
      }
      int parallelism = m->hidden_size * num_tokens;
      store_query_cache<<<GET_BLOCKS(parallelism),
                          min(CUDA_NUM_THREADS, parallelism),
                          0,
                          stream>>>(
          static_cast<DT *>(m->devQKVProjArray),
          static_cast<DT *>(m->query_activation_buffer),
          num_tokens,
          m->hidden_size);
    }
    // Step 1: compute query-key product QK.T/sqrt(d_k)
    {
      // Scale by sqrt(d_k) as per the original attention paper
      DT alpha = 1.0f, beta = 0.0f;
      if (*m->qk_prod_scaling) {
        alpha = static_cast<DT>(1.0f / sqrt(m->kProjSize));
      }
      // after transpositions
      int m_ = num_new_tokens;
      int n = total_tokens;
      int k = m->qProjSize;
      // before transpositions
      int lda = k * m->num_q_heads * QKV_WEIGHT_NUM, ldb = k * m->num_q_heads,
          ldc = m_;
      // N.B. strides are applied before transpose operations
      int strideA = q_block_size;
      int strideB = kt_block_size;
      int strideC = num_new_tokens * total_tokens;

      // matrix A: devQKVProjArray
      // matrix A's layout: [qProjSize, num_heads, 3, num_new_tokens]
      // To get query projection, skip over Q entries from previous requests
      DT const *A = static_cast<DT *>(m->devQKVProjArray) +
                    bc->requestsInfo[i].first_token_offset_in_batch *
                        m->qProjSize * m->num_q_heads * QKV_WEIGHT_NUM;
      // matrix B: key cache
      // matrix B's layout: [kProjSize * num_heads, total_tokens]
      // To get B, skip over K entries from previous requests (all heads +
      // padding)
      DT const *B = static_cast<DT *>(m->keyCache) + i * kt_req_block_size;
      // matrix C: qk_prods
      // matrix C's layout: [num_new_tokens, total_tokens, num_heads]
      // To get C, skip over QK.T products from previous requests
      DT *C = static_cast<DT *>(m->qk_prods);
      checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                           HIPBLAS_OP_T,
                                           HIPBLAS_OP_N,
                                           m_,
                                           n,
                                           k,
                                           &alpha,
                                           A,
                                           cublas_data_type,
                                           lda,
                                           strideA,
                                           B,
                                           cublas_data_type,
                                           ldb,
                                           strideB,
                                           &beta,
                                           C,
                                           cublas_data_type,
                                           ldc,
                                           strideC,
                                           m->num_q_heads,
                                           compute_type,
                                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    }
    // Step 2: Add alibi position bias to qk production
    // matrix C: qk_prods
    // matrix C's layout: [num_new_tokens, total_tokens, num_heads]
    // To get C, skip over QK.T products from previous requests
    DT *C = static_cast<DT *>(m->qk_prods);
    if (*m->position_bias) {
      size_t parallelism = m->num_q_heads * total_tokens * num_new_tokens;
      apply_position_bias_qkprd<<<GET_BLOCKS(parallelism),
                                  min((size_t)CUDA_NUM_THREADS, parallelism),
                                  0,
                                  stream>>>(C,
                                            num_new_tokens,
                                            total_tokens,
                                            m->num_q_heads,
                                            m->global_num_q_heads,
                                            shard_id);
    }

    // Step 3: Apply causal mask. Fill all elements above diagonal in qk prods
    // with -inf to force causal attention.
    assert(num_new_tokens <= total_tokens);
    size_t entries_above_diagonal = num_new_tokens * (num_new_tokens - 1) / 2;
    if (entries_above_diagonal > 0) {
      size_t parallelism = m->num_q_heads * entries_above_diagonal;
      fill_entries_above_diagonal<<<GET_BLOCKS(parallelism),
                                    min((size_t)CUDA_NUM_THREADS, parallelism),
                                    0,
                                    stream>>>(C,
                                              num_new_tokens,
                                              total_tokens,
                                              m->num_q_heads,
                                              entries_above_diagonal,
                                              static_cast<DT>(-INFINITY));
    }

    // Step 4: Compute Softmax(QK.T/sqrt(d_k))
    {
      // Before modifying the parameters below, make sure to read the following
      // description of the HIPDNN_TENSOR_NCHW tensor layout, from
      // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnTensorFormat_t:
      // This tensor format specifies that the data is laid out in the following
      // order: batch size, feature maps, rows, columns. The strides are
      // implicitly defined in such a way that the data are contiguous in memory
      // with no padding between images, feature maps, rows, and columns; the
      // columns are the inner dimension and the images are the outermost
      // dimension.
      int n_param = m->num_q_heads;
      int c_param = total_tokens;
      int h_param = 1;
      int w_param = num_new_tokens;
      checkCUDNN(hipdnnSetTensor4dDescriptor(m->qk_tensor,
                                            HIPDNN_TENSOR_NCHW,
                                            cudnn_data_type,
                                            n_param,
                                            c_param,
                                            h_param,
                                            w_param));
      float softmax_alpha = 1.0f, softmax_beta = 0.0f;
      DT *C_softmax = static_cast<DT *>(m->qk_prods_softmax);
      // The softmax operation below is executed according to the
      // HIPDNN_SOFTMAX_MODE_CHANNEL, which is also described in the docs: The
      // softmax operation is computed per spatial location (H,W) per image (N)
      // across dimension C.
      checkCUDNN(hipdnnSoftmaxForward(m->handle.dnn,
                                     HIPDNN_SOFTMAX_ACCURATE,
                                     HIPDNN_SOFTMAX_MODE_CHANNEL,
                                     &softmax_alpha,
                                     m->qk_tensor,
                                     C,
                                     &softmax_beta,
                                     m->qk_tensor,
                                     C_softmax));
    }
    // Copy C_softmax to m->softmax_activation_buffer if we need to compute
    // PEFT backward
    if (bc->requestsInfo[i].peft_bwd) {
      DT *C_softmax = static_cast<DT *>(m->qk_prods_softmax);
      size_t activation_size_needed =
          sizeof(DT) * max_peft_tokens * max_peft_tokens * m->num_q_heads;
      if (activation_size_needed > m->allocated_peft_buffer_size2) {
        MemoryAllocator *allocator = m->handle.peft_activation_allocator;
        m->softmax_activation_buffer =
            allocator->allocate_instance_untyped(activation_size_needed);
        m->allocated_peft_buffer_size2 = activation_size_needed;
      }
      checkCUDA(hipMemcpyAsync(m->softmax_activation_buffer,
                                C_softmax,
                                sizeof(DT) * total_tokens * num_new_tokens *
                                    m->num_q_heads,
                                hipMemcpyDeviceToDevice,
                                stream));
    }
    // Step 5: Matmul softmax(QK.T/sqrt(d_k)) by V. Implemented as V @
    // softmax(QK.T/sqrt(d_k)).T
    {
      DT alpha = 1.0f, beta = 0.0f;
      // after transpositions
      int m_ = m->vProjSize;
      int n = num_new_tokens;
      int k = total_tokens;
      // before transpositions
      int lda = m_ * m->num_q_heads, ldb = n, ldc = m_ * m->num_q_heads;
      // N.B. strides are applied before transpose operations
      int strideA = vt_block_size;
      int strideB = num_new_tokens * total_tokens;
      int strideC = m->vProjSize;
      // matrix A: value cache
      // matrix A's layout: [vProjSize, num_heads, total_tokens]
      // To get A, skip over V.T entries from previous requests (all heads +
      // padding)
      DT *A = static_cast<DT *>(m->valueCache) + i * vt_req_block_size;
      // matrix B: qk_prods_softmax
      // matrix B's layout: [num_new_tokens, total_tokens, num_heads]
      // To get B, skip over softmax(QK.T/sqrt(d_k)) entries from previous
      // requests (all heads)
      DT *B = static_cast<DT *>(m->qk_prods_softmax);
      // matrix C: attn heads
      // matrix C's layout: [vProjSize, num_heads, num_new_tokens]
      // To get C, skip over softmax(QK.T/sqrt(d_k))V products from previous
      // requests
      // store the result attn heads, also skip the genration tokens
      DT *C = static_cast<DT *>(m->attn_heads) +
              (bc->requestsInfo[i].first_token_offset_in_batch) *
                  m->num_q_heads * m->vProjSize;
      checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                           HIPBLAS_OP_N,
                                           HIPBLAS_OP_T,
                                           m_,
                                           n,
                                           k,
                                           &alpha,
                                           A,
                                           cublas_data_type,
                                           lda,
                                           strideA,
                                           B,
                                           cublas_data_type,
                                           ldb,
                                           strideB,
                                           &beta,
                                           C,
                                           cublas_data_type,
                                           ldc,
                                           strideC,
                                           m->num_q_heads,
                                           compute_type,
                                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    }
    tokens_previous_requests += num_new_tokens;
  }
  if (tokens_previous_requests != (num_tokens - bc->num_generation_tokens)) {
    bc->print();
    printf("tokens_previous_requests: %i\n", tokens_previous_requests);
    printf("num_tokens: %i\n", num_tokens);
    printf("bc->num_generation_tokens: %i\n", bc->num_generation_tokens);
  }
  assert(tokens_previous_requests == (num_tokens - bc->num_generation_tokens));
}

// gridDim = num_heads
// blockDim = num_tokens/num_request * head_size
// QKV tensor layout: |QKV| * num_new_tokens. |Q=K=V=head_size * num_heads|
// one thread process one head_size
template <typename DT,
          int THREADS_PER_BLOCK,
          int Dh,
          int Dh_MAX,
          int THREADS_PER_KEY,
          int THREADS_PER_VALUE>
__global__ void compute_attention_kernel_generation_kernel(
    DT const *query,
    DT const *key_cache,
    DT const *value_cache,
    DT *output_ptr,
    float const scale,
    int max_seq_length,
    int per_head_size,
    int hidden_size,
    BatchConfig::PerRequestInfo *request_infos) {

  // q, k
  using Q_vec = typename VEC_K<DT, THREADS_PER_KEY>::Type;
  using K_vec = typename VEC_K<DT, THREADS_PER_KEY>::Type;
  using V_vec = typename VEC_V<DT>::Type;
  using Out_sum = typename Vec_fp32_<V_vec>::Type;

  constexpr int WARPS_PER_BLOCK = THREADS_PER_BLOCK / WARP_SIZE;

  // eg.  if head_size = 128, thread_per_key = 4, with float32 precision
  // then K_VEC_SIZE = 1,  QK_VEC_SIZE = 4
  //  K_ELTS_PER_THREAD = 128 / 4 = 32
  //  K_VECS_PER_THREAD = 32 / 1 = 32
  constexpr int K_VEC_SIZE = sizeof(K_vec) / sizeof(DT);
  // constexpr int QK_VEC_SIZE = 16 / sizeof(DT);
  // // constexpr int QK_VEC_SIZE = sizeof(Qk_vec_k) / sizeof(DT);
  constexpr int K_ELTS_PER_THREAD = Dh / THREADS_PER_KEY;
  constexpr int K_VECS_PER_THREAD = K_ELTS_PER_THREAD / K_VEC_SIZE;
  // constexpr int QK_ELTS_IN_16B = 16 / sizeof(DT);

  // thread id
  int const tidx = threadIdx.x;
  // head id
  int const head_idx = blockIdx.x;
  // request idx
  int const request_idx = blockIdx.y;

  int const batch_config_request_id =
      request_infos[request_idx].batch_config_request_id;

  int const first_step = 0;

  int const tlength =
      request_infos[batch_config_request_id].first_token_depth_in_request +
      request_infos[batch_config_request_id].num_tokens_in_batch;

  // shared memory objects
  extern __shared__ char smem_[];

  float *qk_smem = reinterpret_cast<float *>(smem_);
  float *out_smem = reinterpret_cast<float *>(smem_);

  float qk_max = -FLT_MAX;

  // first WARPS_PER_BLOCK for store qk_max, second WARPS_PER_BLOCK for sum
  __shared__ float red_smem[WARPS_PER_BLOCK * 2];

  const DT *q_ptr = query + request_idx * hidden_size * QKV_WEIGHT_NUM +
                    head_idx * per_head_size;
  __shared__ Q_vec q_vecs[THREADS_PER_KEY][K_VECS_PER_THREAD];
  // DT const *q_ptr =
  //     query + request_idx * Dh * QKV_WEIGHT_NUM + head_idx * per_head_size;

  // q tensor in this thread
  // if THREADS_PER_KEY is 4, first thread load 0, 4, 8, 12..., total
  // K_VECS_PER_THREAD elements
  // QK_vec_k: 32->1, 64->2, 128->4... head_size
  // K_vec_k: 4->1, 2->2, 1->4 threads_per_key

  // the start offset of the element eg. (0, 1, 2, 3) * K_VEC_SIZE
  int ki = tidx % THREADS_PER_KEY * K_VEC_SIZE;
  int ki_o = tidx % THREADS_PER_KEY;
  // the first key's offset for this thread
  // ko = 0, 0, 0, 0, 1, 1, 1, 1, ....
  int ko = tidx / THREADS_PER_KEY;
  // load q tensor
  Q_vec q_vec[K_VECS_PER_THREAD];
#pragma unroll
  for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
    q_vecs[ki_o][ii] = *reinterpret_cast<Q_vec const *>(
        q_ptr + ki + ii * THREADS_PER_KEY * K_VEC_SIZE);
  }
  __syncthreads();
  // first iter = 128 / 4 = 32
  // K_VECS_PER_THREAD = 32
  //  K_PER_ITER how many keys in this loop
  //  The number of timesteps loaded per iteration.
  constexpr int K_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_KEY;
  //   // The number of keys per warp.
  constexpr int K_PER_WARP = WARP_SIZE / THREADS_PER_KEY;

  DT const *k_cache_batch =
      key_cache + batch_config_request_id * max_seq_length * hidden_size + ki;

  int ti_end =
      div_up(tlength - first_step, K_PER_WARP) * K_PER_WARP + first_step;
  // get k, perform qk proj

  for (int ti = ko; ti < ti_end; ti += K_PER_ITER) {
    K_vec k[K_VECS_PER_THREAD];
    int const ti_circ = ti % max_seq_length;
#pragma unroll
    for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
      int jj = ii * THREADS_PER_KEY * K_VEC_SIZE;
      if (ti < tlength) {
        k[ii] = *reinterpret_cast<K_vec const *>(k_cache_batch +
                                                 ti_circ * hidden_size +
                                                 head_idx * per_head_size + jj);
      }
      // Compute dot product.
      // This includes a reduction across the threads in the same thread group.
    }
    float qk = scale * Qk_dot<DT, THREADS_PER_KEY>::dot(q_vecs[ki_o], k);
    // // todo add positional embedding to the qk production
    // // Store the product to shared memory. There's one qk value per
    // timestep.
    // // Update the max.
    if (ti < tlength && tidx % THREADS_PER_KEY == 0) {
      // todo add alobi here
      bool const mask = ti_circ >= tlength;
      if (mask) {
        assert(false);
      }
      qk_max = mask ? qk_max : fmaxf(qk_max, qk);
      qk_smem[ti - first_step] = mask ? 0.f : qk;
    }
  }

  __syncthreads();

#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= THREADS_PER_KEY; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }

  // Decompose the thread index into warp and lane.
  int const warp = tidx / WARP_SIZE;
  int const lane = tidx % WARP_SIZE;

  // The warp leader writes the max to shared memory.
  if (lane == 0) {
    red_smem[warp] = qk_max;
  }

  // Make sure the products are in shared memory.
  __syncthreads();

  // The warps finalize the reduction.
  qk_max = lane < WARPS_PER_BLOCK ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = WARPS_PER_BLOCK / 2; mask >= 1; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }

  // Broadcast to all the threads in the warp.
  qk_max = __shfl_sync(uint32_t(-1), qk_max, 0);

  float exp_sum = 0.f;
  for (int ti = first_step + tidx; ti < tlength; ti += THREADS_PER_BLOCK) {
    float logit = __expf(qk_smem[ti - first_step] - qk_max);
    exp_sum += logit;
    qk_smem[ti - first_step] = logit;
  }

  // Compute the sum.
  exp_sum = block_sum<WARPS_PER_BLOCK>(&red_smem[WARPS_PER_BLOCK], exp_sum);

  // softmax
  float inv_sum = __fdividef(1.f, exp_sum + 1.e-6);
  for (int ti = first_step + tidx; ti < tlength; ti += THREADS_PER_BLOCK) {
    qk_smem[ti - first_step] *= inv_sum;
  }

  __syncthreads();
  // if (blockIdx.y == 0 && blockIdx.x == 0 && tidx == 0) {
  //   printf("softmax %.10f\n", qk_smem[0]);
  // }

  // value projection
  constexpr int V_VEC_SIZE = 16 / sizeof(DT);
  // A vector of V elements for the current timestep.
  // using V_vec_k = typename V_vec_k_<DT, V_VEC_SIZE>::Type;
  // using V_vec_acum = typename V_vec_acum_fp32_<V_vec_k>::Type;

  // The value computed by this thread.
  int vo = tidx / THREADS_PER_VALUE;
  // The hidden dimensions computed by this particular thread.
  int vi = tidx % THREADS_PER_VALUE * V_VEC_SIZE;
  constexpr int V_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_VALUE;

  Out_sum out;
  zero(out);

  // The base pointer for the value in the cache buffer.
  DT const *v_cache_batch =
      value_cache + batch_config_request_id * max_seq_length * hidden_size + vi;

  if (Dh == Dh_MAX || vi < Dh) {
    for (int ti = first_step + vo; ti < tlength; ti += V_PER_ITER) {
      // Load the values from the cache.
      int const ti_circ = ti % max_seq_length;

      V_vec v = *reinterpret_cast<V_vec const *>(
          v_cache_batch + ti_circ * hidden_size + head_idx * per_head_size);
      float logit = qk_smem[ti - first_step];
      out = FlexFlow::fma(logit, cast_to_float(v), out);
    }
  }

  //   // Make sure we can start writing to shared memory.
  __syncthreads();

  // Run the final reduction amongst the different groups computing different
  // partial outputs.
  if (Dh == Dh_MAX || vi < Dh) {
#pragma unroll
    for (int active_groups = V_PER_ITER; active_groups >= 2;
         active_groups /= 2) {

      // The midpoint in the number of active groups.
      int midpoint = active_groups / 2;

      // The upper part of active threads store to shared memory.
      if (vo >= midpoint && vo < active_groups && (Dh == Dh_MAX || vi < Dh)) {
        *reinterpret_cast<Out_sum *>(out_smem + (vo - midpoint) * Dh + vi) =
            out;
      }
      __syncthreads();

      // The bottom warps update their values.
      if (vo < midpoint && (Dh == Dh_MAX || vi < Dh)) {
        out = add(*reinterpret_cast<Out_sum const *>(out_smem + vo * Dh + vi),
                  out);
      }
      __syncthreads();
    }
  }

  // Output the final values.
  if (vo == 0 && (Dh == Dh_MAX || vi < Dh)) {
    convert_from_float(
        *reinterpret_cast<V_vec *>(output_ptr + request_idx * hidden_size +
                                   head_idx * per_head_size + vi),
        out);
  }
}

// only used by MPT model. https://arxiv.org/abs/2108.12409
template <typename DT>
__global__ void apply_position_bias_qkprd(DT *input_ptr,
                                          int num_tokens,
                                          int num_total_tokens,
                                          int num_heads,
                                          int global_num_q_heads,
                                          int shard_id) {
  CUDA_KERNEL_LOOP(i, num_tokens * num_total_tokens * num_heads) {
    // get head_idx,
    int head_idx = i / (num_tokens * num_total_tokens) + (num_heads * shard_id);
    int position_idx = (i / num_tokens) % num_total_tokens;
    position_idx = position_idx + 1 - num_total_tokens;
    // 8 is alibi_bias_max in
    // https://huggingface.co/mosaicml/mpt-30b/blob/main/config.json
    float base = (float)(head_idx + 1) * 8 / global_num_q_heads;
    float slopes = 1.0 / pow(2, base);
    // if(i == 0){
    //   printf("see position: %d, %f, %f, %f\n", position_idx, base, slopes,
    //   position_idx * slopes);
    // }
    input_ptr[i] += static_cast<DT>(position_idx * slopes);
  }
}

template <typename DT>
__global__ void scaling_query_kernel(DT *input_ptr,
                                     int qProjSize,
                                     int num_tokens,
                                     int num_q_heads,
                                     float scaling_factor,
                                     int hidden_size) {
  CUDA_KERNEL_LOOP(i, num_tokens * hidden_size) {
    int token_idx = i / hidden_size;
    input_ptr[i % hidden_size + token_idx * hidden_size * QKV_WEIGHT_NUM] *=
        scaling_factor;
  }
}

template <typename DT>
__global__ void
    apply_rotary_embedding_hf(DT *input_ptr,
                              hipFloatComplex *complex_input,
                              BatchConfig::PerTokenInfo const *tokenInfos,
                              float rope_theta,
                              bool llama3_rope,
                              float factor,
                              float low_freq_factor,
                              float high_freq_factor,
                              int original_max_position_embeddings,
                              int qProjSize,
                              int kProjSize,
                              int num_tokens,
                              size_t q_array_size,
                              int hidden_size) {
  CUDA_KERNEL_LOOP(i, num_tokens * hidden_size) {
    // create complex number
    bool q_tensor = i < (q_array_size / 2);
    int proj_size = q_tensor ? qProjSize : kProjSize;
    int real_i = q_tensor ? i : i - q_array_size / 2;

    int token_idx = real_i / (hidden_size / 2);
    int idx = real_i % (proj_size / 2);
    int head_idx = (real_i - (token_idx * (hidden_size / 2))) / (proj_size / 2);

    int real_part_index = idx + head_idx * proj_size +
                          token_idx * hidden_size * QKV_WEIGHT_NUM +
                          hidden_size * (q_tensor ? 0 : 1);
    int complex_part_index = real_part_index + (proj_size / 2);

    complex_input[i] = {input_ptr[real_part_index],
                        input_ptr[complex_part_index]};

    // get the freq_cis: shape 1 * (qProjSize/2) = 1 * 64
    // apply a Cartesian coordinate transformation
    // multiple with input & /copy back to q/k

    // get position of token

    // size_t pos = id_map[token_idx].token_position;
    size_t pos = tokenInfos[token_idx].abs_depth_in_request;

    // float before_real = complex_input[i].x, before_complex =
    int pos_i = real_i % (proj_size / 2);

    float freq =
        pos * (1.0 / pow(rope_theta, (float)2 * pos_i / proj_size)); // θ_i

    if (llama3_rope) {
      float pi = HIP_PI_F;
      float wavelen = 2 * pi / freq;
      float low_freq_wavelen =
          original_max_position_embeddings / low_freq_factor;
      float high_freq_wavelen =
          original_max_position_embeddings / high_freq_factor;
      if (wavelen < high_freq_wavelen) {
      } else if (wavelen > low_freq_wavelen) {
        freq = freq / factor;
      } else {
        assert(low_freq_wavelen != high_freq_wavelen);
        float smooth =
            (original_max_position_embeddings / wavelen - low_freq_factor) /
            (high_freq_factor - low_freq_factor);
        freq = ((1 - smooth) * freq / factor + smooth * freq);
      }
    }

    hipFloatComplex complex_pos = {cos(freq), sin(freq)};

    complex_input[i] = hipCmulf(complex_input[i], complex_pos);
    input_ptr[real_part_index] = complex_input[i].x;
    input_ptr[complex_part_index] = complex_input[i].y;
  }
}

template <typename DT>
__global__ void
    apply_rotary_embedding_bwd(DT *input_ptr,
                               hipFloatComplex *complex_input,
                               BatchConfig::PerTokenInfo const *tokenInfos,
                               float rope_theta,
                               bool llama3_rope,
                               float factor,
                               float low_freq_factor,
                               float high_freq_factor,
                               int original_max_position_embeddings,
                               int proj_size,
                               int num_tokens,
                               int hidden_size) {
  CUDA_KERNEL_LOOP(i, num_tokens * hidden_size) {
    // compute indexes to visit first half proj_size of each of q/k tensor.
    // devQKVProj has shape [num_tokens, qProjSize, num_heads, 3] in peft_bwd
    bool q_tensor = i < (num_tokens * hidden_size / 2);
    int real_i = q_tensor ? i : i - num_tokens * hidden_size / 2;
    assert(hidden_size % proj_size == 0);
    int num_heads = hidden_size / proj_size;

    int token_idx = real_i % num_tokens;
    int idx = (real_i / num_tokens) % (proj_size / 2);
    int head_idx = real_i / (num_tokens * proj_size / 2);
    assert(head_idx < num_heads);

    int complex_part_index = (q_tensor ? 0 : 1) * num_tokens * hidden_size +
                             head_idx * num_tokens * proj_size +
                             idx * num_tokens + token_idx;
    int real_part_index = complex_part_index + (proj_size / 2) * num_tokens;

    complex_input[i] = {input_ptr[real_part_index],
                        input_ptr[complex_part_index]};

    size_t pos = tokenInfos[token_idx].abs_depth_in_request;

    float freq =
        pos * (1.0 / pow(rope_theta, (float)2 * idx / proj_size)); // θ_i

    if (llama3_rope) {
      float pi = HIP_PI_F;
      float wavelen = 2 * pi / freq;
      float low_freq_wavelen =
          original_max_position_embeddings / low_freq_factor;
      float high_freq_wavelen =
          original_max_position_embeddings / high_freq_factor;
      if (wavelen < high_freq_wavelen) {
      } else if (wavelen > low_freq_wavelen) {
        freq = freq / factor;
      } else {
        assert(low_freq_wavelen != high_freq_wavelen);
        float smooth =
            (original_max_position_embeddings / wavelen - low_freq_factor) /
            (high_freq_factor - low_freq_factor);
        freq = ((1 - smooth) * freq / factor + smooth * freq);
      }
    }

    hipFloatComplex complex_pos = {cos(freq), sin(freq)};

    complex_input[i] = hipCmulf(complex_input[i], complex_pos);
    input_ptr[real_part_index] = complex_input[i].x;
    input_ptr[complex_part_index] = complex_input[i].y;
  }
}

template <typename DT>
void compute_qkv_kernel(IncMultiHeadSelfAttentionMeta const *m,
                        BatchConfig const *bc,
                        int shard_id,
                        DT *output_ptr,
                        hipStream_t stream) {

  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  assert(m->qSize == m->vSize && m->qSize == m->kSize);

  int num_tokens = bc->num_active_tokens();
  int parallelism = m->kProjSize * num_tokens * m->num_q_heads;
  size_t q_array_size = m->qProjSize * num_tokens * m->num_q_heads;

  if (m->scaling_query) {
    scaling_query_kernel<<<GET_BLOCKS(parallelism),
                           min(CUDA_NUM_THREADS, parallelism),
                           0,
                           stream>>>(output_ptr,
                                     m->qProjSize,
                                     num_tokens,
                                     m->num_q_heads,
                                     m->scaling_factor,
                                     m->hidden_size);
  }

  // Step 3: apply rotary embedding if needed
  if (m->rotary_embedding_meta->apply_rotary_embedding) {
    /*q&k*/
    parallelism = num_tokens * m->hidden_size;
    apply_rotary_embedding_hf<<<GET_BLOCKS(parallelism),
                                min(CUDA_NUM_THREADS, parallelism),
                                0,
                                stream>>>(
        output_ptr,
        m->complex_input,
        m->token_infos,
        m->rotary_embedding_meta->rope_theta,
        (m->rotary_embedding_meta->rope_type == "llama3"),
        m->rotary_embedding_meta->factor,
        m->rotary_embedding_meta->low_freq_factor,
        m->rotary_embedding_meta->high_freq_factor,
        m->rotary_embedding_meta->original_max_position_embeddings,
        m->qProjSize,
        m->kProjSize,
        num_tokens,
        q_array_size,
        m->hidden_size);
  }
}

template <typename DT>
void update_kv_cache_kernel(IncMultiHeadSelfAttentionMeta const *m,
                            BatchConfig const *bc,
                            hipStream_t stream) {
  int num_tokens = bc->num_active_infr_tokens();
  if (num_tokens > 0) {
    int parallelism = m->hidden_size * num_tokens;
    store_kv_cache<<<GET_BLOCKS(parallelism),
                     min(CUDA_NUM_THREADS, parallelism),
                     0,
                     stream>>>(static_cast<DT *>(m->devQKVProjArray),
                               static_cast<DT *>(m->keyCache),
                               static_cast<DT *>(m->valueCache),
                               m->token_infos,
                               num_tokens,
                               BatchConfig::max_sequence_length(),
                               m->hidden_size);
  }
}

#define LAUNCH_ATTENTION_SCORE_KERNEL(                                         \
    DT, Dh, Dh_MAX, THDS_PER_KEY, THREADS_PER_VALUE, THDS_PER_BLOCK, stream)   \
  smem_sz = smem_size_in_bytes<DT>(m->qProjSize,                               \
                                   BatchConfig::max_sequence_length(),         \
                                   THREADS_PER_VALUE,                          \
                                   THDS_PER_BLOCK);                            \
  compute_attention_kernel_generation_kernel<DT,                               \
                                             THDS_PER_BLOCK,                   \
                                             Dh,                               \
                                             Dh_MAX,                           \
                                             THDS_PER_KEY,                     \
                                             THREADS_PER_VALUE>                \
      <<<grid, THDS_PER_BLOCK, smem_sz, stream>>>(                             \
          static_cast<DT *>(m->devQKVProjArray),                               \
          static_cast<DT *>(m->keyCache),                                      \
          static_cast<DT *>(m->valueCache),                                    \
          output_ptr,                                                          \
          scale,                                                               \
          BatchConfig::max_sequence_length(),                                  \
          m->qProjSize,                                                        \
          m->hidden_size,                                                      \
          m->request_infos)

template <typename DT>
void compute_attention_kernel_generation(IncMultiHeadSelfAttentionMeta const *m,
                                         BatchConfig const *bc,
                                         DT *output_ptr,
                                         hipStream_t stream) {
  dim3 grid(m->num_q_heads, bc->num_generation_tokens);
  int const per_head_size = m->qProjSize;
  float scale = (*m->qk_prod_scaling) ? 1.0f / sqrt(m->kProjSize) : 1.0f;
  size_t smem_sz;
  if (per_head_size == 64) {
    constexpr int THREADS_PER_VALUE_64 = threads_per_value_t<DT, 64>::value;
    LAUNCH_ATTENTION_SCORE_KERNEL(
        DT, 64, 64, 4, THREADS_PER_VALUE_64, 128, stream);
  } else if (per_head_size == 128) {
    constexpr int THREADS_PER_VALUE_128 = threads_per_value_t<DT, 128>::value;
    LAUNCH_ATTENTION_SCORE_KERNEL(
        DT, 128, 128, 4, THREADS_PER_VALUE_128, 128, stream);
  } else {
    assert(false && "a unsupported head size");
  }
}

std::string get_fwd_dbg_folder(IncMultiHeadSelfAttentionMeta const *m,
                               int shard_id) {
  std::string op_name_without_uid =
      IncMultiHeadSelfAttention::get_op_name_without_uid(m);
  fs::path dst_filepath = get_dst_folder("fwd", m->decoding_step, shard_id);
  if (m->layer_guid.model_id > 0) {
    assert(false && "Model ID > 0 not supported yet");
  }
  std::string layername = "layers." +
                          std::to_string(m->layer_guid.transformer_layer_id) +
                          "." + op_name_without_uid;
  dst_filepath /= layername;
  return dst_filepath.string();
}

template <typename DT>
void inference_kernel(IncMultiHeadSelfAttentionMeta *m,
                      BatchConfig const *bc,
                      int shard_id,
                      DT const *qkv_ptr,
                      DT *output_ptr,
                      hipStream_t stream) {

  // phase 0: copy calculated qkv into devQKVProjArray
  // [qProjSize, num_heads, 3, num_new_tokens]
  size_t qkv_proj_size =
      m->qProjSize * m->num_q_heads * QKV_WEIGHT_NUM * bc->num_active_tokens();

  hipMemcpyAsync(m->devQKVProjArray,
                  qkv_ptr,
                  qkv_proj_size * sizeof(DT),
                  hipMemcpyDeviceToDevice,
                  stream);

  // phase 1: Implement kernel to apply rotary embedding and scaling
  compute_qkv_kernel(
      m, bc, shard_id, static_cast<DT *>(m->devQKVProjArray), stream);
  update_kv_cache_kernel<DT>(m, bc, stream);

  if (bc->num_generation_tokens > 0) {
    // phase 3: Compute attention score for generation tokens
    compute_attention_kernel_generation<DT>(
        m, bc, static_cast<DT *>(m->attn_heads), stream);
  }

  if (bc->num_tokens > bc->num_generation_tokens) {
    // phase 4: Compute attention score for prompt tokens;
    compute_attention_kernel_prompt<DT>(m, bc, shard_id, stream);
  }

  int num_tokens = bc->num_active_tokens();
  hipMemcpyAsync(output_ptr,
                  m->attn_heads,
                  m->oProjSize * num_tokens * sizeof(DT),
                  hipMemcpyDeviceToDevice,
                  stream);
}

std::string get_peft_dbg_folder(IncMultiHeadSelfAttentionMeta const *m,
                                int shard_id) {
  std::string op_name_without_uid =
      IncMultiHeadSelfAttention::get_op_name_without_uid(m);
  fs::path dst_filepath = get_dst_folder("bwd", m->bwd_step, shard_id);
  if (m->layer_guid.model_id > 0) {
    assert(false && "Model ID > 0 not supported yet");
  }
  std::string layername = "layers." +
                          std::to_string(m->layer_guid.transformer_layer_id) +
                          "." + op_name_without_uid;
  dst_filepath /= layername;
  return dst_filepath.string();
}

__global__ void transposeAdd_half_kernel(
    half *out, half const *in, int width, int height, half alpha, half beta) {
  int t_id = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;
  for (int i = t_id; i < width * height; i += num_threads) {
    int row = i / width;
    int col = i % width;
    out[col * height + row] =
        alpha * in[row * width + col] + beta * out[col * height + row];
  }
}

__global__ void transposeAdd_float_kernel(float *out,
                                          float const *in,
                                          int width,
                                          int height,
                                          float alpha,
                                          float beta) {
  int t_id = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;
  for (int i = t_id; i < width * height; i += num_threads) {
    int row = i / width;
    int col = i % width;
    out[col * height + row] =
        alpha * in[row * width + col] + beta * out[col * height + row];
  }
}

template <typename DT>
void transposeAdd(DT *out,
                  const DT *in,
                  int width,
                  int height,
                  float alpha,
                  float beta,
                  hipStream_t stream) {
  assert(false && "Unsupported data type");
}

template <>
void transposeAdd<float>(float *out,
                         float const *in,
                         int width,
                         int height,
                         float alpha,
                         float beta,
                         hipStream_t stream) {
  transposeAdd_float_kernel<<<4, 1024, 0, stream>>>(
      out, in, width, height, alpha, beta);
}

template <>
void transposeAdd<half>(half *out,
                        half const *in,
                        int width,
                        int height,
                        float alpha,
                        float beta,
                        hipStream_t stream) {
  transposeAdd_half_kernel<<<4, 1024, 0, stream>>>(
      out, in, width, height, __float2half(alpha), __float2half(beta));
}

template <typename DT>
void peft_bwd_kernel(IncMultiHeadSelfAttentionMeta const *m,
                     BatchConfig const *bc,
                     int shard_id,
                     DT *input_grad_ptr,
                     DT const *output_grad_ptr,
                     hipStream_t stream) {
  assert(!m->offload);
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  hipDataType cublas_data_type = ff_to_cuda_datatype(m->output_type[0]);
  hipdnnDataType_t cudnn_data_type = ff_to_cudnn_datatype(m->output_type[0]);
  assert(data_type_size(m->output_type[0]) == sizeof(DT));
  hipDataType compute_type = cublas_data_type;

  for (int i = 0; i < bc->max_requests_per_batch(); i++) {
    if (bc->request_completed[i]) {
      continue;
    }
    if (!bc->requestsInfo[i].peft_bwd) {
      continue;
    }
    int num_tokens = bc->requestsInfo[i].num_tokens_in_batch;
    int num_total_tokens = bc->requestsInfo[i].first_token_depth_in_request +
                           bc->requestsInfo[i].num_tokens_in_batch;
    // Currently assume we are calculating gradients for all tokens
    // of a request
    assert(num_tokens == num_total_tokens);
    int kt_block_size = m->kProjSize;
    int kt_req_block_size =
        kt_block_size * m->num_q_heads * BatchConfig::max_sequence_length();
    int vt_block_size = m->vProjSize;
    int vt_req_block_size =
        vt_block_size * m->num_q_heads * BatchConfig::max_sequence_length();
    assert(m->qProjSize == m->kProjSize && m->kProjSize == m->vProjSize);
    // Step 1: copy gradient before final projection into workspace
    {
      int m_ = m->vProjSize * m->num_q_heads;
      int n_ = num_tokens;
      DT *C = static_cast<DT *>(m->handle.workSpace);
      hipMemcpyAsync(C,
                      output_grad_ptr +
                          bc->requestsInfo[i].first_token_offset_in_batch *
                              m->oProjSize,
                      m_ * n_ * sizeof(DT),
                      hipMemcpyDeviceToDevice,
                      stream);
      if (m->inference_debugging) {
        // save result to file for checking
        std::string filename =
            get_peft_dbg_folder(m, shard_id) + ".o_proj.input_gradient_0";
        save_tensor(C, m_ * n_, filename.c_str());
      }
    }
    // Step 2: compute gradients w.r.t. value
    {
      float alpha = 1.0f, beta = 0.0f;
      // matrix A: qk_prods_softmax
      // matrix A's layout: [num_new_tokens, total_tokens, num_heads]
      DT const *A = static_cast<DT *>(m->qk_prods_softmax);
      // matrix B: attn_heads gradients
      // matrix B's layout: [vProjSize * num_heads, num_new_tokens]
      DT const *B = static_cast<DT *>(m->handle.workSpace);
      // matrix C: gradients for value (saved as part of m->devQKVProjArray)
      // matrix C's layout: [num_tokens, qProjsize * num_heads, 3]
      DT *C = static_cast<DT *>(m->devQKVProjArray) +
              2 * num_tokens *
                  (m->qProjSize * m->num_q_heads); // skip over regions reserved
                                                   // for Q and K gradients
      // after transpositions
      int m_ = num_tokens;   // total_tokens
      int n_ = m->vProjSize; // num_new_tokens
      int k_ = num_tokens;   // num_new_tokens
      // before transpositions
      int lda = num_tokens; // num_new_tokens
      int ldb = m->vProjSize * m->num_q_heads;
      int ldc = num_tokens; // total_tokens
      // N.B. strides are applied before transpose operations
      int strideA = num_tokens * num_tokens; // num_new_tokens * total_tokens
      int strideB = m->vProjSize;
      int strideC = num_tokens * m->vProjSize;
      checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                           HIPBLAS_OP_T,
                                           HIPBLAS_OP_T,
                                           m_,
                                           n_,
                                           k_,
                                           &alpha,
                                           A,
                                           cublas_data_type,
                                           lda,
                                           strideA,
                                           B,
                                           cublas_data_type,
                                           ldb,
                                           strideB,
                                           &beta,
                                           C,
                                           cublas_data_type,
                                           ldc,
                                           strideC,
                                           m->num_q_heads,
                                           compute_type,
                                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
      // save result to file for checking
      if (m->inference_debugging) {
        std::string filename =
            get_peft_dbg_folder(m, shard_id) + ".v_proj.input_gradient_0";
        save_tensor(C, m_ * n_ * m->num_q_heads, filename.c_str());
        std::string filename2 =
            get_peft_dbg_folder(m, shard_id) + ".qk_prods.softmax";
        save_tensor(A, m_ * k_ * m->num_q_heads, filename2.c_str());
      }
    }
    // Step 3: compute gradients w.r.t. the qk_prods_softmax tensor
    {
      float alpha = 1.0f, beta = 0.0f;
      // matrix A: attn_heads gradients
      // matrix A's layout: [vProjSize * num_heads, num_new_tokens]
      DT const *A = static_cast<DT *>(m->handle.workSpace);
      // matrix B: value cache
      // matrix B's layout: [vProjSize * num_heads, max_num_tokens, num_req]
      DT const *B = static_cast<DT *>(m->valueCache) + i * vt_req_block_size;
      // matrix C: qk_prods_softmax gradients
      // matrix C's layout: [num_new_tokens, total_tokens, num_heads]
      DT *C = static_cast<DT *>(m->qk_prods_softmax);
      // after transposition & striding
      int m_ = num_tokens; // num_new_tokens
      int n_ = num_tokens;
      int k_ = m->vProjSize;
      // before transposition and striding
      int lda = m->vProjSize * m->num_q_heads;
      int ldb = m->vProjSize * m->num_q_heads;
      int ldc = num_tokens; // num_new_tokens
      int strideA = m->vProjSize;
      int strideB = m->vProjSize;
      int strideC = num_tokens * num_tokens; // num_new_tokens * total_tokens

      checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                           HIPBLAS_OP_T,
                                           HIPBLAS_OP_N,
                                           m_,
                                           n_,
                                           k_,
                                           &alpha,
                                           A,
                                           cublas_data_type,
                                           lda,
                                           strideA,
                                           B,
                                           cublas_data_type,
                                           ldb,
                                           strideB,
                                           &beta,
                                           C,
                                           cublas_data_type,
                                           ldc,
                                           strideC,
                                           m->num_q_heads,
                                           compute_type,
                                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
      if (m->inference_debugging) {
        std::string filename =
            get_peft_dbg_folder(m, shard_id) + ".qk_prods.softmax_grad";
        save_tensor(
            C, num_tokens * num_tokens * m->num_q_heads, filename.c_str());
        std::string filename2 = get_peft_dbg_folder(m, shard_id) + ".vcache";
        save_tensor(
            B, m->vProjSize * m->num_q_heads * num_tokens, filename2.c_str());
      }
    }
    // Step 4: softmax backpropagation
    {
      float alpha = 1.0f, beta = 0.0f;
      int n_param = m->num_q_heads;
      int c_param = num_tokens;
      int h_param = 1;
      int w_param = num_tokens;
      checkCUDNN(hipdnnSetTensor4dDescriptor(m->qk_tensor,
                                            HIPDNN_TENSOR_NCHW,
                                            cudnn_data_type,
                                            n_param,
                                            c_param,
                                            h_param,
                                            w_param));
      checkCUDNN(hipdnnSoftmaxBackward(m->handle.dnn,
                                      HIPDNN_SOFTMAX_ACCURATE,
                                      HIPDNN_SOFTMAX_MODE_CHANNEL,
                                      &alpha,
                                      m->qk_tensor,
                                      m->softmax_activation_buffer,
                                      m->qk_tensor,
                                      m->qk_prods_softmax,
                                      &beta,
                                      m->qk_tensor,
                                      m->qk_prods));

      if (m->inference_debugging) {
        DT *C = static_cast<DT *>(m->qk_prods);
        std::string filename =
            get_peft_dbg_folder(m, shard_id) + ".qk_prods.softmax_grad_in";
        save_tensor(
            C, num_tokens * num_tokens * m->num_q_heads, filename.c_str());
      }

      //  TODO: fill all elements above diagonal to force causal attention
      size_t entries_above_diagonal = num_tokens * (num_tokens - 1) / 2;
      if (entries_above_diagonal > 0) {
        size_t parallelism = m->num_q_heads * entries_above_diagonal;
        fill_entries_above_diagonal<<<GET_BLOCKS(parallelism),
                                      min((size_t)CUDA_NUM_THREADS,
                                          parallelism),
                                      0,
                                      stream>>>(static_cast<DT *>(m->qk_prods),
                                                num_tokens,
                                                num_tokens,
                                                m->num_q_heads,
                                                entries_above_diagonal,
                                                DT(0.0f));
      }
      if (m->inference_debugging) {
        DT *C = static_cast<DT *>(m->qk_prods);
        std::string filename = get_peft_dbg_folder(m, shard_id) +
                               ".qk_prods.softmax_grad_in.masked";
        save_tensor(
            C, num_tokens * num_tokens * m->num_q_heads, filename.c_str());
      }
    }
    // Step 5: compute gradients w.r.t. key
    {
      float alpha = 1.0f, beta = 0.0f;
      if (*m->qk_prod_scaling) {
        alpha = 1.0f / sqrt(m->kProjSize);
      }
      // matrix A: gradients w.r.t. qk_prods
      // matrix A's layout: [num_new_tokens, num_tokens, num_heads]
      DT const *A = static_cast<DT *>(m->qk_prods);
      // matrix B: query activation (in query_activation_buffer)
      // matrix B's layout: [m->qProjSize * num_heads, num_new_tokens]
      DT const *B = static_cast<DT *>(m->query_activation_buffer);
      // matrix C: gradients for key (saved as part of m->devQKVProjArray)
      // matrix C's layout: [num_tokens, qProjsize * num_heads, 3]
      DT *C =
          static_cast<DT *>(m->devQKVProjArray) +
          num_tokens *
              (m->qProjSize *
               m->num_q_heads); // skip over regions reserved for Q gradients
      // after transposition & striding
      int m_ = num_tokens;
      int n_ = m->kProjSize;
      int k_ = num_tokens; // num_new_tokens
      // before transposition and striding
      int lda = num_tokens; // num_new_tokens
      int ldb = m->kProjSize * m->num_q_heads;
      int ldc = num_tokens;
      int strideA = num_tokens * num_tokens;
      int strideB = m->kProjSize;
      int strideC = num_tokens * m->kProjSize;
      checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                           HIPBLAS_OP_T,
                                           HIPBLAS_OP_T,
                                           m_,
                                           n_,
                                           k_,
                                           &alpha,
                                           A,
                                           cublas_data_type,
                                           lda,
                                           strideA,
                                           B,
                                           cublas_data_type,
                                           ldb,
                                           strideB,
                                           &beta,
                                           C,
                                           cublas_data_type,
                                           ldc,
                                           strideC,
                                           m->num_q_heads,
                                           compute_type,
                                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
      if (m->inference_debugging) {
        std::string filename =
            get_peft_dbg_folder(m, shard_id) + ".query_activation";
        save_tensor(
            B, m->qProjSize * m->num_q_heads * num_tokens, filename.c_str());
        std::string filename2 =
            get_peft_dbg_folder(m, shard_id) + ".devkproj_pre";
        save_tensor(
            C, num_tokens * (m->qProjSize * m->num_q_heads), filename2.c_str());
      }
    }
    // Step 6: compute gradients w.r.t query
    {
      float alpha = 1.0f, beta = 0.0f;
      if (*m->qk_prod_scaling) {
        alpha = 1.0f / sqrt(m->kProjSize);
      }
      // matrix A: gradients w.r.t. qk_prods
      // matrix A's layout: [num_new_tokens, num_tokens, num_heads]
      DT const *A = static_cast<DT *>(m->qk_prods);
      // matrix B: key cache
      // matrix B's layout: [vProjSize * num_heads, max_num_tokens, num_req]
      DT const *B = static_cast<DT *>(m->keyCache) + i * kt_req_block_size;
      // matrix C: gradients for query (saved as part of m->devQKVProjArray)
      // matrix C's layout: [num_tokens, qProjsize * num_heads, 3]
      DT *C = static_cast<DT *>(m->devQKVProjArray);
      // after transposition & striding
      int m_ = num_tokens; // num_new_tokens
      int n_ = m->qProjSize;
      int k_ = num_tokens;
      // before transposition and striding
      int lda = num_tokens; // num_new_tokens
      int ldb = m->qProjSize * m->num_q_heads;
      int ldc = num_tokens;
      int strideA = num_tokens * num_tokens;
      int strideB = m->qProjSize;
      int strideC = num_tokens * m->qProjSize;
      checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                           HIPBLAS_OP_N,
                                           HIPBLAS_OP_T,
                                           m_,
                                           n_,
                                           k_,
                                           &alpha,
                                           A,
                                           cublas_data_type,
                                           lda,
                                           strideA,
                                           B,
                                           cublas_data_type,
                                           ldb,
                                           strideB,
                                           &beta,
                                           C,
                                           cublas_data_type,
                                           ldc,
                                           strideC,
                                           m->num_q_heads,
                                           compute_type,
                                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
      if (m->inference_debugging) {
        std::string filename =
            get_peft_dbg_folder(m, shard_id) + ".devQKVPRojArray_pre";
        save_tensor(C,
                    num_tokens * m->qProjSize * m->num_q_heads * 3,
                    filename.c_str());
      }
    }

    // Step 7: perform rotary position embeddings (RoPE) bwd
    {
      if (m->rotary_embedding_meta->apply_rotary_embedding) {
        assert(m->hidden_size == m->qProjSize * m->num_q_heads);
        assert(m->qProjSize == m->kProjSize);
        /*q&k*/
        int parallelism = num_tokens * m->hidden_size;
        DT *A = static_cast<DT *>(m->devQKVProjArray);
        apply_rotary_embedding_bwd<<<GET_BLOCKS(parallelism),
                                     min(CUDA_NUM_THREADS, parallelism),
                                     0,
                                     stream>>>(
            A,
            m->complex_input,
            m->token_infos,
            m->rotary_embedding_meta->rope_theta,
            (m->rotary_embedding_meta->rope_type == "llama3"),
            m->rotary_embedding_meta->factor,
            m->rotary_embedding_meta->low_freq_factor,
            m->rotary_embedding_meta->high_freq_factor,
            m->rotary_embedding_meta->original_max_position_embeddings,
            m->qProjSize,
            num_tokens,
            m->hidden_size);
        DT *C = static_cast<DT *>(m->devQKVProjArray);
        if (m->inference_debugging) {
          std::string filename =
              get_peft_dbg_folder(m, shard_id) + ".devQKVPRojArray";
          save_tensor(C,
                      num_tokens * m->qProjSize * m->num_q_heads * 3,
                      filename.c_str());
        }
      }

      // matrix C: gradients for key (saved as part of m->devQKVProjArray)
      // matrix C's layout: [num_tokens, qProjsize * num_heads, 3]
      DT *C =
          static_cast<DT *>(m->devQKVProjArray) +
          num_tokens *
              (m->qProjSize *
               m->num_q_heads); // skip over regions reserved for Q gradients
      if (m->inference_debugging) {
        std::string filename = get_peft_dbg_folder(m, shard_id) + ".devkproj";
        save_tensor(
            C, num_tokens * (m->qProjSize * m->num_q_heads), filename.c_str());
      }
    }

    // Step 8: compute gradients w.r.t. input
    {
      float alpha = 1.0f, beta = 0.0f;
      if (!m->reset_input_grads[0]) {
        beta = 1.0f;
      }
      // matrix B: gradients w.r.t. QKV (concatenated in devQKVArray)
      // matrix B's layout: [num_tokens, qProjsize * num_heads, 3]
      DT const *B = static_cast<DT *>(m->devQKVProjArray);
      // matrix C: gradients w.r.t. input
      // matrix C's layout: [m->qSize, num_tokens]
      DT *C = input_grad_ptr +
              bc->requestsInfo[i].first_token_offset_in_batch * m->qSize;
      // int m_ = m->qSize;
      int n_ = num_tokens;
      int k_ = m->num_q_heads * (m->qProjSize + m->kProjSize + m->vProjSize);

      // The original version uses existing result and attention's projection to
      // do further calculation in a way different than the usual dense layer,
      // they are off by a transpose. So an explicit transpose is needed here.
      // The add here is just for gradient accumulation.
      transposeAdd(C, B, n_, k_, alpha, beta, stream);

      if (m->inference_debugging) {
        std::string filename =
            get_peft_dbg_folder(m, shard_id) + ".self_attn.input_gradient_0";
        save_tensor(C, num_tokens * m->qSize, filename.c_str());
      }
    }
  }
}

} // namespace IncMultiHeadAttention
} // namespace Kernels

using namespace Kernels::IncMultiHeadAttention;

/*static*/
void IncMultiHeadSelfAttention::inference_kernel_wrapper(
    IncMultiHeadSelfAttentionMeta *m,
    BatchConfig const *bc,
    int shard_id,
    GenericTensorAccessorR const &input,
    GenericTensorAccessorW const &output) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  assert(input.data_type == output.data_type);

  if (input.data_type == DT_HALF) {
    Kernels::IncMultiHeadAttention::inference_kernel(
        m, bc, shard_id, input.get_half_ptr(), output.get_half_ptr(), stream);
  } else if (input.data_type == DT_FLOAT) {
    Kernels::IncMultiHeadAttention::inference_kernel(
        m, bc, shard_id, input.get_float_ptr(), output.get_float_ptr(), stream);
  } else {
    assert(false && "Unspported data type");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("IncMultiHeadSelfAttention forward time = %.9fms\n", elapsed);
  }
}

/*static*/
void IncMultiHeadSelfAttention::peft_bwd_kernel_wrapper(
    IncMultiHeadSelfAttentionMeta *m,
    BatchConfig const *bc,
    int shard_id,
    GenericTensorAccessorW const &input_grad,
    GenericTensorAccessorR const &output_grad) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  assert(input_grad.data_type == output_grad.data_type);

  if (input_grad.data_type == DT_HALF) {
    assert(!m->offload);
    Kernels::IncMultiHeadAttention::peft_bwd_kernel(m,
                                                    bc,
                                                    shard_id,
                                                    input_grad.get_half_ptr(),
                                                    output_grad.get_half_ptr(),
                                                    stream);
  } else if (input_grad.data_type == DT_FLOAT) {
    assert(!m->offload);
    Kernels::IncMultiHeadAttention::peft_bwd_kernel(m,
                                                    bc,
                                                    shard_id,
                                                    input_grad.get_float_ptr(),
                                                    output_grad.get_float_ptr(),
                                                    stream);
  } else {
    assert(false && "Unspported data type");
  }
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("IncMultiHeadSelfAttention PEFT backward time = %.9fms\n", elapsed);
  }
}

IncMultiHeadSelfAttentionMeta::IncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    IncMultiHeadSelfAttention const *attn,
    MemoryAllocator &gpu_mem_allocator,
    int num_samples,
    int _num_q_heads,
    int _num_kv_heads)
    : IncMultiHeadSelfAttentionMeta(handler,
                                    INC_DECODING_MODE,
                                    attn,
                                    attn->qSize,
                                    attn->kSize,
                                    attn->vSize,
                                    attn->qProjSize,
                                    attn->kProjSize,
                                    attn->vProjSize,
                                    attn->oProjSize,
                                    attn->rotary_embedding_meta,
                                    attn->scaling_query,
                                    attn->qk_prod_scaling,
                                    attn->position_bias,
                                    attn->scaling_factor,
                                    gpu_mem_allocator,
                                    num_samples,
                                    attn->num_q_heads,
                                    attn->num_kv_heads,
                                    _num_q_heads,
                                    _num_kv_heads,
                                    attn->quantization_type,
                                    attn->offload) {}

IncMultiHeadSelfAttentionMeta::IncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    InferenceMode infer_mode,
    Op const *attn,
    int _qSize,
    int _kSize,
    int _vSize,
    int _qProjSize,
    int _kProjSize,
    int _vProjSize,
    int _oProjSize,
    RotaryEmbeddingMeta _rotary_embedding_meta,
    bool _scaling_query,
    bool _qk_prod_scaling,
    bool _position_bias,
    float _scaling_factor,
    MemoryAllocator &gpu_mem_allocator,
    int num_samples,
    int _global_num_q_heads,
    int _global_num_kv_heads,
    int _num_q_heads,
    int _num_kv_heads,
    DataType _quantization_type,
    bool _offload)
    : OpMeta(handler, attn) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  checkCUDNN(hipdnnSetStream(handler.dnn, stream));
  checkCUDNN(hipdnnCreateTensorDescriptor(&qk_tensor));
  qSize = _qSize;
  kSize = _kSize;
  vSize = _vSize;
  // assume dimensions match for now
  assert(qSize == kSize);
  assert(kSize == vSize);
  qProjSize = _qProjSize;
  kProjSize = _kProjSize;
  assert(qProjSize == kProjSize); // required for attention QK.T matmul
  vProjSize = _vProjSize;
  oProjSize = _oProjSize;
  size_t size_of_dt = data_type_size(attn->data_type);
  quantization_type = _quantization_type;
  offload = _offload;

  global_num_q_heads = _global_num_q_heads;
  global_num_kv_heads = _global_num_kv_heads;
  num_q_heads = _num_q_heads;
  num_kv_heads = _num_kv_heads;
  hidden_size = num_q_heads * qProjSize;

  rotary_embedding_meta =
      (RotaryEmbeddingMeta *)calloc(1, sizeof(RotaryEmbeddingMeta));
  *rotary_embedding_meta = _rotary_embedding_meta;
  scaling_query = (bool *)calloc(1, sizeof(bool));
  *scaling_query = _scaling_query;
  scaling_factor = _scaling_factor;
  qk_prod_scaling = (bool *)calloc(1, sizeof(bool));
  *qk_prod_scaling = _qk_prod_scaling;
  position_bias = (bool *)calloc(1, sizeof(bool));
  *position_bias = _position_bias;

  // allocate memory for the seqArray and reserve space
  {
    int max_tokens_per_batch = infer_mode == TREE_VERIFY_MODE
                                   ? BatchConfig::max_verify_tokens_per_batch()
                                   : BatchConfig::max_tokens_per_batch();
    size_t qkv_max_proj_size = max_tokens_per_batch * (qProjSize * num_q_heads +
                                                       kProjSize * num_q_heads +
                                                       vProjSize * num_q_heads);
    size_t key_cache_size = 0, value_cache_size = 0;
    switch (infer_mode) {
      case INC_DECODING_MODE: {
        key_cache_size = num_q_heads * kProjSize *
                         BatchConfig::max_requests_per_batch() *
                         BatchConfig::max_sequence_length();
        value_cache_size = num_q_heads * vProjSize *
                           BatchConfig::max_requests_per_batch() *
                           BatchConfig::max_sequence_length();
        break;
      }
      case BEAM_SEARCH_MODE:
      case TREE_VERIFY_MODE: {
        // a K-ary tree max node is (k^n - 1) / 2
        key_cache_size = num_q_heads * kProjSize *
                         BeamSearchBatchConfig::max_requests_per_batch() *
                         (BatchConfig::max_sequence_length() +
                          BatchConfig::max_spec_tree_token_num());
        value_cache_size = num_q_heads * vProjSize *
                           BeamSearchBatchConfig::max_requests_per_batch() *
                           (BatchConfig::max_sequence_length() +
                            BatchConfig::max_spec_tree_token_num());
        break;
      }
      default:
        assert(false && "Unkown inference mode");
    }
    size_t requestinfo_size = BatchConfig::max_requests_per_batch();
    // size_t tokeninfo_size = max_tokens_per_batch;
    size_t qk_prod_size =
        max_tokens_per_batch * BatchConfig::max_sequence_length() * num_q_heads;
    size_t attn_heads_size = max_tokens_per_batch * num_q_heads * vProjSize;
    size_t complex_size = (max_tokens_per_batch * (qProjSize * num_q_heads +
                                                   kProjSize * num_q_heads)) /
                          2;
    size_t totalSize =
        (qkv_max_proj_size + key_cache_size + value_cache_size +
         2 * qk_prod_size + attn_heads_size) *
            size_of_dt +
        complex_size * sizeof(hipFloatComplex); // more components will
                                               // be added here later
    if (offload) {
      // assert that we have enough reserved work space left
      size_t totalSharedSize =
          infer_mode == TREE_VERIFY_MODE
              ? totalSize -
                    (key_cache_size + value_cache_size + qkv_max_proj_size) *
                        size_of_dt
              : totalSize - (key_cache_size + value_cache_size) * size_of_dt;

      size_t instance_size =
          size_of_dt *
          (infer_mode == TREE_VERIFY_MODE
               ? key_cache_size + value_cache_size + qkv_max_proj_size
               : key_cache_size + value_cache_size);

      assert(gpu_mem_allocator.reserved_total_size -
                 gpu_mem_allocator.reserved_allocated_size >=
             totalSharedSize);
      gpu_mem_allocator.create_legion_instance(reserveInst, instance_size);
    } else {
      gpu_mem_allocator.create_legion_instance(reserveInst, totalSize);
    }

    // in tree_verify, enable devQKVProjArray;
    if (!offload || infer_mode == TREE_VERIFY_MODE) {
      devQKVProjArray = gpu_mem_allocator.allocate_instance_untyped(
          qkv_max_proj_size * size_of_dt);
    } else {
      devQKVProjArray = gpu_mem_allocator.allocate_reserved_untyped(
          qkv_max_proj_size * size_of_dt);
      // offset += qkv_max_proj_size * size_of_dt;
    }

    // use key value cache in all mode.
    keyCache = gpu_mem_allocator.allocate_instance_untyped(key_cache_size *
                                                           size_of_dt);
    valueCache = gpu_mem_allocator.allocate_instance_untyped(value_cache_size *
                                                             size_of_dt);

    token_infos = static_cast<BatchConfig::PerTokenInfo *>(
        handler.batch_config_metadata->tokens_info);
    request_infos = static_cast<BatchConfig::PerRequestInfo *>(
        handler.batch_config_metadata->requestsInfo);

    if (offload) {
      qk_prods = gpu_mem_allocator.allocate_reserved_untyped(qk_prod_size *
                                                             size_of_dt);
      qk_prods_softmax = gpu_mem_allocator.allocate_reserved_untyped(
          qk_prod_size * size_of_dt);
      attn_heads = gpu_mem_allocator.allocate_reserved_untyped(attn_heads_size *
                                                               size_of_dt);
      complex_input =
          gpu_mem_allocator.allocate_reserved<hipFloatComplex>(complex_size);
    } else {
      qk_prods = gpu_mem_allocator.allocate_instance_untyped(qk_prod_size *
                                                             size_of_dt);
      qk_prods_softmax = gpu_mem_allocator.allocate_instance_untyped(
          qk_prod_size * size_of_dt);
      attn_heads = gpu_mem_allocator.allocate_instance_untyped(attn_heads_size *
                                                               size_of_dt);
      complex_input =
          gpu_mem_allocator.allocate_instance<hipFloatComplex>(complex_size);
    }

    // allocate more size for quantization data
    if (quantization_type != DT_NONE) {
      assert(offload);
    }
    if (!offload) {
      assert(gpu_mem_allocator.reserved_total_size ==
             gpu_mem_allocator.reserved_allocated_size);
    }
  }
  allocated_peft_buffer_size1 = 0;
  allocated_peft_buffer_size2 = 0;
  hipStreamSynchronize(stream);
}

IncMultiHeadSelfAttentionMeta::~IncMultiHeadSelfAttentionMeta(void) {
  if (reserveInst != Realm::RegionInstance::NO_INST) {
    reserveInst.destroy();
  }
}

template void
    Kernels::IncMultiHeadAttention::compute_attention_kernel_generation<float>(
        IncMultiHeadSelfAttentionMeta const *m,
        BatchConfig const *bc,
        float *output_ptr,
        hipStream_t stream);

template void
    Kernels::IncMultiHeadAttention::compute_attention_kernel_generation<half>(
        IncMultiHeadSelfAttentionMeta const *m,
        BatchConfig const *bc,
        half *output_ptr,
        hipStream_t stream);

template void Kernels::IncMultiHeadAttention::compute_qkv_kernel<float>(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    int shard_id,
    float *output_ptr,
    hipStream_t stream);

template void Kernels::IncMultiHeadAttention::compute_qkv_kernel<half>(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    int shard_id,
    half *output_ptr,
    hipStream_t stream);

}; // namespace FlexFlow
