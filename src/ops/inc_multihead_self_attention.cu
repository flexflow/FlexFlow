#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#if defined(FF_USE_CUDA) || defined(FF_USE_HIP_CUDA)
#include "hip/hip_complex.h"
#endif
#include "flexflow/ffconst_utils.h"
#include "flexflow/ops/inc_multihead_self_attention.h"
#include "flexflow/ops/kernels/decompress_kernels.h"
#include "flexflow/ops/kernels/inc_multihead_self_attention_kernels.h"
#include "flexflow/ops/kernels/inc_multihead_self_attention_utils.cuh"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

// declare Legion names
using Legion::coord_t;
using Legion::Memory;

#define WARP_SIZE 32

namespace Kernels {
namespace IncMultiHeadAttention {

// gridDim = num_heads
// blockDim = num_tokens/num_request * head_size
// QKV tensor layout: |QKV| * num_new_tokens. |Q=K=V=head_size * num_heads|
// one thread process one head_size
template <typename DT,
          int THREADS_PER_BLOCK,
          int Dh,
          int Dh_MAX,
          int THREADS_PER_KEY,
          int THREADS_PER_VALUE>
__global__ void compute_attention_kernel_generation_kernel(
    DT const *query,
    DT const *key_cache,
    DT const *value_cache,
    DT *output_ptr,
    float const scale,
    int max_seq_length,
    int per_head_size,
    int hidden_size,
    BatchConfig::PerRequestInfo *request_infos,
    bool is_beam,
    int max_beam_width) {

  // q, k
  using Q_vec = typename VEC_K<DT, THREADS_PER_KEY>::Type;
  using K_vec = typename VEC_K<DT, THREADS_PER_KEY>::Type;
  using V_vec = typename VEC_V<DT>::Type;
  using Out_sum = typename Vec_fp32_<V_vec>::Type;

  constexpr int WARPS_PER_BLOCK = THREADS_PER_BLOCK / WARP_SIZE;

  // eg.  if head_size = 128, thread_per_key = 4, with float32 precision
  // then K_VEC_SIZE = 1,  QK_VEC_SIZE = 4
  //  K_ELTS_PER_THREAD = 128 / 4 = 32
  //  K_VECS_PER_THREAD = 32 / 1 = 32
  constexpr int K_VEC_SIZE = sizeof(K_vec) / sizeof(DT);
  // constexpr int QK_VEC_SIZE = 16 / sizeof(DT);
  // // constexpr int QK_VEC_SIZE = sizeof(Qk_vec_k) / sizeof(DT);
  constexpr int K_ELTS_PER_THREAD = Dh / THREADS_PER_KEY;
  constexpr int K_VECS_PER_THREAD = K_ELTS_PER_THREAD / K_VEC_SIZE;
  // constexpr int QK_ELTS_IN_16B = 16 / sizeof(DT);

  // thread id
  int const tidx = threadIdx.x;
  // head id
  int const head_idx = blockIdx.x;
  // request idx
  int const request_idx = blockIdx.y;

  int const beam_request_idx =
      is_beam ? request_idx / max_beam_width : request_idx;
  int const beam_sub_request_idx = is_beam ? request_idx % max_beam_width : 0;

  int const first_step = 0;

  int const tlength =
      request_infos[beam_request_idx].first_token_depth_in_request +
      request_infos[beam_request_idx].num_tokens_in_batch;

  // shared memory objects
  extern __shared__ char smem_[];

  float *qk_smem = reinterpret_cast<float *>(smem_);
  float *out_smem = reinterpret_cast<float *>(smem_);

  float qk_max = -FLT_MAX;

  // first WARPS_PER_BLOCK for store qk_max, second WARPS_PER_BLOCK for sum
  __shared__ float red_smem[WARPS_PER_BLOCK * 2];

  const DT *q_ptr = query + beam_request_idx * hidden_size * QKV_WEIGHT_NUM +
                    head_idx * per_head_size;
  __shared__ Q_vec q_vecs[THREADS_PER_KEY][K_VECS_PER_THREAD];
  // DT const *q_ptr =
  //     query + request_idx * Dh * QKV_WEIGHT_NUM + head_idx * per_head_size;

  // q tensor in this thread
  // if THREADS_PER_KEY is 4, first thread load 0, 4, 8, 12..., total
  // K_VECS_PER_THREAD elements
  // QK_vec_k: 32->1, 64->2, 128->4... head_size
  // K_vec_k: 4->1, 2->2, 1->4 threads_per_key

  // the start offset of the element eg. (0, 1, 2, 3) * K_VEC_SIZE
  int ki = tidx % THREADS_PER_KEY * K_VEC_SIZE;
  int ki_o = tidx % THREADS_PER_KEY;
  // the first key's offset for this thread
  // ko = 0, 0, 0, 0, 1, 1, 1, 1, ....
  int ko = tidx / THREADS_PER_KEY;
  // load q tensor
  Q_vec q_vec[K_VECS_PER_THREAD];
#pragma unroll
  for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
    q_vecs[ki_o][ii] = *reinterpret_cast<Q_vec const *>(
        q_ptr + ki + ii * THREADS_PER_KEY * K_VEC_SIZE);
  }
  __syncthreads();
  // first iter = 128 / 4 = 32
  // K_VECS_PER_THREAD = 32
  //  K_PER_ITER how many keys in this loop
  //  The number of timesteps loaded per iteration.
  constexpr int K_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_KEY;
  //   // The number of keys per warp.
  constexpr int K_PER_WARP = WARP_SIZE / THREADS_PER_KEY;

  DT const *k_cache_batch =
      key_cache +
      (beam_request_idx * max_beam_width + beam_sub_request_idx) *
          max_seq_length * hidden_size +
      ki;

  int ti_end =
      div_up(tlength - first_step, K_PER_WARP) * K_PER_WARP + first_step;
  // get k, perform qk proj

  for (int ti = ko; ti < ti_end; ti += K_PER_ITER) {
    K_vec k[K_VECS_PER_THREAD];
    int const ti_circ = ti % max_seq_length;
#pragma unroll
    for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
      int jj = ii * THREADS_PER_KEY * K_VEC_SIZE;
      if (ti < tlength) {
        k[ii] = *reinterpret_cast<K_vec const *>(k_cache_batch +
                                                 ti_circ * hidden_size +
                                                 head_idx * per_head_size + jj);
      }
      // Compute dot product.
      // This includes a reduction across the threads in the same thread group.
    }
    float qk = scale * Qk_dot<DT, THREADS_PER_KEY>::dot(q_vecs[ki_o], k);
    // // todo add positional embedding to the qk production
    // // Store the product to shared memory. There's one qk value per
    // timestep.
    // // Update the max.
    if (ti < tlength && tidx % THREADS_PER_KEY == 0) {
      // todo add alobi here
      bool const mask = ti_circ >= tlength;
      if (mask) {
        assert(false);
      }
      qk_max = mask ? qk_max : fmaxf(qk_max, qk);
      qk_smem[ti - first_step] = mask ? 0.f : qk;
    }
  }

  __syncthreads();

#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= THREADS_PER_KEY; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }

  // Decompose the thread index into warp and lane.
  int const warp = tidx / WARP_SIZE;
  int const lane = tidx % WARP_SIZE;

  // The warp leader writes the max to shared memory.
  if (lane == 0) {
    red_smem[warp] = qk_max;
  }

  // Make sure the products are in shared memory.
  __syncthreads();

  // The warps finalize the reduction.
  qk_max = lane < WARPS_PER_BLOCK ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = WARPS_PER_BLOCK / 2; mask >= 1; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }

  // Broadcast to all the threads in the warp.
  qk_max = __shfl_sync(uint32_t(-1), qk_max, 0);

  float exp_sum = 0.f;
  for (int ti = first_step + tidx; ti < tlength; ti += THREADS_PER_BLOCK) {
    float logit = __expf(qk_smem[ti - first_step] - qk_max);
    exp_sum += logit;
    qk_smem[ti - first_step] = logit;
  }

  // Compute the sum.
  exp_sum = block_sum<WARPS_PER_BLOCK>(&red_smem[WARPS_PER_BLOCK], exp_sum);

  // softmax
  float inv_sum = __fdividef(1.f, exp_sum + 1.e-6);
  for (int ti = first_step + tidx; ti < tlength; ti += THREADS_PER_BLOCK) {
    qk_smem[ti - first_step] *= inv_sum;
  }

  __syncthreads();
  // if (blockIdx.y == 0 && blockIdx.x == 0 && tidx == 0) {
  //   printf("softmax %.10f\n", qk_smem[0]);
  // }

  // value projection
  constexpr int V_VEC_SIZE = 16 / sizeof(DT);
  // A vector of V elements for the current timestep.
  // using V_vec_k = typename V_vec_k_<DT, V_VEC_SIZE>::Type;
  // using V_vec_acum = typename V_vec_acum_fp32_<V_vec_k>::Type;

  // The value computed by this thread.
  int vo = tidx / THREADS_PER_VALUE;
  // The hidden dimensions computed by this particular thread.
  int vi = tidx % THREADS_PER_VALUE * V_VEC_SIZE;
  constexpr int V_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_VALUE;

  Out_sum out;
  zero(out);

  // The base pointer for the value in the cache buffer.
  DT const *v_cache_batch =
      value_cache +
      (beam_request_idx * max_beam_width + beam_sub_request_idx) *
          max_seq_length * hidden_size +
      vi;

  if (Dh == Dh_MAX || vi < Dh) {
    for (int ti = first_step + vo; ti < tlength; ti += V_PER_ITER) {
      // Load the values from the cache.
      int const ti_circ = ti % max_seq_length;

      V_vec v = *reinterpret_cast<V_vec const *>(
          v_cache_batch + ti_circ * hidden_size + head_idx * per_head_size);
      float logit = qk_smem[ti - first_step];
      out = FlexFlow::fma(logit, cast_to_float(v), out);
    }
  }

  //   // Make sure we can start writing to shared memory.
  __syncthreads();

  // Run the final reduction amongst the different groups computing different
  // partial outputs.
  if (Dh == Dh_MAX || vi < Dh) {
#pragma unroll
    for (int active_groups = V_PER_ITER; active_groups >= 2;
         active_groups /= 2) {

      // The midpoint in the number of active groups.
      int midpoint = active_groups / 2;

      // The upper part of active threads store to shared memory.
      if (vo >= midpoint && vo < active_groups && (Dh == Dh_MAX || vi < Dh)) {
        *reinterpret_cast<Out_sum *>(out_smem + (vo - midpoint) * Dh + vi) =
            out;
      }
      __syncthreads();

      // The bottom warps update their values.
      if (vo < midpoint && (Dh == Dh_MAX || vi < Dh)) {
        out = add(*reinterpret_cast<Out_sum const *>(out_smem + vo * Dh + vi),
                  out);
      }
      __syncthreads();
    }
  }

  // Output the final values.
  if (vo == 0 && (Dh == Dh_MAX || vi < Dh)) {
    convert_from_float(
        *reinterpret_cast<V_vec *>(output_ptr + beam_request_idx * hidden_size +
                                   head_idx * per_head_size + vi),
        out);
  }
}

// only used by MPT model. https://arxiv.org/abs/2108.12409
template <typename DT>
__global__ void apply_position_bias_qkprd(DT *input_ptr,
                                          int num_tokens,
                                          int num_total_tokens,
                                          int num_heads,
                                          int global_num_q_heads,
                                          int shard_id) {
  CUDA_KERNEL_LOOP(i, num_tokens * num_total_tokens * num_heads) {
    // get head_idx,
    int head_idx = i / (num_tokens * num_total_tokens) + (num_heads * shard_id);
    int position_idx = (i / num_tokens) % num_total_tokens;
    position_idx = position_idx + 1 - num_total_tokens;
    // 8 is alibi_bias_max in
    // https://huggingface.co/mosaicml/mpt-30b/blob/main/config.json
    float base = (float)(head_idx + 1) * 8 / global_num_q_heads;
    float slopes = 1.0 / pow(2, base);
    // if(i == 0){
    //   printf("see position: %d, %f, %f, %f\n", position_idx, base, slopes,
    //   position_idx * slopes);
    // }
    input_ptr[i] += static_cast<DT>(position_idx * slopes);
  }
}

template <typename DT>
__global__ void apply_proj_bias_w(DT *input_ptr,
                                  DT const *bias_ptr,
                                  int num_tokens,
                                  int qkv_weight_size,
                                  int oProjSize) {
  CUDA_KERNEL_LOOP(i, num_tokens * oProjSize) {
    int bias_idx = qkv_weight_size + i % oProjSize;
    input_ptr[i] += bias_ptr[bias_idx];
  }
}

template <typename DT>
__global__ void apply_proj_bias_qkv(DT *input_ptr,
                                    DT const *bias_ptr,
                                    int shard_id,
                                    int num_tokens,
                                    int qProjSize,
                                    int kProjSize,
                                    int vProjSize,
                                    int global_num_q_heads,
                                    int num_q_heads,
                                    bool scaling_query,
                                    float scaling_factor,
                                    int hidden_size) {
  CUDA_KERNEL_LOOP(i, num_tokens * hidden_size * QKV_WEIGHT_NUM) {
    // for simplicity, assume q, k, v is in same shape
    // 0->q, 1->k, 2->v
    // int qkv_index = i / (num_tokens * qProjSize) % 3;

    int token_idx = i / (hidden_size * QKV_WEIGHT_NUM);
    size_t in_token_idx = i - token_idx * hidden_size * QKV_WEIGHT_NUM;

    int qkv_index = in_token_idx / hidden_size;

    int proj_size = qkv_index == 0 ? qProjSize : kProjSize;

    int head_idx =
        (in_token_idx - qkv_index * num_q_heads * proj_size) / proj_size;
    int global_head_idx = head_idx + shard_id * num_q_heads;

    size_t pre_length =
        qkv_index == 0
            ? 0
            : (qkv_index == 1 ? qProjSize * global_num_q_heads
                              : qProjSize * global_num_q_heads * KV_WEIGHT_NUM);

    size_t bias_idx = pre_length + global_head_idx * proj_size + i % proj_size;

    input_ptr[i] += bias_ptr[bias_idx];

    if (scaling_query && qkv_index == 0) {
      input_ptr[i] *= scaling_factor;
    }
  }
}

template <typename DT>
__global__ void scaling_query_kernel(DT *input_ptr,
                                     int qProjSize,
                                     int num_tokens,
                                     int num_q_heads,
                                     float scaling_factor,
                                     int hidden_size) {
  CUDA_KERNEL_LOOP(i, num_tokens * hidden_size) {
    int token_idx = i / hidden_size;
    input_ptr[i % hidden_size + token_idx * hidden_size * QKV_WEIGHT_NUM] *=
        scaling_factor;
  }
}

template <typename DT>
__global__ void
    apply_rotary_embedding_native(DT *input_ptr,
                                  hipFloatComplex *complex_input,
                                  BatchConfig::PerTokenInfo const *tokenInfos,
                                  int qProjSize,
                                  int kProjSize,
                                  int num_q_heads,
                                  int num_tokens,
                                  int num_kv_heads,
                                  int q_block_size,
                                  int k_block_size,
                                  int q_array_size) {
  CUDA_KERNEL_LOOP(
      i,
      num_tokens * (qProjSize * num_q_heads + kProjSize * num_kv_heads) / 2) {
    // create complex number
    bool q_tensor = i < (q_array_size / 2);
    int proj_size = q_tensor ? qProjSize : kProjSize;
    int real_i = q_tensor ? i : i - q_array_size / 2;

    int head_idx = real_i / (num_tokens * proj_size / 2);
    int idx = real_i % (num_tokens * proj_size / 2);
    int real_part_index = idx * 2 +
                          head_idx * (q_tensor ? q_block_size : k_block_size) +
                          (q_tensor ? 0 : q_array_size);

    int complex_part_index = real_part_index + 1;

    complex_input[i] = {input_ptr[real_part_index],
                        input_ptr[complex_part_index]};

    int token_idx =
        (real_i - head_idx * (num_tokens * proj_size / 2)) / (proj_size / 2);
    size_t pos = tokenInfos[token_idx].abs_depth_in_request;

    // float before_real = complex_input[i].x, before_complex =
    // complex_input[i].y;

    int pos_i = real_i % (proj_size / 2);
    float freq = pos * (1.0 / pow(10000.0, (float)2 * pos_i / proj_size));
    hipFloatComplex complex_pos = {cos(freq), sin(freq)};

    complex_input[i] = hipCmulf(complex_input[i], complex_pos);
    input_ptr[real_part_index] = complex_input[i].x;
    input_ptr[complex_part_index] = complex_input[i].y;
  }
}

template <typename DT>
__global__ void
    apply_rotary_embedding_hf(DT *input_ptr,
                              hipFloatComplex *complex_input,
                              BatchConfig::PerTokenInfo const *tokenInfos,
                              int qProjSize,
                              int kProjSize,
                              int num_tokens,
                              size_t q_array_size,
                              int hidden_size) {
  CUDA_KERNEL_LOOP(i, num_tokens * hidden_size) {
    // create complex number
    bool q_tensor = i < (q_array_size / 2);
    int proj_size = q_tensor ? qProjSize : kProjSize;
    int real_i = q_tensor ? i : i - q_array_size / 2;

    int token_idx = real_i / (hidden_size / 2);
    int idx = real_i % (proj_size / 2);
    int head_idx = (real_i - (token_idx * (hidden_size / 2))) / (proj_size / 2);

    int real_part_index = idx + head_idx * proj_size +
                          token_idx * hidden_size * QKV_WEIGHT_NUM +
                          hidden_size * (q_tensor ? 0 : 1);
    int complex_part_index = real_part_index + (proj_size / 2);

    complex_input[i] = {input_ptr[real_part_index],
                        input_ptr[complex_part_index]};

    // get the freq_cis: shape 1 * (qProjSize/2) = 1 * 64
    // apply a Cartesian coordinate transformation
    // multiple with input & /copy back to q/k

    // get position of token

    // size_t pos = id_map[token_idx].token_position;
    size_t pos = tokenInfos[token_idx].abs_depth_in_request;

    // float before_real = complex_input[i].x, before_complex =
    int pos_i = real_i % (proj_size / 2);
    float freq = pos * (1.0 / pow(10000.0, (float)2 * pos_i / proj_size));
    hipFloatComplex complex_pos = {cos(freq), sin(freq)};

    complex_input[i] = hipCmulf(complex_input[i], complex_pos);
    input_ptr[real_part_index] = complex_input[i].x;
    input_ptr[complex_part_index] = complex_input[i].y;
  }
}

template <typename DT>
void compute_qkv_kernel(IncMultiHeadSelfAttentionMeta const *m,
                        BatchConfig const *bc,
                        int shard_id,
                        DT const *input_ptr,
                        DT const *weight_ptr,
                        DT *output_ptr,
                        DT const *bias_ptr,
                        hipStream_t stream) {

  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  assert(m->qSize == m->vSize && m->qSize == m->kSize);
  hipDataType cublas_data_type = ff_to_cuda_datatype(m->output_type[0]);
#if defined(CUDA_VERSION) && (CUDA_VERSION < 11000)
  hipDataType compute_type = cublas_data_type;
#else
  // For best performance, set the default cublas compute type to
  // HIPBLAS_COMPUTE_16F for half precision and to
  // HIPBLAS_COMPUTE_32F_FAST_16F for full precision
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
  if (m->output_type[0] == DT_FLOAT) {
    compute_type = HIPBLAS_COMPUTE_32F_FAST_16F;
  }
#endif

  // Step 1: Compute QKV projections
  {
    DT alpha = 1.0f, beta = 0.0f;
    // after transpositions
    int m_q = m->qProjSize * m->num_q_heads;
    int m_k = m->kProjSize * m->num_q_heads;
    int m_v = m->vProjSize * m->num_q_heads;
    assert(m_q == m_k && m_k == m_v); // keep things simple for now
    int n = bc->num_active_tokens();
    int k = m->qSize;
    int m_ = m_q * QKV_WEIGHT_NUM;
    // before transpositions
    int lda = k, ldb = k, ldc = m_;
    // matrix A: QKV weights
    // matrix A's layout: [qSize (hidden_dim), qProjSize, num_heads, 3]
    // matrix B: input
    // matrix B's layout: [qSize (hidden_dim), num_new_tokens]
    // matrix C: devQKVProjArray
    // matrix B's layout: [qProjSize, num_heads, 3, num_new_tokens]
    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_T,
                           HIPBLAS_OP_N,
                           m_,
                           n,
                           k,
                           &alpha,
                           weight_ptr,
                           cublas_data_type,
                           lda,
                           input_ptr,
                           cublas_data_type,
                           ldb,
                           &beta,
                           output_ptr,
                           cublas_data_type,
                           ldc,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  }

  int num_tokens = bc->num_active_tokens();
  int parallelism = m->kProjSize * num_tokens * m->num_q_heads;
  size_t q_array_size = m->qProjSize * num_tokens * m->num_q_heads;

  // Step 2: apply bias for QKV, or scale the query
  if (*m->qkv_bias) {
    apply_proj_bias_qkv<<<GET_BLOCKS(parallelism),
                          min(CUDA_NUM_THREADS, parallelism),
                          0,
                          stream>>>(output_ptr,
                                    bias_ptr,
                                    shard_id,
                                    num_tokens,
                                    m->qProjSize,
                                    m->kProjSize,
                                    m->vProjSize,
                                    m->global_num_q_heads,
                                    m->num_q_heads,
                                    *m->scaling_query,
                                    m->scaling_factor,
                                    m->hidden_size);
  } else if (m->scaling_query) {
    scaling_query_kernel<<<GET_BLOCKS(parallelism),
                           min(CUDA_NUM_THREADS, parallelism),
                           0,
                           stream>>>(output_ptr,
                                     num_tokens,
                                     m->num_q_heads,
                                     m->qProjSize,
                                     m->scaling_factor,
                                     m->hidden_size);
  }

  // Step 3: apply rotary embedding if needed
  if (*m->apply_rotary_embedding) {
    /*q&k*/
    parallelism = num_tokens * m->hidden_size;
    apply_rotary_embedding_hf<<<GET_BLOCKS(parallelism),
                                min(CUDA_NUM_THREADS, parallelism),
                                0,
                                stream>>>(output_ptr,
                                          m->complex_input,
                                          m->token_infos,
                                          m->qProjSize,
                                          m->kProjSize,
                                          num_tokens,
                                          q_array_size,
                                          m->hidden_size);
  }
}

template <typename DT>
void update_kv_cache_kernel(IncMultiHeadSelfAttentionMeta const *m,
                            BatchConfig const *bc,
                            hipStream_t stream) {
  int num_tokens = bc->num_active_tokens();
  if (num_tokens > 0) {
    int parallelism = m->hidden_size * num_tokens;
    store_kv_cache<<<GET_BLOCKS(parallelism),
                     min(CUDA_NUM_THREADS, parallelism),
                     0,
                     stream>>>(static_cast<DT *>(m->devQKVProjArray),
                               static_cast<DT *>(m->keyCache),
                               static_cast<DT *>(m->valueCache),
                               m->token_infos,
                               num_tokens,
                               BatchConfig::max_sequence_length(),
                               m->hidden_size);
  }
}

template <typename DT>
void compute_o_prod_bias(IncMultiHeadSelfAttentionMeta const *m,
                         BatchConfig const *bc,
                         int shard_id,
                         DT *output_ptr,
                         DT const *weight_ptr,
                         DT const *bias_ptr,
                         int num_tokens,
                         hipStream_t stream) {
  hipDataType cublas_data_type = ff_to_cuda_datatype(m->output_type[0]);
  hipdnnDataType_t cudnn_data_type = ff_to_cudnn_datatype(m->output_type[0]);
  assert(data_type_size(m->output_type[0]) == sizeof(DT));
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = cublas_data_type;
#endif
  // Project to output, save result directly on output tensor
  {
    DT alpha = 1.0f, beta = 0.0f;
    // after transpositions
    int m_ = m->oProjSize;
    int k = m->vProjSize * m->num_q_heads;
    int n = num_tokens;
    // before transpositions
    int lda = k, ldb = k, ldc = m_;
    // matrix A: output projection weight
    // matrix A's layout: [vProjSize * num_heads, oProjSize]
    DT const *A = weight_ptr + m->qSize * (m->qProjSize * m->num_q_heads +
                                           m->kProjSize * m->num_q_heads +
                                           m->vProjSize * m->num_q_heads);
    // matrix B: attn heads
    // matrix B's layout: [vProjSize * num_heads, num_new_tokens]
    DT const *B = static_cast<DT *>(m->attn_heads);
    // matrix B: output
    // matrix B's layout: [oProjSize, num_new_tokens]
    DT *C = static_cast<DT *>(output_ptr);

    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_T,
                           HIPBLAS_OP_N,
                           m_,
                           n,
                           k,
                           &alpha,
                           A,
                           cublas_data_type,
                           lda,
                           B,
                           cublas_data_type,
                           ldb,
                           &beta,
                           C,
                           cublas_data_type,
                           ldc,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  }
  // Add final output bias
  if (*m->final_bias && shard_id == 0) {
    int parallelism = m->oProjSize * num_tokens;
    int qkv_weight_size = m->qProjSize * m->global_num_q_heads +
                          m->kProjSize * m->global_num_q_heads +
                          m->vProjSize * m->global_num_q_heads;
    apply_proj_bias_w<<<GET_BLOCKS(parallelism),
                        min(CUDA_NUM_THREADS, parallelism),
                        0,
                        stream>>>(
        output_ptr, bias_ptr, num_tokens, qkv_weight_size, m->oProjSize);
  }
}

#define LAUNCH_ATTENTION_SCORE_KERNEL(                                         \
    DT, Dh, Dh_MAX, THDS_PER_KEY, THREADS_PER_VALUE, THDS_PER_BLOCK, stream)   \
  smem_sz = smem_size_in_bytes<DT>(m->qProjSize,                               \
                                   BatchConfig::max_sequence_length(),         \
                                   THREADS_PER_VALUE,                          \
                                   THDS_PER_BLOCK);                            \
  compute_attention_kernel_generation_kernel<DT,                               \
                                             THDS_PER_BLOCK,                   \
                                             Dh,                               \
                                             Dh_MAX,                           \
                                             THDS_PER_KEY,                     \
                                             THREADS_PER_VALUE>                \
      <<<grid, THDS_PER_BLOCK, smem_sz, stream>>>(                             \
          static_cast<DT *>(m->devQKVProjArray),                               \
          static_cast<DT *>(m->keyCache),                                      \
          static_cast<DT *>(m->valueCache),                                    \
          output_ptr,                                                          \
          scale,                                                               \
          BatchConfig::max_sequence_length(),                                  \
          m->qProjSize,                                                        \
          m->hidden_size,                                                      \
          m->request_infos,                                                    \
          false,                                                               \
          0)

template <typename DT>
void compute_attention_kernel_generation(IncMultiHeadSelfAttentionMeta const *m,
                                         BatchConfig const *bc,
                                         DT *output_ptr,
                                         hipStream_t stream) {
  dim3 grid(m->num_q_heads, bc->num_generation_tokens);
  int const per_head_size = m->qProjSize;
  float scale = (*m->qk_prod_scaling) ? 1.0f / sqrt(m->kProjSize) : 1.0f;
  size_t smem_sz;
  if (per_head_size == 64) {
    constexpr int THREADS_PER_VALUE_64 = threads_per_value_t<DT, 64>::value;
    LAUNCH_ATTENTION_SCORE_KERNEL(
        DT, 64, 64, 4, THREADS_PER_VALUE_64, 128, stream);
  } else if (per_head_size == 128) {
    constexpr int THREADS_PER_VALUE_128 = threads_per_value_t<DT, 128>::value;
    LAUNCH_ATTENTION_SCORE_KERNEL(
        DT, 128, 128, 4, THREADS_PER_VALUE_128, 128, stream);
  } else {
    assert(false && "a unsupported head size");
  }
}

template <typename DT>
void pre_build_weight_kernel(IncMultiHeadSelfAttentionMeta const *m,
                             GenericTensorAccessorR const weight,
                             DataType data_type,
                             hipStream_t stream) {
  // additional processing for weight uploading
  // Note that we update weight_ptr and bias_ptr when uploading weight and
  // bias
  if (m->quantization_type != DT_NONE) {
    // copy weight_ptr to quantized_weight_ptr, do compression and store in
    // m->weight_ptr
    hipMemcpyAsync(m->quantized_weight_ptr,
                    weight.get_byte_ptr(),
                    m->quantized_weightSize,
                    hipMemcpyHostToDevice,
                    stream);

    if (m->quantization_type == DT_INT4) {
      int parallelism = m->qProjSize * m->qSize * m->num_q_heads / 2;
      decompress_int4_attention_weights<<<GET_BLOCKS(parallelism),
                                          min(CUDA_NUM_THREADS, parallelism),
                                          0,
                                          stream>>>(
          m->quantized_weight_ptr,
          static_cast<DT *>(m->weight_ptr),
          m->qProjSize,
          m->qSize,
          m->num_q_heads);
    } else {
      assert(m->quantization_type == DT_INT8);
      int parallelism = m->qProjSize * m->qSize * m->num_q_heads;
      decompress_int8_attention_weights<<<GET_BLOCKS(parallelism),
                                          min(CUDA_NUM_THREADS, parallelism),
                                          0,
                                          stream>>>(
          m->quantized_weight_ptr,
          static_cast<DT *>(m->weight_ptr),
          m->qProjSize,
          m->qSize,
          m->num_q_heads);
    }
  } else {
    if (data_type == DT_FLOAT) {
      hipMemcpyAsync(m->weight_ptr,
                      weight.get_float_ptr(),
                      m->weightSize,
                      hipMemcpyHostToDevice,
                      stream);
    } else if (data_type == DT_HALF) {
      hipMemcpyAsync(m->weight_ptr,
                      weight.get_half_ptr(),
                      m->weightSize,
                      hipMemcpyHostToDevice,
                      stream);
    } else {
      assert(false);
    }
  }
}

template <typename DT>
void inference_kernel(IncMultiHeadSelfAttentionMeta const *m,
                      BatchConfig const *bc,
                      int shard_id,
                      DT const *input_ptr,
                      DT const *weight_ptr,
                      DT *output_ptr,
                      DT const *bias_ptr,
                      hipStream_t stream) {

  if (m->offload && m->biasSize > 0) {
    hipMemcpyAsync(
        m->bias_ptr, bias_ptr, m->biasSize, hipMemcpyHostToDevice, stream);
    bias_ptr = static_cast<DT *>(m->bias_ptr);
  }

  // todo Xinhao copy how many requests if requests are not continous?
  hipMemcpyAsync(m->token_infos,
                  &(bc->tokensInfo),
                  bc->num_active_tokens() * sizeof(BatchConfig::PerTokenInfo),
                  hipMemcpyHostToDevice,
                  stream);
  hipMemcpyAsync(m->request_infos,
                  &(bc->requestsInfo),
                  bc->max_requests_per_batch() *
                      sizeof(BatchConfig::PerRequestInfo),
                  hipMemcpyHostToDevice,
                  stream);

  // phase 1: Implement kernel to compute KQV for input tokens
  compute_qkv_kernel(m,
                     bc,
                     shard_id,
                     input_ptr,
                     weight_ptr,
                     static_cast<DT *>(m->devQKVProjArray),
                     bias_ptr,
                     stream);
  update_kv_cache_kernel<DT>(m, bc, stream);

  if (bc->num_generation_tokens > 0) {
    // phase 3: Compute attention score for generation tokens
    compute_attention_kernel_generation<DT>(
        m, bc, static_cast<DT *>(m->attn_heads), stream);
  }

  if (bc->num_tokens > bc->num_generation_tokens) {
    // phase 4: Compute attention score for prompt tokens;
    compute_attention_kernel_prompt(
        m, bc, shard_id, bias_ptr, weight_ptr, stream);
  }

  // compute output production and bias together for all tokens
  int num_tokens = bc->num_active_tokens();
  compute_o_prod_bias(
      m, bc, shard_id, output_ptr, weight_ptr, bias_ptr, num_tokens, stream);
}

} // namespace IncMultiHeadAttention
} // namespace Kernels

using namespace Kernels::IncMultiHeadAttention;

template <typename DT>
__global__ void store_kv_cache(DT const *devQKVProjArray,
                               DT *kCache_ptr,
                               DT *vCache_ptr,
                               BatchConfig::PerTokenInfo const *tokenInfos,
                               int num_tokens,
                               int max_seq_len,
                               int hidden_size) {
  CUDA_KERNEL_LOOP(i, num_tokens * hidden_size) {
    int token_idx = i / hidden_size;
    int offset = i % hidden_size;

    size_t val_idx =
        token_idx * QKV_WEIGHT_NUM * hidden_size + hidden_size + offset;

    DT kVal = devQKVProjArray[val_idx];
    DT vVal = devQKVProjArray[val_idx + hidden_size];
    int const req_id = tokenInfos[token_idx].request_index;
    int const tok_id = tokenInfos[token_idx].abs_depth_in_request;

    // key cache
    kCache_ptr[req_id * (hidden_size * max_seq_len) + tok_id * hidden_size +
               offset] = kVal;
    vCache_ptr[req_id * (hidden_size * max_seq_len) + tok_id * hidden_size +
               offset] = vVal;
  }
}

template <typename DT>
__global__ void fill_entries_above_diagonal(DT *matrix,
                                            size_t num_rows,
                                            size_t num_cols,
                                            size_t num_q_heads,
                                            size_t entries_above_diagonal,
                                            DT value) {
  CUDA_KERNEL_LOOP(i, entries_above_diagonal * num_q_heads) {
    size_t head_idx = i / entries_above_diagonal;
    size_t entry_idx = i % entries_above_diagonal;
    size_t y = (-1 + sqrt(8 * (float)entry_idx + 1)) / 2;
    size_t x = entry_idx - y * (y + 1) / 2;
    y += (num_cols - num_rows) + 1;
    matrix[head_idx * num_rows * num_cols + num_cols * y + x] = value;
  }
}

template <typename DT>
void compute_attention_kernel_prompt(IncMultiHeadSelfAttentionMeta const *m,
                                     BatchConfig const *bc,
                                     int shard_id,
                                     DT const *bias_ptr,
                                     DT const *weight_ptr,
                                     hipStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  hipDataType cublas_data_type = ff_to_cuda_datatype(m->output_type[0]);
  hipdnnDataType_t cudnn_data_type = ff_to_cudnn_datatype(m->output_type[0]);
  assert(data_type_size(m->output_type[0]) == sizeof(DT));
#if defined(CUDA_VERSION) && (CUDA_VERSION < 11000)
  hipDataType compute_type = cublas_data_type;
#else
  // For best performance, set the default cublas compute type to
  // HIPBLAS_COMPUTE_16F for half precision and to
  // HIPBLAS_COMPUTE_32F_FAST_16F for full precision
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
  if (m->output_type[0] == DT_FLOAT) {
    compute_type = HIPBLAS_COMPUTE_32F_FAST_16F;
  }
#endif
  // int num_requests = bc->num_active_requests();
  int num_tokens = bc->num_active_tokens();
  int tokens_previous_requests = 0;
  int q_block_size = m->qProjSize;
  int kt_block_size = m->kProjSize;
  int kt_req_block_size =
      kt_block_size * m->num_q_heads * BatchConfig::max_sequence_length();
  int vt_block_size = m->vProjSize;
  int vt_req_block_size =
      vt_block_size * m->num_q_heads * BatchConfig::max_sequence_length();
  assert(m->qProjSize == m->kProjSize);

  for (int i = 0; i < bc->max_requests_per_batch(); i++) {
    if (bc->request_completed[i]) {
      continue;
    } else if (tokens_previous_requests < bc->num_generation_tokens) {
      tokens_previous_requests += bc->requestsInfo[i].num_tokens_in_batch;
      continue;
    }
    assert(tokens_previous_requests ==
           bc->requestsInfo[i].first_token_offset_in_batch);
    int num_new_tokens = bc->requestsInfo[i].num_tokens_in_batch;
    int total_tokens = bc->requestsInfo[i].first_token_depth_in_request +
                       bc->requestsInfo[i].num_tokens_in_batch;
    // Step 1: compute query-key product QK.T/sqrt(d_k)
    {
      // Scale by sqrt(d_k) as per the original attention paper
      DT alpha = 1.0f, beta = 0.0f;
      if (*m->qk_prod_scaling) {
        alpha = static_cast<DT>(1.0f / sqrt(m->kProjSize));
      }
      // after transpositions
      int m_ = num_new_tokens;
      int n = total_tokens;
      int k = m->qProjSize;
      // before transpositions
      int lda = k * m->num_q_heads * QKV_WEIGHT_NUM, ldb = k * m->num_q_heads,
          ldc = m_;
      // N.B. strides are applied before transpose operations
      int strideA = q_block_size;
      int strideB = kt_block_size;
      int strideC = num_new_tokens * total_tokens;

      // matrix A: devQKVProjArray
      // matrix A's layout: [qProjSize, num_heads, 3, num_new_tokens]
      // To get query projection, skip over Q entries from previous requests
      DT const *A = static_cast<DT *>(m->devQKVProjArray) +
                    tokens_previous_requests * m->qProjSize * m->num_q_heads *
                        QKV_WEIGHT_NUM;
      // matrix B: key cache
      // matrix B's layout: [kProjSize * num_heads, total_tokens]
      // To get B, skip over K entries from previous requests (all heads +
      // padding)
      DT const *B = static_cast<DT *>(m->keyCache) + i * kt_req_block_size;
      // matrix C: qk_prods
      // matrix C's layout: [num_new_tokens, total_tokens, num_heads]
      // To get C, skip over QK.T products from previous requests
      DT *C = static_cast<DT *>(m->qk_prods);
      checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                           HIPBLAS_OP_T,
                                           HIPBLAS_OP_N,
                                           m_,
                                           n,
                                           k,
                                           &alpha,
                                           A,
                                           cublas_data_type,
                                           lda,
                                           strideA,
                                           B,
                                           cublas_data_type,
                                           ldb,
                                           strideB,
                                           &beta,
                                           C,
                                           cublas_data_type,
                                           ldc,
                                           strideC,
                                           m->num_q_heads,
                                           compute_type,
                                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    }
    // Step 2: Add alibi position bias to qk production
    // matrix C: qk_prods
    // matrix C's layout: [num_new_tokens, total_tokens, num_heads]
    // To get C, skip over QK.T products from previous requests
    DT *C = static_cast<DT *>(m->qk_prods);
    if (*m->position_bias) {
      size_t parallelism = m->num_q_heads * total_tokens * num_new_tokens;
      apply_position_bias_qkprd<<<GET_BLOCKS(parallelism),
                                  min((size_t)CUDA_NUM_THREADS, parallelism),
                                  0,
                                  stream>>>(C,
                                            num_new_tokens,
                                            total_tokens,
                                            m->num_q_heads,
                                            m->global_num_q_heads,
                                            shard_id);
    }

    // Step 3: Apply causal mask. Fill all elements above diagonal in qk prods
    // with -inf to force causal attention.
    assert(num_new_tokens <= total_tokens);
    size_t entries_above_diagonal = num_new_tokens * (num_new_tokens - 1) / 2;
    if (entries_above_diagonal > 0) {
      size_t parallelism = m->num_q_heads * entries_above_diagonal;
      fill_entries_above_diagonal<<<GET_BLOCKS(parallelism),
                                    min((size_t)CUDA_NUM_THREADS, parallelism),
                                    0,
                                    stream>>>(C,
                                              num_new_tokens,
                                              total_tokens,
                                              m->num_q_heads,
                                              entries_above_diagonal,
                                              static_cast<DT>(-INFINITY));
    }

    // Step 4: Compute Softmax(QK.T/sqrt(d_k))
    {
      // Before modifying the parameters below, make sure to read the following
      // description of the HIPDNN_TENSOR_NCHW tensor layout, from
      // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnTensorFormat_t:
      // This tensor format specifies that the data is laid out in the following
      // order: batch size, feature maps, rows, columns. The strides are
      // implicitly defined in such a way that the data are contiguous in memory
      // with no padding between images, feature maps, rows, and columns; the
      // columns are the inner dimension and the images are the outermost
      // dimension.
      int n_param = m->num_q_heads;
      int c_param = total_tokens;
      int h_param = 1;
      int w_param = num_new_tokens;
      checkCUDNN(hipdnnSetTensor4dDescriptor(m->qk_tensor,
                                            HIPDNN_TENSOR_NCHW,
                                            cudnn_data_type,
                                            n_param,
                                            c_param,
                                            h_param,
                                            w_param));
      float softmax_alpha = 1.0f, softmax_beta = 0.0f;
      DT *C_softmax = static_cast<DT *>(m->qk_prods_softmax);
      // The softmax operation below is executed according to the
      // HIPDNN_SOFTMAX_MODE_CHANNEL, which is also described in the docs: The
      // softmax operation is computed per spatial location (H,W) per image (N)
      // across dimension C.
      checkCUDNN(hipdnnSoftmaxForward(m->handle.dnn,
                                     HIPDNN_SOFTMAX_ACCURATE,
                                     HIPDNN_SOFTMAX_MODE_CHANNEL,
                                     &softmax_alpha,
                                     m->qk_tensor,
                                     C,
                                     &softmax_beta,
                                     m->qk_tensor,
                                     C_softmax));
    }
    // Step 5: Matmul softmax(QK.T/sqrt(d_k)) by V. Implemented as V @
    // softmax(QK.T/sqrt(d_k)).T
    {
      DT alpha = 1.0f, beta = 0.0f;
      // after transpositions
      int m_ = m->vProjSize;
      int n = num_new_tokens;
      int k = total_tokens;
      // before transpositions
      int lda = m_ * m->num_q_heads, ldb = n, ldc = m_ * m->num_q_heads;
      // N.B. strides are applied before transpose operations
      int strideA = vt_block_size;
      int strideB = num_new_tokens * total_tokens;
      int strideC = m->vProjSize;
      // matrix A: value cache
      // matrix A's layout: [vProjSize, num_heads, total_tokens]
      // To get A, skip over V.T entries from previous requests (all heads +
      // padding)
      DT *A = static_cast<DT *>(m->valueCache) + i * vt_req_block_size;
      // matrix B: qk_prods_softmax
      // matrix B's layout: [num_new_tokens, total_tokens, num_heads]
      // To get B, skip over softmax(QK.T/sqrt(d_k)) entries from previous
      // requests (all heads)
      DT *B = static_cast<DT *>(m->qk_prods_softmax);
      ;
      // matrix C: attn heads
      // matrix C's layout: [vProjSize, num_heads, num_new_tokens]
      // To get C, skip over softmax(QK.T/sqrt(d_k))V products from previous
      // requests
      // store the result attn heads, also skip the genration tokens
      DT *C = static_cast<DT *>(m->attn_heads) +
              (tokens_previous_requests + bc->num_generation_tokens) *
                  m->num_q_heads * m->vProjSize;
      checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                           HIPBLAS_OP_N,
                                           HIPBLAS_OP_T,
                                           m_,
                                           n,
                                           k,
                                           &alpha,
                                           A,
                                           cublas_data_type,
                                           lda,
                                           strideA,
                                           B,
                                           cublas_data_type,
                                           ldb,
                                           strideB,
                                           &beta,
                                           C,
                                           cublas_data_type,
                                           ldc,
                                           strideC,
                                           m->num_q_heads,
                                           compute_type,
                                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    }
    tokens_previous_requests += num_new_tokens;
  }
  assert(tokens_previous_requests == num_tokens);
}

/*static*/
void IncMultiHeadSelfAttention::inference_kernel_wrapper(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    int shard_id,
    GenericTensorAccessorR const &input,
    GenericTensorAccessorR const &weight,
    GenericTensorAccessorW const &output,
    GenericTensorAccessorR const &bias) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  bool use_bias = *m->qkv_bias || *m->final_bias;

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  // assert(input.data_type == weight.data_type);
  assert(input.data_type == output.data_type);
  if (use_bias) {
    assert(input.data_type == bias.data_type);
  }

  if (input.data_type == DT_HALF) {
    if (m->offload) {
      pre_build_weight_kernel<half>(m, weight, input.data_type, stream);
    }
    half const *bias_ptr =
        use_bias ? bias.get_half_ptr() : static_cast<half const *>(nullptr);
    Kernels::IncMultiHeadAttention::inference_kernel(
        m,
        bc,
        shard_id,
        input.get_half_ptr(),
        m->offload ? static_cast<half *>(m->weight_ptr) : weight.get_half_ptr(),
        output.get_half_ptr(),
        bias_ptr,
        stream);
  } else if (input.data_type == DT_FLOAT) {
    if (m->offload) {
      pre_build_weight_kernel<float>(m, weight, input.data_type, stream);
    }
    float const *bias_ptr =
        use_bias ? bias.get_float_ptr() : static_cast<float const *>(nullptr);
    Kernels::IncMultiHeadAttention::inference_kernel(
        m,
        bc,
        shard_id,
        input.get_float_ptr(),
        m->offload ? static_cast<float *>(m->weight_ptr)
                   : weight.get_float_ptr(),
        output.get_float_ptr(),
        bias_ptr,
        stream);
  } else {
    assert(false && "Unspported data type");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("IncMultiHeadSelfAttention forward time = %.9fms\n", elapsed);
  }
}

IncMultiHeadSelfAttentionMeta::IncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    IncMultiHeadSelfAttention const *attn,
    GenericTensorAccessorR const &weight,
    MemoryAllocator &gpu_mem_allocator,
    int num_samples,
    int _num_q_heads,
    int _num_kv_heads)
    : IncMultiHeadSelfAttentionMeta(handler,
                                    INC_DECODING_MODE,
                                    attn,
                                    attn->qSize,
                                    attn->kSize,
                                    attn->vSize,
                                    attn->qProjSize,
                                    attn->kProjSize,
                                    attn->vProjSize,
                                    attn->oProjSize,
                                    attn->apply_rotary_embedding,
                                    attn->qkv_bias,
                                    attn->scaling_query,
                                    attn->qk_prod_scaling,
                                    attn->position_bias,
                                    attn->final_bias,
                                    attn->scaling_factor,
                                    weight,
                                    gpu_mem_allocator,
                                    num_samples,
                                    attn->num_q_heads,
                                    attn->num_kv_heads,
                                    _num_q_heads,
                                    _num_kv_heads,
                                    attn->quantization_type,
                                    attn->offload) {}

IncMultiHeadSelfAttentionMeta::IncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    InferenceMode infer_mode,
    Op const *attn,
    int _qSize,
    int _kSize,
    int _vSize,
    int _qProjSize,
    int _kProjSize,
    int _vProjSize,
    int _oProjSize,
    bool _apply_rotary_embedding,
    bool _qkv_bias,
    bool _scaling_query,
    bool _qk_prod_scaling,
    bool _position_bias,
    bool _final_bias,
    float _scaling_factor,
    GenericTensorAccessorR const &weight,
    MemoryAllocator &gpu_mem_allocator,
    int num_samples,
    int _global_num_q_heads,
    int _global_num_kv_heads,
    int _num_q_heads,
    int _num_kv_heads,
    DataType _quantization_type,
    bool _offload)
    : OpMeta(handler, attn), weight_ptr(nullptr), bias_ptr(nullptr) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  checkCUDNN(hipdnnSetStream(handler.dnn, stream));
  checkCUDNN(hipdnnCreateTensorDescriptor(&qk_tensor));
  qSize = _qSize;
  kSize = _kSize;
  vSize = _vSize;
  // assume dimensions match for now
  assert(qSize == kSize);
  assert(kSize == vSize);
  qProjSize = _qProjSize;
  kProjSize = _kProjSize;
  assert(qProjSize == kProjSize); // required for attention QK.T matmul
  vProjSize = _vProjSize;
  oProjSize = _oProjSize;
  size_t size_of_dt = data_type_size(attn->data_type);
  quantization_type = _quantization_type;
  offload = _offload;

  global_num_q_heads = _global_num_q_heads;
  global_num_kv_heads = _global_num_kv_heads;
  num_q_heads = _num_q_heads;
  num_kv_heads = _num_kv_heads;
  hidden_size = num_q_heads * qProjSize;

  weightSize =
      ((qSize * qProjSize + oProjSize * (vProjSize > 0 ? vProjSize : vSize)) *
           num_q_heads +
       (kSize * kProjSize + vSize * vProjSize) * num_q_heads) *
      size_of_dt;
  if (quantization_type != DT_NONE) {
    quantized_weightSize = get_quantization_to_byte_size(
        attn->data_type, quantization_type, weightSize);
  }
  // biasSize = _bias ? oProjSize * size_of_dt * 4 : 0;

  int qkv_bias_size =
      qProjSize * num_q_heads + (kProjSize + vProjSize) * num_q_heads;
  int final_bias_size = oProjSize;
  biasSize =
      (_qkv_bias ? qkv_bias_size : 0) + (final_bias ? final_bias_size : 0);

  // has_load_weights = (bool *)calloc(1, sizeof(bool));
  //*has_load_weights = false;
  apply_rotary_embedding = (bool *)calloc(1, sizeof(bool));
  *apply_rotary_embedding = _apply_rotary_embedding;
  qkv_bias = (bool *)calloc(1, sizeof(bool));
  *qkv_bias = _qkv_bias;
  scaling_query = (bool *)calloc(1, sizeof(bool));
  *scaling_query = _scaling_query;
  scaling_factor = _scaling_factor;
  qk_prod_scaling = (bool *)calloc(1, sizeof(bool));
  *qk_prod_scaling = _qk_prod_scaling;
  position_bias = (bool *)calloc(1, sizeof(bool));
  *position_bias = _position_bias;
  final_bias = (bool *)calloc(1, sizeof(bool));
  *final_bias = _final_bias;

  // allocate weight and bias in the reserve space for cpu offloading
  if (offload) {
    weight_ptr = gpu_mem_allocator.allocate_reserved_untyped(weightSize);
    bias_ptr = gpu_mem_allocator.allocate_reserved_untyped(biasSize);
  }

  // allocate memory for the seqArray and reserve space
  {
    int max_tokens_per_batch = BatchConfig::max_tokens_per_batch();
    size_t qkv_max_proj_size = max_tokens_per_batch * (qProjSize * num_q_heads +
                                                       kProjSize * num_q_heads +
                                                       vProjSize * num_q_heads);
    size_t key_cache_size = 0, value_cache_size = 0;
    switch (infer_mode) {
      case INC_DECODING_MODE:
      case TREE_VERIFY_MODE: {
        key_cache_size = num_q_heads * kProjSize *
                         BatchConfig::max_requests_per_batch() *
                         BatchConfig::max_sequence_length();
        value_cache_size = num_q_heads * vProjSize *
                           BatchConfig::max_requests_per_batch() *
                           BatchConfig::max_sequence_length();
        break;
      }
      case BEAM_SEARCH_MODE: {
        key_cache_size = num_q_heads * kProjSize *
                         BeamSearchBatchConfig::max_requests_per_batch() *
                         BatchConfig::max_sequence_length() *
                         BeamSearchBatchConfig::MAX_BEAM_WIDTH;
        value_cache_size = num_q_heads * vProjSize *
                           BeamSearchBatchConfig::max_requests_per_batch() *
                           BatchConfig::max_sequence_length() *
                           BeamSearchBatchConfig::MAX_BEAM_WIDTH;
        break;
      }
      default:
        assert(false && "Unkown inference mode");
    }
    size_t requestinfo_size = BatchConfig::max_requests_per_batch();
    size_t tokeninfo_size = max_tokens_per_batch;
    size_t qk_prod_size =
        max_tokens_per_batch * BatchConfig::max_sequence_length() * num_q_heads;
    size_t attn_heads_size = max_tokens_per_batch * num_q_heads * vProjSize;
    size_t complex_size = (max_tokens_per_batch * (qProjSize * num_q_heads +
                                                   kProjSize * num_q_heads)) /
                          2;
    size_t totalSize =
        (qkv_max_proj_size + key_cache_size + value_cache_size +
         2 * qk_prod_size + attn_heads_size) *
            size_of_dt +
        tokeninfo_size * sizeof(BatchConfig::PerTokenInfo) +
        complex_size * sizeof(hipFloatComplex) +
        requestinfo_size *
            sizeof(BatchConfig::PerRequestInfo); // more components will
                                                 // be added here later
    if (offload) {
      // assert that we have enough reserved work space left
      size_t totalSharedSize =
          infer_mode == TREE_VERIFY_MODE
              ? totalSize -
                    (key_cache_size + value_cache_size + qkv_max_proj_size) *
                        size_of_dt
              : totalSize - (key_cache_size + value_cache_size) * size_of_dt;

      size_t instance_size =
          size_of_dt *
          (infer_mode == TREE_VERIFY_MODE
               ? key_cache_size + value_cache_size + qkv_max_proj_size
               : key_cache_size + value_cache_size);

      if (quantization_type != DT_NONE) {
        totalSharedSize += quantized_weightSize;
      }
      assert(gpu_mem_allocator.reserved_total_size -
                 gpu_mem_allocator.reserved_allocated_size >=
             totalSharedSize);
      gpu_mem_allocator.create_legion_instance(reserveInst, instance_size);
    } else {
      gpu_mem_allocator.create_legion_instance(reserveInst, totalSize);
    }

    // in tree_verify, enable devQKVProjArray;
    if (!offload || infer_mode == TREE_VERIFY_MODE) {
      devQKVProjArray = gpu_mem_allocator.allocate_instance_untyped(
          qkv_max_proj_size * size_of_dt);
    } else {
      devQKVProjArray = gpu_mem_allocator.allocate_reserved_untyped(
          qkv_max_proj_size * size_of_dt);
      // offset += qkv_max_proj_size * size_of_dt;
    }

    // use key value cache in all mode.
    keyCache = gpu_mem_allocator.allocate_instance_untyped(key_cache_size *
                                                           size_of_dt);
    valueCache = gpu_mem_allocator.allocate_instance_untyped(value_cache_size *
                                                             size_of_dt);

    if (offload) {
      token_infos =
          gpu_mem_allocator.allocate_reserved<BatchConfig::PerTokenInfo>(
              tokeninfo_size);
      // offset += sizeof(BatchConfig::PerTokenInfo) * tokeninfo_size;
      qk_prods = gpu_mem_allocator.allocate_reserved_untyped(qk_prod_size *
                                                             size_of_dt);
      // offset += qk_prod_size * size_of_dt;
      qk_prods_softmax = gpu_mem_allocator.allocate_reserved_untyped(
          qk_prod_size * size_of_dt);
      // offset += qk_prod_size * size_of_dt;
      attn_heads = gpu_mem_allocator.allocate_reserved_untyped(attn_heads_size *
                                                               size_of_dt);
      // offset += attn_heads_size * size_of_dt;
      complex_input =
          gpu_mem_allocator.allocate_reserved<hipFloatComplex>(complex_size);
      // offset += complex_size * sizeof(hipFloatComplex);
      request_infos =
          gpu_mem_allocator.allocate_reserved<BatchConfig::PerRequestInfo>(
              requestinfo_size);
    } else {
      token_infos =
          gpu_mem_allocator.allocate_instance<BatchConfig::PerTokenInfo>(
              tokeninfo_size);
      qk_prods = gpu_mem_allocator.allocate_instance_untyped(qk_prod_size *
                                                             size_of_dt);
      qk_prods_softmax = gpu_mem_allocator.allocate_instance_untyped(
          qk_prod_size * size_of_dt);
      attn_heads = gpu_mem_allocator.allocate_instance_untyped(attn_heads_size *
                                                               size_of_dt);
      complex_input =
          gpu_mem_allocator.allocate_instance<hipFloatComplex>(complex_size);
      request_infos =
          gpu_mem_allocator.allocate_instance<BatchConfig::PerRequestInfo>(
              requestinfo_size);
    }

    // allocate more size for quantization data
    if (quantization_type != DT_NONE) {
      assert(offload);
      quantized_weight_ptr =
          gpu_mem_allocator.allocate_reserved<char>(quantized_weightSize);
    }
    if (!offload) {
      assert(gpu_mem_allocator.reserved_total_size ==
             gpu_mem_allocator.reserved_allocated_size);
    }
  }
  hipStreamSynchronize(stream);
}

IncMultiHeadSelfAttentionMeta::~IncMultiHeadSelfAttentionMeta(void) {
  if (reserveInst != Realm::RegionInstance::NO_INST) {
    reserveInst.destroy();
  }
}

template void Kernels::IncMultiHeadAttention::pre_build_weight_kernel<float>(
    IncMultiHeadSelfAttentionMeta const *m,
    GenericTensorAccessorR const weight,
    DataType data_type,
    hipStream_t stream);

template void Kernels::IncMultiHeadAttention::pre_build_weight_kernel<half>(
    IncMultiHeadSelfAttentionMeta const *m,
    GenericTensorAccessorR const weight,
    DataType data_type,
    hipStream_t stream);
}; // namespace FlexFlow
