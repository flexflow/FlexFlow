#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#if defined(FF_USE_CUDA) || defined(FF_USE_HIP_CUDA)
#include "hip/hip_complex.h"
#endif
#include "flexflow/ops/inc_multihead_self_attention.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

// declare Legion names
using Legion::coord_t;
using Legion::Memory;

__global__ void build_w_out_tensor(float const *weight_ptr,
                                   float *contiguous_weight_ptr,
                                   int vProjSize,
                                   int oProjSize,
                                   int num_heads,
                                   int qkv_weight_block_size) {
  CUDA_KERNEL_LOOP(i, vProjSize * oProjSize * num_heads) {
    int row_idx = i % vProjSize;
    int col_idx = (i / vProjSize) % oProjSize;
    int head_idx = i / (vProjSize * oProjSize);
    contiguous_weight_ptr[i] =
        weight_ptr[head_idx * (qkv_weight_block_size + vProjSize * oProjSize) +
                   qkv_weight_block_size + col_idx * vProjSize + row_idx];
  }
}

__global__ void apply_proj_bias_w(float *input_ptr,
                                  float const *bias_ptr,
                                  int num_tokens,
                                  int oProjSize) {
  CUDA_KERNEL_LOOP(i, num_tokens * oProjSize) {
    int bias_idx = 3 * oProjSize + i % oProjSize;
    input_ptr[i] += bias_ptr[bias_idx];
  }
}

__global__ void apply_proj_bias_qkv(float *input_ptr,
                                    float const *bias_ptr,
                                    int num_tokens,
                                    int qProjSize,
                                    int kProjSize,
                                    int vProjSize,
                                    int num_heads,
                                    bool scaling_query,
                                    float scaling_factor) {
  CUDA_KERNEL_LOOP(
      i, num_tokens * (qProjSize + kProjSize + vProjSize) * num_heads) {
    // for simplicity, assume q, k, v is in same shape
    // 0->q, 1->k, 2->v
    int qkv_index = i / (num_tokens * qProjSize) % 3;

    int head_idx = i / (num_tokens * (qProjSize + kProjSize + vProjSize));
    int qkv_block_size = (qProjSize + kProjSize + vProjSize) * num_tokens;
    int q_block_size = qProjSize * num_tokens;

    int idx = i % (num_tokens * (qProjSize));

    int real_part_index =
        head_idx * qkv_block_size + qkv_index * q_block_size + idx;
    int bias_idx = qkv_index * qProjSize * num_heads + head_idx * qProjSize +
                   (idx % qProjSize);
    input_ptr[real_part_index] += bias_ptr[bias_idx];

    if (scaling_query && qkv_index == 0) {
      input_ptr[real_part_index] *= scaling_factor;
    }
  }
}

__global__ void
    apply_rotary_embedding(float *input_ptr,
                           hipFloatComplex *complex_input,
                           BatchConfig::PerTokenInfo const *tokenInfos,
                           int qProjSize,
                           int kProjSize,
                           int num_heads,
                           int num_tokens,
                           int q_block_size,
                           int k_block_size,
                           int v_block_size,
                           bool q_tensor) {
  int proj_size = q_tensor ? qProjSize : kProjSize;
  CUDA_KERNEL_LOOP(i, num_tokens * proj_size * num_heads / 2) {
    // create complex number
    int head_idx = i / (num_tokens * proj_size / 2);
    int idx = i % (num_tokens * proj_size / 2);
    int real_part_index =
        idx * 2 + head_idx * (q_block_size + k_block_size + v_block_size) +
        (q_tensor ? 0 : q_block_size);
    int complex_part_index = real_part_index + 1;

    complex_input[i] = {input_ptr[real_part_index],
                        input_ptr[complex_part_index]};

    // get the freq_cis: shape 1 * (qProjSize/2) = 1 * 64
    // apply a Cartesian coordinate transformation
    // multiple with input & /copy back to q/k

    // get position of token
    //  int head_idx = i / (num_tokens * proj_size);
    int token_idx =
        (i - head_idx * (num_tokens * proj_size / 2)) / (proj_size / 2);
    // size_t pos = id_map[token_idx].token_position;
    size_t pos = tokenInfos[token_idx].abs_depth_in_request;

    // float before_real = complex_input[i].x, before_complex =
    // complex_input[i].y;

    int pos_i = i % (proj_size / 2);
    float freq = pos * (1.0 / pow(10000.0, (float)2 * pos_i / proj_size));
    hipFloatComplex complex_pos = {cos(freq), sin(freq)};

    complex_input[i] = hipCmulf(complex_input[i], complex_pos);
    input_ptr[real_part_index] = complex_input[i].x;
    input_ptr[real_part_index + 1] = complex_input[i].y;

    // if (i % 64 == 1 && head_idx == 0) {
    //   printf("head id: %d, tokenid: %d, pospospos:->  %d, before real part
    //   %f, "
    //          "before complex part: %f, real part: %f,"
    //          "complext part: %f,  freq_cis real: %f, freq_cis commplexx
    //          %f\n", head_idx, token_idx, pos, before_real, before_complex,
    //          complex_input[i].x,
    //          complex_input[i].y,
    //          complex_pos.x,
    //          complex_pos.y);
    // }
  }
}

void compute_qkv_kernel(IncMultiHeadSelfAttentionMeta const *m,
                        BatchConfig const *bc,
                        float const *input_ptr,
                        float const *weight_ptr,
                        float *output_ptr,
                        float const *bias_ptr,
                        hipStream_t stream) {

  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  float alpha = 1.0f, beta = 0.0f;
  assert(m->qSize == m->vSize && m->qSize == m->kSize);
  hipDataType data_type = ff_to_cuda_datatype(DT_FLOAT);
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = HIP_R_32F;
#endif
  // Compute (W^T)x matmul: einsum(ijkl,im->jmkl)
  // Weights: qSize x qProjSize x 3 x num_heads
  // Input: qSize x num_tokens
  // Output >>> qProjSize x num_tokens x 3 x num_heads
  int m_q = m->qProjSize;
  int m_k = m->kProjSize;
  int m_v = m->vProjSize;
  assert(m_q == m_k && m_k == m_v); // keep things simple for now
  int n = bc->num_active_tokens();
  int k = m->qSize;
  int lda = k, ldb = k, ldc_q = m_q, ldc_k = m_k, ldc_v = m_v;
  size_t strideA =
      m->weights_params; // need to also skip over all the parameters for each
                         // head, plus the unused W_o weights
  size_t strideB = 0;    // input stays the same for all heads.
  size_t strideC =
      (m_q + m_k + m_v) * n; // size of the output block for each head.
  // Q
  checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                       HIPBLAS_OP_T,
                                       HIPBLAS_OP_N,
                                       m_q,
                                       n,
                                       k,
                                       &alpha,
                                       weight_ptr,
                                       data_type,
                                       lda,
                                       strideA,
                                       input_ptr,
                                       data_type,
                                       ldb,
                                       strideB,
                                       &beta,
                                       output_ptr,
                                       data_type,
                                       ldc_q,
                                       strideC,
                                       m->num_heads,
                                       compute_type,
                                       CUBLAS_GEMM_DEFAULT_TENSOR_OP));

  checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                       HIPBLAS_OP_T,
                                       HIPBLAS_OP_N,
                                       m_k,
                                       n,
                                       k,
                                       &alpha,
                                       weight_ptr + m_q * k,
                                       data_type,
                                       lda,
                                       strideA,
                                       input_ptr,
                                       data_type,
                                       ldb,
                                       strideB,
                                       &beta,
                                       output_ptr + m_q * n,
                                       data_type,
                                       ldc_k,
                                       strideC,
                                       m->num_heads,
                                       compute_type,
                                       CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  // V
  checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                       HIPBLAS_OP_T,
                                       HIPBLAS_OP_N,
                                       m_v,
                                       n,
                                       k,
                                       &alpha,
                                       weight_ptr + (m_q + m_k) * k,
                                       data_type,
                                       lda,
                                       strideA,
                                       input_ptr,
                                       data_type,
                                       ldb,
                                       strideB,
                                       &beta,
                                       output_ptr + (m_q + m_k) * n,
                                       data_type,
                                       ldc_v,
                                       strideC,
                                       m->num_heads,
                                       compute_type,
                                       CUBLAS_GEMM_DEFAULT_TENSOR_OP));

  // apply rotary emmmbedding for k and v
  // step1 change the k, v to complex tensor
  int num_tokens = bc->num_active_tokens();
  int parallelism = m->kProjSize * num_tokens * m->num_heads;
  int q_block_size = m->qProjSize * num_tokens;
  int k_block_size = m->kProjSize * num_tokens;
  int v_block_size = m->vProjSize * num_tokens;
  hipFloatComplex *complex_input;

  // apply bias for q, k, v
  if (*m->bias) {
    apply_proj_bias_qkv<<<GET_BLOCKS(parallelism),
                          min(CUDA_NUM_THREADS, parallelism),
                          0,
                          stream>>>(output_ptr,
                                    bias_ptr,
                                    num_tokens,
                                    m->qProjSize,
                                    m->kProjSize,
                                    m->vProjSize,
                                    m->num_heads,
                                    *m->scaling_query,
                                    m->scaling_factor);
  }

  if (*m->apply_rotary_embedding) {
    checkCUDA(hipMalloc(&complex_input,
                         num_tokens * m->qProjSize * m->num_heads *
                             sizeof(hipFloatComplex *) / 2));
    /*q*/
    apply_rotary_embedding<<<GET_BLOCKS(parallelism),
                             min(CUDA_NUM_THREADS, parallelism),
                             0,
                             stream>>>(output_ptr,
                                       complex_input,
                                       m->token_infos,
                                       m->qProjSize,
                                       m->kProjSize,
                                       m->num_heads,
                                       num_tokens,
                                       q_block_size,
                                       k_block_size,
                                       v_block_size,
                                       true);
    /*k*/
    apply_rotary_embedding<<<GET_BLOCKS(parallelism),
                             min(CUDA_NUM_THREADS, parallelism),
                             0,
                             stream>>>(output_ptr,
                                       complex_input,
                                       m->token_infos,
                                       m->qProjSize,
                                       m->kProjSize,
                                       m->num_heads,
                                       num_tokens,
                                       q_block_size,
                                       k_block_size,
                                       v_block_size,
                                       false);
  }
}

__global__ void store_kv_cache(float const *devQKVProjArray,
                               float *cache_ptr,
                               BatchConfig::PerTokenInfo const *tokenInfos,
                               int qProjSize,
                               int kProjSize,
                               int vProjSize,
                               int num_tokens,
                               int num_heads,
                               int max_seq_len,
                               bool k_cache) {
  CUDA_KERNEL_LOOP(i,
                   num_tokens * (k_cache ? kProjSize : vProjSize) * num_heads) {
    int proj_size = k_cache ? kProjSize : vProjSize;
    int head_idx = i / (num_tokens * proj_size);
    int token_idx = (i - head_idx * (num_tokens * proj_size)) / proj_size;
    int data_idx = i % proj_size;

    int qkv_block_size = (qProjSize + kProjSize + vProjSize) * num_tokens;
    int current_head_block_size =
        num_tokens * (k_cache ? qProjSize : qProjSize + kProjSize);
    float val =
        devQKVProjArray[head_idx * qkv_block_size + current_head_block_size +
                        token_idx * proj_size + data_idx];
    // int const req_id = id_map[token_idx].request_index;
    // int const tok_id = id_map[token_idx].token_position;
    int const req_id = tokenInfos[token_idx].request_index;
    int const tok_id = tokenInfos[token_idx].abs_depth_in_request;

    cache_ptr[req_id * (num_heads * max_seq_len * proj_size) +
              head_idx * (max_seq_len * proj_size) + tok_id * proj_size +
              data_idx] = val;
  }
}

void update_kv_cache_kernel(IncMultiHeadSelfAttentionMeta const *m,
                            BatchConfig const *bc,
                            hipStream_t stream) {
  int num_tokens = bc->num_active_tokens();
  if (num_tokens > 0) {
    int parallelism = m->kProjSize * num_tokens * m->num_heads;
    store_kv_cache<<<GET_BLOCKS(parallelism),
                     min(CUDA_NUM_THREADS, parallelism),
                     0,
                     stream>>>(m->devQKVProjArray,
                               m->keyCache,
                               m->token_infos,
                               m->qProjSize,
                               m->kProjSize,
                               m->vProjSize,
                               num_tokens,
                               m->num_heads,
                               BatchConfig::MAX_SEQ_LENGTH,
                               /* k_cache = */ true);

    parallelism = m->vProjSize * num_tokens * m->num_heads;
    store_kv_cache<<<GET_BLOCKS(parallelism),
                     min(CUDA_NUM_THREADS, parallelism),
                     0,
                     stream>>>(m->devQKVProjArray,
                               m->valueCache,
                               m->token_infos,
                               m->qProjSize,
                               m->kProjSize,
                               m->vProjSize,
                               num_tokens,
                               m->num_heads,
                               BatchConfig::MAX_SEQ_LENGTH,
                               /* k_cache = */ false);
  }
}

__global__ void fill_entries_above_diagonal(float *matrix,
                                            size_t num_rows,
                                            size_t num_cols,
                                            size_t num_heads,
                                            size_t entries_above_diagonal,
                                            float value) {
  CUDA_KERNEL_LOOP(i, entries_above_diagonal * num_heads) {
    size_t head_idx = i / entries_above_diagonal;
    size_t entry_idx = i % entries_above_diagonal;
    size_t y = (-1 + sqrt(8 * (float)entry_idx + 1)) / 2;
    size_t x = entry_idx - y * (y + 1) / 2;
    y += (num_cols - num_rows) + 1;
    matrix[head_idx * num_rows * num_cols + num_cols * y + x] = value;
  }
}

void compute_attention_kernel(IncMultiHeadSelfAttentionMeta const *m,
                              BatchConfig const *bc,
                              float *output_ptr,
                              float const *bias_ptr,
                              hipStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  hipDataType cublas_data_type = ff_to_cuda_datatype(DT_FLOAT);
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = HIP_R_32F;
#endif
  // int num_requests = bc->num_active_requests();
  int num_tokens = bc->num_active_tokens();
  int tokens_previous_requests = 0;
  int qkv_block_size =
      (m->qProjSize + m->kProjSize + m->vProjSize) * num_tokens;
  int kt_block_size = m->kProjSize * BatchConfig::MAX_SEQ_LENGTH;
  int kt_req_block_size = kt_block_size * m->num_heads;
  int vt_block_size = m->vProjSize * BatchConfig::MAX_SEQ_LENGTH;
  int vt_req_block_size = vt_block_size * m->num_heads;
  assert(m->qProjSize == m->kProjSize);

  for (int i = 0; i < bc->MAX_NUM_REQUESTS; i++) {
    if (bc->request_completed[i]) {
      continue;
    }
    int num_new_tokens = bc->requestsInfo[i].num_tokens_in_batch;
    int total_tokens = bc->requestsInfo[i].token_start_offset +
                       bc->requestsInfo[i].num_tokens_in_batch;
    // bc->token_last_available_idx[i] + 1;
    // Compute (QK^T/sqrt(d_k))
    int m_ = num_new_tokens;
    int n = total_tokens;
    int k = m->qProjSize;
    int lda = k, ldb = k, ldc = m_;
    int strideA = qkv_block_size;
    int strideB = kt_block_size;
    int strideC = num_new_tokens * total_tokens;

    // a flag of using this scaling alpha
    float alpha = 1.0f, beta = 0.0f;
    if (*m->qk_prod_scaling) {
      alpha = 1.0f / (float)sqrt(m->kProjSize), beta = 0.0f;
    }
    // To get A, skip over Q entries from previous requests (same head)
    void const *A = (void const *)(m->devQKVProjArray +
                                   tokens_previous_requests * m->qProjSize);
    // To get B, skip over K entries from previous requests (all heads +
    // padding)
    void const *B = (void const *)(m->keyCache + i * kt_req_block_size);
    // To get C, skip over QK^T products from previous requests
    void *C = (void *)(m->qk_prods);

    checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                         HIPBLAS_OP_T,
                                         HIPBLAS_OP_N,
                                         m_,
                                         n,
                                         k,
                                         &alpha,
                                         A,
                                         cublas_data_type,
                                         lda,
                                         strideA,
                                         B,
                                         cublas_data_type,
                                         ldb,
                                         strideB,
                                         &beta,
                                         C,
                                         cublas_data_type,
                                         ldc,
                                         strideC,
                                         m->num_heads,
                                         compute_type,
                                         CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    // Fill all elements above diagonal in qk prods with -inf to force
    // causal attention.
    assert(num_new_tokens <= total_tokens);
    size_t entries_above_diagonal = num_new_tokens * (num_new_tokens - 1) / 2;
    if (entries_above_diagonal > 0) {
      size_t parallelism = m->num_heads * entries_above_diagonal;
      fill_entries_above_diagonal<<<GET_BLOCKS(parallelism),
                                    min((size_t)CUDA_NUM_THREADS, parallelism),
                                    0,
                                    stream>>>((float *)C,
                                              num_new_tokens,
                                              total_tokens,
                                              m->num_heads,
                                              entries_above_diagonal,
                                              -INFINITY);
    }
    // Compute Softmax(QK^T/sqrt(d_k))
    hipdnnTensorDescriptor_t qk_tensor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&qk_tensor));
    // Before modifying the parameters below, make sure to read the following
    // description of the HIPDNN_TENSOR_NCHW tensor layout, from
    // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnTensorFormat_t:
    // This tensor format specifies that the data is laid out in the following
    // order: batch size, feature maps, rows, columns. The strides are
    // implicitly defined in such a way that the data are contiguous in memory
    // with no padding between images, feature maps, rows, and columns; the
    // columns are the inner dimension and the images are the outermost
    // dimension.
    int n_param = m->num_heads;
    int c_param = total_tokens;
    int h_param = 1;
    int w_param = num_new_tokens;
    checkCUDNN(hipdnnSetTensor4dDescriptor(qk_tensor,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          n_param,
                                          c_param,
                                          h_param,
                                          w_param));
    alpha = 1.0f, beta = 0.0f;
    void *C_softmax = (void *)(m->qk_prods_softmax);
    // The softmax operation below is executed according to the
    // HIPDNN_SOFTMAX_MODE_CHANNEL, which is also described in the docs: The
    // softmax operation is computed per spatial location (H,W) per image (N)
    // across dimension C.
    checkCUDNN(hipdnnSoftmaxForward(m->handle.dnn,
                                   HIPDNN_SOFTMAX_ACCURATE,
                                   HIPDNN_SOFTMAX_MODE_CHANNEL,
                                   &alpha,
                                   qk_tensor,
                                   (void *)((float *)C),
                                   &beta,
                                   qk_tensor,
                                   (void *)((float *)C_softmax)));
    // Matmul softmax(QK^T/sqrt(d_k)) by V
    alpha = 1.0f, beta = 0.0f;
    m_ = num_new_tokens;
    n = m->vProjSize;
    k = total_tokens;
    lda = m_, ldb = n, ldc = m_;
    strideA = num_new_tokens * total_tokens;
    strideB = vt_block_size;
    strideC = num_new_tokens * m->vProjSize;
    // To get A, skip over softmax(QK^T/sqrt(d_k)) entries from previous
    // requests (all heads)
    A = (void const *)C_softmax;
    // To get B, skip over V^T entries from previous requests (all heads +
    // padding)
    B = (void const *)(m->valueCache + i * vt_req_block_size);
    // To get C, skip over softmax(QK^T/sqrt(d_k))V products from previous
    // requests
    C = (void *)(m->attn_heads +
                 tokens_previous_requests * m->num_heads * m->vProjSize);

    checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                         HIPBLAS_OP_N,
                                         HIPBLAS_OP_T,
                                         m_,
                                         n,
                                         k,
                                         &alpha,
                                         A,
                                         cublas_data_type,
                                         lda,
                                         strideA,
                                         B,
                                         cublas_data_type,
                                         ldb,
                                         strideB,
                                         &beta,
                                         C,
                                         cublas_data_type,
                                         ldc,
                                         strideC,
                                         m->num_heads,
                                         compute_type,
                                         CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // Project to output, save result directly on output tensor
    alpha = 1.0f, beta = 0.0f;
    m_ = m->oProjSize;
    k = m->vProjSize * m->num_heads;
    n = num_new_tokens;
    lda = k, ldb = n, ldc = m_;
    A = (void const *)m->W_out_contiguous;
    B = (void const *)C;
    C = (void *)(output_ptr + tokens_previous_requests * m->oProjSize);

    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_T,
                           HIPBLAS_OP_T,
                           m_,
                           n,
                           k,
                           &alpha,
                           A,
                           cublas_data_type,
                           lda,
                           B,
                           cublas_data_type,
                           ldb,
                           &beta,
                           C,
                           cublas_data_type,
                           ldc,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    tokens_previous_requests += num_new_tokens;
  }

  if (*m->bias) {
    int parallelism = m->oProjSize * num_tokens;
    apply_proj_bias_w<<<GET_BLOCKS(parallelism),
                        min(CUDA_NUM_THREADS, parallelism),
                        0,
                        stream>>>(
        output_ptr, bias_ptr, num_tokens, m->oProjSize);
  }

  assert(tokens_previous_requests == num_tokens);
}

/*static*/
void IncMultiHeadSelfAttention::inference_kernel_wrapper(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    float const *input_ptr,
    float const *weight_ptr,
    float *output_ptr,
    float const *bias_ptr) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  // reload the weight_o

  if (!(*m->has_load_weights)) {
    int parallelism = m->vProjSize * m->oProjSize * m->num_heads;
    build_w_out_tensor<<<GET_BLOCKS(parallelism),
                         min(CUDA_NUM_THREADS, parallelism),
                         0,
                         stream>>>(weight_ptr,
                                   m->W_out_contiguous,
                                   m->vProjSize,
                                   m->oProjSize,
                                   m->num_heads,
                                   (m->qSize * m->qProjSize +
                                    m->kSize * m->kProjSize +
                                    m->vSize * m->vProjSize));
    *m->has_load_weights = true;
  }
  // here because we need postion info in infernece 1
  hipMemcpyAsync(m->token_infos,
                  &(bc->tokensInfo),
                  bc->MAX_NUM_TOKENS * sizeof(BatchConfig::PerTokenInfo),
                  hipMemcpyHostToDevice,
                  stream);
  // phase 1: Implement kernel to compute KQV for input tokens
  compute_qkv_kernel(
      m, bc, input_ptr, weight_ptr, m->devQKVProjArray, bias_ptr, stream);

  // phase 2: Update key/val cache
  update_kv_cache_kernel(m, bc, stream);

  // phase 3: Compute attention score
  // 3 kernels for pahse 3: matmul1 - softmax - matmal2
  compute_attention_kernel(m, bc, output_ptr, bias_ptr, stream);

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("IncMultiHeadSelfAttention forward time = %.2fms\n", elapsed);
    // print_tensor<3, float>(acc_query.ptr, acc_query.rect,
    // "[Attention:forward:query]"); print_tensor<3, float>(acc_output.ptr,
    // acc_output.rect, "[Attention:forward:output]");
  }
}

IncMultiHeadSelfAttentionMeta::IncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    IncMultiHeadSelfAttention const *attn,
    float const *weight_ptr,
    Memory gpu_mem,
    int num_samples,
    int _num_heads)
    : OpMeta(handler, attn) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  checkCUDNN(hipdnnSetStream(handler.dnn, stream));

  qSize = attn->qSize;
  kSize = attn->kSize;
  vSize = attn->vSize;
  // assume dimensions match for now
  assert(qSize == kSize);
  assert(kSize == vSize);
  qProjSize = attn->qProjSize;
  kProjSize = attn->kProjSize;
  assert(qProjSize == kProjSize); // required for attention QK^T matmul
  vProjSize = attn->vProjSize;
  oProjSize = attn->oProjSize;

  num_heads = _num_heads;
  weights_params = (qSize * qProjSize + kSize * kProjSize + vSize * vProjSize +
                    oProjSize * (vProjSize > 0 ? vProjSize : vSize));
  weightSize = weights_params * num_heads * sizeof(float);
  has_load_weights = (bool *)calloc(1, sizeof(bool));
  *has_load_weights = false;
  apply_rotary_embedding = (bool *)calloc(1, sizeof(bool));
  *apply_rotary_embedding = attn->apply_rotary_embedding;
  bias = (bool *)calloc(1, sizeof(bool));
  *bias = attn->bias;
  scaling_query = (bool *)calloc(1, sizeof(bool));
  *scaling_query = attn->scaling_query;
  scaling_factor = attn->scaling_factor;
  qk_prod_scaling = (bool *)calloc(1, sizeof(bool));
  *qk_prod_scaling = attn->qk_prod_scaling;
  // Currently do not support adding bias to key/value projection
  assert(!attn->add_bias_kv);

#ifdef INFERENCE_TESTS
  kcache = (float *)calloc(kProjSize * BatchConfig::MAX_SEQ_LENGTH * num_heads *
                               BatchConfig::MAX_NUM_REQUESTS,
                           sizeof(float));
  vcache = (float *)calloc(vProjSize * BatchConfig::MAX_SEQ_LENGTH * num_heads *
                               BatchConfig::MAX_NUM_REQUESTS,
                           sizeof(float));
#endif

  // allocate memory for the seqArray and reserve space
  {
    size_t qkv_proj_dim = qProjSize + kProjSize + vProjSize;
    size_t qkv_max_proj_size =
        BatchConfig::MAX_NUM_TOKENS * qkv_proj_dim * num_heads;
    size_t key_cache_size = num_heads * kProjSize *
                            BatchConfig::MAX_NUM_REQUESTS *
                            BatchConfig::MAX_SEQ_LENGTH;
    size_t value_cache_size = num_heads * vProjSize *
                              BatchConfig::MAX_NUM_REQUESTS *
                              BatchConfig::MAX_SEQ_LENGTH;
    size_t tokeninfo_size = BatchConfig::MAX_NUM_TOKENS;
    size_t qk_prod_size =
        BatchConfig::MAX_NUM_TOKENS * BatchConfig::MAX_SEQ_LENGTH * num_heads;
    size_t attn_heads_size =
        BatchConfig::MAX_NUM_TOKENS * num_heads * vProjSize;
    size_t W_out_block_size = oProjSize * (vProjSize > 0 ? vProjSize : vSize);
    size_t W_out_contiguous_size = W_out_block_size * num_heads;
    size_t totalSize =
        (qkv_max_proj_size + key_cache_size + value_cache_size +
         2 * qk_prod_size + attn_heads_size + W_out_contiguous_size) *
            sizeof(float) +
        tokeninfo_size *
            sizeof(BatchConfig::PerTokenInfo); // more components will
                                               // be added here later

    Realm::Rect<1, coord_t> bounds(Realm::Point<1, coord_t>(0),
                                   Realm::Point<1, coord_t>(totalSize - 1));
    std::vector<size_t> field_sizes;
    field_sizes.push_back(sizeof(char));
    Realm::RegionInstance::create_instance(reserveInst,
                                           gpu_mem,
                                           bounds,
                                           field_sizes,
                                           0,
                                           Realm::ProfilingRequestSet())
        .wait();
    devQKVProjArray = (float *)reserveInst.pointer_untyped(0, sizeof(char));
    keyCache = (float *)devQKVProjArray + qkv_max_proj_size;
    valueCache = (float *)keyCache + key_cache_size;
    token_infos = (BatchConfig::PerTokenInfo *)(valueCache + value_cache_size);
    qk_prods = (float *)(token_infos + tokeninfo_size);
    qk_prods_softmax = (float *)(qk_prods + qk_prod_size);
    attn_heads = (float *)qk_prods_softmax + qk_prod_size;
    W_out_contiguous = (float *)attn_heads + attn_heads_size;
    int parallelism = vProjSize * oProjSize * num_heads;
    build_w_out_tensor<<<GET_BLOCKS(parallelism),
                         min(CUDA_NUM_THREADS, parallelism),
                         0,
                         stream>>>(
        weight_ptr,
        W_out_contiguous,
        vProjSize,
        oProjSize,
        num_heads,
        (qSize * qProjSize + kSize * kProjSize + vSize * vProjSize));
  }

  hipStreamSynchronize(stream);
}

IncMultiHeadSelfAttentionMeta::~IncMultiHeadSelfAttentionMeta(void) {
  reserveInst.destroy();
#ifdef INFERENCE_TESTS
  free(kcache);
  free(vcache);
#endif
}

}; // namespace FlexFlow
