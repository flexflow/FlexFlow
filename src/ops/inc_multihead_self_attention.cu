#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/inc_multihead_self_attention.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

// declare Legion names
using Legion::coord_t;
using Legion::Memory;

/*static*/
void IncMultiHeadSelfAttention::inference_kernel1(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    float const *input_ptr,
    float const *weight_ptr,
    float *output_ptr,
    hipStream_t stream) {

  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));

  float alpha = 1.0f, beta = 0.0f;
  int out_dim = (m->qProjSize + m->kProjSize + m->vProjSize) * m->num_heads;
  int in_dim = m->qSize;
  assert(in_dim == m->vSize && in_dim == m->kSize);
  hipDataType data_type = ff_to_cuda_datatype(DT_FLOAT);
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = HIP_R_32F;
#endif
  checkCUDA(hipblasGemmEx(m->handle.blas,
                         HIPBLAS_OP_T,
                         HIPBLAS_OP_N,
                         out_dim,
                         bc->num_tokens,
                         in_dim,
                         &alpha,
                         weight_ptr,
                         data_type,
                         in_dim,
                         input_ptr,
                         data_type,
                         in_dim,
                         &beta,
                         output_ptr,
                         data_type,
                         out_dim,
                         compute_type,
                         CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

__global__ void store_kv_cache(float const *input_ptr,
                               float const *cache_ptr,
                               request_token_id const *id_map,
                               int max_seq_len,
                               int hid_dim) {
  int const token_idx = blockIdx.x;
  int const element_idx = threadIdx.x;
  int const req_id = id_map[token_idx].request_id;
  int const tok_id = id_map[token_idx].token_id;
  memcpy((float *)input_ptr + token_idx * hid_dim + element_idx,
         (float *)cache_ptr + (req_id * max_seq_len + tok_id) * hid_dim +
             element_idx,
         sizeof(float));
}

/*static*/
void IncMultiHeadSelfAttention::inference_kernel2(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    float const *input_ptr,
    request_token_id const *id_map,
    hipStream_t stream) {
  store_kv_cache<<<bc->num_tokens, m->kProjSize>>>(
      (float *)input_ptr + bc->MAX_NUM_TOKENS * m->qProjSize,
      m->keyCache,
      id_map,
      bc->MAX_SEQUENCE_LENGTH,
      m->kProjSize);
  store_kv_cache<<<bc->num_tokens, m->vProjSize>>>(
      (float *)input_ptr + bc->MAX_NUM_TOKENS * (m->qProjSize + m->kProjSize),
      m->valueCache,
      id_map,
      bc->MAX_SEQUENCE_LENGTH,
      m->vProjSize);
}

/*static*/
void IncMultiHeadSelfAttention::inference_kernel_wrapper(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    float const *input_ptr,
    float const *weight_ptr,
    float *output_ptr) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  // phase 0: convert BatchConfig representation to {rid, tid} struct
  int curr_token_idx = 0;
  int processed_requests = 0;
  printf("Start phase 0: num_tokens: %d, num_requests: %d\n",
         bc->num_tokens,
         bc->num_requests);
  printf("num_active_tokens: %d, num_active_requests: %d\n",
         bc->num_active_tokens(),
         bc->num_active_requests());

  for (uint16_t curr_request_idx = 0; curr_request_idx < bc->MAX_NUM_REQUESTS;
       curr_request_idx++) {
    // printf("request %u: num_tokens: %d, start_idx: %d\n",
    //        curr_request_idx,
    //        bc->num_processing_tokens[curr_request_idx],
    //        bc->token_start_idx[curr_request_idx]);

    if (bc->num_processing_tokens[curr_request_idx] == 0) {
      continue;
    }

    for (int i = 0; i < bc->num_processing_tokens[curr_request_idx]; i++) {
      m->input_token_ids[curr_token_idx].request_id = curr_request_idx;
      m->input_token_ids[curr_token_idx].token_id =
          bc->token_start_idx[curr_request_idx] + i;

      // printf("token %d: request_id: %u, token_id: %u\n",
      //        curr_token_idx,
      //        m->input_token_ids[curr_token_idx].request_id,
      //        m->input_token_ids[curr_token_idx].token_id);
      if (curr_token_idx >= bc->num_active_tokens()) {
        printf("curr_token_idx: %d, curr_request_idx: %d\n",
               curr_token_idx,
               curr_request_idx);
        assert(false); // total number of tokens should matches the batch config
      }
      curr_token_idx += 1;
    }
    processed_requests += 1;
  }
  printf("End of phase 0: curr_token_idx: %d, processed_requests: %d\n",
         curr_token_idx,
         processed_requests);

  assert(processed_requests == bc->num_active_requests());

  // phase 1: Implement kernel to compute KQV for input tokens
  IncMultiHeadSelfAttention::inference_kernel1(
      m, bc, input_ptr, weight_ptr, m->devQKVProjArray, stream);

  // phase 2: Update key/val cache
  IncMultiHeadSelfAttention::inference_kernel2(
      m, bc, m->devQKVProjArray, m->input_token_ids, stream);

  // phase 3: Compute attention score
  // 3 kernels for pahse 3: matmul1 - softmax - matmal2

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("IncMultiHeadSelfAttention forward time = %.2fms\n", elapsed);
    // print_tensor<3, float>(acc_query.ptr, acc_query.rect,
    // "[Attention:forward:query]"); print_tensor<3, float>(acc_output.ptr,
    // acc_output.rect, "[Attention:forward:output]");
  }
}

IncMultiHeadSelfAttentionMeta::IncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    IncMultiHeadSelfAttention const *attn,
    BatchConfig const *bc,
    Memory gpu_mem,
    int num_samples,
    int _num_heads)
    : OpMeta(handler, attn) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  // checkCUDNN(hipdnnSetStream(handler.dnn, stream));

  qSize = attn->qSize;
  kSize = attn->kSize;
  vSize = attn->vSize;
  // assume dimensions match for now
  assert(qSize == kSize);
  assert(kSize == vSize);
  qProjSize = attn->qProjSize;
  kProjSize = attn->kProjSize;
  vProjSize = attn->vProjSize;
  oProjSize = attn->oProjSize;
  num_heads = _num_heads;
  weightSize = (qSize * qProjSize + kSize * kProjSize + vSize * vProjSize +
                oProjSize * (vProjSize > 0 ? vProjSize : vSize)) *
               num_heads * sizeof(float);

  // Currently do not support adding bias to key/value projection
  assert(!attn->add_bias_kv);

  // allocate memory for the seqArray and reserve space
  {
    // size_t totalSize = reserveSpaceSize + sizeof(int) * num_samples * 2 +
    // bc->MAX_NUM_REQUESTS *bc-> MAX_SEQUENCE_LENGTH * sizeof(int); size_t
    // max_num_tokens = bc->MAX_NUM_REQUESTS * bc->MAX_SEQUENCE_LENGTH;
    size_t qkv_proj_dim = qProjSize + kProjSize + vProjSize;
    size_t qkv_max_proj_size = num_samples * qkv_proj_dim * num_heads;
    size_t key_cache_size =
        kProjSize * bc->MAX_NUM_REQUESTS * bc->MAX_SEQUENCE_LENGTH;
    size_t value_cache_size =
        vProjSize * bc->MAX_NUM_REQUESTS * bc->MAX_SEQUENCE_LENGTH;

    size_t totalSize =
        (qkv_max_proj_size + key_cache_size + value_cache_size) *
        sizeof(float); // more components will be added here later

    Realm::Rect<1, coord_t> bounds(Realm::Point<1, coord_t>(0),
                                   Realm::Point<1, coord_t>(totalSize - 1));
    std::vector<size_t> field_sizes;
    field_sizes.push_back(sizeof(char));
    Realm::RegionInstance::create_instance(reserveInst,
                                           gpu_mem,
                                           bounds,
                                           field_sizes,
                                           0,
                                           Realm::ProfilingRequestSet())
        .wait();
    devQKVProjArray = (float *)reserveInst.pointer_untyped(0, sizeof(char));
    keyCache = (float *)devQKVProjArray + qkv_max_proj_size;
    valueCache = (float *)keyCache + key_cache_size;
    // checkCUDA(hipMemcpy(devQoSeqArray,
    //                      qoSeqArray,
    //                      sizeof(int) * num_samples,
    //                      hipMemcpyHostToDevice));
    // devKvSeqArray = (int *)devQoSeqArray + num_samples;
    // checkCUDA(hipMemcpy(devKvSeqArray,
    //                      kvSeqArray,
    //                      sizeof(int) * num_samples,
    //                      hipMemcpyHostToDevice));
    // kvCache = (int *)devKvSeqArray + num_samples;
    // reserveSpace = (int *)kvCache + bc->MAX_NUM_REQUESTS * bc->
    // MAX_SEQUENCE_LENGTH;
  }

  input_token_ids = new request_token_id[bc->MAX_NUM_TOKENS];
}

IncMultiHeadSelfAttentionMeta::~IncMultiHeadSelfAttentionMeta(void) {
  reserveInst.destroy();
}

//__global__ void store_kv_cache(
//    float const *input_ptr, float const *cache_ptr, request_token_id const
//    *id_map, int max_seq_len, int hid_dim) {
//  int const token_idx = blockIdx.x;
//  int const element_idx = threadIdx.x;
//  int const req_id = id_map[token_idx].request_id;
//  int const tok_id = id_map[token_idx].token_id;
//  cache_ptr[(req_id * max_seq_len + tok_id) * hid_dim + element_idx] =
//  input_ptr[token_idx * hid_dim + element_idx];
//}

}; // namespace FlexFlow
