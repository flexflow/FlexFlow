#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#if defined(FF_USE_CUDA) || defined(FF_USE_HIP_CUDA)
#include "hip/hip_complex.h"
#endif
#include "flexflow/ffconst_utils.h"
#include "flexflow/ops/inc_multihead_self_attention.h"
#include "flexflow/ops/kernels/decompress_kernels.h"
#include "flexflow/ops/kernels/inc_multihead_self_attention_kernels.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

// declare Legion names
using Legion::coord_t;
using Legion::Memory;

namespace Kernels {
namespace IncMultiHeadAttention {

// get continous key and value from KV cache.
// KV cache layout: num_tokens * projSize * num_kv_heads * req_num
// layout kProjSize/vProjSize * max_total_tokens * num_kv_heads * req_num
// key and value is also padded to max_total_tokens
// all values exceed the total_tokens is 0

template <typename DT>
__global__ void get_key_value(DT *key,
                              DT *value,
                              DT *kCache_ptr,
                              DT *vCache_ptr,
                              int *total_tokens_per_req,
                              int kProjSize,
                              int vProjSize,
                              int max_total_tokens,
                              int max_total_tokens_all,
                              int num_kv_heads,
                              int max_seq_len) {
  CUDA_KERNEL_LOOP(i, max_total_tokens_all * kProjSize * num_kv_heads) {
    int token_id = i / (kProjSize * num_kv_heads);
    int req_id = token_id / max_total_tokens;
    int offset = i % kProjSize;
    int head_idx = (i - token_id * kProjSize * num_kv_heads) / kProjSize;

    int token_id_in_req = token_id % max_total_tokens;

    // which req it belongs to
    int cache_idx = req_id * num_kv_heads * max_seq_len * kProjSize +
                    head_idx * (max_seq_len * kProjSize) +
                    token_id_in_req * kProjSize + offset;

    int kv_idx = req_id * num_kv_heads * max_total_tokens * kProjSize +
                 head_idx * (max_total_tokens * kProjSize) +
                 token_id_in_req * kProjSize + offset;
    if (token_id_in_req >= total_tokens_per_req[req_id]) {
      key[kv_idx] = 0;
      value[kv_idx] = 0;
    } else {
      key[kv_idx] = kCache_ptr[cache_idx];
      value[kv_idx] = vCache_ptr[cache_idx];
    }
  }
}

// only used by MPT model. https://arxiv.org/abs/2108.12409
template <typename DT>
__global__ void apply_position_bias_qkprd(DT *input_ptr,
                                          int num_tokens,
                                          int num_total_tokens,
                                          int num_heads,
                                          int global_num_q_heads,
                                          int shard_id) {
  CUDA_KERNEL_LOOP(i, num_tokens * num_total_tokens * num_heads) {
    // get head_idx,
    int head_idx = i / (num_tokens * num_total_tokens) + (num_heads * shard_id);
    int position_idx = (i / num_tokens) % num_total_tokens;
    position_idx = position_idx + 1 - num_total_tokens;
    // 8 is alibi_bias_max in
    // https://huggingface.co/mosaicml/mpt-30b/blob/main/config.json
    float base = (float)(head_idx + 1) * 8 / global_num_q_heads;
    float slopes = 1.0 / pow(2, base);
    // if(i == 0){
    //   printf("see position: %d, %f, %f, %f\n", position_idx, base, slopes,
    //   position_idx * slopes);
    // }
    input_ptr[i] += static_cast<DT>(position_idx * slopes);
  }
}

template <typename DT>
__global__ void apply_proj_bias_w(DT *input_ptr,
                                  DT const *bias_ptr,
                                  int num_tokens,
                                  int qkv_weight_size,
                                  int oProjSize) {
  CUDA_KERNEL_LOOP(i, num_tokens * oProjSize) {
    int bias_idx = qkv_weight_size + i % oProjSize;
    input_ptr[i] += bias_ptr[bias_idx];
  }
}

template <typename DT>
__global__ void copy_output(DT const *padded_output,
                            DT *output,
                            int num_total_tokens,
                            int oProjSize,
                            int *real_token_idx) {
  CUDA_KERNEL_LOOP(i, num_total_tokens * oProjSize) {
    int token_id = i / oProjSize;
    int offset = i % oProjSize;
    int real_idx = real_token_idx[token_id];
    if (real_idx >= 0) {
      output[real_idx * oProjSize + offset] = padded_output[i];
    }
  }
}

template <typename DT>
__global__ void pad_input_ptr(DT const *input_ptr,
                              DT *padded_input,
                              BatchConfig const *bc,
                              int num_padded_tokens,
                              int hidden_size,
                              int max_length) {
  CUDA_KERNEL_LOOP(i, num_padded_tokens * hidden_size) {
    int token_id = (i / hidden_size);
    int req_id = token_id / max_length;
    // if (bc->requestsInfo[req_id].num_tokens_in_batch > token_id) {
    //   padded_input[i] = input_ptr[i];
    // } else {
    //   padded_input[i] = (DT)0;
    // }
    padded_input[i] = input_ptr[i];
    // if(i == 0){
    //   printf("?? %d, %d, %d\n", token_id, req_id,
    //   bc->requestsInfo[req_id].num_tokens_in_batch);
    // }
  }
}

template <typename DT>
__global__ void apply_proj_bias_qkv(DT *input_ptr,
                                    DT const *bias_ptr,
                                    int shard_id,
                                    int num_tokens,
                                    int qProjSize,
                                    int kProjSize,
                                    int vProjSize,
                                    int global_num_q_heads,
                                    int global_num_kv_heads,
                                    int num_q_heads,
                                    int num_kv_heads,
                                    bool scaling_query,
                                    float scaling_factor,
                                    int *real_token_idx) {
  CUDA_KERNEL_LOOP(i,
                   num_tokens *
                       (qProjSize * num_q_heads + kProjSize * num_kv_heads +
                        vProjSize * num_kv_heads)) {
    // for simplicity, assume q, k, v is in same shape
    // 0->q, 1->k, 2->v
    // int qkv_index = i / (num_tokens * qProjSize) % 3;
    int token_idx = 0;
    int qkv_index = i < num_tokens * qProjSize * num_q_heads
                        ? 0
                        : (i < num_tokens * (qProjSize * num_q_heads +
                                             kProjSize * num_kv_heads)
                               ? 1
                               : 2);

    // int head_idx = i / (num_tokens * (qProjSize + kProjSize + vProjSize));
    // int qkv_block_size = (qProjSize + kProjSize + vProjSize) * num_tokens;
    int q_block_size = qProjSize * num_tokens * num_q_heads;
    int k_block_size = kProjSize * num_tokens * num_kv_heads;

    // int idx = i % (num_tokens * (qProjSize));

    // int real_part_index =
    //     head_idx * qkv_block_size + qkv_index * q_block_size + idx;
    int bias_idx = 0;
    if (qkv_index == 0) {
      int head_idx = i / (num_tokens * qProjSize);
      int global_head_idx = head_idx + shard_id * num_q_heads;
      int global_i = i + shard_id * num_q_heads * num_tokens * qProjSize;
      bias_idx = global_head_idx * qProjSize +
                 (global_i % (num_tokens * (qProjSize)) % qProjSize);
      token_idx = (i - head_idx * (num_tokens * qProjSize)) / qProjSize;
    } else {

      int idx =
          qkv_index == 1 ? i - q_block_size : i - q_block_size - k_block_size;
      int pre_length = qkv_index == 1 ? qProjSize * global_num_q_heads
                                      : qProjSize * global_num_q_heads +
                                            kProjSize * global_num_kv_heads;

      int head_idx = idx / (num_tokens * kProjSize);
      int global_head_idx = head_idx + shard_id * num_kv_heads;
      int global_idx = idx + shard_id * num_tokens * num_kv_heads * kProjSize;

      bias_idx = pre_length + global_head_idx * kProjSize +
                 (global_idx % (num_tokens * (qProjSize)) % qProjSize);
      token_idx =
          (i - pre_length - head_idx * (num_tokens * kProjSize)) / kProjSize;
    }
    // int bias_idx = qkv_index * qProjSize * global_num_q_heads +
    //                global_head_idx * qProjSize + (idx % qProjSize);

    // if is a padded token, do nothing.
    input_ptr[i] +=
        (real_token_idx[token_idx] >= 0 ? bias_ptr[bias_idx] : (DT)0);

    if (scaling_query && qkv_index == 0) {
      input_ptr[i] *= scaling_factor;
    }
  }
}

template <typename DT>
__global__ void scaling_query_kernel(DT *input_ptr,
                                     int qProjSize,
                                     int num_tokens,
                                     int num_q_heads,
                                     float scaling_factor) {
  CUDA_KERNEL_LOOP(i, num_tokens * (qProjSize * num_q_heads)) {
    input_ptr[i] *= scaling_factor;
  }
}

template <typename DT>
__global__ void
    apply_rotary_embedding_native(DT *input_ptr,
                                  hipFloatComplex *complex_input,
                                  BatchConfig::PerTokenInfo const *tokenInfos,
                                  int qProjSize,
                                  int kProjSize,
                                  int num_q_heads,
                                  int num_tokens,
                                  int num_kv_heads,
                                  int q_block_size,
                                  int k_block_size,
                                  int q_array_size) {
  CUDA_KERNEL_LOOP(
      i,
      num_tokens * (qProjSize * num_q_heads + kProjSize * num_kv_heads) / 2) {
    // create complex number
    bool q_tensor = i < (q_array_size / 2);
    int proj_size = q_tensor ? qProjSize : kProjSize;
    int real_i = q_tensor ? i : i - q_array_size / 2;

    int head_idx = real_i / (num_tokens * proj_size / 2);
    int idx = real_i % (num_tokens * proj_size / 2);
    int real_part_index = idx * 2 +
                          head_idx * (q_tensor ? q_block_size : k_block_size) +
                          (q_tensor ? 0 : q_array_size);

    int complex_part_index = real_part_index + 1;

    complex_input[i] = {input_ptr[real_part_index],
                        input_ptr[complex_part_index]};

    int token_idx =
        (real_i - head_idx * (num_tokens * proj_size / 2)) / (proj_size / 2);
    size_t pos = tokenInfos[token_idx].abs_depth_in_request;

    // float before_real = complex_input[i].x, before_complex =
    // complex_input[i].y;

    int pos_i = real_i % (proj_size / 2);
    float freq = pos * (1.0 / pow(10000.0, (float)2 * pos_i / proj_size));
    hipFloatComplex complex_pos = {cos(freq), sin(freq)};

    complex_input[i] = hipCmulf(complex_input[i], complex_pos);
    input_ptr[real_part_index] = complex_input[i].x;
    input_ptr[complex_part_index] = complex_input[i].y;
  }
}

template <typename DT>
__global__ void
    apply_rotary_embedding_hf(DT *input_ptr,
                              hipFloatComplex *complex_input,
                              BatchConfig::PerTokenInfo const *tokenInfos,
                              int qProjSize,
                              int kProjSize,
                              int num_q_heads,
                              int num_tokens,
                              int num_kv_heads,
                              int q_block_size,
                              int k_block_size,
                              int q_array_size,
                              int *real_token_idx) {
  CUDA_KERNEL_LOOP(
      i,
      num_tokens * (qProjSize * num_q_heads + kProjSize * num_kv_heads) / 2) {
    // create complex number
    bool q_tensor = i < (q_array_size / 2);
    int proj_size = q_tensor ? qProjSize : kProjSize;
    int real_i = q_tensor ? i : i - q_array_size / 2;

    int head_idx = real_i / (num_tokens * proj_size / 2);
    int idx = real_i % (num_tokens * proj_size / 2);
    int token_idx =
        (real_i - head_idx * (num_tokens * proj_size / 2)) / (proj_size / 2);

    int real_part_index = idx + token_idx * (proj_size / 2) +
                          head_idx * (q_tensor ? q_block_size : k_block_size) +
                          (q_tensor ? 0 : q_array_size);
    int complex_part_index = real_part_index + (proj_size / 2);

    complex_input[i] = {input_ptr[real_part_index],
                        input_ptr[complex_part_index]};

    // get the freq_cis: shape 1 * (qProjSize/2) = 1 * 64
    // apply a Cartesian coordinate transformation
    // multiple with input & /copy back to q/k

    // get position of token

    // size_t pos = id_map[token_idx].token_position;
    if (real_token_idx[token_idx] >= 0) {
      token_idx = real_token_idx[token_idx];
      size_t pos = tokenInfos[token_idx].abs_depth_in_request;

      // float before_real = complex_input[i].x, before_complex =
      int pos_i = real_i % (proj_size / 2);
      float freq = pos * (1.0 / pow(10000.0, (float)2 * pos_i / proj_size));
      hipFloatComplex complex_pos = {cos(freq), sin(freq)};

      complex_input[i] = hipCmulf(complex_input[i], complex_pos);
      input_ptr[real_part_index] = complex_input[i].x;
      input_ptr[complex_part_index] = complex_input[i].y;
    }
  }
}

template <typename DT>
void compute_qkv_kernel(IncMultiHeadSelfAttentionMeta const *m,
                        BatchConfig const *bc,
                        int shard_id,
                        DT const *input_ptr,
                        DT const *weight_ptr,
                        DT *output_ptr,
                        DT const *bias_ptr,
                        hipStream_t stream) {

  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  DT alpha = 1.0f, beta = 0.0f;
  assert(m->qSize == m->vSize && m->qSize == m->kSize);
  hipDataType cublas_data_type = ff_to_cuda_datatype(m->output_type[0]);
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = cublas_data_type;
#endif
  // Compute (W^T)x matmul: einsum(ijkl,im->jmkl)
  // Weights: qSize x qProjSize x 3 x num_q_heads
  // Input: qSize x num_tokens
  // Output >>> qProjSize x num_tokens x 3 x num_q_heads
  int m_q = m->qProjSize;
  int m_k = m->kProjSize;
  int m_v = m->vProjSize;
  assert(m_q == m_k && m_k == m_v); // keep things simple for now
  int n = *m->max_req_length * bc->num_active_requests();
  int k = m->qSize;
  int m_ = m_q;
  int lda = k, ldb = k, ldc = m_q;

  size_t strideA = m_q * k; // query weight head size
  size_t strideB = 0;       // input stays the same for all heads.
  size_t strideC = m_q * n; // size of the output block for each head.

  // compute QKV
  checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                       HIPBLAS_OP_T,
                                       HIPBLAS_OP_N,
                                       m_,
                                       n,
                                       k,
                                       &alpha,
                                       weight_ptr,
                                       cublas_data_type,
                                       lda,
                                       strideA,
                                       input_ptr,
                                       cublas_data_type,
                                       ldb,
                                       strideB,
                                       &beta,
                                       output_ptr,
                                       cublas_data_type,
                                       ldc,
                                       strideC,
                                       m->num_q_heads + m->num_kv_heads +
                                           m->num_kv_heads,
                                       compute_type,
                                       CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  // apply rotary emmmbedding for q and k
  // step1 change the k, v to complex tensor
  int num_tokens = *m->max_req_length * bc->num_active_requests();
  int parallelism = m->kProjSize * num_tokens * m->num_q_heads;
  int q_block_size = m->qProjSize * num_tokens;
  int k_block_size = m->kProjSize * num_tokens;
  int q_array_size = m->qProjSize * num_tokens * m->num_q_heads;

  // print_tensor<float>((float *)output_ptr, 32, "qkv");
  // apply bias for q, k, v
  if (*m->qkv_bias) {
    apply_proj_bias_qkv<<<GET_BLOCKS(parallelism),
                          min(CUDA_NUM_THREADS, parallelism),
                          0,
                          stream>>>(output_ptr,
                                    bias_ptr,
                                    shard_id,
                                    num_tokens,
                                    m->qProjSize,
                                    m->kProjSize,
                                    m->vProjSize,
                                    m->global_num_q_heads,
                                    m->global_num_kv_heads,
                                    m->num_q_heads,
                                    m->num_kv_heads,
                                    *m->scaling_query,
                                    m->scaling_factor,
                                    m->real_token_idx_gpu);
  } else if (m->scaling_query) {
    scaling_query_kernel<<<GET_BLOCKS(parallelism),
                           min(CUDA_NUM_THREADS, parallelism),
                           0,
                           stream>>>(output_ptr,
                                     num_tokens,
                                     m->num_q_heads,
                                     m->qProjSize,
                                     m->scaling_factor);
  }
  if (*m->apply_rotary_embedding) {
    /*q&k*/
    parallelism =
        num_tokens *
        (m->qProjSize * m->num_q_heads + m->kProjSize * m->num_kv_heads) / 2;
    apply_rotary_embedding_hf<<<GET_BLOCKS(parallelism),
                                min(CUDA_NUM_THREADS, parallelism),
                                0,
                                stream>>>(output_ptr,
                                          m->complex_input,
                                          m->token_infos,
                                          m->qProjSize,
                                          m->kProjSize,
                                          m->num_q_heads,
                                          num_tokens,
                                          m->num_kv_heads,
                                          q_block_size,
                                          k_block_size,
                                          q_array_size,
                                          m->real_token_idx_gpu);
  }
}

template <typename DT>
void update_kv_cache_kernel(IncMultiHeadSelfAttentionMeta const *m,
                            BatchConfig const *bc,
                            hipStream_t stream) {
  int num_tokens = *m->max_req_length * bc->num_active_requests();

  if (num_tokens > 0) {
    int parallelism =
        (m->kProjSize + m->vProjSize) * num_tokens * m->num_kv_heads;
    store_kv_cache<<<GET_BLOCKS(parallelism),
                     min(CUDA_NUM_THREADS, parallelism),
                     0,
                     stream>>>(static_cast<DT *>(m->devQKVProjArray),
                               static_cast<DT *>(m->keyCache),
                               static_cast<DT *>(m->valueCache),
                               m->token_infos,
                               m->qProjSize,
                               m->kProjSize,
                               m->vProjSize,
                               num_tokens,
                               m->num_q_heads,
                               m->num_kv_heads,
                               BatchConfig::MAX_SEQ_LENGTH,
                               m->real_token_idx_gpu);

    // pad the total_token for key and value

    // store temporary query, key and value
    parallelism = m->num_q_heads * num_tokens * m->qProjSize;
    get_query<<<GET_BLOCKS(parallelism),
                min(CUDA_NUM_THREADS, parallelism),
                0,
                stream>>>(static_cast<DT *>(m->devQKVProjArray),
                          static_cast<DT *>(m->query),
                          m->token_infos,
                          m->qProjSize,
                          num_tokens,
                          *m->max_req_length,
                          m->num_q_heads);
    // print_tensor<float>((float *)m->devQKVProjArray, 32, "query");
    // print_tensor<float>((float *)m->query, 32, "query");
    parallelism = (m->kProjSize + m->vProjSize) * num_tokens * m->num_kv_heads;
    get_key_value<<<GET_BLOCKS(parallelism),
                    min(CUDA_NUM_THREADS, parallelism),
                    0,
                    stream>>>(static_cast<DT *>(m->key),
                              static_cast<DT *>(m->value),
                              static_cast<DT *>(m->keyCache),
                              static_cast<DT *>(m->valueCache),
                              m->total_tokens_per_req_gpu,
                              m->kProjSize,
                              m->vProjSize,
                              *m->max_total_tokens,
                              *m->max_total_tokens * bc->num_active_requests(),
                              m->num_kv_heads,
                              BatchConfig::MAX_SEQ_LENGTH);
  }
}

template <typename DT>
void pre_build_weight_kernel(IncMultiHeadSelfAttentionMeta const *m,
                             GenericTensorAccessorR const weight,
                             DataType data_type,
                             hipStream_t stream) {
  // additional processing for weight uploading
  // Note that we update weight_ptr and bias_ptr when uploading weight and
  // bias
  if (m->quantization_type != DT_NONE) {
    // copy weight_ptr to quantized_weight_ptr, do compression and store in
    // m->weight_ptr
    hipMemcpyAsync(m->quantized_weight_ptr,
                    weight.get_byte_ptr(),
                    m->quantized_weightSize,
                    hipMemcpyHostToDevice,
                    stream);

    if (m->quantization_type == DT_INT4) {
      int parallelism = m->qProjSize * m->qSize * m->num_q_heads / 2;
      decompress_int4_attention_weights<<<GET_BLOCKS(parallelism),
                                          min(CUDA_NUM_THREADS, parallelism),
                                          0,
                                          stream>>>(
          m->quantized_weight_ptr,
          static_cast<DT *>(m->weight_ptr),
          m->qProjSize,
          m->qSize,
          m->num_q_heads);
    } else {
      assert(m->quantization_type == DT_INT8);
      int parallelism = m->qProjSize * m->qSize * m->num_q_heads;
      decompress_int8_attention_weights<<<GET_BLOCKS(parallelism),
                                          min(CUDA_NUM_THREADS, parallelism),
                                          0,
                                          stream>>>(
          m->quantized_weight_ptr,
          static_cast<DT *>(m->weight_ptr),
          m->qProjSize,
          m->qSize,
          m->num_q_heads);
    }
  } else {
    if (data_type == DT_FLOAT) {
      hipMemcpyAsync(m->weight_ptr,
                      weight.get_float_ptr(),
                      m->weightSize,
                      hipMemcpyHostToDevice,
                      stream);
    } else if (data_type == DT_HALF) {
      hipMemcpyAsync(m->weight_ptr,
                      weight.get_half_ptr(),
                      m->weightSize,
                      hipMemcpyHostToDevice,
                      stream);
    } else {
      assert(false);
    }
  }
}

void unpad_output();

template <typename DT>
void pad_input(IncMultiHeadSelfAttentionMeta const *m,
               BatchConfig const *bc,
               DT const *input_ptr,
               hipStream_t stream) {
  *m->max_total_tokens = 0;
  int req_id = 0;
  int max_length = 0;
  for (int i = 0; i < bc->MAX_NUM_REQUESTS; i++) {
    if (bc->request_completed[i]) {
      continue;
    }
    int total_tokens = bc->requestsInfo[i].token_start_offset +
                       bc->requestsInfo[i].num_tokens_in_batch;
    m->total_tokens_per_req[req_id++] = total_tokens;
    *m->max_total_tokens = std::max(*m->max_total_tokens, total_tokens);
    max_length = std::max(max_length, bc->requestsInfo[i].num_tokens_in_batch);
  }
  *m->max_req_length = max_length;

  memset(m->real_token_idx,
         -1,
         sizeof(int) * BatchConfig::MAX_NUM_TOKENS *
             BatchConfig::MAX_NUM_REQUESTS);
  int pre_tokens = 0;
  int pre_req_tokens = 0;

  for (int i = 0; i < bc->num_active_tokens(); i++) {
    int req_idx = bc->tokensInfo[i].request_index;
    int real_idx = req_idx * max_length + (i - pre_req_tokens);
    m->real_token_idx[real_idx] = pre_tokens++;

    if (i < bc->num_active_tokens() - 1 &&
        bc->tokensInfo[i].request_index !=
            bc->tokensInfo[i + 1].request_index) {
      pre_req_tokens += bc->requestsInfo[req_idx].num_tokens_in_batch;
    }
  }

  // copy metadata to gpu
  hipMemcpyAsync(m->total_tokens_per_req_gpu,
                  m->total_tokens_per_req,
                  sizeof(int) * BatchConfig::MAX_NUM_REQUESTS,
                  hipMemcpyHostToDevice,
                  stream);
  hipMemcpyAsync(m->real_token_idx_gpu,
                  m->real_token_idx,
                  sizeof(int) * BatchConfig::MAX_NUM_TOKENS *
                      BatchConfig::MAX_NUM_REQUESTS,
                  hipMemcpyHostToDevice,
                  stream);

  // pad input
  int parallelism =
      m->qProjSize * m->num_q_heads * max_length * bc->num_active_requests();
  pad_input_ptr<<<GET_BLOCKS(parallelism),
                  min(CUDA_NUM_THREADS, parallelism),
                  0,
                  stream>>>(input_ptr,
                            static_cast<DT *>(m->padded_input),
                            bc,
                            max_length * bc->num_active_requests(),
                            m->qSize,
                            max_length);
}

// request afer padding
// r1 -> |t1, t1, t3, pad1, pad2, pad3|
// r2 -> |t1, t1, t3, t4,   t5,     t6|
// r3 -> |t1, t1, t3, t4,   t5,   pad1|
template <typename DT>
void inference_kernel(IncMultiHeadSelfAttentionMeta const *m,
                      BatchConfig const *bc,
                      int shard_id,
                      DT const *input_ptr,
                      DT const *weight_ptr,
                      DT *output_ptr,
                      DT const *bias_ptr,
                      hipStream_t stream) {
  // here because we need position info in inference 1
  // print_tensor<float>((float *)input_ptr, 32, "inputtttt0");
  if (m->offload && m->biasSize > 0) {
    hipMemcpyAsync(
        m->bias_ptr, bias_ptr, m->biasSize, hipMemcpyHostToDevice, stream);
    bias_ptr = static_cast<DT *>(m->bias_ptr);
  }
  pad_input(m, bc, input_ptr, stream);

  // padd all request and copy token infos,
  hipMemcpyAsync(m->token_infos,
                  &(bc->tokensInfo),
                  bc->num_active_tokens() * sizeof(BatchConfig::PerTokenInfo),
                  hipMemcpyHostToDevice,
                  stream);

  // phase 1: Implement kernel to compute KQV for input tokens
  compute_qkv_kernel(m,
                     bc,
                     shard_id,
                     static_cast<DT *>(m->padded_input),
                     weight_ptr,
                     static_cast<DT *>(m->devQKVProjArray),
                     bias_ptr,
                     stream);

  // phase 2: Update key/val cache
  update_kv_cache_kernel<DT>(m, bc, stream);

  // phase 3: Compute attention score
  // 3 kernels for pahse 3: matmul1 - softmax - matmal2
  compute_attention_kernel(
      m, bc, shard_id, output_ptr, bias_ptr, weight_ptr, stream);

  // print_tensor<float>((float *)output_ptr, 32, "output");
  // assert(false);
}

} // namespace IncMultiHeadAttention
} // namespace Kernels

using namespace Kernels::IncMultiHeadAttention;

// change the layout of query
// from num_total_tokens * qProjSize * num_heads
// -> num_new_tokens * qProjSize * num_heads * num_reqs
template <typename DT>
__global__ void get_query(DT const *devQKVProjArray,
                          DT *query,
                          BatchConfig::PerTokenInfo const *tokenInfos,
                          int qProjSize,
                          int total_tokens,
                          int max_new_tokens,
                          int num_q_heads) {
  CUDA_KERNEL_LOOP(i, total_tokens * qProjSize * num_q_heads) {
    int head_id = i / (total_tokens * qProjSize);
    int tokens_id = (i - (head_id * total_tokens * qProjSize)) / qProjSize;

    int token_id_in_req = tokens_id % max_new_tokens;
    int req_id = tokens_id / max_new_tokens;
    int offset = i % qProjSize;

    query[req_id * max_new_tokens * qProjSize * num_q_heads +
          head_id * max_new_tokens * qProjSize + token_id_in_req * qProjSize +
          offset] = devQKVProjArray[i];
  }
}

template <typename DT>
__global__ void store_kv_cache(DT const *devQKVProjArray,
                               DT *kCache_ptr,
                               DT *vCache_ptr,
                               BatchConfig::PerTokenInfo const *tokenInfos,
                               int qProjSize,
                               int kProjSize,
                               int vProjSize,
                               int num_tokens,
                               int num_q_heads,
                               int num_kv_heads,
                               int max_seq_len,
                               int *real_token_idx) {
  CUDA_KERNEL_LOOP(i, num_tokens * (kProjSize + vProjSize) * num_kv_heads) {
    int q_array_size = qProjSize * num_tokens * num_q_heads;
    int k_array_size = kProjSize * num_tokens * num_kv_heads;

    bool k_cache = i < k_array_size;
    int real_i = k_cache ? i : i - k_array_size;

    int proj_size = k_cache ? kProjSize : vProjSize;
    int head_idx = real_i / (num_tokens * proj_size);
    int token_idx = (real_i - head_idx * (num_tokens * proj_size)) / proj_size;
    int data_idx = real_i % proj_size;

    DT val = devQKVProjArray[q_array_size + (k_cache ? 0 : k_array_size) +
                             head_idx * proj_size * num_tokens +
                             token_idx * proj_size + data_idx];

    if (real_token_idx[token_idx] >= 0) {
      token_idx = real_token_idx[token_idx];
      int const req_id = tokenInfos[token_idx].request_index;
      int const tok_id = tokenInfos[token_idx].abs_depth_in_request;

      DT *cache_ptr = k_cache ? kCache_ptr : vCache_ptr;
      cache_ptr[req_id * (num_kv_heads * max_seq_len * proj_size) +
                head_idx * (max_seq_len * proj_size) + tok_id * proj_size +
                data_idx] = val;
    }
  }
}

template <typename DT>
__global__ void fill_entries_above_diagonal(DT *matrix,
                                            size_t num_rows,
                                            size_t num_cols,
                                            size_t num_q_heads,
                                            size_t entries_above_diagonal,
                                            int num_activate_req,
                                            DT value) {
  CUDA_KERNEL_LOOP(i, entries_above_diagonal * num_q_heads * num_activate_req) {
    int req_id = i / (entries_above_diagonal * num_q_heads);
    int pre_eles = entries_above_diagonal * num_q_heads * req_id;
    size_t in_req_idx = i - pre_eles;
    size_t head_idx = in_req_idx / entries_above_diagonal;
    size_t entry_idx = in_req_idx % entries_above_diagonal;
    size_t y = (-1 + sqrt(8 * (float)entry_idx + 1)) / 2;
    size_t x = entry_idx - y * (y + 1) / 2;
    y += (num_cols - num_rows) + 1;
    matrix[req_id * num_rows * num_cols * num_q_heads +
           head_idx * num_rows * num_cols + num_cols * y + x] = value;
  }
}

// input batchszie * num_new_tokens * num_heads * embedding_size/num_heads
// weight size = embeddingsize * embeddingsize

// before padding each request
//  query: batchszie, num_heads, num_new_tokens, embedding_size/num_heads
//  key: batchszie, num_kv_heads, num_total_tokens, embedding_size/num_heads
//  qk_prod: batchsize, num_heads, num_new_tokens, num_total_tokens

// after padding each request
//  query: batchszie, num_heads, num_new_tokens_padded, embedding_size/num_heads
//  key: batchszie, num_kv_heads, MAX_SEQ_LENGTH, embedding_size/num_heads
//  qk_prod: batchsize, num_heads, num_new_tokens_padded, num_total_tokens

// v: batchszie, num_kv_heads, num_total_tokens, embedding_size/num_heads
// score: batchszie, num_heads, num_new_tokens, embedding_size/num_heads

// v: batchszie, num_kv_heads, MAX_SEQ_LENGTH, embedding_size/num_heads
// score: batchszie, num_heads, num_new_tokens_padded, embedding_size/num_heads

// resize num_activate_tokens() * embedding_size

template <typename DT>
void compute_attention_kernel(IncMultiHeadSelfAttentionMeta const *m,
                              BatchConfig const *bc,
                              int shard_id,
                              DT *output_ptr,
                              DT const *bias_ptr,
                              DT const *weight_ptr,
                              hipStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  hipDataType cublas_data_type = ff_to_cuda_datatype(m->output_type[0]);
  hipdnnDataType_t cudnn_data_type = ff_to_cudnn_datatype(m->output_type[0]);
  assert(data_type_size(m->output_type[0]) == sizeof(DT));
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = cublas_data_type;
#endif
  // int num_requests = bc->num_active_requests();
  // int num_tokens = bc->num_active_tokens();
  // int tokens_previous_requests = 0;
  int max_new_tokens = *m->max_req_length;
  int max_total_tokens = *m->max_total_tokens;

  assert(m->qProjSize == m->kProjSize);

  int q_block_size = m->qProjSize * max_new_tokens;
  int kt_block_size = m->kProjSize * max_total_tokens;
  // int kt_req_block_size = kt_block_size * m->num_kv_heads;
  int vt_block_size = m->vProjSize * max_total_tokens;
  // int vt_req_block_size = vt_block_size * m->num_kv_heads;

  int num_activate_req = bc->num_active_requests();
  int m_ = max_new_tokens;
  int n = max_total_tokens;
  int k = m->qProjSize;
  int lda = k, ldb = k, ldc = m_;
  int strideA = q_block_size;
  int strideB = kt_block_size;
  int strideC = max_new_tokens * max_total_tokens;
  DT alpha = 1.0f, beta = 0.0f;
  if (*m->qk_prod_scaling) {
    alpha = static_cast<DT>(1.0f / sqrt(m->kProjSize));
  }
  // To get A, skip over Q entries from previous requests (same head)
  DT const *A = static_cast<DT *>(m->query);
  // To get B, skip over K entries from previous requests (all heads +
  // padding)
  // DT const *B = static_cast<DT *>(m->keyCache) + i * kt_req_block_size;
  DT const *B = static_cast<DT *>(m->key);
  // To get C, skip over QK^T products from previous requests
  DT *C = static_cast<DT *>(m->qk_prods);
  if (m->num_kv_heads == m->num_q_heads) {
    checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                         HIPBLAS_OP_T,
                                         HIPBLAS_OP_N,
                                         m_,
                                         n,
                                         k,
                                         &alpha,
                                         A,
                                         cublas_data_type,
                                         lda,
                                         strideA,
                                         B,
                                         cublas_data_type,
                                         ldb,
                                         strideB,
                                         &beta,
                                         C,
                                         cublas_data_type,
                                         ldc,
                                         strideC,
                                         m->num_q_heads * num_activate_req,
                                         compute_type,
                                         CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // save_tensor<float>((float *)A, 64 * 3 * 10,
    // ("/home/ubuntu/FlexFlow/inference/query" + std::to_string(shard_id) +
    // ".txt").c_str()); print_tensor<float>((float *)C, 32, "qkprod",
    // shard_id); print_tensor<float>((float *)B, 32, "key", shard_id);
    // print_tensor<float>((float *)m->qk_prods, 32, "qkprod");
    // save_tensor<float>((float *)m->qk_prods, 10 * 10 * 12,
    // "/home/xinhaoc/FlexFlow/inference/qk.txt");
  } else {
    strideB = 0;
    // use hipblasGemmStridedBatchedEx
    int one_step_heads = m->num_q_heads / m->num_kv_heads;
    m_ = max_new_tokens;
    n = max_total_tokens;
    k = m->qProjSize;
    lda = k, ldb = k, ldc = m_;
    for (int step = 0; step < m->num_kv_heads; step++) {
      for (int req = 0; req < num_activate_req; req++) {
        A = static_cast<DT *>(m->query) +
            req * max_new_tokens * m->qProjSize * m->num_q_heads;
        B = static_cast<DT *>(m->key) +
            req * max_total_tokens * m->qProjSize * m->num_kv_heads;
        C = static_cast<DT *>(m->qk_prods) +
            req * max_new_tokens * max_total_tokens * m->num_q_heads;
        checkCUDA(
            hipblasGemmStridedBatchedEx(m->handle.blas,
                                       HIPBLAS_OP_T,
                                       HIPBLAS_OP_N,
                                       m_,
                                       n,
                                       k,
                                       &alpha,
                                       A + step * strideA * one_step_heads,
                                       cublas_data_type,
                                       lda,
                                       strideA,
                                       B + step * kt_block_size,
                                       cublas_data_type,
                                       ldb,
                                       strideB,
                                       &beta,
                                       C + step * strideC * one_step_heads,
                                       cublas_data_type,
                                       ldc,
                                       strideC,
                                       one_step_heads,
                                       compute_type,
                                       CUBLAS_GEMM_DEFAULT_TENSOR_OP));
      }
    }
  }
  // add alibi position bias to qk production
  if (*m->position_bias) {
    // size_t parallelism = m->num_q_heads * max_total_tokens * max_new_tokens *
    // num_activate_req; apply_position_bias_qkprd<<<GET_BLOCKS(parallelism),
    //                             min((size_t)CUDA_NUM_THREADS, parallelism),
    //                             0,
    //                             stream>>>(C,
    //                                       max_new_tokens,
    //                                       max_total_tokens,
    //                                       m->num_q_heads,
    //                                       m->global_num_q_heads,
    //                                       num_activate_req,
    //                                       shard_id);
  }

  // Fill all elements above diagonal in qk prods with -inf to force
  // causal attention.
  // assert(num_new_tokens <= total_tokens);
  size_t entries_above_diagonal = max_new_tokens * (max_new_tokens - 1) / 2;
  if (entries_above_diagonal > 0) {
    size_t parallelism =
        m->num_q_heads * entries_above_diagonal * num_activate_req;

    fill_entries_above_diagonal<<<GET_BLOCKS(parallelism),
                                  min((size_t)CUDA_NUM_THREADS, parallelism),
                                  0,
                                  stream>>>(C,
                                            max_new_tokens,
                                            max_total_tokens,
                                            m->num_q_heads,
                                            entries_above_diagonal,
                                            num_activate_req,
                                            static_cast<DT>(-INFINITY));
  }

  // print_tensor<float>((float *)C, 32, "fill in");
  // Compute Softmax(QK^T/sqrt(d_k))
  // Before modifying the parameters below, make sure to read the following
  // description of the HIPDNN_TENSOR_NCHW tensor layout, from
  // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnTensorFormat_t:
  // This tensor format specifies that the data is laid out in the following
  // order: batch size, feature maps, rows, columns. The strides are
  // implicitly defined in such a way that the data are contiguous in memory
  // with no padding between images, feature maps, rows, and columns; the
  // columns are the inner dimension and the images are the outermost
  // dimension.
  int n_param = m->num_q_heads * num_activate_req;
  int c_param = max_total_tokens;
  int h_param = 1;
  int w_param = max_new_tokens;
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->qk_tensor,
                                        HIPDNN_TENSOR_NCHW,
                                        cudnn_data_type,
                                        n_param,
                                        c_param,
                                        h_param,
                                        w_param));
  float softmax_alpha = 1.0f, softmax_beta = 0.0f;
  DT *C_softmax = static_cast<DT *>(m->qk_prods_softmax);
  // The softmax operation below is executed according to the
  // HIPDNN_SOFTMAX_MODE_CHANNEL, which is also described in the docs: The
  // softmax operation is computed per spatial location (H,W) per image (N)
  // across dimension C.
  checkCUDNN(hipdnnSoftmaxForward(m->handle.dnn,
                                 HIPDNN_SOFTMAX_ACCURATE,
                                 HIPDNN_SOFTMAX_MODE_CHANNEL,
                                 &softmax_alpha,
                                 m->qk_tensor,
                                 C,
                                 &softmax_beta,
                                 m->qk_tensor,
                                 C_softmax));
  // print_tensor<float>((float *)C_softmax, 32, "softmax");
  // Matmul softmax(QK^T/sqrt(d_k)) by V
  alpha = 1.0f, beta = 0.0f;
  m_ = max_new_tokens;
  n = m->vProjSize;
  k = max_total_tokens;
  lda = m_, ldb = n, ldc = m_;
  strideA = max_new_tokens * max_total_tokens;
  strideB = vt_block_size;
  strideC = max_new_tokens * m->vProjSize;
  // To get A, skip over softmax(QK^T/sqrt(d_k)) entries from previous
  // requests (all heads)
  A = C_softmax;
  // To get B, skip over V^T entries from previous requests (all heads +
  // padding)
  // B = static_cast<DT *>(m->valueCache) + i * vt_req_block_size;
  B = static_cast<DT *>(m->value);
  // To get C, skip over softmax(QK^T/sqrt(d_k))V products from previous
  // requests
  // C = static_cast<DT *>(m->attn_heads) +
  //     tokens_previous_requests * m->num_q_heads * m->vProjSize;
  C = static_cast<DT *>(m->attn_heads);

  if (m->num_q_heads == m->num_kv_heads) {
    checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                         HIPBLAS_OP_N,
                                         HIPBLAS_OP_T,
                                         m_,
                                         n,
                                         k,
                                         &alpha,
                                         A,
                                         cublas_data_type,
                                         lda,
                                         strideA,
                                         B,
                                         cublas_data_type,
                                         ldb,
                                         strideB,
                                         &beta,
                                         C,
                                         cublas_data_type,
                                         ldc,
                                         strideC,
                                         m->num_q_heads * num_activate_req,
                                         compute_type,
                                         CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    // print_tensor<float>((float *)A + 500, 32, "qk");
    // print_tensor<float>((float *)m->value + 500, 32, "value");
    // print_tensor<float>((float *)C + 3000, 32, "kv prod");
  } else {
    int one_step_heads = m->num_q_heads / m->num_kv_heads;
    n = m->vProjSize;
    lda = m_, ldb = n, ldc = m_;
    strideA = max_new_tokens * max_total_tokens;
    strideB = 0;
    strideC = max_new_tokens * m->vProjSize;
    for (int step = 0; step < m->num_kv_heads; step++) {
      for (int req = 0; req < num_activate_req; req++) {
        A = C_softmax +
            req * max_new_tokens * max_total_tokens * m->num_q_heads;
        B = static_cast<DT *>(m->value) +
            req * max_total_tokens * m->vProjSize * m->num_kv_heads;
        C = static_cast<DT *>(m->attn_heads) +
            req * max_new_tokens * m->vProjSize * m->num_kv_heads;
        checkCUDA(
            hipblasGemmStridedBatchedEx(m->handle.blas,
                                       HIPBLAS_OP_N,
                                       HIPBLAS_OP_T,
                                       m_,
                                       n,
                                       k,
                                       &alpha,
                                       A + step * one_step_heads * strideA,
                                       cublas_data_type,
                                       lda,
                                       strideA,
                                       B + step * vt_block_size,
                                       cublas_data_type,
                                       ldb,
                                       strideB,
                                       &beta,
                                       C + step * one_step_heads * strideC,
                                       cublas_data_type,
                                       ldc,
                                       strideC,
                                       one_step_heads,
                                       compute_type,
                                       CUBLAS_GEMM_DEFAULT_TENSOR_OP));
      }
    }
  }
  // Project to output, save result directly on output tensor
  alpha = 1.0f, beta = 0.0f;
  m_ = m->oProjSize;
  k = m->vProjSize * m->num_q_heads;
  n = max_new_tokens * num_activate_req;
  lda = k, ldb = n, ldc = m_;
  A = weight_ptr + m->qSize * (m->qProjSize * m->num_q_heads +
                               m->kProjSize * m->num_kv_heads +
                               m->vProjSize * m->num_kv_heads);
  B = C;
  C = static_cast<DT *>(m->padded_output);

  checkCUDA(hipblasGemmEx(m->handle.blas,
                         HIPBLAS_OP_T,
                         HIPBLAS_OP_T,
                         m_,
                         n,
                         k,
                         &alpha,
                         A,
                         cublas_data_type,
                         lda,
                         B,
                         cublas_data_type,
                         ldb,
                         &beta,
                         C,
                         cublas_data_type,
                         ldc,
                         compute_type,
                         CUBLAS_GEMM_DEFAULT_TENSOR_OP));

  if (*m->final_bias && shard_id == 0) {
    int parallelism = m->oProjSize * max_new_tokens;
    int qkv_weight_size = m->qProjSize * m->global_num_q_heads +
                          m->kProjSize * m->global_num_kv_heads +
                          m->vProjSize * m->global_num_kv_heads;

    apply_proj_bias_w<<<GET_BLOCKS(parallelism),
                        min(CUDA_NUM_THREADS, parallelism),
                        0,
                        stream>>>(static_cast<DT *>(m->padded_output),
                                  bias_ptr,
                                  max_new_tokens,
                                  qkv_weight_size,
                                  m->oProjSize);
  }

  // copy the output tokens
  // m->padded_output, output_ptr
  // print_tensor<float>((float *)m->padded_output, 32, "padded op");
  int parallelism = m->oProjSize * max_new_tokens * bc->num_active_requests();
  copy_output<<<GET_BLOCKS(parallelism),
                min(CUDA_NUM_THREADS, parallelism),
                0,
                stream>>>(static_cast<DT *>(m->padded_output),
                          output_ptr,
                          max_new_tokens * bc->num_active_requests(),
                          m->oProjSize,
                          m->real_token_idx_gpu);

  // assert(tokens_previous_requests == num_tokens);
}

/*static*/
void IncMultiHeadSelfAttention::inference_kernel_wrapper(
    IncMultiHeadSelfAttentionMeta const *m,
    BatchConfig const *bc,
    int shard_id,
    GenericTensorAccessorR const &input,
    GenericTensorAccessorR const &weight,
    GenericTensorAccessorW const &output,
    GenericTensorAccessorR const &bias) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  bool use_bias = *m->qkv_bias || *m->final_bias;
  // print_tensor<float>(input.get_float_ptr(), 32, "input");

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  // assert(input.data_type == weight.data_type);
  assert(input.data_type == output.data_type);
  if (use_bias) {
    assert(input.data_type == bias.data_type);
  }

  if (input.data_type == DT_HALF) {
    if (m->offload) {
      pre_build_weight_kernel<half>(m, weight, input.data_type, stream);
    }
    half const *bias_ptr =
        use_bias ? bias.get_half_ptr() : static_cast<half const *>(nullptr);
    Kernels::IncMultiHeadAttention::inference_kernel(
        m,
        bc,
        shard_id,
        input.get_half_ptr(),
        m->offload ? static_cast<half *>(m->weight_ptr) : weight.get_half_ptr(),
        output.get_half_ptr(),
        bias_ptr,
        stream);
  } else if (input.data_type == DT_FLOAT) {
    if (m->offload) {
      pre_build_weight_kernel<float>(m, weight, input.data_type, stream);
    }
    float const *bias_ptr =
        use_bias ? bias.get_float_ptr() : static_cast<float const *>(nullptr);
    Kernels::IncMultiHeadAttention::inference_kernel(
        m,
        bc,
        shard_id,
        input.get_float_ptr(),
        m->offload ? static_cast<float *>(m->weight_ptr)
                   : weight.get_float_ptr(),
        output.get_float_ptr(),
        bias_ptr,
        stream);
  } else {
    assert(false && "Unspported data type");
  }
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("IncMultiHeadSelfAttention forward time = %.9fms\n", elapsed);

    // if (input.data_type == DT_HALF) {
    //   print_tensor<half>(input.get_half_ptr(),
    //                      32,
    //                      "[IncMultiHeadSelfAttention:forward:input]");
    //   print_tensor<half>(weight.get_half_ptr(),
    //                      32,
    //                      "[IncMultiHeadSelfAttention:forward:weight]");
    //   print_tensor<half>(output.get_half_ptr(),
    //                      32,
    //                      "[IncMultiHeadSelfAttention:forward:output]");
    //   print_tensor<half>(
    //       bias.get_half_ptr(), 32,
    //       "[IncMultiHeadSelfAttention:forward:bias]");
    // } else {
    //   print_tensor<float>(input.get_float_ptr(),
    //                       32,
    //                       "[IncMultiHeadSelfAttention:forward:input]");
    //   print_tensor<float>(weight.get_float_ptr(),
    //                       32,
    //                       "[IncMultiHeadSelfAttention:forward:weight]");
    //   print_tensor<float>(output.get_float_ptr(),
    //                       32,
    //                       "[IncMultiHeadSelfAttention:forward:output]");
    //   print_tensor<float>(
    //       bias.get_float_ptr(), 32,
    //       "[IncMultiHeadSelfAttention:forward:bias]");
    // }

    // print_tensor<3, float>(acc_query.ptr, acc_query.rect,
    // "[Attention:forward:query]"); print_tensor<3, float>(acc_output.ptr,
    // acc_output.rect, "[Attention:forward:output]");
  }
}

IncMultiHeadSelfAttentionMeta::IncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    IncMultiHeadSelfAttention const *attn,
    GenericTensorAccessorR const &weight,
    MemoryAllocator &gpu_mem_allocator,
    int num_samples,
    int _num_q_heads,
    int _num_kv_heads)
    : IncMultiHeadSelfAttentionMeta(handler,
                                    INC_DECODING_MODE,
                                    attn,
                                    attn->qSize,
                                    attn->kSize,
                                    attn->vSize,
                                    attn->qProjSize,
                                    attn->kProjSize,
                                    attn->vProjSize,
                                    attn->oProjSize,
                                    attn->apply_rotary_embedding,
                                    attn->qkv_bias,
                                    attn->scaling_query,
                                    attn->qk_prod_scaling,
                                    attn->position_bias,
                                    attn->final_bias,
                                    attn->scaling_factor,
                                    weight,
                                    gpu_mem_allocator,
                                    num_samples,
                                    attn->num_q_heads,
                                    attn->num_kv_heads,
                                    _num_q_heads,
                                    _num_kv_heads,
                                    attn->quantization_type,
                                    attn->offload) {}

IncMultiHeadSelfAttentionMeta::IncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    InferenceMode infer_mode,
    Op const *attn,
    int _qSize,
    int _kSize,
    int _vSize,
    int _qProjSize,
    int _kProjSize,
    int _vProjSize,
    int _oProjSize,
    bool _apply_rotary_embedding,
    bool _qkv_bias,
    bool _scaling_query,
    bool _qk_prod_scaling,
    bool _position_bias,
    bool _final_bias,
    float _scaling_factor,
    GenericTensorAccessorR const &weight,
    MemoryAllocator &gpu_mem_allocator,
    int num_samples,
    int _global_num_q_heads,
    int _global_num_kv_heads,
    int _num_q_heads,
    int _num_kv_heads,
    DataType _quantization_type,
    bool _offload)
    : OpMeta(handler, attn), weight_ptr(nullptr), bias_ptr(nullptr) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  checkCUDNN(hipdnnSetStream(handler.dnn, stream));
  checkCUDNN(hipdnnCreateTensorDescriptor(&qk_tensor));
  qSize = _qSize;
  kSize = _kSize;
  vSize = _vSize;
  // assume dimensions match for now
  assert(qSize == kSize);
  assert(kSize == vSize);
  qProjSize = _qProjSize;
  kProjSize = _kProjSize;
  assert(qProjSize == kProjSize); // required for attention QK^T matmul
  vProjSize = _vProjSize;
  oProjSize = _oProjSize;
  size_t size_of_dt = data_type_size(attn->data_type);
  quantization_type = _quantization_type;
  offload = _offload;

  global_num_q_heads = _global_num_q_heads;
  global_num_kv_heads = _global_num_kv_heads;
  num_q_heads = _num_q_heads;
  num_kv_heads = _num_kv_heads;

  weightSize =
      ((qSize * qProjSize + oProjSize * (vProjSize > 0 ? vProjSize : vSize)) *
           num_q_heads +
       (kSize * kProjSize + vSize * vProjSize) * num_kv_heads) *
      size_of_dt;
  if (quantization_type != DT_NONE) {
    quantized_weightSize = get_quantization_to_byte_size(
        attn->data_type, quantization_type, weightSize);
  }
  // biasSize = _bias ? oProjSize * size_of_dt * 4 : 0;

  int qkv_bias_size =
      qProjSize * num_q_heads + (kProjSize + vProjSize) * num_kv_heads;
  int final_bias_size = oProjSize;
  biasSize =
      (_qkv_bias ? qkv_bias_size : 0) + (final_bias ? final_bias_size : 0);

  // has_load_weights = (bool *)calloc(1, sizeof(bool));
  //*has_load_weights = false;
  apply_rotary_embedding = (bool *)calloc(1, sizeof(bool));
  *apply_rotary_embedding = _apply_rotary_embedding;
  qkv_bias = (bool *)calloc(1, sizeof(bool));
  *qkv_bias = _qkv_bias;
  scaling_query = (bool *)calloc(1, sizeof(bool));
  *scaling_query = _scaling_query;
  scaling_factor = _scaling_factor;
  qk_prod_scaling = (bool *)calloc(1, sizeof(bool));
  *qk_prod_scaling = _qk_prod_scaling;
  position_bias = (bool *)calloc(1, sizeof(bool));
  *position_bias = _position_bias;
  final_bias = (bool *)calloc(1, sizeof(bool));
  *final_bias = _final_bias;
  max_req_length = (int *)calloc(1, sizeof(int));
  max_total_tokens = (int *)calloc(1, sizeof(int));

  real_token_idx = (int *)calloc(
      BatchConfig::MAX_NUM_TOKENS * BatchConfig::MAX_NUM_REQUESTS, sizeof(int));
  total_tokens_per_req =
      (int *)calloc(BatchConfig::MAX_NUM_REQUESTS, sizeof(int));

  // allocate weight and bias in the reserve space for cpu offloading
  if (offload) {
    weight_ptr = gpu_mem_allocator.allocate_reserved_untyped(weightSize);
    bias_ptr = gpu_mem_allocator.allocate_reserved_untyped(biasSize);
  }

#ifdef INFERENCE_TESTS
  kcache = (float *)calloc(kProjSize * BatchConfig::MAX_SEQ_LENGTH *
                               num_q_heads * BatchConfig::MAX_NUM_REQUESTS,
                           sizeof(float));
  vcache = (float *)calloc(vProjSize * BatchConfig::MAX_SEQ_LENGTH *
                               num_q_heads * BatchConfig::MAX_NUM_REQUESTS,
                           sizeof(float));
#endif

  // allocate memory for the seqArray and reserve space
  {
    size_t qkv_max_proj_size =
        BatchConfig::MAX_NUM_TOKENS * BatchConfig::MAX_NUM_REQUESTS *
        (qProjSize * num_q_heads + kProjSize * num_kv_heads +
         vProjSize * num_kv_heads);
    size_t key_cache_size = 0, value_cache_size = 0;
    switch (infer_mode) {
      case INC_DECODING_MODE:
      case TREE_VERIFY_MODE: {
        key_cache_size = num_kv_heads * kProjSize *
                         BatchConfig::MAX_NUM_REQUESTS *
                         BatchConfig::MAX_SEQ_LENGTH;
        value_cache_size = num_kv_heads * vProjSize *
                           BatchConfig::MAX_NUM_REQUESTS *
                           BatchConfig::MAX_SEQ_LENGTH;
        break;
      }
      case BEAM_SEARCH_MODE: {
        key_cache_size =
            num_kv_heads * kProjSize * BeamSearchBatchConfig::MAX_NUM_REQUESTS *
            BatchConfig::MAX_SEQ_LENGTH * BeamSearchBatchConfig::MAX_BEAM_WIDTH;
        value_cache_size =
            num_kv_heads * vProjSize * BeamSearchBatchConfig::MAX_NUM_REQUESTS *
            BatchConfig::MAX_SEQ_LENGTH * BeamSearchBatchConfig::MAX_BEAM_WIDTH;
        break;
      }
      default:
        assert(false && "Unkown inference mode");
    }
    size_t tokeninfo_size =
        BatchConfig::MAX_NUM_TOKENS * BatchConfig::MAX_NUM_REQUESTS;
    size_t qk_prod_size = BatchConfig::MAX_NUM_TOKENS *
                          BatchConfig::MAX_NUM_REQUESTS *
                          BatchConfig::MAX_SEQ_LENGTH * num_q_heads;
    size_t attn_heads_size = BatchConfig::MAX_NUM_TOKENS *
                             BatchConfig::MAX_NUM_REQUESTS * num_q_heads *
                             vProjSize;
    size_t complex_size =
        (BatchConfig::MAX_NUM_TOKENS * BatchConfig::MAX_NUM_REQUESTS *
         (qProjSize * num_q_heads + kProjSize * num_kv_heads)) /
        2;
    size_t real_token_idx_size =
        BatchConfig::MAX_NUM_TOKENS * BatchConfig::MAX_NUM_REQUESTS;
    size_t total_tokens_per_req_size = BatchConfig::MAX_NUM_REQUESTS;
    size_t output_size =
        BatchConfig::MAX_NUM_TOKENS * BatchConfig::MAX_NUM_REQUESTS * oProjSize;
    size_t input_size = BatchConfig::MAX_NUM_TOKENS *
                        BatchConfig::MAX_NUM_REQUESTS * num_q_heads * qProjSize;
    size_t query_size = BatchConfig::MAX_NUM_TOKENS *
                        BatchConfig::MAX_NUM_REQUESTS * qProjSize * num_q_heads;
    size_t totalSize =
        (qkv_max_proj_size + key_cache_size + value_cache_size +
         2 * qk_prod_size + attn_heads_size) *
            size_of_dt +
        tokeninfo_size * sizeof(BatchConfig::PerTokenInfo) +
        complex_size * sizeof(hipFloatComplex) +
        real_token_idx_size * sizeof(int) +
        total_tokens_per_req_size * sizeof(int) +
        (key_cache_size + value_cache_size + output_size + query_size +
         input_size) *
            size_of_dt; // temporary storage for key, value, output
                        // more components will
                        // be added here later
    // memory can be shared across layers
    size_t totalSharedSize =
        infer_mode == TREE_VERIFY_MODE
            ? totalSize -
                  (key_cache_size + value_cache_size + qkv_max_proj_size) *
                      size_of_dt
            : totalSize - (key_cache_size + value_cache_size) * size_of_dt;
    // memory can't be shared across layers.
    size_t instance_size =
        size_of_dt *
        (infer_mode == TREE_VERIFY_MODE
             ? key_cache_size + value_cache_size + qkv_max_proj_size
             : key_cache_size + value_cache_size);

    if (offload) {
      // assert that we have enough reserved work space left
      if (quantization_type != DT_NONE) {
        totalSharedSize += quantized_weightSize;
      }
      assert(gpu_mem_allocator.reserved_total_size -
                 gpu_mem_allocator.reserved_allocated_size >=
             totalSharedSize);
    } else {
      assert(handle.workSpaceSize >= totalSharedSize);
    }

    gpu_mem_allocator.create_legion_instance(reserveInst, instance_size);
    // workspace for shared memory across layers
    char *work_space_start_ptr = (char *)handle.workSpace;

    // QKV need to be persistent in Tree_kernel.
    if (infer_mode == TREE_VERIFY_MODE) {
      devQKVProjArray = gpu_mem_allocator.allocate_instance_untyped(
          qkv_max_proj_size * size_of_dt);
    } else if (offload) {
      devQKVProjArray = gpu_mem_allocator.allocate_reserved_untyped(
          qkv_max_proj_size * size_of_dt);
    } else {
      // spec/inc + non-offload
      devQKVProjArray = work_space_start_ptr;
      work_space_start_ptr += qkv_max_proj_size * size_of_dt;
    }

    // use key value cache in all mode.
    keyCache = gpu_mem_allocator.allocate_instance_untyped(key_cache_size *
                                                           size_of_dt);
    valueCache = gpu_mem_allocator.allocate_instance_untyped(value_cache_size *
                                                             size_of_dt);

    if (offload) {
      token_infos =
          gpu_mem_allocator.allocate_reserved<BatchConfig::PerTokenInfo>(
              tokeninfo_size);
      // offset += sizeof(BatchConfig::PerTokenInfo) * tokeninfo_size;
      qk_prods = gpu_mem_allocator.allocate_reserved_untyped(qk_prod_size *
                                                             size_of_dt);
      // offset += qk_prod_size * size_of_dt;
      qk_prods_softmax = gpu_mem_allocator.allocate_reserved_untyped(
          qk_prod_size * size_of_dt);
      // offset += qk_prod_size * size_of_dt;
      attn_heads = gpu_mem_allocator.allocate_reserved_untyped(attn_heads_size *
                                                               size_of_dt);
      // offset += attn_heads_size * size_of_dt;
      complex_input =
          gpu_mem_allocator.allocate_reserved<hipFloatComplex>(complex_size);
      // offset += complex_size * sizeof(hipFloatComplex);
      real_token_idx =
          gpu_mem_allocator.allocate_reserved<int>(real_token_idx_size);
      total_tokens_per_req =
          gpu_mem_allocator.allocate_reserved<int>(total_tokens_per_req_size);

      key = gpu_mem_allocator.allocate_reserved_untyped(key_cache_size *
                                                        size_of_dt);
      query =
          gpu_mem_allocator.allocate_reserved_untyped(query_size * size_of_dt);
      value = gpu_mem_allocator.allocate_reserved_untyped(value_cache_size *
                                                          size_of_dt);

      padded_input =
          gpu_mem_allocator.allocate_reserved_untyped(input_size * size_of_dt);
      padded_output =
          gpu_mem_allocator.allocate_reserved_untyped(output_size * size_of_dt);
    } else {
      token_infos = static_cast<BatchConfig::PerTokenInfo *>(
          (void *)work_space_start_ptr);
      work_space_start_ptr +=
          sizeof(BatchConfig::PerTokenInfo) * tokeninfo_size;
      qk_prods = work_space_start_ptr;
      work_space_start_ptr += qk_prod_size * size_of_dt;
      qk_prods_softmax = work_space_start_ptr;
      work_space_start_ptr += qk_prod_size * size_of_dt;
      attn_heads = work_space_start_ptr;
      work_space_start_ptr += attn_heads_size * size_of_dt;
      complex_input =
          static_cast<hipFloatComplex *>((void *)work_space_start_ptr);
      work_space_start_ptr += sizeof(hipFloatComplex) * complex_size;

      real_token_idx_gpu = static_cast<int *>((void *)work_space_start_ptr);
      work_space_start_ptr += sizeof(int) * real_token_idx_size;
      total_tokens_per_req_gpu =
          static_cast<int *>((void *)work_space_start_ptr);
      work_space_start_ptr += sizeof(int) * total_tokens_per_req_size;

      query = work_space_start_ptr;
      work_space_start_ptr += query_size * size_of_dt;
      key = work_space_start_ptr;
      work_space_start_ptr += key_cache_size * size_of_dt;
      value = work_space_start_ptr;
      work_space_start_ptr += value_cache_size * size_of_dt;
      padded_output = work_space_start_ptr;
      work_space_start_ptr += output_size * size_of_dt;
      padded_input = work_space_start_ptr;
      work_space_start_ptr += input_size * size_of_dt;
    }

    // allocate more size for quantization data
    if (quantization_type != DT_NONE) {
      assert(offload);
      quantized_weight_ptr =
          gpu_mem_allocator.allocate_reserved<char>(quantized_weightSize);
    }
    if (!offload) {
      assert(gpu_mem_allocator.reserved_total_size ==
             gpu_mem_allocator.reserved_allocated_size);
    }
  }
  hipStreamSynchronize(stream);
}

IncMultiHeadSelfAttentionMeta::~IncMultiHeadSelfAttentionMeta(void) {
  if (reserveInst != Realm::RegionInstance::NO_INST) {
    reserveInst.destroy();
  }
#ifdef INFERENCE_TESTS
  free(kcache);
  free(vcache);
#endif
}

template void Kernels::IncMultiHeadAttention::pre_build_weight_kernel<float>(
    IncMultiHeadSelfAttentionMeta const *m,
    GenericTensorAccessorR const weight,
    DataType data_type,
    hipStream_t stream);

template void Kernels::IncMultiHeadAttention::pre_build_weight_kernel<half>(
    IncMultiHeadSelfAttentionMeta const *m,
    GenericTensorAccessorR const weight,
    DataType data_type,
    hipStream_t stream);

}; // namespace FlexFlow
