/* Copyright 2017 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"


Tensor FFModel::softmax(const Tensor& _input, int dim, const char *name)
{
  if (dim < 0)
    dim += _input.numDim;
  Softmax *sm = new Softmax(*this, _input, dim, name);
  layers.push_back(sm);
  return sm->outputs[0];
}

SoftmaxMeta::SoftmaxMeta(FFHandler handler,
                         const Softmax* softmax,
                         const Domain& input_domain)
: OpMeta(handler)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(cudnnSetTensorDescriptorFromDomain(inputTensor, input_domain));
  dim = softmax->dim;
  profiling = softmax->profiling;
  std::strcpy(op_name, softmax->name);
}

Softmax::Softmax(FFModel& model,
                 const Tensor& _input,
                 int _dim,
                 const char* name)
: Op(model, OP_SOFTMAX, name, _input),
  dim(_input.numDim-1-_dim)
{
  // Currently assume we always perform softmax along the inner most dim
  assert(dim == 0);
  outputs[0].numDim = _input.numDim;
  outputs[0].adim[0] = _input.adim[0];
  outputs[0].adim[1] = _input.adim[1];
}


void Softmax::create_weights(FFModel& model)
{
  // Do nothing since we don't ahve weights
}

void Softmax::create_output_and_partition(FFModel& model)
{
  int dim = outputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      create_output_and_partition_with_dim<DIM>(model); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
    {
      // Unsupported dim
      assert(false);
    }
  }
}

template<int NDIM>
void Softmax::create_output_and_partition_with_dim(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = IndexSpaceT<NDIM>(model.get_or_create_task_is(NDIM, pcname));
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<NDIM> part_rect = runtime->get_index_space_domain(ctx, task_is);
  int num_par_c = part_rect.hi[0] - part_rect.lo[0] + 1;
  //int num_par_n = part_rect.hi[1] - part_rect.lo[1] + 1;
  // Current require data parallelism for Softmax
  assert(num_par_c == 1);
  {
    int dims[NDIM];
    for (int i = 0; i < NDIM; i++)
      dims[i] = inputs[0].adim[NDIM-1-i];
    outputs[0] = model.create_tensor<NDIM>(dims, DT_FLOAT, this);
    outputs[0].owner_op = this;
    outputs[0].owner_idx = 0;
  }
  // Compute partition bound for input
  Rect<NDIM> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part.get_index_partition());
  if (input_rect == part_rect) {
    input_lps[0] = inputs[0].part;
    input_grad_lps[0] = inputs[0].part_grad;
  } else {
    model.create_disjoint_partition<NDIM>(
        inputs[0], (IndexSpaceT<NDIM>)task_is, input_lps[0], input_grad_lps[0]);
  }
}

/*
  regions[0]: input
  regions[1]: output
 */
OpMeta* Softmax::init_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  const Softmax* softmax = (Softmax*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  Domain input_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain output_domain = runtime->get_index_space_domain(
    ctx, task->regions[1].region.get_index_space());
  assert(input_domain == output_domain);
  SoftmaxMeta* m = new SoftmaxMeta(handle, softmax, output_domain);
  //checkCUDNN(hipdnnCreateTensorDescriptor(&m->outputTensor));
  return m;
}

__host__
void Softmax::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, task_is);
  ParallelConfig pc;
  std::string pcname = name;
  ff.config.find_parallel_config(2, pcname, pc);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    FFHandler handle = ff.handlers[pc.device_ids[idx++]];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
  }
  IndexLauncher launcher(SOFTMAX_INIT_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Softmax)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, outputs[0].region));
  launcher.add_field(1, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/* static */
void Softmax::forward_kernel(SoftmaxMeta const *m,
                             float const *input_ptr,
                             float *output_ptr,
                             hipStream_t stream)
{
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha = 1.0f, beta = 0.0f;
  checkCUDNN(hipdnnSoftmaxForward(m->handle.dnn,
                                 HIPDNN_SOFTMAX_ACCURATE,
                                 HIPDNN_SOFTMAX_MODE_CHANNEL,
                                 &alpha, m->inputTensor, input_ptr,
                                 &beta, m->inputTensor, output_ptr));
}

void Softmax::forward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return forward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

/*
  regions[0](I): input
  regions[1](O): output
*/
template<int NDIM>
__host__
void Softmax::forward_task_with_dim(
    const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  //const Softmax* softmax = (Softmax*) task->args;
  const SoftmaxMeta* m = *((SoftmaxMeta**) task->local_args);
  TensorAccessorR<float, NDIM> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, NDIM> acc_output(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);

  hipStream_t stream;
  checkCUDA(create_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  forward_kernel(m, acc_input.ptr, acc_output.ptr, stream);
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    //print_tensor<2, float>(acc_input.ptr, acc_input.rect, "[Softmax:forward:input]");
    //print_tensor<2, float>(acc_output.ptr, acc_output.rect, "[Softmax:forward:output]");
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Softmax] forward time = %.2fms\n", m->op_name, elapsed);
  }
}

__host__
void Softmax::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(SOFTMAX_FWD_TASK_ID, task_is,
                         TaskArgument(NULL, 0), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(1, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

/* static */
void Softmax::backward_kernel(float *input_grad_ptr,
                              float const *output_grad_ptr,
                              size_t num_elements,
                              hipStream_t stream)
{
  checkCUDA(hipMemcpyAsync(input_grad_ptr, output_grad_ptr,
                            num_elements * sizeof(float),
                            hipMemcpyDeviceToDevice, stream));
}

void Softmax::backward_task(const Task *task,
                            const std::vector<PhysicalRegion> &regions,
                            Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return backward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

/*
  regions[0](I/O): input_grad
  regions[1](I): output_grad
*/
// Note that the backward task of softmax is actually a no op (i.e., input_grad = output_grad)
// since the upstream cross_entropy_loss function computes performs softmax_cross_entropy_loss
// to avoid intermediate zeros
template<int NDIM>
__host__
void Softmax::backward_task_with_dim(
    const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  //const Softmax* softmax = (Softmax*) task->args;
  const SoftmaxMeta* m = *((SoftmaxMeta**) task->local_args);
  TensorAccessorW<float, NDIM> acc_input_grad(
      regions[0], task->regions[0], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, NDIM> acc_output_grad(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  // make sure the image indices match!
  assert(acc_input_grad.rect == acc_output_grad.rect);

  hipStream_t stream;
  checkCUDA(create_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  backward_kernel(acc_input_grad.ptr, acc_output_grad.ptr, acc_input_grad.rect.volume(), stream);
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    //print_tensor<2, float>(acc_output_grad.ptr, acc_output_grad.rect, "[Softmax:backward:output_grad]");
    //print_tensor<2, float>(acc_input_grad.ptr, acc_input_grad.rect, "[Softmax:backward:input_grad]");
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Softmax backward time = %.2fms\n", elapsed);
  }
}

__host__
void Softmax::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(SOFTMAX_BWD_TASK_ID, task_is,
                         TaskArgument(NULL, 0), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, outputs[0].region_grad));
  launcher.add_field(1, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

bool Softmax::measure_operator_cost(Simulator* sim,
                                    const ParallelConfig& pc,
                                    CostMetrics& cost_metrics)
{
  Tensor sub_output, sub_input;
  if (!outputs[0].get_output_sub_tensor(pc, sub_output, op_type)) {
    return false;
  }
  if (!inputs[0].get_input_sub_tensor(pc, sub_input, op_type)) {
    return false;
  }

  SoftmaxMeta *m = new SoftmaxMeta(sim->handler, this, sub_output.get_domain());

  sim->free_all();
  float *input_ptr = (float *)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert (input_ptr != NULL);
  float *output_ptr = (float *)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert (output_ptr != NULL);

  hipStream_t stream;
  checkCUDA(create_stream(&stream));
  std::function<void()> forward, backward;
  forward = [&] {
    forward_kernel(m, input_ptr, output_ptr, stream);
  };
  if (sim->computationMode == COMP_MODE_TRAINING) {
    float* input_grad_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
    assert(input_grad_ptr != NULL);
    float *output_grad_ptr = (float *)sim->allocate(sub_output.get_volume(), DT_FLOAT);
    assert (output_grad_ptr != NULL);
    backward = [&] {
      backward_kernel(input_grad_ptr, output_grad_ptr, sub_output.get_volume(), stream);
    };
  }

  inner_measure_operator_cost(sim, forward, backward, cost_metrics);

  if (sim->computationMode == COMP_MODE_TRAINING) {
    printf("[Measure Softmax] name(%s) num_elements(%zu) forward_time(%.4lf) backward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time,
        cost_metrics.backward_time);
  } else {
    printf("[Measure Softmax] name(%s) num_elements(%zu) forward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time);
  }
  // Free softmaxmeta
  delete m;
  return true;
}
