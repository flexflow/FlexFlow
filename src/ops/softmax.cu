/* Copyright 2017 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

using namespace Legion;

Tensor FFModel::softmax(const Tensor _input, int dim, const char *name)
{
  if (dim < 0)
    dim += _input->num_dims;
  Softmax *sm = new Softmax(*this, _input, _input->num_dims-1-dim, name);
  layers.push_back(sm);
  return sm->outputs[0];
}

SoftmaxMeta::SoftmaxMeta(FFHandler handler,
                         const Softmax* softmax,
                         const Domain& input_domain)
: OpMeta(handler)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(cudnnSetTensorDescriptorFromDomain(inputTensor, input_domain));
  dim = softmax->dim;
  profiling = softmax->profiling;
  std::strcpy(op_name, softmax->name);
}

Softmax::Softmax(FFModel& model,
                 const Tensor _input,
                 int _dim,
                 const char* name)
: Op(model, OP_SOFTMAX, name, 1/*inputs*/, 0/*weights*/, _input),
  dim(_dim)
{
  // Currently assume we always perform softmax along the inner most dim
  assert(dim == 0);
  ParallelDim dims[MAX_TENSOR_DIM];
  int numdim = _input->num_dims;
  for (int i = 0; i < numdim; i++)
    dims[i] = _input->dims[numdim-1-i];
  outputs[0] = model.create_tensor(numdim, dims, DT_FLOAT, this);
}

void Softmax::create_input_partition(FFModel& model)
{
  int dim = outputs[0]->num_dims;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      create_output_and_partition_with_dim<DIM>(model); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
    {
      // Unsupported dim
      assert(false);
    }
  }
}

template<int NDIM>
void Softmax::create_output_and_partition_with_dim(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = IndexSpaceT<NDIM>(model.get_or_create_task_is(NDIM, pcname));
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<NDIM> part_rect = runtime->get_index_space_domain(ctx, task_is);
  int num_par_c = part_rect.hi[0] - part_rect.lo[0] + 1;
  //int num_par_n = part_rect.hi[1] - part_rect.lo[1] + 1;
  // Current require data parallelism for Softmax
  assert(num_par_c == 1);
  return Op::create_input_partition(model);
#ifdef DEADCODE
  {
    int dims[NDIM];
    for (int i = 0; i < NDIM; i++)
      dims[i] = outputs[0].adim[NDIM-1-i];
    outputs[0] = model.create_tensor<NDIM>(dims, DT_FLOAT, this);
    outputs[0].owner_op = this;
    outputs[0].owner_idx = 0;
  }
  // Compute partition bound for input
  Rect<NDIM> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part.get_index_partition());
  if (input_rect == part_rect) {
    input_lps[0] = inputs[0]->part;
    input_grad_lps[0] = inputs[0]->part_grad;
  } else {
    model.create_disjoint_partition<NDIM>(
        inputs[0], (IndexSpaceT<NDIM>)task_is, input_lps[0], input_grad_lps[0]);
  }
#endif
}

/*
  regions[0]: input
  regions[1]: output
 */
OpMeta* Softmax::init_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  const Softmax* softmax = (Softmax*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  Domain input_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain output_domain = runtime->get_index_space_domain(
    ctx, task->regions[1].region.get_index_space());
  assert(input_domain == output_domain);
  SoftmaxMeta* m = new SoftmaxMeta(handle, softmax, output_domain);
  //checkCUDNN(hipdnnCreateTensorDescriptor(&m->outputTensor));
  return m;
}

__host__
void Softmax::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_init(ff, argmap);
  IndexLauncher launcher(SOFTMAX_INIT_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Softmax)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0]->region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, outputs[0]->region));
  launcher.add_field(1, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  set_opmeta_from_futuremap(ff, fm);
}

/* static */
void Softmax::forward_kernel(SoftmaxMeta const *m,
                             float const *input_ptr,
                             float *output_ptr)
{
  float alpha = 1.0f, beta = 0.0f;
  checkCUDNN(hipdnnSoftmaxForward(m->handle.dnn,
                                 HIPDNN_SOFTMAX_ACCURATE,
                                 HIPDNN_SOFTMAX_MODE_CHANNEL,
                                 &alpha, m->inputTensor, input_ptr,
                                 &beta, m->inputTensor, output_ptr));
}

void Softmax::forward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return forward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

/*
  regions[0](I): input
  regions[1](O): output
*/
template<int NDIM>
__host__
void Softmax::forward_task_with_dim(
    const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  //const Softmax* softmax = (Softmax*) task->args;
  const SoftmaxMeta* m = *((SoftmaxMeta**) task->local_args);
  TensorAccessorR<float, NDIM> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, NDIM> acc_output(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  forward_kernel(m, acc_input.ptr, acc_output.ptr);
  if (m->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    //print_tensor<2, float>(acc_input.ptr, acc_input.rect, "[Softmax:forward:input]");
    //print_tensor<2, float>(acc_output.ptr, acc_output.rect, "[Softmax:forward:output]");
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Softmax] forward time = %.2fms\n", m->op_name, elapsed);
  }
}

__host__
void Softmax::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_forward(ff, argmap);
  IndexLauncher launcher(SOFTMAX_FWD_TASK_ID, task_is,
                         TaskArgument(NULL, 0), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0]->region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0]->region));
  launcher.add_field(1, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

/* static */
void Softmax::backward_kernel(float *input_grad_ptr,
                              float const *output_grad_ptr,
                              size_t num_elements)
{
  checkCUDA(hipMemcpyAsync(input_grad_ptr, output_grad_ptr,
                            num_elements * sizeof(float),
                            hipMemcpyDeviceToDevice));
}

void Softmax::backward_task(const Task *task,
                            const std::vector<PhysicalRegion> &regions,
                            Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return backward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

/*
  regions[0](I/O): input_grad
  regions[1](I): output_grad
*/
// Note that the backward task of softmax is actually a no op (i.e., input_grad = output_grad)
// since the upstream cross_entropy_loss function computes performs softmax_cross_entropy_loss
// to avoid intermediate zeros
template<int NDIM>
__host__
void Softmax::backward_task_with_dim(
    const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  //const Softmax* softmax = (Softmax*) task->args;
  const SoftmaxMeta* m = *((SoftmaxMeta**) task->local_args);
  TensorAccessorW<float, NDIM> acc_input_grad(
      regions[0], task->regions[0], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, NDIM> acc_output_grad(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  // make sure the image indices match!
  assert(acc_input_grad.rect == acc_output_grad.rect);

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  //checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  backward_kernel(acc_input_grad.ptr, acc_output_grad.ptr, acc_input_grad.rect.volume());
  if (m->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    //print_tensor<2, float>(acc_output_grad.ptr, acc_output_grad.rect, "[Softmax:backward:output_grad]");
    //print_tensor<2, float>(acc_input_grad.ptr, acc_input_grad.rect, "[Softmax:backward:input_grad]");
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Softmax backward time = %.2fms\n", elapsed);
  }
}

__host__
void Softmax::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_backward(ff, argmap);
  IndexLauncher launcher(SOFTMAX_BWD_TASK_ID, task_is,
                         TaskArgument(NULL, 0), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, inputs[0]->region_grad));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part_grad, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, outputs[0]->region_grad));
  launcher.add_field(1, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

bool Softmax::measure_operator_cost(Simulator* sim,
                                    const ParallelConfig& pc,
                                    CostMetrics& cost_metrics) const
{
  TensorBase sub_output, sub_input;
  if (!outputs[0]->get_output_sub_tensor(pc, sub_output, op_type)) {
    return false;
  }
  if (!inputs[0]->get_input_sub_tensor(pc, sub_input, op_type)) {
    return false;
  }

  SoftmaxMeta *m = new SoftmaxMeta(sim->handler, this, sub_output.get_domain());

  sim->free_all();
  float *input_ptr = (float *)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert (input_ptr != NULL);
  float *output_ptr = (float *)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert (output_ptr != NULL);

  std::function<void()> forward, backward;
  forward = [&] {
    forward_kernel(m, input_ptr, output_ptr);
  };
  if (sim->computationMode == COMP_MODE_TRAINING) {
    float* input_grad_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
    assert(input_grad_ptr != NULL);
    float *output_grad_ptr = (float *)sim->allocate(sub_output.get_volume(), DT_FLOAT);
    assert (output_grad_ptr != NULL);
    backward = [&] {
      backward_kernel(input_grad_ptr, output_grad_ptr, sub_output.get_volume());
    };
  }

  inner_measure_operator_cost(sim, forward, backward, cost_metrics);

  if (sim->computationMode == COMP_MODE_TRAINING) {
    printf("[Measure Softmax] name(%s) num_elements(%zu) forward_time(%.4lf) backward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time,
        cost_metrics.backward_time);
  } else {
    printf("[Measure Softmax] name(%s) num_elements(%zu) forward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time);
  }
  // Free softmaxmeta
  delete m;
  return true;
}

bool Softmax::get_int_parameter(PMParameter para, int* value) const
{
  switch(para) {
    case PM_SOFTMAX_DIM:
      *value = dim;
      return true;
    default:
      return Op::get_int_parameter(para, value);
  }
}

Node FFModel::get_or_create_softmax_node(const Tensor input,
                                         int softmax_dim)
{
  size_t hash = input->get_owner_independent_hash();
  hash = hash * 31 + std::hash<int>()(softmax_dim);
  const auto& it = cached_softmax_ops.find(hash);
  Softmax* softmax = NULL;
  if (it != cached_softmax_ops.end()) {
    softmax = it->second;
  } else {
    softmax = new Softmax(*this, input, softmax_dim, NULL);
    cached_softmax_ops[hash] = softmax;
  }
  Node ret;
  ret.guid = node_global_guid ++;
  ret.ptr = softmax;
  return ret;
}
