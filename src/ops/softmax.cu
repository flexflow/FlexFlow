/* Copyright 2017 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/softmax.h"
#include "flexflow/utils/cuda_helper.h"
#include "flexflow/utils/hash_utils.h"

namespace FlexFlow {
// declare Legion names
using Legion::Context;
using Legion::Runtime;
using Legion::Domain;
using Legion::Task;
using Legion::Rect;
using Legion::PhysicalRegion;
using Legion::coord_t;

SoftmaxMeta::SoftmaxMeta(FFHandler handler,
                         const Softmax* softmax,
                         const Domain& input_domain)
: OpMeta(handler)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(cudnnSetTensorDescriptorFromDomain(inputTensor, input_domain));
  dim = softmax->dim;
  profiling = softmax->profiling;
  std::strcpy(op_name, softmax->name);
}

/*
  regions[0]: input
  regions[1]: output
 */
OpMeta* Softmax::init_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  const Softmax* softmax = (Softmax*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  Domain input_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain output_domain = runtime->get_index_space_domain(
    ctx, task->regions[1].region.get_index_space());
  assert(input_domain == output_domain);
  SoftmaxMeta* m = new SoftmaxMeta(handle, softmax, output_domain);
  //checkCUDNN(hipdnnCreateTensorDescriptor(&m->outputTensor));
  return m;
}


/* static */
void Softmax::forward_kernel(SoftmaxMeta const *m,
                             float const *input_ptr,
                             float *output_ptr,
                             hipStream_t stream)
{
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha = 1.0f, beta = 0.0f;
  checkCUDNN(hipdnnSoftmaxForward(m->handle.dnn,
                                 HIPDNN_SOFTMAX_ACCURATE,
                                 HIPDNN_SOFTMAX_MODE_CHANNEL,
                                 &alpha, m->inputTensor, input_ptr,
                                 &beta, m->inputTensor, output_ptr));
}

void Softmax::forward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return forward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

/*
  regions[0](I): input
  regions[1](O): output
*/
template<int NDIM>
__host__
void Softmax::forward_task_with_dim(
    const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  //const Softmax* softmax = (Softmax*) task->args;
  const SoftmaxMeta* m = *((SoftmaxMeta**) task->local_args);
  TensorAccessorR<float, NDIM> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, NDIM> acc_output(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  forward_kernel(m, acc_input.ptr, acc_output.ptr, stream);
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    //print_tensor<2, float>(acc_input.ptr, acc_input.rect, "[Softmax:forward:input]");
    //print_tensor<2, float>(acc_output.ptr, acc_output.rect, "[Softmax:forward:output]");
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    log_measure.debug("%s [Softmax] forward time = %.2fms\n", m->op_name, elapsed);
  }
}

/* static */
void Softmax::backward_kernel(float *input_grad_ptr,
                              float const *output_grad_ptr,
                              size_t num_elements,
                              hipStream_t stream)
{
  checkCUDA(hipMemcpyAsync(input_grad_ptr, output_grad_ptr,
                            num_elements * sizeof(float),
                            hipMemcpyDeviceToDevice, stream));
}

void Softmax::backward_task(const Task *task,
                            const std::vector<PhysicalRegion> &regions,
                            Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return backward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

/*
  regions[0](I/O): input_grad
  regions[1](I): output_grad
*/
// Note that the backward task of softmax is actually a no op (i.e., input_grad = output_grad)
// since the upstream cross_entropy_loss function computes performs softmax_cross_entropy_loss
// to avoid intermediate zeros
template<int NDIM>
__host__
void Softmax::backward_task_with_dim(
    const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  //const Softmax* softmax = (Softmax*) task->args;
  const SoftmaxMeta* m = *((SoftmaxMeta**) task->local_args);
  TensorAccessorW<float, NDIM> acc_input_grad(
      regions[0], task->regions[0], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, NDIM> acc_output_grad(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  // make sure the image indices match!
  assert(acc_input_grad.rect == acc_output_grad.rect);

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  backward_kernel(acc_input_grad.ptr, acc_output_grad.ptr, acc_input_grad.rect.volume(), stream);
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    //print_tensor<2, float>(acc_output_grad.ptr, acc_output_grad.rect, "[Softmax:backward:output_grad]");
    //print_tensor<2, float>(acc_input_grad.ptr, acc_input_grad.rect, "[Softmax:backward:input_grad]");
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    log_measure.debug("Softmax backward time = %.2fms\n", elapsed);
  }
}

bool Softmax::measure_operator_cost(Simulator* sim,
                                    const ParallelConfig& pc,
                                    CostMetrics& cost_metrics) const
{
  TensorBase sub_output, sub_input;
  if (!outputs[0]->get_output_sub_tensor(pc, sub_output, op_type)) {
    return false;
  }
  if (!inputs[0]->get_input_sub_tensor(pc, sub_input, op_type)) {
    return false;
  }

  SoftmaxMeta *m = new SoftmaxMeta(sim->handler, this, sub_output.get_domain());

  sim->free_all();
  float *input_ptr = (float *)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert (input_ptr != NULL);
  float *output_ptr = (float *)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert (output_ptr != NULL);

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  std::function<void()> forward, backward;
  forward = [&] {
    forward_kernel(m, input_ptr, output_ptr, stream);
  };
  if (sim->computationMode == COMP_MODE_TRAINING) {
    float* input_grad_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
    assert(input_grad_ptr != NULL);
    float *output_grad_ptr = (float *)sim->allocate(sub_output.get_volume(), DT_FLOAT);
    assert (output_grad_ptr != NULL);
    backward = [&] {
      backward_kernel(input_grad_ptr, output_grad_ptr, sub_output.get_volume(), stream);
    };
  }

  inner_measure_operator_cost(sim, forward, backward, cost_metrics);

  if (sim->computationMode == COMP_MODE_TRAINING) {
    log_measure.debug("[Measure Softmax] name(%s) num_elements(%zu) forward_time(%.4lf) backward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time,
        cost_metrics.backward_time);
  } else {
    log_measure.debug("[Measure Softmax] name(%s) num_elements(%zu) forward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time);
  }
  // Free softmaxmeta
  delete m;
  return true;
}

}; // namespace FlexFlow