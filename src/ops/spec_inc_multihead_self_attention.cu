#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#if defined(FF_USE_CUDA) || defined(FF_USE_HIP_CUDA)
#include "hip/hip_complex.h"
#endif
#include "flexflow/ffconst_utils.h"
#include "flexflow/ops/kernels/inc_multihead_self_attention_kernels.h"
#include "flexflow/ops/spec_inc_multihead_self_attention.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

// declare Legion names
using Legion::coord_t;
using Legion::Memory;
using namespace Kernels::IncMultiHeadAttention;

namespace Kernels {
namespace SpecIncMultiHeadAttention {

template <typename DT>
__global__ void spec_store_kv_cache(
    DT const *devQKVProjArray,
    DT *cache_ptr,
    BatchConfig::PerTokenInfo *tokenInfos,
    BatchConfig::PerRequestInfo *requestInfo,
    BeamSearchBatchConfig::BeamSearchPerTokenInfo *beamTokenInfos,
    BeamSearchBatchConfig::BeamSearchPerRequestInfo *beamRequestInfos,
    int qProjSize,
    int kProjSize,
    int vProjSize,
    int num_tokens,
    int num_heads,
    int max_seq_len,
    int max_beam_width,
    bool k_cache,
    bool is_root) {
  CUDA_KERNEL_LOOP(i,
                   num_tokens * (k_cache ? kProjSize : vProjSize) * num_heads) {
    int proj_size = k_cache ? kProjSize : vProjSize;
    int head_idx = i / (num_tokens * proj_size);
    int token_idx = (i - head_idx * (num_tokens * proj_size)) / proj_size;
    int data_idx = i % proj_size;

    int qkv_block_size = (qProjSize + kProjSize + vProjSize) * num_tokens;
    int current_head_block_size =
        num_tokens * (k_cache ? qProjSize : qProjSize + kProjSize);
    DT val =
        devQKVProjArray[head_idx * qkv_block_size + current_head_block_size +
                        token_idx * proj_size + data_idx];

    // above no need to be changed
    // int const req_id = id_map[token_idx].request_index;
    // int const tok_id = id_map[token_idx].token_position;
    // int const sub_req_id = id_map[token_idx].sub_request_index;
    // int const parent_id = id_map[token_idx].parent_id;
    // int const beam_depth = id_map[token_idx].beam_depth;
    // int const beam_width = id_map[token_idx].beam_width;

    int const req_id = tokenInfos[token_idx].request_index;
    int const tok_id = tokenInfos[token_idx].abs_depth_in_request;
    int const sub_req_id = beamTokenInfos[token_idx].sub_request_index;
    int const parent_id = beamRequestInfos[req_id].parent_id[sub_req_id];
    int const beam_depth = beamRequestInfos[req_id].current_depth;
    int const beam_width = beamRequestInfos[req_id].beam_size;

    // new token
    int new_token_cache_idx = (req_id * max_beam_width + sub_req_id) *
                                  (num_heads * max_seq_len * proj_size) +
                              head_idx * (max_seq_len * proj_size) +
                              tok_id * proj_size + data_idx;
    cache_ptr[new_token_cache_idx] = val;

    // replica in the root iteration
    if (beam_depth == 1) {
      for (int i = 1; i < beam_width; i++) {
        cache_ptr[(req_id * max_beam_width + i) *
                      (num_heads * max_seq_len * proj_size) +
                  head_idx * (max_seq_len * proj_size) + tok_id * proj_size +
                  data_idx] = val;
      }
    }

    // if (head_idx == 0 && beam_depth == 0 && token_idx == 8 && k_cache) {
    //   // printf("token idx %d\n", token_idx);
    //   printf("data idx: %d, tok_id %d, new_token_cache_idx %d, parent_id %d,
    //   "
    //          "sub_req_id %d, num_tokens %d, kProjSize %d, num_heads %d, val "
    //          "%f, beam_width %d\n",
    //          data_idx,
    //          tok_id,
    //          new_token_cache_idx,
    //          parent_id,
    //          sub_req_id,
    //          num_tokens,
    //          kProjSize,
    //          num_heads,
    //          val,
    //          beam_width);
    // }

    // naive cache stealing
    if (sub_req_id != parent_id) {
      if (data_idx == 0 && head_idx == 0 && k_cache) {
        printf("cache stealing!, depth %d req_id %d sub_req_id %d, parentid "
               "%d, tok_id %d\n",
               beam_depth,
               req_id,
               sub_req_id,
               parent_id,
               tok_id);
      }

      for (int depth = 0; depth < beam_depth; depth++) {
        int steal_token_idx = tok_id - beam_depth + depth;
        int steal_from_idx = (req_id * max_beam_width + parent_id) *
                                 (num_heads * max_seq_len * proj_size) +
                             head_idx * (max_seq_len * proj_size) +
                             steal_token_idx * proj_size + data_idx;
        int steal_to_idx = (req_id * max_beam_width + sub_req_id) *
                               (num_heads * max_seq_len * proj_size) +
                           head_idx * (max_seq_len * proj_size) +
                           steal_token_idx * proj_size + data_idx;
        cache_ptr[steal_to_idx] = cache_ptr[steal_from_idx];

        //   if(data_idx == 0 && head_idx == 0 && k_cache && req_id == 1){
        //     printf("cache stealing kernel!, steal_token_idx %d\n",
        //     steal_token_idx);
        // }
      }
    }

    // parallel cache stealing not yet implemented
    // logic shld be
    // launch spec_store_kv_cache with parallelism * current depth
    // from the i here, get depth index
    // if depth index not the current one, check if we need to steal
    // steal if needed

    // cache stealing theory
    // identify which sub request does this token come from
    // for initial token, 0
    // for other, may 0,0,1/ 0,1,2/ 1,1,1 to get which cache to be reuse and
    // which to be delete copy beam_size bunch of blocks when sub_req_id ==
    // parent_id : like 0 -> 0, 1->1, 2->2, do nothing, just append the new k/v
  }
}

template <typename DT>
void update_kv_cache_kernel(SpecIncMultiHeadSelfAttentionMeta const *m,
                            BeamSearchBatchConfig const *bc,
                            hipStream_t stream) {
  int num_tokens = bc->num_active_tokens();
  int curr_depth = bc->beamRequestsInfo[0].current_depth;
  // printf("curr depth: %d\n", curr_depth);
  // assert(curr_depth < 3);
  if (num_tokens > 0) {
    int parallelism = m->kProjSize * num_tokens * m->num_heads;
    spec_store_kv_cache<<<GET_BLOCKS(parallelism),
                          min(CUDA_NUM_THREADS, parallelism),
                          0,
                          stream>>>(static_cast<DT *>(m->devQKVProjArray),
                                    static_cast<DT *>(m->keyCache),
                                    m->token_infos,
                                    m->request_infos,
                                    m->beam_token_infos,
                                    m->beam_request_infos,
                                    m->qProjSize,
                                    m->kProjSize,
                                    m->vProjSize,
                                    num_tokens,
                                    m->num_heads,
                                    BatchConfig::MAX_SEQ_LENGTH,
                                    BeamSearchBatchConfig::MAX_BEAM_WIDTH,
                                    /* k_cache = */ true,
                                    /*root*/ curr_depth == 0);

    parallelism = m->vProjSize * num_tokens * m->num_heads;
    spec_store_kv_cache<<<GET_BLOCKS(parallelism),
                          min(CUDA_NUM_THREADS, parallelism),
                          0,
                          stream>>>(static_cast<DT *>(m->devQKVProjArray),
                                    static_cast<DT *>(m->valueCache),
                                    m->token_infos,
                                    m->request_infos,
                                    m->beam_token_infos,
                                    m->beam_request_infos,
                                    m->qProjSize,
                                    m->kProjSize,
                                    m->vProjSize,
                                    num_tokens,
                                    m->num_heads,
                                    BatchConfig::MAX_SEQ_LENGTH,
                                    BeamSearchBatchConfig::MAX_BEAM_WIDTH,
                                    /* k_cache = */ false,
                                    /*root*/ curr_depth == 0);
  }
}

template <typename DT>
__global__ void spec_fill_entries_above_diagonal(DT *matrix,
                                                 size_t new_tokens,
                                                 size_t total_tokens_in_request,
                                                 size_t num_heads,
                                                 DT value) {
  CUDA_KERNEL_LOOP(i, new_tokens * total_tokens_in_request * num_heads) {
    // size_t head_idx = i / (new_tokens * total_tokens_in_request);
    size_t src_idx = (i / new_tokens) % total_tokens_in_request;
    size_t dst_idx = i % new_tokens + total_tokens_in_request - new_tokens;
    // Casual Mask
    if (src_idx > dst_idx) {
      matrix[i] = value;
    }
  }
}

template <typename DT>
void compute_attention_kernel(SpecIncMultiHeadSelfAttentionMeta const *m,
                              BeamSearchBatchConfig const *bc,
                              DT *output_ptr,
                              DT const *bias_ptr,
                              hipStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  hipDataType cublas_data_type = ff_to_cuda_datatype(m->output_type[0]);
  hipdnnDataType_t cudnn_data_type = ff_to_cudnn_datatype(m->output_type[0]);
  assert(data_type_size(m->output_type[0]) == sizeof(DT));
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = cublas_data_type;
#endif
  // int num_requests = bc->num_active_requests();
  int num_tokens = bc->num_active_tokens();
  int tokens_previous_requests = 0;
  int tokens_prev_requests_squares = 0;
  int qkv_block_size =
      (m->qProjSize + m->kProjSize + m->vProjSize) * num_tokens;
  int kt_block_size = m->kProjSize * BatchConfig::MAX_SEQ_LENGTH;
  int kt_req_block_size = kt_block_size * m->num_heads;
  int vt_block_size = m->vProjSize * BatchConfig::MAX_SEQ_LENGTH;
  int vt_req_block_size = vt_block_size * m->num_heads;
  assert(m->qProjSize == m->kProjSize);

  for (int i = 0; i < bc->MAX_NUM_REQUESTS; i++) {
    if (bc->request_completed[i]) {
      continue;
    }
    for (int sub_req_id = 0; sub_req_id < bc->sub_requests[i]; sub_req_id++) {

      // int num_new_tokens = bc->num_processing_tokens[i];
      // int total_tokens = bc->token_last_available_idx[i] + 1;

      int num_new_tokens = bc->requestsInfo[i].num_tokens_in_batch;
      int total_tokens = bc->requestsInfo[i].token_start_offset +
                         bc->requestsInfo[i].num_tokens_in_batch;
      // Compute (QK^T/sqrt(d_k))
      int m_ = num_new_tokens;
      int n = total_tokens;
      int k = m->qProjSize;
      int lda = k, ldb = k, ldc = m_;
      int strideA = qkv_block_size;
      int strideB = kt_block_size;
      int strideC = num_new_tokens * total_tokens;

      // a flag of using this scaling alpha
      float alpha = 1.0f, beta = 0.0f;
      if (*m->qk_prod_scaling) {
        alpha = 1.0f / (float)sqrt(m->kProjSize), beta = 0.0f;
      }
      // To get A, skip over Q entries from previous requests (same head)
      void const *A = static_cast<DT *>(m->devQKVProjArray) +
                      tokens_previous_requests * m->qProjSize;
      // To get B, skip over K entries from previous requests (all heads +
      // padding)
      void const *B = static_cast<DT *>(m->keyCache) +
                      (i * bc->MAX_BEAM_WIDTH + sub_req_id) * kt_req_block_size;

      // if (i == 0 && sub_req_id == 0 &&
      //     bc->beam_slots.at(0).current_depth == 1) {
      //   int offset = (float *)B - m->keyCache;
      //   printf("key cache offset %d\n", kt_req_block_size);
      // }
      // To get C, skip over QK^T products from previous requests
      void *C = static_cast<DT *>(m->qk_prods) +
                m->num_heads * tokens_prev_requests_squares;

      checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                           HIPBLAS_OP_T,
                                           HIPBLAS_OP_N,
                                           m_,
                                           n,
                                           k,
                                           &alpha,
                                           A,
                                           cublas_data_type,
                                           lda,
                                           strideA,
                                           B,
                                           cublas_data_type,
                                           ldb,
                                           strideB,
                                           &beta,
                                           C,
                                           cublas_data_type,
                                           ldc,
                                           strideC,
                                           m->num_heads,
                                           compute_type,
                                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
      // Fill all elements above diagonal in qk prods with -inf to force
      // causal attention.
      assert(num_new_tokens <= total_tokens);
      if (num_new_tokens > 1) {
        size_t parallelism = m->num_heads * num_new_tokens * total_tokens;
        spec_fill_entries_above_diagonal<<<GET_BLOCKS(parallelism),
                                           min((size_t)CUDA_NUM_THREADS,
                                               parallelism),
                                           0,
                                           stream>>>(
            static_cast<DT *>(C),
            num_new_tokens,
            total_tokens,
            m->num_heads,
            static_cast<DT>(-INFINITY));
      }
      // Compute Softmax(QK^T/sqrt(d_k))
      hipdnnTensorDescriptor_t qk_tensor;
      checkCUDNN(hipdnnCreateTensorDescriptor(&qk_tensor));
      // Before modifying the parameters below, make sure to read the following
      // description of the HIPDNN_TENSOR_NCHW tensor layout, from
      // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnTensorFormat_t:
      // This tensor format specifies that the data is laid out in the following
      // order: batch size, feature maps, rows, columns. The strides are
      // implicitly defined in such a way that the data are contiguous in memory
      // with no padding between images, feature maps, rows, and columns; the
      // columns are the inner dimension and the images are the outermost
      // dimension.
      int n_param = m->num_heads;
      int c_param = total_tokens;
      int h_param = 1;
      int w_param = num_new_tokens;
      checkCUDNN(hipdnnSetTensor4dDescriptor(qk_tensor,
                                            HIPDNN_TENSOR_NCHW,
                                            HIPDNN_DATA_FLOAT,
                                            n_param,
                                            c_param,
                                            h_param,
                                            w_param));
      alpha = 1.0f, beta = 0.0f;
      void *C_softmax = static_cast<DT *>(m->qk_prods_softmax) +
                        m->num_heads * tokens_prev_requests_squares;
      // The softmax operation below is executed according to the
      // HIPDNN_SOFTMAX_MODE_CHANNEL, which is also described in the docs: The
      // softmax operation is computed per spatial location (H,W) per image (N)
      // across dimension C.
      checkCUDNN(hipdnnSoftmaxForward(m->handle.dnn,
                                     HIPDNN_SOFTMAX_ACCURATE,
                                     HIPDNN_SOFTMAX_MODE_CHANNEL,
                                     &alpha,
                                     qk_tensor,
                                     C,
                                     &beta,
                                     qk_tensor,
                                     C_softmax));
      // Matmul softmax(QK^T/sqrt(d_k)) by V
      alpha = 1.0f, beta = 0.0f;
      m_ = num_new_tokens;
      n = m->vProjSize;
      k = total_tokens;
      lda = m_, ldb = n, ldc = m_;
      strideA = num_new_tokens * total_tokens;
      strideB = vt_block_size;
      strideC = num_new_tokens * m->vProjSize;
      // To get A, skip over softmax(QK^T/sqrt(d_k)) entries from previous
      // requests (all heads)
      A = (void const *)C_softmax;
      // To get B, skip over V^T entries from previous requests (all heads +
      // padding)
      B = static_cast<DT *>(m->valueCache) +
          (i * bc->MAX_BEAM_WIDTH + sub_req_id) * vt_req_block_size;
      // To get C, skip over softmax(QK^T/sqrt(d_k))V products from previous
      // requests
      C = static_cast<DT *>(m->attn_heads) +
          tokens_previous_requests * m->num_heads * m->vProjSize;

      checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                           HIPBLAS_OP_N,
                                           HIPBLAS_OP_T,
                                           m_,
                                           n,
                                           k,
                                           &alpha,
                                           A,
                                           cublas_data_type,
                                           lda,
                                           strideA,
                                           B,
                                           cublas_data_type,
                                           ldb,
                                           strideB,
                                           &beta,
                                           C,
                                           cublas_data_type,
                                           ldc,
                                           strideC,
                                           m->num_heads,
                                           compute_type,
                                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
      // Project to output, save result directly on output tensor
      alpha = 1.0f, beta = 0.0f;
      m_ = m->oProjSize;
      k = m->vProjSize * m->num_heads;
      n = num_new_tokens;
      lda = k, ldb = n, ldc = m_;
      A = (void const *)m->W_out_contiguous;
      B = (void const *)C;
      C = (void *)(output_ptr + tokens_previous_requests * m->oProjSize);

      checkCUDA(hipblasGemmEx(m->handle.blas,
                             HIPBLAS_OP_T,
                             HIPBLAS_OP_T,
                             m_,
                             n,
                             k,
                             &alpha,
                             A,
                             cublas_data_type,
                             lda,
                             B,
                             cublas_data_type,
                             ldb,
                             &beta,
                             C,
                             cublas_data_type,
                             ldc,
                             compute_type,
                             CUBLAS_GEMM_DEFAULT_TENSOR_OP));
      tokens_previous_requests += num_new_tokens;
      tokens_prev_requests_squares += num_new_tokens * total_tokens;
    }
    if (*m->bias) {
      int parallelism = m->oProjSize * num_tokens;
      apply_proj_bias_w<<<GET_BLOCKS(parallelism),
                          min(CUDA_NUM_THREADS, parallelism),
                          0,
                          stream>>>(
          output_ptr, bias_ptr, num_tokens, m->oProjSize);
    }
  }

  assert(tokens_previous_requests == num_tokens);
}

template <typename DT>
void inference_kernel(SpecIncMultiHeadSelfAttentionMeta const *m,
                      BeamSearchBatchConfig const *bc,
                      DT const *input_ptr,
                      DT const *weight_ptr,
                      DT *output_ptr,
                      DT const *bias_ptr,
                      hipStream_t stream) {
  // here because we need postion info in infernece 1
  hipMemcpyAsync(m->token_infos,
                  &(bc->tokensInfo),
                  bc->MAX_NUM_TOKENS * sizeof(BatchConfig::PerTokenInfo),
                  hipMemcpyHostToDevice,
                  stream);
  hipMemcpyAsync(m->request_infos,
                  &(bc->requestsInfo),
                  bc->MAX_NUM_REQUESTS * sizeof(BatchConfig::PerRequestInfo),
                  hipMemcpyHostToDevice,
                  stream);
  hipMemcpyAsync(m->beam_token_infos,
                  &(bc->beamTokenInfo),
                  bc->MAX_NUM_TOKENS * bc->MAX_BEAM_WIDTH *
                      sizeof(BeamSearchBatchConfig::BeamSearchPerTokenInfo),
                  hipMemcpyHostToDevice,
                  stream);
  hipMemcpyAsync(m->beam_request_infos,
                  &(bc->beamRequestsInfo),
                  bc->MAX_NUM_REQUESTS *
                      sizeof(BeamSearchBatchConfig::BeamSearchPerRequestInfo),
                  hipMemcpyHostToDevice,
                  stream);
  // phase 1: Implement kernel to compute KQV for input tokens
  compute_qkv_kernel(m,
                     bc,
                     input_ptr,
                     weight_ptr,
                     static_cast<DT *>(m->devQKVProjArray),
                     bias_ptr,
                     stream);
  // phase 2: Update key/val cache
  update_kv_cache_kernel<DT>(m, bc, stream);

  // phase 3: Compute attention score
  // 3 kernels for pahse 3: matmul1 - softmax - matmal2
  compute_attention_kernel(m, bc, output_ptr, bias_ptr, stream);
}

} // namespace SpecIncMultiHeadAttention
} // namespace Kernels

/*static*/
void SpecIncMultiHeadSelfAttention::inference_kernel_wrapper(
    SpecIncMultiHeadSelfAttentionMeta const *m,
    BeamSearchBatchConfig const *bc,
    GenericTensorAccessorR const &input,
    GenericTensorAccessorR const &weight,
    GenericTensorAccessorW const &output,
    GenericTensorAccessorR const &bias) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  bool use_bias = *m->bias;

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  assert(input.data_type == weight.data_type);
  assert(input.data_type == output.data_type);
  if (use_bias) {
    assert(input.data_type == bias.data_type);
  }

  if (input.data_type == DT_HALF) {
    half const *bias_ptr =
        use_bias ? bias.get_half_ptr() : static_cast<half const *>(nullptr);
    Kernels::SpecIncMultiHeadAttention::inference_kernel(m,
                                                         bc,
                                                         input.get_half_ptr(),
                                                         weight.get_half_ptr(),
                                                         output.get_half_ptr(),
                                                         bias_ptr,
                                                         stream);
  } else if (input.data_type == DT_FLOAT) {
    float const *bias_ptr =
        use_bias ? bias.get_float_ptr() : static_cast<float const *>(nullptr);
    Kernels::SpecIncMultiHeadAttention::inference_kernel(m,
                                                         bc,
                                                         input.get_float_ptr(),
                                                         weight.get_float_ptr(),
                                                         output.get_float_ptr(),
                                                         bias_ptr,
                                                         stream);
  } else {
    assert(false && "Unspported data type");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("SpecIncMultiHeadSelfAttention forward time = %.2fms\n", elapsed);
    // print_tensor<3, float>(acc_query.ptr, acc_query.rect,
    // "[Attention:forward:query]"); print_tensor<3, float>(acc_output.ptr,
    // acc_output.rect, "[Attention:forward:output]");
  }
}

SpecIncMultiHeadSelfAttentionMeta::SpecIncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    SpecIncMultiHeadSelfAttention const *attn,
    GenericTensorAccessorR const &weight,
    Memory gpu_mem,
    int num_samples,
    int _num_heads)
    : IncMultiHeadSelfAttentionMeta(handler,
                                    BEAM_SEARCH_MODE,
                                    attn,
                                    attn->qSize,
                                    attn->kSize,
                                    attn->vSize,
                                    attn->qProjSize,
                                    attn->kProjSize,
                                    attn->vProjSize,
                                    attn->oProjSize,
                                    attn->apply_rotary_embedding,
                                    attn->bias,
                                    attn->scaling_query,
                                    attn->qk_prod_scaling,
                                    attn->add_bias_kv,
                                    attn->scaling_factor,
                                    weight,
                                    gpu_mem,
                                    num_samples,
                                    _num_heads) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  checkCUDNN(hipdnnSetStream(handler.dnn, stream));

  // allocate memory for the seqArray and reserve space
  {
    size_t beam_tokeninfo_size = BeamSearchBatchConfig::MAX_NUM_TOKENS *
                                 BeamSearchBatchConfig::MAX_BEAM_WIDTH;
    size_t requestinfo_size = BeamSearchBatchConfig::MAX_NUM_REQUESTS;
    size_t beam_requestinfo_size = BeamSearchBatchConfig::MAX_NUM_REQUESTS;
    size_t total_size =
        requestinfo_size * sizeof(BatchConfig::PerRequestInfo) +
        beam_tokeninfo_size *
            sizeof(BeamSearchBatchConfig::BeamSearchPerTokenInfo) +
        beam_requestinfo_size *
            sizeof(BeamSearchBatchConfig::
                       BeamSearchPerRequestInfo); // more components will
                                                  // be added here later

    Realm::Rect<1, coord_t> bounds(Realm::Point<1, coord_t>(0),
                                   Realm::Point<1, coord_t>(total_size - 1));
    std::vector<size_t> field_sizes;
    field_sizes.push_back(sizeof(char));
    Realm::RegionInstance::create_instance(beam_search_reserve_inst,
                                           gpu_mem,
                                           bounds,
                                           field_sizes,
                                           0,
                                           Realm::ProfilingRequestSet())
        .wait();
    off_t offset = 0;
    beam_token_infos =
        beam_search_reserve_inst
            .pointer<BeamSearchBatchConfig::BeamSearchPerTokenInfo>(offset);
    offset += beam_tokeninfo_size *
              sizeof(BeamSearchBatchConfig::BeamSearchPerTokenInfo);
    request_infos =
        beam_search_reserve_inst.pointer<BatchConfig::PerRequestInfo>(offset);
    offset += requestinfo_size * sizeof(BatchConfig::PerRequestInfo);
    beam_request_infos =
        beam_search_reserve_inst
            .pointer<BeamSearchBatchConfig::BeamSearchPerRequestInfo>(offset);
    offset += beam_requestinfo_size *
              sizeof(BeamSearchBatchConfig::BeamSearchPerRequestInfo);
    assert(offset == total_size);
  }

  hipStreamSynchronize(stream);
}

SpecIncMultiHeadSelfAttentionMeta::~SpecIncMultiHeadSelfAttentionMeta(void) {
  beam_search_reserve_inst.destroy();
}

}; // namespace FlexFlow
