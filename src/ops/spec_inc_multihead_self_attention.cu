#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#if defined(FF_USE_CUDA) || defined(FF_USE_HIP_CUDA)
#include "hip/hip_complex.h"
#endif
#include "flashinfer/prefill_attention_decl.cuh"
#include "flexflow/ffconst_utils.h"
#include "flexflow/ops/kernels/inc_multihead_self_attention_kernels.h"
#include "flexflow/ops/kernels/inc_multihead_self_attention_utils.cuh"
#include "flexflow/ops/spec_inc_multihead_self_attention.h"
#include "flexflow/utils/cuda_helper.h"

#include <sstream>
#include <stdexcept>

#define DISPATCH_HEADDIM(head_dim, HEAD_DIM, ...)                              \
  switch (head_dim) {                                                          \
    case 64: {                                                                 \
      constexpr size_t HEAD_DIM = 64;                                          \
      __VA_ARGS__                                                              \
      break;                                                                   \
    }                                                                          \
    case 128: {                                                                \
      constexpr size_t HEAD_DIM = 128;                                         \
      __VA_ARGS__                                                              \
      break;                                                                   \
    }                                                                          \
    case 256: {                                                                \
      constexpr size_t HEAD_DIM = 256;                                         \
      __VA_ARGS__                                                              \
      break;                                                                   \
    }                                                                          \
    default: {                                                                 \
      std::ostringstream err_msg;                                              \
      err_msg << "Unsupported head_dim: " << head_dim;                         \
      throw std::invalid_argument(err_msg.str());                              \
    }                                                                          \
  }

namespace FlexFlow {

// declare Legion names
using Legion::coord_t;
using Legion::Memory;
using namespace Kernels::IncMultiHeadAttention;

namespace Kernels {
namespace SpecIncMultiHeadSelfAttention {

using flashinfer::BatchPrefillHandler;
using flashinfer::BatchPrefillWithPagedKVCacheWrapperDispatched;
using flashinfer::LogitsPostHook;
using flashinfer::MaskMode;
using flashinfer::paged_kv_t;
using flashinfer::PageStorage;
using flashinfer::PosEncodingMode;
using flashinfer::QKVLayout;

__device__ __forceinline__ size_t get_k_entry_offset(int const req_idx,
                                                     int const token_idx,
                                                     int const max_num_pages,
                                                     int const hidden_size) {
  return ((req_idx * max_num_pages + token_idx / kPagesize) * kPagesize * 2 +
          token_idx % kPagesize) *
         hidden_size;
}

__device__ __forceinline__ size_t get_v_entry_offset(int const req_idx,
                                                     int const token_idx,
                                                     int const max_num_pages,
                                                     int const hidden_size) {
  return ((req_idx * max_num_pages + token_idx / kPagesize) * kPagesize * 2 +
          kPagesize + token_idx % kPagesize) *
         hidden_size;
}

template <typename DT>
__global__ void
    update_qkv_cache_kernel(DT *devQKVProjArray,
                            half *qTmp_ptr,
                            half *kCache_ptr,
                            BatchConfig::PerTokenInfo const *tokenInfos,
                            BatchConfig::PerRequestInfo *request_infos,
                            int const max_num_pages,
                            int hidden_size,
                            int num_new_tokens) {
  int const thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int const token_idx = thread_idx / hidden_size;
  int const offset = thread_idx % hidden_size;
  if (token_idx >= num_new_tokens) {
    return;
  }

  int const req_idx = tokenInfos[token_idx].request_index;
  int const token_abs_idx = tokenInfos[token_idx].abs_index_in_request;

  size_t from_idx = token_idx * QKV_WEIGHT_NUM * hidden_size;
  size_t to_k_idx = get_k_entry_offset(
             req_idx, token_abs_idx, max_num_pages, hidden_size),
         to_v_idx = get_v_entry_offset(
             req_idx, token_abs_idx, max_num_pages, hidden_size);

  // key and value cache should be stored interleaved
  kCache_ptr[to_k_idx + offset] =
      static_cast<half>(devQKVProjArray[from_idx + hidden_size + offset]);
  kCache_ptr[to_v_idx + offset] =
      static_cast<half>(devQKVProjArray[from_idx + hidden_size * 2 + offset]);
  qTmp_ptr[token_idx * hidden_size + offset] =
      static_cast<half>(devQKVProjArray[from_idx + offset]);
}

template <typename DT>
void update_qkv_cache(SpecIncMultiHeadSelfAttentionMeta const *m,
                      BatchConfig const *bc,
                      hipStream_t stream) {
  // update the kv cache, compact the q array
  int num_new_tokens = bc->num_active_tokens();
  int parallelism = m->hidden_size * num_new_tokens;
  int const max_num_pages =
      (BatchConfig::max_sequence_length() +
       BatchConfig::max_spec_tree_token_num() + kPagesize - 1) /
      kPagesize;
  update_qkv_cache_kernel<<<GET_BLOCKS(parallelism),
                            min(CUDA_NUM_THREADS, parallelism),
                            0,
                            stream>>>(static_cast<DT *>(m->devQKVProjArray),
                                      static_cast<half *>(m->queryTmp),
                                      static_cast<half *>(m->keyCache),
                                      m->token_infos,
                                      m->request_infos,
                                      max_num_pages,
                                      m->hidden_size,
                                      num_new_tokens);
}

template <typename DT>
__global__ void produce_output_kernel(half const *input_ptr,
                                      DT *output_ptr,
                                      int parallelism) {
  CUDA_KERNEL_LOOP(idx, parallelism) {
    output_ptr[idx] = static_cast<DT>(input_ptr[idx]);
  }
}

template <typename DT>
void tree_search_attention(SpecIncMultiHeadSelfAttentionMeta *m,
                           BatchConfig const *bc,
                           DT *output_ptr,
                           hipStream_t stream) {
  //   int device;
  //   checkCUDA(hipGetDevice(&device));
  //   hipEvent_t t_start, t_end;
  //   hipEventCreate(&t_start);
  //   hipEventCreate(&t_end);
  //   hipEventRecord(t_start, stream);

  // global constant parameters
  uint32_t const num_q_heads = m->num_q_heads;
  uint32_t const num_kv_heads = m->num_kv_heads;
  uint32_t const head_dim = m->qProjSize;
  uint32_t const batch_size = bc->num_active_requests();
  float const sm_scale =
      (*m->qk_prod_scaling) ? 1.0f / sqrt(m->kProjSize) : 1.0f;
  int32_t q_indptr_h[bc->max_requests_per_batch() + 1], kv_indptr_h[bc->max_requests_per_batch() + 1];
  q_indptr_h[0] = 0;
  kv_indptr_h[0] = 0;
  for (int req_idx = 0, indptr_idx = 0; req_idx < bc->max_requests_per_batch(); req_idx++) {
    if (bc->request_available[req_idx]) {
      int q_len = bc->requestsInfo[req_idx].num_tokens_in_batch;
      int kv_len = bc->requestsInfo[req_idx].num_tokens_in_batch +
                  bc->requestsInfo[req_idx].first_token_index_in_request;
      q_indptr_h[indptr_idx + 1] = q_indptr_h[indptr_idx] + q_len;
      kv_indptr_h[indptr_idx + 1] = kv_indptr_h[indptr_idx] + (kv_len + kPagesize - 1) / kPagesize;
      indptr_idx++;
    }
  }

  //   hipEventCreate(&t_start);
  //   hipEventCreate(&t_end);
  //   hipEventRecord(t_start, stream);

  //   hipEventRecord(t_end, stream);
  //   checkCUDA(hipEventSynchronize(t_end));
  //   elapsed = 0;
  //   checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
  //   hipEventDestroy(t_start);
  //   hipEventDestroy(t_end);
  //   if (device == 0) {
  //     std::cout << "Update custom mask time: " << elapsed << " ms\n";
  //   }

  half *q = static_cast<half *>(m->queryTmp),
       *kv = static_cast<half *>(m->keyCache),
       *o = static_cast<half *>(m->outputTmp);
  paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv(
      num_kv_heads,
      kPagesize,
      head_dim,
      batch_size,
      kv,
      m->handle.attention_metadata.kv_indices,
      m->handle.attention_metadata.kv_indptr,
      m->handle.attention_metadata.kv_last_page_len);

  //   hipEventRecord(t_end, stream);
  //   checkCUDA(hipEventSynchronize(t_end));
  //   float elapsed = 0;
  //   checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
  //   if (device == 0) {
  //     printf("    attn prep time: %.4f ms\n", elapsed);
  //   }
  //   hipEventDestroy(t_start);
  //   hipEventDestroy(t_end);

  //   hipEventCreate(&t_start);
  //   hipEventCreate(&t_end);
  //   hipEventRecord(t_start, stream);

  BatchPrefillHandler *handler =
      static_cast<BatchPrefillHandler *>(m->batch_prefill_handler);
  handler->SetCUDAStream(stream);
  handler->BeginForward<half, int32_t>(m->workspace,
                                       m->workspace_size,
                                       static_cast<int32_t *>(q_indptr_h),
                                       static_cast<int32_t *>(kv_indptr_h),
                                       batch_size,
                                       num_q_heads,
                                       num_kv_heads,
                                       head_dim,
                                       kPagesize);

  //   hipEventRecord(t_end, stream);
  //   checkCUDA(hipEventSynchronize(t_end));
  //   elapsed = 0;
  //   checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
  //   if (device == 0) {
  //     printf("    BeginForward time: %.4f ms\n", elapsed);
  //   }
  //   hipEventDestroy(t_start);
  //   hipEventDestroy(t_end);

  //   hipEventCreate(&t_start);
  //   hipEventCreate(&t_end);
  //   hipEventRecord(t_start, stream);

  DISPATCH_HEADDIM(
    head_dim, HEAD_DIM, {
      hipError_t result;
      if (bc->prompt_phase) {
        result = BatchPrefillWithPagedKVCacheWrapperDispatched<
            PageStorage::kIndices,
            HEAD_DIM,
            LogitsPostHook::kNone,
            QKVLayout::kNHD,
            PosEncodingMode::kNone,
            false,
            MaskMode::kCausal,
            half,
            half,
            int32_t>(handler,
                      q,
                      m->handle.attention_metadata.q_indptr,
                      /*q_offset=*/nullptr,
                      paged_kv,
                      /*custom_mask=*/nullptr,
                      /*qk_indptr=*/nullptr,
                      o,
                      /*lse=*/nullptr,
                      num_q_heads,
                      /*logits_soft_cap=*/0.f,
                      sm_scale,
                      /*rope_scale=*/1.f,
                      /*rope_theta=*/static_cast<float>(1e4),
                      stream);
      } else {
        result = BatchPrefillWithPagedKVCacheWrapperDispatched<
            PageStorage::kIndices,
            HEAD_DIM,
            LogitsPostHook::kNone,
            QKVLayout::kNHD,
            PosEncodingMode::kNone,
            false,
            MaskMode::kCustom,
            half,
            half,
            int32_t>(handler,
                      q,
                      m->handle.attention_metadata.q_indptr,
                      /*q_offset=*/nullptr,
                      paged_kv,
                      m->handle.attention_metadata.custom_mask,
                      m->handle.attention_metadata.qk_indptr,
                      o,
                      /*lse=*/nullptr,
                      num_q_heads,
                      /*logits_soft_cap=*/0.f,
                      sm_scale,
                      /*rope_scale=*/1.f,
                      /*rope_theta=*/static_cast<float>(1e4),
                      stream);
      }
    if (result != hipSuccess) {
      throw std::runtime_error(
          "Failed to run "
          "BatchPrefillWithPagedKVCacheWrapperDispatched" +
          std::string(hipGetErrorString(result)));
    }
  });

  //   hipEventRecord(t_end, stream);
  //   checkCUDA(hipEventSynchronize(t_end));
  //   elapsed = 0;
  //   checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
  //   if (device == 0) {
  //     printf("    actual attn time: %.4f ms\n", elapsed);
  //   }
  //   hipEventDestroy(t_start);
  //   hipEventDestroy(t_end);

  //   hipEventCreate(&t_start);
  //   hipEventCreate(&t_end);
  //   hipEventRecord(t_start, stream);

  {
    int parallelism = m->vProjSize * m->num_q_heads * bc->num_active_tokens();
    produce_output_kernel<<<GET_BLOCKS(parallelism),
                            min(CUDA_NUM_THREADS, parallelism),
                            0,
                            stream>>>(m->outputTmp, output_ptr, parallelism);
  }

  //   hipEventRecord(t_end, stream);
  //   checkCUDA(hipEventSynchronize(t_end));
  //   elapsed = 0;
  //   checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
  //   if (device == 0) {
  //     printf("    produce_output_kernel time: %.4f ms\n", elapsed);
  //   }
  //   hipEventDestroy(t_start);
  //   hipEventDestroy(t_end);
}

template <typename DT>
void inference_kernel(SpecIncMultiHeadSelfAttentionMeta *m,
                      BatchConfig const *bc,
                      int shard_id,
                      DT const *input_ptr,
                      DT const *weight_ptr,
                      DT *output_ptr,
                      DT const *bias_ptr,
                      hipStream_t stream) {
  // phase 1: Implement kernel to compute KQV for input tokens

  // long long time_1 = Realm::Clock::current_time_in_microseconds(), time_2;
  compute_qkv_kernel(m,
                     bc,
                     shard_id,
                     input_ptr,
                     weight_ptr,
                     static_cast<DT *>(m->devQKVProjArray),
                     bias_ptr,
                     stream);
  // phase 2: Update key/val cache
  update_qkv_cache<DT>(m, bc, stream);

  // phase 3: Compute attention score
  // 3 kernels for pahse 3: matmul1 - softmax - matmal2
  tree_search_attention<DT>(m, bc, static_cast<DT *>(m->attn_heads), stream);

  // Debug output:
  //   int size = m->hidden_size * BatchConfig::max_tokens_per_batch();
  //   float *temp_output = new float[size];
  //   hipDeviceSynchronize();
  //   hipMemcpy(
  //       temp_output, m->attn_heads, size * sizeof(float),
  //       hipMemcpyDeviceToHost);

  //   printf("Output: ");
  //   for (int i = 0; i < bc->num_tokens; ++i) {
  //     float temp = 0;
  //     for (int j = 0; j < m->hidden_size; ++j) {
  //       temp += temp_output[i * m->hidden_size + j];
  //     }
  //     printf("%.6f ", temp);
  //   }
  //   printf("\n");

  //   delete[] temp_output;

  // compute output production and bias together for all tokens
  int num_tokens = bc->num_active_tokens();

  compute_o_prod_bias(
      m, bc, shard_id, output_ptr, weight_ptr, bias_ptr, num_tokens, stream);
  // time_2 = Realm::Clock::current_time_in_microseconds();
  // std::cout << "SpecIncMultiHeadSelfAttention kernel time: "
  //           << (time_2 - time_1) << "us" << std::endl;
}

} // namespace SpecIncMultiHeadSelfAttention
} // namespace Kernels

/*static*/
void SpecIncMultiHeadSelfAttention::inference_kernel_wrapper(
    SpecIncMultiHeadSelfAttentionMeta *m,
    BatchConfig const *bc,
    int shard_id,
    GenericTensorAccessorR const &input,
    GenericTensorAccessorR const &weight,
    GenericTensorAccessorW const &output,
    GenericTensorAccessorR const &bias) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  bool use_bias = *m->qkv_bias || *m->final_bias;

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  assert(input.data_type == weight.data_type);
  assert(input.data_type == output.data_type);
  if (use_bias) {
    assert(input.data_type == bias.data_type);
  }

  if (input.data_type == DT_HALF) {
    half const *bias_ptr =
        use_bias ? bias.get_half_ptr() : static_cast<half const *>(nullptr);
    Kernels::SpecIncMultiHeadSelfAttention::inference_kernel<half>(
        m,
        bc,
        shard_id,
        input.get_half_ptr(),
        weight.get_half_ptr(),
        output.get_half_ptr(),
        bias_ptr,
        stream);
  } else if (input.data_type == DT_FLOAT) {
    float const *bias_ptr =
        use_bias ? bias.get_float_ptr() : static_cast<float const *>(nullptr);
    Kernels::SpecIncMultiHeadSelfAttention::inference_kernel<float>(
        m,
        bc,
        shard_id,
        input.get_float_ptr(),
        weight.get_float_ptr(),
        output.get_float_ptr(),
        bias_ptr,
        stream);
  } else {
    assert(false && "Unspported data type");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("SpecIncMultiHeadSelfAttention forward time = %.2fms\n", elapsed);
    // print_tensor<3, float>(acc_query.ptr, acc_query.rect,
    // "[Attention:forward:query]"); print_tensor<3, float>(acc_output.ptr,
    // acc_output.rect, "[Attention:forward:output]");
  }
}

SpecIncMultiHeadSelfAttentionMeta::SpecIncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    SpecIncMultiHeadSelfAttention const *attn,
    GenericTensorAccessorR const &weight,
    MemoryAllocator &gpu_mem_allocator,
    int num_samples,
    int _num_q_heads,
    int _num_kv_heads)
    : IncMultiHeadSelfAttentionMeta(handler,
                                    TREE_SEARCH_MODE,
                                    attn,
                                    attn->qSize,
                                    attn->kSize,
                                    attn->vSize,
                                    attn->qProjSize,
                                    attn->kProjSize,
                                    attn->vProjSize,
                                    attn->oProjSize,
                                    attn->apply_rotary_embedding,
                                    attn->qkv_bias,
                                    attn->scaling_query,
                                    attn->qk_prod_scaling,
                                    attn->position_bias,
                                    attn->final_bias,
                                    attn->scaling_factor,
                                    weight,
                                    gpu_mem_allocator,
                                    num_samples,
                                    attn->num_q_heads,
                                    attn->num_kv_heads,
                                    _num_q_heads,
                                    _num_kv_heads,
                                    DT_NONE,
                                    false) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  checkCUDNN(hipdnnSetStream(handler.dnn, stream));

  {
    workspace_size = 32 * 1024 * 1024; // 32MB
    gpu_mem_allocator.create_legion_instance(
        flashinfer_reserve_inst, workspace_size);
    workspace = static_cast<void *>(
        gpu_mem_allocator.allocate_instance<char>(workspace_size));
    batch_prefill_handler =
        static_cast<void *>(new flashinfer::BatchPrefillHandler);
  }

  hipStreamSynchronize(stream);
}

SpecIncMultiHeadSelfAttentionMeta::~SpecIncMultiHeadSelfAttentionMeta(void) {
  if (flashinfer_reserve_inst != Realm::RegionInstance::NO_INST) {
    flashinfer_reserve_inst.destroy();
  }
  delete static_cast<flashinfer::BatchPrefillHandler *>(batch_prefill_handler);
}

}; // namespace FlexFlow
