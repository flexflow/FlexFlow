#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#if defined(FF_USE_CUDA) || defined(FF_USE_HIP_CUDA)
#include "hip/hip_complex.h"
#endif
#include "flexflow/ffconst_utils.h"
#include "flexflow/ops/kernels/inc_multihead_self_attention_kernels.h"
#include "flexflow/ops/kernels/inc_multihead_self_attention_utils.cuh"
#include "flexflow/ops/spec_inc_multihead_self_attention.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

#define WARP_SIZE 32

// declare Legion names
using Legion::coord_t;
using Legion::Memory;
using namespace Kernels::IncMultiHeadAttention;

namespace Kernels {
namespace SpecIncMultiHeadSelfAttention {

template <typename DT,
          int THREADS_PER_BLOCK,
          int Dh,
          int Dh_MAX,
          int THREADS_PER_KEY,
          int THREADS_PER_VALUE>
__global__ void compute_spec_inc_attention_kernel_generation_kernel(
    DT const *query,
    DT const *key_cache,
    DT const *value_cache,
    DT *output_ptr,
    float const scale,
    int const max_seq_length,
    int per_head_size,
    int hidden_size,
    BatchConfig::PerRequestInfo *request_infos,
    BeamSearchBatchConfig::BeamSearchPerRequestInfo *beam_request_infos,
    BatchConfig::BitMask *causalMask,
    bool *request_completed) {

  // q, k
  using Q_vec = typename VEC_K<DT, THREADS_PER_KEY>::Type;
  using K_vec = typename VEC_K<DT, THREADS_PER_KEY>::Type;
  using V_vec = typename VEC_V<DT>::Type;
  using Out_sum = typename Vec_fp32_<V_vec>::Type;

  constexpr int WARPS_PER_BLOCK = THREADS_PER_BLOCK / WARP_SIZE;

  constexpr int K_VEC_SIZE = sizeof(K_vec) / sizeof(DT);
  constexpr int K_ELTS_PER_THREAD = Dh / THREADS_PER_KEY;
  constexpr int K_VECS_PER_THREAD = K_ELTS_PER_THREAD / K_VEC_SIZE;
  // constexpr int QK_ELTS_IN_16B = 16 / sizeof(DT);

  // thread id
  int const tidx = threadIdx.x;
  // head id
  int const head_idx = blockIdx.x;
  // nth request idx
  int const request_idx = blockIdx.y;

  // request id in batch config
  int const batch_config_request_id =
      request_infos[request_idx].batch_config_request_id;

  // request_idx = re

  BatchConfig::BitMask bitmask = causalMask[batch_config_request_id];

  int const first_step = 0;

  // int const tlength =
  //     request_infos[batch_config_request_id].first_token_depth_in_request +
  //     request_infos[batch_config_request_id].num_tokens_in_batch;

  int const totalCacheSize =
      bitmask.non_tree_cache_size + bitmask.tree_size + bitmask.prompt_size - 1;

  int first_token_idx = 0;
  for (int r = 0; r < batch_config_request_id; r++) {
    first_token_idx += request_completed[r] ? 0 : causalMask[r].this_layer_size;
  }

  int const tree_branch_num =
      beam_request_infos[batch_config_request_id].sub_request_num;

  // shared memory objects
  extern __shared__ char smem_[];

  float *qk_smem = reinterpret_cast<float *>(smem_);
  float *out_smem = reinterpret_cast<float *>(smem_);

  float qk_max = -FLT_MAX;

  // first WARPS_PER_BLOCK for store qk_max, second WARPS_PER_BLOCK for sum
  __shared__ float red_smem[WARPS_PER_BLOCK * 2];

  const DT *q_ptr = query + first_token_idx * hidden_size * QKV_WEIGHT_NUM +
                    head_idx * per_head_size;
  __shared__ Q_vec q_vecs[THREADS_PER_KEY][K_VECS_PER_THREAD];

  // the start offset of the element eg. (0, 1, 2, 3) * K_VEC_SIZE
  int ki = tidx % THREADS_PER_KEY * K_VEC_SIZE;
  int ki_o = tidx % THREADS_PER_KEY;
  // the first key's offset for this thread
  // ko = 0, 0, 0, 0, 1, 1, 1, 1, ....
  int ko = tidx / THREADS_PER_KEY;
  // load q tensor
  Q_vec q_vec[K_VECS_PER_THREAD];

  constexpr int K_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_KEY;
  // The number of keys per warp.
  constexpr int K_PER_WARP = WARP_SIZE / THREADS_PER_KEY;

  DT const *k_cache_batch =
      key_cache + batch_config_request_id * max_seq_length * hidden_size + ki;

  int ti_end =
      div_up(totalCacheSize - first_step, K_PER_WARP) * K_PER_WARP + first_step;

  for (int qi = 0; qi < tree_branch_num; qi += 1) {
#pragma unroll
    for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
      q_vecs[ki_o][ii] = *reinterpret_cast<Q_vec const *>(
          q_ptr + (hidden_size * QKV_WEIGHT_NUM * qi) + ki +
          ii * THREADS_PER_KEY * K_VEC_SIZE);
    }

    int const query_token =
        bitmask.prompt_size + bitmask.tree_size - 1 - tree_branch_num + qi;

    __syncthreads();
    for (int ti = ko; ti < ti_end; ti += K_PER_ITER) {
      K_vec k[K_VECS_PER_THREAD];
      int const ti_circ = ti % max_seq_length;

      for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
        int jj = ii * THREADS_PER_KEY * K_VEC_SIZE;
        if (ti < totalCacheSize) {

          k[ii] = *reinterpret_cast<K_vec const *>(
              k_cache_batch + ti_circ * hidden_size + head_idx * per_head_size +
              jj);
        }
      }
      float qk = scale * Qk_dot<DT, THREADS_PER_KEY>::dot(q_vecs[ki_o], k);

      if (ti < totalCacheSize && tidx % THREADS_PER_KEY == 0) {
        // todo add alobi here
        // bool const mask = ti_circ >= totalCacheSize;
        bool const mask = (ti >= bitmask.non_tree_cache_size &&
                           (!(bitmask.mask[ti - bitmask.non_tree_cache_size] &
                              (1 << query_token))));

        // if (head_idx == 0 && ti == 0 && request_idx == 15 && !mask) {
        //   printf("spec inc attn qkqkqk  request id %d,  %.10f, %d\n",
        //          batch_config_request_id,
        //          ti,
        //          qk,
        //          qi);
        // }
        qk_max = mask ? qk_max : fmaxf(qk_max, qk);
        qk_smem[ti - first_step] = mask ? 0.f : qk;
      }
    }

    __syncthreads();

#pragma unroll
    for (int mask = WARP_SIZE / 2; mask >= THREADS_PER_KEY; mask /= 2) {
      qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
    }

    // Decompose the thread index into warp and lane.
    int const warp = tidx / WARP_SIZE;
    int const lane = tidx % WARP_SIZE;

    // The warp leader writes the max to shared memory.
    if (lane == 0) {
      red_smem[warp] = qk_max;
    }

    // Make sure the products are in shared memory.
    __syncthreads();

    // The warps finalize the reduction.
    qk_max = lane < WARPS_PER_BLOCK ? red_smem[lane] : -FLT_MAX;
#pragma unroll
    for (int mask = WARPS_PER_BLOCK / 2; mask >= 1; mask /= 2) {
      qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
    }

    // Broadcast to all the threads in the warp.
    qk_max = __shfl_sync(uint32_t(-1), qk_max, 0);

    // if (blockIdx.y == 0 && blockIdx.x == 0 && tidx == 0) {
    //   printf("spec inc attn first token qk_max %.10f\n", qk_max);
    // }

    float exp_sum = 0.f;
    for (int ti = first_step + tidx; ti < totalCacheSize;
         ti += THREADS_PER_BLOCK) {
      bool const mask = (ti >= bitmask.non_tree_cache_size &&
                         (!(bitmask.mask[ti - bitmask.non_tree_cache_size] &
                            (1 << query_token))));
      float logit = mask ? 0.0f : __expf(qk_smem[ti - first_step] - qk_max);
      exp_sum += logit;
      qk_smem[ti - first_step] = mask ? 0.0f : logit;
    }

    // Compute the sum.
    exp_sum = block_sum<WARPS_PER_BLOCK>(&red_smem[WARPS_PER_BLOCK], exp_sum);

    // softmax
    float inv_sum = __fdividef(1.f, exp_sum + 1.e-6);
    for (int ti = first_step + tidx; ti < totalCacheSize;
         ti += THREADS_PER_BLOCK) {
      qk_smem[ti - first_step] *= inv_sum;
    }

    __syncthreads();

    // value projection
    constexpr int V_VEC_SIZE = 16 / sizeof(DT);
    // A vector of V elements for the current timestep.
    // using V_vec_k = typename V_vec_k_<DT, V_VEC_SIZE>::Type;
    // using V_vec_acum = typename V_vec_acum_fp32_<V_vec_k>::Type;

    // The value computed by this thread.
    int vo = tidx / THREADS_PER_VALUE;
    // The hidden dimensions computed by this particular thread.
    int vi = tidx % THREADS_PER_VALUE * V_VEC_SIZE;
    constexpr int V_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_VALUE;

    Out_sum out;
    zero(out);

    // The base pointer for the value in the cache buffer.
    DT const *v_cache_batch =
        value_cache + batch_config_request_id * max_seq_length * hidden_size +
        vi;

    if (Dh == Dh_MAX || vi < Dh) {
      for (int ti = first_step + vo; ti < totalCacheSize; ti += V_PER_ITER) {
        // Load the values from the cache.
        int const ti_circ = ti % max_seq_length;
        V_vec v = *reinterpret_cast<V_vec const *>(
            v_cache_batch + ti_circ * hidden_size + head_idx * per_head_size);

        bool const mask = (ti >= bitmask.non_tree_cache_size &&
                           (!(bitmask.mask[ti - bitmask.non_tree_cache_size] &
                              (1 << query_token))));
        float logit = mask ? 0.0f : qk_smem[ti - first_step];
        out = FlexFlow::fma(logit, cast_to_float(v), out);
      }
    }

    //   // Make sure we can start writing to shared memory.
    __syncthreads();

    // Run the final reduction amongst the different groups computing different
    // partial outputs.
    if (Dh == Dh_MAX || vi < Dh) {
#pragma unroll
      for (int active_groups = V_PER_ITER; active_groups >= 2;
           active_groups /= 2) {

        // The midpoint in the number of active groups.
        int midpoint = active_groups / 2;

        // The upper part of active threads store to shared memory.
        if (vo >= midpoint && vo < active_groups && (Dh == Dh_MAX || vi < Dh)) {
          *reinterpret_cast<Out_sum *>(out_smem + (vo - midpoint) * Dh + vi) =
              out;
        }
        __syncthreads();

        // The bottom warps update their values.
        if (vo < midpoint && (Dh == Dh_MAX || vi < Dh)) {
          out = add(*reinterpret_cast<Out_sum const *>(out_smem + vo * Dh + vi),
                    out);
        }
        __syncthreads();
      }
    }

    // Output the final values.
    if (vo == 0 && (Dh == Dh_MAX || vi < Dh)) {
      convert_from_float(*reinterpret_cast<V_vec *>(
                             output_ptr + (first_token_idx + qi) * hidden_size +
                             head_idx * per_head_size + vi),
                         out);
    }
  }
}

template <typename DT>
__global__ void spec_inc_store_kv_cache(
    DT const *devQKVProjArray,
    DT *kCache_ptr,
    DT *vCache_ptr,
    BatchConfig::PerTokenInfo *tokenInfos,
    BatchConfig::PerRequestInfo *requestInfo,
    BeamSearchBatchConfig::BeamSearchPerTokenInfo *beamTokenInfos,
    BeamSearchBatchConfig::BeamSearchPerRequestInfo *beamRequestInfos,
    BatchConfig::BitMask *causalMask,
    int qProjSize,
    int kProjSize,
    int vProjSize,
    int num_tokens,
    int max_seq_len,
    bool is_root,
    int hidden_size) {
  CUDA_KERNEL_LOOP(i, num_tokens * hidden_size) {
    int token_idx = i / (hidden_size);
    int offset = i % hidden_size;

    size_t val_idx =
        token_idx * QKV_WEIGHT_NUM * hidden_size + hidden_size + offset;

    DT kVal = devQKVProjArray[val_idx];
    DT vVal = devQKVProjArray[val_idx + hidden_size];

    int const req_id = tokenInfos[token_idx].request_index;
    // int const tok_id = tokenInfos[token_idx].abs_depth_in_request;

    int const request_token_offset =
        requestInfo[req_id].first_token_offset_in_batch;

    BatchConfig::BitMask bitmask = causalMask[req_id];

    // if prompt token -> token id
    // if tree token:

    int const cache_idx = bitmask.prompt_size + bitmask.non_tree_cache_size +
                          bitmask.tree_size - 1 - bitmask.this_layer_size +
                          token_idx - request_token_offset;

    kCache_ptr[req_id * (hidden_size * max_seq_len) + (cache_idx)*hidden_size +
               offset] = kVal;
    vCache_ptr[req_id * (hidden_size * max_seq_len) + (cache_idx)*hidden_size +
               offset] = vVal;
  }
}

template <typename DT>
void update_kv_cache_kernel(SpecIncMultiHeadSelfAttentionMeta const *m,
                            BeamSearchBatchConfig const *bc,
                            hipStream_t stream) {
  int num_tokens = bc->num_active_tokens();
  int curr_depth = bc->beamRequestsInfo[0].current_depth;
  if (num_tokens > 0) {
    int parallelism = m->hidden_size * KV_WEIGHT_NUM * num_tokens;
    spec_inc_store_kv_cache<<<GET_BLOCKS(parallelism),
                              min(CUDA_NUM_THREADS, parallelism),
                              0,
                              stream>>>(
        static_cast<DT *>(m->devQKVProjArray),
        static_cast<DT *>(m->keyCache),
        static_cast<DT *>(m->valueCache),
        m->token_infos,
        m->request_infos,
        m->beam_token_infos,
        m->beam_request_infos,
        m->causalMask,
        m->qProjSize,
        m->kProjSize,
        m->vProjSize,
        num_tokens,
        BatchConfig::max_sequence_length() +
            BatchConfig::MAX_SPEC_TREE_TOKEN_NUM,
        /*root*/ curr_depth == 0,
        m->hidden_size);
  }
}

#define LAUNCH_SPEC_INC_ATTENTION_SCORE_KERNEL(                                \
    DT, Dh, Dh_MAX, THDS_PER_KEY, THREADS_PER_VALUE, THDS_PER_BLOCK, stream)   \
  smem_sz = smem_size_in_bytes<DT>(m->qProjSize,                               \
                                   BatchConfig::max_sequence_length() +        \
                                       BatchConfig::MAX_SPEC_TREE_TOKEN_NUM,   \
                                   THREADS_PER_VALUE,                          \
                                   THDS_PER_BLOCK);                            \
  compute_spec_inc_attention_kernel_generation_kernel<DT,                      \
                                                      THDS_PER_BLOCK,          \
                                                      Dh,                      \
                                                      Dh_MAX,                  \
                                                      THDS_PER_KEY,            \
                                                      THREADS_PER_VALUE>       \
      <<<grid, THDS_PER_BLOCK, smem_sz, stream>>>(                             \
          static_cast<DT *>(m->devQKVProjArray),                               \
          static_cast<DT *>(m->keyCache),                                      \
          static_cast<DT *>(m->valueCache),                                    \
          output_ptr,                                                          \
          scale,                                                               \
          BatchConfig::max_sequence_length() +                                 \
              BatchConfig::MAX_SPEC_TREE_TOKEN_NUM,                            \
          m->qProjSize,                                                        \
          m->hidden_size,                                                      \
          m->request_infos,                                                    \
          m->beam_request_infos,                                               \
          m->causalMask,                                                       \
          m->request_completed)

template <typename DT>
void compute_spec_inc_attention_kernel_generation(
    SpecIncMultiHeadSelfAttentionMeta const *m,
    BeamSearchBatchConfig const *bc,
    DT *output_ptr,
    hipStream_t stream) {
  // one block == one head per request
  // how many generation requests
  dim3 grid(m->num_q_heads, bc->get_speculative_request_num());
  int const per_head_size = m->qProjSize;
  float scale = (*m->qk_prod_scaling) ? 1.0f / sqrt(m->kProjSize) : 1.0f;
  size_t smem_sz;
  if (per_head_size == 64) {
    constexpr int THREADS_PER_VALUE_64 = threads_per_value_t<DT, 64>::value;
    LAUNCH_SPEC_INC_ATTENTION_SCORE_KERNEL(
        DT, 64, 64, 4, THREADS_PER_VALUE_64, 128, stream);
  } else if (per_head_size == 128) {
    constexpr int THREADS_PER_VALUE_128 = threads_per_value_t<DT, 128>::value;
    LAUNCH_SPEC_INC_ATTENTION_SCORE_KERNEL(
        DT, 128, 128, 4, THREADS_PER_VALUE_128, 128, stream);
  } else {
    assert(false && "a unsupported head size");
  }
}

template <typename DT>
__global__ void spec_fill_entries_above_diagonal(DT *matrix,
                                                 size_t new_tokens,
                                                 size_t total_tokens_in_request,
                                                 size_t num_q_heads,
                                                 DT value) {
  CUDA_KERNEL_LOOP(i, new_tokens * total_tokens_in_request * num_q_heads) {
    // size_t head_idx = i / (new_tokens * total_tokens_in_request);
    size_t src_idx = (i / new_tokens) % total_tokens_in_request;
    size_t dst_idx = i % new_tokens + total_tokens_in_request - new_tokens;
    // Casual Mask
    if (src_idx > dst_idx) {
      matrix[i] = value;
    }
  }
}

template <typename DT>
void compute_attention_kernel_prompt(SpecIncMultiHeadSelfAttentionMeta const *m,
                                     BeamSearchBatchConfig const *bc,
                                     int shard_id,
                                     DT *output_ptr,
                                     DT const *bias_ptr,
                                     DT const *weight_ptr,
                                     hipStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  hipDataType cublas_data_type = ff_to_cuda_datatype(m->output_type[0]);
  hipdnnDataType_t cudnn_data_type = ff_to_cudnn_datatype(m->output_type[0]);
  assert(data_type_size(m->output_type[0]) == sizeof(DT));
#if defined(CUDA_VERSION) && (CUDA_VERSION < 11000)
  hipDataType compute_type = cublas_data_type;
#else
  // For best performance, set the default cublas compute type to
  // HIPBLAS_COMPUTE_16F for half precision and to
  // HIPBLAS_COMPUTE_32F_FAST_16F for full precision
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
  if (m->output_type[0] == DT_FLOAT) {
    compute_type = HIPBLAS_COMPUTE_32F_FAST_16F;
  }
#endif
  // int num_requests = bc->num_active_requests();
  int num_tokens = bc->num_active_tokens();
  int tokens_previous_requests = 0;
  int tokens_prev_requests_squares = 0;
  // int qkv_block_size =
  //     (m->qProjSize + m->kProjSize + m->vProjSize) * num_tokens;
  int q_block_size = m->qProjSize;

  int kt_block_size = m->kProjSize;
  int kt_req_block_size = kt_block_size * m->num_q_heads *
                          (BatchConfig::max_sequence_length() +
                           BatchConfig::MAX_SPEC_TREE_TOKEN_NUM);
  int vt_block_size = m->vProjSize;
  int vt_req_block_size = vt_block_size * m->num_q_heads *
                          (BatchConfig::max_sequence_length() +
                           BatchConfig::MAX_SPEC_TREE_TOKEN_NUM);
  assert(m->qProjSize == m->kProjSize);

  for (int i = 0; i < bc->max_requests_per_batch(); i++) {
    if (bc->request_completed[i] || (!bc->requestsInfo[i].prompt_phase) ||
        (bc->requestsInfo[i].num_tokens_in_batch == 0)) {
      continue;
    } else if (tokens_previous_requests < bc->num_generation_tokens) {
      tokens_previous_requests += bc->requestsInfo[i].num_tokens_in_batch;
      continue;
    }

    // all requests in prompt phase should only have one sub requests;
    assert(bc->sub_requests[i] == 1);
    // int num_new_tokens = bc->num_processing_tokens[i];
    // int total_tokens = bc->token_last_available_idx[i] + 1;

    int num_new_tokens = bc->requestsInfo[i].num_tokens_in_batch;
    int total_tokens = bc->requestsInfo[i].first_token_depth_in_request +
                       bc->requestsInfo[i].num_tokens_in_batch;

    if (num_new_tokens <= 0) {
      continue;
    }

    // Compute (QK^T/sqrt(d_k))
    int m_ = num_new_tokens;
    int n = total_tokens;
    int k = m->qProjSize;
    int lda = k * m->num_q_heads * QKV_WEIGHT_NUM, ldb = k * m->num_q_heads,
        ldc = m_;
    int strideA = q_block_size;
    int strideB = kt_block_size;
    int strideC = num_new_tokens * total_tokens;

    // a flag of using this scaling alpha
    DT alpha = 1.0f, beta = 0.0f;
    if (*m->qk_prod_scaling) {
      alpha = static_cast<DT>(1.0f / sqrt(m->kProjSize));
    }
    // To get A, skip over Q entries from previous requests (same head)
    DT const *A = static_cast<DT *>(m->devQKVProjArray) +
                  bc->requestsInfo[i].first_token_offset_in_batch *
                      m->qProjSize * m->num_q_heads * QKV_WEIGHT_NUM;
    // To get B, skip over K entries from previous requests (all heads +
    // padding)

    // print_tensor<float>((float*)A, 32, "A");
    DT const *B = static_cast<DT *>(m->keyCache) + i * kt_req_block_size;

    // if (i == 0 && sub_req_id == 0 &&
    //     bc->beam_slots.at(0).current_depth == 1) {
    //   int offset = (float *)B - m->keyCache;
    //   printf("key cache offset %d\n", kt_req_block_size);
    // }
    // To get C, skip over QK^T products from previous requests
    DT *C = static_cast<DT *>(m->qk_prods) +
            m->num_q_heads * tokens_prev_requests_squares;
    checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                         HIPBLAS_OP_T,
                                         HIPBLAS_OP_N,
                                         m_,
                                         n,
                                         k,
                                         &alpha,
                                         A,
                                         cublas_data_type,
                                         lda,
                                         strideA,
                                         B,
                                         cublas_data_type,
                                         ldb,
                                         strideB,
                                         &beta,
                                         C,
                                         cublas_data_type,
                                         ldc,
                                         strideC,
                                         m->num_q_heads,
                                         compute_type,
                                         CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // print_tensor<float>((float*)C, 32, "C");
    // add alibi position bias to qk production
    // add alibi position bias to qk production
    if (*m->position_bias) {
      size_t parallelism = m->num_q_heads * total_tokens * num_new_tokens;
      apply_position_bias_qkprd<<<GET_BLOCKS(parallelism),
                                  min((size_t)CUDA_NUM_THREADS, parallelism),
                                  0,
                                  stream>>>(C,
                                            num_new_tokens,
                                            total_tokens,
                                            m->num_q_heads,
                                            m->global_num_q_heads,
                                            shard_id);
    }
    // Fill all elements above diagonal in qk prods with -inf to force
    // causal attention.
    assert(num_new_tokens <= total_tokens);
    if (num_new_tokens > 1) {
      size_t parallelism = m->num_q_heads * num_new_tokens * total_tokens;
      spec_fill_entries_above_diagonal<<<GET_BLOCKS(parallelism),
                                         min((size_t)CUDA_NUM_THREADS,
                                             parallelism),
                                         0,
                                         stream>>>(C,
                                                   num_new_tokens,
                                                   total_tokens,
                                                   m->num_q_heads,
                                                   static_cast<DT>(-INFINITY));
    }
    // Compute Softmax(QK^T/sqrt(d_k))
    // Before modifying the parameters below, make sure to read the following
    // description of the HIPDNN_TENSOR_NCHW tensor layout, from
    // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnTensorFormat_t:
    // This tensor format specifies that the data is laid out in the following
    // order: batch size, feature maps, rows, columns. The strides are
    // implicitly defined in such a way that the data are contiguous in memory
    // with no padding between images, feature maps, rows, and columns; the
    // columns are the inner dimension and the images are the outermost
    // dimension.
    int n_param = m->num_q_heads;
    int c_param = total_tokens;
    int h_param = 1;
    int w_param = num_new_tokens;
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->qk_tensor,
                                          HIPDNN_TENSOR_NCHW,
                                          cudnn_data_type,
                                          n_param,
                                          c_param,
                                          h_param,
                                          w_param));
    float softmax_alpha = 1.0f, softmax_beta = 0.0f;
    DT *C_softmax = static_cast<DT *>(m->qk_prods_softmax) +
                    m->num_q_heads * tokens_prev_requests_squares;
    // The softmax operation below is executed according to the
    // HIPDNN_SOFTMAX_MODE_CHANNEL, which is also described in the docs: The
    // softmax operation is computed per spatial location (H,W) per image (N)
    // across dimension C.
    checkCUDNN(hipdnnSoftmaxForward(m->handle.dnn,
                                   HIPDNN_SOFTMAX_ACCURATE,
                                   HIPDNN_SOFTMAX_MODE_CHANNEL,
                                   &softmax_alpha,
                                   m->qk_tensor,
                                   C,
                                   &softmax_beta,
                                   m->qk_tensor,
                                   C_softmax));
    // Matmul softmax(QK^T/sqrt(d_k)) by V
    alpha = 1.0f, beta = 0.0f;
    m_ = m->vProjSize;
    n = num_new_tokens;
    k = total_tokens;
    lda = m_ * m->num_q_heads, ldb = n, ldc = m_ * m->num_q_heads;
    strideA = vt_block_size;
    strideB = num_new_tokens * total_tokens;
    strideC = m->vProjSize;
    // To get A, skip over V^T entries from previous requests (all heads +
    // padding)
    A = static_cast<DT *>(m->valueCache) + i * vt_req_block_size;
    // To get B, skip over softmax(QK^T/sqrt(d_k)) entries from previous
    // requests (all heads)
    B = C_softmax;
    // To get C, skip over softmax(QK^T/sqrt(d_k))V products from previous
    // requests

    int token_offset = bc->requestsInfo[i].first_token_offset_in_batch;

    C = static_cast<DT *>(m->attn_heads) +
        (token_offset)*m->num_q_heads * m->vProjSize;
    checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                         HIPBLAS_OP_N,
                                         HIPBLAS_OP_T,
                                         m_,
                                         n,
                                         k,
                                         &alpha,
                                         A,
                                         cublas_data_type,
                                         lda,
                                         strideA,
                                         B,
                                         cublas_data_type,
                                         ldb,
                                         strideB,
                                         &beta,
                                         C,
                                         cublas_data_type,
                                         ldc,
                                         strideC,
                                         m->num_q_heads,
                                         compute_type,
                                         CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    tokens_previous_requests += num_new_tokens;
    tokens_prev_requests_squares += num_new_tokens * total_tokens;
  }

  assert(tokens_previous_requests == (num_tokens - bc->num_generation_tokens));
}

template <typename DT>
void inference_kernel(SpecIncMultiHeadSelfAttentionMeta const *m,
                      BeamSearchBatchConfig const *bc,
                      int shard_id,
                      DT const *input_ptr,
                      DT const *weight_ptr,
                      DT *output_ptr,
                      DT const *bias_ptr,
                      hipStream_t stream) {
  // phase 1: Implement kernel to compute KQV for input tokens

  compute_qkv_kernel(m,
                     bc,
                     shard_id,
                     input_ptr,
                     weight_ptr,
                     static_cast<DT *>(m->devQKVProjArray),
                     bias_ptr,
                     stream);
  // phase 2: Update key/val cache
  update_kv_cache_kernel<DT>(m, bc, stream);
  if (bc->num_generation_tokens > 0) {
    compute_spec_inc_attention_kernel_generation<DT>(
        m, bc, static_cast<DT *>(m->attn_heads), stream);
  }
  // phase 3: Compute attention score
  // 3 kernels for pahse 3: matmul1 - softmax - matmal2
  if (bc->num_tokens > bc->num_generation_tokens) {
    compute_attention_kernel_prompt(
        m, bc, shard_id, output_ptr, bias_ptr, weight_ptr, stream);
  }
  // compute output production and bias together for all tokens
  int num_tokens = bc->num_active_tokens();

  compute_o_prod_bias(
      m, bc, shard_id, output_ptr, weight_ptr, bias_ptr, num_tokens, stream);
}

} // namespace SpecIncMultiHeadSelfAttention
} // namespace Kernels

/*static*/
void SpecIncMultiHeadSelfAttention::inference_kernel_wrapper(
    SpecIncMultiHeadSelfAttentionMeta const *m,
    BeamSearchBatchConfig const *bc,
    int shard_id,
    GenericTensorAccessorR const &input,
    GenericTensorAccessorR const &weight,
    GenericTensorAccessorW const &output,
    GenericTensorAccessorR const &bias) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  bool use_bias = *m->qkv_bias || *m->final_bias;

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  assert(input.data_type == weight.data_type);
  assert(input.data_type == output.data_type);
  if (use_bias) {
    assert(input.data_type == bias.data_type);
  }

  if (input.data_type == DT_HALF) {
    half const *bias_ptr =
        use_bias ? bias.get_half_ptr() : static_cast<half const *>(nullptr);
    Kernels::SpecIncMultiHeadSelfAttention::inference_kernel(
        m,
        bc,
        shard_id,
        input.get_half_ptr(),
        weight.get_half_ptr(),
        output.get_half_ptr(),
        bias_ptr,
        stream);
  } else if (input.data_type == DT_FLOAT) {
    float const *bias_ptr =
        use_bias ? bias.get_float_ptr() : static_cast<float const *>(nullptr);
    Kernels::SpecIncMultiHeadSelfAttention::inference_kernel(
        m,
        bc,
        shard_id,
        input.get_float_ptr(),
        weight.get_float_ptr(),
        output.get_float_ptr(),
        bias_ptr,
        stream);
  } else {
    assert(false && "Unspported data type");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("SpecIncMultiHeadSelfAttention forward time = %.2fms\n", elapsed);
    // print_tensor<3, float>(acc_query.ptr, acc_query.rect,
    // "[Attention:forward:query]"); print_tensor<3, float>(acc_output.ptr,
    // acc_output.rect, "[Attention:forward:output]");
  }
}

SpecIncMultiHeadSelfAttentionMeta::SpecIncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    SpecIncMultiHeadSelfAttention const *attn,
    GenericTensorAccessorR const &weight,
    MemoryAllocator &gpu_mem_allocator,
    int num_samples,
    int _num_q_heads,
    int _num_kv_heads)
    : IncMultiHeadSelfAttentionMeta(handler,
                                    BEAM_SEARCH_MODE,
                                    attn,
                                    attn->qSize,
                                    attn->kSize,
                                    attn->vSize,
                                    attn->qProjSize,
                                    attn->kProjSize,
                                    attn->vProjSize,
                                    attn->oProjSize,
                                    attn->apply_rotary_embedding,
                                    attn->qkv_bias,
                                    attn->scaling_query,
                                    attn->qk_prod_scaling,
                                    attn->position_bias,
                                    attn->final_bias,
                                    attn->scaling_factor,
                                    weight,
                                    gpu_mem_allocator,
                                    num_samples,
                                    attn->num_q_heads,
                                    attn->num_kv_heads,
                                    _num_q_heads,
                                    _num_kv_heads,
                                    DT_NONE,
                                    false) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  checkCUDNN(hipdnnSetStream(handler.dnn, stream));

  // allocate memory for the seqArray and reserve space
  {
    beam_token_infos =
        reinterpret_cast<BeamSearchBatchConfig::BeamSearchPerTokenInfo *>(
            reinterpret_cast<char *>(handler.batch_config_metadata) +
            sizeof(BatchConfig::tokensInfo) +
            sizeof(BatchConfig::requestsInfo));

    beam_request_infos =
        reinterpret_cast<BeamSearchBatchConfig::BeamSearchPerRequestInfo *>(
            reinterpret_cast<char *>(handler.batch_config_metadata) +
            sizeof(BatchConfig::tokensInfo) +
            sizeof(BatchConfig::requestsInfo) +
            sizeof(BeamSearchBatchConfig::beamTokenInfo));
    causalMask = reinterpret_cast<BatchConfig::BitMask *>(
        reinterpret_cast<char *>(handler.batch_config_metadata) +
        sizeof(BatchConfig::tokensInfo) + sizeof(BatchConfig::requestsInfo) +
        sizeof(BeamSearchBatchConfig::beamTokenInfo) +
        sizeof(BeamSearchBatchConfig::beamRequestsInfo));

    request_completed = reinterpret_cast<bool *>(
        reinterpret_cast<char *>(handler.batch_config_metadata) +
        sizeof(BatchConfig::tokensInfo) + sizeof(BatchConfig::requestsInfo) +
        sizeof(BeamSearchBatchConfig::beamTokenInfo) +
        sizeof(BeamSearchBatchConfig::beamRequestsInfo) +
        sizeof(BatchConfig::causalMask));
  }

  hipStreamSynchronize(stream);
}

SpecIncMultiHeadSelfAttentionMeta::~SpecIncMultiHeadSelfAttentionMeta(void) {
  if (beam_search_reserve_inst != Realm::RegionInstance::NO_INST) {
    beam_search_reserve_inst.destroy();
  }
}

}; // namespace FlexFlow
