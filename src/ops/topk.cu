#include "hip/hip_runtime.h"
/* Copyright 2021 Facebook
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

// For an input tensor, computes the top k entries in each row
// (resp. vector along the last dimension). Thus,
// values.shape = indices.shape = input.shape[:-1] + [k]
void FFModel::top_k(const Tensor& input,
                    Tensor* outputs,
                    int k,
                    bool sorted,
                    const char *name)
{
  TopK* topk = new TopK(*this, input, k, sorted, name);
  layers.push_back(topk);
  assert(topk->numOutputs == 2);
  outputs[0] = topk->outputs[0];
  outputs[1] = topk->outputs[1];
}

TopK::TopK(FFModel& model,
           const Tensor& _input,
           int _k, bool _sorted,
           const char* name)
: Op(model, OP_TOPK, name, _input),
  k(_k), sorted(_sorted), profiling(model.config.profiling)
{
  numOutputs = 2;
  outputs[0].numDim = inputs[0].numDim;
  outputs[1].numDim = inputs[0].numDim;
  outputs[0].adim[0] = k;
  outputs[1].adim[0] = k;
  for (int i = 1; i < inputs[0].numDim; i++) {
    outputs[0].adim[i] = outputs[1].adim[i] = inputs[0].adim[i];
  }
  numWeights = 0;
}

void TopK::create_weights(FFModel& model)
{
  // Do nothing
}

void TopK::create_output_and_partition(FFModel& model)
{
  int dim = inputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      create_output_and_partition_with_dim<DIM>(model); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
    {
      // Unsupported dim for ElementWiseBinary operator
      assert(false);
    }
  }
}

template<int NDIM>
void TopK::create_output_and_partition_with_dim(FFModel& model)
{
  // Retrive the task indexspace for the op
  task_is = IndexSpaceT<NDIM>(model.get_or_create_task_is(NDIM, name));
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<NDIM> part_rect = runtime->get_index_space_domain(ctx, task_is);
  int dims[NDIM];
  dims[NDIM-1] = k;
  for (int i = 0; i < NDIM-1; i++)
    dims[i] = inputs[0].adim[NDIM-1-i];
  outputs[0] = model.create_tensor<NDIM>(dims, DT_FLOAT, this);
  outputs[0].owner_op = this;
  outputs[0].owner_idx = 0;
  outputs[1] = model.create_tensor<NDIM>(dims, DT_INT32, this);
  outputs[1].owner_op = this;
  outputs[1].owner_idx = 1;
  Rect<NDIM> input_rect;
  input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part.get_index_partition());
  if (input_rect == part_rect) {
    input_lps[0] = inputs[0].part;
    input_grad_lps[0] = inputs[0].part_grad;
  } else {
    model.create_disjoint_partition<NDIM>(
        inputs[0], IndexSpaceT<NDIM>(task_is), input_lps[0], input_grad_lps[0]);
  }
}

OpMeta* TopK::init_task(const Task* task,
                        const std::vector<PhysicalRegion> &regions,
                        Context ctx, Runtime* runtime)
{
  TopK* topk = (TopK*) task->args;
  FFHandler handle = *((FFHandler*)task->local_args);
  TopKMeta* m = new TopKMeta(handle);
  m->profiling = topk->profiling;
  m->sorted = topk->sorted;
  return m;
}

void TopK::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      ParallelConfig pc; \
      std::string pcname = name; \
      ff.config.find_parallel_config(DIM, pcname, pc); \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        FFHandler handle = ff.handlers[pc.device_ids[idx++]]; \
        argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(TOPK_INIT_TASK_ID, task_is,
                         TaskArgument(this, sizeof(TopK)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(input_lps[0], 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
    RegionRequirement(outputs[1].part, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, outputs[1].region));
  launcher.add_field(2, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        meta[idx++] = fm.get_result<OpMeta*>(*it); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

enum class HeapType { kMinHeap, kMaxHeap };
enum class PreferIndices { kLower, kHigher };

template <typename T>
struct Entry {
  int index;
  T value;
};

template <typename T>
struct LinearData {
  typedef Entry<T> Entry;

  __device__ Entry& operator[](std::size_t index) const { return data[index]; }

  __device__ int get_index(int i) const { return data[i].index; }
  __device__ T get_value(int i) const { return data[i].value; }

  Entry* const data;
};

template <typename T>
struct IndirectLinearData {
  typedef Entry<T> Entry;

  __device__ Entry& operator[](std::size_t index) const { return data[index]; }

  __device__ int get_index(int i) const {
    return backing_data[data[i].index].index;
  }
  __device__ T get_value(int i) const { return data[i].value; }

  Entry* const data;
  Entry* const backing_data;
};

template <typename T>
struct StridedData {
  typedef Entry<T> Entry;

  __device__ Entry& operator[](std::size_t index) const {
    return data[index * blockDim.x + threadIdx.x];
  }

  __device__ int get_index(int i) const { return (*this)[i].index; }
  __device__ T get_value(int i) const { return (*this)[i].value; }

  Entry* const data;
};

// A heap of Entry<T> that can either work as a min-heap or as a max-heap.
template <HeapType heapType, PreferIndices preferIndices,
          template <typename> class Data, typename T>
struct IndexedHeap {
  typedef typename Data<T>::Entry Entry;
  const Data<T> data;
  __device__ IndexedHeap(const Data<T>& d) : data(d) {}

  __device__ bool is_above(int left, int right) {
    T left_value = data.get_value(left);
    T right_value = data.get_value(right);
    if (left_value == right_value) {
      if (preferIndices == PreferIndices::kLower) {
        return data.get_index(left) < data.get_index(right);
      } else {
        return data.get_index(left) > data.get_index(right);
      }
    }
    if (heapType == HeapType::kMinHeap) {
      return left_value < right_value;
    } else {
      return left_value > right_value;
    }
  }

  __device__ void assign(int i, const Entry& entry) { data[i] = entry; }

  __device__ void push_up(int i) {
    int child = i;
    int parent;
    for (; child > 0; child = parent) {
      parent = (child - 1) / 2;
      if (!is_above(child, parent)) {
        // Heap property satisfied.
        break;
      }
      swap(child, parent);
    }
  }

  __device__ void swap(int a, int b) {
    auto tmp = data[b];
    data[b] = data[a];
    data[a] = tmp;
  }

  __device__ void push_root_down(int k) { push_down(0, k); }

  // MAX-HEAPIFY in Cormen
  __device__ void push_down(int node, int k) {
    while (true) {
      const int left = 2 * node + 1;
      const int right = left + 1;
      int smallest = node;
      if (left < k && is_above(left, smallest)) {
        smallest = left;
      }
      if (right < k && is_above(right, smallest)) {
        smallest = right;
      }
      if (smallest == node) {
        break;
      }
      swap(smallest, node);
      node = smallest;
    }
  }

  // BUILD-MAX-HEAPIFY in Cormen
  __device__ void build(int k) {
    for (int node = (k - 1) / 2; node >= 0; node--) {
      push_down(node, k);
    }
  }

  // HEAP-EXTRACT-MAX in Cormen
  __device__ void remove_root(int k) {
    data[0] = data[k - 1];
    push_root_down(k - 1);
  }

  // in-place HEAPSORT in Cormen
  // This method destroys the heap property.
  __device__ void sort(int k) {
    for (int slot = k - 1; slot > 0; slot--) {
      // This is like remove_root but we insert the element at the end.
      swap(slot, 0);
      // Heap is now an element smaller.
      push_root_down(/*k=*/slot);
    }
  }

  __device__ void replace_root(const Entry& entry, int k) {
    data[0] = entry;
    push_root_down(k);
  }

  __device__ const Entry& root() { return data[0]; }
};

template <HeapType heapType, PreferIndices preferIndices,
          template <typename> class Data, typename T>
__device__ IndexedHeap<heapType, preferIndices, Data, T> make_indexed_heap(
    typename Data<T>::Entry* data) {
  return IndexedHeap<heapType, preferIndices, Data, T>{Data<T>{data}};
}

// heapTopK walks over [input, input+length) with `step_size` stride starting at
// `start_index`.
// It builds a top-`k` heap that is stored in `heap_entries` using `Accessor` to
// access elements in `heap_entries`. If sorted=true, the elements will be
// sorted at the end.
template <typename T, template <typename> class Data = LinearData>
__device__ void heapTopK(const T* __restrict__ input, int length, int k,
                         Entry<T>* __restrict__ heap_entries,
                         bool sorted = false, int start_index = 0,
                         int step_size = 1)
{
  assert(k <= length);

  auto heap =
      make_indexed_heap<HeapType::kMinHeap, PreferIndices::kHigher, Data, T>(
          heap_entries);

  int heap_end_index = start_index + k * step_size;
  if (heap_end_index > length) {
    heap_end_index = length;
  }
  // Initialize the min-heap.
  for (int index = start_index, slot = 0; index < heap_end_index;
       index += step_size, slot++) {
    heap.assign(slot, {index, input[index]});
  }

  heap.build(k);

  // Now iterate over the remaining items.
  // If an item is smaller than the min element, it is not amongst the top k.
  // Otherwise, replace the min element with it and push upwards.
  for (int index = heap_end_index; index < length; index += step_size) {
    // We prefer elements with lower indices. This is given here.
    // Later elements automatically have higher indices, so can be discarded.
    if (input[index] > heap.root().value) {
      // This element should replace the min.
      heap.replace_root({index, input[index]}, k);
    }
  }

  // Sort if wanted.
  if (sorted) {
    heap.sort(k);
  }
}

// mergeShards performs a top-k merge on `num_shards` many sorted streams that
// are sorted and stored in `entries` in a strided way:
// |s_1 1st|s_2 1st|...s_{num_shards} 1st|s_1 2nd|s_2 2nd|...
// The overall top k elements are written to `top_k_values` and their indices
// to top_k_indices.
// `top_k_heap` is used as temporary storage for the merge heap.
template <typename T> __device__
void mergeShards(int num_shards, int k,
                 Entry<T>* __restrict__ entries,
                 Entry<T>* __restrict__ top_k_heap, T* top_k_values,
                 int* top_k_indices)
{
  // If k < num_shards, we can use a min-heap with k elements to get the top k
  // of the sorted blocks.
  // If k > num_shards, we can initialize a min-heap with the top element from
  // each sorted block.
  const int heap_size = k < num_shards ? k : num_shards;

  // Min-heap part.
  {
    auto min_heap = IndexedHeap<HeapType::kMinHeap, PreferIndices::kHigher,
                                IndirectLinearData, T>{
        IndirectLinearData<T>{top_k_heap, entries}};
    // Initialize the heap as a min-heap.
    for (int slot = 0; slot < heap_size; slot++) {
      min_heap.assign(slot, {slot, entries[slot].value});
    }
    min_heap.build(heap_size);

    // Now perform top k with the remaining shards (if num_shards > heap_size).
    for (int shard = heap_size; shard < num_shards; shard++) {
      const auto entry = entries[shard];
      const auto root = min_heap.root();
      if (entry.value < root.value) {
        continue;
      }
      if (entry.value == root.value &&
          entry.index > entries[root.index].index) {
        continue;
      }
      // This element should replace the min.
      min_heap.replace_root({shard, entry.value}, heap_size);
    }
  }

  // Max-part.
  {
    // Turn the min-heap into a max-heap in-place.
    auto max_heap = IndexedHeap<HeapType::kMaxHeap, PreferIndices::kLower,
                                IndirectLinearData, T>{
        IndirectLinearData<T>{top_k_heap, entries}};
    // Heapify into a max heap.
    max_heap.build(heap_size);

    // Now extract the minimum k-1 times.
    // k is treated specially.
    const int last_k = k - 1;
    for (int rank = 0; rank < last_k; rank++) {
      const Entry<T>& max_element = max_heap.root();
      top_k_values[rank] = max_element.value;
      int shard_index = max_element.index;
      top_k_indices[rank] = entries[shard_index].index;
      int next_shard_index = shard_index + num_shards;
      // For rank < k-1, each top k heap still contains at least 1 element,
      // so we can draw a replacement.
      max_heap.replace_root({next_shard_index, entries[next_shard_index].value},
                            heap_size);
    }

    // rank == last_k.
    const Entry<T>& max_element = max_heap.root();
    top_k_values[last_k] = max_element.value;
    int shard_index = max_element.index;
    top_k_indices[last_k] = entries[shard_index].index;
  }
}

template <typename T>
__global__ void
topk_forward_kernel(const T* __restrict__ input,
                    size_t shared_memory_size,
                    int length, int k, bool sorted,
                    T* __restrict__ output,
                    int* __restrict__ indices)
{
  __shared__ char shared_memory[48 << 10];
  const int batch_index = blockIdx.x;
  const T* batch_input = input + batch_index * length;
  const int thread_index = threadIdx.x;
  const int thread_count = blockDim.x;
  Entry<T>* shared_entries = (Entry<T>*)shared_memory;
  heapTopK<T, StridedData>(batch_input, length, k, shared_entries, true,
                           thread_index, thread_count);
  __syncthreads();
  if (thread_index == 0) {
    const int offset = batch_index * k;
    auto batch_output = output + offset;
    auto batch_indices = indices + offset;
    Entry<T>* top_k_heap = shared_entries + thread_count * k;
     mergeShards(thread_count, k, shared_entries, top_k_heap, batch_output,
                batch_indices);
  }
}

/*static*/
void TopK::forward_kernel(const TopKMeta* m,
                          const float* input_ptr,
                          float* output_ptr,
                          int* indices_ptr,
                          size_t batch_size, int length, int k,
                          bool sorted,
                          hipStream_t stream)
{
  // Adopted from TensorFlow's TopK implementation
  // https://github.com/tensorflow/tensorflow/blob/master/tensorflow/core/kernels/topk_op_gpu.h
  int num_shards = 0;
  {
    constexpr auto shared_memory_size = 48 << 10;
    const auto heap_size = k * sizeof(Entry<float>);
    // shared_memory_size = (num_shards + 1) * heap_size <=>
    num_shards = shared_memory_size / heap_size - 1;
    assert(num_shards > 0);
    if (num_shards > CUDA_NUM_THREADS)
      num_shards = CUDA_NUM_THREADS;
  }
  // We are limited by the amount of shared memory we have per block.
  size_t shared_memory_size = (num_shards + 1) * k * sizeof(Entry<float>);
  //size_t num_blocks = (batch_size + num_shards - 1) / num_shards;
  size_t num_blocks = batch_size;
  assert(num_shards >= (size_t)k);
  num_shards = k;
  topk_forward_kernel<<<num_blocks, num_shards, 0, stream>>>(
    input_ptr, shared_memory_size, length, k, sorted,
    output_ptr, indices_ptr);
}

void TopK::forward_task(const Task* task,
                        const std::vector<PhysicalRegion> &regions,
                        Context ctx, Runtime* runtime)
{


  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  //const TopK* topk = (const TopK*) task->args;
  const TopKMeta* m = *((TopKMeta**)task->local_args);
  Domain in1_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain out1_domain = runtime->get_index_space_domain(
    ctx, task->regions[1].region.get_index_space());
  Domain out2_domain = runtime->get_index_space_domain(
    ctx, task->regions[2].region.get_index_space());

  int in_cols = in1_domain.hi()[0] - in1_domain.lo()[0] + 1;
  int out1_cols = out1_domain.hi()[0] - out1_domain.lo()[0] + 1;
  int out2_cols = out2_domain.hi()[0] - out2_domain.lo()[0] + 1;

  assert(out1_domain == out2_domain);
  for (int i = 1; i < in1_domain.get_dim(); i++) {
    assert(in1_domain.lo()[i] == out1_domain.lo()[i]);
    assert(in1_domain.hi()[i] == out1_domain.hi()[i]);
  }
  const float* in_ptr = helperGetTensorPointerRO<float>(
    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  float* value_ptr = helperGetTensorPointerWO<float>(
    regions[1], task->regions[1], FID_DATA, ctx, runtime);
  int* index_ptr = helperGetTensorPointerWO<int>(
    regions[2], task->regions[2], FID_DATA, ctx, runtime);
  
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  int length = in1_domain.hi()[0] - in1_domain.lo()[0] + 1;
  int k = out1_domain.hi()[0] - out1_domain.lo()[0] + 1; /*TODO: This prints to 5*/
  size_t batch_size = in1_domain.get_volume() / length;

  forward_kernel(m, in_ptr, value_ptr, index_ptr,
      batch_size, length, k, m->sorted, stream);

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
  }
}

void TopK::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(TOPK_FWD_TASK_ID, task_is,
                         TaskArgument(NULL, 0), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(input_lps[0], 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
    RegionRequirement(outputs[1].part, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, outputs[1].region));
  launcher.add_field(2, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

template<typename T>
__global__ void
topk_backward_kernel(const T* __restrict__ value_grad_ptr,
                     const int* __restrict__ indices_ptr,
                     T* __restrict__ in_grad_ptr,
                     size_t batch_size, int length, int k)
{
  coord_t size = (coord_t)batch_size * k;
  CUDA_KERNEL_LOOP(i, size)
  {
    coord_t batch_idx = i / k;
    coord_t src_offset = batch_idx * length + indices_ptr[i];
    in_grad_ptr[src_offset] += value_grad_ptr[i];
  }
}

/*static*/
void TopK::backward_kernel(const TopKMeta* m,
                           const float* value_grad_ptr,
                           const int* indices_ptr,
                           float* in_grad_ptr,
                           size_t batch_size, int length, int k,
                           hipStream_t stream)
{
  topk_backward_kernel<<<GET_BLOCKS(batch_size*k), CUDA_NUM_THREADS, 0, stream>>>(
    value_grad_ptr, indices_ptr, in_grad_ptr, batch_size, length, k);
}

/*
  regions[0](I): out1_grad
  regions[1](I): out2
  regions[2](I/0): in_grad
*/
void TopK::backward_task(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime* runtime)
{
  //const TopK* topk = (const TopK*) task->args;
  const TopKMeta* m = *((TopKMeta**) task->local_args);
  assert(regions.size() == 3);
  Domain out1_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain out2_domain = runtime->get_index_space_domain(
    ctx, task->regions[1].region.get_index_space());
  Domain in_domain = runtime->get_index_space_domain(
    ctx, task->regions[2].region.get_index_space());
  assert(out1_domain == out2_domain);
  for (int i = 1; i < in_domain.get_dim(); i++) {
    assert(in_domain.lo()[i] == out1_domain.lo()[i]);
    assert(in_domain.hi()[i] == out1_domain.hi()[i]);
  }
  const float* value_grad_ptr = helperGetTensorPointerRO<float>(
    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  const int* indices_ptr = helperGetTensorPointerRO<int>(
    regions[1], task->regions[1], FID_DATA, ctx, runtime);
  float* in_grad_ptr = helperGetTensorPointerRW<float>(
    regions[2], task->regions[2], FID_DATA, ctx, runtime);
  
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  int length = in_domain.hi()[0] - in_domain.lo()[0] + 1;
  int k = out1_domain.hi()[0] - out1_domain.lo()[0] + 1;
  size_t batch_size = in_domain.get_volume() / length;
  backward_kernel(m, value_grad_ptr, indices_ptr, in_grad_ptr,
      batch_size, length, k, stream);
  
  // TODO: missing profiling here
}

void TopK::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }

  IndexLauncher launcher(TOPK_BWD_TASK_ID, task_is,
                         TaskArgument(NULL, 0), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  // regions[0](I): value_grad
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
                      READ_ONLY, EXCLUSIVE, outputs[0].region_grad));
  launcher.add_field(0, FID_DATA);
  // regions[1](I): indices
  launcher.add_region_requirement(
    RegionRequirement(outputs[1].part, 0/*projection id*/,
                      READ_ONLY, EXCLUSIVE, outputs[1].region));
  launcher.add_field(1, FID_DATA);
  // regions[2](I/O): input_grad
  launcher.add_region_requirement(
    RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                      READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(2, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

TopKMeta::TopKMeta(FFHandler handler)
: OpMeta(handler)
{
}

bool TopK::measure_operator_cost(Simulator* sim,
                                 const ParallelConfig& pc,
                                 CostMetrics& cost_metrics)
{
  // To be implemented
  assert(false);
  return false;
}
