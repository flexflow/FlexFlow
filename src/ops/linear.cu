#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford, NVIDIA, Facebook
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::dense(const Tensor& input,
                      int outDim,
                      ActiMode activation,
                      bool use_bias,
                      const Op* shared_op,
                      Initializer* kernel_initializer,
                      Initializer* bias_initializer,
                      const char *name)
{
  if (kernel_initializer == NULL) {
    int seed = std::rand();
    kernel_initializer = new GlorotUniform(seed);
  }
  if (bias_initializer == NULL) {
    bias_initializer = new ZeroInitializer();
  }
  Linear *li = new Linear(*this, input, outDim, activation, use_bias,
                          shared_op, kernel_initializer, bias_initializer, name);
  layers.push_back(li);
  return li->outputs[0];
}

Linear::Linear(FFModel& model,
               const Tensor& _input,
               int out_dim,
               ActiMode _activation,
               bool _use_bias,
               const Op* shared_op,
               Initializer* _kernel_initializer,
               Initializer* _bias_initializer,
               const char* name)
: Op(model, OP_LINEAR, shared_op, name, _input),
  in_channels(_input.adim[0]), out_channels(out_dim),
  activation(_activation), use_bias(_use_bias),
  kernel_initializer(_kernel_initializer),
  bias_initializer(_bias_initializer)
{
  numInputs = 1;
  numOutputs = 1;
  outputs[0].numDim = _input.numDim;
  for (int i = 1; i < outputs[0].numDim; i++)
    outputs[0].adim[i] = _input.adim[i];
  outputs[0].adim[0] = out_dim;
  weights[0].numDim = 2;
  weights[0].adim[0] = in_channels;
  weights[0].adim[1] = out_channels;
  numWeights = 1;
  if (use_bias) {
    weights[1].numDim = 1;
    weights[1].adim[0] = out_channels;
    numWeights = 2;
  }
}

void Linear::create_weights(FFModel& model)
{
  int dim = inputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      create_weights_with_dim<DIM>(model); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
    {
      // Unsupported dim
      assert(false);
    }
  }
}

template<int NDIM>
void Linear::create_weights_with_dim(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = IndexSpaceT<NDIM>(model.get_or_create_task_is(NDIM, pcname));

#ifdef FF_USE_NCCL
  ParameterSyncType comm_type = ParameterSyncType::NCCL;
#else
  ParameterSyncType comm_type = ParameterSyncType::PS;
#endif

  // Create kernel tensor
  {
    const int dims[2] = {out_channels, in_channels};
    weights[0] = model.create_linear_weight<2, NDIM>(this, dims, DT_FLOAT,
        kernel_initializer, true/*create_grad*/, comm_type);
  }
  // Create bias tensor
  if (use_bias) {
    const int dims[1] = {out_channels};
    weights[1] = model.create_linear_weight<1, NDIM>(this, dims, DT_FLOAT,
        bias_initializer, true/*create_grad*/, comm_type);
    assert(numWeights == 2);
  } else {
    assert(numWeights == 1);
  }
}

void Linear::create_output_and_partition(FFModel& model)
{
  int dim = inputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      create_output_and_partition_with_dim<DIM>(model); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
    {
      // Unsupported dim for ElementWiseBinary operator
      assert(false);
    }
  }
}

template<int NDIM>
void Linear::create_output_and_partition_with_dim(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = IndexSpaceT<NDIM>(model.get_or_create_task_is(NDIM, pcname));

  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<NDIM> part_rect = runtime->get_index_space_domain(ctx, task_is);
  int num_par_c = part_rect.hi[0] - part_rect.lo[0] + 1;
  int num_par_n = part_rect.hi[NDIM-1] - part_rect.lo[NDIM-1] + 1;
  int in_dim = inputs[0].adim[0];
  assert(in_dim == in_channels);
  int batch_size = inputs[0].adim[NDIM-1];
  {
    int dims[NDIM];
    for (int i = 0; i < NDIM; i++)
      dims[i] = outputs[0].adim[NDIM-1-i];
    outputs[0] = model.create_tensor<NDIM>(dims, DT_FLOAT, this);
    outputs[0].owner_op = this;
    outputs[0].owner_idx = 0;
  }
  // Compute partition bound for input
  Rect<NDIM> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part.get_index_partition());
  // Create replica tensor
  if (num_par_c > 1) {
    {
      Rect<NDIM> extent;
      for (int i = 1; i < NDIM; i++) {
        extent.lo[i] = 0;
        assert(outputs[0].adim[i] % (part_rect.hi[i] - part_rect.lo[i] + 1) == 0);
        extent.hi[i] = outputs[0].adim[i] / (part_rect.hi[i] - part_rect.lo[i] + 1) - 1;
      }
      extent.lo[0] = 0;
      extent.hi[0] = in_dim-1;
      Transform<NDIM, NDIM> transform;
      for (int i = 0; i < NDIM; i++)
        for (int j = 0; j < NDIM; j++)
          transform[i][j] = 0;
      for (int i = 1; i < NDIM; i++)
        transform[i][i] = extent.hi[i] + 1;
      IndexPartition ip = runtime->create_partition_by_restriction(
          ctx, inputs[0].region.get_index_space(), task_is, transform, extent);
      assert(runtime->is_index_partition_complete(ctx, ip));
      input_lps[0] = runtime->get_logical_partition(
          ctx, inputs[0].region, ip);
    }
    if (model.config.computationMode == COMP_MODE_TRAINING) {
      if (NDIM==1) {
        const int dims[2] = {num_par_c, in_dim};
        replica = model.create_linear_replica<2>(dims, (IndexSpaceT<NDIM>)task_is, DT_FLOAT);
      } else if (NDIM==2) {
        const int dims[3] = {num_par_c, batch_size, in_dim};
        replica = model.create_linear_replica<3>(dims, (IndexSpaceT<NDIM>)task_is, DT_FLOAT);
      } else if (NDIM==3) {
        const int dims[4] = {num_par_c, batch_size, inputs[0].adim[1], in_dim};
        replica = model.create_linear_replica<4>(dims, (IndexSpaceT<NDIM>)task_is, DT_FLOAT);
      } else {
        assert(false && "Unsupported dimension for parallelizing Linear operators"
            " using the parameter dim.");
      }
      // Backward use the same ip as inputs[0]
      input_grad_lps[0] = inputs[0].part_grad;
      {
        IndexSpaceT<NDIM> input_task_is = IndexSpaceT<NDIM>(model.get_or_create_task_is(input_rect));
        Rect<NDIM+1> extent;
        for (int i = 0; i < NDIM; i++) {
          extent.lo[i] = 0;
          assert(inputs[0].adim[i] % (input_rect.hi[i] - input_rect.lo[i] + 1) == 0);
          extent.hi[i] = inputs[0].adim[i] / (input_rect.hi[i] - input_rect.lo[i] + 1) - 1;
        }
        extent.lo[NDIM] = 0;
        extent.hi[NDIM] = num_par_c - 1;
        Transform<NDIM+1, NDIM> transform;
        for (int i = 0; i < NDIM+1; i++)
          for (int j = 0; j < NDIM; j++)
            transform[i][j] = 0;
        for (int i = 0; i < NDIM; i++)
          transform[i][i] = inputs[0].adim[i] / (input_rect.hi[i] - input_rect.lo[i] + 1);
        IndexPartition ip = runtime->create_partition_by_restriction(
            ctx, replica.region_grad.get_index_space(), input_task_is,
            transform, extent);
        assert(runtime->is_index_partition_disjoint(ctx, ip));
        assert(runtime->is_index_partition_complete(ctx, ip));
        // Note we use replica.part to save how to partition the replica
        // to compute input_grad_lps
        replica.part = runtime->get_logical_partition(
            ctx, replica.region_grad, ip);
      }
    } // if COMP_MODE_TRAINING
  } else {
    // when num_par_c == 1
    if (input_rect == part_rect) {
      input_lps[0] = inputs[0].part;
      if (model.config.computationMode == COMP_MODE_TRAINING) {
        input_grad_lps[0] = inputs[0].part_grad;
      }
    } else {
      Rect<NDIM> extent;
      for (int i = 0; i < NDIM; i++) {
        extent.lo[i] = 0;
        assert(inputs[0].adim[i] % (part_rect.hi[i] - part_rect.lo[i] + 1) == 0);
        extent.hi[i] = inputs[0].adim[i] / (part_rect.hi[i] - part_rect.lo[i] + 1) - 1;
      }
      Transform<NDIM, NDIM> transform;
      for (int i = 0; i < NDIM; i++)
        for (int j = 0; j < NDIM; j++) {
          transform[i][j] = 0;
          if (i==j)
            transform[i][j] = extent.hi[i] + 1;
        }
      IndexPartition ip = runtime->create_partition_by_restriction(
          ctx, inputs[0].region.get_index_space(), task_is, transform, extent);
      assert(runtime->is_index_partition_disjoint(ctx, ip));
      assert(runtime->is_index_partition_complete(ctx, ip));
      input_lps[0] = runtime->get_logical_partition(
          ctx, inputs[0].region, ip);
      if (model.config.computationMode == COMP_MODE_TRAINING) {
        input_grad_lps[0] = runtime->get_logical_partition(
            ctx, inputs[0].region_grad, ip);
      }
    }
  }
}

/*
  regions[0](O): output
  regions[1](I): kernel
  regions[2](I): bias
*/
OpMeta* Linear::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime* runtime)
{
  Domain out_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (out_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return init_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  return NULL;
}

bool Linear::use_cudnn_activation(ActiMode mode)
{
  switch (mode) {
    case AC_MODE_RELU:
    case AC_MODE_SIGMOID:
    case AC_MODE_TANH:
      return true;
  }
  return false;
}

template<int NDIM>
OpMeta* Linear::init_task_with_dim(const Task *task,
                                   const std::vector<PhysicalRegion> &regions,
                                   Context ctx, Runtime *runtime)
{
  assert(regions.size() == task->regions.size());
  assert(regions.size() == 2 || regions.size() == 3);
  const Linear* linear = (Linear*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  //TensorAccessorR<float, 2> acc_input(
  //    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, NDIM> acc_output(
      regions[0], task->regions[0], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorR<float, 2> acc_kernel(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  // TensorAccessorR<float, 1> acc_bias(
  //     regions[3], task->regions[3], FID_DATA, ctx, runtime);
  //int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int in_dim = acc_kernel.rect.hi[0] - acc_kernel.rect.lo[0] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int batch_size = acc_output.rect.volume() / out_dim;
  printf("init linear (input): in_dim(%d) out_dim(%d) batch_size(%d)\n",
      in_dim, out_dim, batch_size);
  LinearMeta* m = new LinearMeta(handle, batch_size);
  m->activation = linear->activation;
  m->use_bias = linear->use_bias;
  m->profiling = linear->profiling;
  m->trainableInputs[0] = linear->trainableInputs[0];
  std::strcpy(m->op_name, linear->name);

  if (use_cudnn_activation(m->activation)) {
    hipdnnActivationMode_t mode;
    switch (linear->activation) {
      case AC_MODE_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case AC_MODE_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      default:
        // Unsupported activation mode
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          batch_size, out_dim, 1, 1));
  }
  return m;
}

void Linear::init(const FFModel& ff)
{
  int dim = outputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
      return init_with_dim<DIM>(ff);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

template<int NDIM>
void Linear::init_with_dim(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<NDIM> rect = runtime->get_index_space_domain(ctx, task_is);
  ParallelConfig pc;
  std::string pcname = name;
  ff.config.find_parallel_config(NDIM, pcname, pc);
  int idx = 0;
  for (PointInRectIterator<NDIM> it(rect); it(); it++) {
    FFHandler handle = ff.handlers[pc.device_ids[idx++]];
#ifdef FF_USE_NCCL
    handle.ncclComm = pc.nccl_comms[idx-1];
#endif
    argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
  }
  IndexLauncher launcher(LINEAR_INIT_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Linear)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  //launcher.add_region_requirement(
  //    RegionRequirement(input_lps[0], 0/*projection id*/,
  //                      READ_ONLY, EXCLUSIVE, inputs[0].region));
  //launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[0].region));
  launcher.add_field(1, FID_DATA);
  // launcher.add_region_requirement(
  //     RegionRequirement(weights[1].part, 0/*projection id*/,
  //                       READ_ONLY, EXCLUSIVE, weights[1].region));
  // launcher.add_field(3, FID_DATA);
  if (ff.config.computationMode == COMP_MODE_TRAINING) {
    // Add inputs[0].region_grad to avoid Legion warning
    //launcher.add_region_requirement(
    //    RegionRequirement(input_grad_lps[0], 0/*projection id*/,
    //        WRITE_ONLY, EXCLUSIVE, inputs[0].region_grad));
    //launcher.add_field(2, FID_DATA);
  }
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<NDIM> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/*static*/
void Linear::forward_kernel(const LinearMeta* m,
                            const float* input_ptr,
                            float* output_ptr,
                            const float* kernel_ptr,
                            const float* bias_ptr,
                            int in_dim, int out_dim, int batch_size,
                            hipStream_t stream)
{
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha = 1.0f, beta = 0.0f;
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        out_dim, batch_size, in_dim,
                        &alpha, kernel_ptr, in_dim,
                        input_ptr, in_dim, &beta,
                        output_ptr, out_dim));
  // use_bias = True 
  if (bias_ptr != NULL) { 
    checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                          out_dim, batch_size, 1,
                          &alpha, bias_ptr, 1,
                          m->one_ptr, 1, &alpha,
                          output_ptr, out_dim));
  }
  if (use_cudnn_activation(m->activation)) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
        &alpha, m->outputTensor, output_ptr,
        &beta, m->outputTensor, output_ptr));
  } else if (m->activation == AC_MODE_GELU) {
    size_t elements = (size_t)out_dim * (size_t) batch_size;
    constexpr float B = 0.7978845608028654f;   // sqrt(2.0/M_PI)
    constexpr float C = 0.035677408136300125f; // 0.044715 * sqrt(2.0/M_PI)
    gelu_forward_kernel<<<GET_BLOCKS(elements), CUDA_NUM_THREADS>>>(
        elements, B, C, output_ptr);
  } else if (m->activation == AC_MODE_NONE) {
    // Do nothing
  } else {
    assert(false && "Unsupported activation for Linear");
  }
}

__host__
void Linear::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return forward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

/*
  regions[0](I); input
  regions[1](O): output
  regions[2](I): kernel
  regions[3](I): bias
*/
template<int NDIM>
void Linear::forward_task_with_dim(const Task *task,
                                   const std::vector<PhysicalRegion> &regions,
                                   Context ctx, Runtime *runtime)
{
  //Linear* linear = (Linear*) task->args;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  assert(regions.size() == (3 + int(m->use_bias)));
  assert(task->regions.size() == (3 + int(m->use_bias)));
  
  TensorAccessorR<float, NDIM> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, NDIM> acc_output(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorR<float, 2> acc_kernel(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int batch_size = acc_output.rect.volume() / out_dim;
  assert(acc_output.rect.volume() == out_dim * batch_size);
  assert(acc_input.rect.volume() == in_dim * batch_size);
  assert(acc_kernel.rect.volume() == in_dim * out_dim);
  const float* acc_bias_ptr = NULL;
  if (m->use_bias) {
    TensorAccessorR<float, 1> acc_bias(
        regions[3], task->regions[3], FID_DATA, ctx, runtime);
    assert(acc_bias.rect.volume() == out_dim);
    acc_bias_ptr = acc_bias.ptr;
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  Linear::forward_kernel(m, acc_input.ptr, acc_output.ptr,
      acc_kernel.ptr, acc_bias_ptr, in_dim, out_dim, batch_size, stream);

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Linear] forward time = %.2lfms\n", m->op_name, elapsed);
    //print_tensor<NDIM, float>(acc_input.ptr, acc_input.rect, "[Linear:forward:input]");
    //print_tensor<2, float>(acc_kernel.ptr, acc_kernel.rect, "[Linear:forward:kernel]");
    //print_tensor<1, float>(acc_bias.ptr, acc_bias.rect, "[Linear:forward:bias]");
    //print_tensor<NDIM, float>(acc_output.ptr, acc_output.rect, "[Linear:forward:output]");
  }
}

void Linear::forward(const FFModel& ff)
{
  int dim = outputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
      return forward_with_dim<DIM>(ff);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

template<int NDIM>
void Linear::forward_with_dim(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<NDIM> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<NDIM> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(LINEAR_FWD_TASK_ID, task_is,
                         TaskArgument(NULL, 0), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[0].region));
  launcher.add_field(2, FID_DATA);
  if (use_bias) {
    launcher.add_region_requirement(
        RegionRequirement(weights[1].part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, weights[1].region));
    launcher.add_field(3, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

__global__
void sigmoid_backward(float *grad_ptr, const float *output, int n)
{
  CUDA_KERNEL_LOOP(i, n)
  {
    grad_ptr[i] = grad_ptr[i] * output[i] * (1 - output[i]);
  }
}

/*static*/
void Linear::backward_kernel(const LinearMeta* m,
                             const float* input_ptr,
                             float* input_grad_ptr,
                             const float* output_ptr,
                             float* output_grad_ptr,
                             const float* kernel_ptr,
                             float* kernel_grad_ptr,
                             float* bias_grad_ptr,
                             int in_dim, int out_dim, int batch_size,
                             hipStream_t stream)
{
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha = 1.0f;
  int output_size = out_dim * batch_size;
  if (m->activation == AC_MODE_RELU) {
    reluBackward<<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
        output_grad_ptr, output_ptr, output_size);
  } else if (m->activation == AC_MODE_SIGMOID) {
    sigmoid_backward<<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
        output_grad_ptr, output_ptr, output_size);
  } else {
    // TODO: only support relu and sigmoid for now
    assert(m->activation == AC_MODE_NONE);
  }
  // Compute weight gradiant
  // NOTE: we use alpha=1 for kernel_grad to accumulate gradients
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        in_dim, out_dim, batch_size,
                        &alpha, input_ptr, in_dim,
                        output_grad_ptr, out_dim,
                        &alpha, kernel_grad_ptr, in_dim));
  // Compute bias gradiant
  // NOTE: we use alpha=1 for bias_grad to accumulate gradients
  // use_bias = True
  if (bias_grad_ptr != NULL) {
    checkCUDA(hipblasSgemv(m->handle.blas, HIPBLAS_OP_N,
                          out_dim, batch_size,
                          &alpha, output_grad_ptr, out_dim,
                          m->one_ptr, 1,
                          &alpha, bias_grad_ptr, 1));
  }
  // Compute data gradiant
  // NOTE: we use alpha=1 for input_grad to accumulate gradients
  if (input_grad_ptr != NULL) {
    checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_N,
                          in_dim, batch_size, out_dim,
                          &alpha, kernel_ptr, in_dim,
                          output_grad_ptr, out_dim,
                          &alpha, input_grad_ptr, in_dim));
  }
}

void Linear::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return backward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

/*
  regions[0](I): input
  regions[1](I/O): replica_grad or input_grad
  regions[2](I): output
  regions[3](I/O): output_grad
  regions[4](I): filter
  regions[5](I/O): filter_grad
  regions[6](I/O): bias_grad
*/
template<int NDIM>
__host__
void Linear::backward_task_with_dim(const Task *task,
                                    const std::vector<PhysicalRegion> &regions,
                                    Context ctx, Runtime *runtime)
{
  //Linear* linear = (Linear*) task->args;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  assert(regions.size() == (5 + int(m->trainableInputs[0]) + int(m->use_bias)));
  assert(task->regions.size() == (5 + int(m->trainableInputs[0]) + int(m->use_bias)));
  float* input_grad = NULL;
  size_t rid = 0;
  TensorAccessorR<float, NDIM> acc_input(
      regions[rid], task->regions[rid], FID_DATA, ctx, runtime);
  rid++;
  if (m->trainableInputs[0]) {
    Domain domain = runtime->get_index_space_domain(
        ctx, task->regions[rid].region.get_index_space());
    if (domain.get_dim() == NDIM+1) {
      assert(domain.get_volume() == acc_input.rect.volume());
      input_grad = helperGetTensorPointerWO<float>(
          regions[rid], task->regions[rid], FID_DATA, ctx, runtime);
    } else {
      TensorAccessorW<float, NDIM> acc_replica_grad(
          regions[rid], task->regions[rid], FID_DATA, ctx, runtime,
          true/*readOutput*/);
      assert(acc_replica_grad.rect.volume() == acc_input.rect.volume());
      input_grad = acc_replica_grad.ptr;
    }
    rid++;
  }
  TensorAccessorR<float, NDIM> acc_output(
      regions[rid], task->regions[rid], FID_DATA, ctx, runtime);
  rid++;
  TensorAccessorW<float, NDIM> acc_output_grad(
      regions[rid], task->regions[rid], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  rid++;
  TensorAccessorR<float, 2> acc_kernel(
      regions[rid], task->regions[rid], FID_DATA, ctx, runtime);
  rid++;
  TensorAccessorW<float, 2> acc_kernel_grad(
      regions[rid], task->regions[rid], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  rid++;
  // make sure the sizes match
  int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int batch_size = acc_output.rect.volume() / out_dim;
  assert(acc_output.rect.volume() == out_dim * batch_size);
  assert(acc_output_grad.rect.volume() == out_dim * batch_size);
  assert(acc_kernel.rect.volume() == in_dim * out_dim);
  assert(acc_kernel_grad.rect.volume() == in_dim * out_dim);
  float* acc_bias_grad_ptr = NULL;
  if (m->use_bias) {
    TensorAccessorW<float, 1> acc_bias_grad(
        regions[rid], task->regions[rid], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    rid++;
    assert(acc_bias_grad.rect.volume() == out_dim);
    acc_bias_grad_ptr = static_cast<float*>(acc_bias_grad.ptr);
  }
  assert(rid == regions.size());

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  Linear::backward_kernel(m, acc_input.ptr, input_grad,
      acc_output.ptr, acc_output_grad.ptr,
      acc_kernel.ptr, acc_kernel_grad.ptr,
      acc_bias_grad_ptr, in_dim, out_dim, batch_size, stream);
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Linear backward time = %.2lfms\n", elapsed);
    //print_tensor<NDIM, float>(acc_output_grad.ptr, acc_output_grad.rect, "[Linear:backward:output_grad]");
    //print_tensor<2, float>(acc_kernel_grad.ptr, acc_kernel_grad.rect, "[Linear:backward:kernel_grad]");
    //print_tensor<1, float>(acc_bias_grad.ptr, acc_bias_grad.rect, "[Linear:backward:bias_grad]");
    //print_tensor<2, float>(input_grad, acc_input.rect, "[Linear:backward:input_grad]");
  }
}

void Linear::backward2_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return backward2_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}


/*
  regions[0](I/O): input_grad
  regions[1](I): replicas
*/
template<int NDIM>
__host__
void Linear::backward2_task_with_dim(const Task *task,
                                     const std::vector<PhysicalRegion> &regions,
                                     Context ctx, Runtime *runtime)
{
  //const LinearMeta* m = *((LinearMeta**) task->local_args);
  TensorAccessorW<float, NDIM> acc_input_grad(
      regions[0], task->regions[0], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, 3> acc_replica(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  assert(acc_input_grad.rect.hi[0] == acc_replica.rect.hi[0]);
  assert(acc_input_grad.rect.lo[0] == acc_replica.rect.lo[0]);
  assert(acc_input_grad.rect.hi[1] == acc_replica.rect.hi[1]);
  assert(acc_input_grad.rect.lo[1] == acc_replica.rect.lo[1]);

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  int num_replica = acc_replica.rect.hi[NDIM] - acc_replica.rect.lo[NDIM] + 1;
  const float *replica_ptr = acc_replica.ptr;
  for (int i = 0; i < num_replica; i++) {
    size_t num_elements = acc_input_grad.rect.volume();
    apply_add_with_scale<<<GET_BLOCKS(num_elements), CUDA_NUM_THREADS, 0, stream>>>(
        acc_input_grad.ptr, replica_ptr, num_elements, 1.0f);
    replica_ptr += acc_input_grad.rect.volume();
  }
}

void Linear::backward(const FFModel& ff)
{
  int dim = outputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
      return backward_with_dim<DIM>(ff);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

template<int NDIM>
void Linear::backward_with_dim(const FFModel& ff)
{
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  {
    ArgumentMap argmap;
    Rect<NDIM> rect = runtime->get_index_space_domain(ctx, task_is);
    int idx = 0;
    for (PointInRectIterator<NDIM> it(rect); it(); it++) {
      OpMeta* mp = meta[idx++];
      argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
    }
    IndexLauncher launcher(LINEAR_BWD_TASK_ID, task_is,
                           TaskArgument(NULL, 0), argmap,
                           Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                           FFConfig::get_hash_id(std::string(name)));
    int rid = 0;
    // regions[0](I): input
    launcher.add_region_requirement(
        RegionRequirement(input_lps[0], 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, inputs[0].region));
    launcher.add_field(rid++, FID_DATA);
    // regions[1](I/O): replica_grad
    if (trainableInputs[0]) {
      if (replica.region_grad != LogicalRegion::NO_REGION) {
        launcher.add_region_requirement(
            RegionRequirement(replica.part_grad, 0/*projection id*/,
                              WRITE_ONLY, EXCLUSIVE, replica.region_grad));
        launcher.add_field(rid++, FID_DATA);
      } else {
        launcher.add_region_requirement(
            RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                              READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
        launcher.add_field(rid++, FID_DATA);
      }
    }
    // regions[2](I): output
    launcher.add_region_requirement(
        RegionRequirement(outputs[0].part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, outputs[0].region));
    launcher.add_field(rid++, FID_DATA);
    // regions[3](I/O): output_grad
    launcher.add_region_requirement(
        RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, outputs[0].region_grad));
    launcher.add_field(rid++, FID_DATA);
    // regions[4](I): filter
    launcher.add_region_requirement(
        RegionRequirement(weights[0].part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, weights[0].region));
    launcher.add_field(rid++, FID_DATA);
    // regions[5](I/O): filter_grad
    launcher.add_region_requirement(
        RegionRequirement(weights[0].part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, weights[0].region_grad));
    launcher.add_field(rid++, FID_DATA);
    if (use_bias) {
      // regions[6](I/O): bias_grad
      launcher.add_region_requirement(
          RegionRequirement(weights[1].part_grad, 0/*projection id*/,
                            READ_WRITE, EXCLUSIVE, weights[1].region_grad));
      launcher.add_field(rid++, FID_DATA);
    }
    runtime->execute_index_space(ctx, launcher);
  }
  if (replica.region_grad != LogicalRegion::NO_REGION && trainableInputs[0]) {
    // We aggregate parameters from replica tensor to input tensor
    // Note we use input's task_is to reduce extra data transfers
    ArgumentMap argmap;
    Rect<2> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part_grad.get_index_partition());
    IndexSpaceT<2> input_task_is = IndexSpaceT<2>(ff.get_task_is(input_rect));
    // If we are the first layer, our input uses data parallel and does
    // not have an owner
    std::string input_pcname = "";
    if (inputs[0].owner_op != NULL)
      input_pcname = std::string(inputs[0].owner_op->name);
    IndexLauncher launcher(LINEAR_BWD2_TASK_ID, input_task_is,
                           TaskArgument(this, sizeof(Linear)), argmap,
                           Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                           FFConfig::get_hash_id(input_pcname));
    launcher.add_region_requirement(
        RegionRequirement(inputs[0].part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
    launcher.add_field(0, FID_DATA);
    // Note that replica.part save's a partition of replica.region_grad
    launcher.add_region_requirement(
        RegionRequirement(replica.part, 0/*partition id*/,
                          READ_ONLY, EXCLUSIVE, replica.region_grad));
    launcher.add_field(1, FID_DATA);
    runtime->execute_index_space(ctx, launcher);
  }
}

/*
__host__
Parameter* Linear::get_parameter(int index)
{
  if (index == 0) {
    return &weights[0];
  } else if (index == 1){
    return &weights[1];
  } else {
    assert(0);
    return NULL;
  }
}
*/

__host__
void Linear::print_layer(const FFModel& ff)
{
  printf("linear layer\n");
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;

  RegionRequirement kernel_req(weights[0].region, READ_WRITE, EXCLUSIVE, weights[0].region);
  kernel_req.add_field(FID_DATA);
  InlineLauncher kernel_launcher(kernel_req);
  PhysicalRegion kernel_region = runtime->map_region(ctx, kernel_launcher);
  kernel_region.wait_until_valid();

  RegionRequirement bias_req(weights[1].region, READ_WRITE, EXCLUSIVE, weights[1].region);
  bias_req.add_field(FID_DATA);
  InlineLauncher bias_launcher(bias_req);
  PhysicalRegion bias_region = runtime->map_region(ctx, bias_launcher);
  bias_region.wait_until_valid();

  TensorAccessorW<float, 2> acc_kernel(kernel_region, kernel_req, FID_DATA, ctx, runtime, true);
  TensorAccessorW<float, 1> acc_bias(bias_region, bias_req, FID_DATA, ctx, runtime, true);

  const float *kernel_ptr = acc_kernel.ptr;
  const float *bias_ptr = acc_bias.ptr;

  size_t kernel_size = acc_kernel.rect.volume();
  int kernel_dim1 = acc_kernel.rect.hi[0] - acc_kernel.rect.lo[0] + 1;
  int kernel_dim2 = acc_kernel.rect.hi[1] - acc_kernel.rect.lo[1] + 1;
  size_t bias_size = acc_bias.rect.volume();
  printf("kernel, %p, %zu, [%d, %d]\n", kernel_ptr, kernel_size, kernel_dim1, kernel_dim2);
  printf("bias, %p, %zu\n", bias_ptr, bias_size);

  for (int i = 0; i < bias_size; i++) {
    printf("%f ", bias_ptr[i]);
  }
  printf("\n");

  for (int i = 0; i < kernel_size; i++) {
    printf("%f ", kernel_ptr[i]);
  }
  printf("\n");

  runtime->unmap_region(ctx, kernel_region);
  runtime->unmap_region(ctx, bias_region);

}

LinearMeta::LinearMeta(FFHandler handler, int batch_size)
: OpMeta(handler)
{
  // Allocate an all-one's vector
  float* dram_one_ptr = (float *) malloc(sizeof(float) * batch_size);
  for (int i = 0; i < batch_size; i++)
    dram_one_ptr[i] = 1.0f;
  float* fb_one_ptr;
  checkCUDA(hipMalloc(&fb_one_ptr, sizeof(float) * batch_size));
  checkCUDA(hipMemcpy(fb_one_ptr, dram_one_ptr,
                       sizeof(float) * batch_size, hipMemcpyHostToDevice));
  one_ptr = (const float*) fb_one_ptr;
  // Allocate descriptors
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
    checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
}

bool Linear::measure_operator_cost(Simulator* sim,
                                   const ParallelConfig& pc,
                                   CostMetrics& cost_metrics)
{
  Tensor sub_output, sub_input;
  if (!outputs[0].get_output_sub_tensor(pc, sub_output, OP_LINEAR))
    return false;
  if (!inputs[0].get_input_sub_tensor(pc, sub_input, OP_LINEAR))
    return false;
  int input_c = sub_input.adim[0];
  int input_n = sub_input.get_volume() / input_c;
  int output_c = sub_output.adim[0];
  int output_n = sub_output.get_volume() / output_c;
  LinearMeta* m = sim->linear_meta;
  m->activation = activation;
  if (use_cudnn_activation(m->activation)) {
    hipdnnActivationMode_t mode;
    switch (activation) {
      case AC_MODE_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case AC_MODE_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      default:
        // Unsupported activation mode
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
        HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, output_n, output_c, 1, 1));
  }
  // allocate tensors in simulator
  sim->free_all();
  float* input_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert(input_ptr != NULL);
  float *output_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert(output_ptr != NULL);
  float* kernel_ptr = (float*)sim->allocate((size_t)output_c * input_c, DT_FLOAT);
  assert(kernel_ptr != NULL);
  float* bias_ptr = (float*)sim->allocate(output_c, DT_FLOAT);
  assert(bias_ptr != NULL);

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  std::function<void()> forward, backward;
  forward = [&] {
    forward_kernel(m, input_ptr, output_ptr, kernel_ptr, bias_ptr,
        input_c, output_c, input_n, stream);
  };
  if (sim->computationMode == COMP_MODE_TRAINING) {
    float* input_grad_ptr = NULL;
    if (trainableInputs[0]) {
      input_grad_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
    }
    float *output_grad_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
    float* kernel_grad_ptr = (float*)sim->allocate((size_t)output_c * input_c, DT_FLOAT);
    float* bias_grad_ptr = (float*)sim->allocate(output_c, DT_FLOAT);
    assert(bias_grad_ptr != NULL);
    backward = [&] {
      backward_kernel(m, input_ptr, input_grad_ptr, output_ptr, output_grad_ptr,
          kernel_ptr, kernel_grad_ptr, bias_grad_ptr, input_c, output_c, input_n, stream);
    };
  }

  inner_measure_operator_cost(sim, forward, backward, cost_metrics);

  if (sim->computationMode == COMP_MODE_TRAINING) {
    printf("[Measure Linear] name(%s) in(%d %d) out(%d %d) forward_time(%.4lf) backward_time(%.4lf)\n",
           name, input_n, input_c, output_n, output_c,
           cost_metrics.forward_time, cost_metrics.backward_time);
  } else {
    printf("[Measure Linear] name(%s) in(%d %d) out(%d %d) forward_time(%.4lf)\n",
           name, input_n, input_c, output_n, output_c,
           cost_metrics.forward_time);
  }
  return true;
}

ParallelConfig Linear::get_random_parallel_config(const FFModel& ff) const
{
  if (!ff.config.enable_parameter_parallel)
    return Op::get_random_parallel_config(ff);
  std::vector<int> batch_candidates;
  std::vector<int> channel_candidates;
  int batch = outputs[0].adim[outputs[0].numDim-1];
  int channel = outputs[0].adim[0];
  int total_devices = ff.config.workersPerNode * ff.config.numNodes;
  for (int i = 1; i <= ff.config.workersPerNode; i++)
    if (channel % i == 0)
      for (int j = 1; i * j <= total_devices; j++)
        if (batch % j == 0) {
          batch_candidates.push_back(j);
          channel_candidates.push_back(i);
        }
  assert(batch_candidates.size() > 0);
  int idx = std::rand() % batch_candidates.size();
  int num_par_c = channel_candidates[idx];
  int num_par_b = batch_candidates[idx];
  ParallelConfig pc;
  pc.device_type = ParallelConfig::GPU;
  pc.nDims = outputs[0].numDim;
  pc.dim[0] = num_par_c;
  pc.dim[pc.nDims-1] = num_par_b;
  for (int i = 1; i < pc.nDims - 1; i++)
    pc.dim[i] = 1;
  int start_idx = std::rand() % (total_devices - num_par_c * num_par_b + 1);
  start_idx = start_idx - start_idx % num_par_c;
  for (int i = 0; i < num_par_c * num_par_b; i++)
    pc.device_ids[i] = start_idx + i;
  return pc;
}

bool Linear::is_valid_parallel_config(const FFModel& ff, const ParallelConfig& pc) const
{
  if (!ff.config.enable_parameter_parallel)
    return Op::is_valid_parallel_config(ff, pc);
  // Support data and parameter parallel
  if (pc.nDims != outputs[0].numDim)
    return false;
  for (int i = 1; i < pc.nDims-1; i++)
    if (pc.dim[i] != 1)
      return false;
  return true;
}

