#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::dense(const Tensor& input,
                      int outDim,
                      ActiMode activation,
                      bool use_bias,
                      const Op* shared_op,
                      Initializer* kernel_initializer,
                      Initializer* bias_initializer,
                      const char *name)
{
  if (kernel_initializer == NULL) {
    int seed = std::rand();
    kernel_initializer = new GlorotUniform(seed);
  }
  if (bias_initializer == NULL) {
    bias_initializer = new ZeroInitializer();
  }
  Linear *li = new Linear(*this, input, outDim, activation, use_bias,
                          shared_op, kernel_initializer, bias_initializer, name);
  layers.push_back(li);
  return li->outputs[0];
}

Linear* FFModel::dense(int inDim, int outDim,
                       ActiMode activation,
                       bool use_bias,
                       Initializer* kernel_initializer,
                       Initializer* bias_initializer,
                       const char *name)
{
  if (kernel_initializer == NULL) {
    int seed = std::rand();
    kernel_initializer = new GlorotUniform(seed);
  }
  if (bias_initializer == NULL) {
    bias_initializer = new ZeroInitializer();
  }
  Linear *li = new Linear(*this, inDim, outDim, activation, use_bias,
                          kernel_initializer, bias_initializer, name);
  layers.push_back(li);
  return li;
}

Linear::Linear(FFModel& model,
               const Tensor& _input,
               int out_dim,
               ActiMode _activation,
               bool _use_bias,
               const Op* shared_op,
               Initializer* _kernel_initializer,
               Initializer* _bias_initializer,
               const char* name)
: Op(model, OP_LINEAR, shared_op, name, _input),
  in_channels(_input.adim[0]), out_channels(out_dim),
  activation(_activation), use_bias(_use_bias),
  kernel_initializer(_kernel_initializer),
  bias_initializer(_bias_initializer),
  profiling(model.config.profiling)
{
  numInputs = 1;
  numOutputs = 1;
  outputs[0].numDim = _input.numDim;
  for (int i = 1; i < outputs[0].numDim; i++)
    outputs[0].adim[i] = _input.adim[i];
  outputs[0].adim[0] = out_dim;
  weights[0].numDim = 2;
  weights[0].adim[0] = in_channels;
  weights[0].adim[1] = out_channels;
  numWeights = 1;
  if (use_bias) {
    weights[1].numDim = 1;
    weights[1].adim[0] = out_channels;
    numWeights = 2;
  }
}

Linear::Linear(FFModel& model,
               int in_dim, int out_dim,
               ActiMode _activation,
               bool _use_bias,
               Initializer* _kernel_initializer,
               Initializer* _bias_initializer,
               const char* name)
: Op(model, OP_LINEAR, name, 1),
  in_channels(in_dim), out_channels(out_dim),
  activation(_activation), use_bias(_use_bias),
  kernel_initializer(_kernel_initializer),
  bias_initializer(_bias_initializer),
  profiling(model.config.profiling)
{
}

Tensor Linear::init_inout(FFModel& model, const Tensor& _input)
{
  assert(_input.adim[0] == in_channels);
  inputs[0] = _input;
  create_output_and_partition(model);
  return outputs[0];
}

/*
void Linear::add_to_model(FFModel& model)
{
  model.layers.push_back(this);
  model.parameters.push_back(weights[0]);
  if (numWeights > 1) { // bias is used
    assert(numWeights == 2);
    model.parameters.push_back(weights[1]);
  }
}
*/


void Linear::create_weights(FFModel& model)
{
  int dim = inputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      create_weights_with_dim<DIM>(model); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
    {
      // Unsupported dim
      assert(false);
    }
  }
}

template<int NDIM>
void Linear::create_weights_with_dim(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = IndexSpaceT<NDIM>(model.get_or_create_task_is(NDIM, pcname));

#ifdef FF_ENABLE_NCCL
  Parameter::CommType comm_type = Parameter::NCCL;
#else
  Parameter::CommType comm_type = Parameter::PS;
#endif

  // Create kernel tensor
  {
    const int dims[2] = {out_channels, in_channels};
    weights[0] = model.create_linear_weight<2, NDIM>(this, dims, DT_FLOAT,
        kernel_initializer, true/*create_grad*/, comm_type);
  }
  // Create bias tensor
  if (use_bias) {
    const int dims[1] = {out_channels};
    weights[1] = model.create_linear_weight<1, NDIM>(this, dims, DT_FLOAT,
        bias_initializer, true/*create_grad*/, comm_type);
    assert(numWeights == 2);
  } else {
    assert(numWeights == 1);
  }
}

void Linear::create_output_and_partition(FFModel& model)
{
  int dim = inputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      create_output_and_partition_with_dim<DIM>(model); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
    {
      // Unsupported dim for ElementWiseBinary operator
      assert(false);
    }
  }
}

template<int NDIM>
void Linear::create_output_and_partition_with_dim(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = IndexSpaceT<NDIM>(model.get_or_create_task_is(NDIM, pcname));

  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<NDIM> part_rect = runtime->get_index_space_domain(ctx, task_is);
  int num_par_c = part_rect.hi[0] - part_rect.lo[0] + 1;
  int num_par_n = part_rect.hi[NDIM-1] - part_rect.lo[NDIM-1] + 1;
  int in_dim = inputs[0].adim[0];
  assert(in_dim == in_channels);
  int batch_size = inputs[0].adim[NDIM-1];
  {
    int dims[NDIM];
    for (int i = 0; i < NDIM; i++)
      dims[i] = outputs[0].adim[NDIM-1-i];
    outputs[0] = model.create_tensor<NDIM>(dims, DT_FLOAT, this);
    outputs[0].owner_op = this;
    outputs[0].owner_idx = 0;
  }
  // Compute partition bound for input
  Rect<NDIM> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part.get_index_partition());
  // Create replica tensor
  if (num_par_c > 1) {
    const int dims[3] = {num_par_c, batch_size, in_dim};
    replica = model.create_linear_replica<3>(dims, (IndexSpaceT<NDIM>)task_is, DT_FLOAT);
    {
      Rect<NDIM> extent;
      for (int i = 0; i < NDIM; i++) {
        extent.lo[i] = 0;
        assert(outputs[0].adim[i] % (part_rect.hi[i] - part_rect.lo[i] + 1) == 0);
        extent.hi[i] = outputs[0].adim[i] / (part_rect.hi[i] - part_rect.lo[i] + 1) - 1;
      }
      Transform<NDIM, NDIM> transform;
      for (int i = 0; i < NDIM; i++)
        for (int j = 0; j < NDIM; j++)
          transform[i][j] = 0;
      for (int i = 1; i < NDIM; i++)
        transform[i][i] = extent.hi[i] + 1;
      IndexPartition ip = runtime->create_partition_by_restriction(
          ctx, inputs[0].region.get_index_space(), task_is, transform, extent);
      input_lps[0] = runtime->get_logical_partition(
          ctx, inputs[0].region, ip);
    }
    // Backward use the same ip as inputs[0]
    input_grad_lps[0] = inputs[0].part_grad;
    {
      IndexSpaceT<NDIM> input_task_is = IndexSpaceT<NDIM>(model.get_or_create_task_is(input_rect));
      Rect<NDIM+1> extent;
      for (int i = 0; i < NDIM; i++) {
        extent.lo[i] = 0;
        assert(inputs[0].adim[i] % (input_rect.hi[i] - input_rect.lo[i] + 1) == 0);
        extent.hi[i] = inputs[0].adim[i] / (input_rect.hi[i] - input_rect.lo[i] + 1) - 1;
      }
      extent.lo[NDIM] = 0;
      extent.hi[NDIM] = num_par_c - 1;
      Transform<NDIM+1, NDIM> transform;
      for (int i = 0; i < NDIM+1; i++)
        for (int j = 0; j < NDIM; j++)
          transform[i][j] = 0;
      for (int i = 0; i < NDIM; i++)
        transform[i][i] = inputs[0].adim[i] / (input_rect.hi[i] - input_rect.lo[i] + 1);
      IndexPartition ip = runtime->create_partition_by_restriction(
          ctx, replica.region_grad.get_index_space(), input_task_is,
          transform, extent);
      assert(runtime->is_index_partition_disjoint(ctx, ip));
      assert(runtime->is_index_partition_complete(ctx, ip));
      // Note we use replica.part to save how to partition the replica
      // to compute input_grad_lps
      replica.part = runtime->get_logical_partition(
          ctx, replica.region_grad, ip);
    }
  } else {
    if (input_rect == part_rect) {
      input_lps[0] = inputs[0].part;
      input_grad_lps[0] = inputs[0].part_grad;
    } else {
      Rect<NDIM> extent;
      for (int i = 0; i < NDIM; i++) {
        extent.lo[i] = 0;
        assert(inputs[0].adim[i] % (part_rect.hi[i] - part_rect.lo[i] + 1) == 0);
        extent.hi[i] = inputs[0].adim[i] / (part_rect.hi[i] - part_rect.lo[i] + 1) - 1;
      }
      Transform<NDIM, NDIM> transform;
      for (int i = 0; i < NDIM; i++)
        for (int j = 0; j < NDIM; j++) {
          transform[i][j] = 0;
          if (i==j)
            transform[i][j] = extent.hi[i] + 1;
        }
      IndexPartition ip = runtime->create_partition_by_restriction(
          ctx, inputs[0].region.get_index_space(), task_is, transform, extent);
      assert(runtime->is_index_partition_disjoint(ctx, ip));
      assert(runtime->is_index_partition_complete(ctx, ip));
      input_lps[0] = runtime->get_logical_partition(
          ctx, inputs[0].region, ip);
      input_grad_lps[0] = runtime->get_logical_partition(
          ctx, inputs[0].region_grad, ip);
    }
  }
}

/*
  regions[0](O): output
  regions[1](I): kernel
  regions[2](I): bias
*/
OpMeta* Linear::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime* runtime)
{
  Domain out_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (out_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return init_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  return NULL;
}

template<int NDIM>
OpMeta* Linear::init_task_with_dim(const Task *task,
                                   const std::vector<PhysicalRegion> &regions,
                                   Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  const Linear* linear = (Linear*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  //TensorAccessorR<float, 2> acc_input(
  //    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, NDIM> acc_output(
      regions[0], task->regions[0], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorR<float, 2> acc_kernel(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 1> acc_bias(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  //int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int in_dim = acc_kernel.rect.hi[0] - acc_kernel.rect.lo[0] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int batch_size = acc_output.rect.volume() / out_dim;
  //printf("init linear (input): in_dim(%d) out_dim(%d) batch_size(%d)\n",
  //    in_dim, out_dim, batch_size);
  LinearMeta* m = new LinearMeta(handle, batch_size);
  m->activation = linear->activation;

  if (m->activation != AC_MODE_NONE) {
    hipdnnActivationMode_t mode;
    switch (linear->activation) {
      case AC_MODE_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case AC_MODE_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      default:
        // Unsupported activation mode
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          batch_size, out_dim, 1, 1));
  }
#ifdef FF_ENABLE_NCCL
  m->init_nccl_communicator(task, linear->ncclId);
#endif
  return m;
}

void Linear::init(const FFModel& ff)
{
  int dim = outputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
      return init_with_dim<DIM>(ff);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

template<int NDIM>
void Linear::init_with_dim(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<NDIM> rect = runtime->get_index_space_domain(ctx, task_is);
  ParallelConfig pc;
  std::string pcname = name;
  ff.config.find_parallel_config(NDIM, pcname, pc);
  int idx = 0;
  for (PointInRectIterator<NDIM> it(rect); it(); it++) {
    FFHandler handle = ff.handlers[pc.device_ids[idx++]];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
  }
  IndexLauncher launcher(LINEAR_INIT_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Linear)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  //launcher.add_region_requirement(
  //    RegionRequirement(input_lps[0], 0/*projection id*/,
  //                      READ_ONLY, EXCLUSIVE, inputs[0].region));
  //launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[0].region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[1].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[1].region));
  launcher.add_field(2, FID_DATA);
  // Add inputs[0].region_grad to avoid Legion warning
  launcher.add_region_requirement(
      RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(3, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<NDIM> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/*static*/
void Linear::forward_kernel(const LinearMeta* m,
                            const float* input_ptr,
                            float* output_ptr,
                            const float* kernel_ptr,
                            const float* bias_ptr,
                            int in_dim, int out_dim, int batch_size)
{
  float alpha = 1.0f, beta = 0.0f;
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        out_dim, batch_size, in_dim,
                        &alpha, kernel_ptr, in_dim,
                        input_ptr, in_dim, &beta,
                        output_ptr, out_dim));
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        out_dim, batch_size, 1,
                        &alpha, bias_ptr, 1,
                        m->one_ptr, 1, &alpha,
                        output_ptr, out_dim));
  if (m->activation != AC_MODE_NONE) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
        &alpha, m->outputTensor, output_ptr,
        &beta, m->outputTensor, output_ptr));
  }
}

__host__
void Linear::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return forward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

/*
  regions[0](I); input
  regions[1](O): output
  regions[2](I): kernel
  regions[3](I): bias
*/
template<int NDIM>
void Linear::forward_task_with_dim(const Task *task,
                                   const std::vector<PhysicalRegion> &regions,
                                   Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  Linear* linear = (Linear*) task->args;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  TensorAccessorR<float, NDIM> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, NDIM> acc_output(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorR<float, 2> acc_kernel(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 1> acc_bias(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
  int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int batch_size = acc_output.rect.volume() / out_dim;
  assert(acc_output.rect.volume() == out_dim * batch_size);
  assert(acc_input.rect.volume() == in_dim * batch_size);
  assert(acc_kernel.rect.volume() == in_dim * out_dim);
  assert(acc_bias.rect.volume() == out_dim);

  hipEvent_t t_start, t_end;
  if (linear->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  linear->forward_kernel(m, acc_input.ptr, acc_output.ptr,
      acc_kernel.ptr, acc_bias.ptr, in_dim, out_dim, batch_size);

  if (linear->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Linear] forward time = %.2lfms\n", linear->name, elapsed);
    //print_tensor<NDIM, float>(acc_input.ptr, acc_input.rect, "[Linear:forward:input]");
    //print_tensor<2, float>(acc_kernel.ptr, acc_kernel.rect, "[Linear:forward:kernel]");
    //print_tensor<1, float>(acc_bias.ptr, acc_bias.rect, "[Linear:forward:bias]");
    //print_tensor<NDIM, float>(acc_output.ptr, acc_output.rect, "[Linear:forward:output]");
  }
}

void Linear::forward(const FFModel& ff)
{
  int dim = outputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
      return forward_with_dim<DIM>(ff);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

template<int NDIM>
void Linear::forward_with_dim(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<NDIM> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<NDIM> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(LINEAR_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Linear)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[0].region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[1].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[1].region));
  launcher.add_field(3, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

__global__
void sigmoid_backward(float *grad_ptr, const float *output, int n)
{
  CUDA_KERNEL_LOOP(i, n)
  {
    grad_ptr[i] = grad_ptr[i] * output[i] * (1 - output[i]);
  }
}

/*static*/
void Linear::backward_kernel(const LinearMeta* m,
                             const float* input_ptr,
                             float* input_grad_ptr,
                             const float* output_ptr,
                             float* output_grad_ptr,
                             const float* kernel_ptr,
                             float* kernel_grad_ptr,
                             float* bias_grad_ptr,
                             int in_dim, int out_dim, int batch_size)
{
  float alpha = 1.0f;
  int output_size = out_dim * batch_size;
  if (m->activation == AC_MODE_RELU) {
    reluBackward<<<GET_BLOCKS(output_size), CUDA_NUM_THREADS>>>(
        output_grad_ptr, output_ptr, output_size);
  } else if (m->activation == AC_MODE_SIGMOID) {
    sigmoid_backward<<<GET_BLOCKS(output_size), CUDA_NUM_THREADS>>>(
        output_grad_ptr, output_ptr, output_size);
  } else {
    // TODO: only support relu and sigmoid for now
    assert(m->activation == AC_MODE_NONE);
  }
  // Compute weight gradiant
  // NOTE: we use alpha=1 for kernel_grad to accumulate gradients
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        in_dim, out_dim, batch_size,
                        &alpha, input_ptr, in_dim,
                        output_grad_ptr, out_dim,
                        &alpha, kernel_grad_ptr, in_dim));
  // Compute bias gradiant
  // NOTE: we use alpha=1 for bias_grad to accumulate gradients
  checkCUDA(hipblasSgemv(m->handle.blas, HIPBLAS_OP_N,
                        out_dim, batch_size,
                        &alpha, output_grad_ptr, out_dim,
                        m->one_ptr, 1,
                        &alpha, bias_grad_ptr, 1));
  // Compute data gradiant
  // NOTE: we use alpha=1 for input_grad to accumulate gradients
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_N,
                        in_dim, batch_size, out_dim,
                        &alpha, kernel_ptr, in_dim,
                        output_grad_ptr, out_dim,
                        &alpha, input_grad_ptr, in_dim));
}

void Linear::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return backward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

/*
  regions[0](I): input
  regions[1](I/O): replica_grad or input_grad
  regions[2](I): output
  regions[3](I/O): output_grad
  regions[4](I): filter
  regions[5](I/O): filter_grad
  regions[6](I/O): bias_grad
*/
template<int NDIM>
__host__
void Linear::backward_task_with_dim(const Task *task,
                                    const std::vector<PhysicalRegion> &regions,
                                    Context ctx, Runtime *runtime)
{
  assert(regions.size() == 7);
  assert(task->regions.size() == 7);
  Linear* linear = (Linear*) task->args;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  float* input_grad = NULL;
  TensorAccessorR<float, NDIM> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorR<float, NDIM> acc_output(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int batch_size = acc_output.rect.volume() / out_dim;
  Domain domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
  if (domain.get_dim() == NDIM+1) {
    assert(false);
    TensorAccessorW<float, NDIM+1> acc_replica_grad(
        regions[1], task->regions[1], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    assert(acc_replica_grad.rect.volume() == in_dim * batch_size);
    input_grad = acc_replica_grad.ptr;
  } else {
    TensorAccessorW<float, NDIM> acc_replica_grad(
        regions[1], task->regions[1], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    assert(acc_replica_grad.rect.volume() == in_dim * batch_size);
    input_grad = acc_replica_grad.ptr;
  }
  TensorAccessorW<float, NDIM> acc_output_grad(
      regions[3], task->regions[3], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, 2> acc_kernel(
      regions[4], task->regions[4], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> acc_kernel_grad(
      regions[5], task->regions[5], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorW<float, 1> acc_bias_grad(
      regions[6], task->regions[6], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  // make sure the sizes match
  assert(acc_output.rect.volume() == out_dim * batch_size);
  assert(acc_output_grad.rect.volume() == out_dim * batch_size);
  assert(acc_kernel.rect.volume() == in_dim * out_dim);
  assert(acc_kernel_grad.rect.volume() == in_dim * out_dim);
  assert(acc_bias_grad.rect.volume() == out_dim);
  hipEvent_t t_start, t_end;
  if (linear->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  linear->backward_kernel(m, acc_input.ptr, input_grad,
      acc_output.ptr, acc_output_grad.ptr,
      acc_kernel.ptr, acc_kernel_grad.ptr,
      acc_bias_grad.ptr, in_dim, out_dim, batch_size);
  if (linear->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Linear backward time = %.2lfms\n", elapsed);
    //print_tensor<NDIM, float>(acc_output_grad.ptr, acc_output_grad.rect, "[Linear:backward:output_grad]");
    //print_tensor<2, float>(acc_kernel_grad.ptr, acc_kernel_grad.rect, "[Linear:backward:kernel_grad]");
    //print_tensor<1, float>(acc_bias_grad.ptr, acc_bias_grad.rect, "[Linear:backward:bias_grad]");
    //print_tensor<2, float>(input_grad, acc_input.rect, "[Linear:backward:input_grad]");
  }
}

void Linear::backward2_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return backward2_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}


/*
  regions[0](I/O): input_grad
  regions[1](I): replicas
*/
template<int NDIM>
__host__
void Linear::backward2_task_with_dim(const Task *task,
                                     const std::vector<PhysicalRegion> &regions,
                                     Context ctx, Runtime *runtime)
{
  float alpha = 1.0f;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  TensorAccessorW<float, NDIM> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, 3> acc_replica(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  assert(acc_input.rect.hi[0] == acc_replica.rect.hi[0]);
  assert(acc_input.rect.lo[0] == acc_replica.rect.lo[0]);
  assert(acc_input.rect.hi[1] == acc_replica.rect.hi[1]);
  assert(acc_input.rect.lo[1] == acc_replica.rect.lo[1]);
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  int num_replica = acc_replica.rect.hi[NDIM] - acc_replica.rect.lo[NDIM] + 1;
  const float *replica_ptr = acc_replica.ptr;
  for (int i = 1; i < num_replica; i++) {
    checkCUDA(hipblasSaxpy(m->handle.blas, acc_input.rect.volume(),
                          &alpha, replica_ptr, 1, acc_input.ptr, 1));
    replica_ptr += acc_input.rect.volume();
  }
}

void Linear::backward(const FFModel& ff)
{
  int dim = outputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
      return backward_with_dim<DIM>(ff);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

template<int NDIM>
void Linear::backward_with_dim(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<NDIM> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<NDIM> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  {
    IndexLauncher launcher(LINEAR_BWD_TASK_ID, task_is,
                           TaskArgument(this, sizeof(Linear)), argmap,
                           Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                           FFConfig::get_hash_id(std::string(name)));
    // regions[0](I): input
    launcher.add_region_requirement(
        RegionRequirement(input_lps[0], 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, inputs[0].region));
    launcher.add_field(0, FID_DATA);
    // regions[1](I/O): replica_grad
    if (replica.region_grad != LogicalRegion::NO_REGION) {
      launcher.add_region_requirement(
          RegionRequirement(replica.part_grad, 0/*projection id*/,
                            READ_WRITE, EXCLUSIVE, replica.region_grad));
      launcher.add_field(1, FID_DATA);
    } else {
      launcher.add_region_requirement(
          RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                            READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
      launcher.add_field(1, FID_DATA);
    }
    // regions[2](I): output
    launcher.add_region_requirement(
        RegionRequirement(outputs[0].part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, outputs[0].region));
    launcher.add_field(2, FID_DATA);
    // regions[3](I/O): output_grad
    launcher.add_region_requirement(
        RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, outputs[0].region_grad));
    launcher.add_field(3, FID_DATA);
    // regions[4](I): filter
    launcher.add_region_requirement(
        RegionRequirement(weights[0].part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, weights[0].region));
    launcher.add_field(4, FID_DATA);
    // regions[5](I/O): filter_grad
    launcher.add_region_requirement(
        RegionRequirement(weights[0].part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, weights[0].region_grad));
    launcher.add_field(5, FID_DATA);
    // regions[6](I/O): bias_grad
    launcher.add_region_requirement(
        RegionRequirement(weights[1].part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, weights[1].region_grad));
    launcher.add_field(6, FID_DATA);
    runtime->execute_index_space(ctx, launcher);
  }
  if (replica.region_grad != LogicalRegion::NO_REGION) {
    // We aggregate parameters from replica tensor to input tensor
    // Note we use input's task_is to reduce extra data transfers
    Rect<2> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part_grad.get_index_partition());
    IndexSpaceT<2> input_task_is = IndexSpaceT<2>(ff.get_task_is(input_rect));
    IndexLauncher launcher(LINEAR_BWD2_TASK_ID, task_is,
                           TaskArgument(this, sizeof(Linear)), argmap,
                           Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                           FFConfig::get_hash_id(std::string(name)));
    launcher.add_region_requirement(
        RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
    launcher.add_field(0, FID_DATA);
    // Note that replica.part save's a partition of replica.region_grad
    launcher.add_region_requirement(
        RegionRequirement(replica.part, 0/*partition id*/,
                          READ_ONLY, EXCLUSIVE, replica.region_grad));
    launcher.add_field(1, FID_DATA);
    runtime->execute_index_space(ctx, launcher);
  }
}

/*
__host__
Parameter* Linear::get_parameter(int index)
{
  if (index == 0) {
    return &weights[0];
  } else if (index == 1){
    return &weights[1];
  } else {
    assert(0);
    return NULL;
  }
}
*/

__host__
void Linear::print_layer(const FFModel& ff)
{
  printf("linear layer\n");
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;

  RegionRequirement kernel_req(weights[0].region, READ_WRITE, EXCLUSIVE, weights[0].region);
  kernel_req.add_field(FID_DATA);
  InlineLauncher kernel_launcher(kernel_req);
  PhysicalRegion kernel_region = runtime->map_region(ctx, kernel_launcher);
  kernel_region.wait_until_valid();

  RegionRequirement bias_req(weights[1].region, READ_WRITE, EXCLUSIVE, weights[1].region);
  bias_req.add_field(FID_DATA);
  InlineLauncher bias_launcher(bias_req);
  PhysicalRegion bias_region = runtime->map_region(ctx, bias_launcher);
  bias_region.wait_until_valid();

  TensorAccessorW<float, 2> acc_kernel(kernel_region, kernel_req, FID_DATA, ctx, runtime, true);
  TensorAccessorW<float, 1> acc_bias(bias_region, bias_req, FID_DATA, ctx, runtime, true);

  const float *kernel_ptr = acc_kernel.ptr;
  const float *bias_ptr = acc_bias.ptr;

  size_t kernel_size = acc_kernel.rect.volume();
  int kernel_dim1 = acc_kernel.rect.hi[0] - acc_kernel.rect.lo[0] + 1;
  int kernel_dim2 = acc_kernel.rect.hi[1] - acc_kernel.rect.lo[1] + 1;
  size_t bias_size = acc_bias.rect.volume();
  printf("kernel, %p, %d, [%d, %d]\n", kernel_ptr, kernel_size, kernel_dim1, kernel_dim2);
  printf("bias, %p, %d\n", bias_ptr, bias_size);

  for (int i = 0; i < bias_size; i++) {
    printf("%f ", bias_ptr[i]);
  }
  printf("\n");

  for (int i = 0; i < kernel_size; i++) {
    printf("%f ", kernel_ptr[i]);
  }
  printf("\n");

  runtime->unmap_region(ctx, kernel_region);
  runtime->unmap_region(ctx, bias_region);

}

LinearMeta::LinearMeta(FFHandler handler, int batch_size)
: OpMeta(handler)
{
  // Allocate an all-one's vector
  float* dram_one_ptr = (float *) malloc(sizeof(float) * batch_size);
  for (int i = 0; i < batch_size; i++)
    dram_one_ptr[i] = 1.0f;
  float* fb_one_ptr;
  checkCUDA(hipMalloc(&fb_one_ptr, sizeof(float) * batch_size));
  checkCUDA(hipMemcpy(fb_one_ptr, dram_one_ptr,
                       sizeof(float) * batch_size, hipMemcpyHostToDevice));
  one_ptr = (const float*) fb_one_ptr;
  // Allocate descriptors
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
    checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
}

bool Linear::measure_compute_time(Simulator* sim,
                                  const ParallelConfig& pc,
                                  float& forward_time,
                                  float& backward_time)
{
  Tensor sub_output, sub_input;
  if (!outputs[0].get_output_sub_tensor(pc, sub_output, OP_LINEAR))
    return false;
  if (!inputs[0].get_input_sub_tensor(pc, sub_input, OP_LINEAR))
    return false;
  int input_c = sub_input.adim[0];
  int input_n = sub_input.get_volume() / input_c;
  int output_c = sub_output.adim[0];
  int output_n = sub_output.get_volume() / output_c;
  LinearMeta* m = sim->linear_meta;
  m->activation = activation;
  if (activation != AC_MODE_NONE) {
    hipdnnActivationMode_t mode;
    switch (activation) {
      case AC_MODE_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case AC_MODE_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      default:
        // Unsupported activation mode
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
        HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, output_n, output_c, 1, 1));
  }
  // allocate tensors in simulator
  sim->free_all();
  float* input_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert(input_ptr != NULL);
  float *output_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert(output_ptr != NULL);
  float* kernel_ptr = (float*)sim->allocate((size_t)output_c * input_c, DT_FLOAT);
  assert(kernel_ptr != NULL);
  float* bias_ptr = (float*)sim->allocate(output_c, DT_FLOAT);
  assert(bias_ptr != NULL);

  auto forward = [&] {
    forward_kernel(m, input_ptr, output_ptr, kernel_ptr, bias_ptr,
        input_c, output_c, input_n);
  };
  auto backward = [&] {
    backward_kernel(m, input_ptr, input_ptr, output_ptr, output_ptr,
        kernel_ptr, kernel_ptr, bias_ptr, input_c, output_c, input_n);
  };

  inner_measure_compute_time(sim, forward, backward, forward_time, backward_time);

  printf("[Measure Linear] name(%s) in(%d %d) out(%d %d) forward_time(%.4lf) backward_time(%.4lf)\n",
         name, input_n, input_c, output_n, output_c, forward_time, backward_time);
  return true;
}

ParallelConfig Linear::get_random_parallel_config(const FFModel& ff) const
{
  if (!ff.config.enable_parameter_parallel)
    return Op::get_random_parallel_config(ff);
  std::vector<int> batch_candidates;
  std::vector<int> channel_candidates;
  int batch = outputs[0].adim[outputs[0].numDim-1];
  int channel = outputs[0].adim[0];
  int total_devices = ff.config.workersPerNode * ff.config.numNodes;
  for (int i = 1; i <= ff.config.workersPerNode; i++)
    if (channel % i == 0)
      for (int j = 1; i * j <= total_devices; j++)
        if (batch % j == 0) {
          batch_candidates.push_back(j);
          channel_candidates.push_back(i);
        }
  assert(batch_candidates.size() > 0);
  int idx = std::rand() % batch_candidates.size();
  int num_par_c = channel_candidates[idx];
  int num_par_b = batch_candidates[idx];
  ParallelConfig pc;
  pc.device_type = ParallelConfig::GPU;
  pc.nDims = outputs[0].numDim;
  pc.dim[0] = num_par_c;
  pc.dim[pc.nDims-1] = num_par_b;
  for (int i = 1; i < pc.nDims - 1; i++)
    pc.dim[i] = 1;
  int start_idx = std::rand() % (total_devices - num_par_c * num_par_b + 1);
  start_idx = start_idx - start_idx % num_par_c;
  for (int i = 0; i < num_par_c * num_par_b; i++)
    pc.device_ids[i] = start_idx + i;
  return pc;
}

