#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford, NVIDIA, Facebook
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ops/linear.h"
#include "cuda_helper.h"

using namespace Legion;

Tensor FFModel::dense(const Tensor input,
                      int outDim,
                      ActiMode activation,
                      bool use_bias,
                      const Op* shared_op,
                      Initializer* kernel_initializer,
                      Initializer* bias_initializer,
                      const char *name)
{
  Linear* li = new Linear(*this, input, outDim, activation, use_bias, false, name);
  layers.push_back(li);
  return li->outputs[0];
}

Node FFModel::get_or_create_linear_node(const Tensor input,
                                        int out_dim,
                                        ActiMode activation,
                                        bool use_bias)
{
  // replica degree cannot be larger than workersPerNode
  //if (input->dims[input->num_dims-1].degree > config.workersPerNode)
  //  return Node::INVALID_NODE;
  // out_dim must be divisble by replicate_degree
  if (out_dim % input->dims[input->num_dims-1].degree != 0)
    return Node::INVALID_NODE;
  size_t hash = input->get_owner_independent_hash();
  hash = hash * 31 + std::hash<int>()(out_dim);
  hash = hash * 31 + std::hash<int>()(activation);
  hash = hash * 31 + std::hash<int>()(use_bias);
  const auto& it = cached_linear_ops.find(hash);
  Linear* li = NULL;
  if (it != cached_linear_ops.end()) {
    li = it->second;
  } else {
    li = new Linear(*this, input, out_dim, activation, use_bias, false/*allocate_weights*/, NULL);
    cached_linear_ops[hash] = li;
  }

  return this->new_node(li);
}

int Linear::output_replica_dim() const {
  return this->inputs[0]->num_dims - 1;
}

int Linear::output_channel_dim() const {
  return 0;
}

int Linear::input_replica_dim() const {
  return this->inputs[0]->num_dims - 1;
}

int Linear::input_channel_dim() const {
  return 0;
}

namespace Kernel {
  constexpr int INDEX = 0;

  enum {
    CHANNEL_IN = 0,
    CHANNEL_OUT = 1,
  };
};

namespace Bias {
  constexpr int INDEX = 1;

  enum {
    CHANNEL_OUT = 0  
  };
};

int Linear::output_size(ParallelDim output_dims[MAX_TENSOR_DIM]) const {
  Tensor const &input = this->inputs[0];

  const int REPLICA = this->output_replica_dim();
  const int CHANNEL = this->output_channel_dim();

  output_dims[REPLICA].is_replica_dim = true;
  for (int i = 1; i < input->num_dims - 1; i++) {
    output_dims[i].size = input->dims[i].size;
  }
  output_dims[CHANNEL].size = this->out_channels;

  return input->num_dims;
}

int Linear::kernel_size(ParallelDim kernel_dims[MAX_TENSOR_DIM]) const {
  Tensor const &input = this->inputs[0];

  kernel_dims[Kernel::CHANNEL_IN].size = this->in_channels;
  kernel_dims[Kernel::CHANNEL_OUT].size = this->out_channels;
  for (int i = 2; i < input->num_dims; i++) {
    kernel_dims[i].is_replica_dim = true;
  }

  return input->num_dims;
}

int Linear::bias_size(ParallelDim bias_dims[MAX_TENSOR_DIM]) const {
  Tensor const &input = this->inputs[0];

  bias_dims[Bias::CHANNEL_OUT].size = this->out_channels;
  for (int i = 1; i < input->num_dims; i++) {
    bias_dims[i].is_replica_dim = true;
  }

  return input->num_dims;
}

void Linear::register_mappings() {
  this->register_output_mappings();
  this->register_weight_mappings();
}

void Linear::register_output_mappings() {
  this->register_output_parallel_dims({
      { this->input_channel_dim(), this->output_replica_dim() },
      { this->input_replica_dim(), this->output_channel_dim() }
  });

  for (int i = 1; i < this->inputs[0]->num_dims - 1; i++) {
    this->register_output_parallel_dims(i, i);
  }
}

void Linear::register_weight_mappings() {
  const int INPUT_IDX = 0;

  this->register_weight_parallel_dims({
      { this->input_channel_dim(), Kernel::CHANNEL_IN },
      { this->input_replica_dim(), Kernel::CHANNEL_OUT },
  }, INPUT_IDX, Kernel::INDEX);

  for (int i = 1; i < this->inputs[0]->num_dims - 1; i++) {
    this->register_weight_parallel_dims(i, i+1, INPUT_IDX, Kernel::INDEX);
  }

  if (this->use_bias) {
    this->register_weight_parallel_dims(
      this->input_replica_dim(), Bias::CHANNEL_OUT,
      INPUT_IDX, Bias::INDEX);
    for (int i = 0; i < this->inputs[0]->num_dims - 1; i++) {
      this->register_weight_parallel_dims(i, i+1, INPUT_IDX, Bias::INDEX);
    }
  }
}

Linear::Linear(FFModel& model,
               Linear const &other, 
               const Tensor input,
               bool allocate_weights)
: Linear(model, input, other.out_channels, other.activation, other.use_bias, allocate_weights, other.name)
{ }

Linear::Linear(FFModel& model,
               const Tensor _input,
               int out_dim,
               ActiMode _activation,
               bool _use_bias,
               bool allocate_weights,
               const char* name)
: Op(
    model, 
    OP_LINEAR, 
    name, 
    1/*inputs*/, 
    _use_bias ? 2 : 1 /*weights*/, 
    allocate_weights,
    1/*outputs*/, 
    _input),
  in_channels(_input->dims[0].size),
  out_channels(out_dim),
  activation(_activation),
  use_bias(_use_bias)
{
  this->register_mappings();

  std::vector<ParallelDim *> weight_dim_sets;

  int kernel_ndim, bias_ndim;
  ParallelDim kernel_dims[MAX_TENSOR_DIM], 
              bias_dims[MAX_TENSOR_DIM];
  if (allocate_weights) {
    kernel_ndim = this->kernel_size(kernel_dims);
    weight_dim_sets.push_back(kernel_dims);

    if (use_bias) {
      bias_ndim = this->bias_size(bias_dims);
      weight_dim_sets.push_back(bias_dims);
    }
  }

  ParallelDim output_dims[MAX_TENSOR_DIM];
  int output_ndim = this->output_size(output_dims);

  this->solve_parallel_dim_mappings(
      { _input->dims },
      weight_dim_sets,
      { output_dims }
  );

  if (allocate_weights) {
    Initializer *kernel_initializer = new GlorotUniform(std::rand()/*seed*/);

    weights[Kernel::INDEX] = model.create_weight_legion_ordering(
        kernel_ndim, kernel_dims, DT_FLOAT, NULL/*owner_op*/, true/*create_grad*/, kernel_initializer, CHOSEN_SYNC_TYPE);

    if (use_bias) {
      Initializer *bias_initializer = new ZeroInitializer();

      weights[Bias::INDEX] = model.create_weight_legion_ordering(
          bias_ndim, bias_dims, DT_FLOAT, NULL/*owner_op*/, true/*create_grad*/, bias_initializer, CHOSEN_SYNC_TYPE);
    }
  }

  // Create the output tensor
  outputs[0] = model.create_tensor_legion_ordering(output_ndim, output_dims, DT_FLOAT, this);

  assert(check_output_input_weight_parallel_dims(allocate_weights));
}

/*
  regions[0](O): output
  regions[1](I): kernel
  regions[2](I): bias
*/
OpMeta* Linear::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime* runtime)
{
  Domain out_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (out_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return init_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  return NULL;
}

bool Linear::use_cudnn_activation(ActiMode mode)
{
  switch (mode) {
    case AC_MODE_RELU:
    case AC_MODE_SIGMOID:
    case AC_MODE_TANH:
      return true;
  }
  return false;
}

template<int NDIM>
OpMeta* Linear::init_task_with_dim(const Task *task,
                                   const std::vector<PhysicalRegion> &regions,
                                   Context ctx, Runtime *runtime)
{
  assert(regions.size() == task->regions.size());
  assert(regions.size() == 2 || regions.size() == 3);
  const Linear* linear = (Linear*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  //TensorAccessorR<float, 2> acc_input(
  //    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, NDIM> acc_output(
      regions[0], task->regions[0], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorW<float, 3> acc_kernel(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  // TensorAccessorR<float, 1> acc_bias(
  //     regions[3], task->regions[3], FID_DATA, ctx, runtime);
  //int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int in_dim = acc_kernel.rect.hi[0] - acc_kernel.rect.lo[0] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int batch_size = acc_output.rect.volume() / out_dim;
  printf("init linear (input): in_dim(%d) out_dim(%d) batch_size(%d)\n",
      in_dim, out_dim, batch_size);
  LinearMeta* m = new LinearMeta(handle, batch_size);
  m->activation = linear->activation;
  m->use_bias = linear->use_bias;
  m->profiling = linear->profiling;
  std::strcpy(m->op_name, linear->name);

  if (use_cudnn_activation(m->activation)) {
    hipdnnActivationMode_t mode;
    switch (linear->activation) {
      case AC_MODE_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case AC_MODE_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      default:
        // Unsupported activation mode
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          batch_size, out_dim, 1, 1));
  }
  return m;
}

void Linear::init(const FFModel& ff)
{
  assert(check_output_input_weight_same_parallel_is());
  //assert(check_output_input_weight_same_machine_view());
  parallel_is = outputs[0]->parallel_is;
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_init(ff, argmap);
  IndexLauncher launcher(LINEAR_INIT_TASK_ID, parallel_is,
                         TaskArgument(this, sizeof(Linear)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         outputs[0]->machine_view.hash());
  //launcher.add_region_requirement(
  //    RegionRequirement(input_lps[0], 0/*projection id*/,
  //                      READ_ONLY, EXCLUSIVE, inputs[0]->region));
  //launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0]->region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0]->part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, weights[0]->region));
  launcher.add_field(1, FID_DATA);
  // launcher.add_region_requirement(
  //     RegionRequirement(weights[1]->part, 0/*projection id*/,
  //                       READ_ONLY, EXCLUSIVE, weights[1]->region));
  // launcher.add_field(3, FID_DATA);
  if (ff.config.computationMode == COMP_MODE_TRAINING) {
    // Add inputs[0]->region_grad to avoid Legion warning
    launcher.add_region_requirement(
        RegionRequirement(inputs[0]->part_grad, 0/*projection id*/,
            WRITE_ONLY, EXCLUSIVE, inputs[0]->region_grad));
    launcher.add_field(2, FID_DATA);
  }
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  set_opmeta_from_futuremap(ff, fm);
}

/*static*/
void Linear::forward_kernel(const LinearMeta* m,
                            const float* input_ptr,
                            float* output_ptr,
                            const float* kernel_ptr,
                            const float* bias_ptr,
                            int in_dim, int out_dim, int batch_size)
{
  float alpha = 1.0f, beta = 0.0f;
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        out_dim, batch_size, in_dim,
                        &alpha, kernel_ptr, in_dim,
                        input_ptr, in_dim, &beta,
                        output_ptr, out_dim));
  // use_bias = True 
  if (bias_ptr != NULL) { 
    checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                          out_dim, batch_size, 1,
                          &alpha, bias_ptr, 1,
                          m->one_ptr, 1, &alpha,
                          output_ptr, out_dim));
  }
  if (use_cudnn_activation(m->activation)) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
        &alpha, m->outputTensor, output_ptr,
        &beta, m->outputTensor, output_ptr));
  } else if (m->activation == AC_MODE_GELU) {
    size_t elements = (size_t)out_dim * (size_t) batch_size;
    constexpr float B = 0.7978845608028654f;   // sqrt(2.0/M_PI)
    constexpr float C = 0.035677408136300125f; // 0.044715 * sqrt(2.0/M_PI)
    gelu_forward_kernel<<<GET_BLOCKS(elements), CUDA_NUM_THREADS>>>(
        elements, B, C, output_ptr);
  } else if (m->activation == AC_MODE_NONE) {
    // Do nothing
  } else {
    assert(false && "Unsupported activation for Linear");
  }
}

__host__
void Linear::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return forward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

/*
  regions[0](I); input
  regions[1](O): output
  regions[2](I): kernel
  regions[3](I): bias
*/
template<int NDIM>
void Linear::forward_task_with_dim(const Task *task,
                                   const std::vector<PhysicalRegion> &regions,
                                   Context ctx, Runtime *runtime)
{
  //Linear* linear = (Linear*) task->args;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  assert(regions.size() == (3 + int(m->use_bias)));
  assert(task->regions.size() == (3 + int(m->use_bias)));
  
  TensorAccessorR<float, NDIM> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, NDIM> acc_output(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorR<float, 3> acc_kernel(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int batch_size = acc_output.rect.volume() / out_dim;
  assert(acc_output.rect.volume() == out_dim * batch_size);
  assert(acc_input.rect.volume() == in_dim * batch_size);
  assert(acc_kernel.rect.volume() == in_dim * out_dim);
  const float* acc_bias_ptr = NULL;
  if (m->use_bias) {
    TensorAccessorR<float, 3> acc_bias(
        regions[3], task->regions[3], FID_DATA, ctx, runtime);
    assert(acc_bias.rect.volume() == out_dim);
    acc_bias_ptr = acc_bias.ptr;
  }

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  Linear::forward_kernel(m, acc_input.ptr, acc_output.ptr,
      acc_kernel.ptr, acc_bias_ptr, in_dim, out_dim, batch_size);

  if (m->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Linear] forward time = %.2lfms\n", m->op_name, elapsed);
    //print_tensor<NDIM, float>(acc_input.ptr, acc_input.rect, "[Linear:forward:input]");
    //print_tensor<2, float>(acc_kernel.ptr, acc_kernel.rect, "[Linear:forward:kernel]");
    //print_tensor<1, float>(acc_bias.ptr, acc_bias.rect, "[Linear:forward:bias]");
    //print_tensor<NDIM, float>(acc_output.ptr, acc_output.rect, "[Linear:forward:output]");
  }
}

void Linear::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_forward(ff, argmap);
  IndexLauncher launcher(LINEAR_FWD_TASK_ID, parallel_is,
                         TaskArgument(NULL, 0), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         outputs[0]->machine_view.hash());
  launcher.add_region_requirement(
      RegionRequirement(inputs[0]->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0]->region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0]->region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0]->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[0]->region));
  launcher.add_field(2, FID_DATA);
  if (use_bias) {
    launcher.add_region_requirement(
        RegionRequirement(weights[1]->part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, weights[1]->region));
    launcher.add_field(3, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

__global__
void sigmoid_backward(float *grad_ptr, const float *output, int n)
{
  CUDA_KERNEL_LOOP(i, n)
  {
    grad_ptr[i] = grad_ptr[i] * output[i] * (1 - output[i]);
  }
}

/*static*/
void Linear::backward_kernel(const LinearMeta* m,
                             const float* input_ptr,
                             float* input_grad_ptr,
                             const float* output_ptr,
                             float* output_grad_ptr,
                             const float* kernel_ptr,
                             float* kernel_grad_ptr,
                             float* bias_grad_ptr,
                             int in_dim, int out_dim, int batch_size)
{
  float alpha = 1.0f;
  int output_size = out_dim * batch_size;
  if (m->activation == AC_MODE_RELU) {
    reluBackward<<<GET_BLOCKS(output_size), CUDA_NUM_THREADS>>>(
        output_grad_ptr, output_ptr, output_size);
  } else if (m->activation == AC_MODE_SIGMOID) {
    sigmoid_backward<<<GET_BLOCKS(output_size), CUDA_NUM_THREADS>>>(
        output_grad_ptr, output_ptr, output_size);
  } else {
    // TODO: only support relu and sigmoid for now
    assert(m->activation == AC_MODE_NONE);
  }
  // Compute weight gradiant
  // NOTE: we use alpha=1 for kernel_grad to accumulate gradients
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        in_dim, out_dim, batch_size,
                        &alpha, input_ptr, in_dim,
                        output_grad_ptr, out_dim,
                        &alpha, kernel_grad_ptr, in_dim));
  // Compute bias gradiant
  // NOTE: we use alpha=1 for bias_grad to accumulate gradients
  // use_bias = True
  if (bias_grad_ptr != NULL) {
    checkCUDA(hipblasSgemv(m->handle.blas, HIPBLAS_OP_N,
                          out_dim, batch_size,
                          &alpha, output_grad_ptr, out_dim,
                          m->one_ptr, 1,
                          &alpha, bias_grad_ptr, 1));
  }
  // Compute data gradiant
  // NOTE: we use alpha=1 for input_grad to accumulate gradients
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_N,
                        in_dim, batch_size, out_dim,
                        &alpha, kernel_ptr, in_dim,
                        output_grad_ptr, out_dim,
                        &alpha, input_grad_ptr, in_dim));
}

void Linear::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return backward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

/*
  regions[0](I): input
  regions[1](I/O): replica_grad or input_grad
  regions[2](I): output
  regions[3](I/O): output_grad
  regions[4](I): filter
  regions[5](I/O): filter_grad
  regions[6](I/O): bias_grad
*/
template<int NDIM>
__host__
void Linear::backward_task_with_dim(const Task *task,
                                    const std::vector<PhysicalRegion> &regions,
                                    Context ctx, Runtime *runtime)
{
  //Linear* linear = (Linear*) task->args;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  assert(regions.size() == (6 + int(m->use_bias)));
  assert(task->regions.size() == (6 + int(m->use_bias)));
  float* input_grad = NULL;
  TensorAccessorR<float, NDIM> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorR<float, NDIM> acc_output(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int batch_size = acc_output.rect.volume() / out_dim;
  Domain domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
  if (domain.get_dim() == NDIM+1) {
    assert(domain.get_volume() == in_dim * batch_size);
    input_grad = helperGetTensorPointerWO<float>(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
  } else {
    TensorAccessorW<float, NDIM> acc_replica_grad(
        regions[1], task->regions[1], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    assert(acc_replica_grad.rect.volume() == in_dim * batch_size);
    input_grad = acc_replica_grad.ptr;
  }
  TensorAccessorW<float, NDIM> acc_output_grad(
      regions[3], task->regions[3], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, 3> acc_kernel(
      regions[4], task->regions[4], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 3> acc_kernel_grad(
      regions[5], task->regions[5], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  // make sure the sizes match
  assert(acc_output.rect.volume() == out_dim * batch_size);
  assert(acc_output_grad.rect.volume() == out_dim * batch_size);
  assert(acc_kernel.rect.volume() == in_dim * out_dim);
  assert(acc_kernel_grad.rect.volume() == in_dim * out_dim);
  float* acc_bias_grad_ptr = NULL;
  if (m->use_bias) {
    TensorAccessorW<float, 3> acc_bias_grad(
        regions[6], task->regions[6], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    assert(acc_bias_grad.rect.volume() == out_dim);
    acc_bias_grad_ptr = static_cast<float*>(acc_bias_grad.ptr);
  }
  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  Linear::backward_kernel(m, acc_input.ptr, input_grad,
      acc_output.ptr, acc_output_grad.ptr,
      acc_kernel.ptr, acc_kernel_grad.ptr,
      acc_bias_grad_ptr, in_dim, out_dim, batch_size);
  if (m->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Linear backward time = %.2lfms\n", elapsed);
    //print_tensor<NDIM, float>(acc_output_grad.ptr, acc_output_grad.rect, "[Linear:backward:output_grad]");
    //print_tensor<2, float>(acc_kernel_grad.ptr, acc_kernel_grad.rect, "[Linear:backward:kernel_grad]");
    //print_tensor<1, float>(acc_bias_grad.ptr, acc_bias_grad.rect, "[Linear:backward:bias_grad]");
    //print_tensor<2, float>(input_grad, acc_input.rect, "[Linear:backward:input_grad]");
  }
}

void Linear::backward(const FFModel& ff)
{
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  {
    ArgumentMap argmap;
    set_argumentmap_for_backward(ff, argmap);
    IndexLauncher launcher(LINEAR_BWD_TASK_ID, parallel_is,
                           TaskArgument(NULL, 0), argmap,
                           Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                           outputs[0]->machine_view.hash());
    // regions[0](I): input
    launcher.add_region_requirement(
        RegionRequirement(inputs[0]->part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, inputs[0]->region));
    launcher.add_field(0, FID_DATA);
    // regions[1](I/O): replica_grad
    assert(replica == NULL);
    launcher.add_region_requirement(
        RegionRequirement(inputs[0]->part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, inputs[0]->region_grad));
    launcher.add_field(1, FID_DATA);
    // regions[2](I): output
    launcher.add_region_requirement(
        RegionRequirement(outputs[0]->part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, outputs[0]->region));
    launcher.add_field(2, FID_DATA);
    // regions[3](I/O): output_grad
    launcher.add_region_requirement(
        RegionRequirement(outputs[0]->part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, outputs[0]->region_grad));
    launcher.add_field(3, FID_DATA);
    // regions[4](I): filter
    launcher.add_region_requirement(
        RegionRequirement(weights[0]->part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, weights[0]->region));
    launcher.add_field(4, FID_DATA);
    // regions[5](I/O): filter_grad
    launcher.add_region_requirement(
        RegionRequirement(weights[0]->part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, weights[0]->region_grad));
    launcher.add_field(5, FID_DATA);
    if (use_bias) {
      // regions[6](I/O): bias_grad
      launcher.add_region_requirement(
          RegionRequirement(weights[1]->part_grad, 0/*projection id*/,
                            READ_WRITE, EXCLUSIVE, weights[1]->region_grad));
      launcher.add_field(6, FID_DATA);
    }
    runtime->execute_index_space(ctx, launcher);
  }
  assert(replica == NULL);
}

/*
__host__
Parameter* Linear::get_parameter(int index)
{
  if (index == 0) {
    return &weights[0];
  } else if (index == 1){
    return &weights[1];
  } else {
    assert(0);
    return NULL;
  }
}
*/

__host__
void Linear::print_layer(const FFModel& ff)
{
  printf("linear layer\n");
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;

  RegionRequirement kernel_req(weights[0]->region, READ_WRITE, EXCLUSIVE, weights[0]->region);
  kernel_req.add_field(FID_DATA);
  InlineLauncher kernel_launcher(kernel_req);
  PhysicalRegion kernel_region = runtime->map_region(ctx, kernel_launcher);
  kernel_region.wait_until_valid();

  RegionRequirement bias_req(weights[1]->region, READ_WRITE, EXCLUSIVE, weights[1]->region);
  bias_req.add_field(FID_DATA);
  InlineLauncher bias_launcher(bias_req);
  PhysicalRegion bias_region = runtime->map_region(ctx, bias_launcher);
  bias_region.wait_until_valid();

  TensorAccessorW<float, 2> acc_kernel(kernel_region, kernel_req, FID_DATA, ctx, runtime, true);
  TensorAccessorW<float, 1> acc_bias(bias_region, bias_req, FID_DATA, ctx, runtime, true);

  const float *kernel_ptr = acc_kernel.ptr;
  const float *bias_ptr = acc_bias.ptr;

  size_t kernel_size = acc_kernel.rect.volume();
  int kernel_dim1 = acc_kernel.rect.hi[0] - acc_kernel.rect.lo[0] + 1;
  int kernel_dim2 = acc_kernel.rect.hi[1] - acc_kernel.rect.lo[1] + 1;
  size_t bias_size = acc_bias.rect.volume();
  printf("kernel, %p, %zu, [%d, %d]\n", kernel_ptr, kernel_size, kernel_dim1, kernel_dim2);
  printf("bias, %p, %zu\n", bias_ptr, bias_size);

  for (int i = 0; i < bias_size; i++) {
    printf("%f ", bias_ptr[i]);
  }
  printf("\n");

  for (int i = 0; i < kernel_size; i++) {
    printf("%f ", kernel_ptr[i]);
  }
  printf("\n");

  runtime->unmap_region(ctx, kernel_region);
  runtime->unmap_region(ctx, bias_region);

}

LinearMeta::LinearMeta(FFHandler handler, int batch_size)
: OpMeta(handler)
{
  // Allocate an all-one's vector
  float* dram_one_ptr = (float *) malloc(sizeof(float) * batch_size);
  for (int i = 0; i < batch_size; i++)
    dram_one_ptr[i] = 1.0f;
  float* fb_one_ptr;
  checkCUDA(hipMalloc(&fb_one_ptr, sizeof(float) * batch_size));
  checkCUDA(hipMemcpy(fb_one_ptr, dram_one_ptr,
                       sizeof(float) * batch_size, hipMemcpyHostToDevice));
  one_ptr = (const float*) fb_one_ptr;
  // Allocate descriptors
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
    checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
}

bool Linear::measure_operator_cost(Simulator* sim,
                                   const ParallelConfig& pc,
                                   CostMetrics& cost_metrics) const
{
  TensorBase sub_output, sub_input;
  if (!outputs[0]->get_output_sub_tensor(pc, sub_output, OP_LINEAR))
    return false;
  if (!inputs[0]->get_input_sub_tensor(pc, sub_input, OP_LINEAR))
    return false;
  int input_c = sub_input.dims[0].size;
  int input_n = sub_input.get_volume() / input_c;
  int output_c = sub_output.dims[0].size;
  int output_n = sub_output.get_volume() / output_c;
  LinearMeta* m = sim->linear_meta;
  m->activation = activation;
  if (use_cudnn_activation(m->activation)) {
    hipdnnActivationMode_t mode;
    switch (activation) {
      case AC_MODE_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case AC_MODE_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      default:
        // Unsupported activation mode
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
        HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, output_n, output_c, 1, 1));
  }
  // allocate tensors in simulator
  sim->free_all();
  float* input_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  float *output_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  float* kernel_ptr = (float*)sim->allocate((size_t)output_c * input_c, DT_FLOAT);
  float* bias_ptr = (float*)sim->allocate(output_c, DT_FLOAT);
  bool out_of_memory = (input_ptr == NULL) || (output_ptr == NULL)
                       || (kernel_ptr == NULL) || (bias_ptr == NULL);
  if (out_of_memory) {
    cost_metrics.forward_time = Simulator::MAXIMUM_TASK_RUN_TIME;
    cost_metrics.backward_time = Simulator::MAXIMUM_TASK_RUN_TIME;
    return true;
  }
  std::function<void()> forward, backward;
  forward = [&] {
    forward_kernel(m, input_ptr, output_ptr, kernel_ptr, bias_ptr,
        input_c, output_c, input_n);
  };
  if (sim->computationMode == COMP_MODE_TRAINING) {
    float* input_grad_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
    float *output_grad_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
    float* kernel_grad_ptr = (float*)sim->allocate((size_t)output_c * input_c, DT_FLOAT);
    float* bias_grad_ptr = (float*)sim->allocate(output_c, DT_FLOAT);
    out_of_memory = (input_grad_ptr == NULL) || (output_grad_ptr == NULL)
                    || (kernel_grad_ptr == NULL) || (bias_grad_ptr == NULL);
    if (out_of_memory) {
      cost_metrics.forward_time = Simulator::MAXIMUM_TASK_RUN_TIME;
      cost_metrics.backward_time = Simulator::MAXIMUM_TASK_RUN_TIME;
      return true;
    }
    backward = [&] {
      backward_kernel(m, input_ptr, input_grad_ptr, output_ptr, output_grad_ptr,
          kernel_ptr, kernel_grad_ptr, bias_grad_ptr, input_c, output_c, input_n);
    };
  }

  inner_measure_operator_cost(sim, forward, backward, cost_metrics);

  if (sim->computationMode == COMP_MODE_TRAINING) {
    printf("[Measure Linear] name(%s) in(%d %d) out(%d %d) forward_time(%.4lf) backward_time(%.4lf)\n",
           name, input_n, input_c, output_n, output_c,
           cost_metrics.forward_time, cost_metrics.backward_time);
  } else {
    printf("[Measure Linear] name(%s) in(%d %d) out(%d %d) forward_time(%.4lf)\n",
           name, input_n, input_c, output_n, output_c,
           cost_metrics.forward_time);
  }
  return true;
}

bool Linear::estimate_sync_cost(Simulator* sim,
                                const MachineView& view,
                                CostMetrics& cost_metrics) const
{
  // Estimate the cost of sync weights
  TensorBase tensor_base;
  tensor_base.num_dims = 3;
  tensor_base.dims[0] = inputs[0]->dims[0];
  tensor_base.dims[1] = inputs[0]->dims[inputs[0]->num_dims-1];
  tensor_base.dims[2] = inputs[0]->dims[inputs[0]->num_dims-2];
  tensor_base.dims[1].size = out_channels;
  tensor_base.dims[1].degree = 1;
  tensor_base.dims[2].degree = inputs[0]->dims[1].degree * inputs[0]->dims[2].degree;
  tensor_base.dims[2].size = inputs[0]->dims[1].degree * inputs[0]->dims[2].degree;
  cost_metrics.sync_time = sim->default_estimate_sync_cost(&tensor_base, view, 1);
  //printf("[Estimate Linear] name(%s) sync_time(%.4lf)\n", name, cost_metrics.sync_time);
  return true;
}

ParallelConfig Linear::get_random_parallel_config(const FFModel& ff) const
{
  if (!ff.config.enable_parameter_parallel)
    return Op::get_random_parallel_config(ff);
  std::vector<int> batch_candidates;
  std::vector<int> channel_candidates;
  int batch = outputs[0]->dims[outputs[0]->num_dims-1].size;
  int channel = outputs[0]->dims[0].size;
  int total_devices = ff.config.workersPerNode * ff.config.numNodes;
  for (int i = 1; i <= ff.config.workersPerNode; i++)
    if (channel % i == 0)
      for (int j = 1; i * j <= total_devices; j++)
        if (batch % j == 0) {
          batch_candidates.push_back(j);
          channel_candidates.push_back(i);
        }
  assert(batch_candidates.size() > 0);
  int idx = std::rand() % batch_candidates.size();
  int num_par_c = channel_candidates[idx];
  int num_par_b = batch_candidates[idx];
  ParallelConfig pc;
  pc.device_type = ParallelConfig::GPU;
  pc.nDims = outputs[0]->num_dims;
  pc.dim[0] = num_par_c;
  pc.dim[pc.nDims-1] = num_par_b;
  for (int i = 1; i < pc.nDims - 1; i++)
    pc.dim[i] = 1;
  int start_idx = std::rand() % (total_devices - num_par_c * num_par_b + 1);
  start_idx = start_idx - start_idx % num_par_c;
  for (int i = 0; i < num_par_c * num_par_b; i++)
    pc.device_ids[i] = start_idx + i;
  return pc;
}

bool Linear::get_int_parameter(PMParameter para, int* value) const
{
  switch(para) {
    case PM_ACTI:
      *value = (int) activation;
      return true;
    default:
      return Op::get_int_parameter(para, value);
  }
}

bool Linear::is_valid_parallel_config(const FFModel& ff, const ParallelConfig& pc) const
{
  if (!ff.config.enable_parameter_parallel)
    return Op::is_valid_parallel_config(ff, pc);
  // Support data and parameter parallel
  if (pc.nDims != outputs[0]->num_dims)
    return false;
  for (int i = 1; i < pc.nDims-1; i++)
    if (pc.dim[i] != 1)
      return false;
  return true;
}
