#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ffconst_utils.h"
#include "flexflow/ops/layer_norm.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

#define C10_WARP_SIZE 32
constexpr int kCUDABlockReduceNumThreads = 512;
constexpr int kCUDANumThreads = 256;
constexpr int kColwiseReduceTileSize = 32;

LayerNormMeta::LayerNormMeta(FFHandler handle,
                             LayerNorm const *ln,
                             MemoryAllocator &gpu_mem_allocator)
    : OpMeta(handle) {
  elementwise_affine = ln->elementwise_affine;
  use_bias = ln->use_bias;
  effective_batch_size = ln->effective_batch_size;
  effective_num_elements = ln->effective_num_elements;
  profiling = ln->profiling;
  inference_debugging = ln->inference_debugging;
  eps = ln->eps;
  DataType data_type = ln->data_type;
  size_t totalSize = effective_batch_size * data_type_size(data_type) * 6;
  gpu_mem_allocator.create_legion_instance(reserveInst, totalSize);
  mean_ptr = gpu_mem_allocator.allocate_instance_untyped(
      data_type_size(data_type) * effective_batch_size);
  rstd_ptr = gpu_mem_allocator.allocate_instance_untyped(
      data_type_size(data_type) * effective_batch_size);
  ds_ptr = gpu_mem_allocator.allocate_instance_untyped(
      data_type_size(data_type) * effective_batch_size);
  db_ptr = gpu_mem_allocator.allocate_instance_untyped(
      data_type_size(data_type) * effective_batch_size);
  scale_ptr = gpu_mem_allocator.allocate_instance_untyped(
      data_type_size(data_type) * effective_batch_size);
  bias_ptr = gpu_mem_allocator.allocate_instance_untyped(
      data_type_size(data_type) * effective_batch_size);
}

LayerNormMeta::~LayerNormMeta(void) {
  if (reserveInst != Realm::RegionInstance::NO_INST) {
    reserveInst.destroy();
  }
}

template <typename T>
__device__ __forceinline__ T WARP_SHFL_DOWN(T value,
                                            unsigned int delta,
                                            int width = warpSize,
                                            unsigned int mask = 0xffffffff) {
#ifndef __HIP_PLATFORM_HCC__
  return __shfl_down_sync(mask, value, delta, width);
#else
  return __shfl_down(value, delta, width);
#endif
}

template <typename T>
__inline__ __device__ T WarpReduceSum(T val) {
#pragma unroll
  for (int offset = (C10_WARP_SIZE >> 1); offset > 0; offset >>= 1) {
    val += WARP_SHFL_DOWN(val, offset);
  }
  return val;
}

template <typename T>
__inline__ __device__ T BlockReduceSum(T val, T *shared) {
  int const lid = threadIdx.x % C10_WARP_SIZE;
  int const wid = threadIdx.x / C10_WARP_SIZE;
  val = WarpReduceSum(val);
  __syncthreads();
  if (lid == 0) {
    shared[wid] = val;
  }
  __syncthreads();
  val = (threadIdx.x < (blockDim.x / C10_WARP_SIZE)) ? shared[lid] : T(0);
  if (wid == 0) {
    val = WarpReduceSum(val);
  }
  return val;
}

template <typename T>
__inline__ __device__ T BlockReduceSum(T val, T *shared, int max_num_threads) {
  int const lid = threadIdx.x % C10_WARP_SIZE;
  int const wid = threadIdx.x / C10_WARP_SIZE;
  val = WarpReduceSum(val);
  __syncthreads();
  if (lid == 0) {
    shared[wid] = val;
  }
  __syncthreads();
  val = (threadIdx.x < (min(blockDim.x, max_num_threads) / C10_WARP_SIZE))
            ? shared[lid]
            : T(0);
  if (wid == 0) {
    val = WarpReduceSum(val);
  }
  return val;
}

#ifdef DEADCODE
template <typename T>
__global__ void RowwiseMomentsCUDAKernel(
    int64_t N, float eps, T const *X, T *mean, T *rstd) {
  __shared__ float m_shared[C10_WARP_SIZE];
  __shared__ float v_shared[C10_WARP_SIZE];
  const int64_t i = blockIdx.x;
  float sum1 = 0.0f;
  float sum2 = 0.0f;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    sum1 += static_cast<float>(X[index]);
    sum2 += static_cast<float>(X[index]) * static_cast<float>(X[index]);
  }
  sum1 = BlockReduceSum<float>(sum1, m_shared);
  sum2 = BlockReduceSum<float>(sum2, v_shared);
  if (threadIdx.x == 0) {
    float const scale = float(1) / static_cast<float>(N);
    sum1 *= scale;
    sum2 = max(sum2 * scale - sum1 * sum1, float(0));
    mean[i] = static_cast<T>(sum1);
    rstd[i] = static_cast<T>(rsqrt(sum2 + eps));
  }
}

template <typename T>
__global__ void LayerNormForwardCUDAKernel(int64_t N,
                                           T const *X,
                                           T const *mean,
                                           T const *rstd,
                                           T const *gamma,
                                           T const *beta,
                                           T *Y) {
  using T_ACC = T;
  const int64_t i = blockIdx.x;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    const T_ACC beta_v =
        beta == nullptr ? T_ACC(0) : static_cast<T_ACC>(beta[j]);
    Y[index] = (static_cast<T_ACC>(X[index]) - static_cast<T_ACC>(mean[i])) *
                   static_cast<T_ACC>(rstd[i]) * gamma_v +
               beta_v;
  }
}
#endif

template <typename T>
__global__ void LayerNormFusedForwardKernel(int64_t N,
                                            float eps,
                                            T const *X,
                                            T *mean,
                                            T *rstd,
                                            T const *gamma,
                                            T const *beta,
                                            T *Y) {
  __shared__ float m_shared[C10_WARP_SIZE];
  __shared__ float v_shared[C10_WARP_SIZE];
  const int64_t i = blockIdx.x;
  float sum1 = 0.0f;
  float sum2 = 0.0f;
  for (int64_t j = threadIdx.x; j < N;
       j += min(blockDim.x, kCUDABlockReduceNumThreads)) {
    const int64_t index = i * N + j;
    sum1 += static_cast<float>(X[index]);
    sum2 += static_cast<float>(X[index]) * static_cast<float>(X[index]);
  }
  if (threadIdx.x < kCUDABlockReduceNumThreads) {
    sum1 = BlockReduceSum<float>(
        sum1, m_shared, min(blockDim.x, kCUDABlockReduceNumThreads));
    sum2 = BlockReduceSum<float>(
        sum2, v_shared, min(blockDim.x, kCUDABlockReduceNumThreads));
  }
  if (threadIdx.x == 0) {
    float const scale = float(1) / static_cast<float>(N);
    sum1 *= scale;
    sum2 = max(sum2 * scale - sum1 * sum1, float(0));
    mean[i] = static_cast<T>(sum1);
    rstd[i] = static_cast<T>(rsqrt(sum2 + eps));
  }

  __syncthreads();

  using T_ACC = T;
  for (int64_t j = threadIdx.x; j < N; j += min(blockDim.x, kCUDANumThreads)) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    const T_ACC beta_v =
        beta == nullptr ? T_ACC(0) : static_cast<T_ACC>(beta[j]);
    Y[index] = (static_cast<T_ACC>(X[index]) - static_cast<T_ACC>(mean[i])) *
                   static_cast<T_ACC>(rstd[i]) * gamma_v +
               beta_v;
  }
}

/*static*/
template <typename T>
void LayerNorm::forward_kernel(LayerNormMeta const *m,
                               T const *in_ptr,
                               T *out_ptr,
                               T const *gamma_ptr,
                               T const *beta_ptr,
                               hipStream_t stream) {

  std::pair<int, int> kernel1_parallelism =
      std::make_pair(m->effective_batch_size, kCUDABlockReduceNumThreads);
  std::pair<int, int> kernel2_parallelism =
      std::make_pair(m->effective_batch_size, kCUDANumThreads);

  int num_blocks =
      std::max(kernel1_parallelism.first, kernel2_parallelism.first);
  int num_threads =
      std::max(kernel1_parallelism.second, kernel2_parallelism.second);

  LayerNormFusedForwardKernel<T>
      <<<num_blocks, num_threads, 0, stream>>>(m->effective_num_elements,
                                               m->eps,
                                               in_ptr,
                                               static_cast<T *>(m->mean_ptr),
                                               static_cast<T *>(m->rstd_ptr),
                                               gamma_ptr,
                                               beta_ptr,
                                               out_ptr);
}

/*static*/
void LayerNorm::forward_kernel_wrapper(LayerNormMeta const *m,
                                       GenericTensorAccessorR const &input,
                                       GenericTensorAccessorW &output,
                                       GenericTensorAccessorR const &gamma,
                                       GenericTensorAccessorR const &beta) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  if (m->input_type[0] == DT_FLOAT) {
    LayerNorm::forward_kernel<float>(
        m,
        input.get_float_ptr(),
        output.get_float_ptr(),
        m->elementwise_affine ? gamma.get_float_ptr() : nullptr,
        (m->elementwise_affine && m->use_bias) ? beta.get_float_ptr() : nullptr,
        stream);
  } else if (m->input_type[0] == DT_HALF) {
    LayerNorm::forward_kernel<half>(
        m,
        input.get_half_ptr(),
        output.get_half_ptr(),
        m->elementwise_affine ? gamma.get_half_ptr() : nullptr,
        (m->elementwise_affine && m->use_bias) ? beta.get_half_ptr() : nullptr,
        stream);
  } else {
    assert(false && "unsupport datatype in layernorm");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("[LayerNorm] forward time (CF) = %.9fms\n", elapsed);
    // print_tensor<T>(in_ptr, 32, "[LayerNorm:forward:input]");
    // print_tensor<T>(out_ptr, 32, "[LayerNorm:forward:output]");
  }
}

template <typename T>
__global__ void ComputeInternalGradientsCUDAKernel(
    int64_t N, T const *dY, T const *X, T const *gamma, T *ds, T *db) {
  using T_ACC = T;
  __shared__ T_ACC ds_shared[C10_WARP_SIZE];
  __shared__ T_ACC db_shared[C10_WARP_SIZE];
  const int64_t i = blockIdx.x;
  T_ACC sum1 = 0;
  T_ACC sum2 = 0;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    sum1 +=
        static_cast<T_ACC>(dY[index]) * static_cast<T_ACC>(X[index]) * gamma_v;
    sum2 += static_cast<T_ACC>(dY[index]) * gamma_v;
  }
  sum1 = BlockReduceSum<T_ACC>(sum1, ds_shared);
  sum2 = BlockReduceSum<T_ACC>(sum2, db_shared);
  if (threadIdx.x == 0) {
    ds[i] = sum1;
    db[i] = sum2;
  }
}

template <typename T>
__global__ void ComputeGradientFusedParamsCUDAKernel(int64_t M,
                                                     int64_t N,
                                                     T const *mean,
                                                     T const *rstd,
                                                     T const *ds,
                                                     T const *db,
                                                     T *c1,
                                                     T *c2) {
  using T_ACC = T;
  const int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < M) {
    const T_ACC s = T_ACC(1) / static_cast<T_ACC>(N);
    const T_ACC a = (db[index] * static_cast<T_ACC>(mean[index]) - ds[index]) *
                    static_cast<T_ACC>(rstd[index]) *
                    static_cast<T_ACC>(rstd[index]) *
                    static_cast<T_ACC>(rstd[index]) * s;
    c1[index] = a;
    c2[index] = -(a * static_cast<T_ACC>(mean[index]) +
                  db[index] * static_cast<T_ACC>(rstd[index]) * s);
  }
}

template <typename T>
__global__ void LayerNormBackwardCUDAKenrel(int64_t N,
                                            T const *dY,
                                            T const *X,
                                            T const *gamma,
                                            T const *a,
                                            T const *b,
                                            T const *c,
                                            T *dX) {
  using T_ACC = T;
  const int64_t i = blockIdx.x;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    dX[index] =
        static_cast<T_ACC>(a[i]) * static_cast<T_ACC>(dY[index]) * gamma_v +
        b[i] * static_cast<T_ACC>(X[index]) + c[i];
  }
}

template <typename T>
__global__ void GammaBetaBackwardSimpleCUDAKernel(int64_t M,
                                                  int64_t N,
                                                  T const *dY,
                                                  T const *X,
                                                  T const *mean,
                                                  T const *rstd,
                                                  T *dg,
                                                  T *db) {
  using T_ACC = T;
  const int64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j < N) {
    T_ACC sum1 = 0;
    T_ACC sum2 = 0;
    for (int64_t i = 0; i < M; ++i) {
      const int64_t index = i * N + j;
      sum1 += dg == nullptr ? T_ACC(0)
                            : static_cast<T_ACC>(dY[index]) *
                                  (static_cast<T_ACC>(X[index]) -
                                   static_cast<T_ACC>(mean[i])) *
                                  static_cast<T_ACC>(rstd[i]);
      sum2 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index]);
    }
    if (dg != nullptr) {
      dg[j] = sum1;
    }
    if (db != nullptr) {
      db[j] = sum2;
    }
  }
}

template <typename T>
__global__ void GammaBetaBackwardCUDAKernel(int64_t M,
                                            int64_t N,
                                            T const *dY,
                                            T const *X,
                                            T const *mean,
                                            T const *rstd,
                                            T *dg,
                                            T *db) {
  using T_ACC = T;
  __shared__ T_ACC g_shared[kColwiseReduceTileSize][kColwiseReduceTileSize + 1];
  __shared__ T_ACC b_shared[kColwiseReduceTileSize][kColwiseReduceTileSize + 1];
  const int64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  T_ACC dg_sum1 = 0;
  T_ACC dg_sum2 = 0;
  T_ACC db_sum1 = 0;
  T_ACC db_sum2 = 0;
  if (j < N) {
    for (int64_t i = threadIdx.y; i < M; i += blockDim.y * 2) {
      const int64_t i1 = i;
      const int64_t i2 = i + blockDim.y;
      const int64_t index1 = i1 * N + j;
      const int64_t index2 = i2 * N + j;
      dg_sum1 += dg == nullptr ? T_ACC(0)
                               : static_cast<T_ACC>(dY[index1]) *
                                     (static_cast<T_ACC>(X[index1]) -
                                      static_cast<T_ACC>(mean[i1])) *
                                     static_cast<T_ACC>(rstd[i1]);
      db_sum1 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index1]);
      if (i2 < M) {
        dg_sum2 += dg == nullptr ? T_ACC(0)
                                 : static_cast<T_ACC>(dY[index2]) *
                                       (static_cast<T_ACC>(X[index2]) -
                                        static_cast<T_ACC>(mean[i2])) *
                                       static_cast<T_ACC>(rstd[i2]);
        db_sum2 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index2]);
      }
    }
  }
  g_shared[threadIdx.y][threadIdx.x] = dg_sum1;
  g_shared[threadIdx.y + blockDim.y][threadIdx.x] = dg_sum2;
  b_shared[threadIdx.y][threadIdx.x] = db_sum1;
  b_shared[threadIdx.y + blockDim.y][threadIdx.x] = db_sum2;
  __syncthreads();
  T_ACC sum1 = g_shared[threadIdx.x][threadIdx.y];
  T_ACC sum2 = b_shared[threadIdx.x][threadIdx.y];
  sum1 = WarpReduceSum(sum1);
  sum2 = WarpReduceSum(sum2);
  if (threadIdx.x == 0) {
    const int64_t j = blockIdx.x * blockDim.x + threadIdx.y;
    if (j < N) {
      if (dg != nullptr) {
        dg[j] = sum1;
      }
      if (db != nullptr) {
        db[j] = sum2;
      }
    }
  }
  sum1 = g_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum2 = b_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum1 = WarpReduceSum(sum1);
  sum2 = WarpReduceSum(sum2);
  if (threadIdx.x == 0) {
    const int64_t j = blockIdx.x * blockDim.x + threadIdx.y + blockDim.y;
    if (j < N) {
      if (dg != nullptr) {
        dg[j] = sum1;
      }
      if (db != nullptr) {
        db[j] = sum2;
      }
    }
  }
}

template <typename T>
__device__ __inline__ void compute_gI(T const *__restrict__ dY,
                                      T const *__restrict__ X,
                                      T const *__restrict__ mean,
                                      T const *__restrict__ rstd,
                                      T const *__restrict__ gamma,
                                      T *dX,
                                      int const N,
                                      T *buf) {
  auto const i1 = blockIdx.x;
  const T mean_val = mean[i1];
  const T rstd_val = rstd[i1];
  T stats_x1{0}, stats_x2{0};
  constexpr int unroll = 4;
  auto l = unroll * threadIdx.x;
  T const *X_i = X + i1 * N;
  T const *dY_i = dY + i1 * N;
  T *dX_i = dX + i1 * N;
  // vectorized reads don't improve perf, so use regular unrolling

  for (; l + unroll - 1 < N; l += blockDim.x * unroll) {
#pragma unroll
    for (int k = 0; k < unroll; k++) {
      T gamma_val = (gamma != nullptr) ? static_cast<T>(gamma[l + k]) : T(1);
      const T c_h = static_cast<T>(X_i[l + k]);
      const T c_loss = static_cast<T>(dY_i[l + k]);
      stats_x1 += c_loss * gamma_val;
      stats_x2 += c_loss * gamma_val * (c_h - mean_val) * rstd_val;
    }
  }
  for (; l < N; l++) {
    T gamma_val = (gamma != nullptr) ? static_cast<T>(gamma[l]) : T(1);
    const T c_h = static_cast<T>(X_i[l]);
    const T c_loss = static_cast<T>(dY_i[l]);
    stats_x1 += c_loss * gamma_val;
    stats_x2 += c_loss * gamma_val * (c_h - mean_val) * rstd_val;
  }

  stats_x1 = BlockReduceSum(stats_x1, buf);
  stats_x2 = BlockReduceSum(stats_x2, buf);
  if (threadIdx.x == 0) {
    buf[0] = stats_x1;
    buf[1] = stats_x2;
  }
  __syncthreads();
  stats_x1 = buf[0];
  stats_x2 = buf[1];
  T fH = N;
  T term1 = (T(1) / fH) * rstd_val;

  for (int l = threadIdx.x; l < N; l += blockDim.x) {
    const T x = X_i[l];
    const T dy = dY_i[l];
    T gamma_val = (gamma != nullptr) ? static_cast<T>(gamma[l]) : T(1);
    T f_grad_input = fH * gamma_val * dy;
    f_grad_input -= (x - mean_val) * rstd_val * stats_x2;
    f_grad_input -= stats_x1;
    f_grad_input *= term1;
    dX_i[l] = f_grad_input;
  }
}

template <typename T>
__global__ void layer_norm_grad_input_kernel(T const *__restrict__ dY,
                                             T const *__restrict__ X,
                                             T const *__restrict__ mean,
                                             T const *__restrict__ rstd,
                                             T const *__restrict__ gamma,
                                             T *dX,
                                             int const N) {
  alignas(sizeof(double)) extern __shared__ char s_data1[];
  T *buf = reinterpret_cast<T *>(&s_data1);

  compute_gI(dY, X, mean, rstd, gamma, dX, N, buf);
}

/*static*/
template <typename T>
void LayerNorm::backward_kernel(LayerNormMeta const *m,
                                T const *output_grad_ptr,
                                T const *input_ptr,
                                T *input_grad_ptr,
                                T const *gamma_ptr,
                                T *gamma_grad_ptr,
                                T *beta_grad_ptr,
                                hipStream_t stream) {
  const int64_t M = m->effective_batch_size;
  const int64_t N = m->effective_num_elements;
  ComputeInternalGradientsCUDAKernel<T>
      <<<M, kCUDABlockReduceNumThreads, 0, stream>>>(
          N,
          output_grad_ptr,
          input_ptr,
          gamma_ptr,
          static_cast<T *>(m->ds_ptr),
          static_cast<T *>(m->db_ptr));
  const int64_t B = (M + kCUDANumThreads - 1) / kCUDANumThreads;
  ComputeGradientFusedParamsCUDAKernel<T>
      <<<B, kCUDANumThreads, 0, stream>>>(M,
                                          N,
                                          static_cast<T *>(m->mean_ptr),
                                          static_cast<T *>(m->rstd_ptr),
                                          static_cast<T *>(m->ds_ptr),
                                          static_cast<T *>(m->db_ptr),
                                          static_cast<T *>(m->scale_ptr),
                                          static_cast<T *>(m->bias_ptr));
  int const warp_size = C10_WARP_SIZE;
  int const num_threads = 128;
  const dim3 blocks(M);
  int nshared = (num_threads / warp_size) * sizeof(T);
  layer_norm_grad_input_kernel<<<blocks, num_threads, nshared, stream>>>(
      output_grad_ptr,
      input_ptr,
      static_cast<T *>(m->mean_ptr),
      static_cast<T *>(m->rstd_ptr),
      gamma_ptr,
      input_grad_ptr,
      N);

  if (gamma_grad_ptr != NULL || beta_grad_ptr != NULL) {
    if (M < 512) {
      // For small batch size, do colwise reduce directly
      const int64_t B = (N + kCUDANumThreads - 1) / kCUDANumThreads;
      GammaBetaBackwardSimpleCUDAKernel<T>
          <<<B, kCUDANumThreads, 0, stream>>>(M,
                                              N,
                                              output_grad_ptr,
                                              input_ptr,
                                              static_cast<T *>(m->mean_ptr),
                                              static_cast<T *>(m->rstd_ptr),
                                              gamma_grad_ptr,
                                              beta_grad_ptr);
    } else {
      const int64_t B =
          (N + kColwiseReduceTileSize - 1) / kColwiseReduceTileSize;
      constexpr int kThreadX = kColwiseReduceTileSize;
      constexpr int kThreadY = kColwiseReduceTileSize / 2;
      GammaBetaBackwardCUDAKernel<T>
          <<<B, dim3(kThreadX, kThreadY), 0, stream>>>(
              M,
              N,
              output_grad_ptr,
              input_ptr,
              static_cast<T *>(m->mean_ptr),
              static_cast<T *>(m->rstd_ptr),
              gamma_grad_ptr,
              beta_grad_ptr);
    }
  }
}

/*static*/
template <typename T>
void LayerNorm::backward_kernel_wrapper(LayerNormMeta const *m,
                                        T const *output_grad_ptr,
                                        T const *input_ptr,
                                        T *input_grad_ptr,
                                        T const *gamma_ptr,
                                        T *gamma_grad_ptr,
                                        T *beta_grad_ptr) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  if (m->output_type[0] == DT_FLOAT) {
    LayerNorm::backward_kernel<float>(m,
                                      output_grad_ptr,
                                      input_ptr,
                                      input_grad_ptr,
                                      gamma_ptr,
                                      gamma_grad_ptr,
                                      beta_grad_ptr,
                                      stream);
  }
  // }else if(m->output_type[0] == DT_HALF){
  //   LayerNorm::backward_kernel<half>(m,
  //                                   output_grad_ptr,
  //                                   input_ptr,
  //                                   input_grad_ptr,
  //                                   gamma_ptr,
  //                                   gamma_grad_ptr,
  //                                   beta_grad_ptr,
  //                                   stream);
  // }
}

template void
    LayerNorm::backward_kernel_wrapper<float>(LayerNormMeta const *m,
                                              float const *output_grad_ptr,
                                              float const *input_ptr,
                                              float *input_grad_ptr,
                                              float const *gamma_ptr,
                                              float *gamma_grad_ptr,
                                              float *beta_grad_ptr);

}; // namespace FlexFlow
