#include "hip/hip_runtime.h"
/* Copyright 2021 CMU, Facebook
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

#define C10_WARP_SIZE 32
constexpr int kCUDABlockReduceNumThreads = 512;
constexpr int kCUDANumThreads = 256;
constexpr int kColwiseReduceTileSize = 32;

Tensor FFModel::layer_norm(const Tensor& input,
                           const std::vector<int>& axes,
                           bool elementwise_affine,
                           float eps,
                           const char* name)
{
  // axes must be the last axes.size() dimensions
  for (int i = 0; i < axes.size(); i++) {
    bool found = false;
    for (int j = 0; j < axes.size(); j++) 
      if (axes[j] == input.numDim - 1 - i)
        found = true;
    if (!found) {
      assert(false && "axes must be the last axes.size() dimensions");
    }
  }
  LayerNorm *ln = new LayerNorm(*this, input, axes, elementwise_affine, eps, name);
  layers.push_back(ln);
  return ln->outputs[0];
}

LayerNorm::LayerNorm(FFModel& model,
                     const Tensor& _input,
                     const std::vector<int>& axes,
                     bool _elementwise_affine,
                     float _eps,
                     const char *name)
: Op(model, OP_LAYERNORM, name, _input),
  elementwise_affine(_elementwise_affine),
  eps(_eps)
{
  outputs[0].numDim = inputs[0].numDim;
  for (int i = 0; i < outputs[0].numDim; i++)
    outputs[0].adim[i] = inputs[0].adim[i];
  int M = 1;
  for (int i = 0; i < axes.size(); i++)
    M *= inputs[0].adim[inputs[0].numDim-1-axes[i]];
  if (elementwise_affine) {
    numWeights = 2;
    weights[0].numDim = 1;
    weights[0].adim[0] = M;
    weights[1].numDim = 1;
    weights[1].adim[0] = M;
  } else {
    numWeights = 0;
  }
  return;
}

void LayerNorm::create_weights(FFModel& model)
{
  std::string pcname = name;
  task_is = model.get_or_create_task_is(outputs[0].numDim, pcname);

  // Create scale and bias
  Initializer* scale_initializer = new ConstantInitializer(1.0f);
  Initializer* bias_initializer = new ConstantInitializer(0.0f);
  const int dims[1] = {weights[0].adim[0]};
  weights[0] = model.create_conv_weight<1>(this, dims, DT_FLOAT, scale_initializer);
  weights[1] = model.create_conv_weight<1>(this, dims, DT_FLOAT, bias_initializer);
}

void LayerNorm::create_output_and_partition(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = model.get_or_create_task_is(outputs[0].numDim, pcname);
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Domain part_rect = runtime->get_index_space_domain(ctx, task_is);
  {
    int dims[MAX_TENSOR_DIM];
    int ndims = outputs[0].numDim;
    for (int i = 0; i < outputs[0].numDim; i++)
      dims[i] = outputs[0].adim[ndims-1-i];
    switch (ndims) {
#define DIMFUNC(DIM) \
      case DIM: \
      { \
        outputs[0] = model.create_tensor<DIM>(dims, outputs[0].data_type, this); \
        break; \
      }
      LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    }
    outputs[0].owner_op = this;
    outputs[0].owner_idx = 0;
  }
  Domain input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part.get_index_partition());
  // Currently assume output and input must be partitioned in the same way
  if (input_rect == part_rect) {
    input_lps[0] = inputs[0].part;
    input_grad_lps[0] = inputs[0].part_grad;
  } else {
    assert(false && "LayerNorm currently assume output/input have same partition");
  }
}

LayerNormMeta::LayerNormMeta(FFHandler handle, const LayerNorm* ln)
: OpMeta(handle)
{
  effective_batch_size = ln->effective_batch_size;
  effective_num_elements = ln->effective_num_elements;
  eps = ln->eps;
  checkCUDA(hipMalloc(&mean_ptr, sizeof(float) * effective_batch_size));
  checkCUDA(hipMalloc(&rstd_ptr, sizeof(float) * effective_batch_size));
  checkCUDA(hipMalloc(&ds_ptr, sizeof(float) * effective_batch_size));
  checkCUDA(hipMalloc(&db_ptr, sizeof(float) * effective_batch_size));
  checkCUDA(hipMalloc(&scale_ptr, sizeof(float) * effective_batch_size));
  checkCUDA(hipMalloc(&bias_ptr, sizeof(float) * effective_batch_size));
}

__host__
OpMeta* LayerNorm::init_task(const Task *task,
                             const std::vector<PhysicalRegion> &regions,
                             Context ctx, Runtime *runtime)
{
  LayerNorm* ln = (LayerNorm*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  LayerNormMeta* meta = new LayerNormMeta(handle, ln);
  return meta;
}

void LayerNorm::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      ParallelConfig pc; \
      std::string pcname = name; \
      ff.config.find_parallel_config(DIM, pcname, pc); \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        FFHandler handle = ff.handlers[pc.device_ids[idx++]]; \
        argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(LAYERNORM_INIT_TASK_ID, task_is,
    TaskArgument(this, sizeof(LayerNorm)), argmap,
    Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
    FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(0, FID_DATA);
  for (int i = 0; i < numInputs; i++) {
    launcher.add_region_requirement(
      RegionRequirement(input_lps[i], 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[i].region));
    launcher.add_field(i + 1, FID_DATA);
  }
  for (int i = 0; i < numInputs; i++) {
    launcher.add_region_requirement(
      RegionRequirement(input_grad_lps[i], 0/*projection id*/,
        WRITE_ONLY, EXCLUSIVE, inputs[i].region_grad));
    launcher.add_field(i + numInputs + 1, FID_DATA);
  }
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        meta[idx++] = fm.get_result<OpMeta*>(*it); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

/*
  regions[0](I): input
  regions[1](O): output
  regions[2](I/O): gamma
  regions[3](I/O): beta
*/
void LayerNorm::forward_task(const Task *task,
                             const std::vector<PhysicalRegion> &regions,
                             Context ctx, Runtime *runtime)
{
  const LayerNormMeta* m = *((LayerNormMeta**) task->local_args);
  assert(task->regions.size() == regions.size());
  const float *in_ptr = NULL;
  float *out_ptr = NULL, *gamma_ptr = NULL, *beta_ptr = NULL;
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  in_ptr = helperGetTensorPointerRO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  Domain out_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
  out_ptr = helperGetTensorPointerWO<float>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  assert(in_domain == out_domain);
  assert(in_domain.get_volume() == m->effective_num_elements * m->effective_batch_size);
  if (m->elementwise_affine) {
    assert(regions.size() == 4);
    Domain gamma_domain = runtime->get_index_space_domain(
      ctx, task->regions[2].region.get_index_space());
    gamma_ptr = helperGetTensorPointerRW<float>(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
    Domain beta_domain = runtime->get_index_space_domain(
      ctx, task->regions[3].region.get_index_space());
    beta_ptr = helperGetTensorPointerRW<float>(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
    assert(gamma_domain == beta_domain);
    assert(gamma_domain.get_volume() == m->effective_num_elements);
  } else {
    assert(regions.size() == 2);
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  forward_kernel<float>(m, in_ptr, out_ptr, gamma_ptr, beta_ptr, stream);
}

template <typename T>
__device__ __forceinline__ T WARP_SHFL_DOWN(T value, unsigned int delta, int width = warpSize, unsigned int mask = 0xffffffff)
{
#ifndef __HIP_PLATFORM_HCC__
    return __shfl_down_sync(mask, value, delta, width);
#else
    return __shfl_down(value, delta, width);
#endif
}

template <typename T>
__inline__ __device__ T WarpReduceSum(T val) {
#pragma unroll
  for (int offset = (C10_WARP_SIZE >> 1); offset > 0; offset >>= 1) {
    val += WARP_SHFL_DOWN(val, offset);
  }
  return val;
}

template <typename T>
__inline__ __device__ T BlockReduceSum(T val, T* shared) {
  const int lid = threadIdx.x % C10_WARP_SIZE;
  const int wid = threadIdx.x / C10_WARP_SIZE;
  val = WarpReduceSum(val);
  __syncthreads();
  if (lid == 0) {
    shared[wid] = val;
  }
  __syncthreads();
  val = (threadIdx.x < blockDim.x / C10_WARP_SIZE) ? shared[lid] : 0;
  if (wid == 0) {
    val = WarpReduceSum(val);
  }
  return val;
}

template <typename T>
__global__ void RowwiseMomentsCUDAKernel(
    int64_t N,
    T eps,
    const T* X,
    T* mean,
    T* rstd) {
  __shared__ T m_shared[C10_WARP_SIZE];
  __shared__ T v_shared[C10_WARP_SIZE];
  const int64_t i = blockIdx.x;
  T sum1 = 0;
  T sum2 = 0;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    sum1 += static_cast<T>(X[index]);
    sum2 += static_cast<T>(X[index]) * static_cast<T>(X[index]);
  }
  sum1 = BlockReduceSum<T>(sum1, m_shared);
  sum2 = BlockReduceSum<T>(sum2, v_shared);
  if (threadIdx.x == 0) {
    const T scale = T(1) / static_cast<T>(N);
    sum1 *= scale;
    sum2 = max(sum2 * scale - sum1 * sum1, T(0));
    mean[i] = sum1;
    rstd[i] = rsqrt(sum2 + static_cast<T>(eps));
  }
}

template <typename T>
__global__ void LayerNormForwardCUDAKernel(
    int64_t N,
    const T* X,
    const T* mean,
    const T* rstd,
    const T* gamma,
    const T* beta,
    T* Y) {
  using T_ACC = T;
  const int64_t i = blockIdx.x;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    const T_ACC beta_v =
        beta == nullptr ? T_ACC(0) : static_cast<T_ACC>(beta[j]);
    Y[index] = (static_cast<T_ACC>(X[index]) - static_cast<T_ACC>(mean[i])) *
            static_cast<T_ACC>(rstd[i]) * gamma_v +
        beta_v;
  }
}

template<typename T>
void LayerNorm::forward_kernel(const LayerNormMeta* m,
                               const T* in_ptr,
                               T* out_ptr,
                               T* gamma_ptr,
                               T* beta_ptr,
                               hipStream_t stream)
{
  RowwiseMomentsCUDAKernel<float>
      <<<m->effective_batch_size, kCUDABlockReduceNumThreads, 0, stream>>>(
          m->effective_num_elements, m->eps, in_ptr, m->mean_ptr, m->rstd_ptr);
  LayerNormForwardCUDAKernel<float><<<m->effective_batch_size, kCUDANumThreads, 0, stream>>>(
      m->effective_num_elements, in_ptr, m->mean_ptr, m->rstd_ptr, gamma_ptr, beta_ptr, out_ptr);
}

void LayerNorm::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    } 
    LEGION_FOREACH_N(DIMFUNC) 
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(LAYERNORM_FWD_TASK_ID, task_is,
                         TaskArgument(NULL, 0), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(input_lps[0], 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(1, FID_DATA);
  if (elementwise_affine) {
    launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, weights[0].region));
    launcher.add_field(2, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(weights[1].part, 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, weights[1].region));
    launcher.add_field(3, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

template <typename T>
__global__ void ComputeInternalGradientsCUDAKernel(
    int64_t N,
    const T* dY,
    const T* X,
    const T* gamma,
    T* ds,
    T* db) {
  using T_ACC = T;
  __shared__ T_ACC ds_shared[C10_WARP_SIZE];
  __shared__ T_ACC db_shared[C10_WARP_SIZE];
  const int64_t i = blockIdx.x;
  T_ACC sum1 = 0;
  T_ACC sum2 = 0;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    sum1 +=
        static_cast<T_ACC>(dY[index]) * static_cast<T_ACC>(X[index]) * gamma_v;
    sum2 += static_cast<T_ACC>(dY[index]) * gamma_v;
  }
  sum1 = BlockReduceSum<T_ACC>(sum1, ds_shared);
  sum2 = BlockReduceSum<T_ACC>(sum2, db_shared);
  if (threadIdx.x == 0) {
    ds[i] = sum1;
    db[i] = sum2;
  }
}

template <typename T>
__global__ void ComputeGradientFusedParamsCUDAKernel(
    int64_t M,
    int64_t N,
    const T* mean,
    const T* rstd,
    const T* ds,
    const T* db,
    T* c1,
    T* c2) {
  using T_ACC = T;
  const int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < M) {
    const T_ACC s = T_ACC(1) / static_cast<T_ACC>(N);
    const T_ACC a = (db[index] * static_cast<T_ACC>(mean[index]) - ds[index]) *
        static_cast<T_ACC>(rstd[index]) * static_cast<T_ACC>(rstd[index]) *
        static_cast<T_ACC>(rstd[index]) * s;
    c1[index] = a;
    c2[index] =
        -(a * static_cast<T_ACC>(mean[index]) +
          db[index] * static_cast<T_ACC>(rstd[index]) * s);
  }
}

template <typename T>
__global__ void LayerNormBackwardCUDAKenrel(
    int64_t N,
    const T* dY,
    const T* X,
    const T* gamma,
    const T* a,
    const T* b,
    const T* c,
    T* dX) {
  using T_ACC = T;
  const int64_t i = blockIdx.x;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    dX[index] =
        static_cast<T_ACC>(a[i]) * static_cast<T_ACC>(dY[index]) * gamma_v +
        b[i] * static_cast<T_ACC>(X[index]) + c[i];
  }
}

template <typename T>
__global__ void GammaBetaBackwardSimpleCUDAKernel(
    int64_t M,
    int64_t N,
    const T* dY,
    const T* X,
    const T* mean,
    const T* rstd,
    T* dg,
    T* db) {
  using T_ACC = T;
  const int64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j < N) {
    T_ACC sum1 = 0;
    T_ACC sum2 = 0;
    for (int64_t i = 0; i < M; ++i) {
      const int64_t index = i * N + j;
      sum1 += dg == nullptr ? T_ACC(0)
                            : static_cast<T_ACC>(dY[index]) *
              (static_cast<T_ACC>(X[index]) - static_cast<T_ACC>(mean[i])) *
              static_cast<T_ACC>(rstd[i]);
      sum2 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index]);
    }
    if (dg != nullptr) {
      dg[j] = sum1;
    }
    if (db != nullptr) {
      db[j] = sum2;
    }
  }
}

template <typename T>
__global__ void GammaBetaBackwardCUDAKernel(
    int64_t M,
    int64_t N,
    const T* dY,
    const T* X,
    const T* mean,
    const T* rstd,
    T* dg,
    T* db) {
  using T_ACC = T;
  __shared__ T_ACC g_shared[kColwiseReduceTileSize][kColwiseReduceTileSize + 1];
  __shared__ T_ACC b_shared[kColwiseReduceTileSize][kColwiseReduceTileSize + 1];
  const int64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  T_ACC dg_sum1 = 0;
  T_ACC dg_sum2 = 0;
  T_ACC db_sum1 = 0;
  T_ACC db_sum2 = 0;
  if (j < N) {
    for (int64_t i = threadIdx.y; i < M; i += blockDim.y * 2) {
      const int64_t i1 = i;
      const int64_t i2 = i + blockDim.y;
      const int64_t index1 = i1 * N + j;
      const int64_t index2 = i2 * N + j;
      dg_sum1 += dg == nullptr ? T_ACC(0)
                               : static_cast<T_ACC>(dY[index1]) *
              (static_cast<T_ACC>(X[index1]) - static_cast<T_ACC>(mean[i1])) *
              static_cast<T_ACC>(rstd[i1]);
      db_sum1 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index1]);
      if (i2 < M) {
        dg_sum2 += dg == nullptr ? T_ACC(0)
                                 : static_cast<T_ACC>(dY[index2]) *
                (static_cast<T_ACC>(X[index2]) - static_cast<T_ACC>(mean[i2])) *
                static_cast<T_ACC>(rstd[i2]);
        db_sum2 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index2]);
      }
    }
  }
  g_shared[threadIdx.y][threadIdx.x] = dg_sum1;
  g_shared[threadIdx.y + blockDim.y][threadIdx.x] = dg_sum2;
  b_shared[threadIdx.y][threadIdx.x] = db_sum1;
  b_shared[threadIdx.y + blockDim.y][threadIdx.x] = db_sum2;
  __syncthreads();
  T_ACC sum1 = g_shared[threadIdx.x][threadIdx.y];
  T_ACC sum2 = b_shared[threadIdx.x][threadIdx.y];
  sum1 = WarpReduceSum(sum1);
  sum2 = WarpReduceSum(sum2);
  if (threadIdx.x == 0) {
    const int64_t j = blockIdx.x * blockDim.x + threadIdx.y;
    if (j < N) {
      if (dg != nullptr) {
        dg[j] = sum1;
      }
      if (db != nullptr) {
        db[j] = sum2;
      }
    }
  }
  sum1 = g_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum2 = b_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum1 = WarpReduceSum(sum1);
  sum2 = WarpReduceSum(sum2);
  if (threadIdx.x == 0) {
    const int64_t j = blockIdx.x * blockDim.x + threadIdx.y + blockDim.y;
    if (j < N) {
      if (dg != nullptr) {
        dg[j] = sum1;
      }
      if (db != nullptr) {
        db[j] = sum2;
      }
    }
  }
}

/*
  regions[0](I): output_grad
  regions[1](I): input
  regions[2](I/O): input_grad
  regions[3](I): gamma
  regions[4](I/O): gamma_grad
  regions[5](I/O): beta_grad
   */
void LayerNorm::backward_task(const Task *task,
                              const std::vector<PhysicalRegion> &regions,
                              Context ctx, Runtime *runtime) {
  const LayerNormMeta* m = *((LayerNormMeta**) task->local_args);
  assert(task->regions.size() == regions.size());
  const float *in_ptr = NULL, *out_grad_ptr = NULL, *gamma_ptr = NULL;
  float *in_grad_ptr = NULL, *gamma_grad_ptr = NULL, *beta_grad_ptr = NULL;
  Domain out_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  out_grad_ptr = helperGetTensorPointerRO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
  in_ptr = helperGetTensorPointerRO<float>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  Domain in_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[2].region.get_index_space());
  in_grad_ptr = helperGetTensorPointerRW<float>(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  assert(in_domain == out_grad_domain);
  assert(in_domain.get_volume() == m->effective_num_elements * m->effective_batch_size);
  if (m->elementwise_affine) {
    assert(regions.size() == 6);
    Domain gamma_domain = runtime->get_index_space_domain(
      ctx, task->regions[3].region.get_index_space());
    gamma_ptr = helperGetTensorPointerRO<float>(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
    Domain gamma_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[4].region.get_index_space());
    gamma_grad_ptr = helperGetTensorPointerRW<float>(
      regions[4], task->regions[4], FID_DATA, ctx, runtime);
    Domain beta_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[5].region.get_index_space());
    beta_grad_ptr = helperGetTensorPointerRW<float>(
      regions[5], task->regions[5], FID_DATA, ctx, runtime);
    assert(gamma_domain == gamma_grad_domain);
    assert(gamma_domain == beta_grad_domain);
    assert(gamma_domain.get_volume() == m->effective_num_elements);
  } else {
    assert(regions.size() == 3);
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  backward_kernel<float>(m, out_grad_ptr, in_ptr, in_grad_ptr,
      gamma_ptr, gamma_grad_ptr, beta_grad_ptr, stream);
}

template<typename T>
void LayerNorm::backward_kernel(const LayerNormMeta* m,
                                const T* output_grad_ptr,
                                const T* input_ptr,
                                T* input_grad_ptr,
                                const T* gamma_ptr,
                                T* gamma_grad_ptr,
                                T* beta_grad_ptr,
                                hipStream_t stream)
{
  const int64_t M = m->effective_batch_size;
  const int64_t N = m->effective_num_elements;
  ComputeInternalGradientsCUDAKernel<T>
      <<<M, kCUDABlockReduceNumThreads, 0, stream>>>(
          N, output_grad_ptr, input_ptr, gamma_ptr, m->ds_ptr, m->db_ptr);
  const int64_t B = (M + kCUDANumThreads - 1) / kCUDANumThreads;
  ComputeGradientFusedParamsCUDAKernel<T>
      <<<B, kCUDANumThreads, 0, stream>>>(
          M,
          N,
          m->mean_ptr,
          m->rstd_ptr,
          m->ds_ptr,
          m->db_ptr,
          m->scale_ptr,
          m->bias_ptr);
  if (gamma_grad_ptr != NULL || beta_grad_ptr != NULL) {
    if (M < 512) {
      // For small batch size, do colwise reduce directly
      const int64_t B = (N + kCUDANumThreads - 1) / kCUDANumThreads;
      GammaBetaBackwardSimpleCUDAKernel<T>
          <<<B, kCUDANumThreads, 0, stream>>>(
              M,
              N,
              output_grad_ptr,
              input_ptr,
              m->mean_ptr,
              m->rstd_ptr,
              gamma_grad_ptr,
              beta_grad_ptr);
    } else {
      const int64_t B =
          (N + kColwiseReduceTileSize - 1) / kColwiseReduceTileSize;
      constexpr int kThreadX = kColwiseReduceTileSize;
      constexpr int kThreadY = kColwiseReduceTileSize / 2;
      GammaBetaBackwardCUDAKernel<T>
          <<<B, dim3(kThreadX, kThreadY), 0, stream>>>(
              M,
              N,
              output_grad_ptr,
              input_ptr,
              m->mean_ptr,
              m->rstd_ptr,
              gamma_grad_ptr,
              beta_grad_ptr);

    }
  }
}

void LayerNorm::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    } 
    LEGION_FOREACH_N(DIMFUNC) 
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(LAYERNORM_BWD_TASK_ID, task_is,
                         TaskArgument(NULL, 0), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  // regions[0](I): output_grad
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, outputs[0].region_grad));
  launcher.add_field(0, FID_DATA);
  // regions[1](I): input
  launcher.add_region_requirement(
    RegionRequirement(input_lps[0], 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(1, FID_DATA);
  // regions[2](I/O): input_grad
  launcher.add_region_requirement(
    RegionRequirement(input_grad_lps[0], 0/*projection id*/,
      READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(2, FID_DATA);
  if (elementwise_affine) {
    // regions[3](I): gamma
    launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, weights[0].region));
    launcher.add_field(3, FID_DATA);
    // regions[4](I/O): gamma_grad
    launcher.add_region_requirement(
      RegionRequirement(weights[0].part_grad, 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, weights[0].region_grad));
    launcher.add_field(4, FID_DATA);
    // regions[5](I/O): beta_grad
    launcher.add_region_requirement(
      RegionRequirement(weights[1].part_grad, 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, weights[1].region_grad));
    launcher.add_field(5, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

bool LayerNorm::measure_operator_cost(Simulator* sim,
                                      const ParallelConfig& pc,
                                      CostMetrics& cost_metrics)
{
  return false;
}

