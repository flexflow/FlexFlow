/* Copyright 2017 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::concat(int n,
                       const Tensor* tensors,
                       int axis,
                       const char *name)
{
  Concat *cat = new Concat(*this, n, tensors, axis, name);
  layers.push_back(cat);
  return cat->outputs[0];
}

Concat::Concat(FFModel& model,
               int _n, const Tensor* _tensors,
               int _axis,
               const char* name)
: Op(model, OP_CONCAT, name, _n, _tensors), axis(_axis),
   profiling(model.config.profiling)
{
  //TODO: swich to use the Legion dim ordering
  int num_dim = inputs[0].numDim;
  outputs[0].numDim = num_dim;
  for (int i = 0; i < num_dim; i++)
    outputs[0].adim[i] = inputs[0].adim[i];
  for (int i = 1; i < numInputs; i++)
    for (int j = 0; j < num_dim; j++) {
      if (j != num_dim - 1 - axis)
        assert(inputs[i].adim[j] == outputs[0].adim[j]);
      else
        outputs[0].adim[j] += inputs[i].adim[j];
    }
  numOutputs = 1;
  numWeights = 0;
}

void Concat::create_weights(FFModel& model)
{
  // DO nothing
}

void Concat::create_output_and_partition(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = model.get_or_create_task_is(inputs[0].numDim, pcname);

  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  int dims[MAX_TENSOR_DIM], num_dim = inputs[0].numDim;
  assert(num_dim == domain.get_dim());
  for (int i = 0; i < num_dim; i++)
    dims[i] = inputs[0].adim[num_dim-1-i];
  for (int i = 1; i < numInputs; i++)
    for (int j = 0; j < num_dim; j++) {
      if (j != axis)
        assert(inputs[i].adim[num_dim-1-j] == dims[j]);
      else
        dims[j] += inputs[i].adim[num_dim-1-j];
    }
  //for (int i = 0; i < num_dim; i++)
    //printf("concat: dim[%d] = %d\n", i, dims[i]);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> part_rect = domain; \
      outputs[0] = model.create_tensor<DIM>(dims, DT_FLOAT, this); \
      outputs[0].owner_op = this; \
      outputs[0].owner_idx = 0; \
      for (int i = 0; i < numInputs; i++) { \
        Rect<DIM> input_rect = runtime->get_index_partition_color_space( \
            ctx, inputs[i].part.get_index_partition()); \
        if (input_rect == part_rect) { \
          input_lps[i] = inputs[i].part; \
          input_grad_lps[i] = inputs[i].part_grad; \
        } else { \
          model.create_disjoint_partition<DIM>(inputs[i], \
              IndexSpaceT<DIM>(task_is), input_lps[i], input_grad_lps[i]); \
        } \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
    {
      fprintf(stderr, "Unsupported concat dimension number");
      assert(false);
    }
  }
}

void Concat::init_meta(ConcatMeta *m) const
{
  m->axis = this->outputs[0].numDim - 1 - this->axis;
}

__host__
OpMeta* Concat::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  Concat* cc = (Concat*) task->args;
  FFHandler handler = *((const FFHandler*) task->local_args);
  ConcatMeta* m = new ConcatMeta(handler);
  // Note that our internal axis index ordering is opposite to other frameworks
  cc->init_meta(m);
  return m;
}

void Concat::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      ParallelConfig pc; \
      std::string pcname = name; \
      ff.config.find_parallel_config(DIM, pcname, pc); \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        FFHandler handle = ff.handlers[pc.device_ids[idx++]]; \
        argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(CONCAT_INIT_TASK_ID, task_is,
    TaskArgument(this, sizeof(Concat)), argmap,
    Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
    FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(0, FID_DATA);
  for (int i = 0; i < numInputs; i++) {
    launcher.add_region_requirement(
      RegionRequirement(input_lps[i], 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[i].region));
    launcher.add_field(i + 1, FID_DATA);
  }
  for (int i = 0; i < numInputs; i++) {
    launcher.add_region_requirement(
      RegionRequirement(input_grad_lps[i], 0/*projection id*/,
        WRITE_ONLY, EXCLUSIVE, inputs[i].region_grad));
    launcher.add_field(i + numInputs + 1, FID_DATA);
  }
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        meta[idx++] = fm.get_result<OpMeta*>(*it); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

template<int N>
void calc_blk_size(coord_t& num_blocks,
                   coord_t& blk_size,
                   Rect<N> rect,
                   int axis)
{
  num_blocks = 1;
  blk_size = 1;
  for (int d = 0; d < N; d++) {
    if (d <= axis)
      blk_size *= (rect.hi[d] - rect.lo[d] + 1);
    else
      num_blocks *= (rect.hi[d] - rect.lo[d] + 1);
  }
}

/*static*/
void Concat::forward_kernel(float* output,
                            float const * const *inputs,
                            int num_inputs,
                            int axis,
                            const Domain& out_domain,
                            const Domain* in_domain)
{
  coord_t num_blocks = 1, output_blk_size = 1, input_blk_sizes[MAX_NUM_INPUTS];
  assert(num_inputs <= MAX_NUM_INPUTS);
  switch (out_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = out_domain; \
      calc_blk_size<DIM>(num_blocks, output_blk_size, rect, axis); \
      for (int i = 0; i < num_inputs; i++) { \
        rect = in_domain[i]; \
        coord_t input_num_blocks = 1; \
        calc_blk_size<DIM>(input_num_blocks, input_blk_sizes[i], rect, axis); \
        assert(input_num_blocks == num_blocks); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      fprintf(stderr, "Unsupported concat dimension number");
      assert(false);
  }

  for (int i = 0; i < num_inputs; i++) {
    copy_with_stride<<<GET_BLOCKS(input_blk_sizes[i]*num_blocks), CUDA_NUM_THREADS>>>(
        output, inputs[i], num_blocks, output_blk_size, input_blk_sizes[i]);
    //printf("output = %x num_blocks=%d output_blk_size=%d input_blk_size[%d]=%d\n",
    //       output, num_blocks, output_blk_size, i, input_blk_sizes[i]);
    output += input_blk_sizes[i];
  }
}

/*
  regions[0](O): output
  regions[1..numInputs](I): inputs
*/
void Concat::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  const Concat* cc = (Concat*) task->args;
  // Note that our internal axis index ordering is opposite to other frameworks
  int axis = cc->outputs[0].numDim - 1 - cc->axis;
  assert(regions.size() == cc->numInputs + 1);
  assert(task->regions.size() == cc->numInputs + 1);
  Domain out_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  assert(out_domain.get_dim() == cc->outputs[0].numDim);
  Domain in_domain[MAX_NUM_INPUTS];
  for (int i = 0; i < cc->numInputs; i++)
    in_domain[i] = runtime->get_index_space_domain(
        ctx, task->regions[i+1].region.get_index_space());
  float *output = helperGetTensorPointerWO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  const float *inputs[MAX_NUM_INPUTS];
  for (int i = 0; i < cc->numInputs; i++)
    inputs[i] = helperGetTensorPointerRO<float>(
        regions[i+1], task->regions[i+1], FID_DATA, ctx, runtime);
  hipEvent_t t_start, t_end;
  if (cc->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
  forward_kernel(output, inputs, cc->numInputs, axis, out_domain, in_domain);
  if (cc->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    //print_tensor<4, float>(output - output_blk_size, output_rect, "[Concat:forward:output]");
    //printf("output_blk_size=%zu\n", output_blk_size);
    //print_tensor<4, float>(inputs[0], input_rect[0], "[Concat:forward:input0]");
    //print_tensor<4, float>(inputs[1], input_rect[1], "[Concat:forward:input1]");
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    printf("[%s] forward time = %.4f ms\n", cc->name, elapsed);
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
  }
}

void Concat::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  IndexLauncher launcher(CONCAT_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Concat)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(0, FID_DATA);
  for (int i = 0; i < numInputs; i++) {
    launcher.add_region_requirement(
      RegionRequirement(input_lps[i], 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[i].region));
    launcher.add_field(i + 1, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

void Concat::backward_kernel(const float* output_grad,
                             float** input_grads,
                             int num_inputs,
                             int axis,
                             const Domain& out_grad_domain,
                             const Domain* in_grad_domain)
{
  coord_t num_blocks = 1, output_blk_size = 1, input_blk_sizes[MAX_NUM_INPUTS];
  assert(num_inputs <= MAX_NUM_INPUTS);
  switch (out_grad_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = out_grad_domain; \
      calc_blk_size<DIM>(num_blocks, output_blk_size, rect, axis); \
      for (int i = 0; i < num_inputs; i++) { \
        rect = in_grad_domain[i]; \
        coord_t input_num_blocks = 1; \
        calc_blk_size<DIM>(input_num_blocks, input_blk_sizes[i], rect, axis); \
        assert(input_num_blocks == num_blocks); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      fprintf(stderr, "Unsupported concat dimension number");
      assert(false);
  }

  for (int i = 0; i < num_inputs; i++) {
    add_with_stride<<<GET_BLOCKS(input_blk_sizes[i]*num_blocks), CUDA_NUM_THREADS>>>(
        input_grads[i], output_grad, num_blocks, input_blk_sizes[i], output_blk_size);
    output_grad += input_blk_sizes[i];
  }

  //Rect<2> output_rect(Point<2>(0, 0), Point<2>(output_blk_size-1, batch_size - 1));
  //Rect<2> input_rect(Point<2>(0, 0), Point<2>(input_blk_sizes[0]-1, batch_size - 1));
  //print_tensor<2, float>(output_grad - output_blk_size, output_rect, "[Concat:backward:output]");
  //print_tensor<2, float>(input_grads[0], input_rect, "[Concat:backward:input0]");
}

/*
  regions[0](I): output_grad
  regions[1..numInputs](I/O): input_grad
*/
void Concat::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  const Concat* cc = (Concat*) task->args;
  // Note that our internal axis index ordering is opposite to other frameworks
  int axis = cc->outputs[0].numDim - 1 - cc->axis;
  assert(regions.size() == cc->numInputs + 1);
  assert(task->regions.size() == cc->numInputs + 1);
  assert(cc->numInputs <= MAX_NUM_INPUTS);
  Domain out_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  assert(out_grad_domain.get_dim() == cc->outputs[0].numDim);
  Domain in_grad_domains[MAX_NUM_INPUTS];
  for (int i = 0; i < cc->numInputs; i++)
    in_grad_domains[i] = runtime->get_index_space_domain(
        ctx, task->regions[i+1].region.get_index_space());
  const float *output_grad = helperGetTensorPointerRO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  float *input_grads[MAX_NUM_INPUTS];
  for (int i = 0; i < cc->numInputs; i++)
    input_grads[i] = helperGetTensorPointerRW<float>(
        regions[i+1], task->regions[i+1], FID_DATA, ctx, runtime);

  hipEvent_t t_start, t_end;
  if (cc->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
  backward_kernel(output_grad, input_grads, cc->numInputs, axis,
      out_grad_domain, in_grad_domains);
  if (cc->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    printf("[%s] forward time = %.4f ms\n", cc->name, elapsed);
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
  }
}

void Concat::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  IndexLauncher launcher(CONCAT_BWD_TASK_ID, task_is,
    TaskArgument(this, sizeof(Concat)), argmap,
    Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
    FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, outputs[0].region_grad));
  launcher.add_field(0, FID_DATA);
  for (int i = 0; i < numInputs; i++) {
    launcher.add_region_requirement(
      RegionRequirement(input_grad_lps[i], 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, inputs[i].region_grad));
    //LogicalRegion lr = inputs[i].region_grad;
    //printf("concat[%d]: region(%d,%d,%d)\n", i+1, lr.get_index_space().get_id(), lr.get_field_space().get_id(), lr.get_tree_id());
    launcher.add_field(i + 1, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}


bool Concat::measure_compute_time(Simulator* sim,
                                  const ParallelConfig& pc,
                                  float& forward_time,
                                  float& backward_time)
{
  assert (numInputs <= MAX_NUM_INPUTS);
  Tensor sub_inputs[MAX_NUM_INPUTS], sub_output;
  if (!outputs[0].get_output_sub_tensor(pc, sub_output, op_type)) {
    return false;
  }
  for (int i = 0; i < numInputs; i++) {
    if (!inputs[i].get_input_sub_tensor(pc, sub_inputs[i], op_type)) {
      return false;
    }
  }

  ConcatMeta *m = sim->concat_meta;
  this->init_meta(m);

  sim->free_all();
  float *input_ptrs[MAX_NUM_INPUTS];
  float *input_grad_ptrs[MAX_NUM_INPUTS];
  for (int i = 0; i < numInputs; i++) {
    input_ptrs[i] = (float *)sim->allocate(sub_inputs[i].get_volume(), DT_FLOAT);
    assert (input_ptrs[i] != NULL);
    input_grad_ptrs[i] = (float *)sim->allocate(sub_inputs[i].get_volume(), DT_FLOAT);
    assert (input_grad_ptrs[i] != NULL);
  }
  float *output_ptr = (float *)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert (output_ptr != NULL);
  float *output_grad_ptr = (float *)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert (output_grad_ptr != NULL);

  int axis = outputs[0].numDim - 1 - this->axis;

  Domain out_domain = sub_output.get_domain();
  Domain in_domains[MAX_NUM_INPUTS];
  for (int i = 0; i < numInputs; i++) {
    in_domains[i] = sub_inputs[i].get_domain();
  }

  auto forward = [&] {
    forward_kernel(output_ptr, input_ptrs, numInputs, axis, out_domain, in_domains);
  };
  auto backward = [&] {
    backward_kernel(output_grad_ptr, input_grad_ptrs, numInputs, axis, out_domain, in_domains);
  };

  inner_measure_compute_time(sim, forward, backward, forward_time, backward_time);

  printf("[Measure Concat] name(%s) forward_time(%.4lf) backward_time(%.4lf)\n",
      name,
      forward_time,
      backward_time);

  return true;
}
