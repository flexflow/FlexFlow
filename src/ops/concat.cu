/* Copyright 2017 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/concat.h"
#include "flexflow/utils/hash_utils.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

// declare Legion names
using Legion::Context;
using Legion::Runtime;
using Legion::Domain;
using Legion::Task;
using Legion::Rect;
using Legion::PhysicalRegion;
using Legion::coord_t;

__host__
OpMeta* Concat::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  Concat* cc = (Concat*) task->args;
  FFHandler handler = *((const FFHandler*) task->local_args);
  ConcatMeta* m = new ConcatMeta(handler);
  // Note that our internal axis index ordering is opposite to other frameworks
  cc->init_meta(m);
  m->profiling = cc->profiling;
  return m;
}

template<int N>
void calc_blk_size(coord_t& num_blocks,
                   coord_t& blk_size,
                   Rect<N> rect,
                   int axis)
{
  num_blocks = 1;
  blk_size = 1;
  for (int d = 0; d < N; d++) {
    if (d <= axis)
      blk_size *= (rect.hi[d] - rect.lo[d] + 1);
    else
      num_blocks *= (rect.hi[d] - rect.lo[d] + 1);
  }
}

/*static*/
void Concat::forward_kernel(float* output,
                            float const * const *inputs,
                            int num_inputs,
                            int axis,
                            const Domain& out_domain,
                            const Domain* in_domain,
                            hipStream_t stream)
{
  coord_t num_blocks = 1, output_blk_size = 1, input_blk_sizes[MAX_NUM_INPUTS];
  assert(num_inputs <= MAX_NUM_INPUTS);
  switch (out_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = out_domain; \
      calc_blk_size<DIM>(num_blocks, output_blk_size, rect, axis); \
      for (int i = 0; i < num_inputs; i++) { \
        rect = in_domain[i]; \
        coord_t input_num_blocks = 1; \
        calc_blk_size<DIM>(input_num_blocks, input_blk_sizes[i], rect, axis); \
        assert(input_num_blocks == num_blocks); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      fprintf(stderr, "Unsupported concat dimension number");
      assert(false);
  }

  for (int i = 0; i < num_inputs; i++) {
    copy_with_stride<<<GET_BLOCKS(input_blk_sizes[i]*num_blocks), CUDA_NUM_THREADS, 0, stream>>>(
        output, inputs[i], num_blocks, output_blk_size, input_blk_sizes[i]);
    //printf("output = %x num_blocks=%d output_blk_size=%d input_blk_size[%d]=%d\n",
    //       output, num_blocks, output_blk_size, i, input_blk_sizes[i]);
    output += input_blk_sizes[i];
  }
}

/*
  regions[0](O): output
  regions[1..numInputs](I): inputs
*/
void Concat::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  const Concat* cc = (Concat*) task->args;
  // Note that our internal axis index ordering is opposite to other frameworks
  assert(regions.size() == cc->numInputs + 1);
  assert(task->regions.size() == cc->numInputs + 1);
  Domain out_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  //assert(out_domain.get_dim() == cc->outputs[0].num_dims);
  Domain in_domain[MAX_NUM_INPUTS];
  for (int i = 0; i < cc->numInputs; i++)
    in_domain[i] = runtime->get_index_space_domain(
        ctx, task->regions[i+1].region.get_index_space());
  float *output = helperGetTensorPointerWO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  const float *inputs[MAX_NUM_INPUTS];
  for (int i = 0; i < cc->numInputs; i++)
    inputs[i] = helperGetTensorPointerRO<float>(
        regions[i+1], task->regions[i+1], FID_DATA, ctx, runtime);
  
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  
  hipEvent_t t_start, t_end;
  if (cc->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  forward_kernel(output, inputs, cc->numInputs, cc->axis, out_domain, in_domain, stream);
  if (cc->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    //print_tensor<4, float>(output - output_blk_size, output_rect, "[Concat:forward:output]");
    //printf("output_blk_size=%zu\n", output_blk_size);
    //print_tensor<4, float>(inputs[0], input_rect[0], "[Concat:forward:input0]");
    //print_tensor<4, float>(inputs[1], input_rect[1], "[Concat:forward:input1]");
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    printf("[%s] forward time = %.4f ms\n", cc->name, elapsed);
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
  }
}

void Concat::backward_kernel(const float* output_grad,
                             float** input_grads,
                             int num_inputs,
                             int axis,
                             const Domain& out_grad_domain,
                             const Domain* in_grad_domain,
                             hipStream_t stream)
{
  coord_t num_blocks = 1, output_blk_size = 1, input_blk_sizes[MAX_NUM_INPUTS];
  assert(num_inputs <= MAX_NUM_INPUTS);
  switch (out_grad_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = out_grad_domain; \
      calc_blk_size<DIM>(num_blocks, output_blk_size, rect, axis); \
      for (int i = 0; i < num_inputs; i++) { \
        rect = in_grad_domain[i]; \
        coord_t input_num_blocks = 1; \
        calc_blk_size<DIM>(input_num_blocks, input_blk_sizes[i], rect, axis); \
        assert(input_num_blocks == num_blocks); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      fprintf(stderr, "Unsupported concat dimension number");
      assert(false);
  }

  for (int i = 0; i < num_inputs; i++) {
    add_with_stride<<<GET_BLOCKS(input_blk_sizes[i]*num_blocks), CUDA_NUM_THREADS, 0, stream>>>(
        input_grads[i], output_grad, num_blocks, input_blk_sizes[i], output_blk_size);
    output_grad += input_blk_sizes[i];
  }

  //Rect<2> output_rect(Point<2>(0, 0), Point<2>(output_blk_size-1, batch_size - 1));
  //Rect<2> input_rect(Point<2>(0, 0), Point<2>(input_blk_sizes[0]-1, batch_size - 1));
  //print_tensor<2, float>(output_grad - output_blk_size, output_rect, "[Concat:backward:output]");
  //print_tensor<2, float>(input_grads[0], input_rect, "[Concat:backward:input0]");
}

/*
  regions[0](I): output_grad
  regions[1..numInputs](I/O): input_grad
*/
void Concat::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  const Concat* cc = (Concat*) task->args;
  // Note that our internal axis index ordering is opposite to other frameworks
  assert(regions.size() == cc->numInputs + 1);
  assert(task->regions.size() == cc->numInputs + 1);
  assert(cc->numInputs <= MAX_NUM_INPUTS);
  Domain out_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  //assert(out_grad_domain.get_dim() == cc->outputs[0].num_dims);
  Domain in_grad_domains[MAX_NUM_INPUTS];
  for (int i = 0; i < cc->numInputs; i++)
    in_grad_domains[i] = runtime->get_index_space_domain(
        ctx, task->regions[i+1].region.get_index_space());
  const float *output_grad = helperGetTensorPointerRO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  float *input_grads[MAX_NUM_INPUTS];
  for (int i = 0; i < cc->numInputs; i++)
    input_grads[i] = helperGetTensorPointerRW<float>(
        regions[i+1], task->regions[i+1], FID_DATA, ctx, runtime);

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  
  hipEvent_t t_start, t_end;
  if (cc->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  backward_kernel(output_grad, input_grads, cc->numInputs, cc->axis,
      out_grad_domain, in_grad_domains, stream);
  if (cc->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    printf("[%s] forward time = %.4f ms\n", cc->name, elapsed);
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
  }
}

bool Concat::measure_operator_cost(Simulator* sim,
                                   const ParallelConfig& pc,
                                   CostMetrics& cost_metrics) const
{
  assert (numInputs <= MAX_NUM_INPUTS);
  TensorBase sub_inputs[MAX_NUM_INPUTS], sub_output;
  if (!outputs[0]->get_output_sub_tensor(pc, sub_output, op_type)) {
    return false;
  }
  for (int i = 0; i < numInputs; i++) {
    if (!inputs[i]->get_input_sub_tensor(pc, sub_inputs[i], op_type)) {
      return false;
    }
  }

  ConcatMeta *m = sim->concat_meta;
  this->init_meta(m);

  sim->free_all();
  float *input_ptrs[MAX_NUM_INPUTS];
  float *input_grad_ptrs[MAX_NUM_INPUTS];
  bool out_of_memory = false;
  for (int i = 0; i < numInputs; i++) {
    input_ptrs[i] = (float *)sim->allocate(sub_inputs[i].get_volume(), DT_FLOAT);
    out_of_memory = out_of_memory || (input_ptrs[i] == NULL);
  }
  float *output_ptr = (float *)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  out_of_memory = out_of_memory || (output_ptr == NULL);
  if (out_of_memory) {
    cost_metrics.forward_time = Simulator::MAXIMUM_TASK_RUN_TIME;
    cost_metrics.backward_time = Simulator::MAXIMUM_TASK_RUN_TIME;
    return true;
  }

  Domain out_domain = sub_output.get_domain();
  Domain in_domains[MAX_NUM_INPUTS];
  for (int i = 0; i < numInputs; i++) {
    in_domains[i] = sub_inputs[i].get_domain();
  }
  
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  std::function<void()> forward, backward;
  forward = [&] {
    forward_kernel(output_ptr, input_ptrs, numInputs, axis, out_domain, in_domains, stream);
  };
  if (sim->computationMode == COMP_MODE_TRAINING) {
    for (int i = 0; i < numInputs; i++) {
      input_grad_ptrs[i] = (float *)sim->allocate(sub_inputs[i].get_volume(), DT_FLOAT);
      out_of_memory = out_of_memory || (input_grad_ptrs[i] == NULL);
    }
    float *output_grad_ptr = (float *)sim->allocate(sub_output.get_volume(), DT_FLOAT);
    out_of_memory = out_of_memory || (output_grad_ptr == NULL);
    if (out_of_memory) {
      cost_metrics.forward_time = Simulator::MAXIMUM_TASK_RUN_TIME;
      cost_metrics.backward_time = Simulator::MAXIMUM_TASK_RUN_TIME;
      return true;
    }
    backward = [&] {
      backward_kernel(output_grad_ptr, input_grad_ptrs,
        numInputs, axis, out_domain, in_domains, stream);
    };
  }

  inner_measure_operator_cost(sim, forward, backward, cost_metrics);

  if (sim->computationMode == COMP_MODE_TRAINING) {
    printf("[Measure Concat] name(%s) forward_time(%.4lf) backward_time(%.4lf)\n",
        name,
        cost_metrics.forward_time,
        cost_metrics.backward_time);
  } else {
    printf("[Measure Concat] name(%s) forward_time(%.4lf)\n",
        name, cost_metrics.forward_time);
  }

  return true;
}

}; // namespace FlexFlow
