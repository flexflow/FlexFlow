/* Copyright 2020 Facebook
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

using namespace Legion;

Tensor FFModel::batch_matmul(const Tensor A,
                             const Tensor B,
                             int a_seq_length_dim,
                             int b_seq_length_dim)
{
  BatchMatmul* bmm = new BatchMatmul(*this, A, B,
      a_seq_length_dim, b_seq_length_dim);
  layers.push_back(bmm);
  return bmm->outputs[0];
}

// return A*B
BatchMatmul::BatchMatmul(FFModel& model,
                         const Tensor A,
                         const Tensor B,
                         int _a_seq_length_dim,
                         int _b_seq_length_dim)
: Op(model, OP_BATCHMATMUL, "BatchMatmul_", 2/*inputs*/, 0/*weights*/, A, B),
  a_seq_length_dim(A->num_dims-1-_a_seq_length_dim),
  b_seq_length_dim(B->num_dims-1-_b_seq_length_dim)
{
  assert((a_seq_length_dim <= 1) && "FlexFlow currently only supports seq_length_dim of 0 or 1 (in Fortran ordering).");
  assert((b_seq_length_dim <= 1) && "FlexFlow currently only supports seq_length_dim of 0 or 1 (in Fortran ordering).");
  assert(A->num_dims == B->num_dims);
  for (int i = A->num_dims-1; i >= 2; i--)
    assert(A->dims[i] == B->dims[i]);
  assert(A->dims[0] == B->dims[1]);
  ParallelDim dims[MAX_TENSOR_DIM];
  for (int i = 0; i < A->num_dims; i++)
    dims[i] = A->dims[i];
  dims[0] = B->dims[0];
  numOutputs = 1;
  outputs[0] = model.create_tensor_legion_ordering(A->num_dims, dims, DT_FLOAT, this);
  // C is not none
  //if (C != Tensor::NO_TENSOR) {
  //  numInputs = 3;
  //  assert(C.num_dims == outputs[0].num_dims);
  //  for (int i = 0; i < C.num_dims; i++)
  //    assert(C.adim[i] == outputs[0].adim[i]);
  //}
}

#ifdef DEADCODE
void BatchMatmul::create_input_partition(FFModel& model)
{
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Domain part_rect = runtime->get_index_space_domain(ctx, task_is);
  // currently only support data parallel for batch matmul
  // the parallel degree of the inner most two dims must be 1
  assert(part_rect.hi()[0] == part_rect.lo()[0]);
  assert(part_rect.hi()[1] == part_rect.lo()[1]);
  return Op::create_input_partition(model);
#ifdef DEADCODE
  int dims[NDIM];
  for (int i = 0; i < NDIM; i++)
    dims[i] = outputs[0].adim[NDIM-1-i];
  outputs[0] = model.create_tensor<NDIM>(dims, DT_FLOAT, this);
  outputs[0].owner_op = this;
  outputs[0].owner_idx = 0;
  for (int i = 0; i < numInputs; i++) {
    Rect<NDIM> input_rect = runtime->get_index_partition_color_space(
        ctx, inputs[i]->part.get_index_partition());
    if (input_rect == part_rect) {
      input_lps[i] = inputs[i]->part;
      input_grad_lps[i] = inputs[i]->part_grad;
    } else {
      model.create_disjoint_partition<NDIM>(
          inputs[i], IndexSpaceT<NDIM>(task_is), input_lps[i], input_grad_lps[i]);
    }
  }
#endif
}
#endif

__host__
OpMeta* BatchMatmul::init_task(const Task* task,
                               const std::vector<PhysicalRegion>& regions,
                               Context ctx, Runtime* runtime)
{
  const BatchMatmul* bmm = (BatchMatmul*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  BatchMatmulMeta* m = new BatchMatmulMeta(handle);
  m->profiling = bmm->profiling;
  m->a_seq_length_dim = bmm->a_seq_length_dim;
  m->b_seq_length_dim = bmm->b_seq_length_dim;
  return m;
}

void BatchMatmul::init(const FFModel& ff)
{
  int dim = outputs[0]->num_dims;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      init_with_dim<DIM>(ff); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

template<int NDIM>
void BatchMatmul::init_with_dim(const FFModel& ff)
{
  assert(check_output_input_weight_same_parallel_is());
  parallel_is = outputs[0]->parallel_is;
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_init(ff, argmap);
  IndexLauncher launcher(BATCHMATMUL_INIT_TASK_ID, parallel_is,
                         TaskArgument(this, sizeof(BatchMatmul)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(outputs[0]->part, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, outputs[0]->region));
  launcher.add_field(0, FID_DATA);
  for (int i = 0; i < numInputs; i++) {
    launcher.add_region_requirement(
      RegionRequirement(inputs[i]->part, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[i]->region));
    launcher.add_field(i+1, FID_DATA);
  }
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  set_opmeta_from_futuremap(ff, fm);
}

/*
A: (batch, n, k)
B: (batch, k, m)
O: (batch, n, m)
O = A * B
*/
void BatchMatmul::forward_kernel(const BatchMatmulMeta* meta,
                                 float* o_ptr,
                                 const float* a_ptr,
                                 const float* b_ptr,
                                 const float* c_ptr,
                                 int m, int n, int k,
                                 int batch,
                                 int a_seq_length_dim,
                                 int b_seq_length_dim,
                                 int seq_length)
{
  //int a_stride = n * k;
  //int b_stride = m * k;
  //int o_stride = n * m;
  int lda = k; int ldb = m; int ldo = m;
  long long int strideA = (long long int)n*k;
  long long int strideB = (long long int)k*m;
  long long int strideO = (long long int)n*m;
  if ((a_seq_length_dim==0)&&(seq_length>=0)) {
    assert(seq_length <= k);
    k = seq_length;
    assert(b_seq_length_dim == 1);
  } else if ((a_seq_length_dim==1)&&(seq_length>=0)) {
    assert(seq_length <= n);
    n = seq_length;
  } else {
    // currently only support a_seq_length_dim = 0 or 1
    assert((a_seq_length_dim<0)||(seq_length<0));
  }
  if ((b_seq_length_dim==0)&&(seq_length>=0)) {
    assert(seq_length <= m);
    m = seq_length;
  } else if ((b_seq_length_dim==1)&&(seq_length>=0)) {
    assert(a_seq_length_dim == 0);
    assert(k == seq_length);
  } else {
    // currently only support a_seq_length_dim = 0 or 1
    assert((b_seq_length_dim<0)||(seq_length<0));
  }

  float alpha = 1.0f, beta = 0.0f;
  checkCUDA(hipblasSgemmStridedBatched(meta->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_N,
      m, n, k, &alpha, b_ptr, ldb, strideB, a_ptr, lda, strideA,
      &beta, o_ptr, ldo, strideO, batch));
  // current assume c is null
  assert(c_ptr == NULL);
}

/*
  regions[0](O): output
  regions[1](I): A
  regions[2](I): B
  (optional) regions[3](I): C
  output = A * B + C
*/
__host__
void BatchMatmul::forward_task(const Task* task,
                               const std::vector<PhysicalRegion>& regions,
                               Context ctx, Runtime* runtime)
{
  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  //const BatchMatmul* bmm = (const BatchMatmul*) task->args;
  const FFIterationConfig* iter_config = (const FFIterationConfig*) task->args;
  const BatchMatmulMeta* meta = *((BatchMatmulMeta**) task->local_args);
  Domain out_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain a_domain = runtime->get_index_space_domain(
    ctx, task->regions[1].region.get_index_space());
  Domain b_domain = runtime->get_index_space_domain(
    ctx, task->regions[2].region.get_index_space());
  int m = b_domain.hi()[0] - b_domain.lo()[0] + 1;
  assert(m == out_domain.hi()[0] - out_domain.lo()[0] + 1);
  int n = a_domain.hi()[1] - a_domain.lo()[1] + 1;
  assert(n == out_domain.hi()[1] - out_domain.lo()[1] + 1);
  int k = a_domain.hi()[0] - a_domain.lo()[0] + 1;
  assert(k == b_domain.hi()[1] - b_domain.lo()[1] + 1);
  assert(a_domain.get_dim() == b_domain.get_dim());
  assert(a_domain.get_dim() == out_domain.get_dim());
  int batch = 1;
  for (int i = 2; i < a_domain.get_dim(); i++) {
    int dim_size = a_domain.hi()[i] - a_domain.lo()[i] + 1;
    assert(dim_size == b_domain.hi()[i] - b_domain.lo()[i] + 1);
    assert(dim_size == out_domain.hi()[i] - out_domain.lo()[i] + 1);
    batch *= dim_size;
  }
  float* out_ptr = helperGetTensorPointerWO<float>(
    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  const float* a_ptr = helperGetTensorPointerRO<float>(
    regions[1], task->regions[1], FID_DATA, ctx, runtime);
  const float* b_ptr = helperGetTensorPointerRO<float>(
    regions[2], task->regions[2], FID_DATA, ctx, runtime);
  const float* c_ptr = NULL;
  if (regions.size() == 4) {
    Domain c_domain = runtime->get_index_space_domain(
      ctx, task->regions[3].region.get_index_space());
    assert(c_domain == a_domain);
    c_ptr = helperGetTensorPointerRO<float>(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
  }
  hipEvent_t t_start, t_end;
  if (meta->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(meta->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(meta->handle.dnn, stream));
#endif
  forward_kernel(meta, out_ptr, a_ptr, b_ptr, c_ptr,
    m, n, k, batch, meta->a_seq_length_dim, meta->b_seq_length_dim,
    iter_config->seq_length);
  if (meta->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("BatchMatmul forward time = %.2lfms\n", elapsed);
  }
}

void BatchMatmul::forward(const FFModel& ff)
{
  int dim = outputs[0]->num_dims;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      forward_with_dim<DIM>(ff); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

template<int NDIM>
void BatchMatmul::forward_with_dim(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_forward(ff, argmap);
  IndexLauncher launcher(BATCHMATMUL_FWD_TASK_ID, parallel_is,
      TaskArgument(&ff.iter_config, sizeof(FFIterationConfig)), argmap,
      Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
      FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(outputs[0]->part, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, outputs[0]->region));
  launcher.add_field(0, FID_DATA);
  for (int i = 0; i < numInputs; i++) {
    launcher.add_region_requirement(
      RegionRequirement(inputs[i]->part, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[i]->region));
    launcher.add_field(i+1, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

/*
A, AGrad: (batch, n, k)
B, BGrad: (batch, k, m)
O, OGrad: (batch, n, m)
AGrad = OGrad * B^T
BGrad = A^T * OGrad
*/
void BatchMatmul::backward_kernel(const BatchMatmulMeta* meta,
                                  const float* o_ptr,
                                  const float* o_grad_ptr,
                                  const float* a_ptr,
                                  float* a_grad_ptr,
                                  const float* b_ptr,
                                  float* b_grad_ptr,
                                  float* c_grad_ptr,
                                  int m, int n, int k, int batch)
{
  int a_stride = n * k;
  int b_stride = m * k;
  int o_stride = n * m;
  float alpha = 1.0f;
  checkCUDA(hipblasSgemmStridedBatched(meta->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
      k, n, m, &alpha, b_ptr, m, b_stride, o_grad_ptr, m, o_stride,
      &alpha, a_grad_ptr, k, a_stride, batch));
  checkCUDA(hipblasSgemmStridedBatched(meta->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_T,
      m, k, n, &alpha, o_grad_ptr, m, o_stride, a_ptr, k, a_stride,
      &alpha, b_grad_ptr, m, b_stride, batch));
  assert (c_grad_ptr == NULL);
}


/*
  regions[0](I): output
  regions[1](I): output_grad
  regions[2](I): A
  regions[3](I/O): A_grad
  regions[4](I): B
  regions[5](I/O): B_grad
  regions[6](I/O): C_grad
*/
__host__
void BatchMatmul::backward_task(const Task *task,
                                const std::vector<PhysicalRegion> &regions,
                                Context ctx, Runtime *runtime)
{
  // Currently assume C is NULL
  assert(regions.size() == 6);
  assert(task->regions.size() == 6);
  //BatchMatmul* bmm = (BatchMatmul*) task->args;
  const FFIterationConfig* iter_config = (const FFIterationConfig*) task->args;
  const BatchMatmulMeta* meta = *((BatchMatmulMeta**) task->local_args);
  // output domains
  Domain out_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain out_grad_domain = runtime->get_index_space_domain(
    ctx, task->regions[1].region.get_index_space());
  assert(out_domain == out_grad_domain);
  // A domains
  Domain a_domain = runtime->get_index_space_domain(
    ctx, task->regions[2].region.get_index_space());
  Domain a_grad_domain = runtime->get_index_space_domain(
    ctx, task->regions[3].region.get_index_space());
  assert(a_domain == a_grad_domain);
  // B domains
  Domain b_domain = runtime->get_index_space_domain(
    ctx, task->regions[4].region.get_index_space());
  Domain b_grad_domain = runtime->get_index_space_domain(
    ctx, task->regions[4].region.get_index_space());
  assert(b_domain == b_grad_domain);
  // check dins
  int m = b_domain.hi()[0] - b_domain.lo()[0] + 1;
  assert(m == out_domain.hi()[0] - out_domain.lo()[0] + 1);
  int n = a_domain.hi()[1] - a_domain.lo()[1] + 1;
  assert(n == out_domain.hi()[1] - out_domain.lo()[1] + 1);
  int k = a_domain.hi()[0] - a_domain.lo()[0] + 1;
  assert(k == b_domain.hi()[1] - b_domain.lo()[1] + 1);
  assert(a_domain.get_dim() == b_domain.get_dim());
  assert(a_domain.get_dim() == out_domain.get_dim());
  int batch = 1;
  for (int i = 2; i < a_domain.get_dim(); i++) {
    int dim_size = a_domain.hi()[i] - a_domain.lo()[i] + 1;
    assert(dim_size == b_domain.hi()[i] - b_domain.lo()[i] + 1);
    assert(dim_size == out_domain.hi()[i] - out_domain.lo()[i] + 1);
    batch *= dim_size;
  }
  // get pointers
  const float* out_ptr = helperGetTensorPointerRO<float>(
    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  const float* out_grad_ptr = helperGetTensorPointerRO<float>(
    regions[1], task->regions[1], FID_DATA, ctx, runtime);
  const float* a_ptr = helperGetTensorPointerRO<float>(
    regions[2], task->regions[2], FID_DATA, ctx, runtime);
  float* a_grad_ptr = helperGetTensorPointerRW<float>(
    regions[3], task->regions[3], FID_DATA, ctx, runtime);
  const float* b_ptr = helperGetTensorPointerRO<float>(
    regions[4], task->regions[4], FID_DATA, ctx, runtime);
  float* b_grad_ptr = helperGetTensorPointerRW<float>(
    regions[5], task->regions[5], FID_DATA, ctx, runtime);

  float* c_grad_ptr = NULL;
  hipEvent_t t_start, t_end;
  if (meta->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(meta->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(meta->handle.dnn, stream));
#endif
  // TODO: add support for meta->a_seq_length_dim >= 0
  // or meta->b_seq_length_dim >= 0
  assert((meta->a_seq_length_dim<0)||(iter_config->seq_length==0));
  assert((meta->b_seq_length_dim<0)||(iter_config->seq_length==0));
  backward_kernel(meta, out_ptr, out_grad_ptr, a_ptr, a_grad_ptr,
    b_ptr, b_grad_ptr, c_grad_ptr, m, n, k, batch);
  if (meta->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("BatchMatmul backward time = %.2lfms\n", elapsed);
  }
}

void BatchMatmul::backward(const FFModel& ff)
{
  int dim = outputs[0]->num_dims;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      backward_with_dim<DIM>(ff); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

/*
  regions[0](I): output
  regions[1](I): output_grad
  regions[2](I): A
  regions[3](I/O): A_grad
  regions[4](I): B
  regions[5](I/O): B_grad
  regions[6](I/O): C_grad
*/
template<int NDIM>
void BatchMatmul::backward_with_dim(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_backward(ff, argmap);
  IndexLauncher launcher(BATCHMATMUL_BWD_TASK_ID, parallel_is,
      TaskArgument(&ff.iter_config, sizeof(FFIterationConfig)), argmap,
      Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
      FFConfig::get_hash_id(std::string(name)));
  // regions[0](I): output
  launcher.add_region_requirement(
    RegionRequirement(outputs[0]->part, 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, outputs[0]->region));
  launcher.add_field(0, FID_DATA);
  // regions[1](I): output_grad
  launcher.add_region_requirement(
    RegionRequirement(outputs[0]->part_grad, 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, outputs[0]->region_grad));
  launcher.add_field(1, FID_DATA);
  // regions[2](I): A
  launcher.add_region_requirement(
    RegionRequirement(inputs[0]->part, 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, inputs[0]->region));
  launcher.add_field(2, FID_DATA);
  // regions[3](I/O): A_grad
  launcher.add_region_requirement(
    RegionRequirement(inputs[0]->part_grad, 0/*projection id*/,
      READ_WRITE, EXCLUSIVE, inputs[0]->region_grad));
  launcher.add_field(3, FID_DATA);
  // regions[4](I): B
  launcher.add_region_requirement(
    RegionRequirement(inputs[1]->part, 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, inputs[1]->region));
  launcher.add_field(4, FID_DATA);
  // regions[5](I/O): B_grad
  launcher.add_region_requirement(
    RegionRequirement(inputs[1]->part_grad, 0/*projection id*/,
      READ_WRITE, EXCLUSIVE, inputs[1]->region_grad));
  launcher.add_field(5, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

__host__
void BatchMatmul::print_layer(const FFModel& ff)
{
  return;
}

BatchMatmulMeta::BatchMatmulMeta(FFHandler handler)
: OpMeta(handler)
{}

bool BatchMatmul::measure_operator_cost(Simulator* sim,
                                        const ParallelConfig& pc,
                                        CostMetrics& cost_metrics) const
{
  TensorBase sub_output, sub_input0, sub_input1;
  if (! outputs[0]->get_output_sub_tensor(pc, sub_output, OP_BATCHMATMUL)) {
    return false;
  }
  if (! inputs[0]->get_input_sub_tensor(pc, sub_input0, OP_BATCHMATMUL)) {
    return false;
  }
  if (! inputs[1]->get_input_sub_tensor(pc, sub_input1, OP_BATCHMATMUL)) {
    return false;
  }

  int input0_c = sub_input0.dims[0].size;
  int input0_r = sub_input0.dims[1].size;
  int input1_c = sub_input1.dims[0].size;
  int input1_r = sub_input1.dims[1].size;
  int output_c = sub_output.dims[0].size;
  int output_r = sub_output.dims[1].size;

  assert (input0_c == input1_r);
  assert (input0_r == output_r);
  assert (input1_c == output_c);

  assert (sub_input0.dims[2] == sub_input1.dims[2]);
  assert (sub_input1.dims[2] == sub_output.dims[2]);
  int batch = 1;
  assert(sub_input0.num_dims == sub_input1.num_dims);
  for (int i = 2; i < sub_input0.num_dims; i++) {
    assert(sub_input0.dims[i] == sub_input1.dims[i]);
    assert(sub_input0.dims[i] == sub_output.dims[i]);
    batch *= sub_input0.dims[i].size;
  }

  BatchMatmulMeta *meta = sim->batch_matmul_meta;

  // allocate tensors in simulator
  sim->free_all();
  float *a_ptr = (float *)sim->allocate(sub_input0.get_volume(), DT_FLOAT);
  assert (a_ptr != NULL);
  float *b_ptr = (float *)sim->allocate(sub_input1.get_volume(), DT_FLOAT);
  assert (b_ptr != NULL);
  float *c_ptr = NULL;
  float *out_ptr = (float *)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert (out_ptr != NULL);

  int m = input1_c;
  int n = input0_r;
  int k = input0_c;

  std::function<void()> forward, backward;
  forward = [&] {
    forward_kernel(meta, out_ptr, a_ptr, b_ptr, c_ptr, m, n, k, batch);
  };

  if (sim->computationMode == COMP_MODE_TRAINING) {
    float *a_grad_ptr = (float *)sim->allocate(sub_input0.get_volume(), DT_FLOAT);
    float *b_grad_ptr = (float *)sim->allocate(sub_input1.get_volume(), DT_FLOAT);
    float *c_grad_ptr = NULL;
    float *out_grad_ptr = (float *)sim->allocate(sub_output.get_volume(), DT_FLOAT);
    assert (out_grad_ptr != NULL);

    backward = [&] {
      backward_kernel(meta, out_ptr, out_grad_ptr, a_ptr, a_grad_ptr, b_ptr, b_grad_ptr, c_grad_ptr, m, n, k, batch);
    };
  }

  inner_measure_operator_cost(sim, forward, backward, cost_metrics);

  if (sim->computationMode == COMP_MODE_TRAINING) {
    printf("[Measure BatchMatmul] name(%s) adim(%d %d %d) bdim(%d %d %d) odim(%d %d %d) forward_time(%.4lf) backward_time(%.4lf)\n",
        name,
        batch, input0_r, input0_c,
        batch, input1_r, input1_c,
        batch, output_r, output_c,
        cost_metrics.forward_time,
        cost_metrics.backward_time);
  } else {
    printf("[Measure BatchMatmul] name(%s) adim(%d %d %d) bdim(%d %d %d) odim(%d %d %d) forward_time(%.4lf)\n",
        name,
        batch, input0_r, input0_c,
        batch, input1_r, input1_c,
        batch, output_r, output_c,
        cost_metrics.forward_time);
  }

  return true;
}
