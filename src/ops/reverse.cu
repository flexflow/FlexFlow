#include "hip/hip_runtime.h"
/* Copyright 2020 Facebook
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/reverse.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {
// declare Legion names
using Legion::Context;
using Legion::Runtime;
using Legion::Domain;
using Legion::Task;
using Legion::Rect;
using Legion::PhysicalRegion;
using Legion::coord_t;

__host__
OpMeta* Reverse::init_task(const Task* task,
                           const std::vector<PhysicalRegion>& regions,
                           Context ctx, Runtime* runtime)
{
  return NULL;
}

__global__
void reverse_forward_kernel(const float* in_ptr,
                            float* out_ptr,
                            coord_t num_out_blks,
                            coord_t reverse_dim_size,
                            coord_t in_blk_size)
{
  CUDA_KERNEL_LOOP(i, num_out_blks * reverse_dim_size * in_blk_size)
  {
    coord_t blk_idx = i / (reverse_dim_size * in_blk_size);
    i = i - blk_idx * (reverse_dim_size * in_blk_size);
    coord_t reverse_dim_idx = i / in_blk_size;
    i = i - reverse_dim_idx * in_blk_size;
    coord_t in_idx = blk_idx * (reverse_dim_size * in_blk_size)
                   + (reverse_dim_size - 1 - reverse_dim_idx) * in_blk_size + i;
    out_ptr[i] = in_ptr[in_idx];
  }
}

void Reverse::forward_kernel(float const *in_ptr,
                             float *out_ptr,
                             coord_t num_out_blks,
                             coord_t reverse_dim_size,
                             coord_t in_blk_size,
                             coord_t output_size,
                             hipStream_t stream)
{
  reverse_forward_kernel<<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
      in_ptr, out_ptr, num_out_blks, reverse_dim_size, in_blk_size);
}


__host__
void Reverse::forward_task(const Task* task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime* runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  const Reverse* reverse = (const Reverse*) task->args;
  Domain in_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain out_domain = runtime->get_index_space_domain(
    ctx, task->regions[1].region.get_index_space());
  assert(out_domain == in_domain);
  const float* in_ptr = helperGetTensorPointerRO<float>(
    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  float* out_ptr = helperGetTensorPointerWO<float>(
    regions[1], task->regions[1], FID_DATA, ctx, runtime);
  int axis = in_domain.get_dim() - reverse->axis - 1;
  coord_t in_blk_size = 1, reverse_dim_size = 1, num_out_blks = 1;
  for (int i = 0; i < out_domain.get_dim(); i++) {
    if (i < axis)
      in_blk_size *= out_domain.hi()[i] - out_domain.lo()[i] + 1;
    else if (i == axis)
      reverse_dim_size = out_domain.hi()[i] - out_domain.lo()[i] + 1;
    else
      num_out_blks *= out_domain.hi()[i] - out_domain.lo()[i] + 1;
  }
  int output_size = out_domain.get_volume();

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  forward_kernel(in_ptr, out_ptr, num_out_blks, reverse_dim_size, in_blk_size, output_size, stream);
}

void Reverse::backward_kernel(float const *out_grad_ptr,
                              float *in_grad_ptr,
                              coord_t num_out_blks,
                              coord_t reverse_dim_size,
                              coord_t in_blk_size,
                              coord_t input_size,
                              hipStream_t stream)
{
  reverse_forward_kernel<<<GET_BLOCKS(input_size), CUDA_NUM_THREADS, 0, stream>>>(
      out_grad_ptr, in_grad_ptr, num_out_blks, reverse_dim_size, in_blk_size);
}

__host__
void Reverse::backward_task(const Task* task,
                            const std::vector<PhysicalRegion> &regions,
                            Context ctx, Runtime* runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  const Reverse* reverse = (const Reverse*) task->args;
  Domain out_grad_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain in_grad_domain = runtime->get_index_space_domain(
    ctx, task->regions[1].region.get_index_space());
  assert(out_grad_domain == in_grad_domain);
  const float* out_grad_ptr = helperGetTensorPointerRO<float>(
    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  float* in_grad_ptr = helperGetTensorPointerRW<float>(
    regions[1], task->regions[1], FID_DATA, ctx, runtime);
  // We reuse the forward kernel for backward tasks
  int axis = in_grad_domain.get_dim() - reverse->axis - 1;
  coord_t in_blk_size = 1, reverse_dim_size = 1, num_out_blks = 1;
  for (int i = 0; i < in_grad_domain.get_dim(); i++) {
    if (i < axis)
      in_blk_size *= in_grad_domain.hi()[i] - in_grad_domain.lo()[i] + 1;
    else if (i == axis)
      reverse_dim_size = in_grad_domain.hi()[i] - in_grad_domain.lo()[i] + 1;
    else
      num_out_blks *= in_grad_domain.hi()[i] - in_grad_domain.lo()[i] + 1;
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  backward_kernel(out_grad_ptr, in_grad_ptr, num_out_blks, reverse_dim_size, in_blk_size, in_grad_domain.get_volume(), stream);
}

bool Reverse::measure_operator_cost(Simulator* sim,
                                    const ParallelConfig& pc,
                                    CostMetrics& cost_metrics) const
{
  TensorBase sub_input, sub_output;
  if (!outputs[0]->get_output_sub_tensor(pc, sub_output, op_type)) {
    return false;
  }
  if (!inputs[0]->get_input_sub_tensor(pc, sub_input, op_type)) {
    return false;
  }

  sim->free_all();
  float *input_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert (input_ptr != NULL);
  float *output_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert (output_ptr != NULL);

  coord_t in_blk_size = 1, reverse_dim_size = 1, num_out_blks = 1;
  for (int i = 0; i < sub_output.num_dims; i++) {
    if (i < axis) {
      in_blk_size *= sub_output.dims[i].size;
    } else if (i == axis) {
      reverse_dim_size = sub_output.dims[i].size;
    } else {
      num_out_blks *= sub_output.dims[i].size;
    }
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  std::function<void()> forward, backward;
  forward = [&] {
     forward_kernel(input_ptr, output_ptr, num_out_blks, reverse_dim_size, in_blk_size, sub_output.get_volume(), stream);
  };
  if (sim->computationMode == COMP_MODE_TRAINING) {
    float *input_grad_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
    assert (input_grad_ptr != NULL);
    float *output_grad_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
    assert (output_grad_ptr != NULL);
    backward = [&] {
      backward_kernel(output_grad_ptr, input_grad_ptr, num_out_blks, reverse_dim_size, in_blk_size, sub_input.get_volume(), stream);
    };
  }

  inner_measure_operator_cost(sim, forward, backward, cost_metrics);

  if (sim->computationMode == COMP_MODE_TRAINING) {
    printf("[Measure Reverse] name(%s) forward_time(%.4lf) backward_time(%.4lf)\n",
        name,
        cost_metrics.forward_time,
        cost_metrics.backward_time);
  } else {
    printf("[Measure Reverse] name(%s) forward_time(%.4lf)\n",
        name,
        cost_metrics.forward_time);
  }

  return true;
}

}; // namespace FlexFlow