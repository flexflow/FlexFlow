#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

#define MAX_K 4
#define MAX_BATCH_SIZE 32
#define MAX_N 12


Tensor FFModel::aggregate(const Tensor* inputs, /* gate_preds, gate_assign, full_gate_pred, n * exp_pred */
                          int n, float lambda_bal, const char* name)
{
  Aggregate* aggr = new Aggregate(*this, inputs, n, lambda_bal, name);
  layers.push_back(aggr);
  return aggr->outputs[0];
}


Aggregate::Aggregate(FFModel& model,
                    const Tensor* _inputs,
                    int _n, float _lambda_bal, const char* name)
: Op(model, OP_AGGREGATE, name, _n+4, _inputs),
  n(_n), lambda_bal(_lambda_bal),
  profiling(model.config.profiling)
{
  // FIXME: For now, set upper limits Better: Do as follows, but memory is
  // assigned per block, so requires to check that
  // https://stackoverflow.com/questions/5531247/allocating-shared-memory/5531640#5531640
  assert(n <= MAX_N && "Increase MAX_N in #define");
  assert(inputs[0].adim[0] <= MAX_K && "Increase MAX_K in #define");
  assert(inputs[0].adim[1] <= MAX_BATCH_SIZE && "Increase MAX_BATCH_SIZE in #define");

  assert(n+4 == numInputs);
  assert(n > 0);
  assert(inputs[0].numDim == 2);
  assert(inputs[1].numDim == 2);
  assert(inputs[2].numDim == 2);
  assert(inputs[3].numDim == 2);

  for(int i = 0; i < inputs[0].numDim; i++) {
    assert(inputs[0].adim[i] == inputs[1].adim[i]);
    assert(inputs[0].adim[i] == inputs[2].adim[i]);
  }
  assert(inputs[0].adim[1] == inputs[3].adim[1]);
  assert(inputs[3].adim[0] == n);

  // expert inputs
  int num_dim = inputs[4].numDim;
  int out_dim = inputs[4].adim[0];
  for(int i = 1; i < n; i++) {
    assert(inputs[i+4].numDim == num_dim);
    assert(inputs[i+4].adim[0] == out_dim);
  }
  // output
  outputs[0].numDim = num_dim;
  for(int i = 0; i < num_dim-1; i++)
    outputs[0].adim[i] = inputs[4].adim[i];
  outputs[0].adim[num_dim-1] = inputs[0].adim[num_dim-1];

  numWeights = 0;
}


void Aggregate::create_weights(FFModel& model)
{
  // Do nothing
}


void Aggregate::create_output_and_partition(FFModel& model)
{
  // Retrieve the task indexspace for the op
  std::string pcname = name;
  task_is = IndexSpaceT<2>(model.get_or_create_task_is(2, pcname));
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<2> part_rect = runtime->get_index_space_domain(ctx, task_is);

  // Can only partition over the sample dim
  assert(part_rect.hi[0] == part_rect.lo[0]);

  int num_dim = inputs[4].numDim;
  int dims[num_dim];
  dims[0] = inputs[0].adim[1];
  for (int i = 1; i < num_dim; i++)
    dims[i] = inputs[4].adim[num_dim-1-i];

  outputs[0] = model.create_tensor<2>(dims, DT_FLOAT, this);
  outputs[0].owner_op = this;
  outputs[0].owner_idx = 0;


  // Compute partition bound for input
  for(int i = 0; i < n+4; i++) {
    Rect<2> input_rect = runtime->get_index_partition_color_space(
        ctx, inputs[i].part.get_index_partition());
    if (input_rect == part_rect) {
      input_lps[i] = inputs[i].part;
      input_grad_lps[i] = inputs[i].part_grad;
    } else {
      model.create_disjoint_partition<2>(
        inputs[i], (IndexSpaceT<2>)task_is, input_lps[i], input_grad_lps[i]);
    }
  }
}


OpMeta* Aggregate::init_task(const Task* task,
                        const std::vector<PhysicalRegion> &regions,
                        Context ctx, Runtime* runtime)
{
  Aggregate* agg = (Aggregate*) task->args;
  FFHandler handle = *((FFHandler*)task->local_args);
  AggregateMeta* m = new AggregateMeta(handle, agg->n);
  m->profiling = agg->profiling;
  return m;
}


void Aggregate::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      ParallelConfig pc; \
      std::string pcname = name; \
      ff.config.find_parallel_config(DIM, pcname, pc); \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        FFHandler handle = ff.handlers[pc.device_ids[idx++]]; \
        argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(AGGREGATE_INIT_TASK_ID, task_is,
    TaskArgument(this, sizeof(Aggregate)), argmap,
    Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
    FFConfig::get_hash_id(std::string(name)));
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        meta[idx++] = fm.get_result<OpMeta*>(*it); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}


__global__
void agg_forward_kernel(float** exp_preds,
        const int* exp_assign,
        const float* gate_net_preds,
        float* output,
        int n,
        const int k, // num chosen experts
        int exp_samples, // max samples per expert
        const int batch_size,
        int out_dim)
{
  __shared__ float* chosen_exp_preds[MAX_K*MAX_BATCH_SIZE];

  // Get pred pointers, single thread pre block
  if(threadIdx.x == 0) {
    int expert_idx[MAX_N] = {0};
    for(int i = 0; i < batch_size; i++) {
      for(int j = 0; j < k; j++) {
        // Get pointer to chosen expert predictions
        int expert = exp_assign[i*k+j];
        if(expert_idx[expert] >= exp_samples) {
          // dropped sample
          chosen_exp_preds[i*k+j] = 0;
          continue;
        }
        chosen_exp_preds[i*k+j] = exp_preds[expert] + expert_idx[expert]*out_dim;
        expert_idx[expert]++;
      }
    }
  }

  // set output tensor to 0
  CUDA_KERNEL_LOOP(i, batch_size*out_dim)
  {
    output[i] = 0.0f;
  }

  __syncthreads();

  // compute output
  CUDA_KERNEL_LOOP(i, k*out_dim*batch_size)
  {
    if(chosen_exp_preds[i/out_dim] != 0) {
      float res = gate_net_preds[i/out_dim] * chosen_exp_preds[i/out_dim][i%(out_dim)];
      int out_id = (i/(k*out_dim))*out_dim + (i%out_dim);
      atomicAdd(output+out_id, res);
    }
  }
}


__device__
void agg_backward_kernel_gate(const float* output_grad,
              float* full_gate_grads,
              float** exp_preds,
              const int* expert_assign,
              const bool* cache_corr,
              int* expert_bal, float lambda_bal,
              int batch_size, int k, int n, int out_dim)
{
  // gate gradient
  CUDA_KERNEL_LOOP(i, batch_size*k*out_dim)
  {
    if (exp_preds[i/out_dim] != 0 && cache_corr[i/(k*out_dim)]) {
      int out_id = (i/(k*out_dim))*out_dim + (i%out_dim);
      float res = output_grad[out_id] * exp_preds[i/out_dim][i%out_dim];

      float* gate_grad_idx = full_gate_grads + (i/(out_dim*k))*n
        + expert_assign[(i/(out_dim*k))*k+(i/out_dim)%k];
      atomicAdd(gate_grad_idx, res);
    }
  }

  // balance term
  CUDA_KERNEL_LOOP(i, n*batch_size)
  {
    atomicAdd(full_gate_grads+i, lambda_bal*expert_bal[i%n]);
  }

  __syncthreads();

  // make 0 mean
  CUDA_KERNEL_LOOP(i, batch_size*n)
  {
    int start = (i/n)*n;
    float sub = -full_gate_grads[i]/n;
    for(int j = 0; j < n; j++) {
      atomicAdd(full_gate_grads+start+j, sub);
    }
  }
}


__device__
void agg_backward_kernel_exp(const float* output_grad,
              const float* gate_preds,
              float** exp_grads,
              int batch_size,
              int k,
              int out_dim) {
  // compute expert gradients
  CUDA_KERNEL_LOOP(i, k*out_dim*batch_size)
  {
    if (exp_grads[i/out_dim] != 0) {
      int out_id = (i/(k*out_dim))*out_dim + (i%out_dim);
      exp_grads[i/out_dim][i%out_dim] += gate_preds[i/out_dim] * output_grad[out_id];
    }
  }
}


__global__
void agg_backward_kernel(float** exp_preds,
        float** exp_grads,
        const int* exp_assign,
        const int* true_exp_assign,
        const float* gating_net_preds,
        float* full_gating_grads,
        const float* output_grads,
        int n, // num experts
        int k, // num chosen experts
        int exp_samples, // max samples per expert
        float lambda_bal,
        int batch_size,
        int out_dim)
{
  __shared__ float* chosen_exp_preds[MAX_K*MAX_BATCH_SIZE];
  __shared__ float* chosen_exp_grads[MAX_K*MAX_BATCH_SIZE];
  __shared__ int expert_bal[MAX_N];
  __shared__ bool cache_corr[MAX_BATCH_SIZE];

  // Get pred pointers, single thread per block
  if(threadIdx.x == 0) {
    // init arrays
    for(int i = 0; i < n; i++) expert_bal[i] = 0;
    for(int i = 0; i < batch_size; i++) cache_corr[i] = true;

    // Get pointer to chosen expert predictions and expert counts
    for(int i = 0; i < batch_size; i++) {
      for(int j = 0; j < k; j++) {
        int expert = true_exp_assign[k*i + j];
        if(expert != exp_assign[k*i + j])
          cache_corr[i] = false;
        if(expert_bal[expert] >= exp_samples) {
          // dropped sample
          chosen_exp_preds[i*k+j] = 0;
          chosen_exp_grads[i*k+j] = 0;
          expert_bal[expert]++;
          continue;
        }
        chosen_exp_preds[i*k+j] = exp_preds[expert] + expert_bal[expert]*out_dim;
        chosen_exp_grads[i*k+j] = exp_grads[expert] + expert_bal[expert]*out_dim;
        expert_bal[expert]++;
      }
    }
  }

  __syncthreads();

  // FIXME: These 2 functions could execute independently in parallel
  // get expert gradients
  agg_backward_kernel_exp(output_grads, gating_net_preds, chosen_exp_grads,
    batch_size, k, out_dim);

  // get gating net gradients
  agg_backward_kernel_gate(output_grads, full_gating_grads, chosen_exp_preds,
    exp_assign, cache_corr, expert_bal, (lambda_bal*n)/batch_size, batch_size,
    k, n, out_dim);
}


void Aggregate::forward_task(const Task *task,
                             const std::vector<PhysicalRegion>& regions,
                             Context ctx, Runtime* runtime)
{
  int n = ((Aggregate*)task->args)->n;

  assert((int)regions.size() == n+3);
  assert((int)task->regions.size() == n+3);

  const AggregateMeta* m = *((AggregateMeta**)task->local_args);

  // get gate_pred, gate_assign, output
  const AccessorRO<float, 2> acc_gate_pred(regions[0], FID_DATA);
  const AccessorRO<int, 2> acc_gate_assign(regions[1], FID_DATA);
  const AccessorWO<float, 2> acc_output(regions[n+2], FID_DATA);

  Rect<2> rect_gate_pred = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  Rect<2> rect_gate_assign = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
  Rect<2> rect_output = runtime->get_index_space_domain(
      ctx, task->regions[n+2].region.get_index_space());

  coord_t batch_size = rect_gate_pred.hi[1] - rect_gate_pred.lo[1] + 1;
  assert(batch_size == rect_gate_assign.hi[1] - rect_gate_assign.lo[1] + 1);
  assert(rect_gate_pred.hi[0] - rect_gate_pred.lo[0] == rect_gate_assign.hi[0] - rect_gate_assign.lo[0]);
  assert(batch_size == rect_output.hi[1] - rect_output.lo[1] + 1);
  coord_t out_dim = rect_output.hi[0] - rect_output.lo[0] + 1;

  // get exp_preds
  float* exp_preds[n];
  // get first exp_pred and row and out_dim
  Domain exp_domain = runtime->get_index_space_domain(
    ctx, task->regions[2].region.get_index_space());
  exp_preds[0] = helperGetTensorPointerWO<float>(
    regions[2], task->regions[2], FID_DATA, ctx, runtime);
  coord_t rows = exp_domain.hi()[1] - exp_domain.lo()[1] + 1;
  assert(out_dim == exp_domain.hi()[0] - exp_domain.lo()[0] + 1);

  for(int i = 1; i < n; i++) {
    exp_domain = runtime->get_index_space_domain(
      ctx, task->regions[i+2].region.get_index_space());
    exp_preds[i] = helperGetTensorPointerWO<float>(
      regions[i+2], task->regions[i+2], FID_DATA, ctx, runtime);

    assert(rows == exp_domain.hi()[1] - exp_domain.lo()[1] + 1);
    assert(out_dim == exp_domain.hi()[0] - exp_domain.lo()[0] + 1);
  }

  int k = (int)(rect_gate_assign.hi[0] - rect_gate_assign.lo[0] + 1);

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  // call forward_kernel
  hipMemcpy(m->dev_exp_preds, exp_preds, n*sizeof(float*), hipMemcpyHostToDevice);

  agg_forward_kernel<<<GET_BLOCKS(batch_size*k*out_dim), min(CUDA_NUM_THREADS,(int)(batch_size*k*out_dim)), 0, stream>>>(
    m->dev_exp_preds, acc_gate_assign.ptr(rect_gate_assign), acc_gate_pred.ptr(rect_gate_pred),
    acc_output.ptr(rect_output), n, k, rows, batch_size, out_dim);
}


void Aggregate::backward_task(const Task *task,
                              const std::vector<PhysicalRegion>& regions,
                              Context ctx, Runtime* runtime)
{
  const AggregateMeta* m = *((AggregateMeta**)task->local_args);
  int n = ((Aggregate*)task->args)->n;
  float lambda_bal = ((Aggregate*)task->args)->lambda_bal;

  assert((int)regions.size() == 2*n+5);
  assert((int)task->regions.size() == 2*n+5);

  // get gate_pred, gate_grad, gate_assign, output_grad
  const AccessorRO<float, 2> acc_gate_pred(regions[0], FID_DATA);
  const AccessorRO<int, 2> acc_gate_assign(regions[1], FID_DATA);
  const AccessorRO<int, 2> acc_true_gate_assign(regions[2], FID_DATA);
  const AccessorWO<float, 2> full_acc_gate_grad(regions[3], FID_DATA);
  const AccessorRO<float, 2> acc_output_grad(regions[2*n+4], FID_DATA);

  Rect<2> rect_gate_pred = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  Rect<2> rect_gate_assign = runtime->get_index_space_domain(
       ctx, task->regions[1].region.get_index_space());
  Rect<2> rect_true_gate_assign = runtime->get_index_space_domain(
      ctx, task->regions[2].region.get_index_space());
  Rect<2> rect_full_gate_grad = runtime->get_index_space_domain(
      ctx, task->regions[3].region.get_index_space());
  Rect<2> rect_out_grad = runtime->get_index_space_domain(
      ctx, task->regions[2*n+4].region.get_index_space());

  coord_t batch_size = rect_gate_pred.hi[1] - rect_gate_pred.lo[1] + 1;
  assert(batch_size == rect_gate_assign.hi[1] - rect_gate_assign.lo[1] + 1);
  assert(rect_gate_assign == rect_true_gate_assign);
  assert(batch_size == rect_out_grad.hi[1] - rect_out_grad.lo[1] + 1);
  assert(batch_size == rect_full_gate_grad.hi[1] - rect_full_gate_grad.lo[1] + 1);
  coord_t k = rect_gate_assign.hi[0] - rect_gate_assign.lo[0] + 1;
  assert(rect_gate_pred.hi[0] - rect_gate_pred.lo[0] + 1 == k);
  coord_t out_dim = rect_out_grad.hi[0] - rect_out_grad.lo[0] + 1;
  assert(n == rect_full_gate_grad.hi[0] - rect_full_gate_grad.lo[0] + 1);

  // get exp_preds
  float* exp_preds[n];
  // get first exp_pred and row
  Domain exp_domain = runtime->get_index_space_domain(
    ctx, task->regions[4].region.get_index_space());
  exp_preds[0] = helperGetTensorPointerRW<float>(
    regions[4], task->regions[4], FID_DATA, ctx, runtime);
  coord_t rows = exp_domain.hi()[1] - exp_domain.lo()[1] + 1;
  assert(out_dim == exp_domain.hi()[0] - exp_domain.lo()[0] + 1);

  for(int i = 1; i < n; i++) {
    exp_domain = runtime->get_index_space_domain(
      ctx, task->regions[i+4].region.get_index_space());
    exp_preds[i] = helperGetTensorPointerRW<float>(
      regions[i+4], task->regions[i+4], FID_DATA, ctx, runtime);
    assert(rows == exp_domain.hi()[1] - exp_domain.lo()[1] + 1);
    assert(out_dim == exp_domain.hi()[0] - exp_domain.lo()[0] + 1);
  }

  // get chosen_exp_grads
  float* exp_grads[n];
  for(int i = 0; i < n; i++) {
    exp_domain = runtime->get_index_space_domain(
      ctx, task->regions[n+i+4].region.get_index_space());
    exp_grads[i] = helperGetTensorPointerRW<float>(
      regions[n+i+4], task->regions[n+i+4], FID_DATA, ctx, runtime);
    assert(rows == exp_domain.hi()[1] - exp_domain.lo()[1] + 1);
    assert(out_dim == exp_domain.hi()[0] - exp_domain.lo()[0] + 1);
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  // call backward kernel
  hipMemcpy(m->dev_exp_preds, exp_preds, n*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(m->dev_exp_grads, exp_grads, n*sizeof(float*), hipMemcpyHostToDevice);

  agg_backward_kernel<<<GET_BLOCKS(batch_size*k*out_dim), min(CUDA_NUM_THREADS,(int)(batch_size*k*out_dim)), 0, stream>>>(
    m->dev_exp_preds, m->dev_exp_grads, acc_gate_assign.ptr(rect_gate_assign),
    acc_true_gate_assign.ptr(rect_true_gate_assign), acc_gate_pred.ptr(rect_gate_pred),
    full_acc_gate_grad.ptr(rect_full_gate_grad), acc_output_grad.ptr(rect_out_grad),
    n, k, rows, lambda_bal, batch_size, out_dim);
}


void Aggregate::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(AGGREGATE_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Aggregate)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  // gate_preds
  launcher.add_region_requirement(
    RegionRequirement(input_lps[0], 0/*projection id*/,
      READ_WRITE, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  // gate_assign
  launcher.add_region_requirement(
    RegionRequirement(input_lps[1], 0/*projection id*/,
      READ_WRITE, EXCLUSIVE, inputs[1].region));
  launcher.add_field(1, FID_DATA);
  // exp_preds
  for(int i = 0; i < n; i++) {
    launcher.add_region_requirement(
      RegionRequirement(input_lps[i+4], 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, inputs[i+4].region));
    launcher.add_field(i+2, FID_DATA);
  }
  // output
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(n+2, FID_DATA);

  runtime->execute_index_space(ctx, launcher);
}


void Aggregate::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }

  IndexLauncher launcher(AGGREGATE_BWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Aggregate)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));

  // gate_preds
  launcher.add_region_requirement(
    RegionRequirement(input_lps[0], 0/*projection id*/,
      READ_WRITE, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);

  // gate_assign
  launcher.add_region_requirement(
    RegionRequirement(input_lps[1], 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, inputs[1].region));
  launcher.add_field(1, FID_DATA);

  // true gate_assign
  launcher.add_region_requirement(
    RegionRequirement(input_lps[2], 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, inputs[2].region));
  launcher.add_field(2, FID_DATA);

  // full_gate gradients
  launcher.add_region_requirement(
    RegionRequirement(input_grad_lps[3], 0/*projection id*/,
      READ_WRITE, EXCLUSIVE, inputs[3].region_grad));
  launcher.add_field(3, FID_DATA);

  // exp_preds
  for(int i = 0; i < n; i++) {
    launcher.add_region_requirement(
      RegionRequirement(input_lps[i+4], 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, inputs[i+4].region));
    launcher.add_field(i+4, FID_DATA);
  }

  // exp_preds gradients
  for(int i = 0; i < n; i++) {
    launcher.add_region_requirement(
      RegionRequirement(input_grad_lps[i+4], 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, inputs[i+4].region_grad));
    launcher.add_field(i+n+4, FID_DATA);
  }

  // output
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
      READ_WRITE, EXCLUSIVE, outputs[0].region_grad));
  launcher.add_field(2*n+4, FID_DATA);

  runtime->execute_index_space(ctx, launcher);
}


AggregateMeta::AggregateMeta(FFHandler handler, int n)
: OpMeta(handler)
{
  checkCUDA(hipMalloc(&dev_exp_preds, n*sizeof(float*)));
  checkCUDA(hipMalloc(&dev_exp_grads, n*sizeof(float*)));
}
AggregateMeta::~AggregateMeta(void)
{
  checkCUDA(hipFree(&dev_exp_preds));
  checkCUDA(hipFree(&dev_exp_grads));
}


bool Aggregate::measure_operator_cost(Simulator* sim,
                                 const ParallelConfig& pc,
                                 CostMetrics& cost_metrics)
{
  //TODO: implement
  cost_metrics.forward_time = 0.0f;
  cost_metrics.backward_time = 0.0f;
  cost_metrics.memory_requirement = 0;
  return false;
}
