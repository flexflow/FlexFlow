#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ffconst_utils.h"
#include "flexflow/ops/kernels/rms_norm_kernels.h"
#include "flexflow/ops/rms_norm.h"
#include "flexflow/utils/cuda_helper.h"
#include <hipblas.h>

namespace FlexFlow {
// declare Legion names
using Legion::coord_t;

#define C10_WARP_SIZE 32

RMSNormMeta::RMSNormMeta(FFHandler handler,
                         RMSNorm const *rms,
                         MemoryAllocator &gpu_mem_allocator)
    : OpMeta(handler, rms) {
  eps = rms->eps;

  in_dim = rms->data_dim;
  batch_size = rms->effective_batch_size;
  num_elements = in_dim * batch_size;

  DataType data_type = rms->weights[0]->data_type;
  size_t rms_ptr_size = batch_size;
  size_t norm_ptr_size = num_elements;
  size_t totalSize = (rms_ptr_size + norm_ptr_size) * data_type_size(data_type);
  gpu_mem_allocator.create_legion_instance(reserveInst, totalSize);
  rms_ptr = gpu_mem_allocator.allocate_instance_untyped(
      rms_ptr_size * data_type_size(data_type));
  norm_ptr = gpu_mem_allocator.allocate_instance_untyped(
      norm_ptr_size * data_type_size(data_type));
  allocated_peft_buffer_size = 0;
}
RMSNormMeta::~RMSNormMeta(void) {
  if (reserveInst != Realm::RegionInstance::NO_INST) {
    reserveInst.destroy();
  }
}

namespace Kernels {
namespace RMSNorm {

template <typename T>
__device__ __forceinline__ T WARP_SHFL_DOWN(T value,
                                            unsigned int delta,
                                            int width = warpSize,
                                            unsigned int mask = 0xffffffff) {
#ifndef __HIP_PLATFORM_HCC__
  return __shfl_down_sync(mask, value, delta, width);
#else
  return __shfl_down(value, delta, width);
#endif
}

template <typename T>
__inline__ __device__ T WarpReduceSum(T val) {
#pragma unroll
  for (int offset = (C10_WARP_SIZE >> 1); offset > 0; offset >>= 1) {
    val += WARP_SHFL_DOWN(val, offset);
  }
  return val;
}

template <typename T>
__inline__ __device__ T BlockReduceSum(T val, T *shared) {
  int const lid = threadIdx.x % C10_WARP_SIZE;
  int const wid = threadIdx.x / C10_WARP_SIZE;
  val = WarpReduceSum(val);
  __syncthreads();
  if (lid == 0) {
    shared[wid] = val;
  }
  __syncthreads();
  val = (threadIdx.x < (blockDim.x / C10_WARP_SIZE)) ? shared[lid] : T(0);
  if (wid == 0) {
    val = WarpReduceSum(val);
  }
  return val;
}

template <typename T>
__global__ void RMSNormFusedForwardKernel(int64_t N,
                                          float eps,
                                          T const *X,
                                          T *rms,
                                          T *Y,
                                          T const *weights,
                                          T *output) {
  __shared__ float v_shared[C10_WARP_SIZE];
  int64_t const i = blockIdx.x;
  float sum = 0.0f;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    int64_t const index = i * N + j;
    sum += (static_cast<float>(X[index]) * static_cast<float>(X[index]));
  }
  sum = BlockReduceSum<float>(sum, v_shared);

  if (threadIdx.x == 0) {
    rms[i] = static_cast<T>(rsqrt((sum / static_cast<float>(N)) + eps));
  }

  __syncthreads();

  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    Y[index] = static_cast<T>(X[index]) * static_cast<T>(rms[i]);
    output[index] = Y[index] * weights[index % N];
  }
}

template <typename T>
void forward_kernel(RMSNormMeta const *m,
                    T const *input_ptr,
                    T const *weight_ptr,
                    T *output_ptr,
                    hipStream_t stream) {

  RMSNormFusedForwardKernel<T>
      <<<m->batch_size, std::min(CUDA_NUM_THREADS, m->in_dim), 0, stream>>>(
          m->in_dim,
          m->eps,
          input_ptr,
          static_cast<T *>(m->rms_ptr),
          static_cast<T *>(m->norm_ptr),
          weight_ptr,
          output_ptr);
}

void forward_kernel_wrapper(RMSNormMeta const *m,
                            GenericTensorAccessorR const &input,
                            GenericTensorAccessorR const &weight,
                            GenericTensorAccessorW const &output) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  assert(output.data_type == input.data_type);
  assert(weight.data_type == output.data_type);
  if (output.data_type == DT_HALF) {
    forward_kernel(m,
                   input.get_half_ptr(),
                   weight.get_half_ptr(),
                   output.get_half_ptr(),
                   stream);
  } else if (output.data_type == DT_FLOAT) {
    forward_kernel(m,
                   input.get_float_ptr(),
                   weight.get_float_ptr(),
                   output.get_float_ptr(),
                   stream);
  } else {
    assert(false && "Unsupported data type");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("[RMSNorm] forward time (CF) = %.2fms\n", elapsed);
  }
}

void inference_kernel_wrapper(RMSNormMeta *m,
                              BatchConfig const *bc,
                              GenericTensorAccessorR const &input,
                              GenericTensorAccessorR const &weight,
                              GenericTensorAccessorW const &output) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  assert(output.data_type == input.data_type);
  assert(weight.data_type == output.data_type);

  // save input activation if needed for PEFT
  if (bc->num_active_peft_tokens() > 0) {
    // Check that we have at most one request that requires peft_bwd
    int num_peft_requests = 0;
    for (int i = 0; i < bc->max_requests_per_batch(); i++) {
      if (bc->request_completed[i]) {
        continue;
      }
      if (bc->requestsInfo[i].peft_model_id == PEFTModelID::NO_ID) {
        continue;
      }
      if (bc->requestsInfo[i].peft_bwd) {
        num_peft_requests++;
      }
    }
    assert(num_peft_requests <= 1);
    for (int i = 0; i < bc->max_requests_per_batch(); i++) {
      if (bc->request_completed[i]) {
        continue;
      }
      // Skip non-PEFT requests
      if (bc->requestsInfo[i].peft_model_id == PEFTModelID::NO_ID) {
        continue;
      }
      int num_peft_tokens = bc->requestsInfo[i].num_tokens_in_batch;
      int max_peft_tokens = bc->requestsInfo[i].max_sequence_length;
      int first_token_offset = bc->requestsInfo[i].first_token_offset_in_batch;
      int in_dim = input.domain.hi()[0] - input.domain.lo()[0] + 1;
      if (bc->requestsInfo[i].peft_bwd) {
        size_t activation_size_needed =
            data_type_size(m->input_type[0]) * max_peft_tokens * in_dim;
        if (activation_size_needed > m->allocated_peft_buffer_size) {
          MemoryAllocator *allocator = m->handle.peft_activation_allocator;
          m->input_activation =
              allocator->allocate_instance_untyped(activation_size_needed);
          m->allocated_peft_buffer_size = activation_size_needed;
        }

        if (input.data_type == DT_FLOAT) {
          checkCUDA(hipMemcpyAsync(
              m->input_activation,
              input.get_float_ptr() + first_token_offset * in_dim,
              data_type_size(input.data_type) * num_peft_tokens * in_dim,
              hipMemcpyDeviceToDevice,
              stream));
        } else if (input.data_type == DT_HALF) {
          checkCUDA(hipMemcpyAsync(
              m->input_activation,
              input.get_half_ptr() + first_token_offset * in_dim,
              data_type_size(input.data_type) * num_peft_tokens * in_dim,
              hipMemcpyDeviceToDevice,
              stream));
        } else {
          assert(false && "unsupport datatype in layernorm");
        }
      }
    }
  }

  if (output.data_type == DT_HALF) {
    forward_kernel(m,
                   input.get_half_ptr(),
                   weight.get_half_ptr(),
                   output.get_half_ptr(),
                   stream);
  } else if (output.data_type == DT_FLOAT) {
    forward_kernel(m,
                   input.get_float_ptr(),
                   weight.get_float_ptr(),
                   output.get_float_ptr(),
                   stream);
  } else {
    assert(false && "Unsupported data type");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("[RMSNorm] forward time (CF) = %.2fms\n", elapsed);
  }
}

template <typename T>
__global__ void ComputeInternalGradientsCUDAKernel(
    int64_t N, T const *dY, T const *X, T const *gamma, T const *rrms, T *c2) {
  __shared__ T ds_storage[C10_WARP_SIZE];
  const int64_t i = blockIdx.x;
  float ds = 0;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    int const index = i * N + j;
    ds += static_cast<float>(dY[index]) * static_cast<float>(X[index]) *
          static_cast<float>(gamma[j]);
  }
  ds = BlockReduceSum<T>(ds, ds_storage);
  if (threadIdx.x == 0) {
    float const c2_val =
        -ds *
        (static_cast<float>(rrms[i]) * static_cast<float>(rrms[i]) *
         static_cast<float>(rrms[i])) /
        static_cast<float>((int)N);
    c2[i] = static_cast<T>(c2_val);
  }
}

template <typename T>
__global__ void RMSNormBackwardCUDAKernel(int64_t N,
                                          T const *dY,
                                          T const *X,
                                          T const *gamma,
                                          T const *c1,
                                          T const *c2,
                                          T *dX,
                                          bool reset_input_grad) {
  const int64_t i = blockIdx.x;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    float const dX_val =
        static_cast<float>(c1[i]) * static_cast<float>(dY[index]) *
            static_cast<float>(gamma[j]) +
        static_cast<float>(c2[i]) * static_cast<float>(X[index]);
    if (reset_input_grad) {
      dX[index] = dX_val;
    } else {
      dX[index] += dX_val;
    }
  }
}

// Assume the batch size will not be very large, direct implementation is the
// most efficient one.
template <typename T>
__global__ void GammaBackwardCUDAKernel(
    int64_t M, int64_t N, T const *dY, T const *X, T const *rrms, T *dg) {
  const int64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j < N) {
    T sum1 = 0;
    for (int64_t i = 0; i < M; ++i) {
      const int64_t index = i * N + j;
      sum1 += dY[index] * X[index] * rrms[i];
    }
    dg[j] = sum1;
  }
}

template <typename T>
void backward_kernel(RMSNormMeta const *m,
                     T const *output_grad_ptr,
                     T const *input_ptr,
                     T *input_grad_ptr,
                     T const *weight_ptr,
                     T *weight_grad_ptr,
                     hipStream_t stream) {
  int M = m->batch_size;
  int N = m->in_dim;
  ComputeInternalGradientsCUDAKernel<T>
      <<<M, std::min(N, CUDA_NUM_THREADS), 0, stream>>>(
          N,
          output_grad_ptr,
          input_ptr,
          weight_ptr,
          static_cast<T *>(m->rms_ptr),
          static_cast<T *>(m->norm_ptr));

  RMSNormBackwardCUDAKernel<T><<<M, std::min(N, CUDA_NUM_THREADS), 0, stream>>>(
      m->in_dim,
      output_grad_ptr,
      input_ptr,
      weight_ptr,
      static_cast<T *>(m->rms_ptr),
      static_cast<T *>(m->norm_ptr),
      input_grad_ptr,
      m->reset_input_grads[0]);
  GammaBackwardCUDAKernel<T><<<M, std::min(N, CUDA_NUM_THREADS), 0, stream>>>(
      M,
      N,
      output_grad_ptr,
      input_ptr,
      static_cast<T *>(m->rms_ptr),
      weight_grad_ptr);
}

void backward_kernel_wrapper(RMSNormMeta const *m,
                             GenericTensorAccessorR const &output_grad,
                             GenericTensorAccessorR const &input,
                             GenericTensorAccessorW const &input_grad,
                             GenericTensorAccessorR const &weight,
                             GenericTensorAccessorW const &weight_grad) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  assert(input_grad.data_type == input.data_type);
  assert(weight_grad.data_type == weight.data_type);
  assert(output_grad.data_type == input.data_type);
  assert(weight.data_type == output_grad.data_type);

  if (output_grad.data_type == DT_HALF) {
    backward_kernel(m,
                    output_grad.get_half_ptr(),
                    input.get_half_ptr(),
                    input_grad.get_half_ptr(),
                    weight.get_half_ptr(),
                    weight_grad.get_half_ptr(),
                    stream);
  } else if (output_grad.data_type == DT_FLOAT) {
    backward_kernel(m,
                    output_grad.get_float_ptr(),
                    input.get_float_ptr(),
                    input_grad.get_float_ptr(),
                    weight.get_float_ptr(),
                    weight_grad.get_float_ptr(),
                    stream);
  } else {
    assert(false && "Unsupported data type");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("[RMSNorm] backward time (CF) = %.2fms\n", elapsed);
  }
}

template <typename T>
void peft_bwd_kernel(RMSNormMeta const *m,
                     BatchConfig const *bc,
                     T const *output_grad_ptr,
                     T *input_grad_ptr,
                     T const *weight_ptr,
                     hipStream_t stream) {
  for (int i = 0; i < bc->max_requests_per_batch(); i++) {
    if (bc->request_completed[i]) {
      continue;
    }
    // Skip non-PEFT requests
    if (bc->requestsInfo[i].peft_model_id == PEFTModelID::NO_ID) {
      continue;
    }
    // Skip PEFT forward-only requests
    if (!bc->requestsInfo[i].peft_bwd) {
      continue;
    }

    int M = bc->requestsInfo[i].num_tokens_in_batch;
    int N = m->num_elements;
    ComputeInternalGradientsCUDAKernel<T>
        <<<M, std::min(N, CUDA_NUM_THREADS), 0, stream>>>(
            N,
            output_grad_ptr,
            static_cast<T *>(m->input_activation),
            weight_ptr,
            static_cast<T *>(m->rms_ptr),
            static_cast<T *>(m->norm_ptr));
    RMSNormBackwardCUDAKernel<T>
        <<<M, std::min(N, CUDA_NUM_THREADS), 0, stream>>>(
            m->in_dim,
            output_grad_ptr,
            static_cast<T *>(m->input_activation),
            weight_ptr,
            static_cast<T *>(m->rms_ptr),
            static_cast<T *>(m->norm_ptr),
            input_grad_ptr,
            m->reset_input_grads[0]);
  }
}

void peft_bwd_kernel_wrapper(RMSNormMeta const *m,
                             BatchConfig const *bc,
                             GenericTensorAccessorR const &output_grad,
                             GenericTensorAccessorW const &input_grad,
                             GenericTensorAccessorR const &weight) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  assert(input_grad.data_type == output_grad.data_type);
  assert(output_grad.data_type == weight.data_type);

  if (output_grad.data_type == DT_HALF) {
    peft_bwd_kernel(m,
                    bc,
                    output_grad.get_half_ptr(),
                    input_grad.get_half_ptr(),
                    weight.get_half_ptr(),
                    stream);
  } else if (output_grad.data_type == DT_FLOAT) {
    peft_bwd_kernel(m,
                    bc,
                    output_grad.get_float_ptr(),
                    input_grad.get_float_ptr(),
                    weight.get_float_ptr(),
                    stream);
  } else {
    assert(false && "Unsupported data type");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("[RMSNorm] peft_bwd time (CF) = %.2fms\n", elapsed);
  }
}

} // namespace RMSNorm
} // namespace Kernels
} // namespace FlexFlow
