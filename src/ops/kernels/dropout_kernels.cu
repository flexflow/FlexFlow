/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/kernels/dropout_kernels.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

// declare Legion names
using Legion::coord_t;
using Legion::Domain;
using Legion::Memory;

DropoutMeta::DropoutMeta(FFHandler handler,
                         Dropout const *dropout,
                         Memory gpu_mem,
                         Domain const &output_domain)
    : OpMeta(handler) {
  profiling = dropout->profiling;
  inference_debugging = dropout->inference_debugging;
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateDropoutDescriptor(&dropoutDesc));
  checkCUDNN(hipdnnDropoutGetStatesSize(handle.dnn, &(dropoutStateSize)));
  checkCUDNN(cudnnSetTensorDescriptorFromDomain(inputTensor, output_domain));
  checkCUDNN(cudnnSetTensorDescriptorFromDomain(outputTensor, output_domain));
  checkCUDNN(
      cudnnDropoutGetReserveSpaceSize(outputTensor, &(reserveSpaceSize)));
  {
    // allocate memory for dropoutStates and reserveSpace
    size_t totalSize = dropoutStateSize + reserveSpaceSize;
    Realm::Rect<1, coord_t> bounds(Realm::Point<1, coord_t>(0),
                                   Realm::Point<1, coord_t>(totalSize - 1));
    std::vector<size_t> field_sizes;
    field_sizes.push_back(sizeof(char));
    Realm::RegionInstance::create_instance(reserveInst,
                                           gpu_mem,
                                           bounds,
                                           field_sizes,
                                           0,
                                           Realm::ProfilingRequestSet())
        .wait();
    dropoutStates = reserveInst.pointer_untyped(0, sizeof(char));
    reserveSpace = ((char *)dropoutStates) + dropoutStateSize;
  }
  // checkCUDA(hipMalloc(&dropoutStates, dropoutStateSize));
  // checkCUDA(hipMalloc(&reserveSpace, reserveSpaceSize));
  checkCUDNN(hipdnnSetDropoutDescriptor(dropoutDesc,
                                       handle.dnn,
                                       dropout->rate,
                                       dropoutStates,
                                       dropoutStateSize,
                                       dropout->seed));
}

DropoutMeta::~DropoutMeta(void) {
  reserveInst.destroy();
  checkCUDNN(hipdnnDestroyTensorDescriptor(inputTensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(outputTensor));
  checkCUDNN(hipdnnDestroyDropoutDescriptor(dropoutDesc));
}

namespace Kernels {
namespace Dropout {

void forward_kernel_wrapper(DropoutMeta *m,
                            float const *input_ptr,
                            float *output_ptr) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  Internal::forward_kernel(m, input_ptr, output_ptr, stream);
}

void backward_kernel_wrapper(DropoutMeta *m,
                             float const *output_grad_ptr,
                             float *input_grad_ptr) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  Internal::backward_kernel(m, output_grad_ptr, input_grad_ptr, stream);
}

namespace Internal {

void forward_kernel(DropoutMeta *m,
                    float const *input_ptr,
                    float *output_ptr,
                    hipStream_t stream) {
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  checkCUDNN(cudnnDropoutForward(m->handle.dnn,
                                 m->dropoutDesc,
                                 m->inputTensor,
                                 input_ptr,
                                 m->outputTensor,
                                 output_ptr,
                                 m->reserveSpace,
                                 m->reserveSpaceSize));
}

void backward_kernel(DropoutMeta *m,
                     float const *output_grad_ptr,
                     float *input_grad_ptr,
                     hipStream_t stream) {
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  checkCUDNN(cudnnDropoutBackward(m->handle.dnn,
                                  m->dropoutDesc,
                                  m->outputTensor,
                                  output_grad_ptr,
                                  m->inputTensor,
                                  input_grad_ptr,
                                  m->reserveSpace,
                                  m->reserveSpaceSize));
}

} // namespace Internal
} // namespace Dropout
} // namespace Kernels
} // namespace FlexFlow
