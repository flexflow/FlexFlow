#include "flexflow/ops/kernels/conv_2d_kernels.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

Conv2DMeta::Conv2DMeta(FFHandler handler) : OpMeta(handler) {
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
}

namespace Kernels {
namespace Conv2D {

void init_kernel(Conv2DMeta *m,
                 int input_w,
                 int input_h,
                 int input_c,
                 int input_n,
                 int output_w,
                 int output_h,
                 int output_c,
                 int output_n,
                 int kernel_h,
                 int kernel_w,
                 int groups,
                 int stride_h,
                 int stride_w,
                 int pad_h,
                 int pad_w,
                 float const *input_ptr,
                 float *output_ptr,
                 float const *kernel_ptr,
                 float *kernel_grad_ptr,
                 float *forward_time,
                 float *backward_time) {
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        input_n,
                                        input_c,
                                        input_h,
                                        input_w));

  checkCUDNN(hipdnnSetTensor4dDescriptor(
      m->biasTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, output_c, 1, 1));

  // Require that input_c is divisible by conv->groups
  assert(input_c % groups == 0);
  printf("filterDim: kernel(%d %d) c_in(%d), c_out(%d)\n",
         kernel_h,
         kernel_w,
         input_c / groups,
         output_c);
  checkCUDNN(hipdnnSetFilter4dDescriptor(m->filterDesc,
                                        HIPDNN_DATA_FLOAT,
                                        HIPDNN_TENSOR_NCHW,
                                        output_c,
                                        input_c / groups,
                                        kernel_h,
                                        kernel_w));

  checkCUDNN(hipdnnSetConvolution2dDescriptor(m->convDesc,
                                             pad_h, // conv->padding_h,
                                             pad_w, // conv->padding_w,
                                             stride_h,
                                             stride_w,
                                             1 /*upscale_x*/,
                                             1 /*upscale_y*/,
                                             HIPDNN_CROSS_CORRELATION,
                                             HIPDNN_DATA_FLOAT));
  if (groups != 1) {
    checkCUDNN(hipdnnSetConvolutionGroupCount(m->convDesc, groups));
  }

  // enable tensor core when possible
  if (m->handle.allowTensorOpMathConversion) {
    checkCUDNN(hipdnnSetConvolutionMathType(
        m->convDesc, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
  } else {
    checkCUDNN(hipdnnSetConvolutionMathType(m->convDesc, HIPDNN_TENSOR_OP_MATH));
  }

  int n, c, h, w;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(
      m->convDesc, m->inputTensor, m->filterDesc, &n, &c, &h, &w));
  assert(n == output_n);
  assert(c == output_c);
  assert(h == output_h);
  assert(w == output_w);

  checkCUDNN(hipdnnSetTensor4dDescriptor(
      m->outputTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));

  float time;
  // select forward algorithm
  m->fwdAlgo =
      Internal::selectConvolutionForwardAlgorithm(m->handle.dnn,
                                                  m->inputTensor,
                                                  input_ptr,
                                                  m->filterDesc,
                                                  kernel_ptr,
                                                  m->convDesc,
                                                  m->handle.workSpace,
                                                  m->handle.workSpaceSize,
                                                  m->outputTensor,
                                                  output_ptr,
                                                  &time);
  if (forward_time != nullptr) {
    *forward_time += time;
  }

  // select backward filter algorithm
  m->bwdFilterAlgo = Internal::selectConvolutionBackwardFilterAlgorithm(
      m->handle.dnn,
      m->inputTensor,
      input_ptr,
      m->outputTensor,
      output_ptr,
      m->convDesc,
      m->handle.workSpace,
      m->handle.workSpaceSize,
      m->filterDesc,
      kernel_grad_ptr,
      &time);
  if (backward_time != nullptr) {
    *backward_time += time;
  }

  // select backward data algorithm
  m->bwdDataAlgo =
      Internal::selectConvolutionBackwardDataAlgorithm(m->handle.dnn,
                                                       m->filterDesc,
                                                       kernel_ptr,
                                                       m->outputTensor,
                                                       output_ptr,
                                                       m->convDesc,
                                                       m->handle.workSpace,
                                                       m->handle.workSpaceSize,
                                                       m->inputTensor,
                                                       (float *)input_ptr,
                                                       &time);
  if (backward_time != nullptr) {
    *backward_time += time;
  }

  if (m->relu) {
    checkCUDNN(hipdnnSetActivationDescriptor(
        m->actiDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));
  }
}

void forward_kernel_wrapper(Conv2DMeta const *m,
                            float const *input_ptr,
                            float *output_ptr,
                            float const *filter_ptr,
                            float const *bias_ptr) {
  // printf("fwdAlgo(%d), bwdFilterALgo(%d), bwdDataAlgo(%d)\n",
  // (int)m->fwdAlgo,(int) m->bwdFilterAlgo,(int) m->bwdDataAlgo);
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  Internal::forward_kernel(
      m, input_ptr, output_ptr, filter_ptr, bias_ptr, stream);
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    print_tensor<float>(input_ptr, 16, "[Conv2D:forward:input]");
    print_tensor<float>(filter_ptr, 16, "[Conv2D:forward:kernel]");
    print_tensor<float>(bias_ptr, 16, "[Conv2D:forward:bias]");
    print_tensor<float>(output_ptr, 16, "[Conv2D:forward:output]");
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Conv2D] forward time (CF) = %.2fms\n", m->op_name, elapsed);
  }
}

void backward_kernel_wrapper(Conv2DMeta const *m,
                             float const *input_ptr,
                             float *input_grad_ptr,
                             float const *output_ptr,
                             float *output_grad_ptr,
                             float const *kernel_ptr,
                             float *kernel_grad_ptr,
                             float *bias_grad_ptr) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  Internal::backward_kernel(m,
                            input_ptr,
                            input_grad_ptr,
                            output_ptr,
                            output_grad_ptr,
                            kernel_ptr,
                            kernel_grad_ptr,
                            bias_grad_ptr,
                            stream);
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Conv2D] backward time = %.2fms\n", m->op_name, elapsed);
    // print_tensor<4, float>(acc_output_grad.ptr, acc_output_grad.rect,
    // "[Conv2D:backward:output_grad]"); print_tensor<4,
    // float>(acc_kernel_grad.ptr, acc_kernel_grad.rect,
    // "[Conv2D:backward:kernel_grad]"); print_tensor<1,
    // float>(acc_bias_grad.ptr, acc_bias_grad.rect,
    // "[Conv2D:backward:bias_grad]"); print_tensor<4,
    // float>(acc_input_grad.ptr, acc_input_grad.rect,
    // "[Conv2D:backward:input_grad]");
  }
}

namespace Internal {

void forward_kernel(Conv2DMeta const *m,
                    float const *input_ptr,
                    float *output_ptr,
                    float const *filter_ptr,
                    float const *bias_ptr,
                    hipStream_t stream) {
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha = 1.0f, beta = 0.0f;
  checkCUDNN(hipdnnConvolutionForward(m->handle.dnn,
                                     &alpha,
                                     m->inputTensor,
                                     input_ptr,
                                     m->filterDesc,
                                     filter_ptr,
                                     m->convDesc,
                                     m->fwdAlgo,
                                     m->handle.workSpace,
                                     m->handle.workSpaceSize,
                                     &beta,
                                     m->outputTensor,
                                     output_ptr));

  // use_bias == True
  if (bias_ptr != NULL) {
    checkCUDNN(hipdnnAddTensor(m->handle.dnn,
                              &alpha,
                              m->biasTensor,
                              bias_ptr,
                              &alpha,
                              m->outputTensor,
                              output_ptr));
  }
  if (m->relu) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn,
                                      m->actiDesc,
                                      &alpha,
                                      m->outputTensor,
                                      output_ptr,
                                      &beta,
                                      m->outputTensor,
                                      output_ptr));
  }
}

void backward_kernel(Conv2DMeta const *m,
                     float const *input_ptr,
                     float *input_grad_ptr,
                     float const *output_ptr,
                     float *output_grad_ptr,
                     float const *kernel_ptr,
                     float *kernel_grad_ptr,
                     float *bias_grad_ptr,
                     hipStream_t stream) {
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha = 1.0f;
  // float beta = 0.0f;
  if (m->relu) {
    hipdnnDataType_t dataType;
    int n, c, h, w, nStride, cStride, hStride, wStride;
    checkCUDNN(hipdnnGetTensor4dDescriptor(m->outputTensor,
                                          &dataType,
                                          &n,
                                          &c,
                                          &h,
                                          &w,
                                          &nStride,
                                          &cStride,
                                          &hStride,
                                          &wStride));
    reluBackward<<<GET_BLOCKS(n * c * h * w), CUDA_NUM_THREADS, 0, stream>>>(
        output_grad_ptr, output_ptr, n * c * h * w);
  }
  // Compute filter gradiant
  // NOTE: we use alpha for kernel_grad to accumulate gradients
  checkCUDNN(hipdnnConvolutionBackwardFilter(m->handle.dnn,
                                            &alpha,
                                            m->inputTensor,
                                            input_ptr,
                                            m->outputTensor,
                                            output_grad_ptr,
                                            m->convDesc,
                                            m->bwdFilterAlgo,
                                            m->handle.workSpace,
                                            m->handle.workSpaceSize,
                                            &alpha,
                                            m->filterDesc,
                                            kernel_grad_ptr));
  // Compute bias gradiant
  // NOTE: we use alpha for bias_grad to accumulate gradients
  if (bias_grad_ptr != NULL) {
    checkCUDNN(hipdnnConvolutionBackwardBias(m->handle.dnn,
                                            &alpha,
                                            m->outputTensor,
                                            output_grad_ptr,
                                            &alpha,
                                            m->biasTensor,
                                            bias_grad_ptr));
  }
  // Compute data gradiant
  // NOTE: we use alpha for input_grad to accumulate gradients
  if (input_grad_ptr != NULL) {
    checkCUDNN(hipdnnConvolutionBackwardData(m->handle.dnn,
                                            &alpha,
                                            m->filterDesc,
                                            kernel_ptr,
                                            m->outputTensor,
                                            output_grad_ptr,
                                            m->convDesc,
                                            m->bwdDataAlgo,
                                            m->handle.workSpace,
                                            m->handle.workSpaceSize,
                                            &alpha,
                                            m->inputTensor,
                                            input_grad_ptr));
  }
}

hipdnnConvolutionFwdAlgo_t selectConvolutionForwardAlgorithm(
    hipdnnHandle_t handle,
    const hipdnnTensorDescriptor_t xDesc,
    void const *x,
    const hipdnnFilterDescriptor_t wDesc,
    void const *w,
    const hipdnnConvolutionDescriptor_t convDesc,
    void *workSpace,
    size_t workSpaceSize,
    const hipdnnTensorDescriptor_t yDesc,
    void *y,
    float *time) {
  int const reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionFwdAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionForwardAlgorithmEx(handle,
                                                    xDesc,
                                                    x,
                                                    wDesc,
                                                    w,
                                                    convDesc,
                                                    yDesc,
                                                    y,
                                                    reqAlgCnt,
                                                    &cnt,
                                                    perfResults,
                                                    workSpace,
                                                    workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("forwardAlgo(%d) time(%.2lf)\n",
         perfResults[0].algo,
         perfResults[0].time);
  if (time != nullptr) {
    *time = perfResults[0].time;
  }
  return perfResults[0].algo;
}

hipdnnConvolutionBwdDataAlgo_t selectConvolutionBackwardDataAlgorithm(
    hipdnnHandle_t handle,
    const hipdnnFilterDescriptor_t wDesc,
    void const *w,
    const hipdnnTensorDescriptor_t dyDesc,
    void const *dy,
    const hipdnnConvolutionDescriptor_t convDesc,
    void *workSpace,
    size_t workSpaceSize,
    const hipdnnTensorDescriptor_t dxDesc,
    void *dx,
    float *time) {
  int const reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionBwdDataAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionBackwardDataAlgorithmEx(handle,
                                                         wDesc,
                                                         w,
                                                         dyDesc,
                                                         dy,
                                                         convDesc,
                                                         dxDesc,
                                                         dx,
                                                         reqAlgCnt,
                                                         &cnt,
                                                         perfResults,
                                                         workSpace,
                                                         workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("bwdDataAlgo(%d) time(%.2lf)\n",
         perfResults[0].algo,
         perfResults[0].time);
  if (time != nullptr) {
    *time = perfResults[0].time;
  }
  return perfResults[0].algo;
}

hipdnnConvolutionBwdFilterAlgo_t selectConvolutionBackwardFilterAlgorithm(
    hipdnnHandle_t handle,
    const hipdnnTensorDescriptor_t xDesc,
    void const *x,
    const hipdnnTensorDescriptor_t dyDesc,
    void const *dy,
    const hipdnnConvolutionDescriptor_t convDesc,
    void *workSpace,
    size_t workSpaceSize,
    const hipdnnFilterDescriptor_t dwDesc,
    void *dw,
    float *time) {
  int const reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionBwdFilterAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionBackwardFilterAlgorithmEx(handle,
                                                           xDesc,
                                                           x,
                                                           dyDesc,
                                                           dy,
                                                           convDesc,
                                                           dwDesc,
                                                           dw,
                                                           reqAlgCnt,
                                                           &cnt,
                                                           perfResults,
                                                           workSpace,
                                                           workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("bwdFilterAlgo(%d) time(%.2lf)\n",
         perfResults[0].algo,
         perfResults[0].time);
  if (time != nullptr) {
    *time = perfResults[0].time;
  }
  return perfResults[0].algo;
}

} // namespace Internal
} // namespace Conv2D
} // namespace Kernels
} // namespace FlexFlow
