/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/kernels/batch_matmul_kernels.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

BatchMatmulMeta::BatchMatmulMeta(FFHandler handler) : OpMeta(handler) {}

namespace Kernels {
namespace BatchMatmul {

void forward_kernel_wrapper(BatchMatmulMeta const *meta,
                            float *o_ptr,
                            float const *a_ptr,
                            float const *b_ptr,
                            float const *c_ptr,
                            int m,
                            int n,
                            int k,
                            int batch,
                            int a_seq_length_dim,
                            int b_seq_length_dim,
                            int seq_length) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (meta->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  Internal::forward_kernel(meta,
                           o_ptr,
                           a_ptr,
                           b_ptr,
                           c_ptr,
                           m,
                           n,
                           k,
                           batch,
                           stream,
                           a_seq_length_dim,
                           b_seq_length_dim,
                           seq_length);
  if (meta->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("BatchMatmul forward time = %.2lfms\n", elapsed);
  }
}

void backward_kernel_wrapper(BatchMatmulMeta const *meta,
                             float const *o_ptr,
                             float const *o_grad_ptr,
                             float const *a_ptr,
                             float *a_grad_ptr,
                             float const *b_ptr,
                             float *b_grad_ptr,
                             float *c_grad_ptr,
                             int m,
                             int n,
                             int k,
                             int batch) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (meta->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  Internal::backward_kernel(meta,
                            o_ptr,
                            o_grad_ptr,
                            a_ptr,
                            a_grad_ptr,
                            b_ptr,
                            b_grad_ptr,
                            c_grad_ptr,
                            m,
                            n,
                            k,
                            batch,
                            stream);
  if (meta->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("BatchMatmul backward time = %.2lfms\n", elapsed);
  }
}

namespace Internal {

/*
A: (batch, n, k)
B: (batch, k, m)
O: (batch, n, m)
O = A * B
*/

void forward_kernel(BatchMatmulMeta const *meta,
                    float *o_ptr,
                    float const *a_ptr,
                    float const *b_ptr,
                    float const *c_ptr,
                    int m,
                    int n,
                    int k,
                    int batch,
                    hipStream_t stream,
                    int a_seq_length_dim,
                    int b_seq_length_dim,
                    int seq_length) {
  checkCUDA(hipblasSetStream(meta->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(meta->handle.dnn, stream));

  // int a_stride = n * k;
  // int b_stride = m * k;
  // int o_stride = n * m;
  int lda = k;
  int ldb = m;
  int ldo = m;
  long long int strideA = (long long int)n * k;
  long long int strideB = (long long int)k * m;
  long long int strideO = (long long int)n * m;
  if ((a_seq_length_dim == 0) && (seq_length >= 0)) {
    assert(seq_length <= k);
    k = seq_length;
    assert(b_seq_length_dim == 1);
  } else if ((a_seq_length_dim == 1) && (seq_length >= 0)) {
    assert(seq_length <= n);
    n = seq_length;
  } else {
    // currently only support a_seq_length_dim = 0 or 1
    assert((a_seq_length_dim < 0) || (seq_length < 0));
  }
  if ((b_seq_length_dim == 0) && (seq_length >= 0)) {
    assert(seq_length <= m);
    m = seq_length;
  } else if ((b_seq_length_dim == 1) && (seq_length >= 0)) {
    assert(a_seq_length_dim == 0);
    assert(k == seq_length);
  } else {
    // currently only support a_seq_length_dim = 0 or 1
    assert((b_seq_length_dim < 0) || (seq_length < 0));
  }

  float alpha = 1.0f, beta = 0.0f;
  checkCUDA(hipblasSgemmStridedBatched(meta->handle.blas,
                                      HIPBLAS_OP_N,
                                      HIPBLAS_OP_N,
                                      m,
                                      n,
                                      k,
                                      &alpha,
                                      b_ptr,
                                      ldb,
                                      strideB,
                                      a_ptr,
                                      lda,
                                      strideA,
                                      &beta,
                                      o_ptr,
                                      ldo,
                                      strideO,
                                      batch));
  // current assume c is null
  assert(c_ptr == NULL);
}

/*
A, AGrad: (batch, n, k)
B, BGrad: (batch, k, m)
O, OGrad: (batch, n, m)
AGrad = OGrad * B^T
BGrad = A^T * OGrad
*/
void backward_kernel(BatchMatmulMeta const *meta,
                     float const *o_ptr,
                     float const *o_grad_ptr,
                     float const *a_ptr,
                     float *a_grad_ptr,
                     float const *b_ptr,
                     float *b_grad_ptr,
                     float *c_grad_ptr,
                     int m,
                     int n,
                     int k,
                     int batch,
                     hipStream_t stream) {
  checkCUDA(hipblasSetStream(meta->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(meta->handle.dnn, stream));

  int a_stride = n * k;
  int b_stride = m * k;
  int o_stride = n * m;
  float alpha = 1.0f;
  checkCUDA(hipblasSgemmStridedBatched(meta->handle.blas,
                                      HIPBLAS_OP_T,
                                      HIPBLAS_OP_N,
                                      k,
                                      n,
                                      m,
                                      &alpha,
                                      b_ptr,
                                      m,
                                      b_stride,
                                      o_grad_ptr,
                                      m,
                                      o_stride,
                                      &alpha,
                                      a_grad_ptr,
                                      k,
                                      a_stride,
                                      batch));
  checkCUDA(hipblasSgemmStridedBatched(meta->handle.blas,
                                      HIPBLAS_OP_N,
                                      HIPBLAS_OP_T,
                                      m,
                                      k,
                                      n,
                                      &alpha,
                                      o_grad_ptr,
                                      m,
                                      o_stride,
                                      a_ptr,
                                      k,
                                      a_stride,
                                      &alpha,
                                      b_grad_ptr,
                                      m,
                                      b_stride,
                                      batch));
  assert(c_grad_ptr == NULL);
}

} // namespace Internal
} // namespace BatchMatmul
} // namespace Kernels
} // namespace FlexFlow
