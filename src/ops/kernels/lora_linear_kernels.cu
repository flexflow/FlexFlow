/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ffconst_utils.h"
#include "flexflow/ops/kernels/decompress_kernels.h"
#include "flexflow/ops/kernels/lora_linear_kernels.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

LoraLinearMeta::LoraLinearMeta(FFHandler handler, LoraLinear const *li)
    : OpMeta(handler, li) {}

LoraLinearMeta::~LoraLinearMeta(void) {}

namespace Kernels {
namespace LoraLinear {

void inference_kernel_wrapper(LoraLinearMeta *m,
                              BatchConfig const *bc,
                              GenericTensorAccessorR const &input,
                              GenericTensorAccessorW const &output) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  hipEvent_t t_start, t_end;
  int in_dim = input.domain.hi()[0] - input.domain.lo()[0] + 1;
  int out_dim = output.domain.hi()[0] - output.domain.lo()[0] + 1;

  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  if (m->input_type[0] == DT_FLOAT) {
    Internal::inference_kernel<float>(m,
                                      bc,
                                      input.get_float_ptr(),
                                      output.get_float_ptr(),
                                      in_dim,
                                      out_dim,
                                      stream);
  } else if (m->input_type[0] == DT_HALF) {
    Internal::inference_kernel<half>(m,
                                     bc,
                                     input.get_half_ptr(),
                                     output.get_half_ptr(),
                                     in_dim,
                                     out_dim,
                                     stream);
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [LoraLinear] forward time = %.2lfms\n", m->op_name, elapsed);
    // print_tensor<float>((float*)input_ptr, in_dim * batch_size,
    // "[LoraLinear:forward:input]"); print_tensor<float>((float*)weight_ptr,
    // in_dim
    // * out_dim, "[LoraLinear:forward:kernel]");
    // print_tensor<float>((float*)output_ptr, out_dim * batch_size,
    // "[LoraLinear:forward:output]");
  }
}

void peft_bwd_kernel_wrapper(LoraLinearMeta *m,
                             BatchConfig const *bc,
                             GenericTensorAccessorW const &input_grad,
                             GenericTensorAccessorR const &output_grad) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  int in_dim = input_grad.domain.hi()[0] - input_grad.domain.lo()[0] + 1;
  int out_dim = output_grad.domain.hi()[0] - output_grad.domain.lo()[0] + 1;
  if (m->input_type[0] == DT_FLOAT) {
    Internal::peft_bwd_kernel<float>(m,
                                     bc,
                                     input_grad.get_float_ptr(),
                                     output_grad.get_float_ptr(),
                                     in_dim,
                                     out_dim,
                                     stream);
  } else if (m->input_type[0] == DT_HALF) {
    Internal::peft_bwd_kernel<half>(m,
                                    bc,
                                    input_grad.get_half_ptr(),
                                    output_grad.get_half_ptr(),
                                    in_dim,
                                    out_dim,
                                    stream);
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [LoraLinear] PEFT Bwd time = %.2lfms\n", m->op_name, elapsed);
    // print_tensor<float>((float*)input_ptr, in_dim * batch_size,
    // "[LoraLinear:forward:input]"); print_tensor<float>((float*)weight_ptr,
    // in_dim
    // * out_dim, "[LoraLinear:forward:kernel]");
    // print_tensor<float>((float*)output_ptr, out_dim * batch_size,
    // "[LoraLinear:forward:output]");
  }
}

namespace Internal {

template <typename DT>
void inference_kernel(LoraLinearMeta *m,
                      BatchConfig const *bc,
                      DT const *input_ptr,
                      DT *output_ptr,
                      int in_dim,
                      int out_dim,
                      ffStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  DT alpha = 1.0f, beta = 0.0f;
  hipDataType input_type = ff_to_cuda_datatype(m->input_type[0]);
  hipDataType output_type = ff_to_cuda_datatype(m->input_type[1]);
  hipDataType lr_actv_type = output_type;
  assert(input_type == output_type);
  hipDataType weight_type = output_type;
#if defined(CUDA_VERSION) && (CUDA_VERSION < 11000)
  hipDataType compute_type = output_type;
#else
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#endif
  int num_peft_requests = 0;
  for (int i = 0; i < bc->max_requests_per_batch(); i++) {
    if (bc->request_completed[i]) {
      continue;
    }
    if (bc->requestsInfo[i].peft_model_id == PEFTModelID::NO_ID) {
      continue;
    }
    if (bc->requestsInfo[i].peft_bwd) {
      num_peft_requests++;
    }
  }
  // Assert that we have at most one request that requires peft_bwd
  assert(num_peft_requests <= 1);
  for (int i = 0; i < bc->max_requests_per_batch(); i++) {
    if (bc->request_completed[i]) {
      continue;
    }
    // Skip non-PEFT requests
    if (bc->requestsInfo[i].peft_model_id == PEFTModelID::NO_ID) {
      continue;
    }
    int num_peft_tokens = bc->requestsInfo[i].num_tokens_in_batch;
    int first_token_offset = bc->requestsInfo[i].first_token_offset_in_batch;
    assert(m->model_weights.find(bc->requestsInfo[i].peft_model_id) !=
           m->model_weights.end());
    LoraLinearWeight weight =
        m->model_weights[bc->requestsInfo[i].peft_model_id];
    int rank = weight.rank;
    void *intermediate_result_ptr = nullptr;
    if (bc->requestsInfo[i].peft_bwd) {
      MemoryAllocator *allocator = m->handle.peft_activation_allocator;
      m->input_activation = allocator->allocate_instance_untyped(
          data_type_size(m->input_type[0]) * num_peft_tokens * in_dim);
      m->low_rank_activation = allocator->allocate_instance_untyped(
          data_type_size(m->input_type[1]) * num_peft_tokens * rank);
      // copy input activation
      checkCUDA(hipMemcpyAsync(m->input_activation,
                                input_ptr + first_token_offset * in_dim,
                                data_type_size(m->input_type[0]) *
                                    num_peft_tokens * in_dim,
                                hipMemcpyDeviceToDevice,
                                stream));
      intermediate_result_ptr = m->low_rank_activation;
    } else {
      // use workspace to save intermediate result
      assert(m->handle.workSpaceSize >=
             data_type_size(m->input_type[1]) * num_peft_tokens * rank);
      intermediate_result_ptr = m->handle.workSpace;
    }
    // buffer = weight_first * input
    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_T,
                           HIPBLAS_OP_N,
                           rank,
                           num_peft_tokens,
                           in_dim,
                           &alpha,
                           weight.w0_ptr,
                           weight_type,
                           in_dim,
                           input_ptr + first_token_offset * in_dim,
                           input_type,
                           in_dim,
                           &beta,
                           intermediate_result_ptr,
                           lr_actv_type,
                           rank,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // output = weight_second * buffer
    // Note that we use alpha in both places since we do
    // an in-place update for LoraLinear
    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_T,
                           HIPBLAS_OP_N,
                           out_dim,
                           num_peft_tokens,
                           rank,
                           &alpha,
                           weight.w1_ptr,
                           weight_type,
                           rank,
                           intermediate_result_ptr,
                           lr_actv_type,
                           rank,
                           &alpha,
                           output_ptr + first_token_offset * out_dim,
                           output_type,
                           out_dim,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  }
}

template <typename DT>
void peft_bwd_kernel(LoraLinearMeta *m,
                     BatchConfig const *bc,
                     DT *input_grad_ptr,
                     DT const *output_grad_ptr,
                     int in_dim,
                     int out_dim,
                     ffStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  DT alpha = 1.0f;
  hipDataType input_type = ff_to_cuda_datatype(m->input_type[0]);
  hipDataType output_type = ff_to_cuda_datatype(m->output_type[0]);
  assert(input_type == output_type);
  hipDataType weight_type = output_type;
  hipDataType lr_actv_type = output_type;
#if defined(CUDA_VERSION) && (CUDA_VERSION < 11000)
  hipDataType compute_type = cublas_data_type;
#else
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#endif
  for (int i = 0; i < bc->max_requests_per_batch(); i++) {
    if (bc->request_completed[i]) {
      continue;
    }
    // Skip non-PEFT requests
    if (bc->requestsInfo[i].peft_model_id == PEFTModelID::NO_ID) {
      continue;
    }
    // Skip PEFT forward-only requests
    if (!bc->requestsInfo[i].peft_bwd) {
      continue;
    }
    int num_peft_tokens = bc->requestsInfo[i].num_tokens_in_batch;
    int first_token_offset = bc->requestsInfo[i].first_token_offset_in_batch;
    assert(m->model_weights.find(bc->requestsInfo[i].peft_model_id) !=
           m->model_weights.end());
    LoraLinearWeight weight =
        m->model_weights[bc->requestsInfo[i].peft_model_id];
    int rank = weight.rank;
    // Compute w1's gradient
    // NOTE: we use alpha=1 for w1_grad to accumulate gradients
    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_N,
                           HIPBLAS_OP_T,
                           rank,
                           out_dim,
                           num_peft_tokens,
                           &alpha,
                           m->low_rank_activation,
                           lr_actv_type,
                           rank,
                           output_grad_ptr,
                           output_type,
                           out_dim,
                           &alpha,
                           weight.w1_grad_ptr,
                           weight_type,
                           rank,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // Compute gradients w.r.t. low_rank activation
    // and save the results to low_rank_activation
    // NOTE: we use alpha=1 for input_grad to accumulate gradients
    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_N,
                           HIPBLAS_OP_N,
                           rank,
                           num_peft_tokens,
                           out_dim,
                           &alpha,
                           weight.w1_ptr,
                           weight_type,
                           rank,
                           output_grad_ptr,
                           output_type,
                           out_dim,
                           &alpha,
                           m->low_rank_activation,
                           lr_actv_type,
                           rank,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // Compute w0's gradient
    // NOTE: we use alpha=1 for kernel_grad to accumulate gradients
    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_N,
                           HIPBLAS_OP_T,
                           in_dim,
                           rank,
                           num_peft_tokens,
                           &alpha,
                           m->input_activation,
                           input_type,
                           in_dim,
                           m->low_rank_activation,
                           lr_actv_type,
                           rank,
                           &alpha,
                           weight.w0_grad_ptr,
                           weight_type,
                           in_dim,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // Compute input gradient
    // NOTE: we use alpha=1 for input_grad to accumulate gradients
    if (input_grad_ptr != nullptr) {
      checkCUDA(hipblasGemmEx(m->handle.blas,
                             HIPBLAS_OP_N,
                             HIPBLAS_OP_N,
                             in_dim,
                             num_peft_tokens,
                             rank,
                             &alpha,
                             weight.w0_ptr,
                             weight_type,
                             in_dim,
                             m->low_rank_activation,
                             lr_actv_type,
                             rank,
                             &alpha,
                             input_grad_ptr,
                             input_type,
                             in_dim,
                             compute_type,
                             CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    }
  }
}

} // namespace Internal
} // namespace LoraLinear
} // namespace Kernels
} // namespace FlexFlow
