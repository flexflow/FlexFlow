#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ffconst_utils.h"
#include "flexflow/ops/kernels/decompress_kernels.h"
#include "flexflow/ops/kernels/lora_linear_kernels.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

LoraLinearMeta::LoraLinearMeta(FFHandler handler, LoraLinear const *li)
    : OpMeta(handler, li) {
  allocated_peft_buffer_size1 = 0;
  allocated_peft_buffer_size2 = 0;
}

LoraLinearMeta::~LoraLinearMeta(void) {}

namespace Kernels {
namespace LoraLinear {

void inference_kernel_wrapper(LoraLinearMeta *m,
                              BatchConfig const *bc,
                              GenericTensorAccessorR const &input,
                              GenericTensorAccessorW const &output) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  hipEvent_t t_start, t_end;
  int in_dim = input.domain.hi()[0] - input.domain.lo()[0] + 1;
  int out_dim = output.domain.hi()[0] - output.domain.lo()[0] + 1;

  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  if (m->input_type[0] == DT_FLOAT) {
    Internal::inference_kernel<float>(m,
                                      bc,
                                      input.get_float_ptr(),
                                      output.get_float_ptr(),
                                      in_dim,
                                      out_dim,
                                      stream);
  } else if (m->input_type[0] == DT_HALF) {
    Internal::inference_kernel<half>(m,
                                     bc,
                                     input.get_half_ptr(),
                                     output.get_half_ptr(),
                                     in_dim,
                                     out_dim,
                                     stream);
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [LoraLinear] forward time = %.2lfms\n", m->op_name, elapsed);
    // print_tensor<float>((float*)input_ptr, in_dim * batch_size,
    // "[LoraLinear:forward:input]"); print_tensor<float>((float*)weight_ptr,
    // in_dim
    // * out_dim, "[LoraLinear:forward:kernel]");
    // print_tensor<float>((float*)output_ptr, out_dim * batch_size,
    // "[LoraLinear:forward:output]");
  }
}

void peft_bwd_kernel_wrapper(LoraLinearMeta *m,
                             BatchConfig const *bc,
                             GenericTensorAccessorW const &input_grad,
                             GenericTensorAccessorR const &output_grad) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  int in_dim = input_grad.domain.hi()[0] - input_grad.domain.lo()[0] + 1;
  int out_dim = output_grad.domain.hi()[0] - output_grad.domain.lo()[0] + 1;
  if (m->input_type[0] == DT_FLOAT) {
    Internal::peft_bwd_kernel<float>(m,
                                     bc,
                                     input_grad.get_float_ptr(),
                                     output_grad.get_float_ptr(),
                                     in_dim,
                                     out_dim,
                                     stream);
  } else if (m->input_type[0] == DT_HALF) {
    Internal::peft_bwd_kernel<half>(m,
                                    bc,
                                    input_grad.get_half_ptr(),
                                    output_grad.get_half_ptr(),
                                    in_dim,
                                    out_dim,
                                    stream);
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [LoraLinear] PEFT Bwd time = %.2lfms\n", m->op_name, elapsed);
    // print_tensor<float>((float*)input_ptr, in_dim * batch_size,
    // "[LoraLinear:forward:input]"); print_tensor<float>((float*)weight_ptr,
    // in_dim
    // * out_dim, "[LoraLinear:forward:kernel]");
    // print_tensor<float>((float*)output_ptr, out_dim * batch_size,
    // "[LoraLinear:forward:output]");
  }
}

namespace Internal {

template <typename DT>
void inference_kernel(LoraLinearMeta *m,
                      BatchConfig const *bc,
                      DT const *input_ptr,
                      DT *output_ptr,
                      int in_dim,
                      int out_dim,
                      ffStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  DT alpha = 1.0f, beta = 0.0f;
  hipDataType input_type = ff_to_cuda_datatype(m->input_type[0]);
  hipDataType output_type = ff_to_cuda_datatype(m->input_type[1]);
  hipDataType lr_actv_type = output_type;
  assert(input_type == output_type);
  hipDataType weight_type = output_type;
  hipDataType compute_type = output_type;
  // #if defined(CUDA_VERSION) && (CUDA_VERSION < 11000)
  //   hipDataType compute_type = output_type;
  // #else
  //   // For best performance, set the default cublas compute type to
  //   // HIPBLAS_COMPUTE_16F for half precision and to
  //   // HIPBLAS_COMPUTE_32F_FAST_16F for full precision
  //   hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
  //   if (m->input_type[0] == DT_FLOAT) {
  //     compute_type = HIPBLAS_COMPUTE_32F_FAST_16F;
  //   }
  // #endif
  int num_peft_requests = 0;
  for (int i = 0; i < bc->max_requests_per_batch(); i++) {
    if (bc->request_completed[i]) {
      continue;
    }
    if (bc->requestsInfo[i].peft_model_id == PEFTModelID::NO_ID) {
      continue;
    }
    if (bc->requestsInfo[i].peft_bwd) {
      num_peft_requests++;
    }
  }
  // Assert that we have at most one request that requires peft_bwd
  assert(num_peft_requests <= 1);
  for (int i = 0; i < bc->max_requests_per_batch(); i++) {
    if (bc->request_completed[i]) {
      continue;
    }
    // Skip non-PEFT requests
    if (bc->requestsInfo[i].peft_model_id == PEFTModelID::NO_ID) {
      continue;
    }
    int num_peft_tokens = bc->requestsInfo[i].num_tokens_in_batch;
    int max_peft_tokens = bc->requestsInfo[i].max_sequence_length;
    int first_token_offset = bc->requestsInfo[i].first_token_offset_in_batch;
    assert(m->model_state.find(bc->requestsInfo[i].peft_model_id) !=
           m->model_state.end());
    LoraLinearWeight weight =
        m->model_state[bc->requestsInfo[i].peft_model_id].weights;
    int rank = weight.rank;
    void *intermediate_result_ptr = nullptr;
    if (bc->requestsInfo[i].peft_bwd) {
      size_t activation_size_needed1 =
          data_type_size(m->input_type[0]) * max_peft_tokens * in_dim;
      size_t activation_size_needed2 =
          data_type_size(m->input_type[1]) * max_peft_tokens * rank;
      MemoryAllocator *allocator = m->handle.peft_activation_allocator;
      if (activation_size_needed1 > m->allocated_peft_buffer_size1) {
        m->input_activation =
            allocator->allocate_instance_untyped(activation_size_needed1);
        m->allocated_peft_buffer_size1 = activation_size_needed1;
      }
      if (activation_size_needed2 > m->allocated_peft_buffer_size2) {
        m->low_rank_activation =
            allocator->allocate_instance_untyped(activation_size_needed2);
        m->allocated_peft_buffer_size2 = activation_size_needed2;
      }
      // copy input activation
      checkCUDA(hipMemcpyAsync(m->input_activation,
                                input_ptr + first_token_offset * in_dim,
                                data_type_size(m->input_type[0]) *
                                    num_peft_tokens * in_dim,
                                hipMemcpyDeviceToDevice,
                                stream));
      intermediate_result_ptr = m->low_rank_activation;
    } else {
      // use workspace to save intermediate result
      assert(m->handle.workSpaceSize >=
             data_type_size(m->input_type[1]) * num_peft_tokens * rank);
      intermediate_result_ptr = m->handle.workSpace;
    }
    // buffer = weight_first * input
    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_T,
                           HIPBLAS_OP_N,
                           rank,
                           num_peft_tokens,
                           in_dim,
                           &alpha,
                           weight.w0_ptr,
                           weight_type,
                           in_dim,
                           input_ptr + first_token_offset * in_dim,
                           input_type,
                           in_dim,
                           &beta,
                           intermediate_result_ptr,
                           lr_actv_type,
                           rank,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // output = weight_second * buffer
    // Note that we use alpha in both places since we do
    // an in-place update for LoraLinear
    double lora_alpha =
        m->model_state[bc->requestsInfo[i].peft_model_id].lora_alpha;
    DT scaling_constant = (DT)(lora_alpha / rank);
    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_T,
                           HIPBLAS_OP_N,
                           out_dim,
                           num_peft_tokens,
                           rank,
                           &scaling_constant,
                           weight.w1_ptr,
                           weight_type,
                           rank,
                           intermediate_result_ptr,
                           lr_actv_type,
                           rank,
                           &alpha,
                           output_ptr + first_token_offset * out_dim,
                           output_type,
                           out_dim,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  }
}

template <typename DT>
__global__ void sgd_update(size_t count,
                           float lr,
                           float weight_decay,
                           float momentum,
                           bool nesterov,
                           DT const *WGrad,
                           DT *V,
                           DT *W) {
  // Refernce https://pytorch.org/docs/stable/_modules/torch/optim/sgd.html#SGD
  CUDA_KERNEL_LOOP(i, count) {
    DT gt = WGrad[i] + (DT)weight_decay * W[i];
    if (momentum > 0.0f) {
      V[i] = V[i] * (DT)momentum + gt;
      if (nesterov) {
        gt = gt + (DT)momentum * V[i];
      } else {
        gt = V[i];
      }
    }
    W[i] -= (DT)lr * gt;
  }
}

template <typename DT>
void peft_bwd_kernel(LoraLinearMeta *m,
                     BatchConfig const *bc,
                     DT *input_grad_ptr,
                     DT const *output_grad_ptr,
                     int in_dim,
                     int out_dim,
                     ffStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  hipDataType input_type = ff_to_cuda_datatype(m->input_type[0]);
  hipDataType output_type = ff_to_cuda_datatype(m->output_type[0]);
  assert(input_type == output_type);
  hipDataType weight_type = output_type;
  hipDataType lr_actv_type = output_type;
  hipDataType compute_type = output_type;
  // #if defined(CUDA_VERSION) && (CUDA_VERSION < 11000)
  //   hipDataType compute_type = output_type;
  // #else
  //   // For best performance, set the default cublas compute type to
  //   // HIPBLAS_COMPUTE_16F for half precision and to
  //   // HIPBLAS_COMPUTE_32F_FAST_16F for full precision
  //   hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
  //   if (m->output_type[0] == DT_FLOAT) {
  //     compute_type = HIPBLAS_COMPUTE_32F_FAST_16F;
  //   }
  // #endif
  for (int i = 0; i < bc->max_requests_per_batch(); i++) {
    if (bc->request_completed[i]) {
      continue;
    }
    // Skip non-PEFT requests
    if (bc->requestsInfo[i].peft_model_id == PEFTModelID::NO_ID) {
      continue;
    }
    // Skip PEFT forward-only requests
    if (!bc->requestsInfo[i].peft_bwd) {
      continue;
    }
    int num_peft_tokens = bc->requestsInfo[i].num_tokens_in_batch;
    // int first_token_offset = bc->requestsInfo[i].first_token_offset_in_batch;
    assert(m->model_state.find(bc->requestsInfo[i].peft_model_id) !=
           m->model_state.end());
    LoraLinearWeight weight =
        m->model_state[bc->requestsInfo[i].peft_model_id].weights;
    int rank = weight.rank;
    // Compute w1's gradient
    DT alpha = 1.0f, beta = 0.0f;
    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_N,
                           HIPBLAS_OP_T,
                           rank,
                           out_dim,
                           num_peft_tokens,
                           &alpha,
                           m->low_rank_activation,
                           lr_actv_type,
                           rank,
                           output_grad_ptr,
                           output_type,
                           out_dim,
                           &beta,
                           weight.w1_grad_ptr,
                           weight_type,
                           rank,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // Compute gradients w.r.t. low_rank activation
    // and save the results to low_rank_activation
    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_N,
                           HIPBLAS_OP_N,
                           rank,
                           num_peft_tokens,
                           out_dim,
                           &alpha,
                           weight.w1_ptr,
                           weight_type,
                           rank,
                           output_grad_ptr,
                           output_type,
                           out_dim,
                           &beta,
                           m->low_rank_activation,
                           lr_actv_type,
                           rank,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // Compute w0's gradient
    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_N,
                           HIPBLAS_OP_T,
                           in_dim,
                           rank,
                           num_peft_tokens,
                           &alpha,
                           m->input_activation,
                           input_type,
                           in_dim,
                           m->low_rank_activation,
                           lr_actv_type,
                           rank,
                           &beta,
                           weight.w0_grad_ptr,
                           weight_type,
                           in_dim,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // Compute input gradient
    // NOTE: we use beta=1 for input_grad to accumulate gradients when needed
    if (input_grad_ptr != nullptr) {
      beta = m->reset_input_grads[0] ? 0.0f : 1.0f;
      checkCUDA(hipblasGemmEx(m->handle.blas,
                             HIPBLAS_OP_N,
                             HIPBLAS_OP_N,
                             in_dim,
                             num_peft_tokens,
                             rank,
                             &alpha,
                             weight.w0_ptr,
                             weight_type,
                             in_dim,
                             m->low_rank_activation,
                             lr_actv_type,
                             rank,
                             &beta,
                             input_grad_ptr,
                             input_type,
                             in_dim,
                             compute_type,
                             CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    }
    if (bc->requestsInfo[i].gradients_update_mode !=
        GradientsUpdateMode::ACCUMULATE_ONLY) {
      LoraOptimizerConfig const *optimizer_config =
          m->model_state[bc->requestsInfo[i].peft_model_id].optimizer_config;
      assert(optimizer_config != nullptr);
      assert(typeid(*optimizer_config) != typeid(LoraOptimizerConfig));
      int w0_num_elements = rank * in_dim;
      int w1_num_elements = rank * out_dim;

      // Get optimizer config
      if (typeid(*optimizer_config) == typeid(LoraSGDOptimizerConfig)) {
        LoraSGDOptimizerConfig const *sgd_config =
            (LoraSGDOptimizerConfig const *)optimizer_config;
        // LoRA_A weight is split in tensor parallelism, so no need to apply
        // all-reduce
        sgd_update<<<GET_BLOCKS(w0_num_elements),
                     CUDA_NUM_THREADS,
                     0,
                     stream>>>(w0_num_elements,
                               sgd_config->lr,
                               sgd_config->weight_decay,
                               sgd_config->momentum,
                               sgd_config->nesterov,
                               static_cast<DT const *>(weight.w0_grad_ptr),
                               static_cast<DT *>(weight.w0_v_values_ptr),
                               static_cast<DT *>(weight.w0_ptr));
        // LoRA_B weight is replicated w tensor parallelism, so we need to sync
        // and sum first
        ncclDataType_t nccl_data_type = ff_to_nccl_datatype(m->output_type[0]);
        checkCUDA(ncclAllReduce(static_cast<DT const *>(weight.w1_grad_ptr),
                                static_cast<DT *>(weight.w1_grad_ptr),
                                w1_num_elements,
                                nccl_data_type,
                                ncclSum,
                                m->handle.ncclComm,
                                stream));
        double lora_alpha =
            m->model_state[bc->requestsInfo[i].peft_model_id].lora_alpha;
        double lr_with_scaling = sgd_config->lr / (lora_alpha / rank);
        sgd_update<<<GET_BLOCKS(w1_num_elements),
                     CUDA_NUM_THREADS,
                     0,
                     stream>>>(w1_num_elements,
                               lr_with_scaling,
                               sgd_config->weight_decay,
                               sgd_config->momentum,
                               sgd_config->nesterov,
                               static_cast<DT const *>(weight.w1_grad_ptr),
                               static_cast<DT *>(weight.w1_v_values_ptr),
                               static_cast<DT *>(weight.w1_ptr));
      } else if (typeid(*optimizer_config) == typeid(LoraAdamOptimizerConfig)) {
        assert(false && "Adam optimizer type not implemented yet");
      } else {
        assert(false && "Unsupported optimizer type");
      }
    }
  }
}

} // namespace Internal
} // namespace LoraLinear
} // namespace Kernels
} // namespace FlexFlow
