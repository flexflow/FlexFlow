#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/kernels/element_binary_kernels.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {
// declare Legion names
using Legion::coord_t;
using Legion::Domain;

ElementBinaryMeta::ElementBinaryMeta(FFHandler handler) : OpMeta(handler) {
  checkCUDNN(hipdnnCreateTensorDescriptor(&input1Tensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&input2Tensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateOpTensorDescriptor(&opDesc));
  checkCUDNN(hipdnnCreateReduceTensorDescriptor(&reduceAddDesc));
  op_type = OP_NOOP;
  profiling = false;
  inplace_a = false;
  has_same_operands = false;
  broadcast_input1 = false;
  broadcast_input2 = false;
}

namespace Kernels {
namespace ElementBinary {

/*static*/
void init_kernel(ElementBinaryMeta *m,
                 Domain const &input1_domain,
                 Domain const &input2_domain,
                 Domain const &output_domain) {
  hipdnnOpTensorOp_t mode;
  switch (m->op_type) {
    case OP_EW_ADD:
    case OP_EW_SUB:
      mode = HIPDNN_OP_TENSOR_ADD;
      break;
    case OP_EW_MUL:
      mode = HIPDNN_OP_TENSOR_MUL;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnSetOpTensorDescriptor(
      m->opDesc, mode, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN));
  checkCUDNN(hipdnnSetReduceTensorDescriptor(m->reduceAddDesc,
                                            HIPDNN_REDUCE_TENSOR_ADD,
                                            HIPDNN_DATA_FLOAT,
                                            HIPDNN_PROPAGATE_NAN,
                                            HIPDNN_REDUCE_TENSOR_NO_INDICES,
                                            HIPDNN_32BIT_INDICES));
  checkCUDNN(
      cudnnSetTensorDescriptorFromDomain(m->input1Tensor, input1_domain));
  checkCUDNN(
      cudnnSetTensorDescriptorFromDomain(m->input2Tensor, input2_domain));
  checkCUDNN(
      cudnnSetTensorDescriptorFromDomain(m->outputTensor, output_domain));
}

/*static*/
void forward_kernel_wrapper(ElementBinaryMeta const *m,
                            float const *in1_ptr,
                            float const *in2_ptr,
                            float *out_ptr) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  Internal::forward_kernel(m, in1_ptr, in2_ptr, out_ptr, stream);
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    char const *opName;
    switch (m->op_type) {
      case OP_EW_ADD:
        opName = "Add";
        break;
      case OP_EW_SUB:
        opName = "Sub";
        break;
      case OP_EW_MUL:
        opName = "Mul";
        break;
      case OP_EW_DIV:
        opName = "Div";
        break;
      default:
        assert(false);
    }
    printf("[%s] forward time (CF) = %.2fms\n", m->op_name, elapsed);
    // print_tensor<float>(in1_ptr, 32, "[EWB:forward:input1]");
    // print_tensor<float>(in2_ptr, 32, "[EWB:forward:input2]");
    // print_tensor<float>(out_ptr, 32, "[EWB:forward:output]");
  }
}

/*static*/
void backward_kernel_wrapper(ElementBinaryMeta const *m,
                             float const *out_grad_ptr,
                             float const *in1_ptr,
                             float const *in2_ptr,
                             float *in1_grad_ptr,
                             float *in2_grad_ptr) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  Internal::backward_kernel(
      m, out_grad_ptr, in1_ptr, in2_ptr, in1_grad_ptr, in2_grad_ptr, stream);
  // elewise_binary_backward_kernel<<<GET_BLOCKS(out_grad_domain.get_volume()),
  // CUDA_NUM_THREADS>>>( out_grad_domain.get_volume(), alpha, alpha,
  // ele->op_type, out_grad_ptr, in1_ptr, in2_ptr, in1_grad_ptr, in2_grad_ptr);
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    char const *opName;
    switch (m->op_type) {
      case OP_EW_ADD:
        opName = "Add";
        break;
      case OP_EW_SUB:
        opName = "Sub";
        break;
      case OP_EW_MUL:
        opName = "Mul";
        break;
      case OP_EW_DIV:
        opName = "Div";
        break;
      default:
        assert(false);
    }
    printf("[%s] backward time (CB) = %.2fms\n", opName, elapsed);
  }
}

namespace Internal {

__global__ void elewise_binary_forward_kernel(coord_t volume,
                                              float const alpha,
                                              float const beta,
                                              OperatorType type,
                                              float const *in1,
                                              float const *in2,
                                              float *out) {
  switch (type) {
    case OP_EW_ADD: {
      CUDA_KERNEL_LOOP(i, volume) {
        out[i] = alpha * (in1[i] + in2[i]) + beta * out[i];
      }
      break;
    }
    case OP_EW_SUB: {
      CUDA_KERNEL_LOOP(i, volume) {
        out[i] = alpha * (in1[i] - in2[i]) + beta * out[i];
      }
      break;
    }
    case OP_EW_MUL: {
      CUDA_KERNEL_LOOP(i, volume) {
        out[i] = alpha * in1[i] * in2[i] + beta * out[i];
      }
      break;
    }
    case OP_EW_DIV: {
      CUDA_KERNEL_LOOP(i, volume) {
        out[i] = alpha * (in1[i] / in2[i]) + beta * out[i];
      }
      break;
    }
    default:
      assert(false);
  }
}

__global__ void elewise_binary_backward_kernel(coord_t volume,
                                               float const alpha,
                                               float const beta,
                                               OperatorType type,
                                               float const *out_grad,
                                               float const *in1,
                                               float const *in2,
                                               float *in1_grad,
                                               float *in2_grad) {
  CUDA_KERNEL_LOOP(i, volume) {
    switch (type) {
      case OP_EW_ADD: {
        in1_grad[i] = alpha * out_grad[i] + beta * in1_grad[i];
        in2_grad[i] = alpha * out_grad[i] + beta * in2_grad[i];
        break;
      }
      case OP_EW_SUB: {
        in1_grad[i] = alpha * out_grad[i] + beta * in1_grad[i];
        in2_grad[i] = -alpha * out_grad[i] + beta * in2_grad[i];
        break;
      }
      case OP_EW_MUL: {
        in1_grad[i] = alpha * out_grad[i] * in2[i] + beta * in1_grad[i];
        in2_grad[i] = alpha * out_grad[i] * in1[i] + beta * in2_grad[i];
        break;
      }
      case OP_EW_DIV: {
        in1_grad[i] = alpha * out_grad[i] / in2[i] + beta * in1_grad[i];
        in2_grad[i] = -alpha * out_grad[i] * in1[i] / (in2[i] * in2[i]) +
                      beta * in2_grad[i];
        break;
      }
      default:
        assert(false);
    }
  }
}

/*static*/
void forward_kernel(ElementBinaryMeta const *m,
                    float const *in1_ptr,
                    float const *in2_ptr,
                    float *out_ptr,
                    hipStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  float alpha1 = 1.0f, alpha2 = 1.0f, beta = 0.0f;
  switch (m->op_type) {
    case OP_EW_SUB:
      alpha2 = -1.0f;
      break;
    case OP_EW_ADD:
    case OP_EW_MUL:
      break;
    default:
      assert(false);
  }
  // cudnn currently does not support broadcasting the first input in
  // hipdnnOpTensor
  if (m->broadcast_input1) {
    // currently only handle add and sub
    assert(m->op_type == OP_EW_SUB || m->op_type == OP_EW_ADD);
    checkCUDNN(hipdnnOpTensor(m->handle.dnn,
                             m->opDesc,
                             &beta,
                             m->outputTensor,
                             out_ptr,
                             &alpha1,
                             m->input1Tensor,
                             in1_ptr,
                             &beta,
                             m->outputTensor,
                             out_ptr));
    checkCUDNN(hipdnnOpTensor(m->handle.dnn,
                             m->opDesc,
                             &beta,
                             m->outputTensor,
                             out_ptr,
                             &alpha2,
                             m->input2Tensor,
                             in2_ptr,
                             &alpha1,
                             m->outputTensor,
                             out_ptr));
  } else {
    checkCUDNN(hipdnnOpTensor(m->handle.dnn,
                             m->opDesc,
                             &alpha1,
                             m->input1Tensor,
                             in1_ptr,
                             &alpha2,
                             m->input2Tensor,
                             in2_ptr,
                             &beta,
                             m->outputTensor,
                             out_ptr));
  }
}

/*static*/
void backward_kernel(ElementBinaryMeta const *m,
                     float const *out_grad_ptr,
                     float const *in1_ptr,
                     float const *in2_ptr,
                     float *in1_grad_ptr,
                     float *in2_grad_ptr,
                     hipStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  if (m->op_type == OP_EW_ADD || m->op_type == OP_EW_SUB) {
    float alpha = 1.0f, beta = 1.0f;
    if (in1_grad_ptr != nullptr) {
      if (m->broadcast_input1) {
        checkCUDNN(hipdnnReduceTensor(m->handle.dnn,
                                     m->reduceAddDesc,
                                     nullptr /*indices*/,
                                     0 /*indicesSizeInBytes*/,
                                     m->handle.workSpace,
                                     m->handle.workSpaceSize,
                                     &alpha,
                                     m->outputTensor,
                                     out_grad_ptr,
                                     &beta,
                                     m->input1Tensor,
                                     in1_grad_ptr));
      } else {
        checkCUDNN(hipdnnAddTensor(m->handle.dnn,
                                  &alpha,
                                  m->outputTensor,
                                  out_grad_ptr,
                                  &beta,
                                  m->input1Tensor,
                                  in1_grad_ptr));
      }
    }
    if (m->op_type == OP_EW_SUB)
      alpha = -1.0f;
    if (in2_grad_ptr != nullptr) {
      if (m->broadcast_input2) {
        checkCUDNN(hipdnnReduceTensor(m->handle.dnn,
                                     m->reduceAddDesc,
                                     nullptr /*indices*/,
                                     0 /*indicesSizeInBytes*/,
                                     m->handle.workSpace,
                                     m->handle.workSpaceSize,
                                     &alpha,
                                     m->outputTensor,
                                     out_grad_ptr,
                                     &beta,
                                     m->input2Tensor,
                                     in2_grad_ptr));
      } else {
        checkCUDNN(hipdnnAddTensor(m->handle.dnn,
                                  &alpha,
                                  m->outputTensor,
                                  out_grad_ptr,
                                  &beta,
                                  m->input2Tensor,
                                  in2_grad_ptr));
      }
    }
  } else if (m->op_type == OP_EW_MUL) {
    float alpha1 = 1.0f, alpha2 = 1.0f, beta = 1.0f;
    if (in1_grad_ptr != nullptr) {
      checkCUDNN(hipdnnOpTensor(m->handle.dnn,
                               m->opDesc,
                               &alpha1,
                               m->outputTensor,
                               out_grad_ptr,
                               &alpha2,
                               m->input2Tensor,
                               in2_ptr,
                               &beta,
                               m->input1Tensor,
                               in1_grad_ptr));
    }
    if (in2_grad_ptr != nullptr) {
      checkCUDNN(hipdnnOpTensor(m->handle.dnn,
                               m->opDesc,
                               &alpha1,
                               m->outputTensor,
                               out_grad_ptr,
                               &alpha2,
                               m->input2Tensor,
                               in1_ptr,
                               &beta,
                               m->input1Tensor,
                               in2_grad_ptr));
    }
  } else {
    assert(false && "Unsupported ElementWise Binary Type");
  }
}

} // namespace Internal
} // namespace ElementBinary
} // namespace Kernels
}; // namespace FlexFlow
