/* Copyright 2018 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::pool2d(const Tensor& input,
                       int kernelH, int kernelW,
                       int strideH, int strideW,
                       int paddingH, int paddingW,
                       PoolType type, ActiMode activation,
                       char const *name)
{
  assert(input.numDim == 4); /*NCHW*/
  Pool2D *pool = new Pool2D(*this, input, kernelH, kernelW,
                      strideH, strideW, paddingH, paddingW,
                      type, activation, name);
  layers.push_back(pool);
  return pool->outputs[0];
}

Pool2D* FFModel::pool2d(int kernelH, int kernelW,
                        int strideH, int strideW,
                        int paddingH, int paddingW,
                        PoolType type, ActiMode activation,
                        char const *name)
{
  Pool2D *pool = new Pool2D(*this, kernelH, kernelW,
                      strideH, strideW, paddingH, paddingW,
                      type, activation, name);
  layers.push_back(pool);
  return pool;
}

Pool2D::Pool2D(FFModel& model,
               const Tensor& _input,
               int _kernel_h, int _kernel_w,
               int _stride_h, int _stride_w,
               int _padding_h, int _padding_w,
               PoolType _type, ActiMode _activation,
               const char* name)
: Op(model, OP_POOL2D, name, _input),
  kernel_h(_kernel_h), kernel_w(_kernel_w),
  stride_h(_stride_h), stride_w(_stride_w),
  padding_h(_padding_h), padding_w(_padding_w),
  pool_type(_type), activation(_activation),
  profiling(model.config.profiling)
{
  int input_w = inputs[0].adim[0];
  int input_h = inputs[0].adim[1];
  int output_w = 1 + (input_w + 2 * padding_w - kernel_w) / stride_w;
  int output_h = 1 + (input_h + 2 * padding_h - kernel_h) / stride_h;
  int output_c = inputs[0].adim[2];
  int output_n = inputs[0].adim[3];
  outputs[0].numDim = 4;
  outputs[0].adim[0] = output_w;
  outputs[0].adim[1] = output_h;
  outputs[0].adim[2] = output_c;
  outputs[0].adim[3] = output_n;
}

Pool2D::Pool2D(FFModel& model,
               int _kernel_h, int _kernel_w,
               int _stride_h, int _stride_w,
               int _padding_h, int _padding_w,
               PoolType _type, ActiMode _activation,
               const char* name)
: Op(model, OP_POOL2D, name, 1),
  kernel_h(_kernel_h), kernel_w(_kernel_w),
  stride_h(_stride_h), stride_w(_stride_w),
  padding_h(_padding_h), padding_w(_padding_w),
  pool_type(_type), activation(_activation),
  profiling(model.config.profiling)
{
}

Tensor Pool2D::init_inout(FFModel& model, const Tensor& _input)
{
  inputs[0] = _input;
  create_output_and_partition(model);
  return outputs[0];
}


void Pool2D::create_weights(FFModel& model)
{
  // Do nothing since we don't have any weight
}

/*
void Pool2D::add_to_model(FFModel& model)
{
  model.layers.push_back(this);
}
*/

void Pool2D::create_output_and_partition(FFModel& model)
{
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  std::string pcname = name;
  task_is = IndexSpaceT<4>(model.get_or_create_task_is(4, pcname));
  Rect<4> part_rect = runtime->get_index_space_domain(ctx, task_is);

  int input_w = inputs[0].adim[0];
  int input_h = inputs[0].adim[1];
  int output_w = 1 + (input_w + 2 * padding_w - kernel_w) / stride_w;
  int output_h = 1 + (input_h + 2 * padding_h - kernel_h) / stride_h;
  int output_c = inputs[0].adim[2];
  int output_n = inputs[0].adim[3];
  {
    const int dims[4] = {output_n, output_c, output_h, output_w};
    outputs[0] = model.create_tensor<4>(dims, DT_FLOAT, this);
    outputs[0].owner_op = this;
    outputs[0].owner_idx = 0;
  }
  //int num_par_w = part_rect.hi[0] - part_rect.lo[0] + 1;
  //int num_par_h = part_rect.hi[1] - part_rect.lo[1] + 1;
  int num_par_c = part_rect.hi[2] - part_rect.lo[2] + 1;
  //int num_par_n = part_rect.hi[3] - part_rect.lo[3] + 1;
  Rect<4> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part.get_index_partition());
  //TODO: currently do not support splitting over the channel dimension
  assert(num_par_c == 1);
  if (input_rect == part_rect) {
    input_lps[0] = inputs[0].part;
    input_grad_lps[0] = inputs[0].part_grad;
  } else {
    model.create_disjoint_partition(
        inputs[0], (IndexSpaceT<4>)task_is, input_lps[0], input_grad_lps[0]);
  }
}

/*
  regions[0]: input
  regions[1]: output
*/
OpMeta* Pool2D::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  const Pool2D* pool = (Pool2D*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  Pool2DMeta* m = new Pool2DMeta(handle);
  TensorAccessorR<float, 4> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 4> acc_output(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);

  int input_w = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int input_h = acc_input.rect.hi[1] - acc_input.rect.lo[1] + 1;
  int input_c = acc_input.rect.hi[2] - acc_input.rect.lo[2] + 1;
  int input_n = acc_input.rect.hi[3] - acc_input.rect.lo[3] + 1;
  int output_w = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int output_h = acc_output.rect.hi[1] - acc_output.rect.lo[1] + 1;
  int output_c = acc_output.rect.hi[2] - acc_output.rect.lo[2] + 1;
  int output_n = acc_output.rect.hi[3] - acc_output.rect.lo[3] + 1;

  printf("init pool (input): n(%d) c(%d) h(%d) w(%d)\n",
         input_n, input_c, input_h, input_w);
  printf("init pool (output): n(%d) c(%d) h(%d) w(%d)\n",
         output_n, output_c, output_h, output_w);
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        input_n,
                                        input_c,
                                        input_h,
                                        input_w));
  int pad_h = ((output_h - 1) * pool->stride_h + pool->kernel_h - input_h + 1) / 2;
  int pad_w = ((output_w - 1) * pool->stride_w + pool->kernel_w - input_w + 1) / 2;
  if (pad_h != pool->padding_h)
    printf("Warning: changing pool_padding_h to satisfy output_h size\n");
  if (pad_w != pool->padding_w)
    printf("Warning: changing pool_padding_w to satisfy output_w size\n");

  hipdnnPoolingMode_t mode;
  if (pool->pool_type == POOL_MAX)
    mode = HIPDNN_POOLING_MAX;
  else {
    assert(pool->pool_type == POOL_AVG);
    mode = HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;
  }
  checkCUDNN(hipdnnSetPooling2dDescriptor(m->poolDesc,
                                         mode,
                                         HIPDNN_PROPAGATE_NAN,
                                         pool->kernel_h,
                                         pool->kernel_w,
                                         pad_h,//pool->padding_h,
                                         pad_w,//pool->padding_w,
                                         pool->stride_h,
                                         pool->stride_w));
  int n, c, h, w;
  checkCUDNN(hipdnnGetPooling2dForwardOutputDim(m->poolDesc,
                                               m->inputTensor,
                                               &n, &c, &h, &w));
  assert(n == output_n);
  assert(c == output_c);
  assert(h == output_h);
  assert(w == output_w);

  checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        n, c, h, w));
  return m;
}

void Pool2D::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<4> rect = runtime->get_index_space_domain(ctx, task_is);
  ParallelConfig pc;
  std::string pcname = name;
  ff.config.find_parallel_config(4, pcname, pc);
  int idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    FFHandler handle = ff.handlers[pc.device_ids[idx++]];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
  }
  IndexLauncher init_launcher(POOL2D_INIT_TASK_ID, task_is,
                              TaskArgument(this, sizeof(Pool2D)), argmap,
                              Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                              FFConfig::get_hash_id(std::string(name)));
  init_launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  init_launcher.add_field(0, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, outputs[0].region));
  init_launcher.add_field(1, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, init_launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/*static*/
void Pool2D::forward_kernel(const Pool2DMeta* m,
                            const float* input_ptr,
                            float* output_ptr)
{
  float alpha = 1.0f, beta = 0.0f;
  checkCUDNN(hipdnnPoolingForward(m->handle.dnn, m->poolDesc,
                                 &alpha, m->inputTensor, input_ptr,
                                 &beta, m->outputTensor, output_ptr));
}

/*
  regions[0](I): input
  regions[1](O): output
*/
void Pool2D::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  const Pool2D* pool = (Pool2D*) task->args;
  const Pool2DMeta* m = *((Pool2DMeta**) task->local_args);
  TensorAccessorR<float, 4> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 4> acc_output(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  hipEvent_t t_start, t_end;
  if (pool->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  forward_kernel(m, acc_input.ptr, acc_output.ptr);
  if (pool->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    //print_tensor<4, float>(acc_input.ptr, acc_input.rect, "[Pool2D:forward:input]");
    //print_tensor<4, float>(acc_output.ptr, acc_output.rect, "[Pool2D:forward:output]");
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Pool2D] forward time = %.2fms\n", pool->name, elapsed);
  }
}

void Pool2D::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<4> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(POOL2D_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Pool2D)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, outputs[0].region));
  launcher.add_field(1, FID_DATA);

  runtime->execute_index_space(ctx, launcher);
}

/*static*/
void Pool2D::backward_kernel(const Pool2DMeta* m,
                             const float* input_ptr,
                             float* input_grad_ptr,
                             const float* output_ptr,
                             const float* output_grad_ptr)
{
  float alpha = 1.0f;
  checkCUDNN(hipdnnPoolingBackward(m->handle.dnn, m->poolDesc,
                                  &alpha, m->outputTensor, output_ptr,
                                  m->outputTensor, output_grad_ptr,
                                  m->inputTensor, input_ptr,
                                  &alpha, m->inputTensor, input_grad_ptr));
}

/*
  regions[0](I): input
  regions[1](I/O): input_grad
  regions[2](I): output
  regions[3](I): output_grad
*/
void Pool2D::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  const Pool2D* pool = (Pool2D*) task->args;
  const Pool2DMeta* m = *((Pool2DMeta**) task->local_args);
  TensorAccessorR<float, 4> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 4> acc_input_grad(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, 4> acc_output(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 4> acc_output_grad(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);

  hipEvent_t t_start, t_end;
  if (pool->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  backward_kernel(m, acc_input.ptr, acc_input_grad.ptr, acc_output.ptr, acc_output_grad.ptr);
  if (pool->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Pool2D backward time = %.2fms\n", elapsed);
  }
}

void Pool2D::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<4> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(POOL2D_BWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Pool2D)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  // regions[0](I): input
  launcher.add_region_requirement(
      RegionRequirement(inputs[0].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  // regions[1](I/O): input_grad
  launcher.add_region_requirement(
      RegionRequirement(inputs[0].part_grad, 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(1, FID_DATA);
  // regions[2](I): output
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(2, FID_DATA);
  // regions[3](I): output_grad
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, outputs[0].region_grad));
  launcher.add_field(3, FID_DATA);

  runtime->execute_index_space(ctx, launcher);
}

Pool2DMeta::Pool2DMeta(FFHandler handler)
: OpMeta(handler)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
}

bool Pool2D::measure_compute_time(Simulator* sim,
                                  const ParallelConfig& pc,
                                  float& forward_time,
                                  float& backward_time)
{
  Tensor sub_output, sub_input;
  if(!outputs[0].get_output_sub_tensor(pc, sub_output, OP_CONV2D))
    return false;
  if(!inputs[0].get_input_sub_tensor(pc, sub_input, OP_CONV2D))
    return false;
  int input_w = sub_input.adim[0];
  int input_h = sub_input.adim[1];
  int input_c = sub_input.adim[2];
  int input_n = sub_input.adim[3];
  int output_w = sub_output.adim[0];
  int output_h = sub_output.adim[1];
  int output_c = sub_output.adim[2];
  int output_n = sub_output.adim[3];
  int pad_h = ((output_h - 1) * stride_h + kernel_h - input_h + 1) / 2;
  int pad_w = ((output_w - 1) * stride_w + kernel_w - input_w + 1) / 2;
  Pool2DMeta* m = sim->pool2d_meta;
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        input_n,
                                        input_c,
                                        input_h,
                                        input_w));
  hipdnnPoolingMode_t mode;
  if (pool_type == POOL_MAX)
    mode = HIPDNN_POOLING_MAX;
  else {
    assert(pool_type == POOL_AVG);
    mode = HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;
  }
  checkCUDNN(hipdnnSetPooling2dDescriptor(m->poolDesc,
                                         mode,
                                         HIPDNN_PROPAGATE_NAN,
                                         kernel_h,
                                         kernel_w,
                                         pad_h,//pool->padding_h,
                                         pad_w,//pool->padding_w,
                                         stride_h,
                                         stride_w));
  int n, c, h, w;
  checkCUDNN(hipdnnGetPooling2dForwardOutputDim(m->poolDesc,
                                               m->inputTensor,
                                               &n, &c, &h, &w));
  assert(n == output_n);
  assert(c == output_c);
  assert(h == output_h);
  assert(w == output_w);

  checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        n, c, h, w));
  // allocate tensors in simulator
  sim->free_all();
  float* input_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert(input_ptr != NULL);
  float* input_grad_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert(input_grad_ptr != NULL);
  float *output_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert(output_ptr != NULL);
  float *output_grad_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert(output_grad_ptr != NULL);

  auto forward = [&] {
    forward_kernel(m, input_ptr, output_ptr);
  };
  auto backward = [&] {
    backward_kernel(m, input_ptr, input_grad_ptr, output_ptr, output_grad_ptr);
  };

  inner_measure_compute_time(sim, forward, backward, forward_time, backward_time);

  printf("[Measure Pool2D] name(%s) input(%d %d %d %d) output(%d %d %d %d) stride(%d %d) padding(%d %d) forward_time(%.4lf) backward_time(%.4lf)\n",
      name,
      input_n, input_c, input_h, input_w,
      output_n, output_c, output_h, output_w,
      stride_h, stride_w,
      padding_h, padding_w,
      forward_time, backward_time);

  return true;
}
