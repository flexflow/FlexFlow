#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ops/element_unary.h"
#include "hash_utils.h"
#include "cuda_helper.h"

using namespace Legion;

Tensor FFModel::unary(OperatorType op,
                      const Tensor x,
                      bool inplace,
                      const char *name,
		      float scalar)
{
  ElementUnary *ele = new ElementUnary(*this, op, x, inplace, name, scalar);
  layers.push_back(ele);
  return ele->outputs[0];
}

size_t ElementUnary::get_params_hash() const {
  size_t hash = this->inputs[0]->get_owner_independent_hash();
  hash_combine(hash, this->op_type);
  hash_combine(hash, this->inplace);
  if (this->op_type == OP_SCALAR_MULTIPLY) {
    hash_combine(hash, this->scalar);
  }

  return hash;
}

Node FFModel::get_or_create_element_unary_node(const Tensor input,
                                               OperatorType op,
                                               bool inplace,
                                               float scalar)
{
  if (input->dims[input->num_dims-1].degree != 1) {
    return Node::INVALID_NODE;
  }

  size_t hash = input->get_owner_independent_hash();
  hash_combine(hash, op);
  hash_combine(hash, inplace);
  if (op == OP_SCALAR_MULTIPLY) {
    hash_combine(hash, scalar);
  }

  ElementUnary *unary;
  const auto &it = this->cached_element_unary_ops.find(hash);
  if (it != cached_element_unary_ops.end()) { 
    unary = it->second;
  } else {
    unary = new ElementUnary(*this, op, input, inplace, NULL, scalar);
    cached_element_unary_ops[hash] = unary;
  }

  return this->new_node(unary);
}

Tensor FFModel::exp(const Tensor x,
                    const char *name)
{
  return this->unary(OP_EXP, x, false/*inplace*/, name);
}

Tensor FFModel::scalar_multiply(const Tensor x, const float scalar, bool inplace, const char *name)
{
  return this->unary(OP_SCALAR_MULTIPLY, x, inplace, name, scalar);
}

Tensor FFModel::relu(const Tensor x, bool inplace, const char *name)
{
  return this->unary(OP_RELU, x, inplace, name);
}

Tensor FFModel::sigmoid(const Tensor x, const char *name)
{
  return this->unary(OP_SIGMOID, x, false/*inplace*/, name);
}

Tensor FFModel::tanh(const Tensor x, const char *name)
{
  return this->unary(OP_TANH, x, false/*inplace*/, name);
}

Tensor FFModel::identity(const Tensor x, const char *name)
{
  return this->unary(OP_IDENTITY, x, false/*inplace*/, name);
}

Tensor FFModel::gelu(const Tensor x, const char *name)
{
  return this->unary(OP_GELU, x, false/*inplace*/, name);
}

Tensor FFModel::elu(const Tensor x, bool inplace, const char *name)
{
  // Currently assume inplace is false
  assert(!inplace);
  return this->unary(OP_ELU, x, inplace, name);
}

ElementUnary::ElementUnary(FFModel& model,
                           OperatorType _op_type,
                           const Tensor x,
                           bool _inplace,
                           const char* name,
                           float _scalar)
: Op(model, _op_type, name, 1/*inputs*/, 0/*weights*/, 1/*outputs*/, x), inplace(_inplace), scalar(_scalar)
{
  numOutputs = 1;
  int numdim = x->num_dims;
  ParallelDim dims[MAX_TENSOR_DIM];
  for (int i = 0; i < numdim; i++) {
    dims[i] = x->dims[i];
  }
  outputs[0] = model.create_tensor_legion_ordering(numdim, dims, x->data_type, this);
}

bool ElementUnary::can_inplace_output(void)
{
  return true;
}

bool ElementUnary::has_inplace_output(void)
{
  return inplace;
}

void ElementUnary::do_inplace_output(void)
{
  inplace = true;
}

bool ElementUnary::use_cudnn(OperatorType type)
{
  if (type == OP_RELU)
    return true;
  if (type == OP_SIGMOID)
    return true;
  if (type == OP_TANH)
    return true;
  if (type == OP_ELU)
    return true;
  return false;
}

OpMeta* ElementUnary::init_task(const Task *task,
                                const std::vector<PhysicalRegion> &regions,
                                Context ctx, Runtime *runtime)
{
  ElementUnary* eu = (ElementUnary*) task->args;
  FFHandler handle = *((FFHandler*) task->local_args);
  ElementUnaryMeta* m = new ElementUnaryMeta(handle);
  m->op_type = eu->op_type;
  m->profiling = eu->profiling;
  m->inplace = eu->inplace;
  m->scalar = eu->scalar;
  if (m->inplace) {
    assert(regions.size() == 1);
    assert(task->regions.size() == 1);
  } else {
    assert(regions.size() == 2);
    assert(task->regions.size() == 2);
  }

  if (use_cudnn(m->op_type))
  {
    hipdnnActivationMode_t mode;
    switch (m->op_type) {
      case OP_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      case OP_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case OP_TANH:
        mode = HIPDNN_ACTIVATION_TANH;
        break;
      case OP_ELU:
        mode = HIPDNN_ACTIVATION_ELU;
        break;
      default:
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    Domain input_domain = runtime->get_index_space_domain(
        ctx, task->regions[0].region.get_index_space());
    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->inputTensor, input_domain));
    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->outputTensor, input_domain));
  }
  return m;
}

void ElementUnary::init(const FFModel& ff)
{
  assert(check_output_input_weight_same_parallel_is());
  parallel_is = outputs[0]->parallel_is;
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_init(ff, argmap);
  IndexLauncher init_launcher(ELEMENTUNARY_INIT_TASK_ID, parallel_is,
                              TaskArgument(this, sizeof(ElementUnary)), argmap,
                              Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                              outputs[0]->machine_view.hash());
  init_launcher.add_region_requirement(
      RegionRequirement(inputs[0]->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0]->region));
  init_launcher.add_field(0, FID_DATA);
  assert (!inplace);
  if (!inplace) {
    init_launcher.add_region_requirement(
        RegionRequirement(outputs[0]->part, 0/*projection id*/,
                          WRITE_ONLY, EXCLUSIVE, outputs[0]->region));
    init_launcher.add_field(1, FID_DATA);
  }
  FutureMap fm = runtime->execute_index_space(ctx, init_launcher);
  fm.wait_all_results();
  set_opmeta_from_futuremap(ff, fm);
}

__global__
void elewise_unary_forward_kernel(coord_t volume,
                                  const float alpha,
                                  const float beta,
                                  const float scalar,
				  OperatorType type,
                                  const float* in,
                                  float* out)
{
  CUDA_KERNEL_LOOP(i, volume)
  {
    switch (type) {
      case OP_EXP:
      {
        out[i] = alpha * exp(in[i]) + beta * out[i];
        break;
      }
      case OP_IDENTITY:
      {
	out[i] = in[i];
	break;
      }
      case OP_SCALAR_MULTIPLY:
      {
	out[i] = in[i] * scalar;
	break;
      }
      case OP_GELU:
      {
	out[i] = in[i] * 0.5 * erfc(-in[i]*M_SQRT1_2);
	break;
      }
      default:
        assert(false);
    }
  }
}

/*static*/
void ElementUnary::forward_kernel(const ElementUnaryMeta* m,
                                  const float* input_ptr,
                                  float* output_ptr,
                                  size_t num_elements)
{
  float alpha = 1.0f, beta = 0.0f;
  if (use_cudnn(m->op_type)) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
        &alpha, m->inputTensor, input_ptr,
        &beta, m->outputTensor, output_ptr));
  } else {
    elewise_unary_forward_kernel<<<GET_BLOCKS(num_elements), CUDA_NUM_THREADS>>>(
        num_elements, alpha, beta,m->scalar, m->op_type, input_ptr, output_ptr);
  }
}

/*
  regions[0](I): input
  regions[1](O): output
*/
__host__
void ElementUnary::forward_task(const Task* task,
                                const std::vector<PhysicalRegion> &regions,
                                Context ctx, Runtime* runtime)
{
  //const ElementUnary* ele = (const ElementUnary*) task->args;
  const ElementUnaryMeta* m = *((ElementUnaryMeta**) task->local_args);
  Domain input_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  const float* input_ptr = NULL;
  float* output_ptr = NULL;
  if (m->inplace) {
    assert(regions.size() == 1);
    assert(task->regions.size() == 1);
    output_ptr = helperGetTensorPointerRW<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    input_ptr = output_ptr;
  } else {
    assert(regions.size() == 2);
    assert(task->regions.size() == 2);
    Domain output_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
    assert(output_domain == input_domain);
    input_ptr = helperGetTensorPointerRO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    output_ptr = helperGetTensorPointerWO<float>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  forward_kernel(m, input_ptr, output_ptr, input_domain.get_volume());
}

void ElementUnary::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_forward(ff, argmap);
  IndexLauncher launcher(ELEMENTUNARY_FWD_TASK_ID, parallel_is,
                         TaskArgument(NULL, 0), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         outputs[0]->machine_view.hash());
  if (inplace) {
    assert(outputs[0]->part == inputs[0]->part);
    assert(outputs[0]->region == inputs[0]->region);
    launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part, 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, outputs[0]->region));
    launcher.add_field(0, FID_DATA);
  } else {
    launcher.add_region_requirement(
      RegionRequirement(inputs[0]->part, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[0]->region));
    launcher.add_field(0, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part, 0/*projection id*/,
         WRITE_ONLY, EXCLUSIVE, outputs[0]->region));
    launcher.add_field(1, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

__global__
void elewise_unary_backward_kernel(coord_t volume,
                                   const float alpha,
                                   const float beta,
				   const float scalar,
                                   OperatorType type,
                                   const float* output_grad,
                                   const float* input,
                                   float* input_grad)
{
  CUDA_KERNEL_LOOP(i, volume)
  {
    switch (type) {
      case OP_EXP:
      {
        //TODO: change to use output instead of recomputing
        input_grad[i] = alpha * output_grad[i] * exp(input[i]) + beta * input_grad[i];
        break;
      }
      case OP_IDENTITY:
      {
	input_grad[i] = output_grad[i];
	break;
      } 
      case OP_SCALAR_MULTIPLY:
      {
	input_grad[i] = output_grad[i]*scalar;
	break;
      }
      case OP_GELU:
      {
	input_grad[i] = output_grad[i]*(0.5 * erfc(-input[i]*M_SQRT1_2)-0.5*M_SQRT1_2*input[i]*exp(-input[i]*input[i]*0.5));
	break;
      }
      default:
        assert(false);
    }
  }
}

/*static*/
void ElementUnary::backward_kernel(const ElementUnaryMeta* m,
                                   const float* input_ptr,
                                   float* input_grad_ptr,
                                   const float* output_ptr,
                                   const float* output_grad_ptr,
                                   size_t num_elements)
{
  float alpha = 1.0f;
  if (use_cudnn(m->op_type)) {
    checkCUDNN(hipdnnActivationBackward(m->handle.dnn, m->actiDesc,
        &alpha, m->outputTensor, output_ptr, m->outputTensor, output_grad_ptr,
        m->inputTensor, input_ptr, &alpha, m->inputTensor, input_grad_ptr));
  } else {
    elewise_unary_backward_kernel<<<GET_BLOCKS(num_elements), CUDA_NUM_THREADS>>>(
        num_elements, alpha, alpha, m->scalar, m->op_type, output_grad_ptr, input_ptr, input_grad_ptr);
  }
}

/*
  regions[0](I): input
  regions[1](I/O): input_grad
  regions[2](I): output
  regions[3](I): output_grad
*/
__host__
void ElementUnary::backward_task(const Task* task,
                                 const std::vector<PhysicalRegion> &regions,
                                 Context ctx, Runtime* runtime)
{
  //const ElementUnary* ele = (const ElementUnary*) task->args;
  const ElementUnaryMeta* m = *((ElementUnaryMeta**) task->local_args);
  const float* input_ptr = NULL, *output_ptr = NULL, *output_grad_ptr = NULL;
  float* input_grad_ptr = NULL;
  Domain input_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  if (m->inplace) {
    assert(regions.size() == 2);
    assert(task->regions.size() == 2);
    Domain input_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
    assert(input_grad_domain == input_domain);
    input_ptr = helperGetTensorPointerRO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    input_grad_ptr = helperGetTensorPointerRW<float>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
    output_ptr = input_ptr;
    output_grad_ptr = input_grad_ptr;
  } else {
    assert(regions.size() == 4);
    assert(task->regions.size() == 4);
    Domain input_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
    Domain output_domain = runtime->get_index_space_domain(
      ctx, task->regions[2].region.get_index_space());
    Domain output_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[3].region.get_index_space());
    assert(output_grad_domain == input_domain);
    assert(output_grad_domain == output_domain);
    assert(output_grad_domain == input_grad_domain);
    input_ptr = helperGetTensorPointerRO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    input_grad_ptr = helperGetTensorPointerRW<float>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
    output_ptr = helperGetTensorPointerRO<float>(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
    output_grad_ptr = helperGetTensorPointerRO<float>(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  backward_kernel(m, input_ptr, input_grad_ptr, output_ptr, output_grad_ptr, input_domain.get_volume());
}

void ElementUnary::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_backward(ff, argmap);
  IndexLauncher launcher(ELEMENTUNARY_BWD_TASK_ID, parallel_is,
      TaskArgument(NULL, 0), argmap,
      Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
      outputs[0]->machine_view.hash());
  if (inplace) {
    assert(inputs[0]->part == outputs[0]->part);
    assert(inputs[0]->part_grad == outputs[0]->part_grad);
    // regions[2](I): output_grad
    launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, outputs[0]->region));
    launcher.add_field(0, FID_DATA);
    // regions[3](I): output_grad
    launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part_grad, 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, outputs[0]->region_grad));
    launcher.add_field(1, FID_DATA);
  } else {
    // regions[0](I): input
    launcher.add_region_requirement(
      RegionRequirement(inputs[0]->part, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[0]->region));
    launcher.add_field(0, FID_DATA);
    // regions[1](I/O): input_grad
    launcher.add_region_requirement(
      RegionRequirement(inputs[0]->part_grad, 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, inputs[0]->region_grad));
    launcher.add_field(1, FID_DATA);
    // regions[2](I): output_grad
    launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, outputs[0]->region));
    launcher.add_field(2, FID_DATA);
    // regions[3](I): output_grad
    launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part_grad, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, outputs[0]->region_grad));
    launcher.add_field(3, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

ElementUnaryMeta::ElementUnaryMeta(FFHandler handler)
: OpMeta(handler)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
}

bool ElementUnary::measure_operator_cost(Simulator* sim,
                                         const ParallelConfig& pc,
                                         CostMetrics& cost_metrics) const
{
  TensorBase sub_output, sub_input;
  if (!outputs[0]->get_output_sub_tensor(pc, sub_output, op_type))
    return false;
  if (!inputs[0]->get_input_sub_tensor(pc, sub_input, op_type))
    return false;
  ElementUnaryMeta* m = sim->ele_unary_meta;
  m->op_type = op_type;
  if (use_cudnn(m->op_type))
  {
    hipdnnActivationMode_t mode;
    switch (op_type) {
      case OP_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      case OP_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case OP_TANH:
        mode = HIPDNN_ACTIVATION_TANH;
        break;
      case OP_ELU:
        mode = HIPDNN_ACTIVATION_ELU;
        break;
      default:
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    Domain input_domain, output_domain;
    input_domain.dim = sub_input.num_dims;
    for (int i = 0; i < sub_input.num_dims; i++) {
      input_domain.rect_data[i] = 0;
      input_domain.rect_data[i+input_domain.dim] = sub_input.dims[i].size-1;
    }
    output_domain.dim = sub_output.num_dims;
    for (int i = 0; i < sub_output.num_dims; i++) {
      output_domain.rect_data[i] = 0;
      output_domain.rect_data[i+input_domain.dim] = sub_output.dims[i].size-1;
    }
    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->inputTensor, input_domain));
    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->outputTensor, output_domain));
  }
  sim->free_all();
  float* input_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert(input_ptr != NULL);
  float* output_ptr = NULL;
  if (inplace) {
    output_ptr = input_ptr;
  } else {
    output_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  }
  assert(output_ptr != NULL);

  std::function<void()> forward, backward;
  forward = [&] {
    forward_kernel(m, input_ptr, output_ptr, sub_output.get_volume());
  };
  if (sim->computationMode == COMP_MODE_TRAINING) {
    float* input_grad_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
    assert(input_grad_ptr != NULL);
    float* output_grad_ptr = NULL;
    if (inplace) {
      output_grad_ptr = input_grad_ptr;
    } else {
      output_grad_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
    }
    assert(output_grad_ptr != NULL);
    backward = [&] {
      backward_kernel(m, input_ptr, input_grad_ptr, output_ptr, output_grad_ptr,
          sub_output.get_volume());
    };
  }

  inner_measure_operator_cost(sim, forward, backward, cost_metrics);

  if (sim->computationMode == COMP_MODE_TRAINING) {
    printf("[Measure Elewise Unary] name(%s) num_elements(%zu) forward_time(%.4lf) backward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time,
        cost_metrics.backward_time);
  } else {
    printf("[Measure Elewise Unary] name(%s) num_elements(%zu) forward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time);
  }
  return true;
}
