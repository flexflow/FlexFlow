#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::unary(OperatorType op,
                      const Tensor& x,
                      const char *name)
{
  ElementUnary *ele = new ElementUnary(*this, op, x, name);
  layers.push_back(ele);
  return ele->outputs[0];
}

ElementUnary *FFModel::unary(OperatorType op,
                             const char *name)
{
  ElementUnary *ele = new ElementUnary(*this, op, name);
  layers.push_back(ele);
  return ele;
}

Tensor FFModel::exp(const Tensor& x,
                    const char *name)
{
  return this->unary(OP_EXP, x, name);
}

ElementUnary* FFModel::exp(const char *name)
{
  return this->unary(OP_EXP, name);
}

Tensor FFModel::relu(const Tensor& x, const char *name)
{
  return this->unary(OP_RELU, x, name);
}

ElementUnary* FFModel::relu(const char *name)
{
  return this->unary(OP_RELU, name);
}

Tensor FFModel::sigmoid(const Tensor& x, const char *name)
{
  return this->unary(OP_SIGMOID, x, name);
}

ElementUnary* FFModel::sigmoid(const char *name)
{
  return this->unary(OP_SIGMOID, name);
}

Tensor FFModel::tanh(const Tensor& x, const char *name)
{
  return this->unary(OP_TANH, x, name);
}

ElementUnary* FFModel::tanh(const char *name)
{
  return this->unary(OP_TANH, name);
}

Tensor FFModel::elu(const Tensor& x, const char *name)
{
  return this->unary(OP_ELU, x, name);
}

ElementUnary* FFModel::elu(const char *name)
{
  return this->unary(OP_ELU, name);
}

ElementUnary::ElementUnary(FFModel& model,
                           OperatorType _op_type,
                           const Tensor& x,
                           const char* name)
: Op(model, _op_type, name, x)
{
  outputs[0].numDim = inputs[0].numDim;
  for (int i = 0; i < outputs[0].numDim; i++)
    outputs[0].adim[i] = inputs[0].adim[i];
}

ElementUnary::ElementUnary(FFModel& model,
                           OperatorType _op_type,
                           const char* name)
: Op(model, _op_type, name, 1)
{}

Tensor ElementUnary::init_inout(FFModel& model,
                                const Tensor& input)
{
  inputs[0] = input;
  create_output_and_partition(model);
  return outputs[0];
}

bool ElementUnary::use_cudnn(OperatorType type)
{
  if (type == OP_RELU)
    return true;
  if (type == OP_SIGMOID)
    return true;
  if (type == OP_TANH)
    return true;
  if (type == OP_ELU)
    return true;
  return false;
}

/*
void ElementUnary::add_to_model(FFModel& model)
{
  model.layers.push_back(this);
}
*/

void ElementUnary::create_weights(FFModel& model)
{
  // Do nothing
}

void ElementUnary::create_output_and_partition(FFModel& model)
{
  int dim = inputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      task_is = model.get_or_create_task_is(DIM, name); \
      create_output_and_partition_with_dim<DIM>(model); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
    {
      // Unsupported dim for ElementWiseUnary operator
      assert(false);
    }
  }
}

template<int NDIM>
void ElementUnary::create_output_and_partition_with_dim(FFModel& model)
{
  // Retrive the task indexspace for the op
  task_is = IndexSpaceT<NDIM>(model.get_or_create_task_is(NDIM, name));
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<NDIM> part_rect = runtime->get_index_space_domain(ctx, task_is);
  int dims[NDIM];
  for (int i = 0; i < NDIM; i++)
    dims[i] = inputs[0].adim[NDIM-1-i];
  outputs[0] = model.create_tensor<NDIM>(dims, DT_FLOAT, this);
  outputs[0].owner_op = this;
  outputs[0].owner_idx = 0;
  Rect<NDIM> input_rect;
  input_rect = runtime->get_index_partition_color_space(
        ctx, inputs[0].part.get_index_partition());
  if (input_rect == part_rect) {
    input_lps[0] = inputs[0].part;
    input_grad_lps[0] = inputs[0].part_grad;
  } else {
    model.create_disjoint_partition<NDIM>(
        inputs[0], IndexSpaceT<NDIM>(task_is), input_lps[0], input_grad_lps[0]);
  }
}

OpMeta* ElementUnary::init_task(const Task *task,
                                const std::vector<PhysicalRegion> &regions,
                                Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  ElementUnary* eu = (ElementUnary*) task->args;
  FFHandler handle = *((FFHandler*) task->local_args);
  ElementUnaryMeta* m = new ElementUnaryMeta(handle);
  m->op_type = eu->op_type;
  if (use_cudnn(m->op_type))
  {
    hipdnnActivationMode_t mode;
    switch (m->op_type) {
      case OP_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      case OP_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case OP_TANH:
        mode = HIPDNN_ACTIVATION_TANH;
        break;
      case OP_ELU:
        mode = HIPDNN_ACTIVATION_ELU;
        break;
      default:
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    Domain input_domain = runtime->get_index_space_domain(
        ctx, task->regions[0].region.get_index_space());
    Domain output_domain = runtime->get_index_space_domain(
        ctx, task->regions[1].region.get_index_space());

    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->inputTensor, input_domain));
    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->outputTensor, output_domain));
  }
  return m;
}

void ElementUnary::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      ParallelConfig pc; \
      std::string pcname = name; \
      ff.config.find_parallel_config(DIM, pcname, pc); \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        FFHandler handle = ff.handlers[pc.device_ids[idx++]]; \
        argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher init_launcher(ELEMENTUNARY_INIT_TASK_ID, task_is,
                              TaskArgument(this, sizeof(ElementUnary)), argmap,
                              Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                              FFConfig::get_hash_id(std::string(name)));
  init_launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  init_launcher.add_field(0, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  init_launcher.add_field(1, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, init_launcher);
  fm.wait_all_results();
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        meta[idx++] = fm.get_result<OpMeta*>(*it); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

__global__
void elewise_unary_forward_kernel(coord_t volume,
                                  const float alpha,
                                  const float beta,
                                  OperatorType type,
                                  const float* in,
                                  float* out)
{
  CUDA_KERNEL_LOOP(i, volume)
  {
    switch (type) {
      case OP_EXP:
      {
        out[i] = alpha * exp(in[i]) + beta * out[i];
        break;
      }
      default:
        assert(false);
    }
  }
}

/*static*/
void ElementUnary::forward_kernel(const ElementUnaryMeta* m,
                                  const float* input_ptr,
                                  float* output_ptr,
                                  size_t num_elements)
{
  float alpha = 1.0f, beta = 0.0f;
  if (use_cudnn(m->op_type)) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
        &alpha, m->inputTensor, input_ptr,
        &beta, m->outputTensor, output_ptr));
  } else {
    elewise_unary_forward_kernel<<<GET_BLOCKS(num_elements), CUDA_NUM_THREADS>>>(
        num_elements, alpha, beta, m->op_type, input_ptr, output_ptr);
  }
}

/*
  regions[0](I): input
  regions[1](O): output
*/
__host__
void ElementUnary::forward_task(const Task* task,
                                const std::vector<PhysicalRegion> &regions,
                                Context ctx, Runtime* runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  //const ElementUnary* ele = (const ElementUnary*) task->args;
  const ElementUnaryMeta* m = *((ElementUnaryMeta**) task->local_args);
  Domain input_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain output_domain = runtime->get_index_space_domain(
    ctx, task->regions[1].region.get_index_space());
  assert(output_domain == input_domain);

  const float* input_ptr = helperGetTensorPointerRO<float>(
    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  float* output_ptr = helperGetTensorPointerWO<float>(
    regions[1], task->regions[1], FID_DATA, ctx, runtime);

#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  forward_kernel(m, input_ptr, output_ptr, output_domain.get_volume());
}

void ElementUnary::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(ELEMENTUNARY_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(ElementUnary)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(input_lps[0], 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part, 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(1, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

__global__
void elewise_unary_backward_kernel(coord_t volume,
                                   const float alpha,
                                   const float beta,
                                   OperatorType type,
                                   const float* output_grad,
                                   const float* input,
                                   float* input_grad)
{
  CUDA_KERNEL_LOOP(i, volume)
  {
    switch (type) {
      case OP_EXP:
      {
        //TODO: change to use output instead of recomputing
        input_grad[i] = alpha * output_grad[i] * exp(input[i]) + beta * input_grad[i];
        break;
      }
      default:
        assert(false);
    }
  }
}

/*static*/
void ElementUnary::backward_kernel(const ElementUnaryMeta* m,
                                   const float* input_ptr,
                                   float* input_grad_ptr,
                                   const float* output_ptr,
                                   const float* output_grad_ptr,
                                   size_t num_elements)
{
  float alpha = 1.0f;
  if (use_cudnn(m->op_type)) {
    checkCUDNN(hipdnnActivationBackward(m->handle.dnn, m->actiDesc,
        &alpha, m->outputTensor, output_ptr, m->outputTensor, output_grad_ptr,
        m->inputTensor, input_ptr, &alpha, m->inputTensor, input_grad_ptr));
  } else {
    elewise_unary_backward_kernel<<<GET_BLOCKS(num_elements), CUDA_NUM_THREADS>>>(
        num_elements, alpha, alpha, m->op_type, output_grad_ptr, input_ptr, input_grad_ptr);
  }
}

/*
  regions[0](I): input
  regions[1](I/O): input_grad
  regions[2](I): output
  regions[3](I): output_grad
*/
__host__
void ElementUnary::backward_task(const Task* task,
                                 const std::vector<PhysicalRegion> &regions,
                                 Context ctx, Runtime* runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  //const ElementUnary* ele = (const ElementUnary*) task->args;
  const ElementUnaryMeta* m = *((ElementUnaryMeta**) task->local_args);
  Domain input_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain input_grad_domain = runtime->get_index_space_domain(
    ctx, task->regions[1].region.get_index_space());
  Domain output_domain = runtime->get_index_space_domain(
    ctx, task->regions[2].region.get_index_space());
  Domain output_grad_domain = runtime->get_index_space_domain(
    ctx, task->regions[3].region.get_index_space());
  assert(output_grad_domain == input_domain);
  assert(output_grad_domain == output_domain);
  assert(output_grad_domain == input_grad_domain);

  const float* input_ptr = helperGetTensorPointerRO<float>(
    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  float* input_grad_ptr = helperGetTensorPointerRW<float>(
    regions[1], task->regions[1], FID_DATA, ctx, runtime);
  const float* output_ptr = helperGetTensorPointerRO<float>(
    regions[2], task->regions[2], FID_DATA, ctx, runtime);
  const float* output_grad_ptr = helperGetTensorPointerRO<float>(
    regions[3], task->regions[3], FID_DATA, ctx, runtime);
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  backward_kernel(m, input_ptr, input_grad_ptr, output_ptr, output_grad_ptr, input_domain.get_volume());
}

void ElementUnary::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }

  IndexLauncher launcher(ELEMENTUNARY_BWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(ElementUnary)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  // regions[0](I): input
  launcher.add_region_requirement(
    RegionRequirement(input_lps[0], 0/*projection id*/,
                      READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  // regions[1](I/O): input_grad
  launcher.add_region_requirement(
    RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                      READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(1, FID_DATA);
  // regions[2](I): output_grad
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part, 0/*projection id*/,
                      READ_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(2, FID_DATA);
  // regions[3](I): output_grad
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
                      READ_ONLY, EXCLUSIVE, outputs[0].region_grad));
  launcher.add_field(3, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

ElementUnaryMeta::ElementUnaryMeta(FFHandler handler)
: OpMeta(handler)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
}

bool ElementUnary::measure_compute_time(Simulator* sim,
                                        const ParallelConfig& pc,
                                        float& forward_time,
                                        float& backward_time)
{
  Tensor sub_output, sub_input;
  if (!outputs[0].get_output_sub_tensor(pc, sub_output, op_type))
    return false;
  if (!inputs[0].get_input_sub_tensor(pc, sub_input, op_type))
    return false;
  ElementUnaryMeta* m = sim->ele_unary_meta;
  m->op_type = op_type;
  if (use_cudnn(m->op_type))
  {
    hipdnnActivationMode_t mode;
    switch (op_type) {
      case OP_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      case OP_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case OP_TANH:
        mode = HIPDNN_ACTIVATION_TANH;
        break;
      case OP_ELU:
        mode = HIPDNN_ACTIVATION_ELU;
        break;
      default:
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    Domain input_domain, output_domain;
    input_domain.dim = sub_input.numDim;
    for (int i = 0; i < sub_input.numDim; i++) {
      input_domain.rect_data[i] = 0;
      input_domain.rect_data[i+Domain::MAX_RECT_DIM] = sub_input.adim[i]-1;
    }
    output_domain.dim = sub_output.numDim;
    for (int i = 0; i < sub_output.numDim; i++) {
      output_domain.rect_data[i] = 0;
      output_domain.rect_data[i+Domain::MAX_RECT_DIM] = sub_output.adim[i]-1;
    }
    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->inputTensor, input_domain));
    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->outputTensor, output_domain));
  }
  sim->free_all();
  float* input_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert(input_ptr != NULL);
  float* input_grad_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert(input_grad_ptr != NULL);
  float* output_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert(output_ptr != NULL);
  float* output_grad_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert(output_grad_ptr != NULL);

  auto forward = [&] {
    forward_kernel(m, input_ptr, output_ptr, sub_output.get_volume());
  };
  auto backward = [&] {
    backward_kernel(m, input_ptr, input_grad_ptr, output_ptr, output_grad_ptr,
        sub_output.get_volume());
  };

  inner_measure_compute_time(sim, forward, backward, forward_time, backward_time);

  printf("[Measure Elewise Unary] name(%s) num_elements(%zu) forward_time(%.4lf) backward_time(%.4lf)\n",
         name, sub_output.get_volume(), forward_time, backward_time);
  return true;
}
