#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::unary(OperatorType op,
                      const Tensor& x,
                      bool inplace,
                      const char *name,
		      float scalar)
{
  ElementUnary *ele = new ElementUnary(*this, op, x, inplace, name, scalar);
  layers.push_back(ele);
  return ele->outputs[0];
}

Tensor FFModel::exp(const Tensor& x,
                    const char *name)
{
  return this->unary(OP_EXP, x, false/*inplace*/, name);
}

Tensor FFModel::scalar_multiply(const Tensor& x,const float scalar ,bool inplace, const char *name)
{
  return this->unary(OP_SCALAR_MULTIPLY, x, inplace, name, scalar);
}

Tensor FFModel::scalar_add(const Tensor& x,const float scalar ,bool inplace, const char *name)
{
  return this->unary(OP_SCALAR_ADD, x, inplace, name, scalar);
}

Tensor FFModel::scalar_sub(const Tensor& x,const float scalar ,bool inplace, const char *name)
{
  return this->unary(OP_SCALAR_SUB, x, inplace, name, scalar);
}

Tensor FFModel::scalar_truediv(const Tensor& x,const float scalar ,bool inplace, const char *name)
{
  return this->unary(OP_SCALAR_TRUE_DIV, x, inplace, name, scalar);
}

Tensor FFModel::relu(const Tensor& x, bool inplace, const char *name)
{
  return this->unary(OP_RELU, x, inplace, name);
}

Tensor FFModel::sigmoid(const Tensor& x, const char *name)
{
  return this->unary(OP_SIGMOID, x, false/*inplace*/, name);
}

Tensor FFModel::tanh(const Tensor& x, const char *name)
{
  return this->unary(OP_TANH, x, false/*inplace*/, name);
}

Tensor FFModel::identity(const Tensor& x, const char *name)
{
  return this->unary(OP_IDENTITY, x, false/*inplace*/, name);
}

Tensor FFModel::gelu(const Tensor& x, const char *name)
{
  return this->unary(OP_GELU, x, false/*inplace*/, name);
}

Tensor FFModel::elu(const Tensor& x, bool inplace, const char *name)
{
  // Currently assume inplace is false
  assert(!inplace);
  return this->unary(OP_ELU, x, inplace, name);
}

Tensor FFModel::rsqrt(const Tensor& x, bool inplace, const char *name)
{
  return this->unary(OP_RSQRT, x, inplace, name);
}
Tensor FFModel::pow(const Tensor& x, const float exponent, bool inplace, const char *name)
{
  return this->unary(OP_POW, x, inplace, name, exponent);
}

ElementUnary::ElementUnary(FFModel& model,
                           OperatorType _op_type,
                           const Tensor& x,
                           bool _inplace,
                           const char* name,
			   float _scalar)
: Op(model, _op_type, name, x), inplace(_inplace), scalar(_scalar)
{
  outputs[0].numDim = inputs[0].numDim;
  for (int i = 0; i < outputs[0].numDim; i++)
    outputs[0].adim[i] = inputs[0].adim[i];
}

bool ElementUnary::can_inplace_output(void)
{
  return true;
}

bool ElementUnary::has_inplace_output(void)
{
  return inplace;
}

void ElementUnary::do_inplace_output(void)
{
  inplace = true;
}

bool ElementUnary::use_cudnn(OperatorType type)
{
  if (type == OP_RELU)
    return true;
  if (type == OP_SIGMOID)
    return true;
  if (type == OP_TANH)
    return true;
  if (type == OP_ELU)
    return true;
  return false;
}

void ElementUnary::create_weights(FFModel& model)
{
  // Do nothing
}

void ElementUnary::create_output_and_partition(FFModel& model)
{
  int dim = inputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      task_is = model.get_or_create_task_is(DIM, name); \
      create_output_and_partition_with_dim<DIM>(model); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
    {
      // Unsupported dim for ElementWiseUnary operator
      assert(false);
    }
  }
}

template<int NDIM>
void ElementUnary::create_output_and_partition_with_dim(FFModel& model)
{
  // Retrive the task indexspace for the op
  task_is = IndexSpaceT<NDIM>(model.get_or_create_task_is(NDIM, name));
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<NDIM> part_rect = runtime->get_index_space_domain(ctx, task_is);
  Rect<NDIM> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part.get_index_partition());
  if (inplace) {
    // output reuse input tensor
    outputs[0] = inputs[0];
    outputs[0].owner_op = this;
    outputs[0].owner_idx = 0;
    assert(input_rect == part_rect && "Inplace require the same partitioning");
    input_lps[0] = inputs[0].part;
    input_grad_lps[0] = inputs[0].part_grad;
    return; 
  }
  int dims[NDIM];
  for (int i = 0; i < NDIM; i++)
    dims[i] = inputs[0].adim[NDIM-1-i];
  outputs[0] = model.create_tensor<NDIM>(dims, DT_FLOAT, this);
  outputs[0].owner_op = this;
  outputs[0].owner_idx = 0;
  if (input_rect == part_rect) {
    input_lps[0] = inputs[0].part;
    input_grad_lps[0] = inputs[0].part_grad;
  } else {
    model.create_disjoint_partition<NDIM>(
        inputs[0], IndexSpaceT<NDIM>(task_is), input_lps[0], input_grad_lps[0]);
  }
}

OpMeta* ElementUnary::init_task(const Task *task,
                                const std::vector<PhysicalRegion> &regions,
                                Context ctx, Runtime *runtime)
{
  ElementUnary* eu = (ElementUnary*) task->args;
  FFHandler handle = *((FFHandler*) task->local_args);
  ElementUnaryMeta* m = new ElementUnaryMeta(handle);
  m->op_type = eu->op_type;
  m->profiling = eu->profiling;
  m->inplace = eu->inplace;
  m->scalar = eu->scalar;
  if (m->inplace) {
    assert(regions.size() == 1);
    assert(task->regions.size() == 1);
  } else {
    assert(regions.size() == 2);
    assert(task->regions.size() == 2);
  }

  if (use_cudnn(m->op_type))
  {
    hipdnnActivationMode_t mode;
    switch (m->op_type) {
      case OP_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      case OP_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case OP_TANH:
        mode = HIPDNN_ACTIVATION_TANH;
        break;
      case OP_ELU:
        mode = HIPDNN_ACTIVATION_ELU;
        break;
      default:
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    Domain input_domain = runtime->get_index_space_domain(
        ctx, task->regions[0].region.get_index_space());
    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->inputTensor, input_domain));
      checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->outputTensor, input_domain));
  }
  return m;
}

void ElementUnary::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      ParallelConfig pc; \
      std::string pcname = name; \
      ff.config.find_parallel_config(DIM, pcname, pc); \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        FFHandler handle = ff.handlers[pc.device_ids[idx++]]; \
        argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher init_launcher(ELEMENTUNARY_INIT_TASK_ID, task_is,
                              TaskArgument(this, sizeof(ElementUnary)), argmap,
                              Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                              FFConfig::get_hash_id(std::string(name)));
  init_launcher.add_region_requirement(
    RegionRequirement(input_lps[0], 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[0].region));
  init_launcher.add_field(0, FID_DATA);
  if (!inplace) {
    init_launcher.add_region_requirement(
        RegionRequirement(outputs[0].part, 0/*projection id*/,
            WRITE_ONLY, EXCLUSIVE, outputs[0].region));
    init_launcher.add_field(1, FID_DATA);
  }
  FutureMap fm = runtime->execute_index_space(ctx, init_launcher);
  fm.wait_all_results();
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        meta[idx++] = fm.get_result<OpMeta*>(*it); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

__global__
void elewise_unary_forward_kernel(coord_t volume,
                                  const float alpha,
                                  const float beta,
                                  const float scalar,
				  OperatorType type,
                                  const float* in,
                                  float* out)
{
  CUDA_KERNEL_LOOP(i, volume)
  {
    switch (type) {
      case OP_EXP:
      {
        out[i] = alpha * exp(in[i]) + beta * out[i];
        break;
      }
      case OP_IDENTITY:
      {
	out[i] = in[i];
	break;
      }
      case OP_SCALAR_MULTIPLY:
      {
	out[i] = in[i] * scalar;
	break;
      }
      case OP_SCALAR_ADD:
      {
	out[i] = in[i] + scalar;
	break;
      }
      case OP_SCALAR_SUB:
      {
	out[i] = in[i] - scalar;
	break;
      }
      case OP_SCALAR_TRUE_DIV:
      {
	out[i] = in[i] / scalar;
	break;
      }
      case OP_GELU:
      {
	out[i] = in[i] * 0.5 * erfc(-in[i]*M_SQRT1_2);
	break;
      }
      case OP_RSQRT:
      {
        out[i] = 1.0f / sqrt(in[i]);
	break;
      }
      case OP_POW:
      {
        out[i] = powf(in[i], scalar);
        break;
      }
      default:
        assert(false);
    }
  }
}

/*static*/
void ElementUnary::forward_kernel(const ElementUnaryMeta* m,
                                  const float* input_ptr,
                                  float* output_ptr,
                                  size_t num_elements, 
                                  hipStream_t stream)
{
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha = 1.0f, beta = 0.0f;
  if (use_cudnn(m->op_type)) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
        &alpha, m->inputTensor, input_ptr,
        &beta, m->outputTensor, output_ptr));
  } else {
    elewise_unary_forward_kernel<<<GET_BLOCKS(num_elements), CUDA_NUM_THREADS, 0, stream>>>(
        num_elements, alpha, beta,m->scalar, m->op_type, input_ptr, output_ptr);
  }
}

/*
  regions[0](I): input
  regions[1](O): output
*/
__host__
void ElementUnary::forward_task(const Task* task,
                                const std::vector<PhysicalRegion> &regions,
                                Context ctx, Runtime* runtime)
{
  //const ElementUnary* ele = (const ElementUnary*) task->args;
  const ElementUnaryMeta* m = *((ElementUnaryMeta**) task->local_args);
  Domain input_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  const float* input_ptr = NULL;
  float* output_ptr = NULL;
  if (m->inplace) {
    assert(regions.size() == 1);
    assert(task->regions.size() == 1);
    output_ptr = helperGetTensorPointerRW<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    input_ptr = output_ptr;
  } else {
    assert(regions.size() == 2);
    assert(task->regions.size() == 2);
    Domain output_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
    assert(output_domain == input_domain);
    input_ptr = helperGetTensorPointerRO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    output_ptr = helperGetTensorPointerWO<float>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  forward_kernel(m, input_ptr, output_ptr, input_domain.get_volume(), stream);
}

void ElementUnary::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(ELEMENTUNARY_FWD_TASK_ID, task_is,
      TaskArgument(NULL, 0), argmap,
      Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
      FFConfig::get_hash_id(std::string(name)));
  if (inplace) {
    assert(outputs[0].part == input_lps[0]);
    assert(outputs[0].region == inputs[0].region);
    launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, outputs[0].region));
    launcher.add_field(0, FID_DATA);
  } else {
    launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[0].region));
    launcher.add_field(0, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
         WRITE_ONLY, EXCLUSIVE, outputs[0].region));
    launcher.add_field(1, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

__global__
void elewise_unary_backward_kernel(coord_t volume,
                                   const float alpha,
                                   const float beta,
				   const float scalar,
                                   OperatorType type,
                                   const float* output,
                                   const float* output_grad,
                                   const float* input,
                                   float* input_grad)
{
  CUDA_KERNEL_LOOP(i, volume)
  {
    switch (type) {
      case OP_EXP:
      {
        //TODO: change to use output instead of recomputing
        input_grad[i] = alpha * output_grad[i] * exp(input[i]) + beta * input_grad[i];
        break;
      }
      case OP_IDENTITY:
      {
	input_grad[i] = output_grad[i];
	break;
      } 
      case OP_SCALAR_MULTIPLY:
      {
	input_grad[i] = output_grad[i]*scalar;
	break;
      }
      case OP_SCALAR_ADD:
      {
	input_grad[i] = output_grad[i];
	break;
      }
      case OP_SCALAR_SUB:
      {
	input_grad[i] = output_grad[i];
	break;
      }
      case OP_SCALAR_TRUE_DIV:
      {
	input_grad[i] = output_grad[i]/scalar;
	break;
      }
      case OP_GELU:
      {
	input_grad[i] = output_grad[i]*(0.5 * erfc(-input[i]*M_SQRT1_2)-0.5*M_SQRT1_2*input[i]*exp(-input[i]*input[i]*0.5));
	break;
      }
      case OP_RSQRT:
      {
        input_grad[i] = -0.5f * output_grad[i] * output[i] * output[i] * output[i];
	break;
      }
      case OP_POW:
      {
        input_grad[i] = output_grad[i] * scalar * powf(input[i], scalar - 1);
      }
      default:
        assert(false);
    }
  }
}

/*static*/
void ElementUnary::backward_kernel(const ElementUnaryMeta* m,
                                   const float* input_ptr,
                                   float* input_grad_ptr,
                                   const float* output_ptr,
                                   const float* output_grad_ptr,
                                   size_t num_elements,
                                   hipStream_t stream)
{
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha = 1.0f;
  if (use_cudnn(m->op_type)) {
    checkCUDNN(hipdnnActivationBackward(m->handle.dnn, m->actiDesc,
        &alpha, m->outputTensor, output_ptr, m->outputTensor, output_grad_ptr,
        m->inputTensor, input_ptr, &alpha, m->inputTensor, input_grad_ptr));
  } else {
    elewise_unary_backward_kernel<<<GET_BLOCKS(num_elements), CUDA_NUM_THREADS, 0, stream>>>(
        num_elements, alpha, alpha, m->scalar, m->op_type, output_ptr, output_grad_ptr, input_ptr, input_grad_ptr);
  }
}

/*
  regions[0](I): input
  regions[1](I/O): input_grad
  regions[2](I): output
  regions[3](I): output_grad
*/
__host__
void ElementUnary::backward_task(const Task* task,
                                 const std::vector<PhysicalRegion> &regions,
                                 Context ctx, Runtime* runtime)
{
  //const ElementUnary* ele = (const ElementUnary*) task->args;
  const ElementUnaryMeta* m = *((ElementUnaryMeta**) task->local_args);
  const float* input_ptr = NULL, *output_ptr = NULL, *output_grad_ptr = NULL;
  float* input_grad_ptr = NULL;
  Domain input_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  if (m->inplace) {
    assert(regions.size() == 2);
    assert(task->regions.size() == 2);
    Domain input_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
    assert(input_grad_domain == input_domain);
    input_ptr = helperGetTensorPointerRO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    input_grad_ptr = helperGetTensorPointerRW<float>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
    output_ptr = input_ptr;
    output_grad_ptr = input_grad_ptr;
  } else {
    assert(regions.size() == 4);
    assert(task->regions.size() == 4);
    Domain input_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
    Domain output_domain = runtime->get_index_space_domain(
      ctx, task->regions[2].region.get_index_space());
    Domain output_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[3].region.get_index_space());
    assert(output_grad_domain == input_domain);
    assert(output_grad_domain == output_domain);
    assert(output_grad_domain == input_grad_domain);
    input_ptr = helperGetTensorPointerRO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    input_grad_ptr = helperGetTensorPointerRW<float>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
    output_ptr = helperGetTensorPointerRO<float>(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
    output_grad_ptr = helperGetTensorPointerRO<float>(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  backward_kernel(m, input_ptr, input_grad_ptr, output_ptr, output_grad_ptr, input_domain.get_volume(), stream);
}

void ElementUnary::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }

  IndexLauncher launcher(ELEMENTUNARY_BWD_TASK_ID, task_is,
      TaskArgument(NULL, 0), argmap,
      Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
      FFConfig::get_hash_id(std::string(name)));
  if (inplace) {
    assert(input_lps[0] == outputs[0].part);
    assert(input_grad_lps[0] == outputs[0].part_grad);
    // regions[2](I): output_grad
    launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, outputs[0].region));
    launcher.add_field(0, FID_DATA);
    // regions[3](I): output_grad
    launcher.add_region_requirement(
      RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, outputs[0].region_grad));
    launcher.add_field(1, FID_DATA);
  } else {
    // regions[0](I): input
    launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[0].region));
    launcher.add_field(0, FID_DATA);
    // regions[1](I/O): input_grad
    launcher.add_region_requirement(
      RegionRequirement(input_grad_lps[0], 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
    launcher.add_field(1, FID_DATA);
    // regions[2](I): output_grad
    launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, outputs[0].region));
    launcher.add_field(2, FID_DATA);
    // regions[3](I): output_grad
    launcher.add_region_requirement(
      RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, outputs[0].region_grad));
    launcher.add_field(3, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

ElementUnaryMeta::ElementUnaryMeta(FFHandler handler)
: OpMeta(handler)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
}

bool ElementUnary::measure_operator_cost(Simulator* sim,
                                         const ParallelConfig& pc,
                                         CostMetrics& cost_metrics)
{
  Tensor sub_output, sub_input;
  if (!outputs[0].get_output_sub_tensor(pc, sub_output, op_type))
    return false;
  if (!inputs[0].get_input_sub_tensor(pc, sub_input, op_type))
    return false;
  ElementUnaryMeta* m = sim->ele_unary_meta;
  m->op_type = op_type;
  if (use_cudnn(m->op_type))
  {
    hipdnnActivationMode_t mode;
    switch (op_type) {
      case OP_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      case OP_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case OP_TANH:
        mode = HIPDNN_ACTIVATION_TANH;
        break;
      case OP_ELU:
        mode = HIPDNN_ACTIVATION_ELU;
        break;
      default:
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    Domain input_domain, output_domain;
    input_domain.dim = sub_input.numDim;
    for (int i = 0; i < sub_input.numDim; i++) {
      input_domain.rect_data[i] = 0;
      input_domain.rect_data[i+input_domain.dim] = sub_input.adim[i]-1;
    }
    output_domain.dim = sub_output.numDim;
    for (int i = 0; i < sub_output.numDim; i++) {
      output_domain.rect_data[i] = 0;
      output_domain.rect_data[i+input_domain.dim] = sub_output.adim[i]-1;
    }
    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->inputTensor, input_domain));
    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->outputTensor, output_domain));
  }
  sim->free_all();
  float* input_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert(input_ptr != NULL);
  float* output_ptr = NULL;
  if (inplace) {
    output_ptr = input_ptr;
  } else {
    output_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  }
  assert(output_ptr != NULL);

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  std::function<void()> forward, backward;
  forward = [&] {
    forward_kernel(m, input_ptr, output_ptr, sub_output.get_volume(), stream);
  };
  if (sim->computationMode == COMP_MODE_TRAINING) {
    float* input_grad_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
    assert(input_grad_ptr != NULL);
    float* output_grad_ptr = NULL;
    if (inplace) {
      output_grad_ptr = input_grad_ptr;
    } else {
      output_grad_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
    }
    assert(output_grad_ptr != NULL);
    backward = [&] {
      backward_kernel(m, input_ptr, input_grad_ptr, output_ptr, output_grad_ptr,
          sub_output.get_volume(), stream);
    };
  }

  inner_measure_operator_cost(sim, forward, backward, cost_metrics);

  if (sim->computationMode == COMP_MODE_TRAINING) {
    printf("[Measure Elewise Unary] name(%s) num_elements(%zu) forward_time(%.4lf) backward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time,
        cost_metrics.backward_time);
  } else {
    printf("[Measure Elewise Unary] name(%s) num_elements(%zu) forward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time);
  }
  return true;
}
