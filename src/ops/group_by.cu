#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/groupby.h"
#include "flexflow/utils/cuda_helper.h"
#include <math.h>
#include <stdio.h>

#define MAX_K 4
#define MAX_BATCH_SIZE 32
#define MAX_N 12

namespace FlexFlow {
// declare Legion names
using Legion::Context;
using Legion::Runtime;
using Legion::Domain;
using Legion::Task;
using Legion::Rect;
using Legion::PhysicalRegion;
using Legion::coord_t;

OpMeta* Group_by::init_task(const Task* task,
                        const std::vector<PhysicalRegion> &regions,
                        Context ctx, Runtime* runtime)
{
  Group_by* gb = (Group_by*) task->args;
  FFHandler handle = *((FFHandler*)task->local_args);
  GroupByMeta* m = new GroupByMeta(handle, gb->n);
  m->profiling = gb->profiling;
  return m;
}

__global__
void gb_forward_kernel(const float* input,
        const int* exp_assign,
        float** outputs,
        int n, // num experts
        int k, // chosen experts
        float alpha, // factor additional memory assigned
        int batch_size,
        int data_dim)
{
  __shared__ float* chosen_exp_preds[MAX_K*MAX_BATCH_SIZE];

  // Get pred pointers, single thread per block
  if(threadIdx.x == 0) {
    int exp_tensor_rows = ceil(alpha*k/n*batch_size);
    int expert_idx[MAX_N] = {0};
    for(int i = 0; i < k*batch_size; i++) {
      // Get pointer to chosen expert predictions
      int expert = exp_assign[i];
      if(expert_idx[expert] >= exp_tensor_rows) {
        // dropped sample
        chosen_exp_preds[i] = 0;
        continue;
      }
      chosen_exp_preds[i] = outputs[expert] + expert_idx[expert]*data_dim;
      expert_idx[expert]++;
    }
  }

  __syncthreads();

  // compute output
  CUDA_KERNEL_LOOP(i, k*batch_size*data_dim)
  {
    if(chosen_exp_preds[i/data_dim] != 0) {
      float a = input[(i/(k*data_dim))*data_dim + i%data_dim];
      chosen_exp_preds[i/data_dim][i%data_dim] = a;
    }
  }
}


__global__
void gb_backward_kernel(float* input_grad,
        const int* exp_assign,
        float** output_grads,
        int n, // num experts
        int k, // chosen experts
        float alpha, // factor additional memory assigned
        int batch_size,
        int data_dim)
{
  __shared__ float* chosen_exp_grads[MAX_K*MAX_BATCH_SIZE];

  // Get pred pointers, single thread
  if(blockIdx.x * blockDim.x + threadIdx.x == 0) {
    int exp_tensor_rows = ceil(alpha*k/n*batch_size);
    int expert_idx[MAX_N] = {0};
    for(int i = 0; i < k*batch_size; i++) {
      // Get pointer to chosen expert predictions
      int expert = exp_assign[i];
      if(expert_idx[expert] >= exp_tensor_rows) {
        // dropped sample
        chosen_exp_grads[i] = 0;
        continue;
      }
      chosen_exp_grads[i] = output_grads[expert] + expert_idx[expert]*data_dim;
      expert_idx[expert]++;
    }
  }

  __syncthreads();

  // compute output
  CUDA_KERNEL_LOOP(i, k*batch_size*data_dim)
  {
    if(chosen_exp_grads[i/data_dim] != 0) {
      input_grad[(i/(k*data_dim))*data_dim + i%data_dim] = chosen_exp_grads[i/data_dim][i%data_dim];
    }
  }
}


void Group_by::forward_task(const Task *task,
                            const std::vector<PhysicalRegion>& regions,
                            Context ctx, Runtime* runtime)
{
  // Get n, alpha
  const Group_by* gb = (Group_by*) task->args;
  int n = gb->n;
  float alpha = gb->alpha;

  assert((int)regions.size() == n+2);
  assert((int)task->regions.size() == n+2);

  const GroupByMeta* m = *((GroupByMeta**)task->local_args);

  // get input and assign regions
  const AccessorRO<float, 2> acc_input(regions[0], FID_DATA);
  const AccessorRO<int, 2> acc_assign(regions[1], FID_DATA);

  Rect<2> rect_input = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  Rect<2> rect_assign = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());

  coord_t input_rows = rect_input.hi[1] - rect_input.lo[1] + 1;
  coord_t input_cols = rect_input.hi[0] - rect_input.lo[0] + 1;
  assert(input_rows == rect_assign.hi[1] - rect_assign.lo[1] + 1);
  int k = rect_assign.hi[0] - rect_assign.lo[0] + 1;
  int batch_size = input_rows;
  int data_dim = input_cols;

  // get output
  float* outputs[n];
  //int exp_output_rows = (int)ceil(alpha*k/n*batch_size);
  for(int i = 0; i < n; i++) {
    Domain out_domain = runtime->get_index_space_domain(
      ctx, task->regions[i+2].region.get_index_space());
    outputs[i] = helperGetTensorPointerWO<float>(
      regions[i+2], task->regions[i+2], FID_DATA, ctx, runtime);

    //coord_t output_rows = out_domain.hi()[1] - out_domain.lo()[1] + 1;
    coord_t output_cols = out_domain.hi()[0] - out_domain.lo()[0] + 1;
    //assert((int)output_rows == exp_output_rows);
    assert(output_cols == input_cols);
  }

  // TODO: why cublas/cudnn stream is needed here?
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  // call forward kernel
  hipMemcpy(m->dev_region_ptrs, outputs, n*sizeof(float*), hipMemcpyHostToDevice);

  gb_forward_kernel<<<GET_BLOCKS(batch_size*k*data_dim), min(CUDA_NUM_THREADS,(int)(batch_size*k*data_dim)), 0, stream>>>(
    acc_input.ptr(rect_input), acc_assign.ptr(rect_assign), m->dev_region_ptrs, n, k,
    alpha, batch_size, data_dim);
}


void Group_by::backward_task(const Task *task,
                            const std::vector<PhysicalRegion>& regions,
                            Context ctx, Runtime* runtime)
{
  // Get n, alpha
  const GroupByMeta* m = *((GroupByMeta**)task->local_args);
  const Group_by* gb = (Group_by*) task->args;
  int n = gb->n;
  float alpha = gb->alpha;

  assert((int)regions.size() == n+2);
  assert((int)task->regions.size() == n+2);

  // get input and assign regions
  const AccessorWO<float, 2> acc_input_grad(regions[0], FID_DATA);
  const AccessorRO<int, 2> acc_assign(regions[1], FID_DATA);

  Rect<2> rect_input_grad = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  Rect<2> rect_assign = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());

  coord_t input_rows = rect_input_grad.hi[1] - rect_input_grad.lo[1] + 1;
  coord_t input_cols = rect_input_grad.hi[0] - rect_input_grad.lo[0] + 1;
  assert(input_rows == rect_assign.hi[1] - rect_assign.lo[1] + 1);
  int k = rect_assign.hi[0] - rect_assign.lo[0] + 1;
  int batch_size = input_rows;
  int data_dim = input_cols;

  // get output
  float* output_grads[n];
  //int exp_output_rows = (int)ceil(alpha*k/n*batch_size);
  for(int i = 0; i < n; i++) {
    Domain out_domain = runtime->get_index_space_domain(
      ctx, task->regions[i+2].region.get_index_space());
    output_grads[i] = helperGetTensorPointerRW<float>(
      regions[i+2], task->regions[i+2], FID_DATA, ctx, runtime);

    //coord_t output_rows = out_domain.hi()[1] - out_domain.lo()[1] + 1;
    coord_t output_cols = out_domain.hi()[0] - out_domain.lo()[0] + 1;
    //assert((int)output_rows == exp_output_rows);
    assert(output_cols == input_cols);
  }

  // TODO: why cublas/cudnn stream is needed here
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  // call forward kernel
  hipMemcpy(m->dev_region_ptrs, output_grads, n*sizeof(float*), hipMemcpyHostToDevice);

  gb_backward_kernel<<<GET_BLOCKS(batch_size*k*data_dim), min(CUDA_NUM_THREADS,(int)(batch_size*k*data_dim)), 0, stream>>>(
    acc_input_grad.ptr(rect_input_grad), acc_assign.ptr(rect_assign), m->dev_region_ptrs,
    n, k, alpha, batch_size, data_dim);
}

GroupByMeta::GroupByMeta(FFHandler handler, int n)
: OpMeta(handler)
{
  checkCUDA(hipMalloc(&dev_region_ptrs, n*sizeof(float*)));
}
GroupByMeta::~GroupByMeta(void)
{
  checkCUDA(hipFree(&dev_region_ptrs));
}

bool Group_by::measure_operator_cost(Simulator* sim,
                                 const ParallelConfig& pc,
                                 CostMetrics& cost_metrics) const
{
  //TODO: implement
  cost_metrics.forward_time = 0.0f;
  cost_metrics.backward_time = 0.0f;
  cost_metrics.memory_requirement = 0;
  return false;
}

}; // namespace FlexFlow
