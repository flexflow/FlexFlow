#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::binary(OperatorType op,
                       const Tensor& in1,
                       const Tensor& in2,
                       bool inplace_a,
                       char const *name)
{
  ElementBinary *ele = new ElementBinary(*this, op, in1, in2, inplace_a, name);
  layers.push_back(ele);
  return ele->outputs[0];
}

Tensor FFModel::add(const Tensor& in1,
                    const Tensor& in2,
                    bool inplace_a,
                    char const *name)
{
  return this->binary(OP_EW_ADD, in1, in2, inplace_a, name);
}

Tensor FFModel::subtract(const Tensor& in1,
                         const Tensor& in2,
                         bool inplace_a,
                         char const *name)
{
  return this->binary(OP_EW_SUB, in1, in2, inplace_a, name);
}

Tensor FFModel::multiply(const Tensor& in1,
                         const Tensor& in2,
                         bool inplace_a,
                         char const *name)
{
  return this->binary(OP_EW_MUL, in1, in2, inplace_a, name);
}

Tensor FFModel::divide(const Tensor& in1,
                       const Tensor& in2,
                       bool inplace_a,
                       char const *name)
{
  return this->binary(OP_EW_DIV, in1, in2, inplace_a, name);
}

ElementBinary::ElementBinary(FFModel& model,
                             OperatorType _op_type,
                             const Tensor& in1,
                             const Tensor& in2,
                             bool _inplace_a,
                             const char* name)
: Op(
    model,
    _op_type,
    name,
    in1,
    in2
  ),
  inplace_a(_inplace_a)
{
  numOutputs = 1;
  numWeights = 0;
  int dim = std::max(in1.numDim, in2.numDim);
  outputs[0].numDim = dim;
  for (int i = 0; i < dim; i++) {
    if (i >= in1.numDim) {
      outputs[0].adim[i] = in2.adim[i];
    } else if (i >= in2.numDim) {
      outputs[0].adim[i] = in1.adim[i];
    } else if (in1.adim[i] == in2.adim[i]) {
      outputs[0].adim[i] = in1.adim[i];
    } else if (in1.adim[i] == 1 || in2.adim[i] == 1) {
      // This is okay since one of them is 1
      outputs[0].adim[i] = in1.adim[i] * in2.adim[i];
    } else {
      assert(false && "Operands could not be broadcast together");
      exit(0);
    }
  }
}

bool ElementBinary::can_inplace_output(void)
{
  if (op_type == OP_EW_ADD || op_type == OP_EW_MUL) {
    // TODO: Currently assume that we always inplace_a
    if (outputs[0].numDim != inputs[0].numDim)
      return false;
    for (int i = 0; i < inputs[0].numDim; i++) {
      if (inputs[0].adim[i] != outputs[0].adim[i])
        return false;
    }
    return true;
  }
  return false;
}

bool ElementBinary::has_inplace_output(void)
{
  return inplace_a;
}

void ElementBinary::do_inplace_output(void)
{
  inplace_a = true;
}

void ElementBinary::create_weights(FFModel& model)
{
  // Do nothing
}

void ElementBinary::create_output_and_partition(FFModel& model)
{
  int odim = outputs[0].numDim;
  for (int idx = 0; idx < 2; idx ++) {
    int idim = inputs[idx].numDim;
    switch (odim * MAX_TENSOR_DIM + idim) {
#define DIMFUNC(ODIM, IDIM) \
      case ODIM * MAX_TENSOR_DIM + IDIM: \
      { \
        create_output_and_partition_with_dim<ODIM, IDIM>(model, idx); \
        break; \
      }
      LEGION_FOREACH_NN(DIMFUNC)
#undef DIMFUNC
      default:
      {
        // Unsupported dim for ElementWiseBinary operator
        assert(false);
      }
    }
  }
}

template<int ODIM, int IDIM>
void ElementBinary::create_output_and_partition_with_dim(FFModel& model, int idx)
{
  // Retrive the task indexspace for the op
  task_is = IndexSpaceT<ODIM>(model.get_or_create_task_is(ODIM, name));
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Domain part_rect = runtime->get_index_space_domain(ctx, task_is);
  if (inplace_a && idx == 0) {
    assert(IDIM == ODIM);
    outputs[0] = inputs[0];
    outputs[0].owner_op = this;
    outputs[0].owner_idx = 0;
    Domain input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[idx].part.get_index_partition());
    // inplace_a require part_rect == inputs[0].part_rect
    assert(input_rect == part_rect);
    if (input_rect == part_rect) {
      input_lps[idx] = inputs[idx].part;
      input_grad_lps[idx] = inputs[idx].part_grad;
    } else {
      model.create_disjoint_partition<ODIM>(
          inputs[idx], IndexSpaceT<ODIM>(task_is), input_lps[idx], input_grad_lps[idx]);
    }
    return;
  }
  // Create output tensor when idx == 0
  if (idx == 0) {
    int dims[ODIM];
    for (int i = 0; i < ODIM; i++)
      dims[i] = outputs[0].adim[ODIM-1-i];
    outputs[0] = model.create_tensor<ODIM>(dims, DT_FLOAT, this);
    outputs[0].owner_op = this;
    outputs[0].owner_idx = 0;
  }
  Domain input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[idx].part.get_index_partition());
  if (input_rect == part_rect) {
    input_lps[idx] = inputs[idx].part;
    input_grad_lps[idx] = inputs[idx].part_grad;
  } else if (IDIM == ODIM) {
    model.create_disjoint_partition<IDIM>(
        inputs[idx], IndexSpaceT<IDIM>(task_is), input_lps[idx], input_grad_lps[idx]);
  } else {
    model.create_data_parallel_partition_with_diff_dims<IDIM, ODIM>(
        inputs[idx], (IndexSpaceT<ODIM>)task_is, input_lps[idx], input_grad_lps[idx]);
  }
}

__host__
OpMeta* ElementBinary::init_task(const Task* task,
                                 const std::vector<PhysicalRegion> &regions,
                                 Context ctx, Runtime* runtime)
{
  ElementBinary* eb = (ElementBinary*) task->args;
  FFHandler handle = *((FFHandler*) task->local_args);
  ElementBinaryMeta* m = new ElementBinaryMeta(handle);
  m->op_type = eb->op_type;
  m->profiling = eb->profiling;
  m->inplace_a = eb->inplace_a;
  Domain input_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain output_domain;
  if (m->inplace_a) {
    assert(regions.size() == 2);
    assert(task->regions.size() == regions.size());
    output_domain = runtime->get_index_space_domain(
        ctx, task->regions[1].region.get_index_space());
    assert(output_domain == input_domain);
  } else {
    assert(regions.size() == 3);
    assert(task->regions.size() == regions.size());
    output_domain = runtime->get_index_space_domain(
        ctx, task->regions[2].region.get_index_space());
    // check that input can broadcast to output
    for (int i = 0; i < input_domain.dim; i++) {
      int input_dim_size = input_domain.hi()[i] - input_domain.lo()[i] + 1;
      int output_dim_size = output_domain.hi()[i] - output_domain.lo()[i] + 1;
      assert(input_dim_size == output_dim_size || input_dim_size == 1);
    }
  }
  hipdnnOpTensorOp_t mode;
  switch (eb->op_type) {
    case OP_EW_ADD:
    case OP_EW_SUB:
      mode = HIPDNN_OP_TENSOR_ADD;
      break;
    case OP_EW_MUL:
      mode = HIPDNN_OP_TENSOR_MUL;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnSetOpTensorDescriptor(m->opDesc, mode,
      HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN));
  checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->inputTensor, input_domain));
  checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->outputTensor, output_domain));
  return m;
}

void ElementBinary::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      ParallelConfig pc; \
      std::string pcname = name; \
      ff.config.find_parallel_config(DIM, pcname, pc); \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        FFHandler handle = ff.handlers[pc.device_ids[idx++]]; \
        argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(ELEMENTBINARY_INIT_TASK_ID, task_is,
                         TaskArgument(this, sizeof(ElementBinary)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
    RegionRequirement(input_lps[0], 0/*projection id*/,
      READ_WRITE, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
    RegionRequirement(input_lps[1], 0/*projection id*/,
      READ_WRITE, EXCLUSIVE, inputs[1].region));
  launcher.add_field(1, FID_DATA);
  if (!inplace_a) {
    launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
        WRITE_ONLY, EXCLUSIVE, outputs[0].region));
    launcher.add_field(2, FID_DATA);
  } else {
    assert(outputs[0].part == input_lps[0]);
    assert(outputs[0].region == inputs[0].region);
  }
  //launcher.add_region_requirement(
  //  RegionRequirement(input_grad_lps[0], 0/*projection id*/,
  //    WRITE_ONLY, EXCLUSIVE, inputs[0].region_grad));
  //launcher.add_field(3, FID_DATA);
  //if (inputs[0].region_grad != inputs[1].region_grad) {
    // regions[4](I/O): input1_grad
  //  launcher.add_region_requirement(
  //    RegionRequirement(input_grad_lps[1], 0/*projection id*/,
  //                      WRITE_ONLY, EXCLUSIVE, inputs[1].region_grad));
  //  launcher.add_field(4, FID_DATA);
  //}
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        meta[idx++] = fm.get_result<OpMeta*>(*it); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

__global__
void elewise_binary_forward_kernel(coord_t volume,
                                   const float alpha,
                                   const float beta,
                                   OperatorType type,
                                   const float* in1,
                                   const float* in2,
                                   float* out)
{
  switch (type) {
    case OP_EW_ADD:
    {
      CUDA_KERNEL_LOOP(i, volume)
      {
        out[i] = alpha * (in1[i] + in2[i]) + beta * out[i];
      }
      break;
    }
    case OP_EW_SUB:
    {
      CUDA_KERNEL_LOOP(i, volume)
      {
        out[i] = alpha * (in1[i] - in2[i]) + beta * out[i];
      }
      break;
    }
    case OP_EW_MUL:
    {
      CUDA_KERNEL_LOOP(i, volume)
      {
        out[i] = alpha * in1[i] * in2[i] + beta * out[i];
      }
      break;
    }
    case OP_EW_DIV:
    {
      CUDA_KERNEL_LOOP(i, volume)
      {
        out[i] = alpha * (in1[i] / in2[i]) + beta * out[i];
      }
      break;
    }
    default:
      assert(false);
  }
}

/*static*/
void ElementBinary::forward_kernel(const ElementBinaryMeta* m,
                                   const float* in1_ptr,
                                   const float* in2_ptr,
                                   float* out_ptr,
                                   hipStream_t stream)
{
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha1 = 1.0f, alpha2 = 1.0f, beta = 0.0f;
  switch (m->op_type) {
    case OP_EW_SUB:
      alpha2 = -1.0f;
      break;
    case OP_EW_ADD:
    case OP_EW_MUL:
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnOpTensor(m->handle.dnn, m->opDesc,
      &alpha1, m->inputTensor, in1_ptr,
      &alpha2, m->inputTensor, in2_ptr,
      &beta, m->outputTensor, out_ptr));
}

/*
  regions[0](I): in1
  regions[1](I): in2
  regions[2](O): output
*/
__host__
void ElementBinary::forward_task(const Task* task,
                                 const std::vector<PhysicalRegion> &regions,
                                 Context ctx, Runtime* runtime)
{
  //const ElementBinary* ele = (const ElementBinary*) task->args;
  const ElementBinaryMeta* m = *((ElementBinaryMeta**) task->local_args);
  Domain in1_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain in2_domain = runtime->get_index_space_domain(
    ctx, task->regions[1].region.get_index_space());
  assert(in1_domain == in2_domain);
  const float* in1_ptr = NULL, *in2_ptr = NULL;
  float *out_ptr = NULL;
  if (m->inplace_a) {
    assert(regions.size() == 2);
    assert(task->regions.size() == 2);
    out_ptr = helperGetTensorPointerRW<float>(
        regions[0], task->regions[0], FID_DATA, ctx, runtime);
    in2_ptr = helperGetTensorPointerRO<float>(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
    in1_ptr = out_ptr;
  } else {
    assert(regions.size() == 3);
    assert(task->regions.size() == 3);
    Domain out_domain = runtime->get_index_space_domain(
        ctx, task->regions[2].region.get_index_space());
    assert(out_domain == in1_domain);
    in1_ptr = helperGetTensorPointerRO<float>(
        regions[0], task->regions[0], FID_DATA, ctx, runtime);
    in2_ptr = helperGetTensorPointerRO<float>(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
    out_ptr = helperGetTensorPointerWO<float>(
        regions[2], task->regions[2], FID_DATA, ctx, runtime);
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  //print_tensor<float>(in1_ptr, in1_domain.get_volume(), "input1:");
  //print_tensor<float>(in2_ptr, in2_domain.get_volume(), "input2:");
  forward_kernel(m, in1_ptr, in2_ptr, out_ptr, stream);
  //print_tensor<float>(out_ptr, in1_domain.get_volume(), "output:");
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    char const *opName;
    switch (m->op_type) {
      case OP_EW_ADD:
        opName = "Add";
        break;
      case OP_EW_SUB:
        opName = "Sub";
        break;
      case OP_EW_MUL:
        opName = "Mul";
        break;
      case OP_EW_DIV:
        opName = "Div";
        break;
      default:
        assert(false);
    }
    printf("[%s] forward time (CF) = %.2fms\n", opName, elapsed);
  }
}

void ElementBinary::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(ELEMENTBINARY_FWD_TASK_ID, task_is,
      TaskArgument(NULL, 0), argmap,
      Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
      FFConfig::get_hash_id(std::string(name)));
  if (inplace_a) {
    assert(outputs[0].part == input_lps[0]);
    assert(outputs[0].region == inputs[0].region);
    launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
        READ_WRITE, EXCLUSIVE, inputs[0].region));
    launcher.add_field(0, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(input_lps[1], 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[1].region));
    launcher.add_field(1, FID_DATA);
  } else {
    launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[0].region));
    launcher.add_field(0, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(input_lps[1], 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[1].region));
    launcher.add_field(1, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
        WRITE_ONLY, EXCLUSIVE, outputs[0].region));
    launcher.add_field(2, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

__global__
void elewise_binary_backward_kernel(coord_t volume,
                                    const float alpha,
                                    const float beta,
                                    OperatorType type,
                                    const float* out_grad,
                                    const float* in1,
                                    const float* in2,
                                    float* in1_grad,
                                    float* in2_grad)
{
  CUDA_KERNEL_LOOP(i, volume)
  {
    switch (type) {
      case OP_EW_ADD:
      {
        in1_grad[i] = alpha * out_grad[i] + beta * in1_grad[i];
        in2_grad[i] = alpha * out_grad[i] + beta * in2_grad[i];
        break;
      }
      case OP_EW_SUB:
      {
        in1_grad[i] = alpha * out_grad[i] + beta * in1_grad[i];
        in2_grad[i] = - alpha * out_grad[i] + beta * in2_grad[i];
        break;
      }
      case OP_EW_MUL:
      {
        in1_grad[i] = alpha * out_grad[i] * in2[i] + beta * in1_grad[i];
        in2_grad[i] = alpha * out_grad[i] * in1[i] + beta * in2_grad[i];
        break;
      }
      case OP_EW_DIV:
      {
        in1_grad[i] = alpha * out_grad[i] / in2[i] + beta * in1_grad[i];
        in2_grad[i] = - alpha * out_grad[i] * in1[i] / (in2[i] * in2[i]) + beta * in2_grad[i];
        break;
      }
      default:
        assert(false);
    }
  }
}

/*static*/
void ElementBinary::backward_kernel(const ElementBinaryMeta* m,
                                    const float* out_grad_ptr,
                                    const float* in1_ptr,
                                    const float* in2_ptr,
                                    float* in1_grad_ptr,
                                    float* in2_grad_ptr,
                                    hipStream_t stream)
{
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha1 = 1.0f, alpha2 = 1.0f, beta = 1.0f;
  switch (m->op_type) {
    case OP_EW_ADD:
      alpha1 = 1.0f;
      alpha2 = 0.0f;
      break;
    case OP_EW_SUB:
      alpha1 = -1.0f;
      alpha2 = 0.0f;
      break;
    case OP_EW_MUL:
      alpha1 = 1.0f;
      alpha2 = 1.0f;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnOpTensor(m->handle.dnn, m->opDesc,
      &alpha1, m->outputTensor, out_grad_ptr,
      &alpha2, m->inputTensor, in1_ptr,
      &beta, m->inputTensor, in2_grad_ptr));
  switch (m->op_type) {
    case OP_EW_ADD:
    case OP_EW_SUB:
      alpha1 = 1.0f;
      alpha2 = 0.0f;
      break;
    case OP_EW_MUL:
      alpha1 = 1.0f;
      alpha2 = 1.0f;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnOpTensor(m->handle.dnn, m->opDesc,
      &alpha1, m->outputTensor, out_grad_ptr,
      &alpha2, m->inputTensor, in2_ptr,
      &beta, m->inputTensor, in1_grad_ptr));
}

/*
  regions[0](I or I/O): out_grad (I/O if inplace_a)
  regions[1](I): in0
  regions[2](I/O): in0_grad (Missing if in0_grad = out_grad)
  regions[3](I): in1 (Missing if in0 = in1)
  regions[4](I/O): in1_grad (Missing if in0=in1)
*/
void ElementBinary::backward_task(const Task *task,
                            const std::vector<PhysicalRegion> &regions,
                            Context ctx, Runtime* runtime)
{
  //const ElementBinary* ele = (const ElementBinary*) task->args;
  const ElementBinaryMeta* m = *((ElementBinaryMeta**) task->local_args);
  const float *in0_ptr = NULL, *in1_ptr = NULL, *out_grad_ptr = NULL;
  float *in0_grad_ptr = NULL, *in1_grad_ptr = NULL;
  Domain out_grad_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  if (m->inplace_a) {
    in0_grad_ptr = helperGetTensorPointerRW<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    if (regions.size() == 2 || regions.size() == 4);
    assert(task->regions.size() == regions.size());
    if (regions.size() == 2) {
      Domain in0_domain = runtime->get_index_space_domain(
        ctx, task->regions[1].region.get_index_space());
      assert(in0_domain == out_grad_domain);
      in0_ptr = helperGetTensorPointerRO<float>(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
      in1_ptr = in0_ptr;
      in1_grad_ptr = in0_grad_ptr;
      out_grad_ptr = in0_grad_ptr;
    } else {
      Domain in0_domain = runtime->get_index_space_domain(
        ctx, task->regions[1].region.get_index_space());
      Domain in1_domain = runtime->get_index_space_domain(
        ctx, task->regions[2].region.get_index_space());
      assert(in0_domain == out_grad_domain);
      assert(in1_domain == out_grad_domain);
      in0_ptr = helperGetTensorPointerRO<float>(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
      in1_ptr = helperGetTensorPointerRO<float>(
        regions[2], task->regions[2], FID_DATA, ctx, runtime);
      in1_grad_ptr = helperGetTensorPointerRW<float>(
        regions[3], task->regions[3], FID_DATA, ctx, runtime);
      out_grad_ptr = in0_grad_ptr;
    }
  } else {
    assert(regions.size() == 3 || regions.size() == 5);
    assert(task->regions.size() == regions.size());
    out_grad_ptr = helperGetTensorPointerRO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    Domain in0_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
    Domain in0_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[2].region.get_index_space());
    assert(out_grad_domain == in0_grad_domain);
    assert(out_grad_domain == in0_domain);
    in0_ptr = helperGetTensorPointerRO<float>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
    in0_grad_ptr = helperGetTensorPointerRW<float>(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
    if (regions.size() == 3) {
      // in0 == in1
      in1_ptr = in0_ptr;
      in1_grad_ptr = in0_grad_ptr;
    } else {
      Domain in1_domain = runtime->get_index_space_domain(
        ctx, task->regions[3].region.get_index_space());
      Domain in1_grad_domain = runtime->get_index_space_domain(
        ctx, task->regions[4].region.get_index_space());
      assert(out_grad_domain == in1_domain);
      assert(out_grad_domain == in1_grad_domain);
      in1_ptr = helperGetTensorPointerRO<float>(
        regions[3], task->regions[3], FID_DATA, ctx, runtime);
      in1_grad_ptr = helperGetTensorPointerRW<float>(
        regions[4], task->regions[4], FID_DATA, ctx, runtime);
    }
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  backward_kernel(m, out_grad_ptr, in0_ptr, in1_ptr, in0_grad_ptr, in1_grad_ptr, stream);
  //elewise_binary_backward_kernel<<<GET_BLOCKS(out_grad_domain.get_volume()), CUDA_NUM_THREADS>>>(
    //out_grad_domain.get_volume(), alpha, alpha, ele->op_type, out_grad_ptr, in1_ptr, in2_ptr,
    //in1_grad_ptr, in2_grad_ptr);
}

void ElementBinary::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }

  IndexLauncher launcher(ELEMENTBINARY_BWD_TASK_ID, task_is,
      TaskArgument(NULL, 0), argmap,
      Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
      FFConfig::get_hash_id(std::string(name)));
  if (inplace_a) {
    // regions[0](I/O): output_grad
    launcher.add_region_requirement(
      RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, outputs[0].region_grad));
    launcher.add_field(0, FID_DATA);
    // regions[1](I): input0
    launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
    launcher.add_field(1, FID_DATA);
    if (inputs[0].region == inputs[1].region) {
      // regions[3](I): input1
      launcher.add_region_requirement(
        RegionRequirement(input_lps[1], 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, inputs[1].region));
      launcher.add_field(2, FID_DATA);
      // regions[4](I/O): input1_grad
      launcher.add_region_requirement(
        RegionRequirement(input_grad_lps[1], 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, inputs[1].region_grad));
      launcher.add_field(3, FID_DATA);
    }
  } else {
    // regions[0](I): output_grad
    launcher.add_region_requirement(
      RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, outputs[0].region_grad));
    launcher.add_field(0, FID_DATA);
    // regions[1](I): input0
    launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
    launcher.add_field(1, FID_DATA);
    // regions[2](I/O): input0_grad
    launcher.add_region_requirement(
      RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
    launcher.add_field(2, FID_DATA);
    if (inputs[0].region == inputs[1].region) {
      // regions[3](I): input1
      launcher.add_region_requirement(
        RegionRequirement(input_lps[1], 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, inputs[1].region));
      launcher.add_field(3, FID_DATA);
      // regions[4](I/O): input1_grad
      launcher.add_region_requirement(
        RegionRequirement(input_grad_lps[1], 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, inputs[1].region_grad));
      launcher.add_field(4, FID_DATA);
    }
  }
  runtime->execute_index_space(ctx, launcher);
}

ElementBinaryMeta::ElementBinaryMeta(FFHandler handler)
: OpMeta(handler)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateOpTensorDescriptor(&opDesc));
  op_type = OP_ANY;
}

bool ElementBinary::measure_operator_cost(Simulator* sim,
                                          const ParallelConfig& pc,
                                          CostMetrics& cost_metrics)
{
  Tensor sub_output, sub_input1, sub_input0;
  if (!outputs[0].get_output_sub_tensor(pc, sub_output, op_type))
    return false;
  if (!inputs[0].get_input_sub_tensor(pc, sub_input0, op_type))
    return false;
  if (!inputs[1].get_input_sub_tensor(pc, sub_input1, op_type))
    return false;
  ElementBinaryMeta* m = sim->ele_binary_meta;
  m->op_type = op_type;
  hipdnnOpTensorOp_t mode;
  switch (op_type) {
    case OP_EW_ADD:
    case OP_EW_SUB:
      mode = HIPDNN_OP_TENSOR_ADD;
      break;
    case OP_EW_MUL:
      mode = HIPDNN_OP_TENSOR_MUL;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnSetOpTensorDescriptor(m->opDesc, mode,
      HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN));
  Domain input_domain = sub_input0.get_domain();
  Domain output_domain = sub_output.get_domain();
  checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->inputTensor, input_domain));
  checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->outputTensor, output_domain));
  sim->free_all();
  float* input0_ptr = (float*)sim->allocate(sub_input0.get_volume(), DT_FLOAT);
  assert(input0_ptr != NULL);
  float* input1_ptr = (float*)sim->allocate(sub_input1.get_volume(), DT_FLOAT);
  assert(input1_ptr != NULL);
  float* output_ptr = NULL;
  if (inplace_a) {
    output_ptr = input0_ptr;
  } else {
    output_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  }
  assert(output_ptr != NULL);

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  std::function<void()> forward, backward;
  forward = [&] {
    forward_kernel(m, input0_ptr, input1_ptr, output_ptr, stream);
  };
  if (sim->computationMode == COMP_MODE_TRAINING) {
    float* input0_grad_ptr = (float*)sim->allocate(sub_input0.get_volume(), DT_FLOAT);
    assert(input0_grad_ptr != NULL);
    float* input1_grad_ptr = (float*)sim->allocate(sub_input0.get_volume(), DT_FLOAT);
    assert(input1_grad_ptr != NULL);
    float* output_grad_ptr = NULL;
    if (inplace_a) {
      output_grad_ptr = input0_grad_ptr;
    } else {
      output_grad_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
    }
    assert(output_grad_ptr != NULL);
    backward = [&] {
      backward_kernel(m, output_grad_ptr, input0_ptr, input1_ptr, input0_grad_ptr, input1_grad_ptr, stream);
    };
  }

  inner_measure_operator_cost(sim, forward, backward, cost_metrics);

  if (sim->computationMode == COMP_MODE_TRAINING) {
    printf("[Measure Elewise Binary] name(%s) num_elements(%zu) forward_time(%.4lf) backward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time,
        cost_metrics.backward_time);
  } else {
    printf("[Measure Elewise Binary] name(%s) num_elements(%zu) forward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time);
  }

  return true;
}
