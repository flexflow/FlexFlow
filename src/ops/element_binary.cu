#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/element_binary.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {
// declare Legion names
using Legion::Context;
using Legion::Runtime;
using Legion::Domain;
using Legion::Task;
using Legion::Rect;
using Legion::PhysicalRegion;
using Legion::coord_t;

bool ElementBinary::can_inplace_output(void)
{
  if (op_type == OP_EW_ADD)
    return false;
  if (op_type == OP_EW_MUL)
    return false;
  return false;
}

bool ElementBinary::has_inplace_output(void)
{
  return inplace_a;
}

void ElementBinary::do_inplace_output(void)
{
  inplace_a = true;
}

__host__
OpMeta* ElementBinary::init_task(const Task* task,
                                 const std::vector<PhysicalRegion> &regions,
                                 Context ctx, Runtime* runtime)
{
  ElementBinary* eb = (ElementBinary*) task->args;
  FFHandler handle = *((FFHandler*) task->local_args);
  ElementBinaryMeta* m = new ElementBinaryMeta(handle);
  m->op_type = eb->op_type;
  m->profiling = eb->profiling;
  m->inplace_a = eb->inplace_a;
  Domain input_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain output_domain;
  if (m->inplace_a) {
    assert(regions.size() == 2);
    assert(task->regions.size() == regions.size());
    output_domain = runtime->get_index_space_domain(
        ctx, task->regions[1].region.get_index_space());
    assert(output_domain == input_domain);
  } else {
    assert(regions.size() == 3);
    assert(task->regions.size() == regions.size());
    output_domain = runtime->get_index_space_domain(
        ctx, task->regions[2].region.get_index_space());
    assert(output_domain == input_domain);
  }
  hipdnnOpTensorOp_t mode;
  switch (eb->op_type) {
    case OP_EW_ADD:
    case OP_EW_SUB:
      mode = HIPDNN_OP_TENSOR_ADD;
      break;
    case OP_EW_MUL:
      mode = HIPDNN_OP_TENSOR_MUL;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnSetOpTensorDescriptor(m->opDesc, mode,
      HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN));
  checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->inputTensor, input_domain));
  checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->outputTensor, output_domain));
  return m;
}

__global__
void elewise_binary_forward_kernel(coord_t volume,
                                   const float alpha,
                                   const float beta,
                                   OperatorType type,
                                   const float* in1,
                                   const float* in2,
                                   float* out)
{
  switch (type) {
    case OP_EW_ADD:
    {
      CUDA_KERNEL_LOOP(i, volume)
      {
        out[i] = alpha * (in1[i] + in2[i]) + beta * out[i];
      }
      break;
    }
    case OP_EW_SUB:
    {
      CUDA_KERNEL_LOOP(i, volume)
      {
        out[i] = alpha * (in1[i] - in2[i]) + beta * out[i];
      }
      break;
    }
    case OP_EW_MUL:
    {
      CUDA_KERNEL_LOOP(i, volume)
      {
        out[i] = alpha * in1[i] * in2[i] + beta * out[i];
      }
      break;
    }
    case OP_EW_DIV:
    {
      CUDA_KERNEL_LOOP(i, volume)
      {
        out[i] = alpha * (in1[i] / in2[i]) + beta * out[i];
      }
      break;
    }
    default:
      assert(false);
  }
}

/*static*/
void ElementBinary::forward_kernel(const ElementBinaryMeta* m,
                                   const float* in1_ptr,
                                   const float* in2_ptr,
                                   float* out_ptr,
                                   hipStream_t stream)
{
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha1 = 1.0f, alpha2 = 1.0f, beta = 0.0f;
  switch (m->op_type) {
    case OP_EW_SUB:
      alpha2 = -1.0f;
      break;
    case OP_EW_ADD:
    case OP_EW_MUL:
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnOpTensor(m->handle.dnn, m->opDesc,
      &alpha1, m->inputTensor, in1_ptr,
      &alpha2, m->inputTensor, in2_ptr,
      &beta, m->outputTensor, out_ptr));
}

/*
  regions[0](I): in1
  regions[1](I): in2
  regions[2](O): output
*/
__host__
void ElementBinary::forward_task(const Task* task,
                                 const std::vector<PhysicalRegion> &regions,
                                 Context ctx, Runtime* runtime)
{
  //const ElementBinary* ele = (const ElementBinary*) task->args;
  const ElementBinaryMeta* m = *((ElementBinaryMeta**) task->local_args);
  Domain in1_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  Domain in2_domain = runtime->get_index_space_domain(
    ctx, task->regions[1].region.get_index_space());
  assert(in1_domain == in2_domain);
  const float* in1_ptr = NULL, *in2_ptr = NULL;
  float *out_ptr = NULL;
  if (m->inplace_a) {
    assert(regions.size() == 2);
    assert(task->regions.size() == 2);
    out_ptr = helperGetTensorPointerRW<float>(
        regions[0], task->regions[0], FID_DATA, ctx, runtime);
    in2_ptr = helperGetTensorPointerRO<float>(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
    in1_ptr = out_ptr;
  } else {
    assert(regions.size() == 3);
    assert(task->regions.size() == 3);
    Domain out_domain = runtime->get_index_space_domain(
        ctx, task->regions[2].region.get_index_space());
    assert(out_domain == in1_domain);
    in1_ptr = helperGetTensorPointerRO<float>(
        regions[0], task->regions[0], FID_DATA, ctx, runtime);
    in2_ptr = helperGetTensorPointerRO<float>(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
    out_ptr = helperGetTensorPointerWO<float>(
        regions[2], task->regions[2], FID_DATA, ctx, runtime);
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  //print_tensor<float>(in1_ptr, in1_domain.get_volume(), "input1:");
  //print_tensor<float>(in2_ptr, in2_domain.get_volume(), "input2:");
  forward_kernel(m, in1_ptr, in2_ptr, out_ptr, stream);
  //print_tensor<float>(out_ptr, in1_domain.get_volume(), "output:");
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    char const *opName;
    switch (m->op_type) {
      case OP_EW_ADD:
        opName = "Add";
        break;
      case OP_EW_SUB:
        opName = "Sub";
        break;
      case OP_EW_MUL:
        opName = "Mul";
        break;
      case OP_EW_DIV:
        opName = "Div";
        break;
      default:
        assert(false);
    }
    log_measure.debug("[%s] forward time (CF) = %.2fms\n", opName, elapsed);
  }
}

__global__
void elewise_binary_backward_kernel(coord_t volume,
                                    const float alpha,
                                    const float beta,
                                    OperatorType type,
                                    const float* out_grad,
                                    const float* in1,
                                    const float* in2,
                                    float* in1_grad,
                                    float* in2_grad)
{
  CUDA_KERNEL_LOOP(i, volume)
  {
    switch (type) {
      case OP_EW_ADD:
      {
        in1_grad[i] = alpha * out_grad[i] + beta * in1_grad[i];
        in2_grad[i] = alpha * out_grad[i] + beta * in2_grad[i];
        break;
      }
      case OP_EW_SUB:
      {
        in1_grad[i] = alpha * out_grad[i] + beta * in1_grad[i];
        in2_grad[i] = - alpha * out_grad[i] + beta * in2_grad[i];
        break;
      }
      case OP_EW_MUL:
      {
        in1_grad[i] = alpha * out_grad[i] * in2[i] + beta * in1_grad[i];
        in2_grad[i] = alpha * out_grad[i] * in1[i] + beta * in2_grad[i];
        break;
      }
      case OP_EW_DIV:
      {
        in1_grad[i] = alpha * out_grad[i] / in2[i] + beta * in1_grad[i];
        in2_grad[i] = - alpha * out_grad[i] * in1[i] / (in2[i] * in2[i]) + beta * in2_grad[i];
        break;
      }
      default:
        assert(false);
    }
  }
}

/*static*/
void ElementBinary::backward_kernel(const ElementBinaryMeta* m,
                                    const float* out_grad_ptr,
                                    const float* in1_ptr,
                                    const float* in2_ptr,
                                    float* in1_grad_ptr,
                                    float* in2_grad_ptr,
                                    hipStream_t stream)
{
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha1 = 1.0f, alpha2 = 1.0f, beta = 1.0f;
  switch (m->op_type) {
    case OP_EW_ADD:
      alpha1 = 1.0f;
      alpha2 = 0.0f;
      break;
    case OP_EW_SUB:
      alpha1 = -1.0f;
      alpha2 = 0.0f;
      break;
    case OP_EW_MUL:
      alpha1 = 1.0f;
      alpha2 = 1.0f;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnOpTensor(m->handle.dnn, m->opDesc,
      &alpha1, m->outputTensor, out_grad_ptr,
      &alpha2, m->inputTensor, in1_ptr,
      &beta, m->inputTensor, in2_grad_ptr));
  switch (m->op_type) {
    case OP_EW_ADD:
    case OP_EW_SUB:
      alpha1 = 1.0f;
      alpha2 = 0.0f;
      break;
    case OP_EW_MUL:
      alpha1 = 1.0f;
      alpha2 = 1.0f;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnOpTensor(m->handle.dnn, m->opDesc,
      &alpha1, m->outputTensor, out_grad_ptr,
      &alpha2, m->inputTensor, in2_ptr,
      &beta, m->inputTensor, in1_grad_ptr));
}

/*
  regions[0](I or I/O): out_grad (I/O if inplace_a)
  regions[1](I): in0
  regions[2](I/O): in0_grad (Missing if in0_grad = out_grad)
  regions[3](I): in1 (Missing if in0 = in1)
  regions[4](I/O): in1_grad (Missing if in0=in1)
*/
void ElementBinary::backward_task(const Task *task,
                            const std::vector<PhysicalRegion> &regions,
                            Context ctx, Runtime* runtime)
{
  //const ElementBinary* ele = (const ElementBinary*) task->args;
  const ElementBinaryMeta* m = *((ElementBinaryMeta**) task->local_args);
  const float *in0_ptr = NULL, *in1_ptr = NULL, *out_grad_ptr = NULL;
  float *in0_grad_ptr = NULL, *in1_grad_ptr = NULL;
  Domain out_grad_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  if (m->inplace_a) {
    in0_grad_ptr = helperGetTensorPointerRW<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    if (regions.size() == 2 || regions.size() == 4);
    assert(task->regions.size() == regions.size());
    if (regions.size() == 2) {
      Domain in0_domain = runtime->get_index_space_domain(
        ctx, task->regions[1].region.get_index_space());
      assert(in0_domain == out_grad_domain);
      in0_ptr = helperGetTensorPointerRO<float>(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
      in1_ptr = in0_ptr;
      in1_grad_ptr = in0_grad_ptr;
      out_grad_ptr = in0_grad_ptr;
    } else {
      Domain in0_domain = runtime->get_index_space_domain(
        ctx, task->regions[1].region.get_index_space());
      Domain in1_domain = runtime->get_index_space_domain(
        ctx, task->regions[2].region.get_index_space());
      assert(in0_domain == out_grad_domain);
      assert(in1_domain == out_grad_domain);
      in0_ptr = helperGetTensorPointerRO<float>(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
      in1_ptr = helperGetTensorPointerRO<float>(
        regions[2], task->regions[2], FID_DATA, ctx, runtime);
      in1_grad_ptr = helperGetTensorPointerRW<float>(
        regions[3], task->regions[3], FID_DATA, ctx, runtime);
      out_grad_ptr = in0_grad_ptr;
    }
  } else {
    assert(regions.size() == 3 || regions.size() == 5);
    assert(task->regions.size() == regions.size());
    out_grad_ptr = helperGetTensorPointerRO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    Domain in0_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
    Domain in0_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[2].region.get_index_space());
    assert(out_grad_domain == in0_grad_domain);
    assert(out_grad_domain == in0_domain);
    in0_ptr = helperGetTensorPointerRO<float>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
    in0_grad_ptr = helperGetTensorPointerRW<float>(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
    if (regions.size() == 3) {
      // in0 == in1
      in1_ptr = in0_ptr;
      in1_grad_ptr = in0_grad_ptr;
    } else {
      Domain in1_domain = runtime->get_index_space_domain(
        ctx, task->regions[3].region.get_index_space());
      Domain in1_grad_domain = runtime->get_index_space_domain(
        ctx, task->regions[4].region.get_index_space());
      assert(out_grad_domain == in1_domain);
      assert(out_grad_domain == in1_grad_domain);
      in1_ptr = helperGetTensorPointerRO<float>(
        regions[3], task->regions[3], FID_DATA, ctx, runtime);
      in1_grad_ptr = helperGetTensorPointerRW<float>(
        regions[4], task->regions[4], FID_DATA, ctx, runtime);
    }
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  backward_kernel(m, out_grad_ptr, in0_ptr, in1_ptr, in0_grad_ptr, in1_grad_ptr, stream);
  //elewise_binary_backward_kernel<<<GET_BLOCKS(out_grad_domain.get_volume()), CUDA_NUM_THREADS>>>(
    //out_grad_domain.get_volume(), alpha, alpha, ele->op_type, out_grad_ptr, in1_ptr, in2_ptr,
    //in1_grad_ptr, in2_grad_ptr);
}

ElementBinaryMeta::ElementBinaryMeta(FFHandler handler)
: OpMeta(handler)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateOpTensorDescriptor(&opDesc));
  op_type = OP_NOOP;
}

bool ElementBinary::measure_operator_cost(Simulator* sim,
                                          const ParallelConfig& pc,
                                          CostMetrics& cost_metrics) const
{
  TensorBase sub_output, sub_input1, sub_input0;
  if (!outputs[0]->get_output_sub_tensor(pc, sub_output, op_type))
    return false;
  if (!inputs[0]->get_input_sub_tensor(pc, sub_input0, op_type))
    return false;
  if (!inputs[1]->get_input_sub_tensor(pc, sub_input1, op_type))
    return false;
  ElementBinaryMeta* m = sim->ele_binary_meta;
  m->op_type = op_type;
  hipdnnOpTensorOp_t mode;
  switch (op_type) {
    case OP_EW_ADD:
    case OP_EW_SUB:
      mode = HIPDNN_OP_TENSOR_ADD;
      break;
    case OP_EW_MUL:
      mode = HIPDNN_OP_TENSOR_MUL;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnSetOpTensorDescriptor(m->opDesc, mode,
      HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN));
  Domain input_domain = sub_input0.get_domain();
  Domain output_domain = sub_output.get_domain();
  checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->inputTensor, input_domain));
  checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->outputTensor, output_domain));
  sim->free_all();
  float* input0_ptr = (float*)sim->allocate(sub_input0.get_volume(), DT_FLOAT);
  assert(input0_ptr != NULL);
  float* input1_ptr = (float*)sim->allocate(sub_input1.get_volume(), DT_FLOAT);
  assert(input1_ptr != NULL);
  float* output_ptr = NULL;
  if (inplace_a) {
    output_ptr = input0_ptr;
  } else {
    output_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  }
  assert(output_ptr != NULL);

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  std::function<void()> forward, backward;
  forward = [&] {
    forward_kernel(m, input0_ptr, input1_ptr, output_ptr, stream);
  };
  if (sim->computationMode == COMP_MODE_TRAINING) {
    float* input0_grad_ptr = (float*)sim->allocate(sub_input0.get_volume(), DT_FLOAT);
    assert(input0_grad_ptr != NULL);
    float* input1_grad_ptr = (float*)sim->allocate(sub_input0.get_volume(), DT_FLOAT);
    assert(input1_grad_ptr != NULL);
    float* output_grad_ptr = NULL;
    if (inplace_a) {
      output_grad_ptr = input0_grad_ptr;
    } else {
      output_grad_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
    }
    assert(output_grad_ptr != NULL);
    backward = [&] {
      backward_kernel(m, output_grad_ptr, input0_ptr, input1_ptr, input0_grad_ptr, input1_grad_ptr, stream);
    };
  }

  inner_measure_operator_cost(sim, forward, backward, cost_metrics);

  if (sim->computationMode == COMP_MODE_TRAINING) {
    log_measure.debug("[Measure Elewise Binary] name(%s) num_elements(%zu) forward_time(%.4lf) backward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time,
        cost_metrics.backward_time);
  } else {
    log_measure.debug("[Measure Elewise Binary] name(%s) num_elements(%zu) forward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time);
  }

  return true;
}

}; // namespace FlexFlow
