/* Copyright 2021 Facebook
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

using namespace Legion;

Tensor FFModel::multihead_attention(const Tensor query,
                                    const Tensor key,
                                    const Tensor value,
                                    int embed_dim,
                                    int num_heads,
                                    int kdim,
                                    int vdim,
                                    float dropout,
                                    bool bias,
                                    bool add_bias_kv,
                                    bool add_zero_attn,
                                    Initializer* kernel_initializer,
                                    const char* name)
{
  if (kernel_initializer == NULL) {
    int seed = std::rand();
    kernel_initializer = new GlorotUniform(seed);
  }
  //if (bias_initializer == NULL) {
  //  bias_initializer = new ZeroInitializer();
  //}
#ifdef FF_USE_NCCL
  ParameterSyncType comm_type = ParameterSyncType::NCCL;
#else
  ParameterSyncType comm_type = ParameterSyncType::PS;
#endif
  Tensor kernel;
  {
    // Compute weight size
    int qSize = query->dims[0].size;
    int kSize = key->dims[0].size;
    int vSize = value->dims[0].size;
    int qProjSize = kdim;
    int kProjSize = kdim;
    int vProjSize = vdim;
    int oProjSize = embed_dim;
    int qParas = qProjSize * qSize;
    int kParas = kProjSize * kSize;
    int vParas = vProjSize * vSize;
    int oParas = oProjSize * (vProjSize > 0 ? vProjSize : vSize);
    const int dims[2] = {num_heads, qParas + kParas + vParas + oParas};
    kernel = create_weight<2>(dims, DT_FLOAT, NULL/*owner_op*/,
        true/*create_grad*/, kernel_initializer, comm_type);
  }
  MultiHeadAttention* attn = new MultiHeadAttention(*this, query, key, value,
      kernel, embed_dim, num_heads, kdim, vdim, dropout, bias,
      add_bias_kv, add_zero_attn, name);
  layers.push_back(attn);
  return attn->outputs[0];
}

MultiHeadAttention::MultiHeadAttention(
    FFModel& model,
    const Tensor _query,
    const Tensor _key,
    const Tensor _value,
    const Tensor _weight,
    int _embed_dim, int _num_heads,
    int _kdim, int _vdim,
    float _dropout, bool _bias,
    bool _add_bias_kv, bool _add_zero_attn,
    const char* name)
//    Initializer* _bias_initializer)
: Op(model,
     OP_MULTIHEAD_ATTENTION,
     name, 3/*inputs*/, 1/*weights*/,
     _query, _key, _value, _weight),
  dropout(_dropout), bias(_bias),
  add_bias_kv(_add_bias_kv), add_zero_attn(_add_zero_attn),
  qSize(_query->dims[0].size), kSize(_key->dims[0].size), vSize(_value->dims[0].size),
  qProjSize(_kdim), kProjSize(_kdim), vProjSize(_vdim), oProjSize(_embed_dim),
  qoSeqLength(_query->dims[1].size), kvSeqLength(_key->dims[1].size)
  //bias_initializer(_bias_initializer)
{
  // assert key and value have the same sequence length
  assert(_key->dims[1] == _value->dims[1]);
  numOutputs = 1;
  ParallelDim dims[MAX_TENSOR_DIM];
  for (int i = 0; i < _query->num_dims; i++)
    dims[i] = _query->dims[_query->num_dims-1-i];
  dims[_query->num_dims-1].size = _embed_dim;
  outputs[0] = model.create_tensor(_query->num_dims, dims, DT_FLOAT, this);
}

#ifdef DEADCODE
void MultiHeadAttention::create_weights(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = model.get_or_create_task_is(3, pcname);
#ifdef FF_USE_NCCL
  ParameterSyncType comm_type = ParameterSyncType::NCCL;
#else
  ParameterSyncType comm_type = ParameterSyncType::PS;
#endif
  {
    const int dims[2] = {weights[0].dims[1].size, weights[0].dims[0].size};
    weights[0] = model.create_linear_weight<2, 3>(this, dims, DT_FLOAT,
        kernel_initializer, true/*create_grad*/, comm_type);
  }
}
#endif

#ifdef DEADCODE
void MultiHeadAttention::create_input_partition(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = model.get_or_create_task_is(3, pcname);

  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<3> part_rect = runtime->get_index_space_domain(ctx, task_is);
  int num_part_n = part_rect.hi[2] - part_rect.lo[2] + 1;
  int num_part_v = part_rect.hi[1] - part_rect.lo[1] + 1;
  int num_part_c = part_rect.hi[0] - part_rect.lo[0] + 1;
  // Currently assume only partition over the batch dim
  assert(num_part_v == 1);
  assert(num_part_c == 1);
  return Op::create_input_partition(model);
  //{
  //  const int dims[3] = {outputs[0].dims[2].size, outputs[0].dims[1].size, outputs[0].dims[0].size};
  //  outputs[0] = model.create_tensor<3>(dims, DT_FLOAT, this);
  //  outputs[0].owner_op = this;
  //  outputs[0].owner_idx = 0;
  //}
  //for (int i = 0; i < 3; i++) {
  //  Rect<3> input_rect = runtime->get_index_partition_color_space(
  //      ctx, inputs[i]->part.get_index_partition());
  //  if (input_rect == part_rect) {
  //    input_lps[i] = inputs[i]->part;
  //    input_grad_lps[i] = inputs[i]->part_grad;
  //  } else {
  //    model.create_disjoint_partition(
  //        inputs[i], (IndexSpaceT<3>)task_is, input_lps[i], input_grad_lps[i]);
  //  }
  //}
}
#endif

/*
  regions[0](I): query
  regions[1](I): key
  regions[2](I): value
  regions[3](I): weight
  regions[4](O): output
*/
OpMeta* MultiHeadAttention::init_task(
    const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime* runtime)
{
  const MultiHeadAttention* attn = (MultiHeadAttention*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  TensorAccessorR<float, 3> acc_query(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 3> acc_key(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 3> acc_value(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 2> acc_weight(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 3> acc_output(
      regions[4], task->regions[4], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  int num_samples = acc_query.rect.hi[2] - acc_query.rect.lo[2] + 1;
  assert(attn->qoSeqLength == acc_query.rect.hi[1] - acc_query.rect.lo[1] + 1);
  assert(attn->qSize == acc_query.rect.hi[0]-acc_query.rect.lo[0]+1);
  assert(num_samples == acc_key.rect.hi[2]-acc_key.rect.lo[2]+1);
  assert(attn->kvSeqLength == acc_key.rect.hi[1]-acc_key.rect.lo[1]+1);
  assert(attn->kSize == acc_key.rect.hi[0]-acc_key.rect.lo[0]+1);
  assert(num_samples == acc_value.rect.hi[2]-acc_value.rect.lo[2]+1);
  assert(attn->kvSeqLength == acc_value.rect.hi[1]-acc_value.rect.lo[1]+1);
  assert(attn->vSize == acc_value.rect.hi[0]-acc_value.rect.lo[0]+1);
  int num_heads = acc_weight.rect.hi[1]-acc_weight.rect.lo[1]+1;
  assert(num_samples == acc_output.rect.hi[2]-acc_output.rect.lo[2]+1);
  assert(attn->qoSeqLength == acc_output.rect.hi[1]-acc_output.rect.lo[1]+1);
  assert(attn->oProjSize == acc_output.rect.hi[0]-acc_output.rect.lo[0]+1);

  Memory gpu_mem = Machine::MemoryQuery(Machine::get_machine())
         .only_kind(Memory::GPU_FB_MEM).best_affinity_to(task->target_proc).first();
  MultiHeadAttentionMeta* m = new MultiHeadAttentionMeta(handle,
      attn, gpu_mem, num_samples, num_heads);
  m->profiling = attn->profiling;
  assert(acc_weight.rect.volume() * sizeof(float) == m->weightSize);
  return m;
}

void MultiHeadAttention::init(const FFModel& ff)
{
  assert(check_output_input_weight_same_parallel_is());
  parallel_is = outputs[0]->parallel_is;
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_init(ff, argmap);
  IndexLauncher launcher(ATTENTION_INIT_TASK_ID, parallel_is,
      TaskArgument(this, sizeof(MultiHeadAttention)), argmap,
      Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
      outputs[0]->machine_view.hash());
  launcher.add_region_requirement(
      RegionRequirement(inputs[0]->part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[0]->region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(inputs[1]->part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[1]->region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(inputs[2]->part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[2]->region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0]->part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, weights[0]->region));
  launcher.add_field(3, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part, 0/*projection id*/,
          WRITE_ONLY, EXCLUSIVE, outputs[0]->region));
  launcher.add_field(4, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  set_opmeta_from_futuremap(ff, fm);
}

/*static*/
void MultiHeadAttention::forward_kernel(
    const MultiHeadAttentionMeta* m,
    const float* query_ptr,
    const float* key_ptr,
    const float* value_ptr,
    const float* weight_ptr,
    float* output_ptr)
{
  checkCUDNN(cudnnMultiHeadAttnForward(m->handle.dnn,
      m->attnDesc, -1, m->loWinIdx, m->hiWinIdx,
      m->devQoSeqArray, m->devKvSeqArray, m->qDesc,
      query_ptr, NULL/*residual*/, m->kDesc, key_ptr,
      m->vDesc, value_ptr, m->oDesc, output_ptr, m->weightSize,
      weight_ptr, m->handle.workSpaceSize, m->handle.workSpace,
      m->reserveSpaceSize, m->reserveSpace));
}

/*
  regions[0](I): query
  regions[1](I): key
  regions[2](I): value
  regions[3](I): weight
  regions[4](O): output
*/
__host__
void MultiHeadAttention::forward_task(
    const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime* runtime)
{
  assert(regions.size() == 5);
  assert(task->regions.size() == regions.size());
  //const MultiHeadAttention* attn = (MultiHeadAttention*) task->args;
  const MultiHeadAttentionMeta* m = *((MultiHeadAttentionMeta**) task->local_args);
  TensorAccessorR<float, 3> acc_query(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 3> acc_key(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 3> acc_value(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 2> acc_weight(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 3> acc_output(
      regions[4], task->regions[4], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  MultiHeadAttention::forward_kernel(m,
      acc_query.ptr, acc_key.ptr, acc_value.ptr,
      acc_weight.ptr, acc_output.ptr);
  if (m->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("MultiHeadAttention forward time = %.2fms\n", elapsed);
    //print_tensor<3, float>(acc_query.ptr, acc_query.rect, "[Attention:forward:query]");
    //print_tensor<3, float>(acc_output.ptr, acc_output.rect, "[Attention:forward:output]");
  }
}

void MultiHeadAttention::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_forward(ff, argmap);
  IndexLauncher launcher(ATTENTION_FWD_TASK_ID, parallel_is,
      TaskArgument(NULL, 0), argmap,
      Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
      outputs[0]->machine_view.hash());
  launcher.add_region_requirement(
      RegionRequirement(inputs[0]->part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[0]->region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(inputs[1]->part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[1]->region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(inputs[2]->part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[2]->region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0]->part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, weights[0]->region));
  launcher.add_field(3, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part, 0/*projection id*/,
          WRITE_ONLY, EXCLUSIVE, outputs[0]->region));
  launcher.add_field(4, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

/*static*/
void MultiHeadAttention::backward_kernel(
    const MultiHeadAttentionMeta* m,
    const float* query_ptr,
    float* query_grad_ptr,
    const float* key_ptr,
    float* key_grad_ptr,
    const float* value_ptr,
    float* value_grad_ptr,
    const float* weight_ptr,
    float* weight_grad_ptr,
    const float* output_grad_ptr)
{
  checkCUDNN(cudnnMultiHeadAttnBackwardData(m->handle.dnn,
      m->attnDesc, m->loWinIdx, m->hiWinIdx, m->devQoSeqArray,
      m->devKvSeqArray, m->oDesc, output_grad_ptr, m->qDesc,
      query_grad_ptr, query_ptr, m->kDesc, key_grad_ptr, key_ptr,
      m->vDesc, value_grad_ptr, value_ptr, m->weightSize, weight_ptr,
      m->handle.workSpaceSize, m->handle.workSpace, m->reserveSpaceSize,
      m->reserveSpace));
  checkCUDNN(cudnnMultiHeadAttnBackwardWeights(m->handle.dnn,
      m->attnDesc, CUDNN_WGRAD_MODE_ADD, m->qDesc,
      query_ptr, m->kDesc, key_ptr, m->vDesc, value_ptr, m->oDesc,
      output_grad_ptr, m->weightSize, weight_ptr, weight_grad_ptr,
      m->handle.workSpaceSize, m->handle.workSpace,
      m->reserveSpaceSize, m->reserveSpace));
}

/*
  regions[0](I): query
  regions[1](I): key
  regions[2](I): value
  regions[3](I): weight
  regions[4](I): output_grad
  regions[5](I/O): weight_grad
  regions[6](I/O): query_grad
  regions[7](I/O) (optional): key_grad
  regions[8](I/O) (optional): value_grad
*/
__host__
void MultiHeadAttention::backward_task(
    const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime* runtime)
{
  assert(regions.size() >= 7);
  assert(task->regions.size() == regions.size());
  //MultiHeadAttention* attn = (MultiHeadAttention*) task->args;
  const MultiHeadAttentionMeta* m = *((MultiHeadAttentionMeta**) task->local_args);
  TensorAccessorR<float, 3> acc_query(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 3> acc_key(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 3> acc_value(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 2> acc_weight(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 3> acc_output_grad(
      regions[4], task->regions[4], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> acc_weight_grad(
      regions[5], task->regions[5], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorW<float, 3> acc_query_grad(
      regions[6], task->regions[6], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  float *key_grad_ptr, *value_grad_ptr;
  assert(acc_query_grad.rect == acc_query.rect);
  assert(acc_weight_grad.rect.volume() == acc_weight.rect.volume());
  if (regions.size() == 7) {
    // assert query == key and query == value
    assert(regions[0].get_logical_region() == regions[1].get_logical_region());
    assert(regions[0].get_logical_region() == regions[2].get_logical_region());
    key_grad_ptr = acc_query_grad.ptr;
    value_grad_ptr = acc_query_grad.ptr;
  } else if (regions.size() == 8) {
    // assert query == key
    assert(regions[0].get_logical_region() == regions[1].get_logical_region());
    TensorAccessorW<float, 3> acc_value_grad(
        regions[7], task->regions[7], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    assert(acc_value_grad.rect == acc_value.rect);
    key_grad_ptr = acc_query_grad.ptr;
    value_grad_ptr = acc_value_grad.ptr;
  } else {
    assert(regions.size() == 10);
    TensorAccessorW<float, 3> acc_key_grad(
        regions[7], task->regions[7], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    TensorAccessorW<float, 3> acc_value_grad(
        regions[8], task->regions[8], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    assert(acc_key.rect == acc_key_grad.rect);
    assert(acc_value.rect == acc_value_grad.rect);
    value_grad_ptr = acc_value_grad.ptr;
    key_grad_ptr = acc_key_grad.ptr;
  }
  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }

#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  MultiHeadAttention::backward_kernel(m,
      acc_query.ptr, acc_query_grad.ptr,
      acc_key.ptr, key_grad_ptr, acc_value.ptr, value_grad_ptr,
      acc_weight.ptr, acc_weight_grad.ptr,
      acc_output_grad.ptr);
  if (m->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("MultiHeadAttention backward time = %.2fms\n", elapsed);
  }
}

void MultiHeadAttention::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_backward(ff, argmap);
  IndexLauncher launcher(ATTENTION_BWD_TASK_ID, parallel_is,
      TaskArgument(NULL, 0), argmap,
      Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
      outputs[0]->machine_view.hash());
  launcher.add_region_requirement(
      RegionRequirement(inputs[0]->part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[0]->region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(inputs[1]->part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[1]->region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(inputs[2]->part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[2]->region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0]->part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, weights[0]->region));
  launcher.add_field(3, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part_grad, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, outputs[0]->region_grad));
  launcher.add_field(4, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0]->part_grad, 0/*projection id*/,
          READ_WRITE, EXCLUSIVE, weights[0]->region_grad));
  launcher.add_field(5, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(inputs[0]->part_grad, 0/*projection id*/,
          READ_WRITE, EXCLUSIVE, inputs[0]->region_grad));
  launcher.add_field(6, FID_DATA);
  int num_regions = 7;
  if (inputs[1]->region != inputs[0]->region) {
    // when key != query
    launcher.add_region_requirement(
        RegionRequirement(inputs[1]->part_grad, 0/*projection id*/,
            READ_WRITE, EXCLUSIVE, inputs[1]->region_grad));
    launcher.add_field(num_regions++, FID_DATA);
  }
  if ((inputs[2]->region != inputs[0]->region)
  && (inputs[2]->region != inputs[1]->region)) {
    // when value != key and value != query
    launcher.add_region_requirement(
        RegionRequirement(inputs[2]->part_grad, 0/*projection id*/,
            READ_WRITE, EXCLUSIVE, inputs[2]->region_grad));
    launcher.add_field(num_regions++, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

MultiHeadAttentionMeta::MultiHeadAttentionMeta(FFHandler handler,
                                               const MultiHeadAttention* attn,
                                               Memory gpu_mem,
                                               int num_samples,
                                               int num_heads)
: OpMeta(handler)
{
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(handler.dnn, stream));
#endif
  checkCUDNN(cudnnCreateAttnDescriptor(&attnDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&qDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&kDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&vDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&oDesc));
  // Currently do not support adding bias to key/value projection
  assert(!attn->add_bias_kv);
  cudnnAttnQueryMap_t attnMode = CUDNN_ATTN_QUERYMAP_ALL_TO_ONE;
  // Assume no beam search for now
  int maxBeamSize = 1;
  //printf("batchSize(%d) qSize(%d) kSize(%d) vSize(%d) qProjSize(%d) kProjSize(%d)\n",
  //    num_samples, attn->qSize, attn->kSize, attn->vSize, attn->qProjSize, attn->kProjSize);
  //printf("vProjSize(%d) oProjSize(%d) qoSeqLength(%d) kvSeqLength(%d)\n",
  //    attn->vProjSize, attn->oProjSize, attn->qoSeqLength, attn->kvSeqLength);
  hipdnnMathType_t math_type;
  if (handle.allowTensorOpMathConversion) {
    math_type = CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION;
  } else {
    math_type = HIPDNN_TENSOR_OP_MATH;
  }
  checkCUDNN(cudnnSetAttnDescriptor(attnDesc, attnMode, num_heads,
      1.0f/*smScalar*/, HIPDNN_DATA_FLOAT, HIPDNN_DATA_FLOAT, math_type,
      NULL/*attnDropoutDesc*/, NULL/*postDropoutDesc*/,
      attn->qSize, attn->kSize, attn->vSize, attn->qProjSize, attn->kProjSize,
      attn->vProjSize, attn->oProjSize, attn->qoSeqLength, attn->kvSeqLength,
      num_samples, maxBeamSize));
  size_t workSpaceSize;
  checkCUDNN(cudnnGetMultiHeadAttnBuffers(handler.dnn, attnDesc, &weightSize,
      &workSpaceSize, &reserveSpaceSize));
  assert(workSpaceSize <= handler.workSpaceSize);
  //printf("weightSize(%zu) workSpaceSize(%zu) reserveSpaceSize(%zu)\n", weightSize, workSpaceSize, reserveSpaceSize);
  int dimA[CUDNN_SEQDATA_DIM_COUNT];
  cudnnSeqDataAxis_t axes[CUDNN_SEQDATA_DIM_COUNT];
  assert(CUDNN_SEQDATA_DIM_COUNT == 4);
  axes[3] = CUDNN_SEQDATA_VECT_DIM; // 3 = nbDims-1
  axes[2] = CUDNN_SEQDATA_BEAM_DIM;
  axes[1] = CUDNN_SEQDATA_TIME_DIM;
  axes[0] = CUDNN_SEQDATA_BATCH_DIM;
  int *qoSeqArray = (int*) malloc(sizeof(int) * num_samples);
  int *kvSeqArray = (int*) malloc(sizeof(int) * num_samples);
  for (int i = 0; i < num_samples; i++) {
    qoSeqArray[i] = attn->qoSeqLength;
    kvSeqArray[i] = attn->kvSeqLength;
  }
  // Set qDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = attn->qoSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = attn->qSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(qDesc,
        HIPDNN_DATA_FLOAT, CUDNN_SEQDATA_DIM_COUNT, dimA, axes,
        num_samples, qoSeqArray, NULL));
  }
  // Set kDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = attn->kvSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = attn->kSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(kDesc,
        HIPDNN_DATA_FLOAT, CUDNN_SEQDATA_DIM_COUNT, dimA, axes,
        num_samples, kvSeqArray, NULL));
  }
  // Set vDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = attn->kvSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = attn->vSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(vDesc,
        HIPDNN_DATA_FLOAT, CUDNN_SEQDATA_DIM_COUNT, dimA, axes,
        num_samples, kvSeqArray, NULL));
  }
  // Set oDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = attn->qoSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = attn->oProjSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(oDesc,
        HIPDNN_DATA_FLOAT, CUDNN_SEQDATA_DIM_COUNT, dimA, axes,
        num_samples, qoSeqArray, NULL));
  }
  // allocate memory for the seqArray and reserve space
  {
    size_t totalSize = reserveSpaceSize + sizeof(int) * num_samples * 2;
    Realm::Rect<1, coord_t> bounds(Realm::Point<1, coord_t>(0), Realm::Point<1, coord_t>(totalSize-1));
    std::vector<size_t> field_sizes;
    field_sizes.push_back(sizeof(char));
    Realm::RegionInstance::create_instance(reserveInst, gpu_mem, bounds,
        field_sizes, 0, Realm::ProfilingRequestSet()).wait();
    devQoSeqArray = (int*) reserveInst.pointer_untyped(0, sizeof(char));
    checkCUDA(hipMemcpy(devQoSeqArray, qoSeqArray, sizeof(int) * num_samples,
        hipMemcpyHostToDevice));
    devKvSeqArray = (int*)devQoSeqArray + num_samples;
    checkCUDA(hipMemcpy(devKvSeqArray, kvSeqArray, sizeof(int) * num_samples,
        hipMemcpyHostToDevice));
    reserveSpace = (int*)devKvSeqArray + num_samples;
  }
  // allocate memory for loWinIdx/hiWinIdx
  loWinIdx = (int*) malloc(sizeof(int) * attn->qoSeqLength);
  hiWinIdx = (int*) malloc(sizeof(int) * attn->qoSeqLength);
  for (int i = 0; i < attn->qoSeqLength; i++) {
    loWinIdx[i] = 0;
    hiWinIdx[i] = attn->kvSeqLength;
  }
  free(qoSeqArray);
  free(kvSeqArray);
}

MultiHeadAttentionMeta::~MultiHeadAttentionMeta(void)
{
  reserveInst.destroy();
  free(loWinIdx);
  free(hiWinIdx);
  checkCUDNN(cudnnDestroyAttnDescriptor(attnDesc));
  checkCUDNN(cudnnDestroySeqDataDescriptor(qDesc));
  checkCUDNN(cudnnDestroySeqDataDescriptor(kDesc));
  checkCUDNN(cudnnDestroySeqDataDescriptor(vDesc));
  checkCUDNN(cudnnDestroySeqDataDescriptor(oDesc));
}

bool MultiHeadAttention::measure_operator_cost(Simulator* sim,
                                               const ParallelConfig& pc,
                                               CostMetrics& cost_metrics) const
{
  TensorBase sub_output, sub_query, sub_key, sub_value;
  if (!inputs[0]->get_input_sub_tensor(pc, sub_query, OP_MULTIHEAD_ATTENTION))
    return false;
  if (!inputs[1]->get_input_sub_tensor(pc, sub_key, OP_MULTIHEAD_ATTENTION))
    return false;
  if (!inputs[2]->get_input_sub_tensor(pc, sub_value, OP_MULTIHEAD_ATTENTION))
    return false;
  if (!outputs[0]->get_input_sub_tensor(pc, sub_output, OP_MULTIHEAD_ATTENTION))
    return false;
  // Currently assume only data parallel
  Tensor sub_weight = weights[0];
  assert(sub_weight->num_dims == 2);
  int num_heads = sub_weight->dims[1].size;
  assert(sub_query.num_dims == 3);
  int num_samples = sub_query.dims[2].size;
  MultiHeadAttentionMeta* m = new MultiHeadAttentionMeta(sim->handler,
      this, sim->memory, num_samples, num_heads);

  // allocate tensors in simulator
  sim->free_all();
  const float* query_ptr =
      (const float*)sim->allocate(sub_query.get_volume(), DT_FLOAT);
  const float* key_ptr =
      (const float*)sim->allocate(sub_key.get_volume(), DT_FLOAT);
  const float* value_ptr =
      (const float*)sim->allocate(sub_value.get_volume(), DT_FLOAT);
  const float* weight_ptr =
      (const float*)sim->allocate(sub_weight->get_volume(), DT_FLOAT);
  float* output_ptr =
      (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert(output_ptr != NULL);

  std::function<void()> forward, backward;
  forward = [&] {
    forward_kernel(m, query_ptr, key_ptr, value_ptr, weight_ptr, output_ptr);
  };
  if (sim->computationMode == COMP_MODE_TRAINING) {
    float* query_grad_ptr =
        (float*)sim->allocate(sub_query.get_volume(), DT_FLOAT);
    float* key_grad_ptr =
        (float*)sim->allocate(sub_key.get_volume(), DT_FLOAT);
    float* value_grad_ptr =
        (float*)sim->allocate(sub_value.get_volume(), DT_FLOAT);
    float* weight_grad_ptr =
        (float*)sim->allocate(sub_weight->get_volume(), DT_FLOAT);
    float* output_grad_ptr =
        (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
    assert(output_grad_ptr != NULL);

    backward = [&] {
      backward_kernel(m, query_ptr, query_grad_ptr, key_ptr, key_grad_ptr,
        value_ptr, value_grad_ptr, weight_ptr, weight_grad_ptr, output_grad_ptr);
    };
  }

  inner_measure_operator_cost(sim, forward, backward, cost_metrics);

  if (sim->computationMode == COMP_MODE_TRAINING) {
    printf("[Measure MultiHeadAttention] query(%d %d %d) key(%d %d %d) value(%d %d %d) output(%d %d %d)"
         "forward_time(%.4lf) backward_time(%.4lf)\n",
         sub_query.dims[2].size, sub_query.dims[1].size, sub_query.dims[0].size,
         sub_key.dims[2].size, sub_key.dims[1].size, sub_key.dims[0].size,
         sub_value.dims[2].size, sub_value.dims[1].size, sub_value.dims[0].size,
         sub_output.dims[2].size, sub_output.dims[1].size, sub_output.dims[0].size,
         cost_metrics.forward_time, cost_metrics.backward_time);
  } else {
    printf("[Measure MultiHeadAttention] query(%d %d %d) key(%d %d %d) value(%d %d %d) output(%d %d %d)"
         "forward_time(%.4lf)\n",
         sub_query.dims[2].size, sub_query.dims[1].size, sub_query.dims[0].size,
         sub_key.dims[2].size, sub_key.dims[1].size, sub_key.dims[0].size,
         sub_value.dims[2].size, sub_value.dims[1].size, sub_value.dims[0].size,
         sub_output.dims[2].size, sub_output.dims[1].size, sub_output.dims[0].size,
         cost_metrics.forward_time);
  }
  // Free multiheadattentionmeta
  delete m;
  return true;
}
