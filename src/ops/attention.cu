/* Copyright 2021 Facebook
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::multihead_attention(const Tensor& query,
                                    const Tensor& key,
                                    const Tensor& value,
                                    int embed_dim,
                                    int num_heads,
                                    int kdim,
                                    int vdim,
                                    float dropout,
                                    bool bias,
                                    bool add_bias_kv,
                                    bool add_zero_attn,
                                    Initializer* kernel_initializer,
                                    const char* name)
{
  if (kernel_initializer == NULL) {
    int seed = std::rand();
    kernel_initializer = new GlorotUniform(seed);
  }
  //if (bias_initializer == NULL) {
  //  bias_initializer = new ZeroInitializer();
  //}
  MultiHeadAttention* attn = new MultiHeadAttention(*this, query, key, value,
      embed_dim, num_heads, kdim, vdim, dropout, bias, add_bias_kv, add_zero_attn,
      kernel_initializer/*, bias_initializer*/, name);
  layers.push_back(attn);
  return attn->outputs[0];
}

MultiHeadAttention::MultiHeadAttention(FFModel& model,
                                       const Tensor& _query,
                                       const Tensor& _key,
                                       const Tensor& _value,
                                       int _embed_dim, int _num_heads,
                                       int _kdim, int _vdim,
                                       float _dropout, bool _bias,
                                       bool _add_bias_kv, bool _add_zero_attn,
                                       Initializer* _kernel_initializer,
                                       const char* name)
//                                       Initializer* _bias_initializer)
: Op(model,
     OP_MULTIHEAD_ATTENTION,
     name,
     _query, _key, _value),
  dropout(_dropout), bias(_bias),
  add_bias_kv(_add_bias_kv), add_zero_attn(_add_zero_attn),
  qSize(_query.adim[0]), kSize(_key.adim[0]), vSize(_value.adim[0]),
  qProjSize(_kdim), kProjSize(_kdim), vProjSize(_vdim), oProjSize(_embed_dim),
  qoSeqLength(_query.adim[1]), kvSeqLength(_key.adim[1]),
  kernel_initializer(_kernel_initializer),
  profiling(model.config.profiling)
  //bias_initializer(_bias_initializer)
{
  // assert key and value have the same sequence length
  assert(_key.adim[1] == _value.adim[1]);
  numOutputs = 1;
  outputs[0].numDim = _query.numDim;
  for (int i = 1; i < _query.numDim; i++)
    outputs[0].adim[i] = _query.adim[i];
  outputs[0].adim[0] = _embed_dim;
  numWeights = 1;
  weights[0].numDim = 2;
  // Compute weight size
  int qParas = qProjSize * qSize;
  int kParas = kProjSize * kSize;
  int vParas = vProjSize * vSize;
  int oParas = oProjSize * (vProjSize > 0 ? vProjSize : vSize);
  weights[0].adim[0] = qParas + kParas + vParas + oParas;
  weights[0].adim[1] = _num_heads;
}

void MultiHeadAttention::create_weights(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = model.get_or_create_task_is(3, pcname);
#ifdef FF_ENABLE_NCCL
  Parameter::CommType comm_type = Parameter::NCCL;
#else
  Parameter::CommType comm_type = Parameter::PS;
#endif
  {
    const int dims[2] = {weights[0].adim[1], weights[0].adim[0]};
    weights[0] = model.create_linear_weight<2, 3>(this, dims, DT_FLOAT,
        kernel_initializer, true/*create_grad*/, comm_type);
  }
}

void MultiHeadAttention::create_output_and_partition(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = model.get_or_create_task_is(3, pcname);

  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<3> part_rect = runtime->get_index_space_domain(ctx, task_is);
  int num_part_n = part_rect.hi[2] - part_rect.lo[2] + 1;
  int num_part_v = part_rect.hi[1] - part_rect.lo[1] + 1;
  int num_part_c = part_rect.hi[0] - part_rect.lo[0] + 1;
  // Currently assume only partition over the batch dim
  assert(num_part_v == 1);
  assert(num_part_c == 1);
  {
    const int dims[3] = {outputs[0].adim[2], outputs[0].adim[1], outputs[0].adim[0]};
    outputs[0] = model.create_tensor<3>(dims, DT_FLOAT, this);
    outputs[0].owner_op = this;
    outputs[0].owner_idx = 0;
  }
  for (int i = 0; i < 3; i++) {
    Rect<3> input_rect = runtime->get_index_partition_color_space(
        ctx, inputs[i].part.get_index_partition());
    if (input_rect == part_rect) {
      input_lps[i] = inputs[i].part;
      input_grad_lps[i] = inputs[i].part_grad;
    } else {
      model.create_disjoint_partition(
          inputs[i], (IndexSpaceT<3>)task_is, input_lps[i], input_grad_lps[i]);
    }
  }
}

/*
  regions[0](I): query
  regions[1](I): key
  regions[2](I): value
  regions[3](I): weight
  regions[4](O): output
*/
OpMeta* MultiHeadAttention::init_task(
    const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime* runtime)
{
  const MultiHeadAttention* attn = (MultiHeadAttention*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  TensorAccessorR<float, 3> acc_query(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 3> acc_key(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 3> acc_value(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 2> acc_weight(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 3> acc_output(
      regions[4], task->regions[4], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  int num_samples = acc_query.rect.hi[2] - acc_query.rect.lo[2] + 1;
  assert(attn->qoSeqLength == acc_query.rect.hi[1] - acc_query.rect.lo[1] + 1);
  assert(attn->qSize == acc_query.rect.hi[0]-acc_query.rect.lo[0]+1);
  assert(num_samples == acc_key.rect.hi[2]-acc_key.rect.lo[2]+1);
  assert(attn->kvSeqLength == acc_key.rect.hi[1]-acc_key.rect.lo[1]+1);
  assert(attn->kSize == acc_key.rect.hi[0]-acc_key.rect.lo[0]+1);
  assert(num_samples == acc_value.rect.hi[2]-acc_value.rect.lo[2]+1);
  assert(attn->kvSeqLength == acc_value.rect.hi[1]-acc_value.rect.lo[1]+1);
  assert(attn->vSize == acc_value.rect.hi[0]-acc_value.rect.lo[0]+1);
  int num_heads = acc_weight.rect.hi[1]-acc_weight.rect.lo[1]+1;
  assert(num_samples == acc_output.rect.hi[2]-acc_output.rect.lo[2]+1);
  assert(attn->qoSeqLength == acc_output.rect.hi[1]-acc_output.rect.lo[1]+1);
  assert(attn->oProjSize == acc_output.rect.hi[0]-acc_output.rect.lo[0]+1);

  Memory gpu_mem = Machine::MemoryQuery(Machine::get_machine())
         .only_kind(Memory::GPU_FB_MEM).best_affinity_to(task->target_proc).first();
  MultiHeadAttentionMeta* m = new MultiHeadAttentionMeta(handle,
      attn, gpu_mem, num_samples, num_heads);
  assert(acc_weight.rect.volume() * sizeof(float) == m->weightSize);
  return m;
}

void MultiHeadAttention::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<3> rect = runtime->get_index_space_domain(ctx, task_is);
  ParallelConfig pc;
  std::string pcname = name;
  ff.config.find_parallel_config(3, pcname, pc);
  int idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    FFHandler handle = ff.handlers[pc.device_ids[idx++]];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
  }
  IndexLauncher launcher(ATTENTION_INIT_TASK_ID, task_is,
      TaskArgument(this, sizeof(MultiHeadAttention)), argmap,
      Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
      FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(input_lps[1], 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[1].region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(input_lps[2], 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[2].region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, weights[0].region));
  launcher.add_field(3, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
          WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(4, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

void MultiHeadAttention::forward_kernel(
    const MultiHeadAttentionMeta* m,
    const float* query_ptr,
    const float* key_ptr,
    const float* value_ptr,
    const float* weight_ptr,
    float* output_ptr) const
{
  checkCUDNN(cudnnMultiHeadAttnForward(m->handle.dnn,
      m->attnDesc, -1, m->loWinIdx, m->hiWinIdx,
      m->devQoSeqArray, m->devKvSeqArray, m->qDesc,
      query_ptr, NULL/*residual*/, m->kDesc, key_ptr,
      m->vDesc, value_ptr, m->oDesc, output_ptr, m->weightSize,
      weight_ptr, m->handle.workSpaceSize, m->handle.workSpace,
      m->reserveSpaceSize, m->reserveSpace));
}

/*
  regions[0](I): query
  regions[1](I): key
  regions[2](I): value
  regions[3](I): weight
  regions[4](O): output
*/
__host__
void MultiHeadAttention::forward_task(
    const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime* runtime)
{
  assert(regions.size() == 5);
  assert(task->regions.size() == regions.size());
  const MultiHeadAttention* attn = (MultiHeadAttention*) task->args;
  const MultiHeadAttentionMeta* m = *((MultiHeadAttentionMeta**) task->local_args);
  TensorAccessorR<float, 3> acc_query(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 3> acc_key(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 3> acc_value(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 2> acc_weight(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 3> acc_output(
      regions[4], task->regions[4], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  hipEvent_t t_start, t_end;
  if (attn->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  attn->forward_kernel(m, acc_query.ptr, acc_key.ptr, acc_value.ptr,
      acc_weight.ptr, acc_output.ptr);
  if (attn->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("MultiHeadAttention forward time = %.2fms\n", elapsed);
    //print_tensor<3, float>(acc_query.ptr, acc_query.rect, "[Attention:forward:query]");
    //print_tensor<3, float>(acc_output.ptr, acc_output.rect, "[Attention:forward:output]");
  }
}

void MultiHeadAttention::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<3> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(ATTENTION_FWD_TASK_ID, task_is,
      TaskArgument(this, sizeof(MultiHeadAttention)), argmap,
      Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
      FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(input_lps[1], 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[1].region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(input_lps[2], 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[2].region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, weights[0].region));
  launcher.add_field(3, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
          WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(4, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

void MultiHeadAttention::backward_kernel(
    const MultiHeadAttentionMeta* m,
    const float* query_ptr,
    float* query_grad_ptr,
    const float* key_ptr,
    float* key_grad_ptr,
    const float* value_ptr,
    float* value_grad_ptr,
    const float* weight_ptr,
    float* weight_grad_ptr,
    const float* output_grad_ptr) const
{
  checkCUDNN(cudnnMultiHeadAttnBackwardData(m->handle.dnn,
      m->attnDesc, m->loWinIdx, m->hiWinIdx, m->devQoSeqArray,
      m->devKvSeqArray, m->oDesc, output_grad_ptr, m->qDesc,
      query_grad_ptr, query_ptr, m->kDesc, key_grad_ptr, key_ptr,
      m->vDesc, value_grad_ptr, value_ptr, m->weightSize, weight_ptr,
      m->handle.workSpaceSize, m->handle.workSpace, m->reserveSpaceSize,
      m->reserveSpace));
  checkCUDNN(cudnnMultiHeadAttnBackwardWeights(m->handle.dnn,
      m->attnDesc, CUDNN_WGRAD_MODE_ADD, m->qDesc,
      query_ptr, m->kDesc, key_ptr, m->vDesc, value_ptr, m->oDesc,
      output_grad_ptr, m->weightSize, weight_ptr, weight_grad_ptr,
      m->handle.workSpaceSize, m->handle.workSpace,
      m->reserveSpaceSize, m->reserveSpace));
}

/*
  regions[0](I): query
  regions[1](I): key
  regions[2](I): value
  regions[3](I): weight
  regions[4](I): output_grad
  regions[5](I/O): weight_grad
  regions[6](I/O): query_grad
  regions[7](I/O) (optional): key_grad
  regions[8](I/O) (optional): value_grad
*/
__host__
void MultiHeadAttention::backward_task(
    const Task *task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime* runtime)
{
  assert(regions.size() >= 7);
  assert(task->regions.size() == regions.size());
  MultiHeadAttention* attn = (MultiHeadAttention*) task->args;
  const MultiHeadAttentionMeta* m = *((MultiHeadAttentionMeta**) task->local_args);
  TensorAccessorR<float, 3> acc_query(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 3> acc_key(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 3> acc_value(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 2> acc_weight(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 3> acc_output_grad(
      regions[4], task->regions[4], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> acc_weight_grad(
      regions[5], task->regions[5], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorW<float, 3> acc_query_grad(
      regions[6], task->regions[6], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  float *key_grad_ptr, *value_grad_ptr;
  assert(acc_query_grad.rect == acc_query.rect);
  assert(acc_weight_grad.rect == acc_weight.rect);
  if (regions.size() == 7) {
    // assert query == key and query == value
    assert(regions[0].get_logical_region() == regions[1].get_logical_region());
    assert(regions[0].get_logical_region() == regions[2].get_logical_region());
    key_grad_ptr = acc_query_grad.ptr;
    value_grad_ptr = acc_query_grad.ptr;
  } else if (regions.size() == 8) {
    // assert query == key
    assert(regions[0].get_logical_region() == regions[2].get_logical_region());
    key_grad_ptr = acc_query_grad.ptr;
    TensorAccessorW<float, 3> acc_value_grad(
        regions[7], task->regions[7], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    assert(acc_value_grad.rect == acc_value.rect);
    key_grad_ptr = acc_query_grad.ptr;
    value_grad_ptr = acc_value_grad.ptr;
  } else {
    assert(regions.size() == 10);
    TensorAccessorW<float, 3> acc_key_grad(
        regions[7], task->regions[7], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    TensorAccessorW<float, 3> acc_value_grad(
        regions[8], task->regions[8], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    assert(acc_key.rect == acc_key_grad.rect);
    assert(acc_value.rect == acc_value_grad.rect);
    value_grad_ptr = acc_value_grad.ptr;
    key_grad_ptr = acc_key_grad.ptr;
  }
  hipEvent_t t_start, t_end;
  if (attn->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }

#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  attn->backward_kernel(m, acc_query.ptr, acc_query_grad.ptr,
      acc_key.ptr, key_grad_ptr, acc_value.ptr, value_grad_ptr,
      acc_weight.ptr, acc_weight_grad.ptr,
      acc_output_grad.ptr);
  if (attn->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("MultiHeadAttention backward time = %.2fms\n", elapsed);
  }
}

void MultiHeadAttention::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<3> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(ATTENTION_BWD_TASK_ID, task_is,
      TaskArgument(this, sizeof(MultiHeadAttention)), argmap,
      Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
      FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(input_lps[1], 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[1].region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(input_lps[2], 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, inputs[2].region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, weights[0].region));
  launcher.add_field(3, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
          READ_ONLY, EXCLUSIVE, outputs[0].region_grad));
  launcher.add_field(4, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part_grad, 0/*projection id*/,
          READ_WRITE, EXCLUSIVE, weights[0].region_grad));
  launcher.add_field(5, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(input_grad_lps[0], 0/*projection id*/,
          READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(6, FID_DATA);
  int num_regions = 7;
  if (inputs[1].region != inputs[0].region) {
    // when key != query
    launcher.add_region_requirement(
        RegionRequirement(input_grad_lps[1], 0/*projection id*/,
            READ_WRITE, EXCLUSIVE, inputs[1].region_grad));
    launcher.add_field(num_regions++, FID_DATA);
  }
  if ((inputs[2].region != inputs[0].region)
  && (inputs[2].region != inputs[1].region)) {
    // when value != key and value != query
    launcher.add_region_requirement(
        RegionRequirement(input_grad_lps[2], 0/*projection id*/,
            READ_WRITE, EXCLUSIVE, inputs[2].region_grad));
    launcher.add_field(num_regions++, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

MultiHeadAttentionMeta::MultiHeadAttentionMeta(FFHandler handler,
                                               const MultiHeadAttention* attn,
                                               Memory gpu_mem,
                                               int num_samples,
                                               int num_heads)
: OpMeta(handler)
{
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(handler.dnn, stream));
#endif
  checkCUDNN(cudnnCreateAttnDescriptor(&attnDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&qDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&kDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&vDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&oDesc));
  // Currently do not support adding bias to key/value projection
  assert(!attn->add_bias_kv);
  cudnnAttnQueryMap_t attnMode = CUDNN_ATTN_QUERYMAP_ALL_TO_ONE;
  // Assume no beam search for now
  int maxBeamSize = 1;
  //printf("batchSize(%d) qSize(%d) kSize(%d) vSize(%d) qProjSize(%d) kProjSize(%d)\n",
  //    num_samples, attn->qSize, attn->kSize, attn->vSize, attn->qProjSize, attn->kProjSize);
  //printf("vProjSize(%d) oProjSize(%d) qoSeqLength(%d) kvSeqLength(%d)\n",
  //    attn->vProjSize, attn->oProjSize, attn->qoSeqLength, attn->kvSeqLength);
  checkCUDNN(cudnnSetAttnDescriptor(attnDesc, attnMode, num_heads,
      1.0f/*smScalar*/, HIPDNN_DATA_FLOAT, HIPDNN_DATA_FLOAT, HIPDNN_DEFAULT_MATH,
      NULL/*attnDropoutDesc*/, NULL/*postDropoutDesc*/,
      attn->qSize, attn->kSize, attn->vSize, attn->qProjSize, attn->kProjSize,
      attn->vProjSize, attn->oProjSize, attn->qoSeqLength, attn->kvSeqLength,
      num_samples, maxBeamSize));
  size_t workSpaceSize;
  checkCUDNN(cudnnGetMultiHeadAttnBuffers(handler.dnn, attnDesc, &weightSize,
      &workSpaceSize, &reserveSpaceSize));
  assert(workSpaceSize <= handler.workSpaceSize);
  //printf("weightSize(%zu) workSpaceSize(%zu) reserveSpaceSize(%zu)\n", weightSize, workSpaceSize, reserveSpaceSize);
  int dimA[CUDNN_SEQDATA_DIM_COUNT];
  cudnnSeqDataAxis_t axes[CUDNN_SEQDATA_DIM_COUNT];
  assert(CUDNN_SEQDATA_DIM_COUNT == 4);
  axes[3] = CUDNN_SEQDATA_VECT_DIM; // 3 = nbDims-1
  axes[2] = CUDNN_SEQDATA_BEAM_DIM;
  axes[1] = CUDNN_SEQDATA_TIME_DIM;
  axes[0] = CUDNN_SEQDATA_BATCH_DIM;
  int *qoSeqArray = (int*) malloc(sizeof(int) * num_samples);
  int *kvSeqArray = (int*) malloc(sizeof(int) * num_samples);
  for (int i = 0; i < num_samples; i++) {
    qoSeqArray[i] = attn->qoSeqLength;
    kvSeqArray[i] = attn->kvSeqLength;
  }
  // Set qDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = attn->qoSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = attn->qSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(qDesc,
        HIPDNN_DATA_FLOAT, CUDNN_SEQDATA_DIM_COUNT, dimA, axes,
        num_samples, qoSeqArray, NULL));
  }
  // Set kDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = attn->kvSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = attn->kSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(kDesc,
        HIPDNN_DATA_FLOAT, CUDNN_SEQDATA_DIM_COUNT, dimA, axes,
        num_samples, kvSeqArray, NULL));
  }
  // Set vDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = attn->kvSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = attn->vSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(vDesc,
        HIPDNN_DATA_FLOAT, CUDNN_SEQDATA_DIM_COUNT, dimA, axes,
        num_samples, kvSeqArray, NULL));
  }
  // Set oDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = attn->qoSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = attn->oProjSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(oDesc,
        HIPDNN_DATA_FLOAT, CUDNN_SEQDATA_DIM_COUNT, dimA, axes,
        num_samples, qoSeqArray, NULL));
  }
  // allocate memory for the seqArray and reserve space
  {
    size_t totalSize = reserveSpaceSize + sizeof(int) * num_samples * 2;
    Realm::Rect<1, coord_t> bounds(Realm::Point<1, coord_t>(0), Realm::Point<1, coord_t>(totalSize-1));
    std::vector<size_t> field_sizes;
    field_sizes.push_back(sizeof(char));
    Realm::RegionInstance::create_instance(reserveInst, gpu_mem, bounds,
        field_sizes, 0, Realm::ProfilingRequestSet()).wait();
    devQoSeqArray = (int*) reserveInst.pointer_untyped(0, sizeof(char));
    checkCUDA(hipMemcpy(devQoSeqArray, qoSeqArray, sizeof(int) * num_samples,
        hipMemcpyHostToDevice));
    devKvSeqArray = (int*)devQoSeqArray + num_samples;
    checkCUDA(hipMemcpy(devKvSeqArray, kvSeqArray, sizeof(int) * num_samples,
        hipMemcpyHostToDevice));
    reserveSpace = (int*)devKvSeqArray + num_samples;
  }
  // allocate memory for loWinIdx/hiWinIdx
  loWinIdx = (int*) malloc(sizeof(int) * attn->qoSeqLength);
  hiWinIdx = (int*) malloc(sizeof(int) * attn->qoSeqLength);
  for (int i = 0; i < attn->qoSeqLength; i++) {
    loWinIdx[i] = 0;
    hiWinIdx[i] = attn->kvSeqLength;
  }
  free(qoSeqArray);
  free(kvSeqArray);
}

MultiHeadAttentionMeta::~MultiHeadAttentionMeta(void)
{
  reserveInst.destroy();
  free(loWinIdx);
  free(hiWinIdx);
  checkCUDNN(cudnnDestroyAttnDescriptor(attnDesc));
  checkCUDNN(cudnnDestroySeqDataDescriptor(qDesc));
  checkCUDNN(cudnnDestroySeqDataDescriptor(kDesc));
  checkCUDNN(cudnnDestroySeqDataDescriptor(vDesc));
  checkCUDNN(cudnnDestroySeqDataDescriptor(oDesc));
}

bool MultiHeadAttention::measure_compute_time(Simulator* sim,
    const ParallelConfig& pc,
    float& forward_time,
    float& backward_time)
{
  Tensor sub_output, sub_query, sub_key, sub_value;
  if (!inputs[0].get_input_sub_tensor(pc, sub_query, OP_MULTIHEAD_ATTENTION))
    return false;
  if (!inputs[1].get_input_sub_tensor(pc, sub_key, OP_MULTIHEAD_ATTENTION))
    return false;
  if (!inputs[2].get_input_sub_tensor(pc, sub_value, OP_MULTIHEAD_ATTENTION))
    return false;
  if (!outputs[0].get_input_sub_tensor(pc, sub_output, OP_MULTIHEAD_ATTENTION))
    return false;
  // Currently assume only data parallel
  Tensor sub_weight = weights[0];
  assert(sub_weight.numDim == 2);
  int num_heads = sub_weight.adim[1];
  assert(sub_query.numDim == 3);
  int num_samples = sub_query.adim[2];
  MultiHeadAttentionMeta* m = new MultiHeadAttentionMeta(sim->handler,
      this, sim->memory, num_samples, num_heads);

  // allocate tensors in simulator
  sim->free_all();
  const float* query_ptr =
      (const float*)sim->allocate(sub_query.get_volume(), DT_FLOAT);
  float* query_grad_ptr =
      (float*)sim->allocate(sub_query.get_volume(), DT_FLOAT);
  const float* key_ptr =
      (const float*)sim->allocate(sub_key.get_volume(), DT_FLOAT);
  float* key_grad_ptr =
      (float*)sim->allocate(sub_key.get_volume(), DT_FLOAT);
  const float* value_ptr =
      (const float*)sim->allocate(sub_value.get_volume(), DT_FLOAT);
  float* value_grad_ptr =
      (float*)sim->allocate(sub_value.get_volume(), DT_FLOAT);
  const float* weight_ptr =
      (const float*)sim->allocate(sub_weight.get_volume(), DT_FLOAT);
  float* weight_grad_ptr =
      (float*)sim->allocate(sub_weight.get_volume(), DT_FLOAT);
  float* output_ptr =
      (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  float* output_grad_ptr =
      (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert(output_grad_ptr != NULL);

  auto forward = [&] {
    forward_kernel(m, query_ptr, key_ptr, value_ptr, weight_ptr, output_ptr);
  };
  auto backward = [&] {
    backward_kernel(m, query_ptr, query_grad_ptr, key_ptr, key_grad_ptr,
        value_ptr, value_grad_ptr, weight_ptr, weight_grad_ptr, output_grad_ptr);
  };

  inner_measure_compute_time(sim, forward, backward, forward_time, backward_time);

  printf("[Measure MultiHeadAttention] query(%d %d %d) key(%d %d %d) value(%d %d %d) output(%d %d %d)"
         "forward_time(%.4lf) backward_time(%.4lf)\n",
         sub_query.adim[2], sub_query.adim[1], sub_query.adim[0],
         sub_key.adim[2], sub_key.adim[1], sub_key.adim[0],
         sub_value.adim[2], sub_value.adim[1], sub_value.adim[0],
         sub_output.adim[2], sub_output.adim[1], sub_output.adim[0],
         forward_time, backward_time);
  // Free multiheadattentionmeta
  delete m;
  return true;
}
