/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/conv_2d.h"
#include "flexflow/simulator.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

hipdnnConvolutionFwdAlgo_t selectConvolutionForwardAlgorithm(
    hipdnnHandle_t handle,
    const hipdnnTensorDescriptor_t xDesc,
    void const *x,
    const hipdnnFilterDescriptor_t wDesc,
    void const *w,
    const hipdnnConvolutionDescriptor_t convDesc,
    void *workSpace,
    size_t workSpaceSize,
    const hipdnnTensorDescriptor_t yDesc,
    void *y);
hipdnnConvolutionBwdFilterAlgo_t selectConvolutionBackwardFilterAlgorithm(
    hipdnnHandle_t handle,
    const hipdnnTensorDescriptor_t xDesc,
    void const *x,
    const hipdnnTensorDescriptor_t dyDesc,
    void const *dy,
    const hipdnnConvolutionDescriptor_t convDesc,
    void *workSpace,
    size_t workSpaceSize,
    const hipdnnFilterDescriptor_t dwDesc,
    void *dw);
hipdnnConvolutionBwdDataAlgo_t selectConvolutionBackwardDataAlgorithm(
    hipdnnHandle_t handle,
    const hipdnnFilterDescriptor_t wDesc,
    void const *w,
    const hipdnnTensorDescriptor_t dyDesc,
    void const *dy,
    const hipdnnConvolutionDescriptor_t convDesc,
    void *workSpace,
    size_t workSpaceSize,
    const hipdnnTensorDescriptor_t dxDesc,
    void *dx);

/*static*/
void Conv2D::init_kernel(Conv2D const *conv,
                         Conv2DMeta *m,
                         int input_w,
                         int input_h,
                         int input_c,
                         int input_n,
                         int output_w,
                         int output_h,
                         int output_c,
                         int output_n,
                         int pad_h,
                         int pad_w,
                         float const *input_ptr,
                         float *output_ptr,
                         float const *kernel_ptr,
                         float *kernel_grad_ptr) {
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        input_n,
                                        input_c,
                                        input_h,
                                        input_w));

  checkCUDNN(hipdnnSetTensor4dDescriptor(
      m->biasTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, output_c, 1, 1));

  // Require that input_c is divisible by conv->groups
  assert(input_c % conv->groups == 0);
  printf("filterDim: kernel(%d %d) c_in(%d), c_out(%d)\n",
         conv->kernel_h,
         conv->kernel_w,
         input_c / conv->groups,
         output_c);
  checkCUDNN(hipdnnSetFilter4dDescriptor(m->filterDesc,
                                        HIPDNN_DATA_FLOAT,
                                        HIPDNN_TENSOR_NCHW,
                                        output_c,
                                        input_c / conv->groups,
                                        conv->kernel_h,
                                        conv->kernel_w));

  checkCUDNN(hipdnnSetConvolution2dDescriptor(m->convDesc,
                                             pad_h, // conv->padding_h,
                                             pad_w, // conv->padding_w,
                                             conv->stride_h,
                                             conv->stride_w,
                                             1 /*upscale_x*/,
                                             1 /*upscale_y*/,
                                             HIPDNN_CROSS_CORRELATION,
                                             HIPDNN_DATA_FLOAT));
  if (conv->groups != 1) {
    checkCUDNN(hipdnnSetConvolutionGroupCount(m->convDesc, conv->groups));
  }

  // enable tensor core when possible
  if (m->handle.allowTensorOpMathConversion) {
    checkCUDNN(hipdnnSetConvolutionMathType(
        m->convDesc, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
  } else {
    checkCUDNN(hipdnnSetConvolutionMathType(m->convDesc, HIPDNN_TENSOR_OP_MATH));
  }

  int n, c, h, w;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(
      m->convDesc, m->inputTensor, m->filterDesc, &n, &c, &h, &w));
  assert(n == output_n);
  assert(c == output_c);
  assert(h == output_h);
  assert(w == output_w);

  checkCUDNN(hipdnnSetTensor4dDescriptor(
      m->outputTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
  // select forward algorithm
  m->fwdAlgo = selectConvolutionForwardAlgorithm(m->handle.dnn,
                                                 m->inputTensor,
                                                 input_ptr,
                                                 m->filterDesc,
                                                 kernel_ptr,
                                                 m->convDesc,
                                                 m->handle.workSpace,
                                                 m->handle.workSpaceSize,
                                                 m->outputTensor,
                                                 output_ptr);
  // select backward filter algorithm
  m->bwdFilterAlgo =
      selectConvolutionBackwardFilterAlgorithm(m->handle.dnn,
                                               m->inputTensor,
                                               input_ptr,
                                               m->outputTensor,
                                               output_ptr,
                                               m->convDesc,
                                               m->handle.workSpace,
                                               m->handle.workSpaceSize,
                                               m->filterDesc,
                                               kernel_grad_ptr);
  // select backward data algorithm
  m->bwdDataAlgo =
      selectConvolutionBackwardDataAlgorithm(m->handle.dnn,
                                             m->filterDesc,
                                             kernel_ptr,
                                             m->outputTensor,
                                             output_ptr,
                                             m->convDesc,
                                             m->handle.workSpace,
                                             m->handle.workSpaceSize,
                                             m->inputTensor,
                                             (float *)input_ptr);
  if (m->relu) {
    checkCUDNN(hipdnnSetActivationDescriptor(
        m->actiDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));
  }
}

/*static*/
void Conv2D::forward_kernel(Conv2DMeta const *m,
                            float const *input_ptr,
                            float *output_ptr,
                            float const *filter_ptr,
                            float const *bias_ptr,
                            hipStream_t stream) {
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha = 1.0f, beta = 0.0f;
  checkCUDNN(hipdnnConvolutionForward(m->handle.dnn,
                                     &alpha,
                                     m->inputTensor,
                                     input_ptr,
                                     m->filterDesc,
                                     filter_ptr,
                                     m->convDesc,
                                     m->fwdAlgo,
                                     m->handle.workSpace,
                                     m->handle.workSpaceSize,
                                     &beta,
                                     m->outputTensor,
                                     output_ptr));

  // use_bias == True
  if (bias_ptr != NULL) {
    checkCUDNN(hipdnnAddTensor(m->handle.dnn,
                              &alpha,
                              m->biasTensor,
                              bias_ptr,
                              &alpha,
                              m->outputTensor,
                              output_ptr));
  }
  if (m->relu) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn,
                                      m->actiDesc,
                                      &alpha,
                                      m->outputTensor,
                                      output_ptr,
                                      &beta,
                                      m->outputTensor,
                                      output_ptr));
  }
}

/*static*/
void Conv2D::forward_kernel_wrapper(Conv2DMeta const *m,
                                    float const *input_ptr,
                                    float *output_ptr,
                                    float const *filter_ptr,
                                    float const *bias_ptr) {
  // printf("fwdAlgo(%d), bwdFilterALgo(%d), bwdDataAlgo(%d)\n",
  // (int)m->fwdAlgo,(int) m->bwdFilterAlgo,(int) m->bwdDataAlgo);
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  Conv2D::forward_kernel(
      m, input_ptr, output_ptr, filter_ptr, bias_ptr, stream);
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    print_tensor<float>(input_ptr, 16, "[Conv2D:forward:input]");
    print_tensor<float>(filter_ptr, 16, "[Conv2D:forward:kernel]");
    print_tensor<float>(bias_ptr, 16, "[Conv2D:forward:bias]");
    print_tensor<float>(output_ptr, 16, "[Conv2D:forward:output]");
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Conv2D] forward time (CF) = %.2fms\n", m->op_name, elapsed);
  }
}

/*static*/
void Conv2D::backward_kernel(Conv2DMeta const *m,
                             float const *input_ptr,
                             float *input_grad_ptr,
                             float const *output_ptr,
                             float *output_grad_ptr,
                             float const *kernel_ptr,
                             float *kernel_grad_ptr,
                             float *bias_grad_ptr,
                             hipStream_t stream) {
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha = 1.0f;
  // float beta = 0.0f;
  if (m->relu) {
    hipdnnDataType_t dataType;
    int n, c, h, w, nStride, cStride, hStride, wStride;
    checkCUDNN(hipdnnGetTensor4dDescriptor(m->outputTensor,
                                          &dataType,
                                          &n,
                                          &c,
                                          &h,
                                          &w,
                                          &nStride,
                                          &cStride,
                                          &hStride,
                                          &wStride));
    reluBackward<<<GET_BLOCKS(n * c * h * w), CUDA_NUM_THREADS, 0, stream>>>(
        output_grad_ptr, output_ptr, n * c * h * w);
  }
  // Compute filter gradiant
  // NOTE: we use alpha for kernel_grad to accumulate gradients
  checkCUDNN(hipdnnConvolutionBackwardFilter(m->handle.dnn,
                                            &alpha,
                                            m->inputTensor,
                                            input_ptr,
                                            m->outputTensor,
                                            output_grad_ptr,
                                            m->convDesc,
                                            m->bwdFilterAlgo,
                                            m->handle.workSpace,
                                            m->handle.workSpaceSize,
                                            &alpha,
                                            m->filterDesc,
                                            kernel_grad_ptr));
  // Compute bias gradiant
  // NOTE: we use alpha for bias_grad to accumulate gradients
  if (bias_grad_ptr != NULL) {
    checkCUDNN(hipdnnConvolutionBackwardBias(m->handle.dnn,
                                            &alpha,
                                            m->outputTensor,
                                            output_grad_ptr,
                                            &alpha,
                                            m->biasTensor,
                                            bias_grad_ptr));
  }
  // Compute data gradiant
  // NOTE: we use alpha for input_grad to accumulate gradients
  if (input_grad_ptr != NULL) {
    checkCUDNN(hipdnnConvolutionBackwardData(m->handle.dnn,
                                            &alpha,
                                            m->filterDesc,
                                            kernel_ptr,
                                            m->outputTensor,
                                            output_grad_ptr,
                                            m->convDesc,
                                            m->bwdDataAlgo,
                                            m->handle.workSpace,
                                            m->handle.workSpaceSize,
                                            &alpha,
                                            m->inputTensor,
                                            input_grad_ptr));
  }
}

/*static*/
void Conv2D::backward_kernel_wrapper(Conv2DMeta const *m,
                                     float const *input_ptr,
                                     float *input_grad_ptr,
                                     float const *output_ptr,
                                     float *output_grad_ptr,
                                     float const *kernel_ptr,
                                     float *kernel_grad_ptr,
                                     float *bias_grad_ptr) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  Conv2D::backward_kernel(m,
                          input_ptr,
                          input_grad_ptr,
                          output_ptr,
                          output_grad_ptr,
                          kernel_ptr,
                          kernel_grad_ptr,
                          bias_grad_ptr,
                          stream);
  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Conv2D] backward time = %.2fms\n", m->op_name, elapsed);
    // print_tensor<4, float>(acc_output_grad.ptr, acc_output_grad.rect,
    // "[Conv2D:backward:output_grad]"); print_tensor<4,
    // float>(acc_kernel_grad.ptr, acc_kernel_grad.rect,
    // "[Conv2D:backward:kernel_grad]"); print_tensor<1,
    // float>(acc_bias_grad.ptr, acc_bias_grad.rect,
    // "[Conv2D:backward:bias_grad]"); print_tensor<4,
    // float>(acc_input_grad.ptr, acc_input_grad.rect,
    // "[Conv2D:backward:input_grad]");
  }
}

hipdnnConvolutionFwdAlgo_t selectConvolutionForwardAlgorithm(
    hipdnnHandle_t handle,
    const hipdnnTensorDescriptor_t xDesc,
    void const *x,
    const hipdnnFilterDescriptor_t wDesc,
    void const *w,
    const hipdnnConvolutionDescriptor_t convDesc,
    void *workSpace,
    size_t workSpaceSize,
    const hipdnnTensorDescriptor_t yDesc,
    void *y) {
  int const reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionFwdAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionForwardAlgorithmEx(handle,
                                                    xDesc,
                                                    x,
                                                    wDesc,
                                                    w,
                                                    convDesc,
                                                    yDesc,
                                                    y,
                                                    reqAlgCnt,
                                                    &cnt,
                                                    perfResults,
                                                    workSpace,
                                                    workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("forwardAlgo(%d) time(%.2lf)\n",
         perfResults[0].algo,
         perfResults[0].time);
  return perfResults[0].algo;
}

hipdnnConvolutionBwdFilterAlgo_t selectConvolutionBackwardFilterAlgorithm(
    hipdnnHandle_t handle,
    const hipdnnTensorDescriptor_t xDesc,
    void const *x,
    const hipdnnTensorDescriptor_t dyDesc,
    void const *dy,
    const hipdnnConvolutionDescriptor_t convDesc,
    void *workSpace,
    size_t workSpaceSize,
    const hipdnnFilterDescriptor_t dwDesc,
    void *dw) {
  int const reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionBwdFilterAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionBackwardFilterAlgorithmEx(handle,
                                                           xDesc,
                                                           x,
                                                           dyDesc,
                                                           dy,
                                                           convDesc,
                                                           dwDesc,
                                                           dw,
                                                           reqAlgCnt,
                                                           &cnt,
                                                           perfResults,
                                                           workSpace,
                                                           workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("bwdFilterAlgo(%d) time(%.2lf)\n",
         perfResults[0].algo,
         perfResults[0].time);
  return perfResults[0].algo;
}

hipdnnConvolutionBwdDataAlgo_t selectConvolutionBackwardDataAlgorithm(
    hipdnnHandle_t handle,
    const hipdnnFilterDescriptor_t wDesc,
    void const *w,
    const hipdnnTensorDescriptor_t dyDesc,
    void const *dy,
    const hipdnnConvolutionDescriptor_t convDesc,
    void *workSpace,
    size_t workSpaceSize,
    const hipdnnTensorDescriptor_t dxDesc,
    void *dx) {
  int const reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionBwdDataAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionBackwardDataAlgorithmEx(handle,
                                                         wDesc,
                                                         w,
                                                         dyDesc,
                                                         dy,
                                                         convDesc,
                                                         dxDesc,
                                                         dx,
                                                         reqAlgCnt,
                                                         &cnt,
                                                         perfResults,
                                                         workSpace,
                                                         workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("bwdDataAlgo(%d) time(%.2lf)\n",
         perfResults[0].algo,
         perfResults[0].time);
  return perfResults[0].algo;
}

Conv2DMeta::Conv2DMeta(FFHandler handler) : OpMeta(handler) {
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
}

// TODO: refactor it
bool Conv2D::measure_operator_cost(Simulator *sim,
                                   MachineView const &mv,
                                   CostMetrics &cost_metrics) const {
  ParallelTensorBase sub_output, sub_input;
  if (!outputs[0]->get_sub_tensor(mv, sub_output))
    return false;
  if (!inputs[0]->get_sub_tensor(mv, sub_input))
    return false;
  int input_w = sub_input.dims[0].size;
  int input_h = sub_input.dims[1].size;
  int input_c = sub_input.dims[2].size;
  int input_n = sub_input.dims[3].size;
  int output_w = sub_output.dims[0].size;
  int output_h = sub_output.dims[1].size;
  int output_c = sub_output.dims[2].size;
  int output_n = sub_output.dims[3].size;
  int pad_h = ((output_h - 1) * stride_h + kernel_h - input_h + 1) / 2;
  int pad_w = ((output_w - 1) * stride_w + kernel_w - input_w + 1) / 2;

  Conv2DMeta *m = sim->conv2d_meta;
  m->relu = activation == AC_MODE_RELU;
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        input_n,
                                        input_c,
                                        input_h,
                                        input_w));
  checkCUDNN(hipdnnSetTensor4dDescriptor(
      m->biasTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, output_c, 1, 1));
  // require input_c is divisible by groups
  assert(input_c % groups == 0);
  checkCUDNN(hipdnnSetFilter4dDescriptor(m->filterDesc,
                                        HIPDNN_DATA_FLOAT,
                                        HIPDNN_TENSOR_NCHW,
                                        output_c,
                                        input_c / groups,
                                        kernel_h,
                                        kernel_w));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(m->convDesc,
                                             pad_h,
                                             pad_w,
                                             stride_h,
                                             stride_w,
                                             1 /*dilationH*/,
                                             1 /*dilationW*/,
                                             HIPDNN_CROSS_CORRELATION,
                                             HIPDNN_DATA_FLOAT));

  checkCUDNN(hipdnnSetConvolutionGroupCount(m->convDesc, groups));
  if (m->handle.allowTensorOpMathConversion) {
    checkCUDNN(hipdnnSetConvolutionMathType(
        m->convDesc, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
  } else {
    checkCUDNN(hipdnnSetConvolutionMathType(m->convDesc, HIPDNN_TENSOR_OP_MATH));
  }
  int n, c, h, w;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(
      m->convDesc, m->inputTensor, m->filterDesc, &n, &c, &h, &w));
  assert(n == output_n);
  assert(c == output_c);
  assert(h == output_h);
  assert(w == output_w);
  checkCUDNN(hipdnnSetActivationDescriptor(
      m->actiDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0.0));
  checkCUDNN(hipdnnSetTensor4dDescriptor(
      m->outputTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
  // allocate tensors in simulator
  sim->free_all();
  float *input_ptr = (float *)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert(input_ptr != NULL);
  cost_metrics.inputs_memory += cost_metrics.total_mem_diff_from(sim->offset);

  float *output_ptr = (float *)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert(output_ptr != NULL);
  cost_metrics.outputs_memory += cost_metrics.total_mem_diff_from(sim->offset);

  float *weight_ptr = (float *)sim->allocate(
      (size_t)output_c * input_c * kernel_h * kernel_w / groups, DT_FLOAT);
  assert(weight_ptr != NULL);
  float *bias_ptr = (float *)sim->allocate(output_c, DT_FLOAT);
  assert(bias_ptr != NULL);
  cost_metrics.weights_memory += cost_metrics.total_mem_diff_from(sim->offset);

  // select forward algorithm
  {
    int const reqAlgCnt = 8;
    int cnt = 0;
    hipdnnConvolutionFwdAlgoPerf_t perfResults[reqAlgCnt];
    checkCUDNN(hipdnnFindConvolutionForwardAlgorithmEx(m->handle.dnn,
                                                      m->inputTensor,
                                                      input_ptr,
                                                      m->filterDesc,
                                                      weight_ptr,
                                                      m->convDesc,
                                                      m->outputTensor,
                                                      output_ptr,
                                                      reqAlgCnt,
                                                      &cnt,
                                                      perfResults,
                                                      m->handle.workSpace,
                                                      m->handle.workSpaceSize));
    assert(cnt > 0);
    checkCUDNN(perfResults[0].status);
    cost_metrics.forward_time = perfResults[0].time;
    // for (int i = 0; i < cnt; i++)
    //   printf("conv forward: algo(%d) time(%.4lf)\n", perfResults[i].algo,
    //   perfResults[i].time);
  }
  // select backward algorithm
  {
    int const reqAlgCnt = 8;
    int cnt = 0;
    hipdnnConvolutionBwdFilterAlgoPerf_t perfResults[reqAlgCnt];
    checkCUDNN(
        hipdnnFindConvolutionBackwardFilterAlgorithmEx(m->handle.dnn,
                                                      m->inputTensor,
                                                      input_ptr,
                                                      m->outputTensor,
                                                      output_ptr,
                                                      m->convDesc,
                                                      m->filterDesc,
                                                      weight_ptr,
                                                      reqAlgCnt,
                                                      &cnt,
                                                      perfResults,
                                                      m->handle.workSpace,
                                                      m->handle.workSpaceSize));
    assert(cnt > 0);
    checkCUDNN(perfResults[0].status);
    cost_metrics.backward_time = perfResults[0].time;
    // for (int i = 0; i < cnt; i++)
    //   printf("conv backward filter: algo(%d) time(%.4lf)\n",
    //   perfResults[i].algo, perfResults[i].time);
  }
  if (trainableInputs[0]) {
    int const reqAlgCnt = 8;
    int cnt = 0;
    hipdnnConvolutionBwdDataAlgoPerf_t perfResults[reqAlgCnt];
    checkCUDNN(
        hipdnnFindConvolutionBackwardDataAlgorithmEx(m->handle.dnn,
                                                    m->filterDesc,
                                                    weight_ptr,
                                                    m->outputTensor,
                                                    output_ptr,
                                                    m->convDesc,
                                                    m->inputTensor,
                                                    input_ptr,
                                                    reqAlgCnt,
                                                    &cnt,
                                                    perfResults,
                                                    m->handle.workSpace,
                                                    m->handle.workSpaceSize));
    assert(cnt > 0);
    checkCUDNN(perfResults[0].status);
    cost_metrics.backward_time += perfResults[0].time;
    // for (int i = 0; i < cnt; i++)
    //   printf("conv backward data: algo(%d) time(%.4lf)\n",
    //   perfResults[i].algo, perfResults[i].time);
  }
  log_measure.debug("[Measure Conv2D] name(%s) input(%d %d %d %d) weight(%d %d "
                    "%d %d) output(%d %d %d %d) stride(%d %d) padding(%d %d) "
                    "forward_time(%.4lf) backward_time(%.4lf)\n",
                    name,
                    input_n,
                    input_c,
                    input_h,
                    input_w,
                    output_c,
                    input_c / groups,
                    kernel_h,
                    kernel_w,
                    output_n,
                    output_c,
                    output_h,
                    output_w,
                    stride_h,
                    stride_w,
                    padding_h,
                    padding_w,
                    cost_metrics.forward_time,
                    cost_metrics.backward_time);
  return true;
}

}; // namespace FlexFlow
