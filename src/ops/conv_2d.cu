/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::conv2d(const Tensor& input,
                       int outChannels,
                       int kernelH, int kernelW,
                       int strideH, int strideW,
                       int paddingH, int paddingW,
                       int groups,
                       ActiMode activation,
                       bool use_bias,
                       const Op* shared_op,
                       Initializer* kernel_initializer,
                       Initializer* bias_initializer,
                       char const *name)
{
  if (kernel_initializer == NULL) {
    int seed = std::rand();
    kernel_initializer = new GlorotUniform(seed);
  }
  if (bias_initializer == NULL) {
    bias_initializer = new ZeroInitializer();
  }

  assert(input.numDim == 4); /*NCHW*/
  Conv2D *conv = new Conv2D(*this, input, outChannels, kernelH, kernelW,
      strideH, strideW, paddingH, paddingW, groups, activation,
      use_bias, shared_op, kernel_initializer, bias_initializer, name);
  layers.push_back(conv);
  return conv->outputs[0];
}

Conv2D* FFModel::conv2d(int inChannels,
                        int outChannels,
                        int kernelH, int kernelW,
                        int strideH, int strideW,
                        int paddingH, int paddingW,
                        int groups,
                        ActiMode activation,
                        bool use_bias,
                        Initializer* kernel_initializer,
                        Initializer* bias_initializer,
                        char const *name)
{
  if (kernel_initializer == NULL) {
    int seed = std::rand();
    kernel_initializer = new GlorotUniform(seed);
  }
  if (bias_initializer == NULL) {
    bias_initializer = new ZeroInitializer();
  }

  Conv2D *conv = new Conv2D(*this, inChannels, outChannels, kernelH, kernelW,
                            strideH, strideW, paddingH, paddingW, groups, activation,
                            use_bias, kernel_initializer, bias_initializer, name);
  layers.push_back(conv);
  return conv;
}

/*
locals[0] = kernel
locals[1] = bias
*/
Conv2D::Conv2D(FFModel& model,
               const Tensor& _input,
               int out_dim,
               int _kernel_h, int _kernel_w,
               int _stride_h, int _stride_w,
               int _padding_h, int _padding_w,
               int _groups,
               ActiMode _activation,
               bool _use_bias,
               const Op* shared_op,
               Initializer* _kernel_initializer,
               Initializer* _bias_initializer,
               const char* name)
: Op(model, OP_CONV2D, shared_op, name, _input),
  in_channels(_input.adim[2]), out_channels(out_dim),
  kernel_h(_kernel_h), kernel_w(_kernel_w),
  stride_h(_stride_h), stride_w(_stride_w),
  padding_h(_padding_h), padding_w(_padding_w),
  groups(_groups), activation(_activation), use_bias(_use_bias),
  kernel_initializer(_kernel_initializer),
  bias_initializer(_bias_initializer),
  profiling(model.config.profiling)
{
  assert(_input.numDim == 4);
  // Set output shape
  int input_w = inputs[0].adim[0];
  int input_h = inputs[0].adim[1];
  int output_w = 1 + (input_w + 2 * padding_w - kernel_w) / stride_w;
  int output_h = 1 + (input_h + 2 * padding_h - kernel_h) / stride_h;
  int output_c = out_channels;
  int output_n = inputs[0].adim[3];
  numOutputs = 1;
  outputs[0].numDim = 4;
  outputs[0].adim[0] = output_w;
  outputs[0].adim[1] = output_h;
  outputs[0].adim[2] = output_c;
  outputs[0].adim[3] = output_n;
  weights[0].numDim = 4;
  weights[0].adim[0] = kernel_w;
  weights[0].adim[1] = kernel_h;
  // Require input channels is divisible by groups
  assert(in_channels % groups == 0);
  weights[0].adim[2] = in_channels / groups;
  weights[0].adim[3] = out_channels;
  numWeights = 1;
  if (use_bias) {
    weights[1].numDim = 1;
    weights[1].adim[0] = out_channels;
    numWeights = 2;
  }
}

Conv2D::Conv2D(FFModel& model,
               int in_dim, int out_dim,
               int _kernel_h, int _kernel_w,
               int _stride_h, int _stride_w,
               int _padding_h, int _padding_w,
               int _groups,
               ActiMode _activation,
               bool _use_bias,
               Initializer* _kernel_initializer,
               Initializer* _bias_initializer,
               const char* name)
: Op(model, OP_CONV2D, name, 1),
  in_channels(in_dim), out_channels(out_dim),
  kernel_h(_kernel_h), kernel_w(_kernel_w),
  stride_h(_stride_h), stride_w(_stride_w),
  padding_h(_padding_h), padding_w(_padding_w),
  groups(_groups), activation(_activation), use_bias(_use_bias),
  kernel_initializer(_kernel_initializer),
  bias_initializer(_bias_initializer),
  profiling(model.config.profiling)
{
}

Tensor Conv2D::init_inout(FFModel& model, const Tensor& _input)
{
  assert(_input.numDim == 4);
  assert(_input.adim[2] == in_channels);
  inputs[0] = _input;
  create_output_and_partition(model);
  return outputs[0];
}

void Conv2D::create_weights(FFModel& model)
{
  // Retrive the task indexspace for the op
  task_is = (IndexSpaceT<4>)model.get_or_create_task_is(4, name);

  // TODO: temp work, will let users to pick either NCCL or PS
#ifdef FF_ENABLE_NCCL
  Parameter::CommType comm_type = Parameter::NCCL;
#else
  Parameter::CommType comm_type = Parameter::PS;
#endif

  // Create kernel
  {
    const int dims[4] = {out_channels, in_channels / groups, kernel_h, kernel_w};
    weights[0] = model.create_conv_weight<4>(this, dims, DT_FLOAT,
        kernel_initializer, true/*create_grad*/, comm_type);
  }
  // Create bias tensor
  if (use_bias) {
    const int dims[1] = {out_channels};
    weights[1] = model.create_conv_weight<1>(this, dims, DT_FLOAT,
        bias_initializer, true/*create_grad*/, comm_type);
    assert(numWeights == 2);
  } else {
    assert(numWeights == 1);
  }
}

void Conv2D::create_output_and_partition(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = IndexSpaceT<4>(model.get_or_create_task_is(4, pcname));

  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<4> part_rect = runtime->get_index_space_domain(ctx, task_is);
  // Create output tensor
  int input_w = inputs[0].adim[0];
  int input_h = inputs[0].adim[1];
  int output_w = 1 + (input_w + 2 * padding_w - kernel_w) / stride_w;
  int output_h = 1 + (input_h + 2 * padding_h - kernel_h) / stride_h;
  int output_c = out_channels;
  int output_n = inputs[0].adim[3];
  int num_par_w = part_rect.hi[0] - part_rect.lo[0] + 1;
  int num_par_h = part_rect.hi[1] - part_rect.lo[1] + 1;
  int num_par_c = part_rect.hi[2] - part_rect.lo[2] + 1;
  int num_par_n = part_rect.hi[3] - part_rect.lo[3] + 1;
  {
    const int dims[4] = {output_n, output_c, output_h, output_w};
    outputs[0] = model.create_tensor<4>(dims, DT_FLOAT, this);
    outputs[0].owner_op = this;
    outputs[0].owner_idx = 0;
  }
  // Compute partition bound for input
  Rect<4> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part.get_index_partition());
  // Currently assume we didn't split across the channel dimension
  assert(num_par_c == 1);
  if (input_rect == part_rect) {
    input_lps[0] = inputs[0].part;
    input_grad_lps[0] = inputs[0].part_grad;
  } else {
    model.create_disjoint_partition(
        inputs[0], (IndexSpaceT<4>)task_is, input_lps[0], input_grad_lps[0]);
  }
}

hipdnnConvolutionFwdAlgo_t
selectConvolutionForwardAlgorithm(hipdnnHandle_t handle,
                                  const hipdnnTensorDescriptor_t xDesc, const void* x,
                                  const hipdnnFilterDescriptor_t wDesc, const void* w,
                                  const hipdnnConvolutionDescriptor_t convDesc,
                                  void* workSpace, size_t workSpaceSize,
                                  const hipdnnTensorDescriptor_t yDesc, void* y);
hipdnnConvolutionBwdFilterAlgo_t
selectConvolutionBackwardFilterAlgorithm(hipdnnHandle_t handle,
                                         const hipdnnTensorDescriptor_t xDesc, const void* x,
                                         const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                         const hipdnnConvolutionDescriptor_t convDesc,
                                         void* workSpace, size_t workSpaceSize,
                                         const hipdnnFilterDescriptor_t dwDesc, void* dw);
hipdnnConvolutionBwdDataAlgo_t
selectConvolutionBackwardDataAlgorithm(hipdnnHandle_t handle,
                                       const hipdnnFilterDescriptor_t wDesc, const void* w,
                                       const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                       const hipdnnConvolutionDescriptor_t convDesc,
                                       void* workSpace, size_t workSpaceSize,
                                       const hipdnnTensorDescriptor_t dxDesc, void* dx);
/*
  regions[0]: input
  regions[1]: output
  regions[2](I): filter
  regions[3](I): bias
  regions[4](O): filter_grad
  regions[5](O): input_grad
*/
__host__
OpMeta* Conv2D::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 6);
  assert(task->regions.size() == 6);
  const Conv2D* conv = (Conv2D*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  TensorAccessorR<float, 4> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 4> acc_output(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorR<float, 4> acc_kernel(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 1> acc_bias(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 4> acc_kernel_grad(
      regions[4], task->regions[4], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorW<float, 4> acc_input_grad(
      regions[5], task->regions[5], FID_DATA, ctx, runtime,
      false/*readOutput*/);

  Conv2DMeta* m = new Conv2DMeta(handle);
  m->relu = conv->activation == AC_MODE_RELU;

  int input_w = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int input_h = acc_input.rect.hi[1] - acc_input.rect.lo[1] + 1;
  int input_c = acc_input.rect.hi[2] - acc_input.rect.lo[2] + 1;
  int input_n = acc_input.rect.hi[3] - acc_input.rect.lo[3] + 1;
  int output_w = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int output_h = acc_output.rect.hi[1] - acc_output.rect.lo[1] + 1;
  int output_c = acc_output.rect.hi[2] - acc_output.rect.lo[2] + 1;
  int output_n = acc_output.rect.hi[3] - acc_output.rect.lo[3] + 1;
  printf("init conv (input): n(%d) c(%d) h(%d) w(%d)\n",
         input_n, input_c, input_h, input_w);
  printf("init conv (output): n(%d) c(%d) h(%d) w(%d)\n",
          output_n, output_c, output_h, output_w);
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor,
      HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
      input_n, input_c, input_h, input_w));

  checkCUDNN(hipdnnSetTensor4dDescriptor(m->biasTensor,
      HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
      1, output_c, 1, 1));

  // Require that input_c is divisible by conv->groups
  assert(input_c % conv->groups == 0);
  printf("filterDim: kernel(%d %d) c_in(%d), c_out(%d)\n",
      conv->kernel_h, conv->kernel_w, input_c / conv->groups, output_c);
  checkCUDNN(hipdnnSetFilter4dDescriptor(m->filterDesc,
      HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
      output_c, input_c / conv->groups, conv->kernel_h, conv->kernel_w));

  //printf("convDim: padding(%d %d) stride(%d %d)\n", conv->padding_h, conv->padding_w, conv->stride_h, conv->stride_w);
  int pad_h = ((output_h - 1) * conv->stride_h + conv->kernel_h - input_h + 1) / 2;
  int pad_w = ((output_w - 1) * conv->stride_w + conv->kernel_w - input_w + 1) / 2;
  if (pad_h != conv->padding_h)
    printf("Warning: changing conv_padding_h to satisfy output_h size\n");
  if (pad_w != conv->padding_w)
    printf("Warning: changing conv_padding_w to satisfy output_w size\n");

  checkCUDNN(hipdnnSetConvolution2dDescriptor(m->convDesc,
                                             pad_h,//conv->padding_h,
                                             pad_w,//conv->padding_w,
                                             conv->stride_h,
                                             conv->stride_w,
                                             1/*upscale_x*/,
                                             1/*upscale_y*/,
                                             HIPDNN_CROSS_CORRELATION,
                                             HIPDNN_DATA_FLOAT));
  if (conv->groups != 1) {
    checkCUDNN(hipdnnSetConvolutionGroupCount(m->convDesc, conv->groups));
  }
  // enable tensor core when possible
  checkCUDNN(hipdnnSetConvolutionMathType(m->convDesc, HIPDNN_TENSOR_OP_MATH));
  int n, c, h, w;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(m->convDesc,
                                                   m->inputTensor,
                                                   m->filterDesc,
                                                   &n, &c, &h, &w));
  assert(n == output_n);
  assert(c == output_c);
  assert(h == output_h);
  assert(w == output_w);

  checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        n, c, h, w));
  // select forward algorithm
  m->fwdAlgo = selectConvolutionForwardAlgorithm(m->handle.dnn, m->inputTensor, acc_input.ptr,
                                                 m->filterDesc, acc_kernel.ptr, m->convDesc,
                                                 m->handle.workSpace, m->handle.workSpaceSize,
                                                 m->outputTensor, acc_output.ptr);
  // select backward filter algorithm
  m->bwdFilterAlgo = selectConvolutionBackwardFilterAlgorithm(
                         m->handle.dnn, m->inputTensor, acc_input.ptr,
                         m->outputTensor, acc_output.ptr,
                         m->convDesc, m->handle.workSpace, m->handle.workSpaceSize,
                         m->filterDesc, acc_kernel_grad.ptr);
  // select backward data algorithm
  m->bwdDataAlgo = selectConvolutionBackwardDataAlgorithm(
                       m->handle.dnn, m->filterDesc, acc_kernel.ptr,
                       m->outputTensor, acc_output.ptr,
                       m->convDesc, m->handle.workSpace, m->handle.workSpaceSize,
                       m->inputTensor, acc_input_grad.ptr);
  if (m->relu) {
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, HIPDNN_ACTIVATION_RELU,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
  }
#ifdef FF_ENABLE_NCCL
  m->init_nccl_communicator(task, conv->ncclId);
#endif
  return m;
}

void Conv2D::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<4> rect = runtime->get_index_space_domain(ctx, task_is);
  ParallelConfig pc;
  std::string pcname = name;
  ff.config.find_parallel_config(4, pcname, pc);
  int idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    FFHandler handle = ff.handlers[pc.device_ids[idx++]];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
  }
  IndexLauncher launcher(CONV2D_INIT_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Conv2D)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[0].region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[1].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[1].region));
  launcher.add_field(3, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part_grad, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, weights[0].region_grad));
  launcher.add_field(4, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(5, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/*static*/
void Conv2D::forward_kernel(const Conv2DMeta* m,
                            const float* input_ptr,
                            float* output_ptr,
                            const float* filter_ptr,
                            const float* bias_ptr)
{
  float alpha = 1.0f, beta = 0.0f;
  checkCUDNN(hipdnnConvolutionForward(m->handle.dnn, &alpha,
                                     m->inputTensor, input_ptr,
                                     m->filterDesc, filter_ptr,
                                     m->convDesc, m->fwdAlgo,
                                     m->handle.workSpace, m->handle.workSpaceSize,
                                     &beta, m->outputTensor, output_ptr));

  checkCUDNN(hipdnnAddTensor(m->handle.dnn, &alpha, m->biasTensor,
                            bias_ptr, &alpha, m->outputTensor, output_ptr));
  if (m->relu) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
                                      &alpha, m->outputTensor, output_ptr,
                                      &beta, m->outputTensor, output_ptr));
  }
}

/*
  regions[0](I): input
  regions[1](O): output
  regions[2](I): filter
  regions[3](I): bias
*/
__host__
void Conv2D::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  Conv2D* conv = (Conv2D*) task->args;
  const Conv2DMeta* m = *((Conv2DMeta**) task->local_args);
  TensorAccessorR<float, 4> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 4> acc_output(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorR<float, 4> acc_kernel(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 1> acc_bias(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);

  //printf("fwdAlgo(%d), bwdFilterALgo(%d), bwdDataAlgo(%d)\n", (int)m->fwdAlgo,(int) m->bwdFilterAlgo,(int) m->bwdDataAlgo);
  hipEvent_t t_start, t_end;
  if (conv->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }

#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  conv->forward_kernel(m, acc_input.ptr, acc_output.ptr, acc_kernel.ptr, acc_bias.ptr);
  if (conv->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    //print_tensor<4, float>(acc_input.ptr, acc_input.rect, "[Conv2D:forward:input]");
    //print_tensor<4, float>(acc_kernel.ptr, acc_kernel.rect, "[Conv2D:forward:kernel]");
    //print_tensor<1, float>(acc_bias.ptr, acc_bias.rect, "[Conv2D:forward:bias]");
    //print_tensor<4, float>(acc_output.ptr, acc_output.rect, "[Conv2D:forward:output]");
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Conv2D] forward time (CF) = %.2fms\n", conv->name, elapsed);
  }
}

__host__
void Conv2D::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<4> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(CONV2D_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Conv2D)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[0].region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[1].region, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[1].region));
  launcher.add_field(3, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

/*static*/
void Conv2D::backward_kernel(const Conv2DMeta* m,
                             const float* input_ptr,
                             float* input_grad_ptr,
                             const float* output_ptr,
                             float* output_grad_ptr,
                             const float* kernel_ptr,
                             float* kernel_grad_ptr,
                             float* bias_grad_ptr)
{
  float alpha = 1.0f;
  //float beta = 0.0f;
  if (m->relu) {
    hipdnnDataType_t dataType;
    int n, c, h, w, nStride, cStride, hStride, wStride;
    checkCUDNN(hipdnnGetTensor4dDescriptor(m->outputTensor, &dataType,
        &n, &c, &h, &w, &nStride, &cStride, &hStride, &wStride));
    reluBackward<<<GET_BLOCKS(n*c*h*w), CUDA_NUM_THREADS>>>(output_grad_ptr, output_ptr, n*c*h*w);
  }
  // Compute filter gradiant
  // NOTE: we use alpha for kernel_grad to accumulate gradients
  checkCUDNN(hipdnnConvolutionBackwardFilter(m->handle.dnn, &alpha,
                                            m->inputTensor, input_ptr,
                                            m->outputTensor, output_grad_ptr,
                                            m->convDesc, m->bwdFilterAlgo,
                                            m->handle.workSpace, m->handle.workSpaceSize,
                                            &alpha, m->filterDesc, kernel_grad_ptr));
  // Compute bias gradiant
  // NOTE: we use alpha for bias_grad to accumulate gradients
  checkCUDNN(hipdnnConvolutionBackwardBias(m->handle.dnn, &alpha,
                                          m->outputTensor, output_grad_ptr,
                                          &alpha, m->biasTensor, bias_grad_ptr));
  // Compute data gradiant
  // NOTE: we use alpha for input_grad to accumulate gradients
  checkCUDNN(hipdnnConvolutionBackwardData(m->handle.dnn, &alpha,
                                          m->filterDesc, kernel_ptr,
                                          m->outputTensor, output_grad_ptr,
                                          m->convDesc, m->bwdDataAlgo,
                                          m->handle.workSpace, m->handle.workSpaceSize,
                                          &alpha, m->inputTensor, input_grad_ptr));
}

/*
  regions[0](I): input
  regions[1](I/O): input_grad
  regions[2](I): output
  regions[3](I/O): output_grad
  regions[4](I): filter
  regions[5](I/O): filter_grad
  regions[6](I/O): bias_grad
*/
__host__
void Conv2D::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  assert(regions.size() == 7);
  assert(task->regions.size() == 7);
  Conv2D* conv = (Conv2D*) task->args;
  const Conv2DMeta* m = *((Conv2DMeta**) task->local_args);
  TensorAccessorR<float, 4> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 4> acc_input_grad(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, 4> acc_output(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 4> acc_output_grad(
      regions[3], task->regions[3], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, 4> acc_kernel(
      regions[4], task->regions[4], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 4> acc_kernel_grad(
      regions[5], task->regions[5], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorW<float, 1> acc_bias_grad(
      regions[6], task->regions[6], FID_DATA, ctx, runtime,
      true/*readOutput*/);

  hipEvent_t t_start, t_end;
  if (conv->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }

#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  conv->backward_kernel(m, acc_input.ptr, acc_input_grad.ptr,
                        acc_output.ptr, acc_output_grad.ptr,
                        acc_kernel.ptr, acc_kernel_grad.ptr,
                        acc_bias_grad.ptr);
  if (conv->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Conv2D] backward time = %.2fms\n", conv->name, elapsed);
    //print_tensor<4, float>(acc_output_grad.ptr, acc_output_grad.rect, "[Conv2D:backward:output_grad]");
    //print_tensor<4, float>(acc_kernel_grad.ptr, acc_kernel_grad.rect, "[Conv2D:backward:kernel_grad]");
    //print_tensor<1, float>(acc_bias_grad.ptr, acc_bias_grad.rect, "[Conv2D:backward:bias_grad]");
    //print_tensor<4, float>(acc_input_grad.ptr, acc_input_grad.rect, "[Conv2D:backward:input_grad]");
  }
}

__host__
void Conv2D::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<4> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }

  IndexLauncher launcher(CONV2D_BWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Conv2D)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  // regions[0](I): input
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  // regions[1](I/O): input_grad
  launcher.add_region_requirement(
      RegionRequirement(inputs[0].part_grad, 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(1, FID_DATA);
  // regions[2](I): output
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(2, FID_DATA);
  // regions[3](I/O): output_grad
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part_grad, 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, outputs[0].region_grad));
  launcher.add_field(3, FID_DATA);
  // regions[4](I): filter
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[0].region));
  launcher.add_field(4, FID_DATA);
  // regions[5](I/O): filter_grad
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part_grad, 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, weights[0].region_grad));
  launcher.add_field(5, FID_DATA);
  // regions[6](I/O): bias_grad
  launcher.add_region_requirement(
      RegionRequirement(weights[1].part_grad, 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, weights[1].region_grad));
  launcher.add_field(6, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  // TODO: remove this line
  //if (first_layer)
    //fm.wait_all_results();
}

#ifdef DEADCODE
/*
  regions[0](I/O): filter
  regions[1](I): filter_grad
  regions[2](I/O): bias
  regions[3](I): bias_grad
*/
__host__
void Conv2D::update_task(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  const Conv2D* conv = (Conv2D*) task->args;
  const AccessorRW<float, 1> acc_filter(regions[0], FID_DATA);
  const AccessorRO<float, 1> acc_filter_grad(regions[1], FID_DATA);
  const AccessorRW<float, 1> acc_bias(regions[2], FID_DATA);
  const AccessorRO<float, 1> acc_bias_grad(regions[3], FID_DATA);
  Rect<1> rect_filter, rect_filter_grad, rect_bias, rect_bias_grad;
  rect_filter =
    runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_filter_grad =
    runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_bias =
    runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_bias_grad =
    runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  size_t filter_size = rect_filter.volume();
  size_t bias_size = rect_bias.volume();
  assert(filter_size == conv->in_channels * conv->out_channels
                        * conv->kernel_w * conv->kernel_h);
  assert(bias_size == conv->out_channels);
  assert(filter_size * conv->num_replica == rect_filter_grad.volume());
  assert(bias_size * conv->num_replica == rect_bias_grad.volume());
  assert(acc_filter.accessor.is_dense_arbitrary(rect_filter));
  assert(acc_filter_grad.accessor.is_dense_arbitrary(rect_filter_grad));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  assert(acc_bias_grad.accessor.is_dense_arbitrary(rect_bias_grad));
  float *filter_ptr = acc_filter.ptr(rect_filter.lo);
  const float *filter_grad_ptr = acc_filter_grad.ptr(rect_filter_grad.lo);
  float *bias_ptr = acc_bias.ptr(rect_bias.lo);
  const float *bias_grad_ptr = acc_bias_grad.ptr(rect_bias_grad.lo);
  updateGAS(filter_ptr, filter_grad_ptr, filter_size,
            conv->num_replica, conv->learning_rate);
  updateGAS(bias_ptr, bias_grad_ptr, bias_size,
            conv->num_replica, conv->learning_rate);
}

__host__
void Conv2D::update(const FFModel& ff)
{
  // Synchronize the learning rate
  learning_rate = ff.config.learningRate;
  assert(num_replica > 0);
  // Only aggregate parameters if more than one replica
  if (num_replica > 1) {
    Context ctx = ff.config.lg_ctx;
    Runtime* runtime = ff.config.lg_hlr;
    TaskLauncher launcher(CONV2D_UPD_TASK_ID, TaskArgument(this, sizeof(Conv2D)));
    launcher.add_region_requirement(
      RegionRequirement(locals[0].region, READ_WRITE, EXCLUSIVE, locals[0].region));
    launcher.add_field(0, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(locals[0].region_grad, READ_ONLY, EXCLUSIVE, locals[0].region_grad));
    launcher.add_field(1, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(locals[1].region, READ_WRITE, EXCLUSIVE, locals[1].region));
    launcher.add_field(2, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(locals[1].region_grad, READ_ONLY, EXCLUSIVE, locals[1].region_grad));
    launcher.add_field(3, FID_DATA);
    runtime->execute_task(ctx, launcher);
  }
}
#endif

/*
__host__
Parameter* Conv2D::get_parameter(int index)
{
  if (index == 0) {
    return &weights[0];
  } else if (index == 1) {
    return &weights[1];
  } else {
    assert(0);
    return NULL;
  }
}*/

__host__
void Conv2D::print_layer(const FFModel& ff)
{
  printf("conv2d layer\n");
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
#if 0
  TaskLauncher launcher(CONV2D_PRINT_TASK_ID, TaskArgument(NULL, 0));
  launcher.add_region_requirement(
    RegionRequirement(kernel.region, READ_ONLY, EXCLUSIVE, kernel.region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
    RegionRequirement(bias.region, READ_ONLY, EXCLUSIVE, bias.region));
  launcher.add_field(1, FID_DATA);
  Future fu = runtime->execute_task(ctx, launcher);
  fu.wait();
#else
  RegionRequirement kernel_req(weights[0].region, READ_WRITE, EXCLUSIVE, weights[0].region);
  kernel_req.add_field(FID_DATA);
  InlineLauncher kernel_launcher(kernel_req);
  PhysicalRegion kernel_region = runtime->map_region(ctx, kernel_launcher);
  kernel_region.wait_until_valid();

/*
  RegionRequirement kernel_grad_req(kernel.region_grad, READ_WRITE, EXCLUSIVE, kernel.region_grad);
  kernel_grad_req.add_field(FID_DATA);
  InlineLauncher kernel_grad_launcher(kernel_grad_req);
  PhysicalRegion kernel_grad_region = runtime->map_region(ctx, kernel_grad_launcher);
  kernel_grad_region.wait_until_valid();
*/
  RegionRequirement bias_req(weights[1].region, READ_WRITE, EXCLUSIVE, weights[1].region);
  bias_req.add_field(FID_DATA);
  InlineLauncher bias_launcher(bias_req);
  PhysicalRegion bias_region = runtime->map_region(ctx, bias_launcher);
  bias_region.wait_until_valid();
/*
  RegionRequirement bias_grad_req(bias.region_grad, READ_WRITE, EXCLUSIVE, bias.region_grad);
  bias_grad_req.add_field(FID_DATA);
  InlineLauncher bias_grad_launcher(bias_grad_req);
  PhysicalRegion bias_grad_region = runtime->map_region(ctx, bias_grad_launcher);
  bias_grad_region.wait_until_valid();
  */
  TensorAccessorW<float, 4> acc_kernel(kernel_region, kernel_req, FID_DATA, ctx, runtime, true);
//  const AccessorRW<float, 1> acc_kernel_grad(kernel_grad_region, FID_DATA);
  TensorAccessorW<float, 1> acc_bias(bias_region, bias_req, FID_DATA, ctx, runtime, true);
  //const AccessorRW<float, 1> acc_bias_grad(bias_grad_region, FID_DATA);

  const float *kernel_ptr = acc_kernel.ptr;
  //float *kernel_grad_ptr = acc_kernel_grad.ptr;
  const float *bias_ptr = acc_bias.ptr;
  //float *bias_grad_ptr = acc_bias_grad.ptr;

  size_t kernel_size = acc_kernel.rect.volume();
  int kernel_dim1 = acc_kernel.rect.hi[0] - acc_kernel.rect.lo[0] + 1;
  int kernel_dim2 = acc_kernel.rect.hi[1] - acc_kernel.rect.lo[1] + 1;
  int kernel_dim3 = acc_kernel.rect.hi[2] - acc_kernel.rect.lo[2] + 1;
  int kernel_dim4 = acc_kernel.rect.hi[3] - acc_kernel.rect.lo[3] + 1;
  //size_t kernel_grad_size = rect_kernel_grad.volume();
  size_t bias_size = acc_bias.rect.volume();
  //size_t bias_grad_size = rect_bias_grad.volume();
  printf("kernel, %p, %d, [%d, %d, %d, %d]\n", kernel_ptr, kernel_size, kernel_dim1, kernel_dim2, kernel_dim3, kernel_dim4);
  //printf("kernel_grad, %d\n", kernel_grad_size);
  printf("bias, %p, %d\n", bias_ptr, bias_size);
  //printf("bias_grad, %d\n", bias_grad_size);


  for (int i = 0; i < bias_size; i++) {
    printf("%f ", bias_ptr[i]);
  }
  printf("\n");

/*
  for (int i = 0; i < bias_grad_size; i++) {
    printf("%f ", bias_grad_ptr);
    bias_grad_ptr ++;
  }
  printf("\n");*/

  for (int i = 0; i < kernel_size; i++) {
    printf("%f ", kernel_ptr[i]);
  }
  printf("\n");

/*
  for (int i = 0; i < kernel_grad_size; i++) {
    printf("%f ", kernel_grad_ptr);
    kernel_grad_ptr ++;
  }
  printf("\n");
  */
  runtime->unmap_region(ctx, kernel_region);
 // runtime->unmap_region(ctx, kernel_grad_region);
  runtime->unmap_region(ctx, bias_region);
//  runtime->unmap_region(ctx, bias_grad_region);
#endif
}

hipdnnConvolutionFwdAlgo_t
selectConvolutionForwardAlgorithm(hipdnnHandle_t handle,
                                  const hipdnnTensorDescriptor_t xDesc, const void* x,
                                  const hipdnnFilterDescriptor_t wDesc, const void* w,
                                  const hipdnnConvolutionDescriptor_t convDesc,
                                  void* workSpace, size_t workSpaceSize,
                                  const hipdnnTensorDescriptor_t yDesc, void* y)
{
  const int reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionFwdAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionForwardAlgorithmEx(
      handle, xDesc, x, wDesc, w, convDesc, yDesc, y,
      reqAlgCnt, &cnt, perfResults, workSpace, workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("forwardAlgo(%d) time(%.2lf)\n", perfResults[0].algo, perfResults[0].time);
  return perfResults[0].algo;
}

hipdnnConvolutionBwdFilterAlgo_t
selectConvolutionBackwardFilterAlgorithm(hipdnnHandle_t handle,
                                         const hipdnnTensorDescriptor_t xDesc, const void* x,
                                         const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                         const hipdnnConvolutionDescriptor_t convDesc,
                                         void* workSpace, size_t workSpaceSize,
                                         const hipdnnFilterDescriptor_t dwDesc, void* dw)
{
  const int reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionBwdFilterAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionBackwardFilterAlgorithmEx(
      handle, xDesc, x, dyDesc, dy, convDesc, dwDesc, dw,
      reqAlgCnt, &cnt, perfResults, workSpace, workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("bwdFilterAlgo(%d) time(%.2lf)\n", perfResults[0].algo, perfResults[0].time);
  return perfResults[0].algo;
}

hipdnnConvolutionBwdDataAlgo_t
selectConvolutionBackwardDataAlgorithm(hipdnnHandle_t handle,
                                       const hipdnnFilterDescriptor_t wDesc, const void* w,
                                       const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                       const hipdnnConvolutionDescriptor_t convDesc,
                                       void* workSpace, size_t workSpaceSize,
                                       const hipdnnTensorDescriptor_t dxDesc, void* dx)
{
  const int reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionBwdDataAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionBackwardDataAlgorithmEx(
      handle, wDesc, w, dyDesc, dy, convDesc, dxDesc, dx,
      reqAlgCnt, &cnt, perfResults, workSpace, workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("bwdDataAlgo(%d) time(%.2lf)\n", perfResults[0].algo, perfResults[0].time);
  return perfResults[0].algo;
}

Conv2DMeta::Conv2DMeta(FFHandler handler)
: OpMeta(handler)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
}

bool Conv2D::measure_compute_time(Simulator* sim,
                                  const ParallelConfig& pc,
                                  float& forward_time,
                                  float& backward_time)
{
  Tensor sub_output, sub_input;
  if(!outputs[0].get_output_sub_tensor(pc, sub_output, OP_CONV2D))
    return false;
  if(!inputs[0].get_input_sub_tensor(pc, sub_input, OP_CONV2D))
    return false;
  int input_w = sub_input.adim[0];
  int input_h = sub_input.adim[1];
  int input_c = sub_input.adim[2];
  int input_n = sub_input.adim[3];
  int output_w = sub_output.adim[0];
  int output_h = sub_output.adim[1];
  int output_c = sub_output.adim[2];
  int output_n = sub_output.adim[3];
  int pad_h = ((output_h - 1) * stride_h + kernel_h - input_h + 1) / 2;
  int pad_w = ((output_w - 1) * stride_w + kernel_w - input_w + 1) / 2;

  Conv2DMeta* m = sim->conv2d_meta;
  m->relu = activation == AC_MODE_RELU;
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, input_n, input_c, input_h, input_w));
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->biasTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, 1, output_c, 1, 1));
  // require input_c is divisible by groups
  assert(input_c % groups == 0);
  checkCUDNN(hipdnnSetFilter4dDescriptor(m->filterDesc, HIPDNN_DATA_FLOAT,
      HIPDNN_TENSOR_NCHW, output_c, input_c / groups, kernel_h, kernel_w));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(m->convDesc, pad_h, pad_w,
      stride_h, stride_w, 1/*dilationH*/, 1/*dilationW*/,
      HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

  checkCUDNN(hipdnnSetConvolutionGroupCount(m->convDesc, groups));
  checkCUDNN(hipdnnSetConvolutionMathType(m->convDesc, HIPDNN_TENSOR_OP_MATH));
  int n, c, h, w;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(m->convDesc,
      m->inputTensor, m->filterDesc, &n, &c, &h, &w));
  assert(n == output_n);
  assert(c == output_c);
  assert(h == output_h);
  assert(w == output_w);
  checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, HIPDNN_ACTIVATION_RELU,
      HIPDNN_NOT_PROPAGATE_NAN, 0.0));
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, n, c, h, w));
  // allocate tensors in simulator
  sim->free_all();
  float* input_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert(input_ptr != NULL);
  float *output_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert(output_ptr != NULL);
  float* weight_ptr = (float*)sim->allocate((size_t)output_c * input_c * kernel_h * kernel_w / groups, DT_FLOAT);
  assert(weight_ptr != NULL);
  float* bias_ptr = (float*)sim->allocate(output_c, DT_FLOAT);
  assert(bias_ptr != NULL);

  // select forward algorithm
  {
    const int reqAlgCnt = 8;
    int cnt = 0;
    hipdnnConvolutionFwdAlgoPerf_t perfResults[reqAlgCnt];
    checkCUDNN(hipdnnFindConvolutionForwardAlgorithmEx(
        m->handle.dnn, m->inputTensor, input_ptr,
        m->filterDesc, weight_ptr, m->convDesc, m->outputTensor, output_ptr,
        reqAlgCnt, &cnt, perfResults,
        m->handle.workSpace, m->handle.workSpaceSize));
    assert(cnt > 0);
    checkCUDNN(perfResults[0].status);
    forward_time = perfResults[0].time;
    //for (int i = 0; i < cnt; i++)
    //  printf("conv forward: algo(%d) time(%.4lf)\n", perfResults[i].algo, perfResults[i].time);
  }
  // select forward algorithm
  {
    const int reqAlgCnt = 8;
    int cnt = 0;
    hipdnnConvolutionBwdFilterAlgoPerf_t perfResults[reqAlgCnt];
    checkCUDNN(hipdnnFindConvolutionBackwardFilterAlgorithmEx(
        m->handle.dnn, m->inputTensor, input_ptr,
        m->outputTensor, output_ptr, m->convDesc, m->filterDesc, weight_ptr,
        reqAlgCnt, &cnt, perfResults,
        m->handle.workSpace, m->handle.workSpaceSize));
    assert(cnt > 0);
    checkCUDNN(perfResults[0].status);
    backward_time = perfResults[0].time;
  }
  {
    const int reqAlgCnt = 8;
    int cnt = 0;
    hipdnnConvolutionBwdDataAlgoPerf_t perfResults[reqAlgCnt];
    checkCUDNN(hipdnnFindConvolutionBackwardDataAlgorithmEx(
        m->handle.dnn, m->filterDesc, weight_ptr,
        m->outputTensor, output_ptr, m->convDesc, m->inputTensor, input_ptr,
        reqAlgCnt, &cnt, perfResults,
        m->handle.workSpace, m->handle.workSpaceSize));
    assert(cnt > 0);
    checkCUDNN(perfResults[0].status);
    backward_time += perfResults[0].time;
  }
  printf("[Measure Conv2D] name(%s) input(%d %d %d %d) weight(%d %d %d %d) output(%d %d %d %d) stride(%d %d) padding(%d %d) forward_time(%.4lf) backward_time(%.4lf)\n",
         name,
         input_n, input_c, input_h, input_w,
         output_c, input_c / groups, kernel_h, kernel_w,
         output_n, output_c, output_h, output_w,
         stride_h, stride_w,
         padding_h, padding_w,
         forward_time, backward_time);
  return true;
}

