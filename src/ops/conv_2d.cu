/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ops/conv_2d.h"
#include "cuda_helper.h"
#include "hash_utils.h"

using namespace Legion;
Tensor FFModel::conv2d(const Tensor input,
                       int outChannels,
                       int kernelH, int kernelW,
                       int strideH, int strideW,
                       int paddingH, int paddingW,
                       ActiMode activation,
                       int groups,
                       bool use_bias,
                       const Op* shared_op,
                       Initializer* kernel_initializer,
                       Initializer* bias_initializer,
                       char const *name)
{
  assert(input->num_dims == 5); /*RNCHW*/

  Conv2D *conv = new Conv2D(
      *this, 
      input, 
      outChannels,
      kernelH, kernelW,
      strideH, strideW, 
      paddingH, paddingW, 
      activation,
      groups,
      use_bias,
      false,
      name
  );
  layers.push_back(conv);
  return conv->outputs[0];
}

namespace Input {
  static constexpr int INDEX = 0;

  enum {
    WIDTH = 0,
    HEIGHT = 1,
    CHANNEL = 2,
    SAMPLE = 3,
    REPLICA = 4,
    NUMDIM
  };
}

namespace Output {
  enum {
    WIDTH = 0,
    HEIGHT = 1,
    CHANNEL = 2,
    SAMPLE = 3,
    REPLICA = 4,
    NUMDIM
  };
}

namespace Kernel {
  static constexpr int INDEX = 0;

  enum {
    WIDTH = 0,
    HEIGHT = 1,
    CHANNEL_IN = 2,
    CHANNEL_OUT = 3,
    REPLICA = 4,
    NUMDIM
  };
}

namespace Bias {
  static constexpr int INDEX = 1;

  enum {
    CHANNEL = 0,
    REPLICA_1 = 1,
    REPLICA_2 = 2,
    REPLICA_3 = 3,
    REPLICA_4 = 4,
    NUMDIM
  };
}


Conv2DParams Conv2D::get_params() const {
  Conv2DParams params;
  params.out_channels = this->out_channels;
  params.kernel_h = this->kernel_h;
  params.kernel_w = this->kernel_w;
  params.stride_h = this->stride_h;
  params.stride_w = this->stride_w;
  params.padding_h = this->padding_h;
  params.padding_w = this->padding_w;
  params.activation = this->activation;
  params.groups = this->groups;
  params.use_bias = this->use_bias;

  return params;
}

Node FFModel::get_or_create_conv2d_node(const Tensor input,
                                        const Conv2DParams& params) 
{
  if (!params.is_valid(input)) {
    return Node::INVALID_NODE;
  }

  size_t hash = input->get_owner_independent_hash();
  hash_combine(hash, params.out_channels);
  hash_combine(hash, params.kernel_h);
  hash_combine(hash, params.kernel_w);
  hash_combine(hash, params.stride_h);
  hash_combine(hash, params.stride_w);
  hash_combine(hash, params.padding_h);
  hash_combine(hash, params.padding_w);
  hash_combine(hash, params.activation);
  hash_combine(hash, params.groups);
  hash_combine(hash, params.use_bias);

  Conv2D *conv = NULL;

  const auto &it = this->cached_conv2d_ops.find(hash);
  if (it != cached_conv2d_ops.end()) {
    conv = it->second;
  } else {
    conv = new Conv2D(*this, 
                      input, 
                      params.out_channels, 
                      params.kernel_h, params.kernel_w, 
                      params.stride_h, params.stride_w,
                      params.padding_h, params.padding_w,
                      params.activation, 
                      params.groups, 
                      params.use_bias,
                      false/*allocate_weights*/,
                      NULL);
    cached_conv2d_ops[hash] = conv;
  }

  return this->new_node(conv);
}

Node FFModel::get_or_create_conv2d_node(const Tensor input,
                                        int outChannels,
                                        int kernelH, int kernelW,
                                        int strideH, int strideW,
                                        int paddingH, int paddingW,
                                        ActiMode activation,
                                        int groups,
                                        bool use_bias) 
{
  Conv2DParams params;
  params.out_channels = outChannels;
  params.kernel_h = kernelH;
  params.kernel_w = kernelW;
  params.stride_h = strideH;
  params.stride_w = strideW;
  params.padding_h = paddingH;
  params.padding_w = paddingW;
  params.activation = activation;
  params.groups = groups;
  params.use_bias = use_bias;

  return this->get_or_create_conv2d_node(input, params);
}

void Conv2DParams::mark_replica_dims(const Tensor input,
                               ParallelDim output_dims[MAX_TENSOR_DIM], 
                               ParallelDim kernel_dims[MAX_TENSOR_DIM], 
                               ParallelDim bias_dims[MAX_TENSOR_DIM]) const 
{
  if (output_dims != nullptr) {
    output_dims[Output::REPLICA].is_replica_dim = true;
  }
  if (kernel_dims != nullptr) {
    kernel_dims[Output::REPLICA].is_replica_dim = true;
  }
  if (bias_dims != nullptr) {
    bias_dims[Bias::REPLICA_1].is_replica_dim = true;
    bias_dims[Bias::REPLICA_2].is_replica_dim = true;
    bias_dims[Bias::REPLICA_3].is_replica_dim = true;
    bias_dims[Bias::REPLICA_4].is_replica_dim = true;
  }
}

int Conv2DParams::output_size(const Tensor input, ParallelDim output_dims[MAX_TENSOR_DIM]) const {
  int input_w = input->dims[Input::WIDTH].size;
  int input_h = input->dims[Input::HEIGHT].size;

  output_dims[Output::SAMPLE].size = input->dims[Input::SAMPLE].size;
  output_dims[Output::CHANNEL].size = out_channels;
  output_dims[Output::HEIGHT].size = 1 + (input_h + 2 * padding_h - kernel_h) / stride_h;
  output_dims[Output::WIDTH].size = 1 + (input_w + 2 * padding_w - kernel_w) / stride_w;

  return input->num_dims;
};

int Conv2DParams::kernel_size(const Tensor input, ParallelDim kernel_dims[MAX_TENSOR_DIM]) const {
  kernel_dims[Kernel::CHANNEL_OUT].size = this->out_channels;
  kernel_dims[Kernel::CHANNEL_IN].size = input->dims[Input::CHANNEL].size / this->groups;
  kernel_dims[Kernel::HEIGHT].size = this->kernel_h * input->dims[Input::HEIGHT].degree;
  kernel_dims[Kernel::WIDTH].size = this->kernel_w * input->dims[Input::WIDTH].degree;

  return Kernel::NUMDIM;
}

int Conv2DParams::bias_size(const Tensor input, ParallelDim bias_dims[MAX_TENSOR_DIM]) const {
  bias_dims[Bias::CHANNEL].size = this->out_channels;

  return Bias::NUMDIM;
};

void Conv2DParams::solve_dims(const Tensor input, 
                              ParallelDim output_dims[MAX_TENSOR_DIM], int* output_ndims,
                              ParallelDim kernel_dims[MAX_TENSOR_DIM], int* kernel_ndims,  
                              ParallelDim bias_dims[MAX_TENSOR_DIM], int* bias_ndims) const 
{
  assert ((output_dims == nullptr) == (output_ndims == nullptr));
  assert ((kernel_dims == nullptr) == (kernel_ndims == nullptr));
  assert ((bias_dims == nullptr) == (bias_ndims == nullptr));

  std::vector<ParallelDimMappingRecord> mapping;
  Conv2D::construct_mappings(mapping, this->use_bias);

  this->mark_replica_dims(input, output_dims, kernel_dims, bias_dims);

  std::vector<ParallelDim *> output_dim_sets;
  if (output_dims != nullptr) {
    output_dim_sets.push_back(output_dims);
  }

  std::vector<ParallelDim *> weight_dim_sets;
  if (kernel_dims != nullptr) {
    weight_dim_sets.push_back(kernel_dims);
  }
  if (bias_dims != nullptr && this->use_bias) {
    weight_dim_sets.push_back(bias_dims);
  }

  solve_parallel_dim_mappings(
      mapping, 
      {input->dims},
      weight_dim_sets,
      output_dim_sets
  );

  if (output_dims != nullptr) {
    *output_ndims = this->output_size(input, output_dims);
  }
  if (kernel_dims != nullptr) {
    *kernel_ndims = this->kernel_size(input, kernel_dims);
  }
  if (bias_dims != nullptr && this->use_bias) {
    *bias_ndims = this->bias_size(input, bias_dims);
  }
}

/*static*/
void Conv2D::construct_mappings(std::vector<ParallelDimMappingRecord>& out, bool use_bias) {
  Conv2D::construct_output_mappings(out);
  Conv2D::construct_weight_mappings(out, use_bias);
}

/*static*/
void Conv2D::construct_output_mappings(std::vector<ParallelDimMappingRecord>& out) {
  Op::construct_output_parallel_dims(
    out, 
    {
      {Input::CHANNEL, MappingOperation::REPLICATE, Output::REPLICA},
      {Input::SAMPLE, MappingOperation::PARTITION, Output::SAMPLE},
      {Input::REPLICA, MappingOperation::PARTITION, Output::CHANNEL},
      {Input::HEIGHT, MappingOperation::PARTITION, Output::HEIGHT},
      {Input::WIDTH, MappingOperation::PARTITION, Output::WIDTH}
    }
  );
}

/*static*/
void Conv2D::construct_weight_mappings(std::vector<ParallelDimMappingRecord>& out, bool use_bias) {
  Op::construct_weight_parallel_dims(
    out,
    {
      {Input::REPLICA, MappingOperation::PARTITION, Kernel::CHANNEL_OUT},
      {Input::SAMPLE, MappingOperation::REPLICATE, Kernel::REPLICA},
      {Input::CHANNEL, MappingOperation::PARTITION, Kernel::CHANNEL_IN}, 
      {Input::HEIGHT, MappingOperation::REPLICATE, Kernel::HEIGHT}, // Kernel::{HEIGHT, WEIGHT} would both work here
      {Input::WIDTH, MappingOperation::REPLICATE, Kernel::WIDTH}, // same as above
    }, 
    Input::INDEX, Kernel::INDEX
  );

  if (use_bias) {
    Op::construct_weight_parallel_dims(
      out,
      {
        {Input::REPLICA, Bias::REPLICA_1},
        {Input::SAMPLE, Bias::REPLICA_2},
        {Input::CHANNEL, Bias::CHANNEL},
        {Input::HEIGHT, Bias::REPLICA_3},
        {Input::WIDTH, Bias::REPLICA_4}
      }, 
      Input::INDEX, Bias::INDEX
    );
  }
}

Conv2D::Conv2D(FFModel& model,
               Conv2D const &other,
               const Tensor input,
               bool allocate_weights)
: Conv2D(model, 
         input, 
         other.out_channels, 
         other.kernel_h,
         other.kernel_w,
         other.stride_h,
         other.stride_w,
         other.padding_h,
         other.padding_w,
         other.activation,
         other.groups,
         allocate_weights,
         other.use_bias,
         other.name) 
{ }

bool Conv2DParams::is_valid(const Tensor input) const {
  ParallelDim output_dims[MAX_TENSOR_DIM],
              kernel_dims[MAX_TENSOR_DIM],
              bias_dims[MAX_TENSOR_DIM];
  int output_ndims, 
      kernel_ndims,
      bias_ndims;

  this->solve_dims(
      input, 
      output_dims, &output_ndims, 
      kernel_dims, &kernel_ndims,
      bias_dims, &bias_ndims
  );

  bool is_valid = true;
  is_valid &= input->check_valid();
  is_valid &= ParallelDim::dims_are_valid(output_dims, output_ndims);
  is_valid &= ParallelDim::dims_are_valid(kernel_dims, kernel_ndims);
  if (use_bias) { 
    is_valid &= ParallelDim::dims_are_valid(bias_dims, bias_ndims);
  }

  return is_valid;
}

Conv2D::Conv2D(FFModel& model,
               const Tensor input,
               int outChannels,
               int kernelH, int kernelW,
               int strideH, int strideW, 
               int paddingH, int paddingW,
               ActiMode activation,
               int groups,
               bool allocate_weights,
               bool use_bias,
               const char* name)
: Op(model, OP_CONV2D, name, 1/*inputs*/, use_bias ? 2 : 1/*weights*/, allocate_weights, 1/*outputs*/, input),
  in_channels(input->dims[Input::CHANNEL].size),
  out_channels(outChannels),
  kernel_h(kernelH), kernel_w(kernelW),
  stride_h(strideH), stride_w(strideW),
  padding_h(paddingH), padding_w(paddingW),
  activation(activation),
  groups(groups),
  use_bias(use_bias)
{
  assert (input->num_dims == Input::NUMDIM);
  assert (this->stride_h > 0);
  assert (this->stride_w > 0);

  ParallelDim output_dims[MAX_TENSOR_DIM],
              kernel_dims[MAX_TENSOR_DIM], 
              bias_dims[MAX_TENSOR_DIM];
  int output_ndims,
      kernel_ndims,
      bias_ndims;

  this->construct_mappings(
      *this->parallel_dims_mapping, this->use_bias);
  this->get_params().solve_dims(
      this->inputs[0],
      output_dims, &output_ndims,
      kernel_dims, &kernel_ndims,
      bias_dims, &bias_ndims);

  if (allocate_weights) {
    Initializer *kernel_initializer = new GlorotUniform(std::rand()/*seed*/);

    weights[Kernel::INDEX] = model.create_weight_legion_ordering(
        kernel_ndims, kernel_dims, DT_FLOAT, NULL/*owner_op*/, true/*create_grad*/, kernel_initializer, CHOSEN_SYNC_TYPE);
    
    if (use_bias) {
      Initializer *bias_initializer = new ZeroInitializer();

      weights[Bias::INDEX] = model.create_weight_legion_ordering(
          bias_ndims, bias_dims, DT_FLOAT, NULL/*owner_op*/, true/*create_grad*/, bias_initializer, CHOSEN_SYNC_TYPE);
    }
  }

  outputs[0] = model.create_tensor_legion_ordering(output_ndims, output_dims, DT_FLOAT, this);

  assert(check_output_input_weight_parallel_dims(allocate_weights));
}

hipdnnConvolutionFwdAlgo_t
selectConvolutionForwardAlgorithm(hipdnnHandle_t handle,
                                  const hipdnnTensorDescriptor_t xDesc, const void* x,
                                  const hipdnnFilterDescriptor_t wDesc, const void* w,
                                  const hipdnnConvolutionDescriptor_t convDesc,
                                  void* workSpace, size_t workSpaceSize,
                                  const hipdnnTensorDescriptor_t yDesc, void* y);
hipdnnConvolutionBwdFilterAlgo_t
selectConvolutionBackwardFilterAlgorithm(hipdnnHandle_t handle,
                                         const hipdnnTensorDescriptor_t xDesc, const void* x,
                                         const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                         const hipdnnConvolutionDescriptor_t convDesc,
                                         void* workSpace, size_t workSpaceSize,
                                         const hipdnnFilterDescriptor_t dwDesc, void* dw);
hipdnnConvolutionBwdDataAlgo_t
selectConvolutionBackwardDataAlgorithm(hipdnnHandle_t handle,
                                       const hipdnnFilterDescriptor_t wDesc, const void* w,
                                       const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                       const hipdnnConvolutionDescriptor_t convDesc,
                                       void* workSpace, size_t workSpaceSize,
                                       const hipdnnTensorDescriptor_t dxDesc, void* dx);
/*
  regions[0]: input
  regions[1]: output
  regions[2](I): filter
  regions[3](I): bias
  regions[4](O): filter_grad
  regions[5](O): input_grad
*/
__host__
OpMeta* Conv2D::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  const Conv2D* conv = (Conv2D*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  TensorAccessorR<float, Input::NUMDIM> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, Output::NUMDIM> acc_output(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorR<float, Kernel::NUMDIM> acc_kernel(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  // TensorAccessorR<float, 1> acc_bias(
  //     regions[3], task->regions[3], FID_DATA, ctx, runtime);
  TensorAccessorW<float, Kernel::NUMDIM> acc_kernel_grad(
      regions[3], task->regions[3], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  //TensorAccessorW<float, 4> acc_input_grad(
  //    regions[4], task->regions[4], FID_DATA, ctx, runtime,
  //    false/*readOutput*/);

  Conv2DMeta* m = new Conv2DMeta(handle);
  m->relu = conv->activation == AC_MODE_RELU;
  m->use_bias = conv->use_bias;
  m->profiling = conv->profiling;
  std::strcpy(m->op_name, conv->name);

  int input_w = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int input_h = acc_input.rect.hi[1] - acc_input.rect.lo[1] + 1;
  int input_c = acc_input.rect.hi[2] - acc_input.rect.lo[2] + 1;
  int input_n = acc_input.rect.hi[3] - acc_input.rect.lo[3] + 1;
  int output_w = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int output_h = acc_output.rect.hi[1] - acc_output.rect.lo[1] + 1;
  int output_c = acc_output.rect.hi[2] - acc_output.rect.lo[2] + 1;
  int output_n = acc_output.rect.hi[3] - acc_output.rect.lo[3] + 1;

  printf("init conv (input): n(%d) c(%d) h(%d) w(%d)\n",
         input_n, input_c, input_h, input_w);
  printf("init conv (output): n(%d) c(%d) h(%d) w(%d)\n",
          output_n, output_c, output_h, output_w);
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor,
      HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
      input_n, input_c, input_h, input_w));

  checkCUDNN(hipdnnSetTensor4dDescriptor(m->biasTensor,
      HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
      1, output_c, 1, 1));

  // Require that input_c is divisible by conv->groups
  assert(input_c % conv->groups == 0);
  printf("filterDim: kernel(%d %d) c_in(%d), c_out(%d)\n",
      conv->kernel_h, conv->kernel_w, input_c / conv->groups, output_c);
  checkCUDNN(hipdnnSetFilter4dDescriptor(m->filterDesc,
      HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
      output_c, input_c / conv->groups, conv->kernel_h, conv->kernel_w));

  //printf("convDim: padding(%d %d) stride(%d %d)\n", conv->padding_h, conv->padding_w, conv->stride_h, conv->stride_w);
  int pad_h = ((output_h - 1) * conv->stride_h + conv->kernel_h - input_h + 1) / 2;
  int pad_w = ((output_w - 1) * conv->stride_w + conv->kernel_w - input_w + 1) / 2;
  if (pad_h != conv->padding_h)
    printf("Warning: changing conv_padding_h to satisfy output_h size\n");
  if (pad_w != conv->padding_w)
    printf("Warning: changing conv_padding_w to satisfy output_w size\n");

  checkCUDNN(hipdnnSetConvolution2dDescriptor(m->convDesc,
                                             pad_h,//conv->padding_h,
                                             pad_w,//conv->padding_w,
                                             conv->stride_h,
                                             conv->stride_w,
                                             1/*upscale_x*/,
                                             1/*upscale_y*/,
                                             HIPDNN_CROSS_CORRELATION,
                                             HIPDNN_DATA_FLOAT));
  if (conv->groups != 1) {
    checkCUDNN(hipdnnSetConvolutionGroupCount(m->convDesc, conv->groups));
  }

  // enable tensor core when possible
  if (m->handle.allowTensorOpMathConversion) {
    checkCUDNN(hipdnnSetConvolutionMathType(m->convDesc, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
  } else {
    checkCUDNN(hipdnnSetConvolutionMathType(m->convDesc, HIPDNN_TENSOR_OP_MATH));
  }

  int n, c, h, w;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(m->convDesc,
                                                   m->inputTensor,
                                                   m->filterDesc,
                                                   &n, &c, &h, &w));
  assert(n == output_n);
  assert(c == output_c);
  assert(h == output_h);
  assert(w == output_w);

  checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        n, c, h, w));
  // select forward algorithm
  m->fwdAlgo = selectConvolutionForwardAlgorithm(m->handle.dnn, m->inputTensor, acc_input.ptr,
                                                 m->filterDesc, acc_kernel.ptr, m->convDesc,
                                                 m->handle.workSpace, m->handle.workSpaceSize,
                                                 m->outputTensor, acc_output.ptr);
  // select backward filter algorithm
  m->bwdFilterAlgo = selectConvolutionBackwardFilterAlgorithm(
                         m->handle.dnn, m->inputTensor, acc_input.ptr,
                         m->outputTensor, acc_output.ptr,
                         m->convDesc, m->handle.workSpace, m->handle.workSpaceSize,
                         m->filterDesc, acc_kernel_grad.ptr);
  // select backward data algorithm
  m->bwdDataAlgo = selectConvolutionBackwardDataAlgorithm(
                       m->handle.dnn, m->filterDesc, acc_kernel.ptr,
                       m->outputTensor, acc_output.ptr,
                       m->convDesc, m->handle.workSpace, m->handle.workSpaceSize,
                       m->inputTensor, (float*)acc_input.ptr);
  if (m->relu) {
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, HIPDNN_ACTIVATION_RELU,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
  }
  return m;
}

void Conv2D::init(const FFModel& ff)
{
  assert(check_output_input_weight_same_parallel_is());
  parallel_is = outputs[0]->parallel_is;
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_init(ff, argmap);
  IndexLauncher launcher(CONV2D_INIT_TASK_ID, parallel_is,
                         TaskArgument(this, sizeof(Conv2D)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         outputs[0]->machine_view.hash());
  launcher.add_region_requirement(
      RegionRequirement(inputs[0]->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0]->region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0]->region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0]->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[0]->region));
  launcher.add_field(2, FID_DATA);
  // launcher.add_region_requirement(
  //     RegionRequirement(weights[1]->part, 0/*projection id*/,
  //                       READ_ONLY, EXCLUSIVE, weights[1]->region));
  // launcher.add_field(3, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0]->part_grad, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, weights[0]->region_grad));
  launcher.add_field(3, FID_DATA);
  //launcher.add_region_requirement(
  //    RegionRequirement(inputs[0]->part_grad, 0/*projection id*/,
  //                      WRITE_ONLY, EXCLUSIVE, inputs[0]->region_grad));
  //launcher.add_field(4, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  set_opmeta_from_futuremap(ff, fm);
}

/*static*/
void Conv2D::forward_kernel(const Conv2DMeta* m,
                            const float* input_ptr,
                            float* output_ptr,
                            const float* filter_ptr,
                            const float* bias_ptr)
{
  float alpha = 1.0f, beta = 0.0f;
  checkCUDNN(hipdnnConvolutionForward(m->handle.dnn, &alpha,
                                     m->inputTensor, input_ptr,
                                     m->filterDesc, filter_ptr,
                                     m->convDesc, m->fwdAlgo,
                                     m->handle.workSpace, m->handle.workSpaceSize,
                                     &beta, m->outputTensor, output_ptr));

  // use_bias == True
  if (bias_ptr != NULL) {
    checkCUDNN(hipdnnAddTensor(m->handle.dnn, &alpha, m->biasTensor,
                              bias_ptr, &alpha, m->outputTensor, output_ptr));
  }
  if (m->relu) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
                                      &alpha, m->outputTensor, output_ptr,
                                      &beta, m->outputTensor, output_ptr));
  }
}

/*
  regions[0](I): input
  regions[1](O): output
  regions[2](I): filter
  regions[3](I): bias
*/
__host__
void Conv2D::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  //Conv2D* conv = (Conv2D*) task->args;
  const Conv2DMeta* m = *((Conv2DMeta**) task->local_args);
  assert(regions.size() == (3 + int(m->use_bias)));
  assert(task->regions.size() == (3 + int(m->use_bias)));
  TensorAccessorR<float, Input::NUMDIM> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, Output::NUMDIM> acc_output(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorR<float, Kernel::NUMDIM> acc_kernel(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  const float* acc_bias_ptr = NULL;
  if (m->use_bias) { 
    TensorAccessorR<float, Bias::NUMDIM> acc_bias(
        regions[3], task->regions[3], FID_DATA, ctx, runtime);
    acc_bias_ptr = acc_bias.ptr;
  }

  //printf("fwdAlgo(%d), bwdFilterALgo(%d), bwdDataAlgo(%d)\n", (int)m->fwdAlgo,(int) m->bwdFilterAlgo,(int) m->bwdDataAlgo);
  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }

#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  Conv2D::forward_kernel(m, acc_input.ptr, acc_output.ptr, acc_kernel.ptr, acc_bias_ptr);
  if (m->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    //print_tensor<4, float>(acc_input.ptr, acc_input.rect, "[Conv2D:forward:input]");
    //print_tensor<4, float>(acc_kernel.ptr, acc_kernel.rect, "[Conv2D:forward:kernel]");
    //print_tensor<1, float>(acc_bias.ptr, acc_bias.rect, "[Conv2D:forward:bias]");
    //print_tensor<4, float>(acc_output.ptr, acc_output.rect, "[Conv2D:forward:output]");
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Conv2D] forward time (CF) = %.2fms\n", m->op_name, elapsed);
  }
}

__host__
void Conv2D::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_forward(ff, argmap);
  IndexLauncher launcher(CONV2D_FWD_TASK_ID, parallel_is,
                         TaskArgument(NULL, 0), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         outputs[0]->machine_view.hash());
  launcher.add_region_requirement(
      RegionRequirement(inputs[0]->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0]->region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0]->region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(weights[0]->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[0]->region));
  launcher.add_field(2, FID_DATA);
  if (use_bias) {
    launcher.add_region_requirement(
        RegionRequirement(weights[1]->region, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, weights[1]->region));
    launcher.add_field(3, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

/*static*/
void Conv2D::backward_kernel(const Conv2DMeta* m,
                             const float* input_ptr,
                             float* input_grad_ptr,
                             const float* output_ptr,
                             float* output_grad_ptr,
                             const float* kernel_ptr,
                             float* kernel_grad_ptr,
                             float* bias_grad_ptr)
{
  float alpha = 1.0f;
  //float beta = 0.0f;
  if (m->relu) {
    hipdnnDataType_t dataType;
    int n, c, h, w, nStride, cStride, hStride, wStride;
    checkCUDNN(hipdnnGetTensor4dDescriptor(m->outputTensor, &dataType,
        &n, &c, &h, &w, &nStride, &cStride, &hStride, &wStride));
    reluBackward<<<GET_BLOCKS(n*c*h*w), CUDA_NUM_THREADS>>>(output_grad_ptr, output_ptr, n*c*h*w);
  }
  // Compute filter gradiant
  // NOTE: we use alpha for kernel_grad to accumulate gradients
  checkCUDNN(hipdnnConvolutionBackwardFilter(m->handle.dnn, &alpha,
                                            m->inputTensor, input_ptr,
                                            m->outputTensor, output_grad_ptr,
                                            m->convDesc, m->bwdFilterAlgo,
                                            m->handle.workSpace, m->handle.workSpaceSize,
                                            &alpha, m->filterDesc, kernel_grad_ptr));
  // Compute bias gradiant
  // NOTE: we use alpha for bias_grad to accumulate gradients
  if (bias_grad_ptr != NULL) {
    checkCUDNN(hipdnnConvolutionBackwardBias(m->handle.dnn, &alpha,
                                            m->outputTensor, output_grad_ptr,
                                            &alpha, m->biasTensor, bias_grad_ptr));
                                          }
  // Compute data gradiant
  // NOTE: we use alpha for input_grad to accumulate gradients
  checkCUDNN(hipdnnConvolutionBackwardData(m->handle.dnn, &alpha,
                                          m->filterDesc, kernel_ptr,
                                          m->outputTensor, output_grad_ptr,
                                          m->convDesc, m->bwdDataAlgo,
                                          m->handle.workSpace, m->handle.workSpaceSize,
                                          &alpha, m->inputTensor, input_grad_ptr));
}

/*
  regions[0](I): input
  regions[1](I/O): input_grad
  regions[2](I): output
  regions[3](I/O): output_grad
  regions[4](I): filter
  regions[5](I/O): filter_grad
  regions[6](I/O): bias_grad
*/
__host__
void Conv2D::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  //Conv2D* conv = (Conv2D*) task->args;
  const Conv2DMeta* m = *((Conv2DMeta**) task->local_args);
  assert(regions.size() == (6 + int(m->use_bias)));
  assert(task->regions.size() == (6 + int(m->use_bias)));
  TensorAccessorR<float, Input::NUMDIM> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, Input::NUMDIM> acc_input_grad(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, Output::NUMDIM> acc_output(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  TensorAccessorW<float, Output::NUMDIM> acc_output_grad(
      regions[3], task->regions[3], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, Kernel::NUMDIM> acc_kernel(
      regions[4], task->regions[4], FID_DATA, ctx, runtime);
  TensorAccessorW<float, Kernel::NUMDIM> acc_kernel_grad(
      regions[5], task->regions[5], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  float* acc_bias_grad_ptr = NULL;
  if (m->use_bias) { 
    TensorAccessorW<float, Bias::NUMDIM> acc_bias_grad(
        regions[6], task->regions[6], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    acc_bias_grad_ptr = static_cast<float*>(acc_bias_grad.ptr);
  }
  

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }

#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  Conv2D::backward_kernel(m, acc_input.ptr, acc_input_grad.ptr,
                          acc_output.ptr, acc_output_grad.ptr,
                          acc_kernel.ptr, acc_kernel_grad.ptr,
                          acc_bias_grad_ptr);
  if (m->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Conv2D] backward time = %.2fms\n", m->op_name, elapsed);
    //print_tensor<4, float>(acc_output_grad.ptr, acc_output_grad.rect, "[Conv2D:backward:output_grad]");
    //print_tensor<4, float>(acc_kernel_grad.ptr, acc_kernel_grad.rect, "[Conv2D:backward:kernel_grad]");
    //print_tensor<1, float>(acc_bias_grad.ptr, acc_bias_grad.rect, "[Conv2D:backward:bias_grad]");
    //print_tensor<4, float>(acc_input_grad.ptr, acc_input_grad.rect, "[Conv2D:backward:input_grad]");
  }
}

__host__
void Conv2D::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  set_argumentmap_for_backward(ff, argmap);
  IndexLauncher launcher(CONV2D_BWD_TASK_ID, parallel_is,
                         TaskArgument(NULL, 0), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         outputs[0]->machine_view.hash());
  // regions[0](I): input
  launcher.add_region_requirement(
      RegionRequirement(inputs[0]->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0]->region));
  launcher.add_field(0, FID_DATA);
  // regions[1](I/O): input_grad
  launcher.add_region_requirement(
      RegionRequirement(inputs[0]->part_grad, 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, inputs[0]->region_grad));
  launcher.add_field(1, FID_DATA);
  // regions[2](I): output
  launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, outputs[0]->region));
  launcher.add_field(2, FID_DATA);
  // regions[3](I/O): output_grad
  launcher.add_region_requirement(
      RegionRequirement(outputs[0]->part_grad, 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, outputs[0]->region_grad));
  launcher.add_field(3, FID_DATA);
  // regions[4](I): filter
  launcher.add_region_requirement(
      RegionRequirement(weights[0]->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, weights[0]->region));
  launcher.add_field(4, FID_DATA);
  // regions[5](I/O): filter_grad
  launcher.add_region_requirement(
      RegionRequirement(weights[0]->part_grad, 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, weights[0]->region_grad));
  launcher.add_field(5, FID_DATA);
  if (use_bias) {
    // regions[6](I/O): bias_grad
    launcher.add_region_requirement(
        RegionRequirement(weights[1]->part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, weights[1]->region_grad));
    launcher.add_field(6, FID_DATA);
  }
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  // TODO: remove this line
  //if (first_layer)
    //fm.wait_all_results();
}

__host__
void Conv2D::print_layer(const FFModel& ff)
{
  printf("conv2d layer\n");
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
#if 0
  TaskLauncher launcher(CONV2D_PRINT_TASK_ID, TaskArgument(NULL, 0));
  launcher.add_region_requirement(
    RegionRequirement(kernel->region, READ_ONLY, EXCLUSIVE, kernel->region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
    RegionRequirement(bias->region, READ_ONLY, EXCLUSIVE, bias->region));
  launcher.add_field(1, FID_DATA);
  Future fu = runtime->execute_task(ctx, launcher);
  fu.wait();
#else
  RegionRequirement kernel_req(weights[0]->region, READ_WRITE, EXCLUSIVE, weights[0]->region);
  kernel_req.add_field(FID_DATA);
  InlineLauncher kernel_launcher(kernel_req);
  PhysicalRegion kernel_region = runtime->map_region(ctx, kernel_launcher);
  kernel_region.wait_until_valid();

/*
  RegionRequirement kernel_grad_req(kernel->region_grad, READ_WRITE, EXCLUSIVE, kernel->region_grad);
  kernel_grad_req.add_field(FID_DATA);
  InlineLauncher kernel_grad_launcher(kernel_grad_req);
  PhysicalRegion kernel_grad_region = runtime->map_region(ctx, kernel_grad_launcher);
  kernel_grad_region.wait_until_valid();
*/
  RegionRequirement bias_req(weights[1]->region, READ_WRITE, EXCLUSIVE, weights[1]->region);
  bias_req.add_field(FID_DATA);
  InlineLauncher bias_launcher(bias_req);
  PhysicalRegion bias_region = runtime->map_region(ctx, bias_launcher);
  bias_region.wait_until_valid();
/*
  RegionRequirement bias_grad_req(bias->region_grad, READ_WRITE, EXCLUSIVE, bias->region_grad);
  bias_grad_req.add_field(FID_DATA);
  InlineLauncher bias_grad_launcher(bias_grad_req);
  PhysicalRegion bias_grad_region = runtime->map_region(ctx, bias_grad_launcher);
  bias_grad_region.wait_until_valid();
  */
  TensorAccessorW<float, Kernel::NUMDIM> acc_kernel(kernel_region, kernel_req, FID_DATA, ctx, runtime, true);
//  const AccessorRW<float, 1> acc_kernel_grad(kernel_grad_region, FID_DATA);
  TensorAccessorW<float, Bias::NUMDIM> acc_bias(bias_region, bias_req, FID_DATA, ctx, runtime, true);
  //const AccessorRW<float, 1> acc_bias_grad(bias_grad_region, FID_DATA);

  const float *kernel_ptr = acc_kernel.ptr;
  //float *kernel_grad_ptr = acc_kernel_grad.ptr;
  const float *bias_ptr = acc_bias.ptr;
  //float *bias_grad_ptr = acc_bias_grad.ptr;

  size_t kernel_size = acc_kernel.rect.volume();
  int kernel_dim1 = acc_kernel.rect.hi[0] - acc_kernel.rect.lo[0] + 1;
  int kernel_dim2 = acc_kernel.rect.hi[1] - acc_kernel.rect.lo[1] + 1;
  int kernel_dim3 = acc_kernel.rect.hi[2] - acc_kernel.rect.lo[2] + 1;
  int kernel_dim4 = acc_kernel.rect.hi[3] - acc_kernel.rect.lo[3] + 1;
  //size_t kernel_grad_size = rect_kernel_grad.volume();
  size_t bias_size = acc_bias.rect.volume();
  //size_t bias_grad_size = rect_bias_grad.volume();
  printf("kernel, %p, %zu, [%d, %d, %d, %d]\n", kernel_ptr, kernel_size, kernel_dim1, kernel_dim2, kernel_dim3, kernel_dim4);
  //printf("kernel_grad, %d\n", kernel_grad_size);
  printf("bias, %p, %zu\n", bias_ptr, bias_size);
  //printf("bias_grad, %d\n", bias_grad_size);


  for (int i = 0; i < bias_size; i++) {
    printf("%f ", bias_ptr[i]);
  }
  printf("\n");

/*
  for (int i = 0; i < bias_grad_size; i++) {
    printf("%f ", bias_grad_ptr);
    bias_grad_ptr ++;
  }
  printf("\n");*/

  for (int i = 0; i < kernel_size; i++) {
    printf("%f ", kernel_ptr[i]);
  }
  printf("\n");

/*
  for (int i = 0; i < kernel_grad_size; i++) {
    printf("%f ", kernel_grad_ptr);
    kernel_grad_ptr ++;
  }
  printf("\n");
  */
  runtime->unmap_region(ctx, kernel_region);
 // runtime->unmap_region(ctx, kernel_grad_region);
  runtime->unmap_region(ctx, bias_region);
//  runtime->unmap_region(ctx, bias_grad_region);
#endif
}

hipdnnConvolutionFwdAlgo_t
selectConvolutionForwardAlgorithm(hipdnnHandle_t handle,
                                  const hipdnnTensorDescriptor_t xDesc, const void* x,
                                  const hipdnnFilterDescriptor_t wDesc, const void* w,
                                  const hipdnnConvolutionDescriptor_t convDesc,
                                  void* workSpace, size_t workSpaceSize,
                                  const hipdnnTensorDescriptor_t yDesc, void* y)
{
  const int reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionFwdAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionForwardAlgorithmEx(
      handle, xDesc, x, wDesc, w, convDesc, yDesc, y,
      reqAlgCnt, &cnt, perfResults, workSpace, workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("forwardAlgo(%d) time(%.2lf)\n", perfResults[0].algo, perfResults[0].time);
  return perfResults[0].algo;
}

hipdnnConvolutionBwdFilterAlgo_t
selectConvolutionBackwardFilterAlgorithm(hipdnnHandle_t handle,
                                         const hipdnnTensorDescriptor_t xDesc, const void* x,
                                         const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                         const hipdnnConvolutionDescriptor_t convDesc,
                                         void* workSpace, size_t workSpaceSize,
                                         const hipdnnFilterDescriptor_t dwDesc, void* dw)
{
  const int reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionBwdFilterAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionBackwardFilterAlgorithmEx(
      handle, xDesc, x, dyDesc, dy, convDesc, dwDesc, dw,
      reqAlgCnt, &cnt, perfResults, workSpace, workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("bwdFilterAlgo(%d) time(%.2lf)\n", perfResults[0].algo, perfResults[0].time);
  return perfResults[0].algo;
}

hipdnnConvolutionBwdDataAlgo_t
selectConvolutionBackwardDataAlgorithm(hipdnnHandle_t handle,
                                       const hipdnnFilterDescriptor_t wDesc, const void* w,
                                       const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                       const hipdnnConvolutionDescriptor_t convDesc,
                                       void* workSpace, size_t workSpaceSize,
                                       const hipdnnTensorDescriptor_t dxDesc, void* dx)
{
  const int reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionBwdDataAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionBackwardDataAlgorithmEx(
      handle, wDesc, w, dyDesc, dy, convDesc, dxDesc, dx,
      reqAlgCnt, &cnt, perfResults, workSpace, workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("bwdDataAlgo(%d) time(%.2lf)\n", perfResults[0].algo, perfResults[0].time);
  return perfResults[0].algo;
}

Conv2DMeta::Conv2DMeta(FFHandler handler)
: OpMeta(handler)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
}

bool Conv2D::measure_operator_cost(Simulator* sim,
                                   const ParallelConfig& pc,
                                   CostMetrics& cost_metrics) const
{
  TensorBase sub_output, sub_input;
  if(!outputs[0]->get_output_sub_tensor(pc, sub_output, OP_CONV2D))
    return false;
  if(!inputs[0]->get_input_sub_tensor(pc, sub_input, OP_CONV2D))
    return false;
  int input_w = sub_input.dims[0].size;
  int input_h = sub_input.dims[1].size;
  int input_c = sub_input.dims[2].size;
  int input_n = sub_input.dims[3].size;
  int output_w = sub_output.dims[0].size;
  int output_h = sub_output.dims[1].size;
  int output_c = sub_output.dims[2].size;
  int output_n = sub_output.dims[3].size;
  int pad_h = ((output_h - 1) * stride_h + kernel_h - input_h + 1) / 2;
  int pad_w = ((output_w - 1) * stride_w + kernel_w - input_w + 1) / 2;

  Conv2DMeta* m = sim->conv2d_meta;
  m->relu = activation == AC_MODE_RELU;
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, input_n, input_c, input_h, input_w));
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->biasTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, 1, output_c, 1, 1));
  // require input_c is divisible by groups
  assert(input_c % groups == 0);
  checkCUDNN(hipdnnSetFilter4dDescriptor(m->filterDesc, HIPDNN_DATA_FLOAT,
      HIPDNN_TENSOR_NCHW, output_c, input_c / groups, kernel_h, kernel_w));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(m->convDesc, pad_h, pad_w,
      stride_h, stride_w, 1/*dilationH*/, 1/*dilationW*/,
      HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

  checkCUDNN(hipdnnSetConvolutionGroupCount(m->convDesc, groups));
  if (m->handle.allowTensorOpMathConversion) {
    checkCUDNN(hipdnnSetConvolutionMathType(m->convDesc, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
  } else {
    checkCUDNN(hipdnnSetConvolutionMathType(m->convDesc, HIPDNN_TENSOR_OP_MATH));
  }
  int n, c, h, w;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(m->convDesc,
      m->inputTensor, m->filterDesc, &n, &c, &h, &w));
  assert(n == output_n);
  assert(c == output_c);
  assert(h == output_h);
  assert(w == output_w);
  checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, HIPDNN_ACTIVATION_RELU,
      HIPDNN_NOT_PROPAGATE_NAN, 0.0));
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, n, c, h, w));
  // allocate tensors in simulator
  sim->free_all();
  float* input_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert(input_ptr != NULL);
  float *output_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert(output_ptr != NULL);
  float* weight_ptr = (float*)sim->allocate((size_t)output_c * input_c * kernel_h * kernel_w / groups, DT_FLOAT);
  assert(weight_ptr != NULL);
  float* bias_ptr = (float*)sim->allocate(output_c, DT_FLOAT);
  assert(bias_ptr != NULL);

  // compute memory usage
  // Assume:
  //   1. all memory allocations use Simulator::allocate
  //   2. we call Simulator::free_all before measure an operator
  // Therefore, the memory usage of an operator is sim->offset
  cost_metrics.memory_requirement = (size_t)sim->offset;

  // select forward algorithm
  {
    const int reqAlgCnt = 8;
    int cnt = 0;
    hipdnnConvolutionFwdAlgoPerf_t perfResults[reqAlgCnt];
    checkCUDNN(hipdnnFindConvolutionForwardAlgorithmEx(
        m->handle.dnn, m->inputTensor, input_ptr,
        m->filterDesc, weight_ptr, m->convDesc, m->outputTensor, output_ptr,
        reqAlgCnt, &cnt, perfResults,
        m->handle.workSpace, m->handle.workSpaceSize));
    assert(cnt > 0);
    checkCUDNN(perfResults[0].status);
    cost_metrics.forward_time = perfResults[0].time;
    //for (int i = 0; i < cnt; i++)
    //  printf("conv forward: algo(%d) time(%.4lf)\n", perfResults[i].algo, perfResults[i].time);
  }
  // select forward algorithm
  {
    const int reqAlgCnt = 8;
    int cnt = 0;
    hipdnnConvolutionBwdFilterAlgoPerf_t perfResults[reqAlgCnt];
    checkCUDNN(hipdnnFindConvolutionBackwardFilterAlgorithmEx(
        m->handle.dnn, m->inputTensor, input_ptr,
        m->outputTensor, output_ptr, m->convDesc, m->filterDesc, weight_ptr,
        reqAlgCnt, &cnt, perfResults,
        m->handle.workSpace, m->handle.workSpaceSize));
    assert(cnt > 0);
    checkCUDNN(perfResults[0].status);
    cost_metrics.backward_time = perfResults[0].time;
  }
  {
    const int reqAlgCnt = 8;
    int cnt = 0;
    hipdnnConvolutionBwdDataAlgoPerf_t perfResults[reqAlgCnt];
    checkCUDNN(hipdnnFindConvolutionBackwardDataAlgorithmEx(
        m->handle.dnn, m->filterDesc, weight_ptr,
        m->outputTensor, output_ptr, m->convDesc, m->inputTensor, input_ptr,
        reqAlgCnt, &cnt, perfResults,
        m->handle.workSpace, m->handle.workSpaceSize));
    assert(cnt > 0);
    checkCUDNN(perfResults[0].status);
    cost_metrics.backward_time += perfResults[0].time;
  }
  printf("[Measure Conv2D] name(%s) input(%d %d %d %d) weight(%d %d %d %d) output(%d %d %d %d) stride(%d %d) padding(%d %d) forward_time(%.4lf) backward_time(%.4lf)\n",
         name,
         input_n, input_c, input_h, input_w,
         output_c, input_c / groups, kernel_h, kernel_w,
         output_n, output_c, output_h, output_w,
         stride_h, stride_w,
         padding_h, padding_w,
         cost_metrics.forward_time, cost_metrics.backward_time);
  return true;
}

bool Conv2D::estimate_sync_cost(Simulator* sim, 
                                const MachineView& view,
                                CostMetrics& cost_metrics) const 
{
  ParallelDim kernel_dims[MAX_TENSOR_DIM],
              bias_dims[MAX_TENSOR_DIM];
  int kernel_ndims,
      bias_ndims;
  
  this->get_params().solve_dims(this->inputs[0], 
                                nullptr, nullptr,
                                kernel_dims, &kernel_ndims,
                                bias_dims, &bias_ndims);

  cost_metrics.sync_time = sim->default_estimate_sync_cost(kernel_dims, kernel_ndims, view);

  if (this->use_bias) {
    cost_metrics.sync_time += sim->default_estimate_sync_cost(bias_dims, bias_ndims, view);
  }

  return true;
}
