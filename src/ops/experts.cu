#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/experts.h"
#include "flexflow/utils/cuda_helper.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

// Thrust-related headers
#define THRUST_IGNORE_DEPRECATED_CPP_DIALECT 1
#include <thrust/binary_search.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/unique.h>

namespace FlexFlow {

/* __global__ void experts_forward_prepare_kernel(
    int num_experts,
    int num_tokens,
    int num_chosen_experts,
    int out_dim,
    int in_dim,

    int const *indices, // @In: Indices for chosen experts (num_chosen_experts,
                        // batch_size)

    float const **weights,    // @In: Experts' weights
    float **weight_idx_array, // @Out: Aarray for GemmBatchedEx

    float const *input,       // @In: Tokens' values (in_dim, batch_size)
    float **token_idx_arrary, // @Out: Barray for GemmBatchedEx

    float const *gemm_results, // @In: GemmBatchedEx's result tensor (out_dim,
                               // batch_size)
    float **result_idx_arrary, // @Out: Carray for GemmBatchedEx

    float const *coefficients, // @In: topk_gate_predss coefficients tensor
                               // (num_chosen_experts, batch_size)
    float **dev_coefficient_idx_array, // @Out: Barray for Aggregation

    float const *output, // @In: experts ops output tensor (out_dim, batch_size)
    float **dev_output_idx_array // @Out: Carray for Aggregation
) {
  // Initialize Aarray and Barray for Cublas GemmBatchedEx
  CUDA_KERNEL_LOOP(i, num_tokens * num_chosen_experts) {
    int expert_index = indices[i];
    // int token_index = i / num_chosen_experts;   // if indices are row
    // majored, in shape (batch_size, num_chosen_experts)
    int token_index = i % num_tokens; // if indices are column majored, in shape
                                      // (num_chosen_experts, batch_size)
    if (expert_index && expert_index < num_experts) {
      weight_idx_array[i] = const_cast<float *>(
          weights[expert_index]); // <--- FIX: this line has issue with memory
                                  // access
      token_idx_arrary[i] =
          const_cast<float *>(input + token_index * in_dim * sizeof(float));
      result_idx_arrary[i] =
          const_cast<float *>(gemm_results + i * out_dim * sizeof(float));

      // prepare aggreation required ptr array
      dev_coefficient_idx_array[i] =
          const_cast<float *>(coefficients + i * sizeof(float));
      dev_output_idx_array[i] =
          const_cast<float *>(output + token_index * in_dim * sizeof(float));
    }
  }
}
 */
__global__ void experts_forward_prepare_kernel(
    int num_valid_assignments,
    int expert_capacity,
    int lb_index,
    int experts_start_idx,
    int num_experts_per_block,
    int num_chosen_experts,
    thrust::device_ptr<int> sorted_indices,
    thrust::device_ptr<int> expert_start_indexes,
    thrust::device_ptr<int> exp_local_label_to_index,
    thrust::device_ptr<int> destination_start_indices,
    thrust::device_ptr<int> original_indices,
    float const *input,             // @In: Tokens' values (in_dim, batch_size)
    float const **token_idx_array,  // @Out: Barray for GemmBatchedEx
    float const **weights,          // @In: Experts' weights
    float const **weight_idx_array, // @Out: Aarray for GemmBatchedEx
    float const *coefficients,      // @In: topk_gate_predss coefficients tensor
                                    // (num_chosen_experts, batch_size)
    float const **coefficient_idx_array // @Out: Barray for Aggregation
) {
  CUDA_KERNEL_LOOP(i, num_valid_assignments) {
    int global_expert_label = sorted_indices[lb_index + i];
    assert(global_expert_label >= experts_start_idx &&
           global_expert_label < experts_start_idx + num_experts_per_block);
    int local_expert_label = global_expert_label - experts_start_idx;

    int expert_index = exp_local_label_to_index[local_expert_label];
    int within_expert_offset = i - expert_start_indexes[expert_index];
    if (within_expert_offset < expert_capacity) {
      token_idx_array[destination_start_indices[expert_index] +
                      within_expert_offset] =
          &input[original_indices[i + lb_index] / num_chosen_experts];
      weight_idx_array[destination_start_indices[expert_index] +
                       within_expert_offset] = weights[local_expert_label];
      coefficient_idx_array[destination_start_indices[expert_index] +
                            within_expert_offset] =
          &coefficients[original_indices[i + lb_index]];
    }
  }
}

struct is_less_than_capacity {
  int _expert_capacity;
  is_less_than_capacity(int expert_capacity)
      : _expert_capacity(expert_capacity){};
  __host__ __device__ bool operator()(int x) {
    return x <= _expert_capacity;
  }
};

/*static*/
void Experts::forward_kernel_wrapper(ExpertsMeta const *m,
                                     float const *input,
                                     int const *indices,
                                     float const *topk_gate_preds,
                                     float *output,
                                     float const **weights,
                                     int chosen_experts,
                                     int batch_size,
                                     int out_dim) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  int num_experts_per_block = m->num_experts;
  int experts_start_idx = m->experts_start_idx;
  // bool use_bias = m->use_bias;
  // ActiMode activation = m->activation;
  // int data_dim = m->data_dim;
  int num_chosen_experts = m->num_chosen_experts;
  int num_tokens = m->effective_batch_size;
  int expert_capacity = m->expert_capacity;

  assert(chosen_experts == num_chosen_experts);
  assert(num_tokens == batch_size);

  int num_indices = num_tokens * num_chosen_experts;
  // sort the indices and coefficients by expert. Keep track of the original
  // position of each index/coefficient using the original_indices array
  thrust::device_ptr<int const> thrust_indices =
      thrust::device_pointer_cast(indices);
  thrust::device_ptr<int> sorted_indices =
      thrust::device_pointer_cast(m->sorted_indices);
  thrust::copy(thrust::device,
               thrust_indices,
               thrust_indices + num_indices,
               sorted_indices);
  thrust::device_ptr<int> original_indices =
      thrust::device_pointer_cast(m->original_indices);
  thrust::sequence(
      thrust::device, original_indices, original_indices + num_indices);
  thrust::stable_sort_by_key(thrust::device,
                             sorted_indices,
                             sorted_indices + num_indices,
                             original_indices);

  // get lower and upper bound of indices corresponding to experts in the block
  thrust::device_ptr<int> lb = thrust::lower_bound(
      sorted_indices, sorted_indices + num_indices, experts_start_idx);
  thrust::device_ptr<int> ub =
      thrust::upper_bound(sorted_indices,
                          sorted_indices + num_indices,
                          experts_start_idx + num_experts_per_block);
  int lb_index = lb - sorted_indices;
  int ub_index = ub - sorted_indices;
  int num_valid_assignments = ub_index - lb_index;
  if (num_valid_assignments == 0) {
    return;
  }
  thrust::device_ptr<float const> thrust_inputs =
      thrust::device_pointer_cast(input);
  /* for (int i=0; i<num_tokens; i++) {
    std::cout << "Token " << i << ":\t";
    thrust::copy_n(thrust_inputs, data_dim,
    std::ostream_iterator<int>(std::cout, ",")); std::cout << std::endl;
  } */
  // create "exp_local_label_to_index", a mapping from local expert label to its
  // non-zero expert index
  thrust::device_ptr<int> non_zero_expert_labels =
      thrust::device_pointer_cast(m->non_zero_expert_labels);
  thrust::device_ptr<int> non_zero_expert_labels_end =
      thrust::unique_copy(lb, ub, non_zero_expert_labels);
  int non_zero_experts_count =
      non_zero_expert_labels_end - non_zero_expert_labels;
  using namespace thrust::placeholders;
  thrust::for_each(thrust::device,
                   non_zero_expert_labels,
                   non_zero_expert_labels + non_zero_experts_count,
                   _1 -=
                   experts_start_idx); // convert global indexes to local ones
  thrust::device_ptr<int> temp_sequence =
      thrust::device_pointer_cast(m->temp_sequence);
  thrust::sequence(
      thrust::device, temp_sequence, temp_sequence + non_zero_experts_count);
  thrust::device_ptr<int> exp_local_label_to_index =
      thrust::device_pointer_cast(m->exp_local_label_to_index);
  thrust::scatter(thrust::device,
                  temp_sequence,
                  temp_sequence + non_zero_experts_count,
                  non_zero_expert_labels,
                  exp_local_label_to_index);

  // get local start index (within lower/upper bound) for each expert receiving
  // non-zero tokens
  thrust::device_ptr<int> expert_start_indexes =
      thrust::device_pointer_cast(m->expert_start_indexes);
  thrust::sequence(thrust::device,
                   expert_start_indexes,
                   expert_start_indexes + non_zero_experts_count);
  int start_indexes =
      (thrust::unique_by_key(lb, ub, expert_start_indexes)).first - lb;
  assert(start_indexes == non_zero_experts_count);

  // get number of token assignment to each expert
  thrust::device_ptr<int> num_assignments_per_expert =
      thrust::device_pointer_cast(m->num_assignments_per_expert);
  thrust::transform(expert_start_indexes + 1,
                    expert_start_indexes + non_zero_experts_count,
                    num_assignments_per_expert,
                    expert_start_indexes,
                    thrust::minus<int>());

  // build destination_start_index array, telling us the first slot that belongs
  // to each expert in the destination array (after factoring in expert
  // capacity)
  thrust::device_ptr<int> destination_start_indices =
      thrust::device_pointer_cast(m->destination_start_indices);
  thrust::replace_copy_if(thrust::device,
                          num_assignments_per_expert,
                          num_assignments_per_expert + non_zero_experts_count,
                          destination_start_indices,
                          is_less_than_capacity(expert_capacity),
                          expert_capacity);
  thrust::exclusive_scan(thrust::device,
                         destination_start_indices,
                         destination_start_indices + non_zero_experts_count,
                         destination_start_indices,
                         0);
  experts_forward_prepare_kernel<<<GET_BLOCKS(num_valid_assignments),
                                   min(CUDA_NUM_THREADS,
                                       (int)num_valid_assignments),
                                   0,
                                   stream>>>(num_valid_assignments,
                                             expert_capacity,
                                             lb_index,
                                             experts_start_idx,
                                             num_experts_per_block,
                                             num_chosen_experts,
                                             sorted_indices,
                                             expert_start_indexes,
                                             exp_local_label_to_index,
                                             destination_start_indices,
                                             original_indices,
                                             input,
                                             m->token_idx_array,
                                             m->dev_weights,
                                             m->weight_idx_array,
                                             topk_gate_preds,
                                             m->coefficient_idx_array);

  // Batched Gemm Excution for every chosen_expert-token pairs
  /* experts_forward_GemmBatched_kernel(handle,
                                     m->dev_weight_idx_array,
                                     m->dev_token_idx_arrary,
                                     m->dev_result_idx_array,
                                     //  bias_ptr,
                                     data_dim,
                                     out_dim,
                                     num_tokens,
                                     num_chosen_experts,
                                     stream);

  experts_forward_aggregate_kernel(handle,
                                   m->dev_result_idx_array,
                                   m->dev_coefficient_idx_array,
                                   m->dev_output_idx_array,
                                   out_dim,
                                   num_tokens,
                                   num_chosen_experts,
                                   stream); */

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("[Experts] forward time = %.2lfms\n", elapsed);
  }
}

void experts_forward_GemmBatched_kernel(ExpertsMeta const *m,
                                        float **weight_ptr,
                                        float **input_ptr,
                                        float **output_ptr,
                                        //  void const *bias_ptr,
                                        int in_dim,
                                        int out_dim,
                                        int num_tokens,
                                        int num_chosen_experts,
                                        ffStream_t stream) {

  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha = 1.0f, beta = 0.0f;

  // hipDataType input_type = ff_to_cuda_datatype(m->input_type);
  // hipDataType weight_type = ff_to_cuda_datatype(m->weight_type);
  // hipDataType output_type = ff_to_cuda_datatype(m->output_type);
  hipDataType input_type = HIP_R_32F;
  hipDataType weight_type = HIP_R_32F;
  hipDataType output_type = HIP_R_32F;

  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

  hipblasGemmBatchedEx(
      m->handle.blas,
      HIPBLAS_OP_T, // Tranpose Weight, shape (in_dim, out_dim) => (out_dim,
                   // in_dim)
      HIPBLAS_OP_N, // Input_token, shape (in_dim, 1)
      out_dim,     // num_row of (A, C) = out_dim
      1,           // num_col of (B, C) = 1
      in_dim,      // num_col of A and num_rows of B = in_dim
      &alpha,
      (void const **)
          weight_ptr, // Aarray (num_tokens * chosen_experts, in_dim, out_dim)
      weight_type,
      in_dim, // Leading Dimension of weight before transpose
      (void const **)
          input_ptr, // Barray (num_tokens * chosen_experts, in_dim, 1)
      input_type,
      in_dim, // Leading Dimension of input_token
      &beta,
      (void **)output_ptr, // Carray (num_tokens * chosen_experts, out_dim, 1)
      output_type,
      out_dim,                         // Leading Dimension of output
      num_tokens * num_chosen_experts, // Total submatrixs
      compute_type,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

void experts_forward_aggregate_kernel(hipblasHandle_t const handle,
                                      float **results_ptr,
                                      float **coefficient_ptr,
                                      float **output_ptr,
                                      //  int in_dim,
                                      int out_dim,
                                      int num_tokens,
                                      int num_chosen_experts,
                                      ffStream_t stream) {

  float alpha = 1.0f, beta = 0.0f;

  hipDataType result_type = HIP_R_32F;
  hipDataType coefficient_type = HIP_R_32F;
  hipDataType output_type = HIP_R_32F;

  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

  hipblasGemmBatchedEx(
      handle,
      HIPBLAS_OP_N, // Intermediate reulsts, shape (out_dim, 1)
      HIPBLAS_OP_N, // Coefficient, shape (1, 1)
      out_dim,     // num_row of (A, C) = out_dim
      1,           // num_col of (B, C) = 1
      1,           // num_col of A and num_rows of B = in_dim
      &alpha,
      (void const **)
          results_ptr, // Aarray (num_tokens * chosen_experts, out_dim, 1)
      result_type,
      out_dim, // Leading Dimension of result tensor
      (void const **)
          coefficient_ptr, // Barray (num_tokens * chosen_experts, 1, 1)
      coefficient_type,
      1, // Leading Dimension of coefficient tensor
      &beta,
      (void **)output_ptr, // Carray (num_tokens * chosen_experts, out_dim, 1)
      output_type,
      out_dim,                         // Leading Dimension of output
      num_tokens * num_chosen_experts, // Total submatrixs
      compute_type,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

ExpertsMeta::ExpertsMeta(FFHandler handler,
                         int _num_experts,
                         int _experts_start_idx,
                         int _data_dim,
                         int _out_dim,
                         int _effective_batch_size,
                         int _num_chosen_experts,
                         float _alpha,
                         bool _use_bias,
                         ActiMode _activation)
    : OpMeta(handler), num_experts(_num_experts),
      experts_start_idx(_experts_start_idx), data_dim(_data_dim),
      out_dim(_out_dim), effective_batch_size(_effective_batch_size),
      num_chosen_experts(_num_chosen_experts), alpha(_alpha),
      use_bias(_use_bias), activation(_activation) {
  expert_capacity =
      ceil(alpha * num_chosen_experts / num_experts * effective_batch_size);

  checkCUDA(
      hipMalloc(&sorted_indices,
                 num_chosen_experts * effective_batch_size * sizeof(int)));
  checkCUDA(
      hipMalloc(&original_indices,
                 num_chosen_experts * effective_batch_size * sizeof(int)));
  checkCUDA(hipMalloc(&non_zero_expert_labels, num_experts * sizeof(int)));
  checkCUDA(hipMalloc(&temp_sequence, num_experts * sizeof(int)));
  checkCUDA(hipMalloc(&exp_local_label_to_index, num_experts * sizeof(int)));
  checkCUDA(hipMalloc(&expert_start_indexes, num_experts * sizeof(int)));
  checkCUDA(hipMalloc(&num_assignments_per_expert, num_experts * sizeof(int)));
  checkCUDA(hipMalloc(&destination_start_indices, num_experts * sizeof(int)));
  checkCUDA(
      hipMalloc(&token_idx_array,
                 num_chosen_experts * effective_batch_size * sizeof(float *)));
  checkCUDA(hipMalloc(&dev_weights, num_experts * sizeof(float *)));
  checkCUDA(
      hipMalloc(&weight_idx_array,
                 num_chosen_experts * effective_batch_size * sizeof(float *)));
  checkCUDA(
      hipMalloc(&coefficient_idx_array,
                 num_chosen_experts * effective_batch_size * sizeof(float *)));
}
ExpertsMeta::~ExpertsMeta(void) {

  checkCUDA(hipFree(&sorted_indices));
  checkCUDA(hipFree(&original_indices));
  checkCUDA(hipFree(&non_zero_expert_labels));
  checkCUDA(hipFree(&temp_sequence));
  checkCUDA(hipFree(&exp_local_label_to_index));
  checkCUDA(hipFree(&expert_start_indexes));
  checkCUDA(hipFree(&num_assignments_per_expert));
  checkCUDA(hipFree(&destination_start_indices));
  checkCUDA(hipFree(&token_idx_array));
  checkCUDA(hipFree(&dev_weights));
  checkCUDA(hipFree(&weight_idx_array));
  checkCUDA(hipFree(&coefficient_idx_array));
}

}; // namespace FlexFlow
