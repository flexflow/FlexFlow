#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#if defined(FF_USE_CUDA) || defined(FF_USE_HIP_CUDA)
#include "hip/hip_complex.h"
#endif
#include "flexflow/ffconst_utils.h"
#include "flexflow/ops/kernels/inc_multihead_self_attention_kernels.h"
#include "flexflow/ops/tree_inc_multihead_self_attention.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

// declare Legion names
using Legion::coord_t;
using Legion::Memory;

using namespace Kernels::IncMultiHeadAttention;

namespace Kernels {
namespace TreeIncMultiHeadAttention {

template <typename DT>
__global__ void commit_tokens_kernel(
    DT const *devQKVProjArray,
    DT *cache_ptr,
    TreeVerifyBatchConfig::CommittedTokensInfo const *committedTokenInfos,
    int qProjSize,
    int kProjSize,
    int vProjSize,
    int num_tokens_to_commit,
    int num_active_tokens_in_last_batch,
    int num_heads,
    int max_seq_len,
    bool k_cache) {

  CUDA_KERNEL_LOOP(
      i, num_tokens_to_commit * (k_cache ? kProjSize : vProjSize) * num_heads) {
    int proj_size = k_cache ? kProjSize : vProjSize;
    int data_idx = i % proj_size;
    int head_idx = i / (num_tokens_to_commit * proj_size);
    int token_pos =
        (i - head_idx * (num_tokens_to_commit * proj_size)) / proj_size;
    int token_idx_in_last_batch = committedTokenInfos[token_pos].token_index;
    assert(token_idx_in_last_batch < num_active_tokens_in_last_batch);

    int qkv_block_size =
        (qProjSize + kProjSize + vProjSize) * num_active_tokens_in_last_batch;
    int current_head_block_size = num_active_tokens_in_last_batch *
                                  (k_cache ? qProjSize : qProjSize + kProjSize);
    DT val =
        devQKVProjArray[head_idx * qkv_block_size + current_head_block_size +
                        token_idx_in_last_batch * proj_size + data_idx];
    // int const req_id = id_map[token_idx].request_index;
    // int const tok_id = id_map[token_idx].token_position;
    int const req_id = committedTokenInfos[token_pos].request_index;
    int const tok_id = committedTokenInfos[token_pos].token_depth;

    cache_ptr[req_id * (num_heads * max_seq_len * proj_size) +
              head_idx * (max_seq_len * proj_size) + tok_id * proj_size +
              data_idx] = val;
  }
}

template <typename DT>
void commit_tokens(TreeIncMultiHeadSelfAttentionMeta const *m,
                   TreeVerifyBatchConfig const *bc,
                   hipStream_t stream) {
  int num_tokens_to_commit = bc->num_tokens_to_commit;
  if (num_tokens_to_commit > 0) {
    int parallelism = m->kProjSize * num_tokens_to_commit * m->num_heads;
    commit_tokens_kernel<<<GET_BLOCKS(parallelism),
                           min(CUDA_NUM_THREADS, parallelism),
                           0,
                           stream>>>(
        static_cast<DT *>(m->devQKVProjArray),
        static_cast<DT *>(m->keyCache),
        m->committed_token_infos,
        m->qProjSize,
        m->kProjSize,
        m->vProjSize,
        num_tokens_to_commit,
        m->num_active_tokens, // number of active tokens in previous batch
        m->num_heads,
        BatchConfig::MAX_SEQ_LENGTH,
        /* k_cache = */ true);

    parallelism = m->vProjSize * num_tokens_to_commit * m->num_heads;
    commit_tokens_kernel<<<GET_BLOCKS(parallelism),
                           min(CUDA_NUM_THREADS, parallelism),
                           0,
                           stream>>>(
        static_cast<DT *>(m->devQKVProjArray),
        static_cast<DT *>(m->valueCache),
        m->committed_token_infos,
        m->qProjSize,
        m->kProjSize,
        m->vProjSize,
        num_tokens_to_commit,
        m->num_active_tokens, // number of active tokens in previous batch
        m->num_heads,
        BatchConfig::MAX_SEQ_LENGTH,
        /* k_cache = */ false);
  }
}

template <typename DT>
__global__ void update_tree_branch_kv_cache(
    DT const *devQKVProjArray,
    DT *cache_ptr,
    TreeVerifyBatchConfig::PerTokenInfo const *tokenInfos,
    int qProjSize,
    int kProjSize,
    int vProjSize,
    int num_tokens_in_branch,
    int processed_tokens_in_batch,
    int total_tokens_in_batch,
    int num_heads,
    int max_seq_len,
    bool k_cache) {
  CUDA_KERNEL_LOOP(
      i, num_tokens_in_branch * (k_cache ? kProjSize : vProjSize) * num_heads) {
    int proj_size = k_cache ? kProjSize : vProjSize;
    int data_idx = i % proj_size;
    int token_idx =
        (i / proj_size) % num_tokens_in_branch; // index in the tree branch
    int head_idx = i / (proj_size * num_tokens_in_branch);

    token_idx += processed_tokens_in_batch; // get index in the whole batch
    int qkv_block_size = (qProjSize + kProjSize + vProjSize) *
                         total_tokens_in_batch; // skip over previous heads
    int current_head_block_size =
        total_tokens_in_batch *
        (k_cache ? qProjSize
                 : qProjSize + kProjSize); // skip over Q entries (and K entries
                                           // if we are working on the V cache)
    DT val =
        devQKVProjArray[head_idx * qkv_block_size + current_head_block_size +
                        token_idx * proj_size + data_idx];
    int const req_id = tokenInfos[token_idx].request_index;
    int const tok_id = tokenInfos[token_idx].abs_depth_in_request;

    cache_ptr[req_id * (num_heads * max_seq_len * proj_size) +
              head_idx * (max_seq_len * proj_size) + tok_id * proj_size +
              data_idx] = val;
  }
}

template <typename DT>
__global__ void tree_fill_entries_above_diagonal(DT *matrix,
                                                 size_t new_tokens,
                                                 size_t total_tokens_in_request,
                                                 size_t num_heads,
                                                 DT value) {
  CUDA_KERNEL_LOOP(i, new_tokens * total_tokens_in_request * num_heads) {
    // size_t head_idx = i / (new_tokens * total_tokens_in_request);
    size_t src_idx = (i / new_tokens) % total_tokens_in_request;
    size_t dst_idx = i % new_tokens + total_tokens_in_request - new_tokens;
    // Casual Mask
    if (src_idx > dst_idx) {
      matrix[i] = value;
    }
  }
}

template <typename DT>
void compute_attention_kernel(TreeIncMultiHeadSelfAttentionMeta const *m,
                              TreeVerifyBatchConfig const *bc,
                              DT *output_ptr,
                              DT const *bias_ptr,
                              hipStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  hipDataType cublas_data_type = ff_to_cuda_datatype(m->output_type[0]);
  hipdnnDataType_t cudnn_data_type = ff_to_cudnn_datatype(m->output_type[0]);
  assert(data_type_size(m->output_type[0]) == sizeof(DT));
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = cublas_data_type;
#endif
  // int num_requests = bc->num_active_requests();
  int processed_tokens_in_batch = 0;
  int qkv_block_size =
      (m->qProjSize + m->kProjSize + m->vProjSize) * bc->num_active_tokens();
  int kt_block_size = m->kProjSize * BatchConfig::MAX_SEQ_LENGTH;
  int kt_req_block_size = kt_block_size * m->num_heads;
  int vt_block_size = m->vProjSize * BatchConfig::MAX_SEQ_LENGTH;
  int vt_req_block_size = vt_block_size * m->num_heads;
  assert(m->qProjSize == m->kProjSize);

  for (int i = 0; i < bc->MAX_NUM_REQUESTS; i++) {
    if (bc->request_completed[i]) {
      continue;
    }
    int last_token_idx_of_the_request =
        processed_tokens_in_batch + bc->requestsInfo[i].num_tokens_in_batch - 1;
    while (processed_tokens_in_batch <= last_token_idx_of_the_request) {
      int num_new_tokens = 1;
      int j = processed_tokens_in_batch;
      while ((j + 1 <= last_token_idx_of_the_request) &&
             (bc->tokensInfo[j].abs_depth_in_request + 1 ==
              bc->tokensInfo[j + 1].abs_depth_in_request)) {
        j++;
        num_new_tokens++;
      }

      int total_tokens_in_request = bc->tokensInfo[j].abs_depth_in_request + 1;
      assert(num_new_tokens >= 1 && total_tokens_in_request >= num_new_tokens);
      {
        // update K-V cache
        int parallelism = m->kProjSize * num_new_tokens * m->num_heads;
        update_tree_branch_kv_cache<<<GET_BLOCKS(parallelism),
                                      min(CUDA_NUM_THREADS, parallelism),
                                      0,
                                      stream>>>(
            static_cast<DT *>(m->devQKVProjArray),
            static_cast<DT *>(m->keyCache),
            m->token_infos,
            m->qProjSize,
            m->kProjSize,
            m->vProjSize,
            num_new_tokens,            // num_tokens_in_branch
            processed_tokens_in_batch, // num_processed_tokens_in_batch
            m->num_active_tokens,      // total_tokens_in_batch
            m->num_heads,
            BatchConfig::MAX_SEQ_LENGTH,
            /* k_cache = */ true);

        parallelism = m->vProjSize * num_new_tokens * m->num_heads;
        update_tree_branch_kv_cache<<<GET_BLOCKS(parallelism),
                                      min(CUDA_NUM_THREADS, parallelism),
                                      0,
                                      stream>>>(
            static_cast<DT *>(m->devQKVProjArray),
            static_cast<DT *>(m->valueCache),
            m->token_infos,
            m->qProjSize,
            m->kProjSize,
            m->vProjSize,
            num_new_tokens,            // num_tokens_in_branch
            processed_tokens_in_batch, // num_processed_tokens_in_batch
            m->num_active_tokens,      // total_tokens_in_batch
            m->num_heads,
            BatchConfig::MAX_SEQ_LENGTH,
            /* k_cache = */ false);
      }

      // bc->token_last_available_idx[i] + 1;
      // Compute (QK^T/sqrt(d_k))
      int m_ = num_new_tokens;
      int n = total_tokens_in_request;
      int k = m->qProjSize;
      int lda = k, ldb = k, ldc = m_;
      int strideA = qkv_block_size;
      int strideB = kt_block_size;
      int strideC = num_new_tokens * total_tokens_in_request;

      // a flag of using this scaling alpha
      DT alpha = 1.0f, beta = 0.0f;
      if (*m->qk_prod_scaling) {
        alpha = static_cast<DT>(1.0f / sqrt(m->kProjSize));
      }
      // To get A, skip over Q entries from previous requests (same head)
      void const *A = static_cast<DT *>(m->devQKVProjArray) +
                      processed_tokens_in_batch * m->qProjSize;
      // To get B, skip over K entries from previous requests (all heads +
      // padding)
      void const *B = static_cast<DT *>(m->keyCache) + i * kt_req_block_size;
      // To get C, skip over QK^T products from previous requests
      void *C = static_cast<DT *>(m->qk_prods);

      checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                           HIPBLAS_OP_T,
                                           HIPBLAS_OP_N,
                                           m_,
                                           n,
                                           k,
                                           &alpha,
                                           A,
                                           cublas_data_type,
                                           lda,
                                           strideA,
                                           B,
                                           cublas_data_type,
                                           ldb,
                                           strideB,
                                           &beta,
                                           C,
                                           cublas_data_type,
                                           ldc,
                                           strideC,
                                           m->num_heads,
                                           compute_type,
                                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));

      // Fill all elements above diagonal in qk prods with -inf to force
      // causal attention.
      assert(num_new_tokens <= total_tokens_in_request);
      if (num_new_tokens > 1) {
        size_t parallelism =
            m->num_heads * num_new_tokens * total_tokens_in_request;
        tree_fill_entries_above_diagonal<<<GET_BLOCKS(parallelism),
                                           min((size_t)CUDA_NUM_THREADS,
                                               parallelism),
                                           0,
                                           stream>>>(
            static_cast<DT *>(C),
            num_new_tokens,
            total_tokens_in_request,
            m->num_heads,
            static_cast<DT>(-INFINITY));
      }
      // Compute Softmax(QK^T/sqrt(d_k))
      hipdnnTensorDescriptor_t qk_tensor;
      checkCUDNN(hipdnnCreateTensorDescriptor(&qk_tensor));
      // Before modifying the parameters below, make sure to read the following
      // description of the HIPDNN_TENSOR_NCHW tensor layout, from
      // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnTensorFormat_t:
      // This tensor format specifies that the data is laid out in the following
      // order: batch size, feature maps, rows, columns. The strides are
      // implicitly defined in such a way that the data are contiguous in memory
      // with no padding between images, feature maps, rows, and columns; the
      // columns are the inner dimension and the images are the outermost
      // dimension.
      int n_param = m->num_heads;
      int c_param = total_tokens_in_request;
      int h_param = 1;
      int w_param = num_new_tokens;
      checkCUDNN(hipdnnSetTensor4dDescriptor(qk_tensor,
                                            HIPDNN_TENSOR_NCHW,
                                            cudnn_data_type,
                                            n_param,
                                            c_param,
                                            h_param,
                                            w_param));
      float softmax_alpha = 1.0f, softmax_beta = 0.0f;
      void *C_softmax = (void *)(m->qk_prods_softmax);
      // The softmax operation below is executed according to the
      // HIPDNN_SOFTMAX_MODE_CHANNEL, which is also described in the docs: The
      // softmax operation is computed per spatial location (H,W) per image (N)
      // across dimension C.
      checkCUDNN(hipdnnSoftmaxForward(m->handle.dnn,
                                     HIPDNN_SOFTMAX_ACCURATE,
                                     HIPDNN_SOFTMAX_MODE_CHANNEL,
                                     &softmax_alpha,
                                     qk_tensor,
                                     C,
                                     &softmax_beta,
                                     qk_tensor,
                                     C_softmax));
      // Matmul softmax(QK^T/sqrt(d_k)) by V
      alpha = 1.0f, beta = 0.0f;
      m_ = num_new_tokens;
      n = m->vProjSize;
      k = total_tokens_in_request;
      lda = m_, ldb = n, ldc = m_;
      strideA = num_new_tokens * total_tokens_in_request;
      strideB = vt_block_size;
      strideC = num_new_tokens * m->vProjSize;
      // To get A, skip over softmax(QK^T/sqrt(d_k)) entries from previous
      // requests (all heads)
      A = static_cast<DT *>(C_softmax);
      // To get B, skip over V^T entries from previous requests (all heads +
      // padding)
      B = static_cast<DT *>(m->valueCache) + i * vt_req_block_size;
      // To get C, skip over softmax(QK^T/sqrt(d_k))V products from previous
      // requests
      C = static_cast<DT *>(m->attn_heads) +
          processed_tokens_in_batch * m->num_heads * m->vProjSize;

      checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                           HIPBLAS_OP_N,
                                           HIPBLAS_OP_T,
                                           m_,
                                           n,
                                           k,
                                           &alpha,
                                           A,
                                           cublas_data_type,
                                           lda,
                                           strideA,
                                           B,
                                           cublas_data_type,
                                           ldb,
                                           strideB,
                                           &beta,
                                           C,
                                           cublas_data_type,
                                           ldc,
                                           strideC,
                                           m->num_heads,
                                           compute_type,
                                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
      // Project to output, save result directly on output tensor
      alpha = 1.0f, beta = 0.0f;
      m_ = m->oProjSize;
      k = m->vProjSize * m->num_heads;
      n = num_new_tokens;
      lda = m_, ldb = n, ldc = m_;
      A = m->W_out_contiguous;
      B = C;
      C = (output_ptr + processed_tokens_in_batch * m->oProjSize);

      checkCUDA(hipblasGemmEx(m->handle.blas,
                             HIPBLAS_OP_N,
                             HIPBLAS_OP_T,
                             m_,
                             n,
                             k,
                             &alpha,
                             A,
                             cublas_data_type,
                             lda,
                             B,
                             cublas_data_type,
                             ldb,
                             &beta,
                             C,
                             cublas_data_type,
                             ldc,
                             compute_type,
                             CUBLAS_GEMM_DEFAULT_TENSOR_OP));
      processed_tokens_in_batch += num_new_tokens;
    }
    // Before moving to the next request
    // check that we have finished all tokens of the request
    assert(last_token_idx_of_the_request + 1 == processed_tokens_in_batch);
  }
  if (*m->bias) {
    int parallelism = m->oProjSize * processed_tokens_in_batch;
    apply_proj_bias_w<<<GET_BLOCKS(parallelism),
                        min(CUDA_NUM_THREADS, parallelism),
                        0,
                        stream>>>(output_ptr,
                                  bias_ptr,
                                  processed_tokens_in_batch,
                                  m->oProjSize,
                                  (m->qProjSize + m->kProjSize + m->vProjSize) *
                                      m->num_heads);
  }

  assert(processed_tokens_in_batch == bc->num_active_tokens());
}

template <typename DT>
void inference_kernel(TreeIncMultiHeadSelfAttentionMeta *m,
                      TreeVerifyBatchConfig const *bc,
                      DT const *input_ptr,
                      DT const *weight_ptr,
                      DT *output_ptr,
                      DT const *bias_ptr,
                      hipStream_t stream) {
  // copy committed tokens info to GPU for the commit_tokens kernel
  // Note that m->num_active_tokens stores the number of active
  // tokens in the previous batch, which is needed for committing
  // keys/values to the key-value cache
  hipMemcpyAsync(m->committed_token_infos,
                  &(bc->committed_tokens),
                  bc->MAX_NUM_TOKENS *
                      sizeof(TreeVerifyBatchConfig::CommittedTokensInfo),
                  hipMemcpyHostToDevice,
                  stream);
  commit_tokens<DT>(m, bc, stream);

  // After commit we update m->num_active_tokens to be the number of active
  // tokens for the current batch
  m->num_active_tokens = bc->num_active_tokens();

  // here because we need postion info in infernece 1
  hipMemcpyAsync(m->token_infos,
                  &(bc->tokensInfo),
                  bc->MAX_NUM_TOKENS *
                      sizeof(TreeVerifyBatchConfig::PerTokenInfo),
                  hipMemcpyHostToDevice,
                  stream);
  // phase 1: Implement kernel to compute KQV for input tokens
  compute_qkv_kernel(m,
                     bc,
                     input_ptr,
                     weight_ptr,
                     static_cast<DT *>(m->devQKVProjArray),
                     bias_ptr,
                     stream);

  // phase 2: No need to update key/val cache
  // IncMultiHeadSelfAttention::update_kv_cache_kernel(
  //    m, bc, stream);

  // phase 3: Compute attention score
  // 3 kernels for pahse 3: matmul1 - softmax - matmal2
  compute_attention_kernel(m, bc, output_ptr, bias_ptr, stream);
}

} // namespace TreeIncMultiHeadAttention
} // namespace Kernels

/*static*/
void TreeIncMultiHeadSelfAttention::inference_kernel_wrapper(
    TreeIncMultiHeadSelfAttentionMeta *m,
    TreeVerifyBatchConfig const *bc,
    GenericTensorAccessorR const &input,
    GenericTensorAccessorR const &weight,
    GenericTensorAccessorW const &output,
    GenericTensorAccessorR const &bias) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  bool use_bias = *m->bias;

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  assert(input.data_type == weight.data_type);
  assert(input.data_type == output.data_type);
  if (use_bias) {
    assert(input.data_type == bias.data_type);
  }

  if (input.data_type == DT_HALF) {
    half const *bias_ptr =
        use_bias ? bias.get_half_ptr() : static_cast<half const *>(nullptr);
    Kernels::TreeIncMultiHeadAttention::inference_kernel(m,
                                                         bc,
                                                         input.get_half_ptr(),
                                                         weight.get_half_ptr(),
                                                         output.get_half_ptr(),
                                                         bias_ptr,
                                                         stream);
  } else if (input.data_type == DT_FLOAT) {
    float const *bias_ptr =
        use_bias ? bias.get_float_ptr() : static_cast<float const *>(nullptr);
    Kernels::TreeIncMultiHeadAttention::inference_kernel(m,
                                                         bc,
                                                         input.get_float_ptr(),
                                                         weight.get_float_ptr(),
                                                         output.get_float_ptr(),
                                                         bias_ptr,
                                                         stream);
  } else {
    assert(false && "Unspported data type");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("TreeIncMultiHeadSelfAttention forward time = %.2fms\n", elapsed);
    // print_tensor<3, float>(acc_query.ptr, acc_query.rect,
    // "[Attention:forward:query]"); print_tensor<3, float>(acc_output.ptr,
    // acc_output.rect, "[Attention:forward:output]");
  }
}

TreeIncMultiHeadSelfAttentionMeta::TreeIncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    TreeIncMultiHeadSelfAttention const *attn,
    GenericTensorAccessorR const &weight,
    Memory gpu_mem,
    int num_samples,
    int _num_heads)
    : IncMultiHeadSelfAttentionMeta(handler,
                                    TREE_VERIFY_MODE,
                                    attn,
                                    attn->qSize,
                                    attn->kSize,
                                    attn->vSize,
                                    attn->qProjSize,
                                    attn->kProjSize,
                                    attn->vProjSize,
                                    attn->oProjSize,
                                    attn->apply_rotary_embedding,
                                    attn->bias,
                                    attn->scaling_query,
                                    attn->qk_prod_scaling,
                                    attn->add_bias_kv,
                                    attn->scaling_factor,
                                    weight,
                                    gpu_mem,
                                    num_samples,
                                    _num_heads,
                                    attn->output_bias),
      num_active_tokens(0) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  checkCUDNN(hipdnnSetStream(handler.dnn, stream));

  // allocate memory for the seqArray and reserve space
  {
    size_t committed_tokeninfo_size = TreeVerifyBatchConfig::MAX_NUM_TOKENS;
    size_t totalSize = committed_tokeninfo_size *
                       sizeof(TreeVerifyBatchConfig::CommittedTokensInfo);

    Realm::Rect<1, coord_t> bounds(Realm::Point<1, coord_t>(0),
                                   Realm::Point<1, coord_t>(totalSize - 1));
    std::vector<size_t> field_sizes;
    field_sizes.push_back(sizeof(char));
    Realm::RegionInstance::create_instance(committed_token_reserve_inst,
                                           gpu_mem,
                                           bounds,
                                           field_sizes,
                                           0,
                                           Realm::ProfilingRequestSet())
        .wait();
    committed_token_infos =
        committed_token_reserve_inst
            .pointer<TreeVerifyBatchConfig::CommittedTokensInfo>(0);
  }

  hipStreamSynchronize(stream);
}

TreeIncMultiHeadSelfAttentionMeta::~TreeIncMultiHeadSelfAttentionMeta(void) {
  committed_token_reserve_inst.destroy();
}

}; // namespace FlexFlow
