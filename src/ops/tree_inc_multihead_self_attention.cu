#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#if defined(FF_USE_CUDA) || defined(FF_USE_HIP_CUDA)
#include "hip/hip_complex.h"
#endif
#include "flexflow/ffconst_utils.h"
#include "flexflow/ops/kernels/inc_multihead_self_attention_kernels.h"
#include "flexflow/ops/tree_inc_multihead_self_attention.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

// declare Legion names
using Legion::coord_t;
using Legion::Memory;

using namespace Kernels::IncMultiHeadAttention;

namespace Kernels {
namespace TreeIncMultiHeadAttention {

template <typename DT>
__global__ void commit_tokens_kernel(
    DT const *devQKVProjArray,
    DT *kCache_ptr,
    DT *vCache_ptr,
    TreeVerifyBatchConfig::CommittedTokensInfo const *committedTokenInfos,
    int qProjSize,
    int kProjSize,
    int vProjSize,
    int num_tokens_to_commit,
    int num_active_tokens_in_last_batch,
    int max_seq_len,
    int hidden_size) {

  CUDA_KERNEL_LOOP(i, num_tokens_to_commit * hidden_size * 2) {

    int token_pos = i / (hidden_size * KV_WEIGHT_NUM);
    int token_idx_in_last_batch = committedTokenInfos[token_pos].token_index;
    int offset = i % hidden_size;
    assert(token_idx_in_last_batch < num_active_tokens_in_last_batch);

    size_t val_idx = token_idx_in_last_batch * QKV_WEIGHT_NUM * hidden_size +
                     hidden_size + offset;

    DT kVal = devQKVProjArray[val_idx];
    DT vVal = devQKVProjArray[val_idx + hidden_size];

    int const req_id = committedTokenInfos[token_pos].request_index;
    int const tok_id = committedTokenInfos[token_pos].token_depth;

    kCache_ptr[req_id * (hidden_size * max_seq_len) + tok_id * hidden_size +
               offset] = kVal;
    vCache_ptr[req_id * (hidden_size * max_seq_len) + tok_id * hidden_size +
               offset] = vVal;
  }
}

template <typename DT>
void commit_tokens(TreeIncMultiHeadSelfAttentionMeta const *m,
                   TreeVerifyBatchConfig const *bc,
                   hipStream_t stream) {
  int num_tokens_to_commit = bc->num_tokens_to_commit;
  if (num_tokens_to_commit > 0) {
    int parallelism = m->hidden_size * KV_WEIGHT_NUM * num_tokens_to_commit;
    commit_tokens_kernel<<<GET_BLOCKS(parallelism),
                           min(CUDA_NUM_THREADS, parallelism),
                           0,
                           stream>>>(
        static_cast<DT *>(m->devQKVProjArray),
        static_cast<DT *>(m->keyCache),
        static_cast<DT *>(m->valueCache),
        m->committed_token_infos,
        m->qProjSize,
        m->kProjSize,
        m->vProjSize,
        num_tokens_to_commit,
        m->num_active_tokens, // number of active tokens in previous batch
        BatchConfig::max_sequence_length(),
        m->hidden_size);
  }
}

template <typename DT>
__global__ void update_tree_branch_kv_cache(
    DT const *devQKVProjArray,
    DT *kCache_ptr,
    DT *vCache_ptr,
    TreeVerifyBatchConfig::PerTokenInfo const *tokenInfos,
    int qProjSize,
    int kProjSize,
    int vProjSize,
    int num_tokens_in_branch,
    int processed_tokens_in_batch,
    int total_tokens_in_batch,
    int max_seq_len,
    int hidden_size) {
  CUDA_KERNEL_LOOP(i, num_tokens_in_branch * hidden_size * 2) {

    int token_idx = i / (hidden_size * KV_WEIGHT_NUM);
    int offset = i % hidden_size;

    token_idx += processed_tokens_in_batch; // get index in the whole batch
    size_t val_idx =
        token_idx * QKV_WEIGHT_NUM * hidden_size + hidden_size + offset;

    DT kVal = devQKVProjArray[val_idx];
    DT vVal = devQKVProjArray[val_idx + hidden_size];

    int const req_id = tokenInfos[token_idx].request_index;
    int const tok_id = tokenInfos[token_idx].abs_depth_in_request;
    kCache_ptr[req_id * (hidden_size * max_seq_len) + tok_id * hidden_size +
               offset] = kVal;
    vCache_ptr[req_id * (hidden_size * max_seq_len) + tok_id * hidden_size +
               offset] = vVal;
  }
}

template <typename DT>
__global__ void tree_fill_entries_above_diagonal(DT *matrix,
                                                 size_t new_tokens,
                                                 size_t total_tokens_in_request,
                                                 size_t num_q_heads,
                                                 DT value) {
  CUDA_KERNEL_LOOP(i, new_tokens * total_tokens_in_request * num_q_heads) {
    // size_t head_idx = i / (new_tokens * total_tokens_in_request);
    size_t src_idx = (i / new_tokens) % total_tokens_in_request;
    size_t dst_idx = i % new_tokens + total_tokens_in_request - new_tokens;
    // Casual Mask
    if (src_idx > dst_idx) {
      matrix[i] = value;
    }
  }
}

template <typename DT>
void compute_attention_kernel(TreeIncMultiHeadSelfAttentionMeta const *m,
                              TreeVerifyBatchConfig const *bc,
                              int shard_id,
                              DT *output_ptr,
                              DT const *bias_ptr,
                              DT const *weight_ptr,
                              hipStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  hipDataType cublas_data_type = ff_to_cuda_datatype(m->output_type[0]);
  hipdnnDataType_t cudnn_data_type = ff_to_cudnn_datatype(m->output_type[0]);
  assert(data_type_size(m->output_type[0]) == sizeof(DT));
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = cublas_data_type;
#endif
  // int num_requests = bc->num_active_requests();
  int processed_tokens_in_batch = 0;
  // int qkv_block_size =
  //     (m->qProjSize + m->kProjSize + m->vProjSize) * bc->num_active_tokens();
  int q_block_size = m->qProjSize;
  int kt_block_size = m->kProjSize;
  int kt_req_block_size =
      kt_block_size * m->num_q_heads * BatchConfig::max_sequence_length();
  int vt_block_size = m->vProjSize;
  int vt_req_block_size =
      vt_block_size * m->num_q_heads * BatchConfig::max_sequence_length();
  assert(m->qProjSize == m->kProjSize);

  for (int i = 0; i < bc->max_requests_per_batch(); i++) {
    if (bc->request_completed[i]) {
      continue;
    }
    assert(processed_tokens_in_batch ==
           bc->requestsInfo[i].first_token_offset_in_batch);
    int last_token_idx_of_the_request =
        processed_tokens_in_batch + bc->requestsInfo[i].num_tokens_in_batch - 1;
    while (processed_tokens_in_batch <= last_token_idx_of_the_request) {
      int num_new_tokens = 1;
      int j = processed_tokens_in_batch;
      while ((j + 1 <= last_token_idx_of_the_request) &&
             (bc->tokensInfo[j].abs_depth_in_request + 1 ==
              bc->tokensInfo[j + 1].abs_depth_in_request)) {
        j++;
        num_new_tokens++;
      }

      int total_tokens_in_request = bc->tokensInfo[j].abs_depth_in_request + 1;
      assert(num_new_tokens >= 1 && total_tokens_in_request >= num_new_tokens);
      {
        // update K-V cache
        int parallelism = m->hidden_size * KV_WEIGHT_NUM * num_new_tokens;
        update_tree_branch_kv_cache<<<GET_BLOCKS(parallelism),
                                      min(CUDA_NUM_THREADS, parallelism),
                                      0,
                                      stream>>>(
            static_cast<DT *>(m->devQKVProjArray),
            static_cast<DT *>(m->keyCache),
            static_cast<DT *>(m->valueCache),
            m->token_infos,
            m->qProjSize,
            m->kProjSize,
            m->vProjSize,
            num_new_tokens,            // num_tokens_in_branch
            processed_tokens_in_batch, // num_processed_tokens_in_batch
            m->num_active_tokens,      // total_tokens_in_batch
            BatchConfig::max_sequence_length(),
            m->hidden_size);
      }

      // bc->token_last_available_idx[i] + 1;
      // Compute (QK^T/sqrt(d_k))
      int m_ = num_new_tokens;
      int n = total_tokens_in_request;
      int k = m->qProjSize;
      int lda = k * m->num_q_heads * QKV_WEIGHT_NUM, ldb = k * m->num_q_heads,
          ldc = m_;
      int strideA = q_block_size;
      int strideB = kt_block_size;
      int strideC = num_new_tokens * total_tokens_in_request;

      // a flag of using this scaling alpha
      DT alpha = 1.0f, beta = 0.0f;
      if (*m->qk_prod_scaling) {
        alpha = static_cast<DT>(1.0f / sqrt(m->kProjSize));
      }
      // To get A, skip over Q entries from previous requests (same head)
      DT const *A = static_cast<DT *>(m->devQKVProjArray) +
                    processed_tokens_in_batch * m->qProjSize * m->num_q_heads *
                        QKV_WEIGHT_NUM;
      // To get B, skip over K entries from previous requests (all heads +
      // padding)
      DT const *B = static_cast<DT *>(m->keyCache) + i * kt_req_block_size;
      // To get C, skip over QK^T products from previous requests
      DT *C = static_cast<DT *>(m->qk_prods);

      checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                           HIPBLAS_OP_T,
                                           HIPBLAS_OP_N,
                                           m_,
                                           n,
                                           k,
                                           &alpha,
                                           A,
                                           cublas_data_type,
                                           lda,
                                           strideA,
                                           B,
                                           cublas_data_type,
                                           ldb,
                                           strideB,
                                           &beta,
                                           C,
                                           cublas_data_type,
                                           ldc,
                                           strideC,
                                           m->num_q_heads,
                                           compute_type,
                                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
      // add alibi position bias to qk production
      // add alibi position bias to qk production
      if (*m->position_bias) {
        size_t parallelism =
            m->num_q_heads * total_tokens_in_request * num_new_tokens;
        apply_position_bias_qkprd<<<GET_BLOCKS(parallelism),
                                    min((size_t)CUDA_NUM_THREADS, parallelism),
                                    0,
                                    stream>>>(C,
                                              num_new_tokens,
                                              total_tokens_in_request,
                                              m->num_q_heads,
                                              m->global_num_q_heads,
                                              shard_id);
      }

      // Fill all elements above diagonal in qk prods with -inf to force
      // causal attention.
      assert(num_new_tokens <= total_tokens_in_request);
      if (num_new_tokens > 1) {
        size_t parallelism =
            m->num_q_heads * num_new_tokens * total_tokens_in_request;
        tree_fill_entries_above_diagonal<<<GET_BLOCKS(parallelism),
                                           min((size_t)CUDA_NUM_THREADS,
                                               parallelism),
                                           0,
                                           stream>>>(
            C,
            num_new_tokens,
            total_tokens_in_request,
            m->num_q_heads,
            static_cast<DT>(-INFINITY));
      }
      // Compute Softmax(QK^T/sqrt(d_k))
      // Before modifying the parameters below, make sure to read the following
      // description of the HIPDNN_TENSOR_NCHW tensor layout, from
      // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnTensorFormat_t:
      // This tensor format specifies that the data is laid out in the following
      // order: batch size, feature maps, rows, columns. The strides are
      // implicitly defined in such a way that the data are contiguous in memory
      // with no padding between images, feature maps, rows, and columns; the
      // columns are the inner dimension and the images are the outermost
      // dimension.
      int n_param = m->num_q_heads;
      int c_param = total_tokens_in_request;
      int h_param = 1;
      int w_param = num_new_tokens;
      checkCUDNN(hipdnnSetTensor4dDescriptor(m->qk_tensor,
                                            HIPDNN_TENSOR_NCHW,
                                            cudnn_data_type,
                                            n_param,
                                            c_param,
                                            h_param,
                                            w_param));
      float softmax_alpha = 1.0f, softmax_beta = 0.0f;
      DT *C_softmax = static_cast<DT *>(m->qk_prods_softmax);
      // The softmax operation below is executed according to the
      // HIPDNN_SOFTMAX_MODE_CHANNEL, which is also described in the docs: The
      // softmax operation is computed per spatial location (H,W) per image (N)
      // across dimension C.
      checkCUDNN(hipdnnSoftmaxForward(m->handle.dnn,
                                     HIPDNN_SOFTMAX_ACCURATE,
                                     HIPDNN_SOFTMAX_MODE_CHANNEL,
                                     &softmax_alpha,
                                     m->qk_tensor,
                                     C,
                                     &softmax_beta,
                                     m->qk_tensor,
                                     C_softmax));
      // Matmul softmax(QK^T/sqrt(d_k)) by V
      alpha = 1.0f, beta = 0.0f;
      m_ = m->vProjSize;
      n = num_new_tokens;
      k = total_tokens_in_request;
      lda = m_ * m->num_q_heads, ldb = n, ldc = m_ * m->num_q_heads;
      strideA = vt_block_size;
      strideB = num_new_tokens * total_tokens_in_request;
      strideC = m->vProjSize;
      // To get A, skip over V^T entries from previous requests (all heads +
      // padding)
      A = static_cast<DT *>(m->valueCache) + i * vt_req_block_size;
      // To get B, skip over softmax(QK^T/sqrt(d_k)) entries from previous
      // requests (all heads)
      B = C_softmax;
      // To get C, skip over softmax(QK^T/sqrt(d_k))V products from previous
      // requests
      C = static_cast<DT *>(m->attn_heads) +
          processed_tokens_in_batch * m->num_q_heads * m->vProjSize;
      checkCUDA(hipblasGemmStridedBatchedEx(m->handle.blas,
                                           HIPBLAS_OP_N,
                                           HIPBLAS_OP_T,
                                           m_,
                                           n,
                                           k,
                                           &alpha,
                                           A,
                                           cublas_data_type,
                                           lda,
                                           strideA,
                                           B,
                                           cublas_data_type,
                                           ldb,
                                           strideB,
                                           &beta,
                                           C,
                                           cublas_data_type,
                                           ldc,
                                           strideC,
                                           m->num_q_heads,
                                           compute_type,
                                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
      processed_tokens_in_batch += num_new_tokens;
    }
    // Before moving to the next request
    // check that we have finished all tokens of the request
    assert(last_token_idx_of_the_request + 1 == processed_tokens_in_batch);
  }
  // Project to output, save result directly on output tensor
  DT alpha = 1.0f, beta = 0.0f;
  int m_ = m->oProjSize;
  int k = m->vProjSize * m->num_q_heads;
  int n = processed_tokens_in_batch;
  int lda = k, ldb = k, ldc = m_;
  DT const *A = weight_ptr + m->qSize * (m->qProjSize * m->num_q_heads +
                                         m->kProjSize * m->num_q_heads +
                                         m->vProjSize * m->num_q_heads);
  DT const *B = static_cast<DT *>(m->attn_heads);
  DT *C = static_cast<DT *>(output_ptr);

  checkCUDA(hipblasGemmEx(m->handle.blas,
                         HIPBLAS_OP_T,
                         HIPBLAS_OP_N,
                         m_,
                         n,
                         k,
                         &alpha,
                         A,
                         cublas_data_type,
                         lda,
                         B,
                         cublas_data_type,
                         ldb,
                         &beta,
                         C,
                         cublas_data_type,
                         ldc,
                         compute_type,
                         CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  if (*m->final_bias && shard_id == 0) {
    int parallelism = m->oProjSize * processed_tokens_in_batch;
    int qkv_weight_size = m->qProjSize * m->global_num_q_heads +
                          m->kProjSize * m->global_num_q_heads +
                          m->vProjSize * m->global_num_q_heads;
    apply_proj_bias_w<<<GET_BLOCKS(parallelism),
                        min(CUDA_NUM_THREADS, parallelism),
                        0,
                        stream>>>(output_ptr,
                                  bias_ptr,
                                  processed_tokens_in_batch,
                                  qkv_weight_size,
                                  m->oProjSize);
  }

  assert(processed_tokens_in_batch == bc->num_active_tokens());
}

template <typename DT>
void inference_kernel(TreeIncMultiHeadSelfAttentionMeta *m,
                      TreeVerifyBatchConfig const *bc,
                      int shard_id,
                      DT const *input_ptr,
                      DT const *weight_ptr,
                      DT *output_ptr,
                      DT const *bias_ptr,
                      hipStream_t stream) {
  // additional processing for weight uploading
  if (m->handle.offload_reserve_space != nullptr) {
    // Note that we update weight_ptr and bias_ptr when uploading weight and
    // bias
    hipMemcpyAsync(m->weight_ptr,
                    weight_ptr,
                    m->weightSize,
                    hipMemcpyHostToDevice,
                    stream);
    weight_ptr = static_cast<DT *>(m->weight_ptr);
    if (m->biasSize > 0) {
      hipMemcpyAsync(
          m->bias_ptr, bias_ptr, m->biasSize, hipMemcpyHostToDevice, stream);
      bias_ptr = static_cast<DT *>(m->bias_ptr);
    }
  }
  // copy committed tokens info to GPU for the commit_tokens kernel
  // Note that m->num_active_tokens stores the number of active
  // tokens in the previous batch, which is needed for committing
  // keys/values to the key-value cache
  hipMemcpyAsync(m->committed_token_infos,
                  &(bc->committed_tokens),
                  bc->num_tokens_to_commit *
                      sizeof(TreeVerifyBatchConfig::CommittedTokensInfo),
                  hipMemcpyHostToDevice,
                  stream);
  commit_tokens<DT>(m, bc, stream);

  // After commit we update m->num_active_tokens to be the number of active
  // tokens for the current batch
  m->num_active_tokens = bc->num_active_tokens();

  // here because we need postion info in infernece 1
  if (m->offload && m->biasSize > 0) {
    hipMemcpyAsync(
        m->bias_ptr, bias_ptr, m->biasSize, hipMemcpyHostToDevice, stream);
    bias_ptr = static_cast<DT *>(m->bias_ptr);
  }
  hipMemcpyAsync(m->token_infos,
                  &(bc->tokensInfo),
                  bc->num_active_tokens() *
                      sizeof(TreeVerifyBatchConfig::PerTokenInfo),
                  hipMemcpyHostToDevice,
                  stream);
  // phase 1: Implement kernel to compute KQV for input tokens
  compute_qkv_kernel(m,
                     bc,
                     shard_id,
                     input_ptr,
                     weight_ptr,
                     static_cast<DT *>(m->devQKVProjArray),
                     bias_ptr,
                     stream);

  // phase 2: No need to update key/val cache
  // IncMultiHeadSelfAttention::update_kv_cache_kernel(
  //    m, bc, stream);

  // phase 3: Compute attention score
  // 3 kernels for pahse 3: matmul1 - softmax - matmal2
  compute_attention_kernel(
      m, bc, shard_id, output_ptr, bias_ptr, weight_ptr, stream);
}

} // namespace TreeIncMultiHeadAttention
} // namespace Kernels

/*static*/
void TreeIncMultiHeadSelfAttention::inference_kernel_wrapper(
    TreeIncMultiHeadSelfAttentionMeta *m,
    TreeVerifyBatchConfig const *bc,
    int shard_id,
    GenericTensorAccessorR const &input,
    GenericTensorAccessorR const &weight,
    GenericTensorAccessorW const &output,
    GenericTensorAccessorR const &bias) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  bool use_bias = *m->qkv_bias || *m->final_bias;

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  // assert(input.data_type == weight.data_type);
  assert(input.data_type == output.data_type);
  if (use_bias) {
    assert(input.data_type == bias.data_type);
  }

  if (input.data_type == DT_HALF) {
    if (m->offload) {
      pre_build_weight_kernel<half>(m, weight, input.data_type, stream);
    }

    half const *bias_ptr =
        use_bias ? bias.get_half_ptr() : static_cast<half const *>(nullptr);
    Kernels::TreeIncMultiHeadAttention::inference_kernel(
        m,
        bc,
        shard_id,
        input.get_half_ptr(),
        m->offload ? static_cast<half *>(m->weight_ptr) : weight.get_half_ptr(),
        output.get_half_ptr(),
        bias_ptr,
        stream);
  } else if (input.data_type == DT_FLOAT) {
    if (m->offload) {
      pre_build_weight_kernel<float>(m, weight, input.data_type, stream);
    }
    float const *bias_ptr =
        use_bias ? bias.get_float_ptr() : static_cast<float const *>(nullptr);
    Kernels::TreeIncMultiHeadAttention::inference_kernel(
        m,
        bc,
        shard_id,
        input.get_float_ptr(),
        m->offload ? static_cast<float *>(m->weight_ptr)
                   : weight.get_float_ptr(),
        output.get_float_ptr(),
        bias_ptr,
        stream);
  } else {
    assert(false && "Unspported data type");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("TreeIncMultiHeadSelfAttention forward time = %.2fms\n", elapsed);
    // print_tensor<3, float>(acc_query.ptr, acc_query.rect,
    // "[Attention:forward:query]"); print_tensor<3, float>(acc_output.ptr,
    // acc_output.rect, "[Attention:forward:output]");
  }
}

TreeIncMultiHeadSelfAttentionMeta::TreeIncMultiHeadSelfAttentionMeta(
    FFHandler handler,
    TreeIncMultiHeadSelfAttention const *attn,
    GenericTensorAccessorR const &weight,
    MemoryAllocator &gpu_mem_allocator,
    int num_samples,
    int _num_q_heads,
    int _num_kv_heads)
    : IncMultiHeadSelfAttentionMeta(handler,
                                    TREE_VERIFY_MODE,
                                    attn,
                                    attn->qSize,
                                    attn->kSize,
                                    attn->vSize,
                                    attn->qProjSize,
                                    attn->kProjSize,
                                    attn->vProjSize,
                                    attn->oProjSize,
                                    attn->apply_rotary_embedding,
                                    attn->qkv_bias,
                                    attn->scaling_query,
                                    attn->qk_prod_scaling,
                                    attn->position_bias,
                                    attn->final_bias,
                                    attn->scaling_factor,
                                    weight,
                                    gpu_mem_allocator,
                                    num_samples,
                                    attn->num_q_heads,
                                    attn->num_kv_heads,
                                    _num_q_heads,
                                    _num_kv_heads,
                                    attn->quantization_type,
                                    attn->offload),
      num_active_tokens(0) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  checkCUDNN(hipdnnSetStream(handler.dnn, stream));

  // allocate memory for the seqArray and reserve space
  {
    int max_tokens_per_batch = BatchConfig::max_tokens_per_batch();
    size_t committed_tokeninfo_size = max_tokens_per_batch;
    size_t total_size = committed_tokeninfo_size *
                        sizeof(TreeVerifyBatchConfig::CommittedTokensInfo);
    if (offload) {
      // assert that we have enough reserved work space left
      assert(gpu_mem_allocator.reserved_total_size -
                 gpu_mem_allocator.reserved_allocated_size >=
             total_size);
      committed_token_infos =
          gpu_mem_allocator
              .allocate_reserved<TreeVerifyBatchConfig::CommittedTokensInfo>(
                  committed_tokeninfo_size);
    } else {
      gpu_mem_allocator.create_legion_instance(committed_token_reserve_inst,
                                               total_size);
      committed_token_infos =
          gpu_mem_allocator
              .allocate_instance<TreeVerifyBatchConfig::CommittedTokensInfo>(
                  committed_tokeninfo_size);
    }
  }

  hipStreamSynchronize(stream);
}

TreeIncMultiHeadSelfAttentionMeta::~TreeIncMultiHeadSelfAttentionMeta(void) {
  if (committed_token_reserve_inst != Realm::RegionInstance::NO_INST) {
    committed_token_reserve_inst.destroy();
  }
}

}; // namespace FlexFlow
