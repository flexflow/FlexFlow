#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ffconst_utils.h"
#include "flexflow/ops/add_bias_residual_layer_norm.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

#define C10_WARP_SIZE 32
constexpr int kCUDABlockReduceNumThreads = 512;
constexpr int kCUDANumThreads = 256;
constexpr int kColwiseReduceTileSize = 32;

AddBiasResidualLayerNormMeta::AddBiasResidualLayerNormMeta(
    FFHandler handle,
    AddBiasResidualLayerNorm const *ln,
    MemoryAllocator &gpu_mem_allocator)
    : OpMeta(handle, ln) {
  elementwise_affine = ln->elementwise_affine;
  use_bias = ln->use_bias;
  effective_batch_size = ln->effective_batch_size;
  effective_num_elements = ln->effective_num_elements;
  profiling = ln->profiling;
  inference_debugging = ln->inference_debugging;
  eps = ln->eps;
  DataType data_type = ln->data_type;
  size_t totalSize = effective_batch_size * data_type_size(data_type) * 3;
  gpu_mem_allocator.create_legion_instance(reserveInst, totalSize);
  mean_ptr = gpu_mem_allocator.allocate_instance_untyped(
      data_type_size(data_type) * effective_batch_size);
  rstd_ptr = gpu_mem_allocator.allocate_instance_untyped(
      data_type_size(data_type) * effective_batch_size);
  bias_ptr = gpu_mem_allocator.allocate_instance_untyped(
      data_type_size(data_type) * effective_batch_size);
}

AddBiasResidualLayerNormMeta::~AddBiasResidualLayerNormMeta(void) {
  if (reserveInst != Realm::RegionInstance::NO_INST) {
    reserveInst.destroy();
  }
}

template <typename T>
__device__ __forceinline__ T WARP_SHFL_DOWN(T value,
                                            unsigned int delta,
                                            int width = warpSize,
                                            unsigned int mask = 0xffffffff) {
#ifndef __HIP_PLATFORM_HCC__
  return __shfl_down_sync(mask, value, delta, width);
#else
  return __shfl_down(value, delta, width);
#endif
}

template <typename T>
__inline__ __device__ T WarpReduceSum(T val) {
#pragma unroll
  for (int offset = (C10_WARP_SIZE >> 1); offset > 0; offset >>= 1) {
    val += WARP_SHFL_DOWN(val, offset);
  }
  return val;
}

template <typename T>
__inline__ __device__ T BlockReduceSum(T val, T *shared) {
  int const lid = threadIdx.x % C10_WARP_SIZE;
  int const wid = threadIdx.x / C10_WARP_SIZE;
  val = WarpReduceSum(val);
  __syncthreads();
  if (lid == 0) {
    shared[wid] = val;
  }
  __syncthreads();
  val = (threadIdx.x < (blockDim.x / C10_WARP_SIZE)) ? shared[lid] : T(0);
  if (wid == 0) {
    val = WarpReduceSum(val);
  }
  return val;
}

template <typename T>
__inline__ __device__ T BlockReduceSum(T val, T *shared, int max_num_threads) {
  int const lid = threadIdx.x % C10_WARP_SIZE;
  int const wid = threadIdx.x / C10_WARP_SIZE;
  val = WarpReduceSum(val);
  __syncthreads();
  if (lid == 0) {
    shared[wid] = val;
  }
  __syncthreads();
  val = (threadIdx.x < (min(blockDim.x, max_num_threads) / C10_WARP_SIZE))
            ? shared[lid]
            : T(0);
  if (wid == 0) {
    val = WarpReduceSum(val);
  }
  return val;
}

template <typename T>
__global__ void LayerNormFusedForwardKernel(int64_t N,
                                            int64_t attn_bias_dim,
                                            float eps,
                                            T const *input_ptr,
                                            T const *attn_bias_ptr,
                                            T const *residual_ptr,
                                            T *X,
                                            T *mean,
                                            T *rstd,
                                            T const *gamma,
                                            T const *beta,
                                            T *Y) {
  __shared__ float m_shared[C10_WARP_SIZE];
  __shared__ float v_shared[C10_WARP_SIZE];
  const int64_t i = blockIdx.x;
  float sum1 = 0.0f;
  float sum2 = 0.0f;
  for (int64_t j = threadIdx.x; j < N;
       j += min(blockDim.x, kCUDABlockReduceNumThreads)) {
    const int64_t index = i * N + j;
    const int64_t bias_idx = index % attn_bias_dim;
    X[index] = input_ptr[index] + attn_bias_ptr[bias_idx] + residual_ptr[index];
    sum1 += static_cast<float>(X[index]);
    sum2 += static_cast<float>(X[index]) * static_cast<float>(X[index]);
  }
  if (threadIdx.x < kCUDABlockReduceNumThreads) {
    sum1 = BlockReduceSum<float>(
        sum1, m_shared, min(blockDim.x, kCUDABlockReduceNumThreads));
    sum2 = BlockReduceSum<float>(
        sum2, v_shared, min(blockDim.x, kCUDABlockReduceNumThreads));
  }
  if (threadIdx.x == 0) {
    float const scale = float(1) / static_cast<float>(N);
    sum1 *= scale;
    sum2 = max(sum2 * scale - sum1 * sum1, float(0));
    mean[i] = static_cast<T>(sum1);
    rstd[i] = static_cast<T>(rsqrt(sum2 + eps));
  }

  __syncthreads();

  using T_ACC = T;
  for (int64_t j = threadIdx.x; j < N; j += min(blockDim.x, kCUDANumThreads)) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    const T_ACC beta_v =
        beta == nullptr ? T_ACC(0) : static_cast<T_ACC>(beta[j]);
    Y[index] = (static_cast<T_ACC>(X[index]) - static_cast<T_ACC>(mean[i])) *
                   static_cast<T_ACC>(rstd[i]) * gamma_v +
               beta_v;
  }
}

/*static*/
template <typename T>
void AddBiasResidualLayerNorm::inference_kernel(
    AddBiasResidualLayerNormMeta const *m,
    int attn_bias_dim,
    int residual_volume,
    T const *input_ptr,
    T const *attn_bias_ptr,
    T const *residual_ptr,
    T *added_output_ptr,
    T *output_ptr,
    T const *gamma_ptr,
    T const *beta_ptr,
    hipStream_t stream) {

  std::pair<int, int> kernel1_parallelism =
      std::make_pair(m->effective_batch_size, kCUDABlockReduceNumThreads);
  std::pair<int, int> kernel2_parallelism =
      std::make_pair(m->effective_batch_size, kCUDANumThreads);

  int num_blocks =
      std::max(kernel1_parallelism.first, kernel2_parallelism.first);
  int num_threads =
      std::max(kernel1_parallelism.second, kernel2_parallelism.second);

  LayerNormFusedForwardKernel<T>
      <<<num_blocks, num_threads, 0, stream>>>(m->effective_num_elements,
                                               attn_bias_dim,
                                               m->eps,
                                               input_ptr,
                                               attn_bias_ptr,
                                               residual_ptr,
                                               added_output_ptr,
                                               static_cast<T *>(m->mean_ptr),
                                               static_cast<T *>(m->rstd_ptr),
                                               gamma_ptr,
                                               beta_ptr,
                                               output_ptr);
}

/*static*/
void AddBiasResidualLayerNorm::inference_kernel_wrapper(
    AddBiasResidualLayerNormMeta *m,
    BatchConfig const *bc,
    GenericTensorAccessorR const &input,
    GenericTensorAccessorR const &attn_bias,
    GenericTensorAccessorR const &residual,
    GenericTensorAccessorW &added_output,
    GenericTensorAccessorW &output,
    GenericTensorAccessorR const &gamma,
    GenericTensorAccessorR const &beta) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  // save input activation if needed for PEFT
  if (bc->num_active_peft_tokens() > 0) {
    // Check that we have at most one request that requires peft_bwd
    int num_peft_requests = 0;
    for (int i = 0; i < bc->max_requests_per_batch(); i++) {
      if (bc->request_completed[i]) {
        continue;
      }
      if (bc->requestsInfo[i].peft_model_id == PEFTModelID::NO_ID) {
        continue;
      }
      if (bc->requestsInfo[i].peft_bwd) {
        num_peft_requests++;
      }
    }
    assert(num_peft_requests <= 1);

    int tokens_previous_requests = 0;
    for (int i = 0; i < bc->max_requests_per_batch(); i++) {
      if (bc->request_completed[i]) {
        continue;
      }
      // Skip non-PEFT requests
      if (bc->requestsInfo[i].peft_model_id == PEFTModelID::NO_ID) {
        // FIXME: use the new approach to computing token offset
        tokens_previous_requests += bc->requestsInfo[i].num_tokens_in_batch;
        continue;
      }
      int num_peft_tokens = bc->requestsInfo[i].num_tokens_in_batch;
      int in_dim =
          added_output.domain.hi()[0] - added_output.domain.lo()[0] + 1;
      if (bc->requestsInfo[i].peft_bwd) {
        MemoryAllocator *allocator = m->handle.peft_activation_allocator;
        m->input_activation = allocator->allocate_instance_untyped(
            data_type_size(m->input_type[0]) * num_peft_tokens * in_dim);
        // copy input activation
        if (m->input_type[0] == DT_FLOAT) {
          checkCUDA(hipMemcpyAsync(
              m->input_activation,
              added_output.get_float_ptr() + tokens_previous_requests * in_dim,
              data_type_size(m->input_type[0]) * num_peft_tokens * in_dim,
              hipMemcpyDeviceToDevice,
              stream));
        } else if (m->input_type[0] == DT_HALF) {
          checkCUDA(hipMemcpyAsync(
              m->input_activation,
              added_output.get_half_ptr() + tokens_previous_requests * in_dim,
              data_type_size(m->input_type[0]) * num_peft_tokens * in_dim,
              hipMemcpyDeviceToDevice,
              stream));
        } else {
          assert(false && "unsupport datatype in layernorm");
        }
      }
    }
  }
  // inference kernel
  int attn_bias_dim = attn_bias.domain.hi()[0] - attn_bias.domain.lo()[0] + 1;
  int residual_volume = residual.domain.get_volume();
  if (m->input_type[0] == DT_FLOAT) {
    AddBiasResidualLayerNorm::inference_kernel<float>(
        m,
        attn_bias_dim,
        residual_volume,
        input.get_float_ptr(),
        attn_bias.get_float_ptr(),
        residual.get_float_ptr(),
        added_output.get_float_ptr(),
        output.get_float_ptr(),
        m->elementwise_affine ? gamma.get_float_ptr() : nullptr,
        (m->elementwise_affine && m->use_bias) ? beta.get_float_ptr() : nullptr,
        stream);
  } else if (m->input_type[0] == DT_HALF) {
    AddBiasResidualLayerNorm::inference_kernel<half>(
        m,
        attn_bias_dim,
        residual_volume,
        input.get_half_ptr(),
        attn_bias.get_half_ptr(),
        residual.get_half_ptr(),
        added_output.get_half_ptr(),
        output.get_half_ptr(),
        m->elementwise_affine ? gamma.get_half_ptr() : nullptr,
        (m->elementwise_affine && m->use_bias) ? beta.get_half_ptr() : nullptr,
        stream);
  } else {
    assert(false && "unsupport datatype in layernorm");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("[AddBiasResidualLayerNorm] forward time (CF) = %.9fms\n", elapsed);
    // if (m->input_type[0] == DT_FLOAT) {
    //   print_tensor<float>(input.get_float_ptr(),
    //                       32,
    //                       "[AddBiasResidualLayerNorm:forward:input]");
    //   print_tensor<float>(attn_bias.get_float_ptr(),
    //                       32,
    //                       "[AddBiasResidualLayerNorm:forward:attn_bias]");
    //   print_tensor<float>(residual.get_float_ptr(),
    //                       32,
    //                       "[AddBiasResidualLayerNorm:forward:residual]");
    //   print_tensor<float>(added_output.get_float_ptr(),
    //                       32,
    //                       "[AddBiasResidualLayerNorm:forward:added_output]");
    //   print_tensor<float>(output.get_float_ptr(),
    //                       32,
    //                       "[AddBiasResidualLayerNorm:forward:output]");
    //   print_tensor<float>(gamma.get_float_ptr(),
    //                       32,
    //                       "[AddBiasResidualLayerNorm:forward:gamma]");
    //   print_tensor<float>(
    //       beta.get_float_ptr(), 32,
    //       "[AddBiasResidualLayerNorm:forward:beta]");
    // } else {
    //   print_tensor<half>(
    //       input.get_half_ptr(), 32,
    //       "[AddBiasResidualLayerNorm:forward:input]");
    //   print_tensor<half>(attn_bias.get_half_ptr(),
    //                      32,
    //                      "[AddBiasResidualLayerNorm:forward:attn_bias]");
    //   print_tensor<half>(residual.get_half_ptr(),
    //                      32,
    //                      "[AddBiasResidualLayerNorm:forward:residual]");
    //   print_tensor<half>(added_output.get_half_ptr(),
    //                      32,
    //                      "[AddBiasResidualLayerNorm:forward:added_output]");
    //   print_tensor<half>(output.get_half_ptr(),
    //                      32,
    //                      "[AddBiasResidualLayerNorm:forward:output]");
    //   print_tensor<half>(
    //       gamma.get_half_ptr(), 32,
    //       "[AddBiasResidualLayerNorm:forward:gamma]");
    //   print_tensor<half>(
    //       beta.get_half_ptr(), 32,
    //       "[AddBiasResidualLayerNorm:forward:beta]");
    // }
    // print_tensor<T>(in_ptr, 32, "[AddBiasResidualLayerNorm:forward:input]");
    // print_tensor<T>(out_ptr, 32,
    // "[AddBiasResidualLayerNorm:forward:output]");
  }
}

template <typename T>
__global__ void ComputeInternalGradientsCUDAKernel(
    int64_t N, T const *dY, T const *X, T const *gamma, T *ds, T *db) {
  using T_ACC = T;
  __shared__ T_ACC ds_shared[C10_WARP_SIZE];
  __shared__ T_ACC db_shared[C10_WARP_SIZE];
  const int64_t i = blockIdx.x;
  T_ACC sum1 = 0;
  T_ACC sum2 = 0;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    sum1 +=
        static_cast<T_ACC>(dY[index]) * static_cast<T_ACC>(X[index]) * gamma_v;
    sum2 += static_cast<T_ACC>(dY[index]) * gamma_v;
  }
  sum1 = BlockReduceSum<T_ACC>(sum1, ds_shared);
  sum2 = BlockReduceSum<T_ACC>(sum2, db_shared);
  if (threadIdx.x == 0) {
    ds[i] = sum1;
    db[i] = sum2;
  }
}

template <typename T>
__global__ void ComputeGradientFusedParamsCUDAKernel(int64_t M,
                                                     int64_t N,
                                                     T const *mean,
                                                     T const *rstd,
                                                     T const *ds,
                                                     T const *db,
                                                     T *c1,
                                                     T *c2) {
  using T_ACC = T;
  const int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < M) {
    const T_ACC s = T_ACC(1) / static_cast<T_ACC>((int)N);
    const T_ACC a = (db[index] * static_cast<T_ACC>(mean[index]) - ds[index]) *
                    static_cast<T_ACC>(rstd[index]) *
                    static_cast<T_ACC>(rstd[index]) *
                    static_cast<T_ACC>(rstd[index]) * s;
    c1[index] = a;
    c2[index] = -(a * static_cast<T_ACC>(mean[index]) +
                  db[index] * static_cast<T_ACC>(rstd[index]) * s);
  }
}

template <typename T>
__global__ void GammaBetaBackwardSimpleCUDAKernel(int64_t M,
                                                  int64_t N,
                                                  T const *dY,
                                                  T const *X,
                                                  T const *mean,
                                                  T const *rstd,
                                                  T *dg,
                                                  T *db) {
  using T_ACC = T;
  const int64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j < N) {
    T_ACC sum1 = 0;
    T_ACC sum2 = 0;
    for (int64_t i = 0; i < M; ++i) {
      const int64_t index = i * N + j;
      sum1 += dg == nullptr ? T_ACC(0)
                            : static_cast<T_ACC>(dY[index]) *
                                  (static_cast<T_ACC>(X[index]) -
                                   static_cast<T_ACC>(mean[i])) *
                                  static_cast<T_ACC>(rstd[i]);
      sum2 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index]);
    }
    if (dg != nullptr) {
      dg[j] = sum1;
    }
    if (db != nullptr) {
      db[j] = sum2;
    }
  }
}

template <typename T>
__global__ void GammaBetaBackwardCUDAKernel(int64_t M,
                                            int64_t N,
                                            T const *dY,
                                            T const *X,
                                            T const *mean,
                                            T const *rstd,
                                            T *dg,
                                            T *db) {
  using T_ACC = T;
  __shared__ T_ACC g_shared[kColwiseReduceTileSize][kColwiseReduceTileSize + 1];
  __shared__ T_ACC b_shared[kColwiseReduceTileSize][kColwiseReduceTileSize + 1];
  const int64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  T_ACC dg_sum1 = 0;
  T_ACC dg_sum2 = 0;
  T_ACC db_sum1 = 0;
  T_ACC db_sum2 = 0;
  if (j < N) {
    for (int64_t i = threadIdx.y; i < M; i += blockDim.y * 2) {
      const int64_t i1 = i;
      const int64_t i2 = i + blockDim.y;
      const int64_t index1 = i1 * N + j;
      const int64_t index2 = i2 * N + j;
      dg_sum1 += dg == nullptr ? T_ACC(0)
                               : static_cast<T_ACC>(dY[index1]) *
                                     (static_cast<T_ACC>(X[index1]) -
                                      static_cast<T_ACC>(mean[i1])) *
                                     static_cast<T_ACC>(rstd[i1]);
      db_sum1 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index1]);
      if (i2 < M) {
        dg_sum2 += dg == nullptr ? T_ACC(0)
                                 : static_cast<T_ACC>(dY[index2]) *
                                       (static_cast<T_ACC>(X[index2]) -
                                        static_cast<T_ACC>(mean[i2])) *
                                       static_cast<T_ACC>(rstd[i2]);
        db_sum2 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index2]);
      }
    }
  }
  g_shared[threadIdx.y][threadIdx.x] = dg_sum1;
  g_shared[threadIdx.y + blockDim.y][threadIdx.x] = dg_sum2;
  b_shared[threadIdx.y][threadIdx.x] = db_sum1;
  b_shared[threadIdx.y + blockDim.y][threadIdx.x] = db_sum2;
  __syncthreads();
  T_ACC sum1 = g_shared[threadIdx.x][threadIdx.y];
  T_ACC sum2 = b_shared[threadIdx.x][threadIdx.y];
  sum1 = WarpReduceSum(sum1);
  sum2 = WarpReduceSum(sum2);
  if (threadIdx.x == 0) {
    const int64_t j = blockIdx.x * blockDim.x + threadIdx.y;
    if (j < N) {
      if (dg != nullptr) {
        dg[j] = sum1;
      }
      if (db != nullptr) {
        db[j] = sum2;
      }
    }
  }
  sum1 = g_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum2 = b_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum1 = WarpReduceSum(sum1);
  sum2 = WarpReduceSum(sum2);
  if (threadIdx.x == 0) {
    const int64_t j = blockIdx.x * blockDim.x + threadIdx.y + blockDim.y;
    if (j < N) {
      if (dg != nullptr) {
        dg[j] = sum1;
      }
      if (db != nullptr) {
        db[j] = sum2;
      }
    }
  }
}

template <typename T>
__device__ __inline__ void compute_gI(T const *__restrict__ dY,
                                      T const *__restrict__ X,
                                      T const *__restrict__ mean,
                                      T const *__restrict__ rstd,
                                      T const *__restrict__ gamma,
                                      T *dX,
                                      T *dX_residual,
                                      bool reset_input_grad,
                                      bool reset_residual_grad,
                                      int const N,
                                      T *buf) {
  auto const i1 = blockIdx.x;
  const T mean_val = mean[i1];
  const T rstd_val = rstd[i1];
  T stats_x1{0}, stats_x2{0};
  constexpr int unroll = 4;
  auto l = unroll * threadIdx.x;
  T const *X_i = X + i1 * N;
  T const *dY_i = dY + i1 * N;
  T *dX_i = dX + i1 * N;
  T *dX_residual_i = dX_residual + i1 * N;
  // vectorized reads don't improve perf, so use regular unrolling

  for (; l + unroll - 1 < N; l += blockDim.x * unroll) {
#pragma unroll
    for (int k = 0; k < unroll; k++) {
      T gamma_val = (gamma != nullptr) ? static_cast<T>(gamma[l + k]) : T(1);
      const T c_h = static_cast<T>(X_i[l + k]);
      const T c_loss = static_cast<T>(dY_i[l + k]);
      stats_x1 += c_loss * gamma_val;
      stats_x2 += c_loss * gamma_val * (c_h - mean_val) * rstd_val;
    }
  }
  for (; l < N; l++) {
    T gamma_val = (gamma != nullptr) ? static_cast<T>(gamma[l]) : T(1);
    const T c_h = static_cast<T>(X_i[l]);
    const T c_loss = static_cast<T>(dY_i[l]);
    stats_x1 += c_loss * gamma_val;
    stats_x2 += c_loss * gamma_val * (c_h - mean_val) * rstd_val;
  }

  stats_x1 = BlockReduceSum(stats_x1, buf);
  stats_x2 = BlockReduceSum(stats_x2, buf);
  if (threadIdx.x == 0) {
    buf[0] = stats_x1;
    buf[1] = stats_x2;
  }
  __syncthreads();
  stats_x1 = buf[0];
  stats_x2 = buf[1];
  T fH = N;
  T term1 = (T(1) / fH) * rstd_val;

  for (int l = threadIdx.x; l < N; l += blockDim.x) {
    const T x = X_i[l];
    const T dy = dY_i[l];
    T gamma_val = (gamma != nullptr) ? static_cast<T>(gamma[l]) : T(1);
    T f_grad_input = fH * gamma_val * dy;
    f_grad_input -= (x - mean_val) * rstd_val * stats_x2;
    f_grad_input -= stats_x1;
    f_grad_input *= term1;
    if (reset_input_grad) {
      dX_i[l] = f_grad_input;
    } else {
      dX_i[l] += f_grad_input;
    }
    if (reset_residual_grad) {
      dX_residual_i[l] = f_grad_input;
    } else {
      dX_residual_i[l] += f_grad_input;
    }
  }
}

template <typename T>
__global__ void layer_norm_grad_input_kernel(T const *__restrict__ dY,
                                             T const *__restrict__ X,
                                             T const *__restrict__ mean,
                                             T const *__restrict__ rstd,
                                             T const *__restrict__ gamma,
                                             T *dX,
                                             T *dX_residual,
                                             bool reset_input_grad,
                                             bool reset_residual_grad,
                                             int const N) {
  alignas(sizeof(double)) extern __shared__ char s_data1[];
  T *buf = reinterpret_cast<T *>(&s_data1);

  compute_gI(dY,
             X,
             mean,
             rstd,
             gamma,
             dX,
             dX_residual,
             reset_input_grad,
             reset_residual_grad,
             N,
             buf);
}

/*static*/
template <typename T>
void AddBiasResidualLayerNorm::backward_kernel(
    AddBiasResidualLayerNormMeta const *m,
    T const *output_grad_ptr,
    T const *added_output_ptr,
    T *input_grad_ptr,
    T *residual_grad_ptr,
    T *attn_bias_grad_ptr,
    T const *gamma_ptr,
    T *gamma_grad_ptr,
    T *beta_grad_ptr,
    hipStream_t stream) {
  const int64_t M = m->effective_batch_size;
  const int64_t N = m->effective_num_elements;
  ComputeInternalGradientsCUDAKernel<T>
      <<<M, kCUDABlockReduceNumThreads, 0, stream>>>(
          N,
          output_grad_ptr,
          added_output_ptr,
          gamma_ptr,
          static_cast<T *>(m->ds_ptr),
          static_cast<T *>(m->db_ptr));
  const int64_t B = (M + kCUDANumThreads - 1) / kCUDANumThreads;
  ComputeGradientFusedParamsCUDAKernel<T>
      <<<B, kCUDANumThreads, 0, stream>>>(M,
                                          N,
                                          static_cast<T *>(m->mean_ptr),
                                          static_cast<T *>(m->rstd_ptr),
                                          static_cast<T *>(m->ds_ptr),
                                          static_cast<T *>(m->db_ptr),
                                          static_cast<T *>(m->scale_ptr),
                                          static_cast<T *>(m->bias_ptr));
  int const warp_size = C10_WARP_SIZE;
  int const num_threads = 128;
  const dim3 blocks(M);
  int nshared = (num_threads / warp_size) * sizeof(T);
  layer_norm_grad_input_kernel<<<blocks, num_threads, nshared, stream>>>(
      output_grad_ptr,
      added_output_ptr,
      static_cast<T *>(m->mean_ptr),
      static_cast<T *>(m->rstd_ptr),
      gamma_ptr,
      input_grad_ptr,
      residual_grad_ptr,
      m->reset_input_grads[0],
      m->reset_input_grads[1],
      N);

  if (gamma_grad_ptr != NULL || beta_grad_ptr != NULL) {
    if (M < 512) {
      // For small batch size, do colwise reduce directly
      const int64_t B = (N + kCUDANumThreads - 1) / kCUDANumThreads;
      GammaBetaBackwardSimpleCUDAKernel<T>
          <<<B, kCUDANumThreads, 0, stream>>>(M,
                                              N,
                                              output_grad_ptr,
                                              added_output_ptr,
                                              static_cast<T *>(m->mean_ptr),
                                              static_cast<T *>(m->rstd_ptr),
                                              gamma_grad_ptr,
                                              beta_grad_ptr);
    } else {
      const int64_t B =
          (N + kColwiseReduceTileSize - 1) / kColwiseReduceTileSize;
      constexpr int kThreadX = kColwiseReduceTileSize;
      constexpr int kThreadY = kColwiseReduceTileSize / 2;
      GammaBetaBackwardCUDAKernel<T>
          <<<B, dim3(kThreadX, kThreadY), 0, stream>>>(
              M,
              N,
              output_grad_ptr,
              added_output_ptr,
              static_cast<T *>(m->mean_ptr),
              static_cast<T *>(m->rstd_ptr),
              gamma_grad_ptr,
              beta_grad_ptr);
    }
  }
}

/*static*/
void AddBiasResidualLayerNorm::backward_kernel_wrapper(
    AddBiasResidualLayerNormMeta const *m,
    GenericTensorAccessorR const &output_grad,
    GenericTensorAccessorR &added_output,
    GenericTensorAccessorW &input_grad,
    GenericTensorAccessorW const &residual_grad,
    GenericTensorAccessorW const &attn_bias_grad,
    GenericTensorAccessorR const &gamma,
    GenericTensorAccessorW const &gamma_grad,
    GenericTensorAccessorW const &beta_grad) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  if (m->output_type[0] == DT_FLOAT) {
    AddBiasResidualLayerNorm::backward_kernel(
        m,
        output_grad.get_float_ptr(),
        added_output.get_float_ptr(),
        input_grad.get_float_ptr(),
        residual_grad.get_float_ptr(),
        attn_bias_grad.get_float_ptr(),
        m->elementwise_affine ? gamma.get_float_ptr() : nullptr,
        m->elementwise_affine ? gamma_grad.get_float_ptr() : nullptr,
        (m->elementwise_affine && m->use_bias) ? beta_grad.get_float_ptr()
                                               : nullptr,
        stream);
  } else if (m->output_type[0] == DT_HALF) {
    AddBiasResidualLayerNorm::backward_kernel(
        m,
        output_grad.get_half_ptr(),
        added_output.get_half_ptr(),
        input_grad.get_half_ptr(),
        residual_grad.get_half_ptr(),
        attn_bias_grad.get_half_ptr(),
        m->elementwise_affine ? gamma.get_half_ptr() : nullptr,
        m->elementwise_affine ? gamma_grad.get_half_ptr() : nullptr,
        (m->elementwise_affine && m->use_bias) ? beta_grad.get_half_ptr()
                                               : nullptr,
        stream);
  } else {
    assert(false && "Unsupported data type");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("[AddBiasResidualLayerNorm] backward time (CF) = %.2fms\n", elapsed);
  }
}

/*static*/
template <typename T>
void AddBiasResidualLayerNorm::peft_bwd_kernel(
    AddBiasResidualLayerNormMeta const *m,
    T const *output_grad_ptr,
    T *input_grad_ptr,
    T *residual_grad_ptr,
    T const *gamma_ptr,
    hipStream_t stream) {
  const int64_t M = m->effective_batch_size;
  const int64_t N = m->effective_num_elements;

  int const warp_size = C10_WARP_SIZE;
  int const num_threads = 128;
  const dim3 blocks(M);
  int nshared = (num_threads / warp_size) * sizeof(T);
  layer_norm_grad_input_kernel<<<blocks, num_threads, nshared, stream>>>(
      output_grad_ptr,
      static_cast<T const *>(m->input_activation),
      static_cast<T *>(m->mean_ptr),
      static_cast<T *>(m->rstd_ptr),
      gamma_ptr,
      input_grad_ptr,
      residual_grad_ptr,
      m->reset_input_grads[0],
      m->reset_input_grads[1],
      N);
}

/*static*/
void AddBiasResidualLayerNorm::peft_bwd_kernel_wrapper(
    AddBiasResidualLayerNormMeta const *m,
    GenericTensorAccessorR const &output_grad,
    GenericTensorAccessorW &input_grad,
    GenericTensorAccessorW const &residual_grad,
    GenericTensorAccessorR const &gamma) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }

  if (m->output_type[0] == DT_FLOAT) {
    peft_bwd_kernel(m,
                    output_grad.get_float_ptr(),
                    input_grad.get_float_ptr(),
                    residual_grad.get_float_ptr(),
                    m->elementwise_affine ? gamma.get_float_ptr() : nullptr,
                    stream);
  } else if (m->output_type[0] == DT_HALF) {
    peft_bwd_kernel(m,
                    output_grad.get_half_ptr(),
                    input_grad.get_half_ptr(),
                    residual_grad.get_half_ptr(),
                    m->elementwise_affine ? gamma.get_half_ptr() : nullptr,
                    stream);
  } else {
    assert(false && "Unsupported data type");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("[AddBiasResidualLayerNorm] peft_bwd time (CF) = %.2fms\n", elapsed);
  }
}

}; // namespace FlexFlow
