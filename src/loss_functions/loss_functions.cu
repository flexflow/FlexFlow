#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

using namespace Legion;

Loss::Loss(const std::string& loss)
{
  if (loss == "categorical_crossentropy")
    loss_type = LOSS_CATEGORICAL_CROSSENTROPY;
  else if (loss == "sparse_categorical_crossentropy")
    loss_type = LOSS_SPARSE_CATEGORICAL_CROSSENTROPY;
  else if (loss == "mean_squared_error")
    loss_type = LOSS_MEAN_SQUARED_ERROR_AVG_REDUCE;
  else
    // Unrecognized loss type
    assert(false);
}

Loss::Loss(LossType _loss_type)
: loss_type(_loss_type)
{}

__global__
void sparse_categorical_crossentropy_loss_backward(
    float *logit_grad,
    const int *label,
    coord_t num_samples,
    coord_t num_classes)
{
  CUDA_KERNEL_LOOP(i, num_samples)
  {
    int label_idx = label[i];
    logit_grad[i * num_classes + label_idx] -= 1.0f;
  }
}

__global__
void categorical_crossentropy_loss_backward(
    float *logit_grad,
    const float *logit,
    const float *label,
    coord_t num_elements)
{
  CUDA_KERNEL_LOOP(i, num_elements)
  {
    logit_grad[i] = logit[i] - label[i];
  }
}

__global__
void mean_squared_error_avg_loss_backward(
    float *logit_grad,
    const float *logit,
    const float *label,
    coord_t num_elements)
{
  CUDA_KERNEL_LOOP(i, num_elements)
  {
    logit_grad[i] = logit[i] - label[i];
  }
}

__host__
void Loss::backward_task(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime)
{
  Domain domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return backward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

template<int NDIM>
__host__
void Loss::backward_task_with_dim(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime)
{
  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  const Loss* loss = (Loss*) task->args;
  if (loss->loss_type == LOSS_SPARSE_CATEGORICAL_CROSSENTROPY) {
    //sparse_categorical_crossentropy has label of dim: (batch_size, 1)
    TensorAccessorW<float, NDIM> acc_logit_grad(
        regions[0], task->regions[0], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    TensorAccessorR<float, NDIM> acc_logit(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
    TensorAccessorR<int, NDIM> acc_label(
        regions[2], task->regions[2], FID_DATA, ctx, runtime);
    int num_samples = acc_logit.rect.hi[NDIM-1] - acc_logit.rect.lo[NDIM-1] + 1;
    int num_classes = acc_logit.rect.volume() / num_samples;
    assert(acc_logit_grad.rect == acc_logit.rect);
    for (int i = 1; i < NDIM; i++) {
      assert(acc_label.rect.hi[i] == acc_logit.rect.hi[i]);
      assert(acc_label.rect.lo[i] == acc_logit.rect.lo[i]);
    }
    assert(acc_label.rect.lo[0] == acc_label.rect.hi[0]);
    checkCUDA(hipMemcpy(acc_logit_grad.ptr, acc_logit.ptr,
                         acc_logit.rect.volume() * sizeof(float),
                         hipMemcpyDeviceToDevice));
    sparse_categorical_crossentropy_loss_backward<<<GET_BLOCKS(num_samples), CUDA_NUM_THREADS>>>(
        acc_logit_grad.ptr, acc_label.ptr, num_samples, num_classes);
    // Scale logit gradients by op->scale_factor
    scale_kernel<<<GET_BLOCKS(acc_logit_grad.rect.volume()), CUDA_NUM_THREADS>>>(
        acc_logit_grad.ptr, acc_logit_grad.rect.volume(), 0, loss->scale_factor);
  } else {
    TensorAccessorW<float, NDIM> acc_logit_grad(
        regions[0], task->regions[0], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    TensorAccessorR<float, NDIM> acc_logit(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
    TensorAccessorR<float, NDIM> acc_label(
        regions[2], task->regions[2], FID_DATA, ctx, runtime);
    // other loss require label and logit have identical shape
    assert(acc_logit.rect == acc_label.rect);
    assert(acc_logit_grad.rect == acc_logit.rect);
    int num_samples = acc_logit.rect.hi[NDIM-1] - acc_logit.rect.lo[NDIM-1] + 1;
    int num_channels = acc_logit.rect.volume() / num_samples;
    if (loss->loss_type == LOSS_CATEGORICAL_CROSSENTROPY) {
      categorical_crossentropy_loss_backward<<<GET_BLOCKS(acc_logit.rect.volume()), CUDA_NUM_THREADS>>>(
          acc_logit_grad.ptr, acc_logit.ptr, acc_label.ptr,
          acc_logit.rect.volume());
      // Scale logit gradients by loss->scale_factor
      scale_kernel<<<GET_BLOCKS(acc_logit_grad.rect.volume()), CUDA_NUM_THREADS>>>(
          acc_logit_grad.ptr, acc_logit_grad.rect.volume(), 0, loss->scale_factor);
    } else if (loss->loss_type == LOSS_MEAN_SQUARED_ERROR_AVG_REDUCE) {
      mean_squared_error_avg_loss_backward<<<GET_BLOCKS(acc_logit.rect.volume()), CUDA_NUM_THREADS>>>(
          acc_logit_grad.ptr, acc_logit.ptr, acc_label.ptr,
          acc_logit.rect.volume());
      // Scale logit gradients by loss->scale_factor
      scale_kernel<<<GET_BLOCKS(acc_logit_grad.rect.volume()), CUDA_NUM_THREADS>>>(
          acc_logit_grad.ptr, acc_logit_grad.rect.volume(), 0, loss->scale_factor);
    } else {
      fprintf(stderr, "Unsupported loss --- report this error to the FlexFlow developers\n");
      assert(false);
    }
  }
}

void Loss::backward(FFModel* model,
                    const Tensor logit,
                    const Tensor label)
#ifdef DEADCODE
{
  assert(logit->num_dims == label->num_dims);
  int dim = logit->num_dims;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      backward_with_dim<DIM>(model, logit, label); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
    {
      assert(false);
    }
  }
}

template<int NDIM>
void Loss::backward_with_dim(FFModel* model,
                             const Tensor logit,
                             const Tensor label)
#endif
{
  // Compute scale factor for loss backpropagation
  scale_factor = 1.0f/ logit->dims[logit->num_dims-1].size;
  //scale_factor = 1.0f;
  // Use the same parallel strategy as the owner of logit
  std::string pcname = logit->owner_op->name;
  Context ctx = model->config.lg_ctx;
  Runtime* runtime = model->config.lg_hlr;
  Domain part_domain = runtime->get_index_space_domain(ctx, logit->parallel_is);
  Domain logit_domain = runtime->get_index_partition_color_space(
      ctx, logit->part.get_index_partition());
  Domain label_domain = runtime->get_index_partition_color_space(
      ctx, label->part.get_index_partition());
  if((logit_domain != part_domain) || (label_domain != part_domain)) {
    fprintf(stderr, "Encounter inconsistency in parallelizing loss computation");
    assert(false);
  }
  ArgumentMap argmap;
  IndexLauncher launcher(LOSS_BWD_TASK_ID, logit->parallel_is,
                         TaskArgument(this, sizeof(Loss)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         logit->machine_view.hash());
  launcher.add_region_requirement(
      RegionRequirement(logit->part_grad, 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, logit->region_grad));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(logit->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, logit->region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(label->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, label->region));
  launcher.add_field(2, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

