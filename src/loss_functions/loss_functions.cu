#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Loss::Loss(const std::string& loss, bool _repl_labels)
{
  repl_labels = _repl_labels;
  if (loss == "categorical_crossentropy")
    loss_type = LOSS_CATEGORICAL_CROSSENTROPY;
  else if (loss == "sparse_categorical_crossentropy")
    loss_type = LOSS_SPARSE_CATEGORICAL_CROSSENTROPY;
  else if (loss == "mean_squared_error")
    loss_type = LOSS_MEAN_SQUARED_ERROR_AVG_REDUCE;
  else
    // Unrecognized loss type
    assert(false);
}

Loss::Loss(LossType _loss_type, bool _repl_labels)
: loss_type(_loss_type), repl_labels(_repl_labels)
{}

__global__
void sparse_categorical_crossentropy_loss_backward(
    float *logit_grad,
    const int *label,
    coord_t num_samples,
    coord_t num_classes,
    const int k)
{
  CUDA_KERNEL_LOOP(i, num_samples)
  {
    int label_idx = label[i/k];
    logit_grad[i * num_classes + label_idx] -= 1.0f;
  }
}

__global__
void categorical_crossentropy_loss_backward(
    float *logit_grad,
    const float *logit,
    const float *label,
    coord_t num_elements)
{
  CUDA_KERNEL_LOOP(i, num_elements)
  {
    logit_grad[i] = logit[i] - label[i];
  }
}

__global__
void mean_squared_error_avg_loss_backward(
    float *logit_grad,
    const float *logit,
    const float *label,
    coord_t num_elements)
{
  CUDA_KERNEL_LOOP(i, num_elements)
  {
    logit_grad[i] = logit[i] - label[i];
  }
}

__host__
void Loss::backward_task(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime)
{
  Domain domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return backward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

template<int NDIM>
__host__
void Loss::backward_task_with_dim(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime)
{
  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  const Loss* loss = (Loss*) task->args;
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  if (loss->loss_type == LOSS_SPARSE_CATEGORICAL_CROSSENTROPY) {
    //sparse_categorical_crossentropy has label of dim: (batch_size, 1)
    TensorAccessorW<float, NDIM> acc_logit_grad(
        regions[0], task->regions[0], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    TensorAccessorR<float, NDIM> acc_logit(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
    TensorAccessorR<int, NDIM> acc_label(
        regions[2], task->regions[2], FID_DATA, ctx, runtime);
    int num_samples = acc_logit.rect.hi[NDIM-1] - acc_logit.rect.lo[NDIM-1] + 1;
    int num_classes = acc_logit.rect.volume() / num_samples;
    assert(acc_logit_grad.rect == acc_logit.rect);
    int k = 1;
    if(loss->repl_labels) {
      k = (acc_logit.rect.hi[NDIM-1]-acc_logit.rect.lo[NDIM-1]+1) /
        (acc_label.rect.hi[NDIM-1]-acc_label.rect.lo[NDIM-1]+1);
    }
    for (int i = 1; i < NDIM-1; i++) {
      assert(acc_label.rect.hi[i] == acc_logit.rect.hi[i]);
      assert(acc_label.rect.lo[i] == acc_logit.rect.lo[i]);
    }
    assert(k*(acc_label.rect.hi[NDIM-1]-acc_label.rect.lo[NDIM-1]+1)
      == acc_logit.rect.hi[NDIM-1]-acc_logit.rect.lo[NDIM-1]+1);
    assert(acc_label.rect.lo[0] == acc_label.rect.hi[0]);
    checkCUDA(hipMemcpy(acc_logit_grad.ptr, acc_logit.ptr,
                         acc_logit.rect.volume() * sizeof(float),
                         hipMemcpyDeviceToDevice));
    sparse_categorical_crossentropy_loss_backward<<<GET_BLOCKS(num_samples), CUDA_NUM_THREADS, 0, stream>>>(
        acc_logit_grad.ptr, acc_label.ptr, num_samples, num_classes, k);
    // Scale logit gradients by op->scale_factor
    scale_kernel<<<GET_BLOCKS(acc_logit_grad.rect.volume()), CUDA_NUM_THREADS, 0, stream>>>(
        acc_logit_grad.ptr, acc_logit_grad.rect.volume(), 0, loss->scale_factor*k);
  } else {
    if(loss->repl_labels) assert(false && "Loss not yet supported for aggr_spec.");
    TensorAccessorW<float, NDIM> acc_logit_grad(
        regions[0], task->regions[0], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    TensorAccessorR<float, NDIM> acc_logit(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
    TensorAccessorR<float, NDIM> acc_label(
        regions[2], task->regions[2], FID_DATA, ctx, runtime);
    // other loss require label and logit have identical shape
    assert(acc_logit.rect == acc_label.rect);
    assert(acc_logit_grad.rect == acc_logit.rect);
    int num_samples = acc_label.rect.hi[NDIM-1] - acc_label.rect.lo[NDIM-1] + 1;
    int num_channels = acc_logit.rect.volume() / num_samples;
    if (loss->loss_type == LOSS_CATEGORICAL_CROSSENTROPY) {
      categorical_crossentropy_loss_backward<<<GET_BLOCKS(acc_logit.rect.volume()), CUDA_NUM_THREADS, 0, stream>>>(
          acc_logit_grad.ptr, acc_logit.ptr, acc_label.ptr,
          acc_logit.rect.volume());
      // Scale logit gradients by loss->scale_factor
      scale_kernel<<<GET_BLOCKS(acc_logit_grad.rect.volume()), CUDA_NUM_THREADS, 0, stream>>>(
          acc_logit_grad.ptr, acc_logit_grad.rect.volume(), 0, loss->scale_factor);
    } else if (loss->loss_type == LOSS_MEAN_SQUARED_ERROR_AVG_REDUCE) {
      mean_squared_error_avg_loss_backward<<<GET_BLOCKS(acc_logit.rect.volume()), CUDA_NUM_THREADS, 0, stream>>>(
          acc_logit_grad.ptr, acc_logit.ptr, acc_label.ptr,
          acc_logit.rect.volume());
      // Scale logit gradients by loss->scale_factor
      scale_kernel<<<GET_BLOCKS(acc_logit_grad.rect.volume()), CUDA_NUM_THREADS, 0, stream>>>(
          acc_logit_grad.ptr, acc_logit_grad.rect.volume(), 0, loss->scale_factor);
    } else {
      fprintf(stderr, "Unsupported loss --- report this error to the FlexFlow developers\n");
      assert(false);
    }
  }
}

void Loss::backward(FFModel* model,
                    const Tensor* logit,
                    const Tensor* label)
{
  assert(logit->numDim == label->numDim);
  int dim = logit->numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      backward_with_dim<DIM>(model, logit, label); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
    {
      assert(false);
    }
  }
}

template<int NDIM>
void Loss::backward_with_dim(FFModel* model,
                             const Tensor* logit,
                             const Tensor* label)
{
  // Compute scale factor for loss backpropagation
  scale_factor = 1.0f/ logit->adim[logit->numDim-1];
  //scale_factor = 1.0f;
  // Use the same parallel strategy as the owner of logit
  std::string pcname = logit->owner_op->name;
  IndexSpaceT<NDIM> task_is = IndexSpaceT<NDIM>(
    model->get_or_create_task_is(NDIM, pcname));
  Context ctx = model->config.lg_ctx;
  Runtime* runtime = model->config.lg_hlr;
  Rect<NDIM> part_rect = runtime->get_index_space_domain(ctx, task_is);
  Rect<NDIM> logit_rect = runtime->get_index_partition_color_space(
      ctx, logit->part.get_index_partition());
  Rect<NDIM> label_rect = runtime->get_index_partition_color_space(
      ctx, label->part.get_index_partition());
  if((logit_rect != part_rect) || (label_rect != part_rect)) {
    fprintf(stderr, "Encounter inconsistency in parallelizing loss computation");
    assert(false);
  }
  ArgumentMap argmap;
  IndexLauncher launcher(LOSS_BWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Loss)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(pcname));
  launcher.add_region_requirement(
      RegionRequirement(logit->part_grad, 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, logit->region_grad));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(logit->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, logit->region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(label->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, label->region));
  launcher.add_field(2, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}
