#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/model.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

using namespace Legion;

__global__
void sparse_categorical_crossentropy_loss_backward(
    float *logit_grad,
    const int *label,
    coord_t num_samples,
    coord_t num_classes,
    const int k)
{
  CUDA_KERNEL_LOOP(i, num_samples)
  {
    int label_idx = label[i/k];
    logit_grad[i * num_classes + label_idx] -= 1.0f;
  }
}

__global__
void categorical_crossentropy_loss_backward(
    float *logit_grad,
    const float *logit,
    const float *label,
    coord_t num_elements)
{
  CUDA_KERNEL_LOOP(i, num_elements)
  {
    logit_grad[i] = logit[i] - label[i];
  }
}

__global__
void mean_squared_error_avg_loss_backward(
    float *logit_grad,
    const float *logit,
    const float *label,
    coord_t num_elements)
{
  CUDA_KERNEL_LOOP(i, num_elements)
  {
    logit_grad[i] = logit[i] - label[i];
  }
}

__host__
void Loss::backward_task(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime)
{
  Domain domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return backward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}

template<int NDIM>
__host__
void Loss::backward_task_with_dim(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime)
{
  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  const Loss* loss = (Loss*) task->args;
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  if (loss->loss_type == LOSS_SPARSE_CATEGORICAL_CROSSENTROPY) {
    //sparse_categorical_crossentropy has label of dim: (batch_size, 1)
    TensorAccessorW<float, NDIM> acc_logit_grad(
        regions[0], task->regions[0], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    TensorAccessorR<float, NDIM> acc_logit(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
    TensorAccessorR<int, NDIM> acc_label(
        regions[2], task->regions[2], FID_DATA, ctx, runtime);
    int num_samples = acc_logit.rect.hi[NDIM-1] - acc_logit.rect.lo[NDIM-1] + 1;
    int num_classes = acc_logit.rect.volume() / num_samples;
    assert(acc_logit_grad.rect == acc_logit.rect);
    int k = 1;
    if(loss->repl_labels) {
      k = (acc_logit.rect.hi[NDIM-1]-acc_logit.rect.lo[NDIM-1]+1) /
        (acc_label.rect.hi[NDIM-1]-acc_label.rect.lo[NDIM-1]+1);
    }
    for (int i = 1; i < NDIM-1; i++) {
      assert(acc_label.rect.hi[i] == acc_logit.rect.hi[i]);
      assert(acc_label.rect.lo[i] == acc_logit.rect.lo[i]);
    }
    assert(k*(acc_label.rect.hi[NDIM-1]-acc_label.rect.lo[NDIM-1]+1)
      == acc_logit.rect.hi[NDIM-1]-acc_logit.rect.lo[NDIM-1]+1);
    assert(acc_label.rect.lo[0] == acc_label.rect.hi[0]);
    checkCUDA(hipMemcpy(acc_logit_grad.ptr, acc_logit.ptr,
                         acc_logit.rect.volume() * sizeof(float),
                         hipMemcpyDeviceToDevice));
    sparse_categorical_crossentropy_loss_backward<<<GET_BLOCKS(num_samples), CUDA_NUM_THREADS, 0, stream>>>(
        acc_logit_grad.ptr, acc_label.ptr, num_samples, num_classes, k);
    // Scale logit gradients by op->scale_factor
    scale_kernel<<<GET_BLOCKS(acc_logit_grad.rect.volume()), CUDA_NUM_THREADS, 0, stream>>>(
        acc_logit_grad.ptr, acc_logit_grad.rect.volume(), 0, loss->scale_factor*k);
  } else {
    if(loss->repl_labels) assert(false && "Loss not yet supported for aggr_spec.");
    TensorAccessorW<float, NDIM> acc_logit_grad(
        regions[0], task->regions[0], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    TensorAccessorR<float, NDIM> acc_logit(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
    TensorAccessorR<float, NDIM> acc_label(
        regions[2], task->regions[2], FID_DATA, ctx, runtime);
    // other loss require label and logit have identical shape
    assert(acc_logit.rect == acc_label.rect);
    assert(acc_logit_grad.rect == acc_logit.rect);
    int num_samples = acc_label.rect.hi[NDIM-1] - acc_label.rect.lo[NDIM-1] + 1;
    int num_channels = acc_logit.rect.volume() / num_samples;
    if (loss->loss_type == LOSS_CATEGORICAL_CROSSENTROPY) {
      categorical_crossentropy_loss_backward<<<GET_BLOCKS(acc_logit.rect.volume()), CUDA_NUM_THREADS, 0, stream>>>(
          acc_logit_grad.ptr, acc_logit.ptr, acc_label.ptr,
          acc_logit.rect.volume());
      // Scale logit gradients by loss->scale_factor
      scale_kernel<<<GET_BLOCKS(acc_logit_grad.rect.volume()), CUDA_NUM_THREADS, 0, stream>>>(
          acc_logit_grad.ptr, acc_logit_grad.rect.volume(), 0, loss->scale_factor);
    } else if (loss->loss_type == LOSS_MEAN_SQUARED_ERROR_AVG_REDUCE) {
      mean_squared_error_avg_loss_backward<<<GET_BLOCKS(acc_logit.rect.volume()), CUDA_NUM_THREADS, 0, stream>>>(
          acc_logit_grad.ptr, acc_logit.ptr, acc_label.ptr,
          acc_logit.rect.volume());
      // Scale logit gradients by loss->scale_factor
      scale_kernel<<<GET_BLOCKS(acc_logit_grad.rect.volume()), CUDA_NUM_THREADS, 0, stream>>>(
          acc_logit_grad.ptr, acc_logit_grad.rect.volume(), 0, loss->scale_factor);
    } else {
      fprintf(stderr, "Unsupported loss --- report this error to the FlexFlow developers\n");
      assert(false);
    }
  }
}

}; // namespace FlexFlow