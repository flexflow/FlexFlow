/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "kernels/attention_kernels.h"
#include "kernels/cuda_helper.h"

namespace FlexFlow {
namespace Kernels {
namespace MultiHeadAttention {

void init_kernel(MHAPerDeviceState *m,
                 int num_samples,
                 int num_heads,
                 int qSize,
                 int kSize,
                 int vSize,
                 int qProjSize,
                 int kProjSize,
                 int vProjSize,
                 int oProjSize,
                 int qoSeqLength,
                 int kvSeqLength,
                 bool add_bias_kv) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  checkCUDNN(cudnnCreateAttnDescriptor(&m->attnDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&m->qDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&m->kDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&m->vDesc));
  checkCUDNN(cudnnCreateSeqDataDescriptor(&m->oDesc));
  // Currently do not support adding bias to key/value projection
  assert(!add_bias_kv);
  cudnnAttnQueryMap_t attnMode = CUDNN_ATTN_QUERYMAP_ALL_TO_ONE;
  // Assume no beam search for now
  int maxBeamSize = 1;
  // printf("batchSize(%d) qSize(%d) kSize(%d) vSize(%d) qProjSize(%d)
  // kProjSize(%d)\n",
  //     num_samples, attn->qSize, attn->kSize, attn->vSize, attn->qProjSize,
  //     attn->kProjSize);
  // printf("vProjSize(%d) oProjSize(%d) qoSeqLength(%d) kvSeqLength(%d)\n",
  //     attn->vProjSize, attn->oProjSize, attn->qoSeqLength,
  //     attn->kvSeqLength);
  hipdnnMathType_t math_type;
  if (m->handle.allowTensorOpMathConversion) {
    math_type = CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION;
  } else {
    math_type = HIPDNN_TENSOR_OP_MATH;
  }
  checkCUDNN(cudnnSetAttnDescriptor(m->attnDesc,
                                    attnMode,
                                    num_heads,
                                    1.0f /*smScalar*/,
                                    HIPDNN_DATA_FLOAT,
                                    HIPDNN_DATA_FLOAT,
                                    math_type,
                                    NULL /*attnDropoutDesc*/,
                                    NULL /*postDropoutDesc*/,
                                    qSize,
                                    kSize,
                                    vSize,
                                    qProjSize,
                                    kProjSize,
                                    vProjSize,
                                    oProjSize,
                                    qoSeqLength,
                                    kvSeqLength,
                                    num_samples,
                                    maxBeamSize));
  size_t workSpaceSize;
  checkCUDNN(cudnnGetMultiHeadAttnBuffers(m->handle.dnn,
                                          m->attnDesc,
                                          &m->weightSize,
                                          &workSpaceSize,
                                          &m->reserveSpaceSize));
  assert(workSpaceSize <= m->handle.workSpaceSize);
  // printf("weightSize(%zu) workSpaceSize(%zu) reserveSpaceSize(%zu)\n",
  // weightSize, workSpaceSize, reserveSpaceSize);
  int dimA[CUDNN_SEQDATA_DIM_COUNT];
  cudnnSeqDataAxis_t axes[CUDNN_SEQDATA_DIM_COUNT];
  assert(CUDNN_SEQDATA_DIM_COUNT == 4);
  axes[3] = CUDNN_SEQDATA_VECT_DIM; // 3 = nbDims-1
  axes[2] = CUDNN_SEQDATA_BEAM_DIM;
  axes[1] = CUDNN_SEQDATA_TIME_DIM;
  axes[0] = CUDNN_SEQDATA_BATCH_DIM;
  int *qoSeqArray = (int *)malloc(sizeof(int) * num_samples);
  int *kvSeqArray = (int *)malloc(sizeof(int) * num_samples);
  for (int i = 0; i < num_samples; i++) {
    qoSeqArray[i] = qoSeqLength;
    kvSeqArray[i] = kvSeqLength;
  }
  // Set qDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = qoSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = qSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(m->qDesc,
                                         HIPDNN_DATA_FLOAT,
                                         CUDNN_SEQDATA_DIM_COUNT,
                                         dimA,
                                         axes,
                                         num_samples,
                                         qoSeqArray,
                                         NULL));
  }
  // Set kDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = kvSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = kSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(m->kDesc,
                                         HIPDNN_DATA_FLOAT,
                                         CUDNN_SEQDATA_DIM_COUNT,
                                         dimA,
                                         axes,
                                         num_samples,
                                         kvSeqArray,
                                         NULL));
  }
  // Set vDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = kvSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = vSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(m->vDesc,
                                         HIPDNN_DATA_FLOAT,
                                         CUDNN_SEQDATA_DIM_COUNT,
                                         dimA,
                                         axes,
                                         num_samples,
                                         kvSeqArray,
                                         NULL));
  }
  // Set oDesc
  {
    dimA[CUDNN_SEQDATA_BEAM_DIM] = 1;
    dimA[CUDNN_SEQDATA_BATCH_DIM] = num_samples;
    dimA[CUDNN_SEQDATA_TIME_DIM] = qoSeqLength;
    dimA[CUDNN_SEQDATA_VECT_DIM] = oProjSize;
    checkCUDNN(cudnnSetSeqDataDescriptor(m->oDesc,
                                         HIPDNN_DATA_FLOAT,
                                         CUDNN_SEQDATA_DIM_COUNT,
                                         dimA,
                                         axes,
                                         num_samples,
                                         qoSeqArray,
                                         NULL));
  }
  // allocate memory for the seqArray and reserve space
  {
    size_t totalSize = m->reserveSpaceSize + sizeof(int) * num_samples * 2;

    m->devQoSeqArray = (int *)m->gpu_alloc(totalSize);
    checkCUDA(hipMemcpy(m->devQoSeqArray,
                         qoSeqArray,
                         sizeof(int) * num_samples,
                         hipMemcpyHostToDevice));
    m->devKvSeqArray = m->devQoSeqArray + num_samples;
    checkCUDA(hipMemcpy(m->devKvSeqArray,
                         kvSeqArray,
                         sizeof(int) * num_samples,
                         hipMemcpyHostToDevice));
    m->reserveSpace = m->devKvSeqArray + num_samples;
  }
  // allocate memory for loWinIdx/hiWinIdx
  m->loWinIdx = (int *)malloc(sizeof(int) * qoSeqLength);
  m->hiWinIdx = (int *)malloc(sizeof(int) * qoSeqLength);
  for (int i = 0; i < qoSeqLength; i++) {
    m->loWinIdx[i] = 0;
    m->hiWinIdx[i] = kvSeqLength;
  }
  free(qoSeqArray);
  free(kvSeqArray);
}

/* void forward_kernel_wrapper(MHAPerDeviceState const *m, */
/*                                                 float const *query_ptr, */
/*                                                 float const *key_ptr, */
/*                                                 float const *value_ptr, */
/*                                                 float const *weight_ptr, */
/*                                                 float *output_ptr) { */
/*   wrapper(Internal::forward_kernel, m->profiling, ) */
/*   hipStream_t stream; */
/*   checkCUDA(get_legion_stream(&stream)); */

/*   hipEvent_t t_start, t_end; */
/*   if (m->profiling) { */
/*     hipEventCreate(&t_start); */
/*     hipEventCreate(&t_end); */
/*     hipEventRecord(t_start, stream); */
/*   } */
/*   Internal::forward_kernel( */
/*       m, query_ptr, key_ptr, value_ptr, weight_ptr, output_ptr, stream); */
/*   if (m->profiling) { */
/*     hipEventRecord(t_end, stream); */
/*     checkCUDA(hipEventSynchronize(t_end)); */
/*     float elapsed = 0; */
/*     checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end)); */
/*     hipEventDestroy(t_start); */
/*     hipEventDestroy(t_end); */
/*     printf("MultiHeadAttention forward time = %.2fms\n", elapsed); */
/*     // print_tensor<3, float>(acc_query.ptr, acc_query.rect, */
/*     // "[Attention:forward:query]"); print_tensor<3, float>(acc_output.ptr,
 */
/*     // acc_output.rect, "[Attention:forward:output]"); */
/*   } */
/* } */

/* void backward_kernel_wrapper( */
/*     MHAPerDeviceState const *m, */
/*     float const *query_ptr, */
/*     float *query_grad_ptr, */
/*     float const *key_ptr, */
/*     float *key_grad_ptr, */
/*     float const *value_ptr, */
/*     float *value_grad_ptr, */
/*     float const *weight_ptr, */
/*     float *weight_grad_ptr, */
/*     float const *output_grad_ptr) { */
/*   hipStream_t stream; */
/*   checkCUDA(get_legion_stream(&stream)); */

/*   hipEvent_t t_start, t_end; */
/*   if (m->profiling) { */
/*     hipEventCreate(&t_start); */
/*     hipEventCreate(&t_end); */
/*     hipEventRecord(t_start, stream); */
/*   } */

/*   Internal::backward_kernel(m, */
/*                                       query_ptr, */
/*                                       query_grad_ptr, */
/*                                       key_ptr, */
/*                                       key_grad_ptr, */
/*                                       value_ptr, */
/*                                       value_grad_ptr, */
/*                                       weight_ptr, */
/*                                       weight_grad_ptr, */
/*                                       output_grad_ptr, */
/*                                       stream); */
/*   if (m->profiling) { */
/*     hipEventRecord(t_end, stream); */
/*     checkCUDA(hipEventSynchronize(t_end)); */
/*     float elapsed = 0; */
/*     checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end)); */
/*     hipEventDestroy(t_start); */
/*     hipEventDestroy(t_end); */
/*     printf("MultiHeadAttention backward time = %.2fms\n", elapsed); */
/*   } */
/* } */

/* namespace Internal { */

void forward_kernel(hipStream_t stream,
                    MHAPerDeviceState *m,
                    float const *query_ptr,
                    float const *key_ptr,
                    float const *value_ptr,
                    float const *weight_ptr,
                    float *output_ptr) {
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  checkCUDNN(cudnnMultiHeadAttnForward(m->handle.dnn,
                                       m->attnDesc,
                                       -1,
                                       m->loWinIdx,
                                       m->hiWinIdx,
                                       m->devQoSeqArray,
                                       m->devKvSeqArray,
                                       m->qDesc,
                                       query_ptr,
                                       nullptr /*residual*/,
                                       m->kDesc,
                                       key_ptr,
                                       m->vDesc,
                                       value_ptr,
                                       m->oDesc,
                                       output_ptr,
                                       m->weightSize,
                                       weight_ptr,
                                       m->handle.workSpaceSize,
                                       m->handle.workSpace,
                                       m->reserveSpaceSize,
                                       m->reserveSpace));
}

void backward_kernel(hipStream_t stream,
                     MHAPerDeviceState *m,
                     float const *query_ptr,
                     float *query_grad_ptr,
                     float const *key_ptr,
                     float *key_grad_ptr,
                     float const *value_ptr,
                     float *value_grad_ptr,
                     float const *weight_ptr,
                     float *weight_grad_ptr,
                     float const *output_grad_ptr) {
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  checkCUDNN(cudnnMultiHeadAttnBackwardData(m->handle.dnn,
                                            m->attnDesc,
                                            m->loWinIdx,
                                            m->hiWinIdx,
                                            m->devQoSeqArray,
                                            m->devKvSeqArray,
                                            m->oDesc,
                                            output_grad_ptr,
                                            m->qDesc,
                                            query_grad_ptr,
                                            query_ptr,
                                            m->kDesc,
                                            key_grad_ptr,
                                            key_ptr,
                                            m->vDesc,
                                            value_grad_ptr,
                                            value_ptr,
                                            m->weightSize,
                                            weight_ptr,
                                            m->handle.workSpaceSize,
                                            m->handle.workSpace,
                                            m->reserveSpaceSize,
                                            m->reserveSpace));
  checkCUDNN(cudnnMultiHeadAttnBackwardWeights(m->handle.dnn,
                                               m->attnDesc,
                                               CUDNN_WGRAD_MODE_ADD,
                                               m->qDesc,
                                               query_ptr,
                                               m->kDesc,
                                               key_ptr,
                                               m->vDesc,
                                               value_ptr,
                                               m->oDesc,
                                               output_grad_ptr,
                                               m->weightSize,
                                               weight_ptr,
                                               weight_grad_ptr,
                                               m->handle.workSpaceSize,
                                               m->handle.workSpace,
                                               m->reserveSpaceSize,
                                               m->reserveSpace));
}

/* } // namespace Internal */
} // namespace MultiHeadAttention
} // namespace Kernels

MHAPerDeviceState::MHAPerDeviceState(FFHandler handler,
                                     Memory gpu_mem,
                                     int num_samples,
                                     int num_heads,
                                     int qSize,
                                     int kSize,
                                     int vSize,
                                     int qProjSize,
                                     int kProjSize,
                                     int vProjSize,
                                     int oProjSize,
                                     int qoSeqLength,
                                     int kvSeqLength,
                                     bool add_bias_kv)
    : PerDeviceOpState(handler) {}

MHAPerDeviceState::MHAPerDeviceState(FFHandler handler,
                                     std::unique_ptr<IAllocator> allocator,
                                     MultiHeadAttentionAttrs const &attrs,
                                     ArrayShape const &query_shape,
                                     ArrayShape const &key_shape,
                                     ArrayShape const &value_shape) {
  : MHAPerDeviceState(handler, 
                      allocator, 
                      query_shape[2],
                      attrs.num_heads, 
                      query_shape[0],
                      key_shape[0],
                      value_shape[0],
                      qProjSize(attrs),
                      kProjSize(attrs),
                      vProjSize(attrs),
                      oProjSize(attrs),
                      query_shape[1],
                      key_shape[1],
                      attrs.add_bias_kv)
{ }

  MHAPerDeviceState::~MHAPerDeviceState(void) {
    free(loWinIdx);
    free(hiWinIdx);
    checkCUDNN(cudnnDestroyAttnDescriptor(attnDesc));
    checkCUDNN(cudnnDestroySeqDataDescriptor(qDesc));
    checkCUDNN(cudnnDestroySeqDataDescriptor(kDesc));
    checkCUDNN(cudnnDestroySeqDataDescriptor(vDesc));
    checkCUDNN(cudnnDestroySeqDataDescriptor(oDesc));
  }

} // namespace FlexFlow
