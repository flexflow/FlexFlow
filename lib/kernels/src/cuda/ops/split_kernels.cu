/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "device.h"
#include "kernels/split_kernels.h"

namespace FlexFlow {
// declare Legion names
using coord_t = long long;

namespace Kernels {
namespace Split {

void forward_kernel(hipStream_t stream,
                    float **out_ptrs,
                    float const *in_ptr,
                    coord_t const *out_blk_sizes,
                    coord_t in_blk_size,
                    coord_t num_blks,
                    int numOutputs) {

  for (int i = 0; i < numOutputs; i++) {
    copy_with_stride<<<GET_BLOCKS(out_blk_sizes[i] * num_blks),
                       CUDA_NUM_THREADS,
                       0,
                       stream>>>(
        out_ptrs[i], in_ptr, num_blks, out_blk_sizes[i], in_blk_size);
    in_ptr += out_blk_sizes[i];
  }
}

void backward_kernel(hipStream_t stream,
                     float *in_grad_ptr,
                     float const **out_grad_ptr,
                     coord_t const *out_blk_sizes,
                     coord_t in_blk_size,
                     coord_t num_blks,
                     int numOutputs) {

  for (int i = 0; i < numOutputs; i++) {
    add_with_stride<<<GET_BLOCKS(out_blk_sizes[i] * num_blks),
                      CUDA_NUM_THREADS,
                      0,
                      stream>>>(
        in_grad_ptr, out_grad_ptr[i], num_blks, in_blk_size, out_blk_sizes[i]);
    in_grad_ptr += out_blk_sizes[i];
  }
  // checkCUDA(hipDeviceSynchronize());
}

} // namespace Split
} // namespace Kernels
} // namespace FlexFlow
