#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "device.h"
#include "kernels/datatype_dispatch.h"
#include "kernels/gather_kernels.h"

namespace FlexFlow {
namespace Kernels {
namespace Gather {

template <typename IndexType>
__global__ void gather_forward(float const *input,
                               IndexType const *index,
                               float *output,
                               coord_t output_size,
                               coord_t stride,
                               coord_t input_dim_size,
                               coord_t output_dim_size) {
  CUDA_KERNEL_LOOP(o, output_size) {
    // output tensor shape: [*, output_dim_size, stride]
    // output tensor stride: [output_dim_size * stride, stride, 1]
    // output tensor index: [outer_index, index_2, left_over]
    // input tensor shape: [*, input_dim_size, stride]
    // input tensor stride: [input_dim_size * stride, stride, 1]
    // the index of the corresponding input tensor should be:
    // [outer_index, index[0], left_over]
    // Therefore, input_index = outer_index * (stride * input_dim_size)
    //                        + index[0] * stride + left_over;
    coord_t outer_index = o / (stride * output_dim_size);
    // coord_t index_2 = (o / stride) % dim_size
    coord_t left_over = o % stride;
    coord_t input_idx =
        outer_index * (stride * input_dim_size) + index[o] * stride + left_over;
    output[o] = input[input_idx];
  }
}

template <typename IndexType>
__global__ void gather_backward(float const *output_grad,
                                IndexType const *index,
                                float *input_grad,
                                coord_t output_size,
                                coord_t stride,
                                coord_t input_dim_size,
                                coord_t output_dim_size) {
  CUDA_KERNEL_LOOP(o, output_size) {
    // output tensor shape: [*, output_dim_size, stride]
    // output tensor stride: [output_dim_size * stride, stride, 1]
    // output tensor index: [outer_index, index_2, left_over]
    // input tensor shape: [*, input_dim_size, stride]
    // input tensor stride: [input_dim_size * stride, stride, 1]
    // the index of the corresponding input tensor should be:
    // [outer_index, index[0], left_over]
    // Therefore, input_index = outer_index * (stride * input_dim_size)
    //                        + index[0] * stride + left_over;
    coord_t outer_index = o / (stride * output_dim_size);
    // coord_t index_2 = (o / stride) % dim_size
    coord_t left_over = o % stride;
    coord_t input_idx =
        outer_index * (stride * input_dim_size) + index[o] * stride + left_over;

    atomicAdd(&input_grad[input_idx], output_grad[o]);
  }
}

template <DataType IndexType>
struct ForwardKernel {
  void operator()(ffStream_t stream,
                  GenericTensorAccessorR const &input,
                  GenericTensorAccessorR const &index,
                  GenericTensorAccessorW const &output,
                  coord_t output_size,
                  coord_t stride,
                  coord_t input_dim_size,
                  coord_t output_dim_size) {
    gather_forward<<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
        input.get_float_ptr(),
        index.get<IndexType>(),
        output.get_float_ptr(),
        output_size,
        stride,
        input_dim_size,
        output_dim_size);
  }
};

template <DataType IndexType>
struct BackwardKernel {
  void operator()(ffStream_t stream,
                  GenericTensorAccessorR const &output_grad,
                  GenericTensorAccessorR const &index,
                  GenericTensorAccessorW const &input_grad,
                  coord_t output_size,
                  coord_t stride,
                  coord_t input_dim_size,
                  coord_t output_dim_size) {
    gather_backward<<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
        output_grad.get_float_ptr(),
        index.get<IndexType>(),
        input_grad.get_float_ptr(),
        output_size,
        stride,
        input_dim_size,
        output_dim_size);
  }
};

void forward_kernel(ffStream_t stream,
                    GatherPerDeviceState const &m,
                    GenericTensorAccessorR const &input,
                    GenericTensorAccessorR const &index,
                    GenericTensorAccessorW const &output) {
  checkCUDA(get_legion_stream(&stream));

  coord_t stride =
      output.shape
          .sub_shape(std::nullopt, legion_dim_t{m.legion_dim.value() + 1})
          .get_volume();
  coord_t output_dim_size = output.shape[m.legion_dim];
  coord_t input_dim_size = input.shape[m.legion_dim];

  assert(index.data_type == DataType::INT32 ||
         index.data_type == DataType::INT64);

  DataTypeDispatch1<ForwardKernel>{}(index.data_type,
                                     stream,
                                     input,
                                     index,
                                     output,
                                     output.shape.get_volume(),
                                     stride,
                                     input_dim_size,
                                     output_dim_size);
}

void backward_kernel(ffStream_t stream,
                     GatherPerDeviceState const &m,
                     GenericTensorAccessorR const &output_grad,
                     GenericTensorAccessorR const &index,
                     GenericTensorAccessorW const &input_grad) {
  checkCUDA(get_legion_stream(&stream));

  coord_t stride =
      output_grad.shape
          .sub_shape(std::nullopt, legion_dim_t{m.legion_dim.value() + 1})
          .get_volume();
  coord_t output_dim_size = output_grad.shape[m.legion_dim];
  coord_t input_dim_size = input_grad.shape[m.legion_dim];

  assert(index.data_type == DataType::INT32 ||
         index.data_type == DataType::INT64);

  DataTypeDispatch1<BackwardKernel>{}(index.data_type,
                                      stream,
                                      output_grad,
                                      index,
                                      input_grad,
                                      output_grad.shape.get_volume(),
                                      stride,
                                      input_dim_size,
                                      output_dim_size);
}

} // namespace Gather
} // namespace Kernels
} // namespace FlexFlow
