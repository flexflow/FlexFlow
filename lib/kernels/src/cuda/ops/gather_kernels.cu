#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "device.h"
#include "kernels/datatype_dispatch.h"
#include "kernels/gather_kernels.h"

namespace FlexFlow {
namespace Kernels {
namespace Gather {

template <typename IndexType>
__global__ void gather_forward(float const *input,
                               IndexType const *index,
                               float *output,
                               coord_t output_size,
                               coord_t stride,
                               coord_t input_dim_size,
                               coord_t output_dim_size) {
  CUDA_KERNEL_LOOP(o, output_size) {
    // output tensor shape: [*, output_dim_size, stride]
    // output tensor stride: [output_dim_size * stride, stride, 1]
    // output tensor index: [outter_index, index_2, left_over]
    // input tensor shape: [*, input_dim_size, stride]
    // input tensor stride: [input_dim_size * stride, stride, 1]
    // the index of the corresponding input tensor should be:
    // [outter_index, index[0], left_over]
    // Therefore, input_index = outter_index * (stride * input_dim_size)
    //                        + index[0] * stride + left_over;
    coord_t outter_index = o / (stride * output_dim_size);
    // coord_t index_2 = (o / stride) % dim_size
    coord_t left_over = o % stride;
    coord_t input_idx = outter_index * (stride * input_dim_size) +
                        index[o] * stride + left_over;
    output[o] = input[input_idx];
  }
}

template <typename IndexType>
__global__ void gather_backward(float const *output_grad,
                                IndexType const *index,
                                float *input_grad,
                                coord_t output_size,
                                coord_t stride,
                                coord_t input_dim_size,
                                coord_t output_dim_size) {
  CUDA_KERNEL_LOOP(o, output_size) {
    // output tensor shape: [*, output_dim_size, stride]
    // output tensor stride: [output_dim_size * stride, stride, 1]
    // output tensor index: [outter_index, index_2, left_over]
    // input tensor shape: [*, input_dim_size, stride]
    // input tensor stride: [input_dim_size * stride, stride, 1]
    // the index of the corresponding input tensor should be:
    // [outter_index, index[0], left_over]
    // Therefore, input_index = outter_index * (stride * input_dim_size)
    //                        + index[0] * stride + left_over;
    coord_t outer_index = o / (stride * output_dim_size);
    // coord_t index_2 = (o / stride) % dim_size
    coord_t left_over = o % stride;
    coord_t input_idx =
        outer_index * (stride * input_dim_size) + index[o] * stride + left_over;

    atomicAdd(&input_grad[input_idx], output_grad[o]);
  }
}

template <DataType IndexType>
struct ForwardKernel {
  void operator()(ffStream_t stream,
                  GenericTensorAccessorR const &input,
                  GenericTensorAccessorR const &index,
                  GenericTensorAccessorW const &output,
                  coord_t output_size,
                  coord_t stride,
                  coord_t input_dim_size,
                  coord_t output_dim_size) {
    gather_forward<<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
        input.get_float_ptr(),
        index.get<IndexType>(),
        output.get_float_ptr(),
        output_size,
        stride,
        input_dim_size,
        output_dim_size);
  }
};

template <DataType IndexType>
struct BackwardKernel {
  void operator()(ffStream_t stream,
                  GenericTensorAccessorR const &output_grad,
                  GenericTensorAccessorR const &index,
                  GenericTensorAccessorW const &input_grad,
                  coord_t output_size,
                  coord_t stride,
                  coord_t input_dim_size,
                  coord_t output_dim_size) {
    gather_backward<<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
        output_grad.get_float_ptr(),
        index.get<IndexType>(),
        input_grad.get_float_ptr(),
        output_size,
        stride,
        input_dim_size,
        output_dim_size);
  }
};

void forward_kernel(ffStream_t stream,
                    GatherPerDeviceState const &m,
                    GenericTensorAccessorR const &input,
                    GenericTensorAccessorR const &index,
                    GenericTensorAccessorW const &output) {
  checkCUDA(get_legion_stream(&stream));

  // Reference code for what's below -- not sure if I got the domain/array shape
  // stuff right coord_t stride = 1; for (int i = 0; i < m->legion_dim; i++) {
  //   stride *= (output.domain.hi()[i] - output.domain.lo()[i] + 1);
  // }
  // coord_t output_dim_size =
  //     output.domain.hi()[m->legion_dim] - output.domain.lo()[m->legion_dim] +
  //     1;
  // coord_t input_dim_size =
  //     input.domain.hi()[m->legion_dim] - input.domain.lo()[m->legion_dim] +
  //     1;

  coord_t stride = 1;
  for (int i = 0; i < m.legion_dim; i++) {
    stride *= output.shape[legion_dim_t(i)] + 1;
  }

  coord_t output_dim_size = output.shape[legion_dim_t(m.legion_dim)] + 1;
  coord_t input_dim_size = input.shape[legion_dim_t(m.legion_dim)] + 1;

  assert(index.data_type == DataType::INT32 ||
         index.data_type == DataType::INT64);

  DataTypeDispatch1<ForwardKernel>{}(index.data_type,
                                     stream,
                                     input,
                                     index,
                                     output,
                                     output.shape.get_volume(),
                                     stride,
                                     input_dim_size,
                                     output_dim_size);
}

void backward_kernel(ffStream_t stream,
                     GatherPerDeviceState const &m,
                     GenericTensorAccessorR const &output_grad,
                     GenericTensorAccessorR const &index,
                     GenericTensorAccessorW const &input_grad) {
  checkCUDA(get_legion_stream(&stream));

  // Reference code for what's below -- not sure if I got the domain/array shape
  // stuff right coord_t stride = 1; for (int i = 0; i < m->legion_dim; i++) {
  //   stride *= (output_grad.domain.hi()[i] - output_grad.domain.lo()[i] + 1);
  // }
  // coord_t output_dim_size = output_grad.domain.hi()[m->legion_dim] -
  //                           output_grad.domain.lo()[m->legion_dim] + 1;
  // coord_t input_dim_size = input_grad.domain.hi()[m->legion_dim] -
  //                          input_grad.domain.lo()[m->legion_dim] + 1;

  coord_t stride = 1;
  for (int i = 0; i < m.legion_dim; i++) {
    stride *= output_grad.shape[legion_dim_t(i)] + 1;
  }

  coord_t output_dim_size = output_grad.shape[legion_dim_t(m.legion_dim)] + 1;
  coord_t input_dim_size = input_grad.shape[legion_dim_t(m.legion_dim)] + 1;

  assert(index.data_type == DataType::INT32 ||
         index.data_type == DataType::INT64);

  DataTypeDispatch1<BackwardKernel>{}(index.data_type,
                                      stream,
                                      output_grad,
                                      index,
                                      input_grad,
                                      output_grad.shape.get_volume(),
                                      stride,
                                      input_dim_size,
                                      output_dim_size);
}

} // namespace Gather
} // namespace Kernels
} // namespace FlexFlow
