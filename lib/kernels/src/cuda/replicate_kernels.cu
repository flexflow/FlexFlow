#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "kernels/replicate_kernels.h"

namespace FlexFlow {
namespace Kernels {
namespace Replicate {

template <DataType T>
struct ForwardKernel {
  void operator()(hipStream_t stream,
                  GenericTensorAccessorR const &input,
                  GenericTensorAccessorW const &output) {

    checkCUDA(hipMemcpyAsync(input.get<T>(),
                              output.get<T>(),
                              input.shape.num_elements() * sizeof(T),
                              hipMemcpyDeviceToDevice,
                              stream));
  }
}

template <DataType T>
struct BackwardKernel {
  void operator()(hipStream_t stream,
                  GenericTensorAccessorW const &input,
                  GenericTensorAccessorR const &output,
                  size_t num_replicas) {
    size_t total_elements = input.shape.num_elements() * num_replicas;
    replicate_backward_kernel<T>
        <<<GET_BLOCKS(total_elements), CUDA_NUM_THREADS, 0, stream>>>(
            input.get<T>(),
            output.get<T>(),
            input.shape.num_elements(),
            num_replicas);
  }
}

template <typename T>
__global__ void replicate_backward_kernel(T const *input_ptr,
                                          T *output_ptr,
                                          size_t num_elements,
                                          size_t num_replicas) {
  CUDA_KERNEL_LOOP(i, num_elements) {
    for (size_t j = 0; j < num_replicas; j++) {
      output_ptr[i] += input_ptr[i + j * num_elements];
    }
  }
}

void forward_kernel(hipStream_t stream,
                    GenericTensorAccessorR const &input,
                    GenericTensorAccessorW const &output) {
  DataTypeDispatch1<ForwardKernel>{}(input->data_type, stream, input, output);
}

void backward_kernel(hipStream_t stream,
                     GenericTensorAccessorW const &input,
                     GenericTensorAccessorR const &output,
                     size_t num_replicas) {
  DataTypeDispatch1<BackwardKernel>{}(
      input->data_type, stream, input, output, num_replicas);
}

} // namespace Replicate
} // namespace Kernels
} // namespace FlexFlow
