#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include "kernels/datatype_dispatch.h"
#include "kernels/gather_kernels.h"

namespace FlexFlow {

GatherPerDeviceState::GatherPerDeviceState(FFHandler handler)
    : PerDeviceOpState(handler){};

namespace Kernels {
namespace Gather {

template <DataType IndexTxype>
struct ForwardKernel {
  void operator()(hipStream_t stream,
                  GatherPerDeviceState const *m,
                  GenericTensorAccessorR const &input,
                  GenericTensorAccessorR const &index,
                  GenericTensorAccessorW const &output,
                  size_t stride,
                  size_t input_dim_size,
                  size_t output_dim_size) {
    /*size_t stride = 1;
    for (int i = 0; i < m->legion_dim; i++) {
      stride *= (output.domain.hi()[i] - output.domain.lo()[i] + 1);
    }
    size_t dim_size =
        output.domain.hi()[m->legion_dim] - output.domain.lo()[m->legion_dim] +
    1;
*/
    gather_forward<IndexType><<<GET_BLOCKS(output.domain.get_volume()),
                                CUDA_NUM_THREADS,
                                0,
                                stream>>>(input.get<DT_FLOAT>(),
                                          index.get<IndexType>(),
                                          output.get<DT_FLOAT>(),
                                          output.domain.get_volume(),
                                          stride,
                                          dim_size);
  }
}

void forward_kernel(hipStream_t stream,
                    GatherPerDeviceState const *m,
                    GenericTensorAccessorR const &input,
                    GenericTensorAccessorR const &index,
                    GenericTensorAccessorW const &output,
                    size_t stride,
                    size_t input_dim_size,
                    size_t output_dim_size) {
  DataTypeDispatch1<ForwardKernel>{}(m->index_data_type,
                                     stream,
                                     m,
                                     input,
                                     index,
                                     output,
                                     stride,
                                     input_dim_size,
                                     output_dim_size);
}

template <DataType IndexType>
struct BackwardKernel {
  void operator()(hipStream_t stream,
                  GatherPerDeviceState const *m,
                  GenericTensorAccessorR const &output_grad,
                  GenericTensorAccessorR const &index,
                  GenericTensorAccessorW const &input_grad,
                  size_t stride,
                  size_t input_dim_size,
                  size_t output_dim_size) {
    /*size_t stride = 1;
    for (int i = 0; i < m->legion_dim; i++) {
      stride *= (output_grad.domain.hi()[i] - output_grad.domain.lo()[i] + 1);
    }
    size_t dim_size = output_grad.domain.hi()[m->legion_dim] -
                      output_grad.domain.lo()[m->legion_dim] + 1;
    */
    gather_backward<IndexType><<<GET_BLOCKS(output_grad.domain.get_volume()),
                                 CUDA_NUM_THREADS,
                                 0,
                                 stream>>>(output_grad.get<DT_FLOAT>(),
                                           index.get<IndexType>(),
                                           input_grad.get<DT_FLOAT>(),
                                           output_grad.domain.get_volume(),
                                           stride,
                                           input_dim_size,
                                           output_dim_size);
  }
}

void backward_kernel(hipStream_t stream,
                     GatherPerDeviceState const *m,
                     GenericTensorAccessorR const &output_grad,
                     GenericTensorAccessorR const &index,
                     GenericTensorAccessorW const &input_grad,
                     size_t stride,
                     size_t input_dim_size,
                     size_t output_dim_size) {
  DataTypeDispatch1<BackwardKernel>{}(m->index_data_type,
                                      stream,
                                      m,
                                      output_grad,
                                      index,
                                      input_grad,
                                      stride,
                                      input_dim_size,
                                      output_dim_size);
}

template <typename IndexType>
__global__ void gather_forward(float const *input,
                               IndexType const *index,
                               float *output,
                               size_t output_size,
                               size_t stride,
                               size_t input_dim_size,
                               size_t output_dim_size) {
  CUDA_KERNEL_LOOP(o, output_size) {
    // output tensor shape: [*, output_dim_size, stride]
    // output tensor stride: [output_dim_size * stride, stride, 1]
    // output tensor index: [outer_index, index_2, left_over]
    // input tensor shape: [*, input_dim_size, stride]
    // input tensor stride: [input_dim_size * stride, stride, 1]
    // the index of the corresponding input tensor should be:
    // [outer_index, index[0], left_over]
    // Therefore, input_index = outer_index * (stride * input_dim_size)
    //                        + index[0] * stride + left_over;
    size_t outer_index = o / (stride * output_dim_size);
    // coord_t index_2 = (o / stride) % dim_size
    size_t left_over = o % stride;
    size_t input_idx =
        outer_index * (stride * input_dim_size) + index[o] * stride + left_over;
    output[o] = input[input_idx];
  }
}

template <typename IndexType>
__global__ void gather_backward(float const *output_grad,
                                IndexType const *index,
                                float *input_grad,
                                size_t output_size,
                                size_t stride,
                                size_t input_dim_size,
                                size_t output_dim_size) {
  CUDA_KERNEL_LOOP(o, output_size) {
    // output tensor shape: [*, output_dim_size, stride]
    // output tensor stride: [output_dim_size * stride, stride, 1]
    // output tensor index: [outer_index, index_2, left_over]
    // input tensor shape: [*, input_dim_size, stride]
    // input tensor stride: [input_dim_size * stride, stride, 1]
    // the index of the corresponding input tensor should be:
    // [outer_index, index[0], left_over]
    // Therefore, input_index = outer_index * (stride * input_dim_size)
    //                        + index[0] * stride + left_over;
    size_t outer_index = o / (stride * output_dim_size);
    // coord_t index_2 = (o / stride) % dim_size
    size_t left_over = o % stride;
    size_t input_idx =
        outer_index * (stride * input_dim_size) + index[o] * stride + left_over;

    atomicAdd(&input_grad[input_idx], output_grad[o]);
  }
}

} // namespace Gather
} // namespace Kernels
} // namespace FlexFlow
