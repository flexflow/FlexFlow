/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "device.h"
#include "kernels/allocation.h"
#include "kernels/batch_norm_kernels.h"
#include "kernels/device.h"
#include "kernels/ff_handle.h"

namespace FlexFlow {
namespace Kernels {
namespace BatchNorm {

void forward_kernel(hipStream_t stream,
                    BatchNormPerDeviceState const *m,
                    float const *input_ptr,
                    float *output_ptr,
                    float const *scale_ptr,
                    float const *bias_ptr) {
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha = 1.0f, beta = 0.0f;
  checkCUDNN(hipdnnBatchNormalizationForwardTraining(m->handle.dnn,
                                                    m->mode,
                                                    &alpha,
                                                    &beta,
                                                    m->inputTensor,
                                                    input_ptr,
                                                    m->outputTensor,
                                                    output_ptr,
                                                    m->biasTensor,
                                                    scale_ptr,
                                                    bias_ptr,
                                                    1.0,
                                                    m->runningMean,
                                                    m->runningVar,
                                                    HIPDNN_BN_MIN_EPSILON,
                                                    m->saveMean,
                                                    m->saveVar));
}

void backward_kernel(hipStream_t stream,
                     BatchNormPerDeviceState *m,
                     float const *input_ptr,
                     float *output_grad_ptr,
                     float const *output_ptr,
                     float *input_grad_ptr,
                     float const *scale_ptr,
                     float *scale_grad_ptr,
                     float *bias_grad_ptr,
                     size_t numElements) {
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  float alpha = 1.0f;
  if (m->relu) {
    reluBackward<<<GET_BLOCKS(numElements), CUDA_NUM_THREADS, 0, stream>>>(
        output_grad_ptr, output_ptr, numElements);
  }
  checkCUDNN(hipdnnBatchNormalizationBackward(m->handle.dnn,
                                             m->mode,
                                             &alpha,
                                             &alpha,
                                             &alpha,
                                             &alpha,
                                             m->inputTensor,
                                             input_ptr,
                                             m->outputTensor,
                                             output_grad_ptr,
                                             m->inputTensor,
                                             input_grad_ptr,
                                             m->biasTensor,
                                             scale_ptr,
                                             scale_grad_ptr,
                                             bias_grad_ptr,
                                             HIPDNN_BN_MIN_EPSILON,
                                             m->saveMean,
                                             m->saveVar));
}

BatchNormPerDeviceState init_kernel(PerDeviceFFHandle handle,
                                    Allocator allocator,
                                    float *runningMean,
                                    int output_n,
                                    int output_c,
                                    int output_h,
                                    int output_w,
                                    bool relu) {
  ffTensorDescriptor_t inputTensor;
  ffTensorDescriptor_t outputTensor;
  ffTensorDescriptor_t biasTensor;
  ffActivationDescriptor_t actiDesc;
  ffBatchNormMode_t mode;
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  mode = HIPDNN_BATCHNORM_SPATIAL;
#if CUDNN_VERSION >= 7000
  mode = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;
#endif
  fprintf(
      stderr, "output(%d,%d,%d,%d)\n", output_n, output_c, output_h, output_w);
  checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        output_n,
                                        output_c,
                                        output_h,
                                        output_w));
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        output_n,
                                        output_c,
                                        output_h,
                                        output_w));
  checkCUDNN(hipdnnSetTensor4dDescriptor(
      biasTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, output_c, 1, 1));
  // allocate memory for runningMean, runningVar, saveMean, saveVar
  size_t totalSize = sizeof(float) * output_c * 4;
  runningMean = (float *)allocator.allocate(totalSize);
  float *runningVar = (float *)runningMean + output_c;
  float *saveMean = (float *)runningVar + output_c;
  float *saveVar = (float *)saveMean + output_c;
  hipStream_t stream;

  assign_kernel<<<GET_BLOCKS(output_c), CUDA_NUM_THREADS, 0, stream>>>(
      runningMean, output_c, 0.0f);
  assign_kernel<<<GET_BLOCKS(output_c), CUDA_NUM_THREADS, 0, stream>>>(
      runningVar, output_c, 0.0f);

  if (relu) {
    checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(
        actiDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));
  }

  BatchNormPerDeviceState per_device_state = {handle,
                                              allocator,
                                              inputTensor,
                                              outputTensor,
                                              biasTensor,
                                              actiDesc,
                                              mode,
                                              runningMean,
                                              runningVar,
                                              saveMean,
                                              saveVar,
                                              output_n,
                                              output_c,
                                              output_h,
                                              output_w,
                                              relu};
  return per_device_state;
}

void cleanup_kernel(Allocator allocator,
                    ffTensorDescriptor_t inputTensor,
                    ffTensorDescriptor_t biasTensor,
                    ffTensorDescriptor_t outputTensor,
                    ffActivationDescriptor_t actiDesc,
                    bool relu,
                    float *runningMean) {
  allocator.deallocate(runningMean);
  checkCUDNN(hipdnnDestroyTensorDescriptor(inputTensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(biasTensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(outputTensor));
  if (relu) {
    checkCUDNN(hipdnnDestroyActivationDescriptor(actiDesc));
  }
}

} // namespace BatchNorm
} // namespace Kernels
} // namespace FlexFlow
