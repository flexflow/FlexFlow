#include "hip/hip_runtime.h"
#include "flexflow/model.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

#ifdef FF_USE_CUDA
hipError_t get_legion_stream(hipStream_t *stream) {
#ifdef DISABLE_LEGION_CUDA_HIJACK
  *stream = (hipStream_t)0;
  return hipSuccess;
#else
  return hipStreamCreate(stream);
#endif
}
#elif FF_USE_HIP_CUDA
extern "C" {
hipStream_t hipGetTaskStream();
}
hipError_t get_legion_stream(hipStream_t *stream) {
#ifdef DISABLE_LEGION_CUDA_HIJACK
  *stream = (hipStream_t)0;
#else
  *stream = hipGetTaskStream();
#endif
  return hipSuccess;
}
#else
#error "Unknown device, please make sure if CUDA is enabled"
#endif

}; // namespace FlexFlow

using FlexFlow::get_legion_stream;

__global__ void scale_kernel(float *ptr, coord_t size, float a, float b) {
  CUDA_KERNEL_LOOP(i, size) {
    ptr[i] = (b - a) * ptr[i] + a;
  }
}

__global__ void ones_kernel(float *ptr, coord_t size) {
  CUDA_KERNEL_LOOP(i, size) {
    ptr[i] = 1.0f;
  }
}

template <typename DT>
__global__ void assign_kernel(DT *ptr, coord_t size, DT value) {
  CUDA_KERNEL_LOOP(i, size) {
    ptr[i] = value;
  }
}

template <typename DT>
__global__ void copy_kernel(DT *dst, const DT *src, coord_t size) {
  CUDA_KERNEL_LOOP(i, size) {
    dst[i] = src[i];
  }
}

template <typename DT>
__global__ void reluBackward(DT *grad_ptr, const DT *output, size_t n) {
  CUDA_KERNEL_LOOP(i, n) {
    grad_ptr[i] = (output[i] > 0.0f) ? grad_ptr[i] : 0;
  }
}

__host__ void relu_backward_kernel(DataType data_type,
                                   void *output_grad_ptr,
                                   void const *output_ptr,
                                   size_t output_size,
                                   hipStream_t stream) {
  if (data_type == DT_FLOAT) {
    reluBackward<float>
        <<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
            (float *)output_grad_ptr, (float const *)output_ptr, output_size);
  } else if (data_type == DT_DOUBLE) {
    reluBackward<double>
        <<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
            (double *)output_grad_ptr, (double const *)output_ptr, output_size);
  } else {
    assert(false && "Unsupported data type in Linear backward");
    exit(1);
  }
}

template <typename DT>
__global__ void
    sigmoid_backward_function(DT *grad_ptr, const DT *output, size_t n) {
  CUDA_KERNEL_LOOP(i, n) {
    grad_ptr[i] = grad_ptr[i] * output[i] * (1.0f - output[i]);
  }
}

__host__ void sigmoid_backward_kernel(DataType data_type,
                                      void *output_grad_ptr,
                                      void const *output_ptr,
                                      size_t output_size,
                                      hipStream_t stream) {
  if (data_type == DT_FLOAT) {
    sigmoid_backward_function<float>
        <<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
            (float *)output_grad_ptr, (float const *)output_ptr, output_size);
  } else if (data_type == DT_DOUBLE) {
    sigmoid_backward_function<double>
        <<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
            (double *)output_grad_ptr, (double const *)output_ptr, output_size);
  } else {
    assert(false && "Unsupported data type in Linear backward");
    exit(1);
  }
}

__global__ void gelu_forward_kernel(size_t size,
                                    float const B,
                                    float const C,
                                    float *input) {
  CUDA_KERNEL_LOOP(i, size) {
    float const in = input[i];
    float const cdf = 0.5f + 0.5f * tanh(in * (C * in * in + B));
    input[i] = in * cdf;
  }
}

__global__ void
    apply_add(float *data_ptr, float const *replica_ptr, size_t size) {
  CUDA_KERNEL_LOOP(i, size) {
    data_ptr[i] += replica_ptr[i];
  }
}

template <typename T>
__global__ void
    apply_add_with_scale(T *data_ptr, T const *grad_ptr, size_t size, T scale) {
  CUDA_KERNEL_LOOP(i, size) {
    data_ptr[i] += grad_ptr[i] * scale;
  }
}

template <typename T>
__global__ void add_kernel(T *data_ptr, T const *grad_ptr, size_t size) {
  CUDA_KERNEL_LOOP(i, size) {
    data_ptr[i] += grad_ptr[i];
  }
}

__global__ void add_with_stride(float *output,
                                float const *input,
                                int num_blocks,
                                int output_blk_size,
                                int input_blk_size) {
  int min_blk_size = min(output_blk_size, input_blk_size);
  CUDA_KERNEL_LOOP(i, num_blocks * min_blk_size) {
    int blk_idx = i / min_blk_size;
    int blk_offset = i % min_blk_size;
    int input_offset = blk_idx * input_blk_size + blk_offset;
    int output_offset = blk_idx * output_blk_size + blk_offset;
    output[output_offset] += input[input_offset];
  }
}

__global__ void copy_with_stride(float *output,
                                 float const *input,
                                 int num_blocks,
                                 int output_blk_size,
                                 int input_blk_size) {
  int min_blk_size = min(output_blk_size, input_blk_size);
  CUDA_KERNEL_LOOP(i, num_blocks * min_blk_size) {
    int blk_idx = i / min_blk_size;
    int blk_offset = i % min_blk_size;
    int input_offset = blk_idx * input_blk_size + blk_offset;
    int output_offset = blk_idx * output_blk_size + blk_offset;
    output[output_offset] = input[input_offset];
  }
}

__host__ void updateGAS(float *para_ptr,
                        float const *grad_ptr,
                        size_t replica_size,
                        int num_replica,
                        float learning_rate) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  // Step 1: gater gradients to the first replica
  for (int i = 1; i < num_replica; i++) {
    float const *replica = grad_ptr + i * replica_size;
    apply_add<<<GET_BLOCKS(replica_size), CUDA_NUM_THREADS, 0, stream>>>(
        (float *)grad_ptr, replica, replica_size);
  }
  // Step 2: scale the first replica
  float scale_factor = 1.0f / num_replica * (-learning_rate);
  apply_add_with_scale<<<GET_BLOCKS(replica_size),
                         CUDA_NUM_THREADS,
                         0,
                         stream>>>(
      para_ptr, grad_ptr, replica_size, scale_factor);
}

template <typename T>
__host__ void
    print_tensor(T const *ptr, size_t num_elements, char const *prefix) {
  // device synchronize to make sure the data are ready
  // checkCUDA(hipDeviceSynchronize());
  T *host_ptr;
  checkCUDA(hipHostAlloc(&host_ptr,
                          sizeof(T) * num_elements,
                          hipHostMallocPortable | hipHostMallocMapped));
  checkCUDA(hipMemcpy(
      host_ptr, ptr, sizeof(T) * num_elements, hipMemcpyDeviceToHost));
  // checkCUDA(hipDeviceSynchronize());
  int idx = 0;
  printf("%s", prefix);
  for (idx = 0; idx < num_elements; idx++) {
    printf(" %.4lf", (float)host_ptr[idx]);
    if (idx >= 16) {
      break;
    }
  }
  printf("\n");
  checkCUDA(hipHostFree(host_ptr));
}

hipdnnStatus_t
    cudnnSetTensorDescriptorFromArrayShape(hipdnnTensorDescriptor_t tensor,
                                           ArrayShape const &shape) {
  ArrayShape flipped = shape.reversed_dim_order();

  if (flipped.get_dim() == 5) {
    assert(flipped[0] == 1);
    flipped = flipped.sub_shape(1, std::nullopt);
  }

  assert(flipped.get_dim() > 0);
  assert(flipped.get_dim() < 4);

  return hipdnnSetTensor4dDescriptor(tensor,
                                    HIPDNN_TENSOR_NCHW,
                                    HIPDNN_DATA_FLOAT,
                                    flipped.at_maybe(0).value_or(1),
                                    flipped.at_maybe(1).value_or(2),
                                    flipped.at_maybe(2).value_or(3),
                                    flipped.at_maybe(3).value_or(3));
}

hipdnnDataType_t ff_to_cudnn_datatype(DataType type) {
  switch (type) {
    case DT_FLOAT:
      return HIPDNN_DATA_FLOAT;
    case DT_DOUBLE:
      return HIPDNN_DATA_DOUBLE;
    case DT_INT32:
      return HIPDNN_DATA_INT32;
    default:
      assert(false && "Unsupported cudnn data type");
  }
  return HIPDNN_DATA_FLOAT;
}

hipDataType ff_to_cuda_datatype(DataType type) {
  switch (type) {
    case DT_FLOAT:
      return HIP_R_32F;
    case DT_DOUBLE:
      return HIP_R_64F;
    case DT_INT32:
      return HIP_R_32I;
    default:
      assert(false && "Unspoorted cuda data type");
  }
  return HIP_R_32F;
}

template __global__ void
    assign_kernel<half>(half *ptr, coord_t size, half value);
template __global__ void
    assign_kernel<float>(float *ptr, coord_t size, float value);
template __global__ void
    assign_kernel<double>(double *ptr, coord_t size, double value);
template __global__ void
    assign_kernel<int32_t>(int32_t *ptr, coord_t size, int32_t value);
template __global__ void
    assign_kernel<int64_t>(int64_t *ptr, coord_t size, int64_t value);

template __global__ void
    add_kernel<float>(float *dst, float const *src, size_t size);
template __global__ void
    add_kernel<double>(double *dst, double const *src, size_t size);
template __global__ void
    add_kernel<int32_t>(int32_t *dst, int32_t const *src, size_t size);
template __global__ void
    add_kernel<int64_t>(int64_t *dst, int64_t const *src, size_t size);

template __global__ void
    copy_kernel<float>(float *dst, float const *src, coord_t size);
template __global__ void
    copy_kernel<int32_t>(int32_t *dst, int32_t const *src, coord_t size);
template __global__ void
    copy_kernel<int64_t>(int64_t *dst, int64_t const *src, coord_t size);

template __global__ void apply_add_with_scale<float>(float *data_ptr,
                                                     float const *grad_ptr,
                                                     size_t size,
                                                     float scale);
template __global__ void apply_add_with_scale<double>(double *data_ptr,
                                                      double const *grad_ptr,
                                                      size_t size,
                                                      double scale);
template __global__ void apply_add_with_scale<int32_t>(int32_t *data_ptr,
                                                       int32_t const *grad_ptr,
                                                       size_t size,
                                                       int32_t scale);
template __global__ void apply_add_with_scale<int64_t>(int64_t *data_ptr,
                                                       int64_t const *grad_ptr,
                                                       size_t size,
                                                       int64_t scale);

template __host__ void
    print_tensor<float>(float const *ptr, size_t rect, char const *prefix);
template __host__ void
    print_tensor<double>(double const *ptr, size_t rect, char const *prefix);
template __host__ void
    print_tensor<int32_t>(int32_t const *ptr, size_t rect, char const *prefix);
template __host__ void
    print_tensor<int64_t>(int64_t const *ptr, size_t rect, char const *prefix);
