/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "alexnet.h"
#include "flexflow/utils/cuda_helper.h"

void DataLoader::load_input(Task const *task,
                            std::vector<PhysicalRegion> const &regions,
                            Context ctx,
                            Runtime *runtime) {
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  SampleIdxs *meta = (SampleIdxs *)task->local_args;
  TensorAccessorR<float, 5> acc_full_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 5> acc_batch_input(regions[1],
                                            task->regions[1],
                                            FID_DATA,
                                            ctx,
                                            runtime,
                                            false /*readOutput*/);
  coord_t batch_size =
      acc_batch_input.rect.hi[3] - acc_batch_input.rect.lo[3] + 1;
  coord_t channels =
      acc_batch_input.rect.hi[2] - acc_batch_input.rect.lo[2] + 1;
  coord_t height = acc_batch_input.rect.hi[1] - acc_batch_input.rect.lo[1] + 1;
  coord_t width = acc_batch_input.rect.hi[0] - acc_batch_input.rect.lo[0] + 1;
  // FIXME: currently assume continous indices
  assert(batch_size == meta->num_samples);
  for (int i = 1; i < batch_size; i++) {
    assert(meta->idxs[i] == meta->idxs[0] + i);
  }
  coord_t start_idx = meta->idxs[0];
  float const *input_zc =
      acc_full_input.ptr + start_idx * channels * height * width;
  copy_kernel<<<GET_BLOCKS(acc_batch_input.rect.volume()), CUDA_NUM_THREADS>>>(
      acc_batch_input.ptr, input_zc, acc_batch_input.rect.volume());
  checkCUDA(hipDeviceSynchronize());
}

void DataLoader::load_label(Task const *task,
                            std::vector<PhysicalRegion> const &regions,
                            Context ctx,
                            Runtime *runtime) {
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  SampleIdxs *meta = (SampleIdxs *)task->local_args;
  TensorAccessorR<int, 3> acc_full_label(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<int, 3> acc_batch_label(regions[1],
                                          task->regions[1],
                                          FID_DATA,
                                          ctx,
                                          runtime,
                                          false /*readOutput*/);
  int batch_size = acc_batch_label.rect.hi[1] - acc_batch_label.rect.lo[1] + 1;
  // FIXME: currently assume continous indices
  assert(batch_size == meta->num_samples);
  for (int i = 1; i < batch_size; i++) {
    assert(meta->idxs[i] == meta->idxs[0] + i);
  }
  int const *input_zc = acc_full_label.ptr + meta->idxs[0];
  copy_kernel<<<GET_BLOCKS(acc_batch_label.rect.volume()), CUDA_NUM_THREADS>>>(
      acc_batch_label.ptr, input_zc, acc_batch_label.rect.volume());
  checkCUDA(hipDeviceSynchronize());
}
