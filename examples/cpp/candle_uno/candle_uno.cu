/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "candle_uno.h"
#include "flexflow/utils/cuda_helper.h"

void DataLoader::load_input(Task const *task,
                            std::vector<PhysicalRegion> const &regions,
                            Context ctx,
                            Runtime *runtime) {
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  SampleIdxs *meta = (SampleIdxs *)task->local_args;
  TensorAccessorR<float, 2> acc_full_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> acc_batch_input(regions[1],
                                            task->regions[1],
                                            FID_DATA,
                                            ctx,
                                            runtime,
                                            false /*readOutput*/);
  int batch_size = acc_batch_input.rect.hi[1] - acc_batch_input.rect.lo[1] + 1;
  int num_feats = acc_batch_input.rect.hi[0] - acc_batch_input.rect.lo[0] + 1;
  assert(acc_batch_input.rect.hi[0] == acc_full_input.rect.hi[0]);
  assert(acc_batch_input.rect.lo[0] == acc_full_input.rect.lo[0]);
  // FIXME: currently assume continous indices
  assert(batch_size == meta->num_samples);
  for (int i = 1; i < batch_size; i++) {
    assert(meta->idxs[i] == meta->idxs[0] + i);
  }
  float const *input_zc = acc_full_input.ptr + meta->idxs[0] * num_feats;
  copy_kernel<<<GET_BLOCKS(acc_batch_input.rect.volume()), CUDA_NUM_THREADS>>>(
      acc_batch_input.ptr, input_zc, acc_batch_input.rect.volume());
  checkCUDA(hipDeviceSynchronize());
}
