/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/utils/cuda_helper.h"
#include "transformers.h"

void DataLoader::load_input(Task const *task,
                            std::vector<PhysicalRegion> const &regions,
                            Context ctx,
                            Runtime *runtime) {
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  SampleIdxs *meta = (SampleIdxs *)task->local_args;
  if (meta->num_samples == 0) {
    return;
  }
  float const *full_input_ptr = helperGetTensorPointerRO<float>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  float *batch_input_ptr = helperGetTensorPointerWO<float>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);

  Domain full_input_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  Domain batch_input_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());

  coord_t token_dim =
      batch_input_domain.hi()[0] - batch_input_domain.lo()[0] + 1;
  coord_t sequence_length =
      batch_input_domain.hi()[1] - batch_input_domain.lo()[1] + 1;
  coord_t batch_size =
      batch_input_domain.hi()[2] - batch_input_domain.lo()[2] + 1;

  // Currently assume continous indices
  assert(meta->num_samples <= batch_size);
  for (int i = 1; i < meta->num_samples; i++) {
    if (meta->guids[i] == meta->guids[i-1])
      assert(meta->idxs[i] == meta->idxs[i-1]+1);
  }
  // keep things simple for now
  assert(batch_input_domain.get_volume() == batch_size * sequence_length * token_dim);
  // currently use sequence length = 1, since we just concatenate all tensors
  assert(sequence_length == 1);
  
  // pad inputs if needed (this is really only useful for debugging)
  if (meta->num_samples < batch_size) {
    checkCUDA(hipMemset(batch_input_ptr + token_dim * meta->num_samples,
                         0,
                         token_dim * (batch_size - meta->num_samples) * sizeof(float)));
  }

  
  size_t guid = meta->guids[0];
  size_t start_idx = meta->idxs[0];
  size_t dst_idx = 0;
  size_t total_tokens = 0;
  for (size_t i=1; i <= meta->num_samples; i++) {
    if (i == meta->num_samples || meta->guids[i] != guid) {
      size_t size_to_copy = token_dim * (meta->idxs[i-1] - start_idx + 1);
      total_tokens += size_to_copy / token_dim;
      float const *input_zc = full_input_ptr + (guid * token_dim * max_sequence_length) + start_idx * token_dim;
      float *dst_ptr = batch_input_ptr + dst_idx * token_dim;
      copy_kernel<<<GET_BLOCKS(size_to_copy), CUDA_NUM_THREADS>>>(dst_ptr, input_zc, size_to_copy);
      if (i < meta->num_samples) {
        guid = meta->guids[i];
        start_idx = meta->idxs[i];
      }
      dst_idx = i;
    }
  }
  assert(total_tokens == meta->num_samples);
  checkCUDA(hipDeviceSynchronize());
}
