/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/utils/cuda_helper.h"
#include "xdl.h"

void DataLoader::load_sparse_input(Task const *task,
                                   std::vector<PhysicalRegion> const &regions,
                                   Context ctx,
                                   Runtime *runtime) {
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  int hash = *((int *)task->args);
  int num_sparse_inputs = hash / 1000;
  int my_input_idx = hash % 1000;
  SampleIdxs *meta = (SampleIdxs *)task->local_args;
  TensorAccessorR<int64_t, 2> acc_full_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<int64_t, 3> acc_batch_input(regions[1],
                                              task->regions[1],
                                              FID_DATA,
                                              ctx,
                                              runtime,
                                              false /*readOutput*/);
  int batch_size = acc_batch_input.rect.hi[1] - acc_batch_input.rect.lo[1] + 1;
  int in_dim = acc_batch_input.rect.hi[0] - acc_batch_input.rect.lo[0] + 1;
  assert(acc_full_input.rect.hi[0] - acc_full_input.rect.lo[0] + 1 ==
         num_sparse_inputs * in_dim);
  int64_t *input_zc;
  checkCUDA(hipHostAlloc(&input_zc,
                          sizeof(int64_t) * acc_batch_input.rect.volume(),
                          hipHostMallocPortable | hipHostMallocMapped));
  assert(batch_size == meta->num_samples);
  for (int i = 0; i < batch_size; i++) {
    int full_offset =
        meta->idxs[i] * num_sparse_inputs * in_dim + my_input_idx * in_dim;
    int batch_offset = i * in_dim;
    assert(full_offset + in_dim <= (int)acc_full_input.rect.volume());
    for (int j = 0; j < in_dim; j++) {
      input_zc[batch_offset + j] = acc_full_input.ptr[full_offset + j];
    }
  }
  checkCUDA(hipMemcpy(acc_batch_input.ptr,
                       input_zc,
                       sizeof(int64_t) * acc_batch_input.rect.volume(),
                       hipMemcpyHostToDevice));
  checkCUDA(hipHostFree(input_zc));
  checkCUDA(hipDeviceSynchronize());
  // print_tensor<2, int>(acc_batch_input.ptr, acc_batch_input.rect,
  // "[DataLoader:load_sparse]");
}

void DataLoader::load_label(Task const *task,
                            std::vector<PhysicalRegion> const &regions,
                            Context ctx,
                            Runtime *runtime) {
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  SampleIdxs *meta = (SampleIdxs *)task->local_args;
  TensorAccessorR<float, 2> acc_full_label(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 3> acc_batch_label(regions[1],
                                            task->regions[1],
                                            FID_DATA,
                                            ctx,
                                            runtime,
                                            false /*readOutput*/);
  int batch_size = acc_batch_label.rect.hi[1] - acc_batch_label.rect.lo[1] + 1;
  int num_label = acc_batch_label.rect.hi[0] - acc_batch_label.rect.lo[0] + 1;
  assert(num_label == 1); // Kaggle dataset a has single label
  assert(acc_batch_label.rect.hi[0] == acc_full_label.rect.hi[0]);
  assert(acc_batch_label.rect.lo[0] == acc_full_label.rect.lo[0]);
  float *label_zc;
  checkCUDA(hipHostAlloc(&label_zc,
                          sizeof(float) * acc_batch_label.rect.volume(),
                          hipHostMallocPortable | hipHostMallocMapped));
  assert(batch_size == meta->num_samples);
  for (int i = 0; i < batch_size; i++) {
    int base_offset = meta->idxs[i] * num_label;
    for (int j = 0; j < num_label; j++) {
      label_zc[i * num_label + j] = acc_full_label.ptr[base_offset + j];
    }
    // printf("meta->idxs[%d]=%d label=%.2lf\n", i, meta->idxs[i], label_zc[i]);
  }
  checkCUDA(hipMemcpy(acc_batch_label.ptr,
                       label_zc,
                       sizeof(float) * acc_batch_label.rect.volume(),
                       hipMemcpyHostToDevice));
  checkCUDA(hipHostFree(label_zc));
}
